#include "hip/hip_runtime.h"
/// LSU EE X70X-X (Fall 2012), GPU Programming
//
 /// CUDA code for computing intersections and time-stepping physics model.

// $Id:$

/// Purpose
//
//   Demonstrate Several Graphical and Simulation Techniques.
//   This file contains GPU/cuda code.
//   See demo-x-collide.cc for main program.

#include <gp/cuda-util-kernel.h>
#include "k-main.cuh"


///
/// Variables Read or Written By With Host Code
///

 /// Ball Information Structure
//
// This is in soa (structure of arrays) form, rather than
// in the programmer-friendly aos (array of structure) form.
// In soa form it is easier for multiple thread to read contiguous
// blocks of data.
//
__constant__ CUDA_Ball_X balls_x;

///
 /// Ball Contact (tact) Pair Information
///

 /// Balls needed by block.
//
// This array identifies those balls that will be used by each block
// during each contact pass. When a thread starts balls are placed in
// shared memory, then contact between a pair of balls is tested for
// and resolved.
//
__constant__ int *block_balls_needed;

 /// Shared memory array holding balls updated cooperating threads in a block.
#undef USE_STRUCT
#ifdef USE_STRUCT
extern __shared__ CUDA_Phys_W sm_balls[];
#else
extern __shared__ float3 sm_balls[];
__shared__ uchar4 sm_balls_misc[300];
#endif

 /// Pairs of Balls to Check
//
__constant__ SM_Idx2 *tacts_schedule;

 /// Box/Box Intersect
//
__constant__ XX_Pair *xx_pairs;
__constant__ float4 *xx_sects_center;
__constant__ float4 *xx_sects_dir;
__constant__ float4 *xx_sects_debug;


__constant__ float3 gravity_accel_dt;
__constant__ float opt_bounce_loss, opt_bounce_loss_box;
__constant__ float opt_friction_coeff, opt_friction_roll;
__constant__ float platform_xmin, platform_xmax;
__constant__ float platform_zmin, platform_zmax;
__constant__ float platform_xmid, platform_xrad;
__constant__ float delta_t;
__constant__ float elasticity_inv_dt;
__constant__ bool opt_debug, opt_debug2;

__constant__ CUDA_Wheel wheel;
extern __shared__ float block_torque_dt[];

static __host__ void collect_symbols();


///
/// Useful Functions and Types
///

typedef float3 pCoor;
typedef float3 pVect;

__device__ float3 make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}
__device__ float3 m3(float4 a){ return make_float3(a); }
__device__ float3 xyz(float4 a){ return m3(a); }
__device__ float4 m4(float3 v, float w) { return make_float4(v.x,v.y,v.z,w); }

__device__ pVect operator +(pVect a,pVect b)
{ return make_float3(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pVect operator -(pVect a,pVect b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(float4 a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator *(float s, pVect v)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ float4 operator *(float s, float4 v)
{return make_float4(s*v.x,s*v.y,s*v.z,s*v.w);}
__device__ pVect operator *(pVect u, pVect v)
{return make_float3(u.x*v.x,u.y*v.y,u.z*v.z);}
__device__ pVect operator -(pVect v) { return make_float3(-v.x,-v.y,-v.z); }
__device__ float3 operator -=(float3& a, pVect b) {a = a - b; return a;}
__device__ float3 operator +=(float3& a, pVect b) {a = a + b; return a;}

struct pNorm {
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

// Make a Coordinate
__device__ pCoor 
mc(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pCoor mc(float4 c){ return make_float3(c.x,c.y,c.z); }

__device__ void set_f3(float3& a, float4 b){a.x = b.x; a.y = b.y; a.z = b.z;}
__device__ void set_f4(float4& a, float3 b)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = 1;}
__device__ void set_f4(float4& a, float3 b, float c)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = c;}

// Make a Vector
__device__ pVect
mv(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pVect mv(float3 a, float3 b) { return b-a; }
__device__ pVect mv(float a) { return make_float3(a,a,a); }

__device__ float dot(float4 a, float4 b)
{ return a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w;}
__device__ float dot(pVect a, pVect b){ return a.x*b.x + a.y*b.y + a.z*b.z;}
__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }
__device__ float dot3(float4 a, float4 b){ return dot(m3(a),m3(b)); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

// Make a Normal (a structure containing a normalized vector and length)
__device__ pNorm mn(pVect v)
{
  pNorm n;
  n.mag_sq = mag_sq(v);
  if ( n.mag_sq == 0 )
    {
      n.magnitude = 0;
      n.v.x = n.v.y = n.v.z = 0;
    }
  else
    {
      n.magnitude = sqrtf(n.mag_sq);
      n.v = (1.0f/n.magnitude) * v;
    }
  return n;
}
__device__ pNorm mn(float4 a, float4 b) {return mn(b-a);}
__device__ pNorm mn(pCoor a, pCoor b) {return mn(b-a);}
__device__ pNorm mn(float x, float y, float z) {return mn(mv(x,y,z));}
__device__ pNorm mn(float4 v4)
{ pNorm n; n.v = m3(v4);  n.magnitude = v4.w;  return n; }
__device__ pNorm mn(float3 v3, float mag)
{ pNorm n; n.v = v3;  n.magnitude = mag;  return n; }

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

struct pQuat {
  float w;
  pVect v;
};

// Make Quaternion
__device__ float4 mq(pNorm axis, float angle)
{
  return m4( __sinf(angle/2) * axis.v, __cosf(angle/2) );
}

__device__ float4 quat_normalize(float4 q)
{
  float len_sq = dot(q,q);
  float norm_factor = 1.0f / sqrtf(len_sq);
  return norm_factor * q;
}

// Make float4
__device__ float4 m4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }
__device__ float4 m4(pNorm v, float w) { return m4(v.v,w); }

__device__ pVect fabs(pVect v){ return mv(fabs(v.x),fabs(v.y),fabs(v.z)); }
__device__ float min(pVect v){ return min(min(v.x,v.y),v.z); }
__device__ float max(pVect v){ return max(max(v.x,v.y),v.z); }
__device__ float sum(pVect v){ return v.x+v.y+v.z; }

// Cross Product of Two Vectors
__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}
__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }
__device__ pVect crossf3(float4 a, float4 b) { return cross(m3(a),m3(b)); }

// Cross Product of Vectors Between Coordinates
__device__ float3
 cross3(float3 a, float3 b, float3 c)
{
  float3 ab = a - b;
  float3 cb = c - b;
  return cross(ab,cb);
}
__device__ pVect cross3(pVect a, pVect b, pNorm c) { return cross3(a,b,c.v); }

__device__ float4 quat_mult(float4 a, float4 b)
{
  float w = a.w * b.w - dot3(a,b);
  float3 v = a.w * m3(b) + b.w * m3(a) + crossf3(a,b);
  return make_float4(v.x,v.y,v.z,w);
};


__device__ void
pMatrix_set_rotation(pMatrix3x3& m, pVect u, float theta)
{
  const float cos_theta = __cosf(theta);
  const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta );
  m.r0.x = u.x * u.x + cos_theta * ( 1 - u.x * u.x );
  m.r0.y = u.x * u.y * ( 1 - cos_theta ) - u.z * sin_theta;
  m.r0.z = u.z * u.x * ( 1 - cos_theta ) + u.y * sin_theta;
  m.r1.x = u.x * u.y * ( 1 - cos_theta ) + u.z * sin_theta;
  m.r1.y = u.y * u.y + cos_theta * ( 1 - u.y * u.y );
  m.r1.z = u.y * u.z * ( 1 - cos_theta ) - u.x * sin_theta;
  m.r2.x = u.z * u.x * ( 1 - cos_theta ) - u.y * sin_theta;
  m.r2.y = u.y * u.z * ( 1 - cos_theta ) + u.x * sin_theta;
  m.r2.z = u.z * u.z + cos_theta * ( 1 - u.z * u.z );
}

__device__ float3 operator *(pMatrix3x3 m, float3 coor)
{ return make_float3(dot(m.r0,coor), dot(m.r1,coor), dot(m.r2,coor)); }


//
/// Ball Physics Functions
//
// See demo-x-collide.cc for details.

__device__ pVect
point_rot_vel(float3 omega, float r, pNorm direction)
{
  /// Return velocity of point on surface of sphere of radius r.
  //
  return r * cross( omega, direction );
}

__device__ float
get_fdt_to_do(float r, float mass_inv) { return 2.5f * mass_inv / r; }

__device__ float3
tan_force_dt
(pNorm tact_dir, float3 force_dt, float fdt_to_do)
{
  /// Change rotation rate due to force_dt at tact_dir in direction force_dir.
  //
  return cross(tact_dir, fdt_to_do * force_dt );
}


///
/// Major Ball Physics Routines
///

// A time step is computed using two kernels, pass_pairs and
// pass_platform. The pass_pairs kernel, which might be launched
// several times, handles collisions between balls. The pass_platform
// kernel handles collision between balls and the platform, and also
// updates position and orientation, and spins the wheel.


__device__ bool
tile_ball_collide
(CUDA_Tile_W& tile, CUDA_Ball_W& ball, pCoor& tact_pos, pVect& tact_dir)
{
  // If tile in contact with ball return true and write contact
  // point on tile to tact_pos and ball-center-to-tact-pos direction
  // to tact_dir.

  pVect tile_to_ball = mv(tile.pt_ll,ball.position);

  // Distance from tile's plane to the ball.
  const float dist = dot(tile_to_ball,tile.normal);
  const float radius = ball.radius;

  if ( fabs(dist) > radius ) return false;

  // The closest point on tile plane to the ball.
  pCoor pt_closest = ball.position - dist * tile.normal; 

  // How far up the tile in the y direction the center of the ball sits
  const float dist_ht = dot(tile.norm_up,tile_to_ball);  

  if ( dist_ht < -radius ) return false;
  if ( dist_ht > tile.height + radius ) return false;

  // How far up the tile in the x direction the center of the ball sits
  const float dist_wd = dot(tile.norm_rt,tile_to_ball);
  if ( dist_wd < -radius ) return false;
  if ( dist_wd > tile.width + radius ) return false;

  // If ball touching tile surface (not including an edge or corner)
  // then set up the pseudo ball for collision handling
  if ( dist_ht >= 0 && dist_ht <= tile.height
       && dist_wd >= 0 && dist_wd <= tile.width )
    {
      tact_pos = pt_closest;
      tact_dir = dist > 0 ? -tile.normal : tile.normal;
      return true;
    }

  float3 pt_lr = tile.pt_ll + tile.width * tile.norm_rt;
  float3 pt_ul = tile.pt_ll + tile.height * tile.norm_up;
  float3 pt_ur = pt_lr + tile.height * tile.norm_up;

  // Test whether the ball is touching a corner
  if ( ( dist_ht < 0 || dist_ht > tile.height ) 
       && ( dist_wd < 0 || dist_wd > tile.width) )
    {
      pCoor ref_pt;

      // We need to place the pseudo ball based upon the vector from
      // ball position to the corner. First step is to figure out which
      // corner.

      if ( dist_ht < 0 && dist_wd < 0 ) 
        {
          ref_pt = tile.pt_ll;
        }
      else if ( dist_ht < 0 && dist_wd > tile.width ) 
        {
          ref_pt = pt_lr;
        }
      else if ( dist_ht > tile.height && dist_wd < 0 ) 
        {
          ref_pt = pt_ul;
        }
      else 
        {
          ref_pt = pt_ur;
        }

      tact_pos = ref_pt;
      tact_dir = normalize(mv(ball.position,ref_pt));
      return true;
    }

  // Else the ball is touching an edge

  const bool tact_horiz = dist_ht < 0 || dist_ht > tile.height;
  const pVect corner_to_tact =
    tact_horiz ? dist_wd * tile.norm_rt : dist_ht * tile.norm_up;
  const pCoor ref_pt =
    tact_horiz ? ( dist_ht < 0 ? tile.pt_ll : pt_ul ) :
    ( dist_wd < 0 ? tile.pt_ll : pt_lr );

  // Find the closest edge point of the tile to the ball
  tact_pos = ref_pt + corner_to_tact;
  tact_dir = normalize(mv(ball.position,tact_pos));

  return true;
}

__device__ void
wheel_collect_tile_force(CUDA_Tile_W& tile, pCoor tact, pVect delta_mo)
{
  pVect to_center = mv(wheel.center,tact);
  // Formula below needs to be checked.
  const float torque_dt = dot(wheel.axis_dir,cross(to_center,delta_mo));
  tile.torque += torque_dt;
}


///
/// Collision (Penetration) Detection and Resolution Routines
///

// Used in both passes.


__device__ bool
penetration_balls_resolve
(CUDA_Ball_W& ball1, CUDA_Ball_W& ball2, bool b2_real, Force_Types ft)
{
  /// Update velocity and angular momentum for a pair of balls in contact.

  // Later, separate friction and other forces.
  if ( ft == FT_Friction ) return false;

  pVect zero_vec = mv(0,0,0);
  pNorm dist = mn(ball1.position,ball2.position);

  float3 v1 = ball1.velocity;
  float3 v2 = ball2.velocity;
  float3 omega1 = ball1.omega;
  float3 omega2 = ball2.omega;
  const float mass_inv1 = ball1.mass_inv;
  const float mass_inv2 = ball2.mass_inv;
  const float r1 = ball1.radius;
  const float r2 = ball2.radius;

  const float radii_sum = r1 + r2;

  if ( dist.magnitude >= radii_sum ) return false;

  /// WARNING:  This doesn't work: somefunc(-dist); 
  pNorm ndist = -dist;

  // Compute relative (approach) velocity.
  //
  pVect prev_appr_vel = ball1.prev_velocity - ball2.prev_velocity;
  const float prev_approach_speed = dot( prev_appr_vel, dist );

  const float loss_factor = 1 - opt_bounce_loss;

  // Compute change in speed based on how close balls touching, ignoring
  // energy loss.
  //
  const float appr_force_dt_no_loss =
    ( radii_sum - dist.magnitude ) * 
    ( radii_sum - dist.magnitude ) * elasticity_inv_dt;

  // Change in speed accounting for energy loss. Only applied when
  // balls separating.
  //
  const float appr_force_dt =
    prev_approach_speed > 0
    ? appr_force_dt_no_loss : loss_factor * appr_force_dt_no_loss;

  const float appr_deltas_1 = appr_force_dt * mass_inv1;

  /// Update Linear Velocity
  //
  v1 -= appr_deltas_1 * dist;
  if ( b2_real ) v2 += appr_force_dt * mass_inv2 * dist;


  const float fdt_to_do_1 = get_fdt_to_do(r1,mass_inv1);
  const float fdt_to_do_2 = get_fdt_to_do(r2,mass_inv2);

  // Find speed on surface of balls at point of contact.
  //
  pVect tact1_rot_vel = point_rot_vel(omega1,r1,dist);
  pVect tact2_rot_vel = point_rot_vel(omega2,r2,ndist);

  // Find relative velocity of surfaces at point of contact
  // in the plane formed by their surfaces.
  //
  pVect tan_vel = prev_appr_vel - prev_approach_speed * dist;
  pNorm tact_vel_dir = mn(tact1_rot_vel - tact2_rot_vel + tan_vel);

  // Find change in velocity due to friction.
  //
  const float fric_force_dt_potential =
    appr_force_dt_no_loss * opt_friction_coeff;

  const float mass_inv_sum = b2_real ? mass_inv1 + mass_inv2 : mass_inv1;

  const float force_dt_limit = tact_vel_dir.magnitude / ( 3.5f * mass_inv_sum );

  // If true, surfaces are not sliding or will stop sliding after
  // frictional forces applied. (If a ball surface isn't sliding
  // against another surface than it must be rolling.)
  //
  const bool will_roll = force_dt_limit <= fric_force_dt_potential;

  const float sliding_fric_force_dt =
    will_roll ? force_dt_limit : fric_force_dt_potential;

  const float dv_tolerance = 0.000001f;

  const float sliding_fric_dv_1 = sliding_fric_force_dt * mass_inv1;
  const float3 sliding_fric_fdt_vec = sliding_fric_force_dt * tact_vel_dir;

  if ( sliding_fric_dv_1 > dv_tolerance )
    {
      // Apply tangential force (resulting in angular momentum change) and
      // linear force (resulting in velocity change).
      //
      omega1 += tan_force_dt(dist, sliding_fric_fdt_vec, -fdt_to_do_1);
      v1 -= sliding_fric_dv_1 * tact_vel_dir;
    }

  const float sliding_fric_dv_2 = sliding_fric_force_dt * mass_inv2;

  if ( b2_real && sliding_fric_dv_2 > dv_tolerance )
    {
      // Apply frictional forces for ball 2.
      //
      omega2 += tan_force_dt(ndist, sliding_fric_fdt_vec, fdt_to_do_2);
      v2 += sliding_fric_dv_2 * tact_vel_dir;;
    }

  {
    /// Torque
    //
    //
    // Account for forces of surfaces twisting against each
    // other. (For example, if one ball is spinning on top of
    // another.)
    //
    const float appr_omega = dot(omega2,dist) - dot(omega1,dist);
    const float fdt_to_do_sum =
      b2_real ? fdt_to_do_1 + fdt_to_do_2 : fdt_to_do_1;
    const float fdt_limit = fabs(appr_omega) / fdt_to_do_sum;
    const bool rev = appr_omega < 0;
    const float fdt_raw = min(fdt_limit,fric_force_dt_potential);
    const pVect fdt_v = ( rev ? -fdt_raw : fdt_raw ) * dist;
    omega1 += fdt_to_do_1 * fdt_v;
    if ( b2_real ) omega2 -= fdt_to_do_2 * fdt_v;
  }

  ball1.velocity = v1;
  ball1.omega = omega1;
  if ( !b2_real ) return true;
  ball2.velocity = v2;
  ball2.omega = omega2;

  return true;

  {
    /// Rolling Friction
    //
    // The rolling friction model used here is ad-hoc.

    pVect tan_b12_vel = b2_real ? 0.5f * tan_vel : zero_vec;
    const float torque_limit_sort_of = appr_force_dt_no_loss
      * sqrt( radii_sum - dist.mag_sq / radii_sum );
      //  * sqrt( ball1.radius - 0.25 * dist.mag_sq * r_inv );

    pVect tact1_rot_vel = point_rot_vel(omega1,r1,dist);
    pVect tact1_roll_vel = tact1_rot_vel + tan_b12_vel;
    pNorm tact1_roll_vel_dir = mn(tact1_roll_vel);
    pVect lost_vel = zero_vec;

    const float rfric_loss_dv_1 =
      torque_limit_sort_of * 2.5f * mass_inv1 *
      ( tact1_roll_vel_dir.magnitude * opt_friction_roll /
        ( 1 + tact1_roll_vel_dir.magnitude * opt_friction_roll ) );
    
    pVect lost_vel1 =
      min(tact1_roll_vel_dir.magnitude, rfric_loss_dv_1) * tact1_roll_vel_dir;

    lost_vel = -lost_vel1;
    
    if ( b2_real )
      {
        pVect tact2_rot_vel = point_rot_vel(omega2,r2,ndist);
        pVect tact2_roll_vel = tact2_rot_vel - tan_b12_vel;
        pNorm tact2_roll_vel_dir = mn(tact2_roll_vel);
        const float rfric_loss_dv_2 =
          torque_limit_sort_of * 2.5f * mass_inv2 *
          ( tact2_roll_vel_dir.magnitude * opt_friction_roll /
            ( 1 + tact2_roll_vel_dir.magnitude * opt_friction_roll ) );
        pVect lost_vel2 =
          min(tact2_roll_vel_dir.magnitude, rfric_loss_dv_2 )
          * tact2_roll_vel_dir;

        lost_vel += lost_vel2;
      }

    omega1 += tan_force_dt(dist, 0.4f / mass_inv1 * lost_vel, fdt_to_do_1);
    if ( b2_real )
      omega2 += tan_force_dt(dist, 0.4f / mass_inv2 * lost_vel, fdt_to_do_2);
  }
  return true;
}

//
// Generic operations used by box code.
//

__device__ float3
sign_mask(int idx, float3 v)
{
  return make_float3
    (idx & 4 ? v.x : -v.x, idx & 2 ? v.y : -v.y, idx & 1 ? v.z : -v.z );
}

// Multiply transpose of matrix m by column vector v.
__device__ float3 mm_transpose(pMatrix3x3 m, float3 v)
{ return v.x * m.r0 + v.y * m.r1 + v.z * m.r2; }

__device__ float
set_min(float &a, float b)
{
  if ( b < a ) a = b;
  return a;
}

__device__ float
set_max(float &a, float b)
{
  if ( b > a ) a = b;
  return a;
}

// Set matrix m to a rotation matrix based on quaternion q.
__device__ void
pMatrix_set_rotation(pMatrix3x3& m, float4 q)
{
  m.r0.x = 1.f - 2.f * q.y * q.y - 2.f * q.z * q.z;
  m.r0.y = 2.f * q.x * q.y - 2.f * q.w * q.z;
  m.r0.z = 2.f * q.x * q.z + 2.f * q.w * q.y;
  m.r1.x = 2.f * q.x * q.y + 2.f * q.w * q.z;
  m.r1.y = 1.f - 2.f * q.x * q.x - 2.f * q.z * q.z;
  m.r1.z = 2.f * q.y * q.z - 2.f * q.w * q.x;
  m.r2.x = 2.f * q.x * q.z - 2.f * q.w * q.y;
  m.r2.y = 2.f * q.y * q.z + 2.f * q.w * q.x;
  m.r2.z = 1.f - 2.f * q.x * q.x - 2.f * q.y * q.y;
}

// Set transpose of matrix m to a rotation matrix based on quaternion q.
__device__ void
pMatrix_set_rotation_transpose(pMatrix3x3& m, float4 q)
{
  m.r0.x = 1.f - 2.f * q.y * q.y - 2.f * q.z * q.z;
  m.r1.x = 2.f * q.x * q.y - 2.f * q.w * q.z;
  m.r2.x = 2.f * q.x * q.z + 2.f * q.w * q.y;
  m.r0.y = 2.f * q.x * q.y + 2.f * q.w * q.z;
  m.r1.y = 1.f - 2.f * q.x * q.x - 2.f * q.z * q.z;
  m.r2.y = 2.f * q.y * q.z - 2.f * q.w * q.x;
  m.r0.z = 2.f * q.x * q.z - 2.f * q.w * q.y;
  m.r1.z = 2.f * q.y * q.z + 2.f * q.w * q.x;
  m.r2.z = 1.f - 2.f * q.x * q.x - 2.f * q.y * q.y;
}

//
// Box operations.
//

struct pLine {
  __device__ pLine() {};
  __device__ pLine(pCoor s, pVect d, float l):start(s),dir(d),len(l){};
  pCoor start;
  pVect dir;
  float len;
};


__device__ int8_t
get_edge_vtx_idx(int edge)
{
  // Index: xyz (z is LSB).
#if 1
  const int axis = edge >> 2;
  const int mask = 0xc >> axis;
  const int face_vtx = edge & 3;
  const int box_vtx_check = ( face_vtx & mask ) + face_vtx;
  return box_vtx_check;
#else
  static const int8_t bi[12] =
    {
      0, 1, 2, 3,
      0, 1, 4, 5,
      0, 2, 4, 6
    };
  return bi[edge];
#endif
}

__device__ float3
box_get_vertices(CUDA_Box_W& box, int vertex)
{
  return box.position + mm_transpose(box.rot_inv,sign_mask(vertex,box.to_111));
}

__device__ float3
box_get_axis_norm(CUDA_Box_W& box, int axis)
{
  return axis == 0 ? box.rot_inv.r0 :
    axis == 1 ? box.rot_inv.r1 : box.rot_inv.r2;
}

__device__ float3
box_get_face_norm(CUDA_Box_W& box, int face)
{
  pVect norm_raw = box_get_axis_norm(box,face>>1);
  return face & 1 ? norm_raw : -norm_raw;
}

__device__ float
box_get_axis_len(CUDA_Box_W& box, int axis)
{
  return 2.0f * 
    ( axis == 0 ? box.to_111.x : axis == 1 ? box.to_111.y : box.to_111.z );
}

__device__ pLine
box_get_edge(CUDA_Box_W& box, int edge)
{
  const int axis = edge >> 2;
  const int8_t box_vtx = get_edge_vtx_idx(edge);
  return
    pLine(box_get_vertices(box,box_vtx), 
          box_get_axis_norm(box,axis), 
          box_get_axis_len(box,axis));
}

__device__ void
box_set_mi_vec(CUDA_Box_W& box,float3 to_111)
{
  pVect dsq = to_111 * to_111;
  float dsqs = dsq.x + dsq.y + dsq.z;
  float mass_factor = 1.0f / ( box.mass_inv * 3.0f );
  box.mi_vec = mass_factor * ( mv(dsqs) - dsq );
}

__device__ void
box_set_mi_vec(CUDA_Box_W& box)
{
  box_set_mi_vec(box,box.to_111);
}

__device__ float
box_get_moment_of_inertia_inv(CUDA_Box_W& box, pNorm axis);


__device__ float3
box_get_vel(CUDA_Box_W&box, float3 pos)
{
  pVect cent_to_pt = mv(box.position,pos);
  pVect rot_vel = cross(box.omega,cent_to_pt);
  return rot_vel + box.velocity;
}

__device__ void
box_geometry_update(CUDA_Box_W& box)
{
  pMatrix_set_rotation_transpose(box.rot_inv, box.orientation);
  box_set_mi_vec(box);
}

__device__ void
box_apply_force_dt(CUDA_Box_W& box, float3 tact, float3 force)
{
  box.velocity += box.mass_inv * force;
  pVect cent_to_tact = mv(box.position,tact);
  pVect torque = cross(cent_to_tact,force);
  pNorm torqueN = mn(torque);
  float mi_inv = box_get_moment_of_inertia_inv(box,torqueN);
  box.omega += mi_inv * torque;
}

__device__ float
box_get_moment_of_inertia_inv(CUDA_Box_W& box, pNorm axis)
{
  if ( axis.mag_sq < 1e-11f || box.mass_inv == 0 ) return 0;
  pVect tl = box.rot_inv * axis.v;
  pVect tls = tl * tl;
  float mi = dot(tls,box.mi_vec);
  return 1.0f / mi;
}

__device__ float
box_get_moment_of_inertia_inv(CUDA_Box_W& box, float3 tact, pNorm dir)
{
  pVect cent_to_tact = mv(box.position,tact);
  pNorm torque_axis = mn(cross(cent_to_tact,dir));
  return box_get_moment_of_inertia_inv(box,torque_axis);
}

__device__ void
box_apply_force_fric_dt
(CUDA_Box_W& box, float3 tact, pNorm force_dir, float force_mag_dt)
{
  box_apply_force_dt(box,tact,force_mag_dt*force_dir);
}

__device__ CUDA_SectTT
sect_init()
{
  CUDA_SectTT sect;
  sect.exists = false;
  return sect;
}

#include "k-boxes.h"

///
/// Pass Box/Box Intersect
///

__global__ void pass_xx_intersect(int xx_pairs_count);

__host__ void
pass_xx_intersect_launch(dim3 dg, dim3 db, int xx_pairs_count)
{
  const int shared_amt = 0;
  pass_xx_intersect<<<dg,db,shared_amt>>>(xx_pairs_count);
}

__device__ void
penetration_boxes_resolve_force
(CUDA_Box_W& box1, CUDA_Box_W& box2, float3 pos, pNorm sep_normal)
{
  const float pen_dist = 0.1f * sep_normal.magnitude;

  pVect vel1 = box_get_vel(box1,pos);
  pVect vel2 = box_get_vel(box2,pos);
  pVect velto1 = vel2 - vel1;

  const float sep_vel = dot(velto1,sep_normal.v);

  const float loss_factor = 1 - opt_bounce_loss_box;
  const float force_dt_no_loss = elasticity_inv_dt * pen_dist;
  const bool separating = sep_vel >= 0;
  const float appr_force_dt = separating
    ? force_dt_no_loss * loss_factor : force_dt_no_loss;

  pVect sep_force = appr_force_dt * sep_normal.v;

  box_apply_force_dt(box1, pos, -sep_force );
  box_apply_force_dt(box2, pos, sep_force );
}

__device__ void
penetration_boxes_resolve_fric
(CUDA_Box_W& box1, CUDA_Box_W& box2, float3 pos, pNorm sep_normal)
{
  const float pen_dist = 0.1f * sep_normal.magnitude;
  const float force_dt_no_loss = elasticity_inv_dt * pen_dist;
  const float fric_force_dt_potential =
    force_dt_no_loss * opt_friction_coeff;
  
  /// Torque
  //
  //
  // Account for forces of surfaces twisting against each
  // other. (For example, if one box is spinning on top of
  // another.)
  //
  const float appr_omega =
    dot(box2.omega,sep_normal) - dot(box1.omega,sep_normal);
  {
    const float mi1_inv = box_get_moment_of_inertia_inv(box1,sep_normal);
    const float mi2_inv = box_get_moment_of_inertia_inv(box2,sep_normal);
    const float fdt_limit = fabs(appr_omega) / ( mi1_inv + mi2_inv );
    const bool rev = appr_omega < 0;
    const float fdt_raw = min(fdt_limit,fric_force_dt_potential);
    const pVect fdt_v = ( rev ? -fdt_raw : fdt_raw ) * sep_normal;
    box1.omega += mi1_inv * fdt_v;
    box2.omega -= mi2_inv * fdt_v;
  }

  pVect vel1b = box_get_vel(box1,pos);
  pVect vel2b = box_get_vel(box2,pos);
  pVect velto1b = vel2b - vel1b;

  const float sep_velb = dot(velto1b,sep_normal);
  pNorm tan_vel = mn(velto1b - sep_velb * sep_normal);

  const float fdt_limit =
    0.5f *
    tan_vel.magnitude /
    ( box1.mass_inv + box2.mass_inv
      + box_get_moment_of_inertia_inv(box1,pos,tan_vel)
      + box_get_moment_of_inertia_inv(box2,pos,tan_vel) );

  const float fric_force_dt = min(fdt_limit,fric_force_dt_potential);

  box_apply_force_fric_dt(box1,pos, tan_vel, fric_force_dt);
  box_apply_force_fric_dt(box2,pos, -tan_vel, fric_force_dt);
}


__device__ bool
penetration_boxes_resolve
(CUDA_Phys_W& phys1, CUDA_Phys_W& phys2, int tsidx, Force_Types ft)
{
  /// Update velocity and angular momentum for a pair of boxes in contact.

  CUDA_Box_W& box1 = phys1.box;
  CUDA_Box_W& box2 = phys2.box;

  float4 dir_and_mag = xx_sects_dir[tsidx];
  if ( dir_and_mag.w == 0 ) return false;
  float4 center_and_um = xx_sects_center[tsidx];
  float3 center = m3(center_and_um);
  pNorm sep_normal = mn(dir_and_mag);
  if ( ft & FT_NonFriction )
    penetration_boxes_resolve_force(box1,box2,center,sep_normal);
  if ( ft & FT_Friction )
    penetration_boxes_resolve_fric(box1,box2,center,sep_normal);
  return true;
}


///
/// Pairs Pass
///
//
// Resolve ball collisions with each other.

__global__ void pass_pairs
(int prefetch_offset, int schedule_offset, int round_cnt, 
 int max_balls_per_thread, int balls_per_block, Force_Types ft);

__host__ void
pass_pairs_launch
(dim3 dg, dim3 db, int prefetch_offset, int schedule_offset, int round_cnt,
 int max_balls_per_thread, int balls_per_block, Force_Types ft)
{
#ifdef USE_STRUCT
  const int shared_amt = balls_per_block * sizeof(CUDA_Phys_W);
#else
  const int shared_amt = balls_per_block * sizeof(sm_balls[0]) * 8;
#endif
  pass_pairs<<<dg,db,shared_amt>>>
    (prefetch_offset, schedule_offset, round_cnt,
     max_balls_per_thread, balls_per_block, ft);
}

#ifndef USE_STRUCT
struct SM_Offsets {
  int idx_pos;
  int idx_vel;
  int idx_omega;
  int idx_prev_vel;
  int idx_rad_etc;
  int idx_to_111;
  int idx_ori_xyz;
  int factor;
};

__device__ CUDA_Phys_W
get_sm_ball(SM_Offsets& smo, int idx)
{
  CUDA_Phys_W phys;
  const int sidx = idx * smo.factor;
  phys.box.velocity = sm_balls[smo.idx_vel+sidx];
  phys.box.prev_velocity = sm_balls[smo.idx_prev_vel+sidx];
  phys.box.position = sm_balls[smo.idx_pos+sidx];
  phys.box.omega = sm_balls[smo.idx_omega+sidx];
  phys.box.radius = sm_balls[smo.idx_rad_etc+sidx].x;
  phys.box.mass_inv = sm_balls[smo.idx_rad_etc+sidx].y;
  phys.read_only = phys.box.mass_inv == 0;
  return phys;
}

__device__ void
upgrade_sm_box(CUDA_Phys_W& phys, SM_Offsets& smo, int idx)
{
  const int sidx = idx * smo.factor;
  float4 ori;
  set_f4(ori,sm_balls[smo.idx_ori_xyz+sidx],
         sm_balls[smo.idx_rad_etc+sidx].z);
  pMatrix_set_rotation_transpose(phys.box.rot_inv,ori);
  float3 to_111 = sm_balls[smo.idx_to_111+sidx];
  phys.box.to_111 = to_111;
  box_set_mi_vec(phys.box);
}

__device__ void
put_sm_phys(SM_Offsets& smo, int sidx, CUDA_Phys_W& phys)
{
  sm_balls[smo.idx_vel+sidx] = phys.ball.velocity;
  sm_balls[smo.idx_omega+sidx] = phys.ball.omega;
}
#endif

__global__ void
pass_pairs(int prefetch_offset, int schedule_offset, int round_cnt,
           int max_balls_per_thread, int balls_per_block, Force_Types ft)
{
  const int tid = threadIdx.x;

  // Initialized variables used to access balls_needed and tacts_schedule
  // arrays.
  //
  const int si_block_size = blockIdx.x * max_balls_per_thread * blockDim.x;
  const int si_block_base = prefetch_offset + si_block_size + tid;
  const int sp_block_size = blockIdx.x * round_cnt * blockDim.x;
  const int sp_block_base = schedule_offset + sp_block_size + tid;

  /// Prefetch objects to shared memory.
  //
#ifdef USE_STRUCT
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= balls_per_block ) continue;
      const int m_idx = block_balls_needed[ si_block_base + i * blockDim.x ];
      CUDA_Phys_W& phys = sm_balls[idx];
      CUDA_Ball_W& ball = phys.ball;
      CUDA_Box_W& box = phys.box;
      phys.m_idx = m_idx;
      if ( m_idx < 0 ) continue;

      int4 tact_counts = balls_x.tact_counts[m_idx];
      phys.pt_type = tact_counts.x;
      phys.contact_count = tact_counts.y;
      phys.debug_pair_calls = tact_counts.z;
      phys.part_of_wheel = bool(tact_counts.w & 2);
      phys.read_only = tact_counts.w & 1;

      ball.velocity = xyz(balls_x.velocity[m_idx]);
      ball.prev_velocity = xyz(balls_x.prev_velocity[m_idx]);
      ball.position = xyz(balls_x.position[m_idx]);
      ball.omega = xyz(balls_x.omega[m_idx]);
      float4 ball_props = balls_x.ball_props[m_idx];
      ball.radius = ball_props.x;
      ball.mass_inv = ball_props.y;
      ball.pad1 = ball_props.z;
      ball.pad2 = ball_props.w;
      if ( phys.pt_type == PT_Box )
        {
          set_f3(box.to_111, balls_x.to_111[m_idx]);
          box.orientation = balls_x.orientation[m_idx];
          box_geometry_update(box);
        }
    }
#else

  SM_Offsets smo;
  smo.idx_pos = 0;
  smo.idx_vel = 1;
  smo.idx_omega = 2;
  smo.idx_prev_vel = 3;
  smo.idx_rad_etc = 4;
  smo.idx_to_111 = 5;
  smo.idx_ori_xyz = 6;
  smo.factor = 7;

  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= balls_per_block ) continue;
      const int m_idx = block_balls_needed[ si_block_base + i * blockDim.x ];

      if ( m_idx < 0 ) continue;

      int4 tact_counts = balls_x.tact_counts[m_idx];
      const int pt_type = tact_counts.x;
      sm_balls_misc[idx].x = tact_counts.x; // pt_type
      sm_balls_misc[idx].y = tact_counts.y; // contact count
      sm_balls_misc[idx].z = tact_counts.z; // debug_pair_calls
      sm_balls_misc[idx].w = tact_counts.w; // Part of wheel is bit 0x2

      const int sidx = idx * smo.factor;

      sm_balls[smo.idx_vel+sidx] = m3(balls_x.velocity[m_idx]);
      sm_balls[smo.idx_prev_vel+sidx] = m3(balls_x.prev_velocity[m_idx]);
      sm_balls[smo.idx_pos+sidx] = m3(balls_x.position[m_idx]);
      sm_balls[smo.idx_omega+sidx] = m3(balls_x.omega[m_idx]);
      float4 props =balls_x.ball_props[m_idx];
      sm_balls[smo.idx_rad_etc+sidx] = m3(props);
      if ( pt_type == PT_Box )
        {
          sm_balls[smo.idx_to_111+sidx] = m3(balls_x.to_111[m_idx]);
          const float4 orientation = balls_x.orientation[m_idx];
          sm_balls[smo.idx_ori_xyz+sidx] = m3(orientation);
          sm_balls[smo.idx_rad_etc+sidx].z = orientation.w;
        }
    }
#endif

  const pVect zero_vec = mv(0,0,0);

  /// Resolve Collisions
  //
  for ( int round=0; round<round_cnt; round++ )
    {
      const int tsidx = sp_block_base + round * blockDim.x;
      SM_Idx2 indices = tacts_schedule[ tsidx ];
      const int ix = indices.x;
      const int iy = indices.y;

      // Wait for all threads to reach this point (to avoid having
      // two threads operate on the same ball simultaneously).
      //
      __syncthreads();

      if ( indices.x == indices.y ) continue;

#ifdef USE_STRUCT
      CUDA_Phys_W& physx = sm_balls[ix];
      CUDA_Phys_W& physy = sm_balls[iy];
      const unsigned char ptx = physx.pt_type;
      const unsigned char pty = physy.pt_type;
#else
      const int six = ix * smo.factor;
      const int siy = iy * smo.factor;
      CUDA_Phys_W physx = get_sm_ball(smo,ix);
      CUDA_Phys_W physy = get_sm_ball(smo,iy);
      const int ptx = sm_balls_misc[ix].x;
      const int pty = sm_balls_misc[iy].x;
#endif

      if ( ft & FT_NonFriction )
        {
#ifdef USE_STRUCT          
          physx.debug_pair_calls++; physy.debug_pair_calls++;
#else
          sm_balls_misc[ix].z++; sm_balls_misc[iy].z++;
#endif
        }

      char rv;

      if ( ptx == PT_Box && pty == PT_Box )
        {
#ifndef USE_STRUCT
          upgrade_sm_box(physx,smo,ix);
          upgrade_sm_box(physy,smo,iy);
#endif
          rv = penetration_boxes_resolve(physx,physy,tsidx,ft);
        }
      else if ( ptx == PT_Ball && pty == PT_Box )
        {
#ifndef USE_STRUCT
          upgrade_sm_box(physy,smo,iy);
#endif
          rv = penetration_box_ball_resolve(physy,physx,ft);
        }
      else if ( pty == PT_Ball )
        {
          CUDA_Ball_W& ballx = physx.ball;
          CUDA_Ball_W& bally = physy.ball;
          rv = penetration_balls_resolve(ballx,bally,true,ft);
        }
      else if ( pty == PT_Box )
        {
          // Note: Tile / Box collisions not yet handled.
          rv = 0;
        }
      else
        {
          CUDA_Ball_W& ballx = physx.ball;
          CUDA_Tile_W& tiley = physy.tile;
          pCoor tact_pos;
          pVect tact_dir;
          rv = tile_ball_collide(tiley, ballx, tact_pos, tact_dir);
          if ( !rv ) continue;
          CUDA_Ball_W pball;
          pball.radius = 1;
          pball.omega = pball.prev_velocity = pball.velocity = zero_vec;
          pball.position = tact_pos + tact_dir;
          pVect vbefore = physx.ball.velocity;
          penetration_balls_resolve(ballx, pball, false, ft);
          pVect delta_mo = ( 1.0f / ballx.mass_inv )
            * ( ballx.velocity - vbefore );
#ifdef USE_STRUCT
          const bool part_of_wheel = physy.part_of_wheel;
#else
          const bool part_of_wheel = sm_balls_misc[iy].w & 2;
#endif
          if ( part_of_wheel )
            {
              wheel_collect_tile_force(tiley, tact_pos, delta_mo);
              // Note: Need to fix this.
            }
#ifndef USE_STRUCT
          put_sm_phys(smo,six,physx);
          sm_balls_misc[ix].y += 1;
          continue;
#endif
        }

#ifdef USE_STRUCT
      physx.contact_count += rv; physy.contact_count += rv;
#else
      put_sm_phys(smo,six,physx);
      put_sm_phys(smo,siy,physy);
      sm_balls_misc[ix].y += rv; sm_balls_misc[iy].y += rv;
#endif
    }

  __syncthreads();

  /// Copy Ball Data to Memory
  //
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= balls_per_block ) continue;

#ifdef USE_STRUCT
      CUDA_Phys_W& phys = sm_balls[idx];
      const int m_idx = phys.m_idx;
      if ( m_idx < 0 ) continue;
      if ( phys.read_only ) continue;
#else
      const int sidx = idx * smo.factor;
      const int m_idx = block_balls_needed[ si_block_base + i * blockDim.x ];
      if ( m_idx < 0 ) continue;
      const float mass_inv = sm_balls[smo.idx_rad_etc+sidx].y;
      const bool read_only = mass_inv == 0;
      if ( read_only ) continue;
#endif

#ifdef USE_STRUCT
      CUDA_Ball_W& ball = phys.ball;

      int4 tact_counts;
      tact_counts.x = phys.pt_type;
      tact_counts.y = phys.contact_count;
      tact_counts.z = phys.debug_pair_calls;
      tact_counts.w = phys.part_of_wheel;
      balls_x.tact_counts[m_idx] = tact_counts;
      const char pt_type = phys.pt_type;
      set_f4(balls_x.velocity[m_idx], ball.velocity);
      if ( pt_type == PT_Tile ) continue;
      set_f4(balls_x.omega[m_idx], ball.omega);
#else
      balls_x.tact_counts[m_idx].y = sm_balls_misc[idx].y;
      balls_x.tact_counts[m_idx].z = sm_balls_misc[idx].z;
      const unsigned char pt_type = sm_balls_misc[idx].x;
      set_f4(balls_x.velocity[m_idx], sm_balls[smo.idx_vel+sidx]);
      if ( pt_type == PT_Tile ) continue;
      set_f4(balls_x.omega[m_idx], sm_balls[smo.idx_omega+sidx]);
#endif
    }
}


///
/// Platform Pass
///
//
// Resolve ball collisions with platform, also update ball position
// and orientation.

__device__ void platform_collision(CUDA_Phys_W& phys);
__device__ void platform_collision_box(CUDA_Phys_W& phys);
__global__ void pass_platform(int ball_count);
__device__ void pass_platform_ball(CUDA_Phys_W& phys, int idx);
__device__ void pass_platform_tile(CUDA_Phys_W& phys, int idx);
__device__ void pass_platform_box(CUDA_Phys_W& phys, int idx);


__host__ hipError_t
cuda_get_attr_plat_pairs
(struct hipFuncAttributes *attr_platform,
 struct hipFuncAttributes *attr_pairs,
 struct hipFuncAttributes *attr_xx_intersect)
{
  collect_symbols();

  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.
  hipError_t e1 = hipFuncGetAttributes(attr_platform,reinterpret_cast<const void*>(pass_platform));
  if ( e1 ) return e1;
  hipError_t e2 = hipFuncGetAttributes(attr_pairs,reinterpret_cast<const void*>(pass_pairs));
  if ( e2 ) return e2;
  hipError_t e3 = hipFuncGetAttributes(attr_xx_intersect,reinterpret_cast<const void*>(pass_xx_intersect));
  return e3;
}

__host__ void
pass_platform_launch
(dim3 dg, dim3 db, int ball_count)
{
  const int block_lg = 32 - __builtin_clz(db.x-1);
  const int shared_amt = sizeof(float) << block_lg;
  pass_platform<<<dg,db,shared_amt>>>(ball_count);
}

__global__ void
pass_platform(int ball_count)
{
  /// Main CUDA routine for resolving collisions with platform and
  /// updating ball position and orientation.

  // One ball per thread.

  const int idx_base = blockIdx.x * blockDim.x;
  const int idx = idx_base + threadIdx.x;

  if ( idx >= ball_count ) return;

  CUDA_Phys_W phys;

  /// Copy ball data from memory to local variables.
  //
  //  Local variables hopefully will be in GPU registers, not
  //  slow local memory.
  //
  int4 tact_counts = balls_x.tact_counts[idx];
  phys.pt_type = tact_counts.x;
  phys.contact_count = tact_counts.y;
  phys.part_of_wheel = tact_counts.w & 1;

  if ( phys.pt_type == PT_Ball )     pass_platform_ball(phys, idx);
  else if ( phys.pt_type == PT_Box ) pass_platform_box(phys, idx);
  else                               pass_platform_tile(phys, idx);

  /// Copy other updated data to memory.
  //
  tact_counts.y = phys.contact_count << 8;
  tact_counts.z = tact_counts.z << 16;
  balls_x.tact_counts[idx] = tact_counts;
}

__device__ void
pass_platform_ball(CUDA_Phys_W& phys, int idx)
{
  // One ball per thread.

  CUDA_Ball_W& ball = phys.ball;

  /// Copy ball data from memory to local variables.
  //
  //  Local variables hopefully will be in GPU registers, not
  //  slow local memory.
  //

  ball.prev_velocity = xyz(balls_x.prev_velocity[idx]);
  ball.velocity = xyz(balls_x.velocity[idx]) + gravity_accel_dt;
  set_f3(ball.position,balls_x.position[idx]);
  set_f3(ball.omega, balls_x.omega[idx]);
  float4 ball_props = balls_x.ball_props[idx];
  ball.radius = ball_props.x;
  ball.mass_inv = ball_props.y;

  /// Handle Ball/Platform Collision
  //
  platform_collision(phys);

  /// Update Position and Orientation
  //
  ball.position +=
    0.5f * delta_t * ( ball.prev_velocity + ball.velocity );

  pNorm axis = mn(ball.omega);
  balls_x.orientation[idx] =
    quat_normalize
    ( quat_mult
      ( mq( axis, delta_t * axis.magnitude ), balls_x.orientation[idx] ));

  /// Copy other updated data to memory.
  //
  set_f4(balls_x.velocity[idx], ball.velocity);
  set_f4(balls_x.prev_velocity[idx], ball.velocity);
  set_f4(balls_x.omega[idx], ball.omega);
  set_f4(balls_x.position[idx], ball.position, ball.radius);
}


__device__ void
pass_platform_tile(CUDA_Phys_W& phys, int idx)
{
  if ( !phys.part_of_wheel ) return;

  const int tid = threadIdx.x;
  float4 tile_props = balls_x.velocity[idx];
  float torque = tile_props.z;
  block_torque_dt[tid] = torque;
  tile_props.z = 0;
  balls_x.velocity[idx] = tile_props;

  float omega = wheel.omega[0];

  const float3 pt_ll = xyz(balls_x.position[idx]);
  const float3 norm_rt = xyz(balls_x.omega[idx]);
  const float3 norm_up = xyz(balls_x.prev_velocity[idx]);
  const float3 normal = xyz(balls_x.ball_props[idx]);

  float torque_sum = 0;
  // Assuming that all are on same warp. :-)
  for ( int i=wheel.idx_start; i<wheel.idx_stop; i++ )
    torque_sum += block_torque_dt[i];

  omega -= torque_sum * wheel.moment_of_inertia_inv;

  const float friction_delta_omega = 
    wheel.friction_torque * wheel.moment_of_inertia_inv * delta_t;
  if ( fabs(omega) <= friction_delta_omega ) omega = 0;
  else if ( omega > 0 )                      omega -= friction_delta_omega;
  else                                       omega += friction_delta_omega;

  const float delta_theta = omega * delta_t;

  pMatrix3x3 rot;
  pMatrix_set_rotation(rot,wheel.axis_dir,delta_theta);
  const float3 rpt_ll = wheel.center + rot * ( pt_ll - wheel.center );
  const float3 rnorm_rt = rot * norm_rt;
  const float3 rnorm_up = rot * norm_up;
  const float3 rnormal = rot * normal;

  set_f4(balls_x.position[idx],rpt_ll);
  set_f4(balls_x.omega[idx], rnorm_rt);
  set_f4(balls_x.prev_velocity[idx], rnorm_up);
  set_f4(balls_x.ball_props[idx], rnormal);
  if ( idx == wheel.idx_start ) wheel.omega[0] = omega;
}

__device__ void
pass_platform_box(CUDA_Phys_W& phys, int idx)
{
  // One box per thread.

  CUDA_Box_W& box = phys.box;

  /// Copy data from memory to local variables.
  //
  //  Local variables hopefully will be in GPU registers, not
  //  slow local memory.
  //

  float4 box_props = balls_x.ball_props[idx];
  box.mass_inv = box_props.y;
  if ( box.mass_inv == 0 ) return; // Read only.
  box.prev_velocity = xyz(balls_x.prev_velocity[idx]);
  box.velocity = xyz(balls_x.velocity[idx]) + gravity_accel_dt;
  set_f3(box.position,balls_x.position[idx]);
  set_f3(box.omega, balls_x.omega[idx]);
  set_f3(box.to_111, balls_x.to_111[idx]);
  box.orientation = balls_x.orientation[idx];

  /// Handle Ball/Platform Collision
  //
  platform_collision_box(phys);

  /// Update Position and Orientation
  //
  box.position +=
    0.5f * delta_t * ( box.prev_velocity + box.velocity );

  pNorm axis = mn(box.omega);
  balls_x.orientation[idx] =
    quat_normalize
    ( quat_mult
      ( mq( axis, delta_t * axis.magnitude ), box.orientation ));

  /// Copy other updated data to memory.
  //
  set_f4(balls_x.velocity[idx], box.velocity);
  set_f4(balls_x.prev_velocity[idx], box.velocity);
  set_f4(balls_x.omega[idx], box.omega);
  set_f4(balls_x.position[idx], box.position, box_props.x);
}

__device__ void
platform_collision_box(CUDA_Phys_W& phys)
{
  CUDA_Box_W& box = phys.box;

  float radius = length(box.to_111);

  if ( box.position.y - radius >= 0 ) return;
  if ( box.position.z + radius <= platform_zmin ) return;
  if ( box.position.z - radius >= platform_zmax ) return;

  float3 axis = mv(platform_xmid,0,box.position.z);
  pVect btoa = mv(box.position,axis);
  if ( dot(btoa,btoa) < (platform_xrad-radius)*(platform_xrad-radius) ) return;

  box_geometry_update(box);

  int inside = 0;
  int outside_under = 0;
  float pen_dists[8];
  CUDA_SectTT psects[5];
  int ps_next = 0;
  float min_pd = 0;  // For vertices between ends.
  float max_pd = 0;

  // Find vertices that are under the platform.
  //
  for ( int v=0; v<8; v++ )
    {
      int v_bit = 1 << v;
      float3 pos = box_get_vertices(box,v);
      if ( pos.y > 0 ) { pen_dists[v] = 0; continue; }
      float3 axis = mc(platform_xmid,0,pos.z);
      pNorm tact_dir = mn(axis,pos);
      float pen_dist = tact_dir.magnitude - platform_xrad;
      pen_dists[v] = pen_dist;
      if ( pos.z < platform_zmin || pos.z > platform_zmax )
        {
          if ( pen_dist > 0 ) outside_under |= v_bit;
          continue;
        }
      set_min(min_pd,pen_dist);
      set_max(max_pd,pen_dist);
      if ( pen_dist > 1 ) continue;
      inside |= v_bit;
      if ( pen_dist <= 0 ) continue;
      CUDA_SectTT* sect = &psects[ps_next++];
      sect->start = pos;
      sect->dir = tact_dir.v;
      sect->pen_dist = pen_dist;
    }

  bool object_inside = max_pd < -min_pd;
  if ( !object_inside ) return;

  // Examine vertices that are off the edge of the platform (in the
  // z direction), to see if an adjoining edge intersects the platform
  // edge.
  //
  for ( int v=0; v<8; v++ )
    {
      int v_bit = 1 << v;
      if ( ! ( v_bit & outside_under )  ) continue;

      // Outside Vertex (beyond z_max or z_min).
      //
      pCoor pos = box_get_vertices(box,v);
      float pen_dist_out = pen_dists[v];
      float v_z = pos.z;
      float ref_z =
        v_z >= platform_zmax ? platform_zmax : platform_zmin;
      float outside_z_len = fabs(v_z - ref_z);

      // Look for adjoining vertices that are over the platform.
      //
      for ( int axis = 0; axis < 3; axis++ )
        {
          int vn = v ^ ( 1 << axis );
          int vn_bit = 1 << vn;
          if ( ! ( inside & vn_bit ) ) continue;
          float pen_len = pen_dists[vn] - pen_dist_out;
          // Inside Vertex
          pCoor pos_in = box_get_vertices(box,vn);

          // Compute the contact point at penetration distance.
          //
          float z_len = fabs(v_z - pos_in.z);
          if ( z_len < 0.0001f ) continue;
          float scale = outside_z_len / z_len;
          pVect to_inside = mv(pos,pos_in);
          pCoor tact = pos + scale * to_inside;
          float pen_tact = pen_dist_out + scale * pen_len;
          if ( pen_tact <= 0 ) continue;
          CUDA_SectTT* sect = &psects[ps_next++];
          sect->start = tact;
          sect->pen_dist = pen_tact;
          pNorm dir = mn(cross(to_inside,mv(-tact.y,tact.x,0)));
          sect->dir = pen_len >= 0 ? normalize(mv(tact.x,tact.y,0)) : dir.v;
        }
    }

  //  if ( ps_next > 0 ) phys.contact_count++;

  for ( int i=0; i<ps_next; i++ )
    {
      CUDA_SectTT *sect = &psects[i];
      pCoor pos = sect->start;
      pVect tact_dir = sect->dir;
      pNorm ctopos = mn(box.position,pos);
      pVect vel = box_get_vel(box,pos);
      float pen_dist = sect->pen_dist;
      float rad_vel = dot(vel,tact_dir);
      double loss_factor = 1 - opt_bounce_loss;
      float force_dt_no_loss = elasticity_inv_dt * pen_dist;
      float max_fdt_in = rad_vel / box.mass_inv;
      float appr_force_dt = rad_vel > 0
        ? min(max_fdt_in,force_dt_no_loss) : force_dt_no_loss * loss_factor;
      box_apply_force_dt(box,pos, - appr_force_dt * tact_dir );
    }

  for ( int i=0; i<ps_next; i++ )
    {
      CUDA_SectTT *sect = &psects[i];
      pCoor pos = sect->start;
      pVect tact_dir = sect->dir;
      float pen_dist = sect->pen_dist;
      float force_dt_no_loss = elasticity_inv_dt * pen_dist;
      pVect vel2 = box_get_vel(box,pos);
      float rad_vel2 = dot(vel2,tact_dir);
      pNorm tan_vel = mn( vel2 - rad_vel2 * tact_dir );
      float mi_inv = box_get_moment_of_inertia_inv(box,pos,tan_vel);
      float fdt_limit = 
        tan_vel.magnitude / ( box.mass_inv + mi_inv );
      float fric_force_dt_no_loss =
        force_dt_no_loss * opt_friction_coeff;
      float fric_force_dt = min(fdt_limit, fric_force_dt_no_loss);
      box_apply_force_fric_dt(box,pos, tan_vel, -fric_force_dt);
    }
}

__device__ void
platform_collision(CUDA_Phys_W& phys)
{
  /// Check if ball in contact with platform, if so apply forces.

  CUDA_Ball_W& ball = phys.ball;

  pCoor pos = ball.position;
  const float r = ball.radius;
  bool collision_possible =
    pos.y < r
    && pos.x >= platform_xmin - r && pos.x <= platform_xmax + r
    && pos.z >= platform_zmin - r && pos.z <= platform_zmax + r;

  if ( !collision_possible ) return;

  CUDA_Ball_W pball;

  pCoor axis = mc(platform_xmid,0,pos.z);
  const float short_xrad = platform_xrad - r;
  const float short_xrad_sq = short_xrad * short_xrad;
  const float long_xrad = platform_xrad + r;
  const float long_xrad_sq = long_xrad * long_xrad;

  // Test for different ways ball can touch platform. If contact
  // is found find position of an artificial platform ball (pball)
  // that touches the real ball at the same place and angle as
  // the platform. This pball will be used for the ball-ball penetration
  // routine, penetration_balls_resolve.

  if ( pos.y > 0 )
    {
      // Possible contact with upper edge of platform.
      //
      pCoor tact
        = mc(pos.x > platform_xmid ? platform_xmax : platform_xmin, 0, pos.z);
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= r * r ) return;
      pball.position = tact + r * tact_dir;
    }
  else if ( pos.z > platform_zmax || pos.z < platform_zmin )
    {
      // Possible contact with side (curved) edges of platform.
      //
      pNorm ball_dir = mn(axis,pos);
      if ( ball_dir.mag_sq <= short_xrad_sq ) return;
      const float zedge =
        pos.z > platform_zmax ? platform_zmax : platform_zmin;
      pCoor axis_edge = mc(platform_xmid,0,zedge);
      pCoor tact = axis_edge + platform_xrad * ball_dir;
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= r * r ) return;
      pball.position = tact + r * tact_dir;
    }
  else
    {
      // Possible contact with surface of platform.
      //
      pNorm tact_dir = mn(axis,pos);
      if ( tact_dir.mag_sq <= short_xrad_sq
           || tact_dir.mag_sq >= long_xrad_sq ) return;
      
      pball.position = axis +
        ( platform_xrad + ( tact_dir.magnitude < platform_xrad ? r : -r ) )
        * tact_dir;
    }

  // Finish initializing platform ball, and call routine to
  // resolve penetration.
  //
  pVect zero_vec = mv(0,0,0);
  pball.omega = zero_vec;
  pball.prev_velocity = pball.velocity = zero_vec;
  pball.radius = ball.radius;
  pball.mass_inv = ball.mass_inv;
  if ( penetration_balls_resolve(phys.ball,pball,false,FT_All) )
    phys.contact_count++;
}

 /// Compute Phys Proximity Pairs

// Mapping from z-sort index to ball array index.
__constant__ int *z_sort_indices;

// Pre-computed z_max values.
__constant__ float *z_sort_z_max;

// Computed proximity values, sent to CPU.
__constant__ int64_t *cuda_prox;

// An array that can be used to pass values back to the CPU for
// use in debugging.
__constant__ float3 *pass_sched_debug;

texture<float4> balls_pos_tex;
texture<float4> balls_vel_tex;

__global__ void pass_sched(int ball_count, float lifetime_delta_t);
__device__ float ball_min_z_get
(float3 position, float3 velocity, float radius, float lifetime_delta_t);

__host__ bool
pass_sched_launch
(dim3 dg, dim3 db, int ball_count, float lifetime_delta_t,
 void *pos_array_dev, void *vel_array_dev)
{
  size_t offset;
  const size_t size = ball_count * sizeof(float4);
  const hipChannelFormatDesc fd =
    hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
  hipBindTexture(&offset, balls_pos_tex, pos_array_dev, fd, size);
  if ( offset ) return false;
  hipBindTexture(&offset, balls_vel_tex, vel_array_dev, fd, size);
  if ( offset ) return false;

  pass_sched<<<dg,db>>>(ball_count,lifetime_delta_t);

  return true;
}

__global__ void
pass_sched(int ball_count, float lifetime_delta_t)
{
  // Determine which balls that are in proximity to a ball. This
  // routine only works for balls, if a tile is found an I-give-up
  // value is returned, and the CPU will have to determine proximity.

  const int idx_base = blockIdx.x * blockDim.x;

  // idx9 is an index into z-sorted arrays.
  const int idx9 = idx_base + threadIdx.x;

  if ( idx9 >= ball_count ) return;

  // bidx9 is an index into the balls arrays.
  const int bidx9 = z_sort_indices[idx9];

  // If bidx9 is negative then Phys at index bidx9 is not a ball,
  // so just return a give-up code 't' (tile).
  if ( bidx9 < 0 )
    {
      cuda_prox[idx9] = ( 't' << 8 ) | 0xff;
      return;
    }

  // Fetch position, radius (packed in position vector), and velocity.
  //
  const float4 pos_rad9 = tex1Dfetch(balls_pos_tex,bidx9);
  const float3 pos9 = xyz(pos_rad9);
  const float radius9 = pos_rad9.w;
  const float4 vel9_pad = tex1Dfetch(balls_vel_tex,bidx9);
  const float3 vel9 = xyz(vel9_pad);

  const float z_min = ball_min_z_get(pos9,vel9,radius9,lifetime_delta_t);

  // Number of nearby balls.
  int proximity_cnt = 0;

  // Reason for giving up, 0 means we didn't give up (yet).
  char incomplete = 0;

  // The list of balls in proximity, packed into a single integer.
  Prox_Offsets offsets = 0;

  for ( int idx1 = idx9-1; !incomplete && idx1 >= 0; idx1-- )
    {
      const float z_max = z_sort_z_max[idx1];

      // Break if this and subsequent z-ordered balls could not
      // possibly be in proximity.
      if ( z_max < z_min ) break;

      const int bidx1 = z_sort_indices[idx1];

      // If there's a tile here give up.
      // (t is for tile)
      if ( bidx1 < 0 ) { incomplete = 't'; continue; }

      const float4 pos_rad = tex1Dfetch(balls_pos_tex,bidx1);
      const float3 pos1 = xyz(pos_rad);
      const float4 vel_pad1 = tex1Dfetch(balls_vel_tex,bidx1);
      const float3 vel1 = xyz(vel_pad1);
      const float radius1 = pos_rad.w;

      // Use the pNorm constructor to compute the distance between two balls.
      pNorm dist = mn(pos1,pos9);

      // Balls are considered in proximity if they can be
      // this close over schedule lifetime.
      const float region_length_small = 1.11f * ( radius9 + radius1 );
      
      // Check if balls will be close enough over lifetime.
      pVect delta_v = vel9 - vel1;
      const float delta_d = lifetime_delta_t * length(delta_v);
      const float dist2 = dist.magnitude - delta_d;

      if ( dist2 > region_length_small ) continue; 

      // At this point the balls are considered in proximity, now
      // squeeze the value of bidx1 into eight bits by taking
      // the difference of z-sort indices, which should be close
      // together.
      const int offset = idx9 - idx1;

      // Ooops, exceeded the limit on the number of proximities.
      // (f is for full)
      if ( proximity_cnt >= cuda_prox_per_ball ) incomplete = 'f';

      // Ooops, the offset won't fit into 8 bits.
      // (o is for overflow)
      else if ( offset >= 255 )                  incomplete = 'o';

      // Everything is fine, slide the offset on to the list.
      else offsets = ( offsets << 8 ) | offset;

      proximity_cnt++;
    }

  // If code could not compute all proximities replace offsets with
  // the error code.
  if ( incomplete ) offsets = ( incomplete << 8 ) | 0xff;

  cuda_prox[idx9] = offsets;
}

__device__ float
ball_min_z_get
(float3 position, float3 velocity, float radius, float lifetime_delta_t)
{
  const float m = fabs(velocity.x) + fabs(velocity.y) + fabs(velocity.z);
  const float z_min = position.z + position.x - m * lifetime_delta_t
    - 2 * radius;
  return z_min;
}

static __host__ void collect_symbols()
{
  CU_SYM(balls_x);
  CU_SYM(block_balls_needed);
  CU_SYM(tacts_schedule);
  CU_SYM(xx_pairs);
  CU_SYM(xx_sects_center);
  CU_SYM(xx_sects_dir);
  CU_SYM(xx_sects_debug);
  CU_SYM(gravity_accel_dt);
  CU_SYM(opt_bounce_loss); CU_SYM(opt_bounce_loss_box);
  CU_SYM(opt_friction_coeff); CU_SYM(opt_friction_roll);
  CU_SYM(platform_xmin); CU_SYM(platform_xmax);
  CU_SYM(platform_zmin); CU_SYM(platform_zmax);
  CU_SYM(platform_xmid); CU_SYM(platform_xrad);
  CU_SYM(delta_t);
  CU_SYM(elasticity_inv_dt);
  CU_SYM(opt_debug); CU_SYM(opt_debug2);
  CU_SYM(wheel);
  CU_SYM(z_sort_indices);
  CU_SYM(z_sort_z_max);
  CU_SYM(cuda_prox);
  CU_SYM(pass_sched_debug);
}

