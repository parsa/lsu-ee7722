
#include <hip/hip_runtime.h>
///
/// Useful Functions and Types
///

typedef float3 pCoor;
typedef float3 pVect;

struct pMatrix3x3 { float3 r0, r1, r2; };
__device__ float3
make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}

__device__ float3 m3(float a, float b, float c){ return make_float3(a,b,c); }
__device__ float3 m3(float4 a){ return make_float3(a); }
__device__ float3 xyz(float4 a){ return m3(a); }
__device__ float4 m4(float3 v, float w) { return make_float4(v.x,v.y,v.z,w); }
__device__ float4 m4(float3 v) { return make_float4(v.x,v.y,v.z,0); }

__device__ float4 make_float4(float f) { return make_float4(f,f,f,f); }
__device__ float4 m4(float f) { return make_float4(f); }

__device__ pVect operator +(pVect a,pVect b)
{ return make_float3(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pVect operator -(pVect a,pVect b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(float4 a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator *(float s, pVect v)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator *(pVect v, float s)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ float4 operator *(float s, float4 v)
{return make_float4(s*v.x,s*v.y,s*v.z,s*v.w);}
__device__ pVect operator *(pVect u, pVect v)
{return make_float3(u.x*v.x,u.y*v.y,u.z*v.z);}
__device__ pVect operator -(pVect v) { return make_float3(-v.x,-v.y,-v.z); }
__device__ float3 operator -=(float3& a, pVect b) {a = a - b; return a;}
__device__ float3 operator +=(float3& a, pVect b) {a = a + b; return a;}

__device__ float3 operator *=(float3& a, float b)
{ a.x *= b;  a.y *= b; a.z *= b;  return a;}

struct pNorm {
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

// Make a Coordinate
__device__ pCoor 
mc(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pCoor mc(float4 c){ return make_float3(c.x,c.y,c.z); }

__device__ void set_f3(float3& a, float4 b){a.x = b.x; a.y = b.y; a.z = b.z;}
__device__ void set_f4(float4& a, float3 b)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = 1;}
__device__ void set_f4(float4& a, float3 b, float c)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = c;}

// Make a Vector
__device__ pVect
mv(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pVect mv(float3 a, float3 b) { return b-a; }
__device__ pVect mv(float a) { return make_float3(a,a,a); }

__device__ float dot(float4 a, float4 b)
{ return a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w;}
__device__ float dot(pVect a, pVect b){ return a.x*b.x + a.y*b.y + a.z*b.z;}
__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }
__device__ float dot(pNorm a, pVect b){ return dot(a.v,b); }
__device__ float dot3(float4 a, float4 b){ return dot(m3(a),m3(b)); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

// Make a Normal (a structure containing a normalized vector and length)
__device__ pNorm mn(pVect v)
{
  pNorm n;
  n.mag_sq = mag_sq(v);
  if ( n.mag_sq == 0 )
    {
      n.magnitude = 0;
      n.v.x = n.v.y = n.v.z = 0;
    }
  else
    {
      const float rsq = 1.0f/sqrtf(n.mag_sq);
      n.magnitude = 1.0f/rsq;
      n.v = rsq * v;
    }
  return n;
}
__device__ pNorm mn(float4 a, float4 b) {return mn(b-a);}
__device__ pNorm mn(pCoor a, pCoor b) {return mn(b-a);}
__device__ pNorm mn(float x, float y, float z) {return mn(mv(x,y,z));}
__device__ pNorm mn(float4 v4)
{ pNorm n; n.v = m3(v4);  n.magnitude = v4.w;  return n; }
__device__ pNorm mn(float3 v3, float mag)
{ pNorm n; n.v = v3;  n.magnitude = mag;  return n; }

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

struct pQuat {
  float3 v;
  float w;
};

// Make Quaternion
__device__ pQuat mq(pNorm axis, float angle)
{
  pQuat q;
  q.v = __sinf(angle/2) * axis.v;
  q.w = __cosf(angle/2);
  return q;
}

__device__ pQuat cast_quat(float4 v)
{
  pQuat q;
  q.v.x = v.x;
  q.v.y = v.y;
  q.v.z = v.z;
  q.w = v.w;
  return q;
}

__device__ pQuat cq(float4 v){ return cast_quat(v); }

__device__ pQuat quat_normalize(pQuat q)
{
  float len_sq = dot(q.v,q.v) + q.w * q.w;
  float norm_factor = 1.0f / sqrtf(len_sq);
  pQuat r;
  r.v = norm_factor * q.v;
  r.w = norm_factor * q.w;
  return r;
}

// Make float4
__device__ float4 c4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }
__device__ float4 m4(pNorm v, float w) { return m4(v.v,w); }

__device__ pVect fabs(pVect v){ return mv(fabs(v.x),fabs(v.y),fabs(v.z)); }
__device__ float min(pVect v){ return min(min(v.x,v.y),v.z); }
__device__ float max(pVect v){ return max(max(v.x,v.y),v.z); }
__device__ float sum(pVect v){ return v.x+v.y+v.z; }

// Cross Product of Two Vectors
__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}
__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }
__device__ pVect crossf3(float4 a, float4 b) { return cross(m3(a),m3(b)); }

// Cross Product of Vectors Between Coordinates
__device__ float3
 cross3(float3 a, float3 b, float3 c)
{
  float3 ab = a - b;
  float3 cb = c - b;
  return cross(ab,cb);
}
__device__ pVect cross3(pVect a, pVect b, pNorm c) { return cross3(a,b,c.v); }


__device__ pQuat quat_mult(pQuat a, pQuat b)
{
  float w = a.w * b.w - dot(a.v,b.v);
  float3 v = a.w * b.v + b.w * a.v + cross(a.v,b.v);
  pQuat q;
  q.w = w;
  q.v = v;
  return q;
  //  return cast_quat(v.x,v.y,v.z,w);
};

__device__ inline pQuat operator * (pQuat q, pQuat v)
{ return quat_mult(q,v); }


__device__ void
pMatrix_set_rotation(pMatrix3x3& m, pVect u, float theta)
{
  const float cos_theta = __cosf(theta);
  const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta );
  m.r0.x = u.x * u.x + cos_theta * ( 1 - u.x * u.x );
  m.r0.y = u.x * u.y * ( 1 - cos_theta ) - u.z * sin_theta;
  m.r0.z = u.z * u.x * ( 1 - cos_theta ) + u.y * sin_theta;
  m.r1.x = u.x * u.y * ( 1 - cos_theta ) + u.z * sin_theta;
  m.r1.y = u.y * u.y + cos_theta * ( 1 - u.y * u.y );
  m.r1.z = u.y * u.z * ( 1 - cos_theta ) - u.x * sin_theta;
  m.r2.x = u.z * u.x * ( 1 - cos_theta ) - u.y * sin_theta;
  m.r2.y = u.y * u.z * ( 1 - cos_theta ) + u.x * sin_theta;
  m.r2.z = u.z * u.z + cos_theta * ( 1 - u.z * u.z );
}

// Set matrix m to a rotation matrix based on quaternion q.
__device__ void
pMatrix_set_rotation(pMatrix3x3& m, float4 q)
{
  m.r0.x = 1.f - 2.f * q.y * q.y - 2.f * q.z * q.z;
  m.r0.y = 2.f * q.x * q.y - 2.f * q.w * q.z;
  m.r0.z = 2.f * q.x * q.z + 2.f * q.w * q.y;
  m.r1.x = 2.f * q.x * q.y + 2.f * q.w * q.z;
  m.r1.y = 1.f - 2.f * q.x * q.x - 2.f * q.z * q.z;
  m.r1.z = 2.f * q.y * q.z - 2.f * q.w * q.x;
  m.r2.x = 2.f * q.x * q.z - 2.f * q.w * q.y;
  m.r2.y = 2.f * q.y * q.z + 2.f * q.w * q.x;
  m.r2.z = 1.f - 2.f * q.x * q.x - 2.f * q.y * q.y;
}

__device__ void
pMatrix_set_rotation(pMatrix3x3& m, pQuat q)
{
  pMatrix_set_rotation(m,c4(q));
}

__device__ pMatrix3x3 mrot(pQuat q)
{
  pMatrix3x3 m;
  pMatrix_set_rotation(m,q);
  return m;
}

__device__ float3 operator *(pMatrix3x3 m, float3 coor)
{ return make_float3(dot(m.r0,coor), dot(m.r1,coor), dot(m.r2,coor)); }

