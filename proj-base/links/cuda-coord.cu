
#include <hip/hip_runtime.h>
///
/// Useful Functions and Types
///

namespace pCUDA_coord
{

typedef float4 pCoor;

class pVect {
public:
  __device__ pVect(){};
  __device__ pVect(float f):x(f),y(f),z(f){};
  __device__ pVect(float xp, float yp, float zp):x(xp),y(yp),z(zp){};
  float x, y, z;
};

// Intended for situations where a power-of-2 stride element is needed.
class __builtin_align__(16) pVect4 {
public:
  float x, y, z, w;
  __device__ pVect4(){}
  __device__ pVect4(float f):x(f),y(f),z(f),w(f){}
  __device__ pVect4(pVect v){ x=v.x; y=v.y; z=v.z; w=0; }
  __device__ operator const pVect () const { return pVect(x,y,z); }
  __device__ void operator = (pVect v) { x=v.x; y=v.y; z=v.z; }
  __device__ void operator += (pVect v) {x+=v.x; y+=v.y; z+=v.z; }
  __device__ void operator *= (float f) {x*=f; y*=f; z*=f; }
};

class pCoor3 {
  public:
  __device__ pCoor3(pCoor c) { x=c.x; y=c.y; z=c.z; }
  float x, y, z;
};


__device__ pVect operator +(pVect a,pVect b)
{ return pVect(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pCoor operator +(pCoor a,pVect b)
{ return make_float4(a.x+b.x,a.y+b.y,a.z+b.z,a.w); }
__device__ pVect operator -(pVect a,pVect b)
{ return pVect(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,pCoor b)
{ return pVect(a.x-b.x,a.y-b.y,a.z-b.z); }

__device__ pVect operator *(float s, pVect v)
{return pVect(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator *(pVect v, float s)
{return pVect(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator *(pVect u, pVect v)
{return pVect(u.x*v.x,u.y*v.y,u.z*v.z);}

__device__ pVect operator -(pVect v) { return pVect(-v.x,-v.y,-v.z); }
__device__ pVect operator -=(pVect& a, pVect b) {a = a - b; return a;}
__device__ pVect operator +=(pVect& a, pVect b) {a = a + b; return a;}
__device__ pCoor operator +=(pCoor& a, pVect b) {a = a + b; return a;}
__device__ pVect operator *=(pVect& a, float b)
{ a.x *= b;  a.y *= b; a.z *= b;  return a;}

class pNorm {
public:
  __device__ pNorm(pVect v){ set_vect(v); }
  __device__ pNorm(float x, float y, float z){ set_vect(pVect(x,y,z)); }
  __device__ pNorm(pCoor p1, pCoor p2){ set_vect(p2-p1); }
  __device__ pNorm(){};
  __device__ void set_vect(pVect v);
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

__device__ float dot(float4 a, float4 b)
{ return a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w;}
__device__ float dot(pVect a, pVect b){ return a.x*b.x + a.y*b.y + a.z*b.z;}
__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }
__device__ float dot(pNorm a, pVect b){ return dot(a.v,b); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

__device__ void
pNorm::set_vect(pVect u)
{
  mag_sq = dot(u,u);
  if ( mag_sq == 0 )
    {
      magnitude = 0;
      v.x = v.y = v.z = 0;
    }
  else
    {
      const float rsq = 1.0f/sqrtf(mag_sq);
      magnitude = 1.0f/rsq;
      v = rsq * u;
    }
}

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

class __builtin_align__(16) pQuat {
public:
  __device__ pQuat(){};
  __device__ pQuat(pNorm axis, float angle)
  {
    v = __sinf(angle/2) * axis.v;
    w = __cosf(angle/2);
  }

  pVect v;
  float w;
};

__device__ pQuat cast_quat(float4 v)
{
  pQuat q;
  q.v.x = v.x;
  q.v.y = v.y;
  q.v.z = v.z;
  q.w = v.w;
  return q;
}

__device__ pQuat cq(float4 v){ return cast_quat(v); }

__device__ pQuat quat_normalize(pQuat q)
{
  float len_sq = dot(q.v,q.v) + q.w * q.w;
  float norm_factor = 1.0f / sqrtf(len_sq);
  pQuat r;
  r.v = norm_factor * q.v;
  r.w = norm_factor * q.w;
  return r;
}

// Make float4
__device__ float4 c4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }
__device__ float4 m4(pNorm v, float w) { return m4(v.v,w); }

__device__ pVect fabs(pVect v)
{ return pVect(::fabs(v.x),::fabs(v.y),::fabs(v.z)); }
__device__ float min(pVect v){ return ::min(::min(v.x,v.y),v.z); }
__device__ float max(pVect v){ return ::max(::max(v.x,v.y),v.z); }
__device__ float sum(pVect v){ return v.x+v.y+v.z; }

// Cross Product of Two Vectors
__device__ pVect
cross(pVect a, pVect b)
{
  return pVect
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}
__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }

// Cross Product of Vectors Between Coordinates
__device__ pVect
 cross3(pCoor a, pCoor b, pCoor c)
{
  pVect ab = a - b;
  pVect cb = c - b;
  return cross(ab,cb);
}

__device__ pQuat quat_mult(pQuat a, pQuat b)
{
  float w = a.w * b.w - dot(a.v,b.v);
  pVect v = a.w * b.v + b.w * a.v + cross(a.v,b.v);
  pQuat q;
  q.w = w;
  q.v = v;
  return q;
};

__device__ inline pQuat operator * (pQuat q, pQuat v)
{ return quat_mult(q,v); }


class __builtin_align__(16) pMatrix3x3p {
public:
  __device__ pMatrix3x3p() {};
  __device__ pMatrix3x3p(pQuat q){ set_rotation(q); }
  __device__ pMatrix3x3p(pMatrix3x3p& m){r0=m.r0; r1=m.r1; r2=m.r2;}
  __device__ void set_rotation(pQuat q);
  __device__ void set_rotation(pVect u, float theta);
  pVect4 r0, r1, r2;
};

class pMatrix {
public:
  pVect4 r[4];
};

__device__ void
pMatrix3x3p::set_rotation(pVect u, float theta)
{
  const float cos_theta = __cosf(theta);
  const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta );
  r0.x = u.x * u.x + cos_theta * ( 1 - u.x * u.x );
  r0.y = u.x * u.y * ( 1 - cos_theta ) - u.z * sin_theta;
  r0.z = u.z * u.x * ( 1 - cos_theta ) + u.y * sin_theta;
  r1.x = u.x * u.y * ( 1 - cos_theta ) + u.z * sin_theta;
  r1.y = u.y * u.y + cos_theta * ( 1 - u.y * u.y );
  r1.z = u.y * u.z * ( 1 - cos_theta ) - u.x * sin_theta;
  r2.x = u.z * u.x * ( 1 - cos_theta ) - u.y * sin_theta;
  r2.y = u.y * u.z * ( 1 - cos_theta ) + u.x * sin_theta;
  r2.z = u.z * u.z + cos_theta * ( 1 - u.z * u.z );
}

// Set matrix m to a rotation matrix based on quaternion q.
__device__ void
pMatrix3x3p::set_rotation(pQuat q)
{
  r0.x = 1.f - 2.f * q.v.y * q.v.y - 2.f * q.v.z * q.v.z;
  r0.y = 2.f * q.v.x * q.v.y - 2.f * q.w * q.v.z;
  r0.z = 2.f * q.v.x * q.v.z + 2.f * q.w * q.v.y;
  r1.x = 2.f * q.v.x * q.v.y + 2.f * q.w * q.v.z;
  r1.y = 1.f - 2.f * q.v.x * q.v.x - 2.f * q.v.z * q.v.z;
  r1.z = 2.f * q.v.y * q.v.z - 2.f * q.w * q.v.x;
  r2.x = 2.f * q.v.x * q.v.z - 2.f * q.w * q.v.y;
  r2.y = 2.f * q.v.y * q.v.z + 2.f * q.w * q.v.x;
  r2.z = 1.f - 2.f * q.v.x * q.v.x - 2.f * q.v.y * q.v.y;
  r0.w = r1.w = r2.w = 0;  // Zero these to get vector writes.
}

__device__ pVect operator *(pMatrix3x3p m, pVect coor)
{ return
    pVect(dot(m.r0,coor), dot(m.r1,coor), dot(m.r2,coor)); }

} // namespace pCUDA_coord
