#include "hip/hip_runtime.h"


#include "cuda-coord.cu"

using namespace pCUDA_coord;

#include "links.cuh"
#include <gp/cuda-util-kernel.h>

__constant__ CPU_GPU_Common dc;

__global__ void time_step_gpu_links(float delta_t);
__global__ void time_step_gpu_balls(float delta_t);

__host__ hipError_t
cuda_setup(GPU_Info *gpu_info)
{
  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.

  hipError_t e1 = hipSuccess;

  gpu_info->GET_INFO(time_step_gpu_links);
  gpu_info->GET_INFO(time_step_gpu_balls);

  return e1;
}

void
data_cpu_to_gpu_common(CPU_GPU_Common *host_c)
{
  CE( hipMemcpyToSymbol(HIP_SYMBOL( dc), host_c, sizeof(*host_c) ) );
}

__host__ void
launch_time_step(float delta_t, int gsize, int blksize)
{
  time_step_gpu_links<<<gsize,blksize>>>(delta_t);
  time_step_gpu_balls<<<gsize,blksize>>>(delta_t);
}



__device__ bool
platform_collision_possible(pCoor pos)
{
  // Assuming no motion in x or z axes.
  //
  return pos.x >= dc.platform_xmin && pos.x <= dc.platform_xmax
    && pos.z >= dc.platform_zmin && pos.z <= dc.platform_zmax;
}

__device__ void
pAtomic_Add(pVect4& d, pVect s)
{
  atomicAdd(&d.x,s.x);
  atomicAdd(&d.y,s.y);
  atomicAdd(&d.z,s.z);
}

__global__ void
time_step_gpu_links(float delta_t)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

#define link(mem) dc.links.mem[li]
#define ball1(mem) dc.balls.mem[ball_1_idx]
#define ball2(mem) dc.balls.mem[ball_2_idx]


  for ( int li=tid; li<dc.n_links; li += n_threads )
    {
      if ( !link(is_simulatable) ) continue;

      // Spring Force from Neighbor Balls
      //
      const int ball_1_idx = link(ball1_idx);
      const int ball_2_idx = link(ball2_idx);

      pCoor ball1_pos = ball1(position);
      pCoor ball2_pos = ball2(position);

      // Find position and velocity of the point where the link touches
      // the surface of ball 1 ...
      //
      pVect dir1 = ball1(omatrix) * link(cb1);
      pCoor pos1 = ball1_pos + dir1;
      pVect vel1 = ball1(velocity) + cross( ball1(omega), dir1 );

      // ... and ball 2.
      //
      pVect dir2 = ball2(omatrix) * link(cb2);
      pCoor pos2 = ball2_pos + dir2;
      pVect vel2 = ball2(velocity) + cross( ball2(omega), dir2 );

      // Construct a normalized (Unit) Vector from ball to neighbor
      // based on link connection points and ball centers.
      //
      pNorm link_dir(pos1,pos2);
      pNorm c_to_c(ball1_pos,ball2_pos);

      const float link_length = link_dir.magnitude;

      // Compute the speed of ball's end of link towards neighbor's end of link.
      //
      pVect delta_v = vel2 - vel1;
      float delta_s = dot( delta_v, link_dir );

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch = link_length - link(distance_relaxed);

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0f ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? dc.opt_spring_constant : dc.opt_spring_constant * 0.7f;

      const float force_mag = spring_constant * spring_stretch;
      pVect spring_force_12 = force_mag * link_dir;

      // Apply forces affecting linear momentum.
      //

      pAtomic_Add( ball1(force),  spring_force_12 );
      pAtomic_Add( ball2(force), -spring_force_12 );

      if ( ! link(is_surface_connection) ) continue;

      pNorm dir1n(dir1);
      pNorm dir2n(dir2);

      // Apply torque.
      //
      pVect torque1 = cross(dir1n, spring_force_12);
      pVect torque2 = cross(spring_force_12,dir2n);

      pAtomic_Add( ball1(torque),  torque1 );
      pAtomic_Add( ball2(torque),  torque2 );
    }
}

__global__ void
time_step_gpu_balls(float delta_t)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

#define ball(mem) dc.balls.mem[bi]

  ///
  /// Update Position of Each Ball
  ///

  for ( int bi=tid; bi<dc.n_balls; bi += n_threads )
    {
      if ( ball(locked) )
        {
          ball(velocity) = pVect4(0);
          ball(omega) = pVect4(0);
          continue;
        }

      // Update Velocity
      //
      // This code assumes that force on ball is constant over time
      // step. This is clearly wrong when balls are moving with
      // respect to each other because the springs are changing
      // length. This inaccuracy will make the simulation unstable
      // when spring constant is large for the time step.
      //
      const float mass = ball(mass);
      pCoor ball_position = ball(position);
      pVect4 ball_velocity = ball(velocity);
      pVect4 ball_force = ball(force);
      ball(force) = pVect4(0);
      ball_force += mass * dc.gravity_accel;

      pVect delta_v = ( delta_t / mass ) * ball_force;

      if ( platform_collision_possible(ball_position) && ball_position.y < 0 )
        {
          const float spring_constant_plat =
            ball_velocity.y < 0 ? 100000 : 50000;
          const float fric_coefficient = 0.1;
          const float force_up = -ball_position.y * spring_constant_plat;
          const float delta_v_up = force_up / mass * delta_t;
          const float fric_force_mag = fric_coefficient * force_up;
          pNorm surface_v(ball_velocity.x,0,ball_velocity.z);
          const float delta_v_surf = fric_force_mag / mass * delta_t;

          if ( delta_v_surf > surface_v.magnitude )
            {
              // Ignoring other forces?
              delta_v = pVect(-ball_velocity.x,delta_v.y,-ball_velocity.z);
            }
          else
            {
              delta_v -= delta_v_surf * surface_v;
            }
          delta_v.y += delta_v_up;
        }

      ball_velocity += delta_v;

      // Air Resistance
      //
      const float fs = powf(1+dc.opt_air_resistance,-delta_t);
      ball_velocity *= fs;
      ball(velocity) = ball_velocity;

      // Update Position
      //
      // Assume that velocity is constant.
      //
      ball_position += ball_velocity * delta_t;
      ball(position) = ball_position;

      pVect4 ball_omega = ball(omega);

      ball_omega += delta_t * ball(fdt_to_do) * ball(torque);
      ball(omega) = ball_omega;
      ball(torque) = pVect4(0);

      pNorm axis(ball_omega);

      // Update Orientation
      //
      // If ball isn't spinning fast skip expensive rotation.
      //
      if ( axis.mag_sq < 0.000001f ) continue;

      pQuat orientation =
        pQuat( axis, delta_t * axis.magnitude ) * ball(orientation);
      ball(orientation) = orientation;
      ball(omatrix) = pMatrix3x3p(orientation);
    }

#undef ball

}
