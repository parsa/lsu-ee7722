#include "hip/hip_runtime.h"


#include "cuda-coord.cu"

using namespace pCUDA_coord;

#include "links.cuh"
#include <gp/cuda-util-kernel.h>
#include <gp/cuda-gpuinfo.h>
#include <assert.h>

__constant__ CPU_GPU_Common dc;

void
data_cpu_to_gpu_common(CPU_GPU_Common *host_c)
{
  CE( hipMemcpyToSymbol(HIP_SYMBOL( dc), host_c, sizeof(*host_c) ) );
}

__global__ void time_step_gpu_zero();
__global__ void time_step_gpu_links(float delta_t);
__global__ void time_step_gpu_balls(float delta_t);

__host__ void
launch_time_step(double delta_t)
{
  time_step_gpu_zero<<<13,256>>>();
  time_step_gpu_links<<<13,256>>>(delta_t);
  time_step_gpu_balls<<<13,256>>>(delta_t);
}



__device__ bool
platform_collision_possible(pCoor pos)
{
  // Assuming no motion in x or z axes.
  //
  return pos.x >= dc.platform_xmin && pos.x <= dc.platform_xmax
    && pos.z >= dc.platform_zmin && pos.z <= dc.platform_zmax;
}


__global__ void
time_step_gpu_links(float delta_t)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

#define link(mem) dc.links.mem[li]
#define ball1(mem) dc.balls.mem[ball_1_idx]
#define ball2(mem) dc.balls.mem[ball_2_idx]


  for ( int li=tid; li<dc.n_links; li += n_threads )
    {
      if ( !link(is_simulatable) ) continue;

      // Spring Force from Neighbor Balls
      //
      const int ball_1_idx = link(ball1_idx);
      const int ball_2_idx = link(ball2_idx);

      pCoor ball1_pos = ball1(position);
      pCoor ball2_pos = ball2(position);

      // Find position and velocity of the point where the link touches
      // the surface of ball 1 ...
      //
      pVect dir1 = ball1(omatrix) * link(cb1);
      pCoor pos1 = ball1_pos + dir1;
      pVect vel1 = ball1(velocity) + cross( ball1(omega), dir1 );

      // ... and ball 2.
      //
      pVect dir2 = ball2(omatrix) * link(cb2);
      pCoor pos2 = ball2_pos + dir2;
      pVect vel2 = ball2(velocity) + cross( ball2(omega), dir2 );

      // Construct a normalized (Unit) Vector from ball to neighbor
      // based on link connection points and ball centers.
      //
      pNorm link_dir(pos1,pos2);
      pNorm c_to_c(ball1_pos,ball2_pos);

      const float link_length = link_dir.magnitude;

      // Compute the speed of ball's end of link towards neighbor's end of link.
      //
      pVect delta_v = vel2 - vel1;
      float delta_s = dot( delta_v, link_dir );

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch = link_length - link(distance_relaxed);

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0 ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? dc.opt_spring_constant : dc.opt_spring_constant * 0.7;

      const float force_mag = spring_constant * spring_stretch;
      pVect spring_force_12 = force_mag * link_dir;

      // Apply forces affecting linear momentum.
      //
      //  link(spring_force_12) = spring_force_12;

#define ATOMIC(d,s) \
      atomicAdd(&d.x,s.x); atomicAdd(&d.y,s.y); atomicAdd(&d.z,s.z);

      ATOMIC(ball1(force),spring_force_12);
      ATOMIC(ball2(force),-spring_force_12);
#if 0
      ball1->force += link->spring_force_12;
      ball2->force -= link->spring_force_12;
      ball1->torque += link->torque1;
      ball2->torque -= link->torque2;
#endif

      if ( ! link(is_surface_connection) ) continue;

      pNorm dir1n(dir1);
      pNorm dir2n(dir2);

      // Apply torque.
      //
      pVect torque1 = cross(dir1n, spring_force_12);
      pVect torque2 = cross(dir2n, spring_force_12);

      ATOMIC(ball1(torque),torque1);
      ATOMIC(ball2(torque),-torque2);
    }

#if 0
  // Note: Because two links can reference the same ball this should
  // not be done in parallel.
  for ( Link *link: links )
    {
      if ( !link->is_simulatable ) continue;
      Ball* const ball1 = link->ball1;
      Ball* const ball2 = link->ball2;
      ball1->force += link->spring_force_12;
      ball2->force -= link->spring_force_12;
      ball1->torque += link->torque1;
      ball2->torque -= link->torque2;
    }
#endif

}

__global__ void
time_step_gpu_zero()
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;
#define ball(mem) dc.balls.mem[bi]

  for ( int bi=tid; bi<dc.n_balls; bi += n_threads )
    {
      ball(force) = pVect4(0);
      ball(torque) = pVect4(0);
    }
}

__global__ void
time_step_gpu_balls(float delta_t)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;


#if 0
  // Note: Because two links can reference the same ball this should
  // not be done in parallel.
  for ( Link *link: links )
    {
      if ( !link->is_simulatable ) continue;
      Ball* const ball1 = link->ball1;
      Ball* const ball2 = link->ball2;
      ball1->force += link->spring_force_12;
      ball2->force -= link->spring_force_12;
      ball1->torque += link->torque1;
      ball2->torque -= link->torque2;
    }
#endif

#define ball(mem) dc.balls.mem[bi]


  ///
  /// Update Position of Each Ball
  ///

  for ( int bi=tid; bi<dc.n_balls; bi += n_threads )
    {
      if ( ball(locked) )
        {
          ball(velocity) = pVect(0);
          ball(omega) = pVect(0);
          continue;
        }

      // Update Velocity
      //
      // This code assumes that force on ball is constant over time
      // step. This is clearly wrong when balls are moving with
      // respect to each other because the springs are changing
      // length. This inaccuracy will make the simulation unstable
      // when spring constant is large for the time step.
      //
      const float mass = ball(mass);
      pCoor ball_position = ball(position);
      pVect ball_velocity = ball(velocity);
      pVect ball_force = ball(force);
      ball(force) = pVect(0);
      ball_force += mass * dc.gravity_accel;

      pVect delta_v = ( delta_t / mass ) * ball_force;

      if ( platform_collision_possible(ball_position) && ball_position.y < 0 )
        {
          const float spring_constant_plat =
            ball_velocity.y < 0 ? 100000 : 50000;
          const float fric_coefficient = 0.1;
          const float force_up = -ball_position.y * spring_constant_plat;
          const float delta_v_up = force_up / mass * delta_t;
          const float fric_force_mag = fric_coefficient * force_up;
          pNorm surface_v(ball_velocity.x,0,ball_velocity.z);
          const float delta_v_surf = fric_force_mag / mass * delta_t;

          if ( delta_v_surf > surface_v.magnitude )
            {
              // Ignoring other forces?
              delta_v = pVect(-ball_velocity.x,delta_v.y,-ball_velocity.z);
            }
          else
            {
              delta_v -= delta_v_surf * surface_v;
            }
          delta_v.y += delta_v_up;
        }

      ball_velocity += delta_v;

      // Air Resistance
      //
      const float fs = powf(1+dc.opt_air_resistance,-delta_t);
      ball_velocity *= fs;
      ball(velocity) = ball_velocity;

      // Update Position
      //
      // Assume that velocity is constant.
      //
      ball_position += ball_velocity * delta_t;
      ball(position) = ball_position;

      pVect ball_omega = ball(omega);

      ball_omega += delta_t * ball(fdt_to_do) * ball(torque);
      ball(omega) = ball_omega;

      pNorm axis(ball_omega);

      // Update Orientation
      //
      // If ball isn't spinning fast skip expensive rotation.
      //
      //  if ( axis.mag_sq < 0.000001 ) continue;

      pQuat orientation =
        pQuat( axis, delta_t * axis.magnitude ) * ball(orientation);
      ball(orientation) = orientation;
      ball(omatrix) = pMatrix3x3(orientation);
    }

#undef ball

}
