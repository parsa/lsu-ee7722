// Simple CUDA Example, without LSU ECE helper classes.

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

struct Vertex
{
  float a[4];
};

struct App
{
  int num_threads;
  int array_size;
  Vertex *v_in;
  float *m_out;
  Vertex *d_v_in;
  float *d_m_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread = d_app.array_size / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = d_app.d_v_in[h];
      float sos = 0;

      for ( int i=0; i<4; i++ ) sos += p.a[i] * p.a[i];

      d_app.d_m_out[h] = sqrtf( sos );
    }
}

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  app.num_threads = abs(nt_raw);

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);
  const int out_array_size_bytes = app.array_size * sizeof(app.m_out[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.m_out = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes     ) );
  CE( hipMalloc( &app.d_m_out, out_array_size_bytes ) );

  printf("Preparing for %d threads %d elements.\n",
         app.num_threads, app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  const double time_start = time_fp();

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol
      ( HIP_SYMBOL(d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  const int threads_per_block = 256;
  const int blocks_per_grid =
    ( app.num_threads + threads_per_block-1 ) / threads_per_block;

  cuda_thread_start<<<blocks_per_grid,threads_per_block>>>();

  // Copy output array from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.m_out, app.d_m_out, out_array_size_bytes, hipMemcpyDeviceToHost) );

  const double data_size = app.array_size * ( sizeof(Vertex) + sizeof(float) );
  const double fp_op_count = app.array_size * 5;
  const double elapsed_time = time_fp() - time_start;

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);
}