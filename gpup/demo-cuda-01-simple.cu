/// LSU EE 4702-1 (Fall 2015), GPU Programming
//

 /// Simple CUDA Example, without LSU ECE helper classes.

/// References
//
//

#if 0
/// Background

 /// CUDA
 //
 //  NVIDIA's system for programming NVIDIA GPUs.
 //
 //  Intended for non-graphical computation, widely used for
 //  scientific computation.
 //

 /// CUDA Components.
 //
 //  - CUDA C
 //    Language used for writing code that runs on the GPU.
 //
 //  - CUDA Runtime API
 //    Library used for managing the execution of code on the GPU.
 //
 //  - CUDA Compiler Toolchain
 //    The "compiler" nvcc, and related tools.
 //
 //  - CUDA Compatible GPU
 //    Probably just NVIDIA GPUs.


 /// CUDA C
 //
 //  Language used for writing code that runs on the GPU.
 //
 //  A file can contain both CUDA C and C for the host compiler ...
 //  ... that is the case for this file.
 //
 //  In this file CUDA C is in routine: cuda_thread_start()
 //
 //  Syntactically similar to C++.
 //
 //  Major Differences
 //
 //    Executes as a hierarchy of threads.
 //
 //    Specialized address spaces.


 /// CUDA C Runtime API
 //
 //  Library calls used on CPU side to manage execution on GPU.
 //
 //  Activities Performed with API
 //
 //    o Send data from CPU to GPU.
 //    o Start execution of GPU code.
 //    o Send data from GPU to CPU.


 /// CUDA Address Spaces
 //

 /// Global
 //
 //  Works like "regular" memory on CPU, but it's usually separated.

 /// Constant
 //
 //  Limited amount of storage, read-only on GPU.


#endif


#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

struct App
{
  int num_threads;
  int array_size;
  float *v_in;
  float *m_out;
  float *d_v_in;
  float *d_m_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  // Warning: The order in which d_v_in is accessed is inefficient.
  //          See demo-cuda-02-basics for a better ordering.
  //
  const int elt_per_thread = d_app.array_size / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    d_app.d_m_out[h] = d_app.d_v_in[h] + 1;

}

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  app.num_threads = abs(nt_raw);

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);
  const int out_array_size_bytes = app.array_size * sizeof(app.m_out[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new float[app.array_size];
  app.m_out = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes     ) );
  CE( hipMalloc( &app.d_m_out, out_array_size_bytes ) );

  printf("Preparing for %d threads %d elements.\n",
         app.num_threads, app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ ) app.v_in[i] = drand48();

  const double time_start = time_fp();

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol
      ( HIP_SYMBOL(d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  const int threads_per_block = 256;
  const int blocks_per_grid =
    ( app.num_threads + threads_per_block-1 ) / threads_per_block;

  /// KERNEL LAUNCH
  cuda_thread_start<<< blocks_per_grid, threads_per_block >>>();

  // Copy output array from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.m_out, app.d_m_out, out_array_size_bytes, hipMemcpyDeviceToHost) );

  const double data_size = 
    app.array_size * ( sizeof(app.v_in[0]) + sizeof(app.m_out[0]) );
  const double fp_op_count = app.array_size * 5;
  const double elapsed_time = time_fp() - time_start;

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);
}
