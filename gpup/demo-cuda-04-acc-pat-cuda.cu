#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2016), GPU Programming
//

#if 0
 /// CUDA Demo 04
 //
 //  This code demonstrates different methods of all-to-all access in
 //  CUDA, such as the accesses to array in the code below:
 //
     for ( int x=0; x<n; x++ )
       for ( int y=0; y<n; y++ )
         sum += array[ x ] * array[ y ];
 //
 ///  See routines time_step_intersect_1 and time_step_intersect_2 in
 //  demo-cuda-04-acc-pat-cuda.cu. Instead of array, this code
 //  accesses helix_position.
#endif

#include "cuda-coord.cu"
#include "demo-cuda-04-acc-pat.cuh"
#include <gp/cuda-util-kernel.h>

// Physical State Variables
//
__constant__ float4 *helix_position;
__constant__ float3 *helix_velocity;     // Note: float4 would be faster.
__constant__ float4 *helix_orientation;
__constant__ float3 *helix_omega;        // Note: float4 would be faster.

__device__ Timing_Data timing_data;   // Measure execution time of intersect.
__constant__ Helix_Info hi;  // Scalar Constants

__global__ void
time_step_intersect_1()
{
  // Find intersections of one helix segment with some other
  // segments. Each block handles several "a" segments, the threads in
  // the block check for intersection with other segments, called "b"
  // segments.

  __shared__ clock_t time_start;
  if ( !threadIdx.x ) time_start = clock64();


  // Note: The size of the helix_position array is hi.phys_helix_segments.

  // Compute how many "a" elements will be handled by each block.
  //
  const int a_per_block = hi.phys_helix_segments / gridDim.x;

  // Compute how many threads handle each "a" element.
  //
  const int thd_per_a = blockDim.x / a_per_block;

  // Compute the smallest "a" element index that this block will handle.
  //
  const int a_idx_block = blockIdx.x * a_per_block;

  /// Assignment of "a" and "b" Values to Threads
  //
  //  The table below is an example of how this routine
  //  assigns "a" and "b" elements to threads.  The table
  //  is based upon the following values:
  //
  //    blockDim = 8,     blockIdx = 4,   hi.phys_helix_segments = 1024
  //    a_per_block = 4,  thd_per_a = 2,  a_idx_block = 16
  //
  // tIx     al   a      b ---> 
  //   0     0    16     0  2  4 ... 1022
  //   1     1    17     0  2  4 ... 1022
  //   2     2    18     0  2  4 ... 1022
  //   3     3    19     0  2  4 ... 1022
  //   4     0    16     1  3  5 ... 1023
  //   5     1    17     1  3  5 ... 1023
  //   6     2    18     1  3  5 ... 1023
  //   7     3    19     1  3  5 ... 1023
  //   |     |     |     |
  //   |     |     |     |
  //   |     |     |     |--------> b_idx_start
  //   |     |     |--------------> a_idx
  //   |     |--------------------> a_local_idx
  //   |--------------------------> threadIdx.x

  // Compute a_idx and b_idx_start to realize ordering above.
  //
  const int a_local_idx = threadIdx.x % a_per_block;
  const int a_idx = a_idx_block + a_local_idx;
  const int b_idx_start = threadIdx.x / a_per_block;


  const float3 a_position = m3(helix_position[a_idx]);
  const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
  const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;

  // Declare dynamically allocated shared memory. Will be split
  // between array for forces, force, and position cache, pos_cache.
  //
  extern __shared__ float3 shared[];

  pVect* const force = shared;
  if ( threadIdx.x < a_per_block ) force[threadIdx.x] = mv(0,0,0);

  __syncthreads();


  float3* const pos_cache = &shared[blockDim.x];

  for ( int j=b_idx_start; j<hi.phys_helix_segments; j += thd_per_a )
    {
      if ( hi.opt_use_shared )
        {
          __syncthreads();
          if ( threadIdx.x < thd_per_a )
            pos_cache[threadIdx.x] =
              m3(helix_position[ j - b_idx_start + threadIdx.x ] );
          __syncthreads();
        }

      float3 b_position = 
        hi.opt_use_shared ? pos_cache[b_idx_start] : m3(helix_position[j]);

      pVect ab = mv(a_position,b_position);

      // Skip if segment is too close.
      if ( abs(a_idx-j) < min_idx_dist ) continue;

      // Skip if no chance of intersection.
      if ( mag_sq(ab) >= four_wire_radius_sq ) continue;

      // Compute intersection force based on bounding sphere, an
      // admittedly crude approximation.
      //
      pNorm dist = mn(ab);
      const float pen = 2 * hi.wire_radius - dist.magnitude;
      float3 f = pen * hi.opt_spring_constant * dist;

      // Add force to shared variable. This is time consuming
      // (especially in CC 3.x and older GPUs) but done
      // infrequently. (A segment can normally only intersect a a few
      // other segments.)
      //
      atomicAdd(&force[a_local_idx].x,f.x);
      atomicAdd(&force[a_local_idx].y,f.y);
      atomicAdd(&force[a_local_idx].z,f.z);
      //
      // Optimization Note: Could acquire a lock and then update
      // all three components.
    }

  // Wait for all threads to finish.
  __syncthreads();

  // Leave it to thread 0 to update velocity.
  if ( threadIdx.x >= a_per_block ) return;

  // Update velocity and write it.
  //
  float3 velocity = helix_velocity[a_idx];
  velocity -= hi.delta_t_mass_inv * force[a_local_idx];
  if ( hi.opt_end_fixed && a_idx + 1 == hi.phys_helix_segments )
    velocity = mv(0,0,0);
  helix_velocity[a_idx] = velocity;

  if ( !threadIdx.x && !blockIdx.x )
    {
      timing_data.inter_time += clock64() - time_start;
      timing_data.inter_count++;
    }
}

__global__ void
time_step_intersect_2()
{
  // Find intersections of one helix segment with some other
  // segments. Each block handles several "a" segments, the threads in the
  // block check for intersection with other segments, called "b"
  // segments.

  __shared__ clock_t time_start;
  if ( !threadIdx.x ) time_start = clock64();

  // Note: The size of the helix_position array is hi.phys_helix_segments.

  // Compute how many "a" elements will be handled by each block.
  //
  const int a_per_block = hi.phys_helix_segments / gridDim.x;

  // Compute how many threads handle each "a" element.
  //
  const int thd_per_a = blockDim.x / a_per_block;

  // Compute the smallest "a" element index that this block will handle.
  //
  const int a_idx_block = blockIdx.x * a_per_block;

  /// Assignment of "a" and "b" Values to Threads
  //
  //  The table below is an example of how this routine
  //  assigns "a" and "b" elements to threads.  The table
  //  is based upon the following values:
  //
  //    blockDim = 8,     blockIdx = 4,   hi.phys_helix_segments = 1024
  //    a_per_block = 4,  thd_per_a = 2,  a_idx_block = 16
  //
  // tIx     al   a      b ---> 
  //   0     0    16     0  2  4 ...
  //   1     0    16     1  3  5
  //   2     1    17     0  2  4
  //   3     1    17     1  3  5
  //   4     2    18     0  2  4
  //   5     2    18     1  3  5
  //   6     3    19     0  2  4
  //   7     3    19     1  3  5 
  //   |     |     |     |
  //   |     |     |     |
  //   |     |     |     |--------> b_idx_start
  //   |     |     |--------------> a_idx
  //   |     |--------------------> a_local_idx
  //   |--------------------------> threadIdx.x

  // Compute a_idx and b_idx_start to realize ordering above.
  //
  const int a_local_idx = threadIdx.x / thd_per_a;
  const int a_idx = a_idx_block + a_local_idx;
  const int b_idx_start = threadIdx.x % thd_per_a;

  const float3 a_position = m3(helix_position[a_idx]);
  const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
  const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;

  // Declare dynamically allocated shared memory. Will be split
  // between array for forces, force, and position cache, pos_cache.
  //
  extern __shared__ float3 shared[];

  pVect* const force = shared;
  if ( threadIdx.x < a_per_block ) force[threadIdx.x] = mv(0,0,0);

  // Wait for thread 0 to initialize force.
  __syncthreads();


  float3* const pos_cache = &shared[blockDim.x];

  for ( int j=b_idx_start; j<hi.phys_helix_segments; j += thd_per_a )
    {
      if ( hi.opt_use_shared )
        {
          __syncthreads();
          if ( threadIdx.x < thd_per_a )
            pos_cache[threadIdx.x] = m3(helix_position[j]);
          __syncthreads();
        }
      float3 b_position = 
        hi.opt_use_shared ? pos_cache[b_idx_start] : m3(helix_position[j]);

      pVect ab = mv(a_position,b_position);

      // Skip if segment is too close.
      if ( abs(a_idx-j) < min_idx_dist ) continue;

      // Skip if no chance of intersection.
      if ( mag_sq(ab) >= four_wire_radius_sq ) continue;

      // Compute intersection force based on bounding sphere, an
      // admittedly crude approximation.
      //
      pNorm dist = mn(ab);
      const float pen = 2 * hi.wire_radius - dist.magnitude;
      float3 f = pen * hi.opt_spring_constant * dist;

      // Add force to shared variable. This is time consuming but
      // done infrequently. (A segment can normally only intersect a
      // a few other segments.)
      //
      atomicAdd(&force[a_local_idx].x,f.x);
      atomicAdd(&force[a_local_idx].y,f.y);
      atomicAdd(&force[a_local_idx].z,f.z);
      //
      // Optimization Note: Could acquire a lock and then update
      // all three components.
    }

  // Wait for all threads to finish.
  __syncthreads();

  // Leave it to thread 0 to update velocity.
  if ( threadIdx.x >= a_per_block ) return;

  {
    // Re-compute a_idx so that first a_per_block threads can write
    // velocities.

    const int a_local_idx = threadIdx.x;
    const int a_idx = a_idx_block + a_local_idx;

    // Update velocity and write it.
    //
    float3 velocity = helix_velocity[a_idx];
    velocity -= hi.delta_t_mass_inv * force[a_local_idx];
    if ( hi.opt_end_fixed && a_idx + 1 == hi.phys_helix_segments )
      velocity = mv(0,0,0);
    helix_velocity[a_idx] = velocity;

    if ( !threadIdx.x && !blockIdx.x )
      {
        timing_data.inter_time += clock64() - time_start;
        timing_data.inter_count++;
      }
  }
}


__global__ void time_step();
__global__ void time_step_intersect_1();
__global__ void time_step_intersect_2();
__global__ void time_step_update_pos();


__host__ hipError_t
cuda_setup(GPU_Info *gpu_info)
{
  // Pass the device address to host code. (See gp/cuda-util-kernel.h ).
  CU_SYM(helix_position);
  CU_SYM(helix_velocity);
  CU_SYM(helix_orientation);
  CU_SYM(helix_omega);
  CU_SYM(hi);
  CU_SYM(timing_data);

  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.

  hipError_t e1 = hipSuccess;

  /// WARNING: Code in render expects time_step_intersect_1 and
  /// time_step_intersect_2 to be 2nd and 3rd (at index 1 and 2) of
  /// gpu_info::ki.
  gpu_info->GET_INFO(time_step);
  gpu_info->GET_INFO(time_step_intersect_1);
  gpu_info->GET_INFO(time_step_intersect_2);
  gpu_info->GET_INFO(time_step_update_pos);

  return e1;
}

__host__ void time_step_launch(int grid_size, int block_size)
{
  time_step<<<grid_size,block_size>>>();
}

__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude);


__global__ void
time_step()
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Use tid for helix segment number.

  if ( tid + 1 > hi.phys_helix_segments ) return;

  // The position of segment 0 is fixed, so don't evolve it.
  if ( tid == 0 ) return;

  pVect vZero = mv(0,0,0);
  pVect gravity_force = hi.helix_seg_mass_inv * hi.gravity_accel;

  pQuat c_orientation = cq(helix_orientation[tid]);
  float3 c_position = m3(helix_position[tid]);

  pMatrix3x3 c_rot;
  // Initialize c_rot to a rotation matrix based on quaternion c_orientation.
  pMatrix_set_rotation(c_rot,c_orientation);

  float3 c_u = c_rot * mv(0,0,1);  // mv: Make Vector.
  float3 c_v = c_rot * mv(0,1,0);
  float3 c_ctr_to_right_dir = c_rot * mv(1,0,0);
  pVect c_ctr_to_right = hi.helix_seg_hlength * c_ctr_to_right_dir;
  float3 c_pos_right = c_position + c_ctr_to_right;
  float3 c_pos_left = c_position - c_ctr_to_right;

  float3 force = hi.opt_gravity ? gravity_force : vZero;
  float3 torque = vZero;

  const int pieces = 3;
  const float delta_theta = 2 * M_PI / pieces;

  /// Compute forces due to right neighbor.
  //
  if ( tid + 1 < hi.phys_helix_segments )
    {
      pQuat r_orientation = cq(helix_orientation[tid+1]);
      float3 r_position = m3(helix_position[tid+1]);
      pMatrix3x3 r_rot;
      pMatrix_set_rotation(r_rot,r_orientation);
      float3 r_u = r_rot * mv(0,0,1);
      float3 r_v = r_rot * mv(0,1,0);
      float3 r_ctr_to_right_dir = r_rot * mv(1,0,0);
      pVect r_ctr_to_right = hi.helix_seg_hlength * r_ctr_to_right_dir;
      float3 r_pos_left = r_position - r_ctr_to_right;

      pQuat cn_rot_q = c_orientation * hi.helix_rn_trans;
      pMatrix3x3 cn_rot;
      pMatrix_set_rotation(cn_rot,cn_rot_q);
      pVect n_ru = cn_rot * mv(0,0,1);
      pVect n_rv = cn_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_right + cosf(theta) * n_ru + sinf(theta) * n_rv;
          pCoor r_pt = r_pos_left + cosf(theta) * r_u + sinf(theta) * r_v;
          pNorm dist = mn(c_pt,r_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// Compute forces due to left neighbor.
  //
  if ( tid > 0 )
    {
      pQuat l_orientation = cq(helix_orientation[tid-1]);
      float3 l_position = m3(helix_position[tid-1]);
      pMatrix3x3 l_rot;
      pMatrix_set_rotation(l_rot,l_orientation);
      float3 l_u = l_rot * mv(0,0,1);
      float3 l_v = l_rot * mv(0,1,0);
      float3 l_ctr_to_right_dir = l_rot * mv(1,0,0);
      pVect l_ctr_to_right = hi.helix_seg_hlength * l_ctr_to_right_dir;
      float3 l_pos_right = l_position + l_ctr_to_right;

      pQuat ln_rot_q = l_orientation * hi.helix_rn_trans;
      pMatrix3x3 ln_rot;
      pMatrix_set_rotation(ln_rot,ln_rot_q);
      pVect n_cu = ln_rot * mv(0,0,1);
      pVect n_cv = ln_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_left + cosf(theta) * c_u + sinf(theta) * c_v;
          pCoor l_pt = l_pos_right + cosf(theta) * n_cu + sinf(theta) * n_cv;
          pNorm dist = mn(c_pt,l_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  float3 velocity = helix_velocity[tid];
  velocity *= 0.99999f;
  float3 omega = helix_omega[tid];
  omega *= 0.99999f;
  velocity += hi.delta_t_mass_inv * force;
  const float torque_axial_mag = dot( torque, c_ctr_to_right_dir );
  pVect torque_axial = torque_axial_mag * c_ctr_to_right_dir;
  pVect do_axial = hi.delta_t_ma_axis * torque_axial;
  pVect torque_other = torque - torque_axial;
  pVect do_other = hi.delta_t_ma_perp_axis * torque_other;
  omega += do_axial + do_other;

  // Update velocity and omega. Don't update position or orientation
  // because we don't want threads in this kernel to accidentally read
  // the updated values.

  helix_omega[tid] = omega;
  helix_velocity[tid] = velocity;
}


__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude)
{
  // Update force and torque of segment for a force acting on FORCE_POS
  // pointing in direction DIR of magnitude MAGNITUDE.
  //
  force += magnitude * dir;
  pVect arm = mv(position,force_pos);
  pVect axis = cross( arm, dir );
  pVect amt = magnitude * axis;
  torque += amt;
}



__host__ void
time_step_intersect_launch
(int grid_size, int block_size, int version, int dynamic_sm_amt)
{
  switch ( version ) {
    case 1: time_step_intersect_1<<<grid_size,block_size,dynamic_sm_amt>>>();
    case 2: time_step_intersect_2<<<grid_size,block_size,dynamic_sm_amt>>>();
  }
}

__global__ void
time_step_update_pos()
{
  // Update position and orientation of spring segments.

  // Use tid for helix segment number.
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Skip out-of-range segments.
  if ( tid >= hi.phys_helix_segments ) return;
  if ( tid == 0 ) return;

  // Update Orientation
  //
  pQuat orientation = cq(helix_orientation[tid]);
  float3 omega = helix_omega[tid];
  pNorm axis = mn(omega);
  helix_orientation[tid] =
    c4( quat_normalize
    ( quat_mult ( mq( axis, hi.delta_t * axis.magnitude ), orientation)));

  // Return if at last segment and it is fixed. Note that even
  // if the segment's position is fixed, it can still rotate.
  //
  if ( hi.opt_end_fixed && tid + 1 == hi.phys_helix_segments ) return;

  // Update Velocity
  //
  float3 position = m3(helix_position[tid]);
  float3 velocity = helix_velocity[tid];
  helix_position[tid] = m4(position + hi.delta_t * velocity,1);
}

__host__ void
time_step_update_pos_launch
(int grid_size, int block_size)
{
  time_step_update_pos<<<grid_size,block_size>>>();
}
