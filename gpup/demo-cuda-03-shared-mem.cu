#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2016), GPU Programming
//

 /// Shared memory CUDA Example, without LSU ECE helper classes.

/// References
//
//

#if 0
/// Background

 /// Shared Address Space and Shared Memory
 //
 //  References
 //    General description: CUDA C Programming Guide Section 3.23 (v8)
 //    Amount of SM: CUDA C Programming Guide Section Table 12 (Appendix G)
 //
 //
 //  :Def: Shared Address Space
 //  An address space provided by CUDA (through CC 5.x) in which:
 //
 //   - Each block has its own address space.
 //
 //   - Address space is shared by all threads in a block.
 //
 //   - Locations can be read and written.
 //
 //   - Size of space is 48 kiB in CC 2.X to CC 5.x (so far).
 //
 //   - Shared address space uses shared memory.
 //
 //
 //  :Def: Shared Memory
 //  Hardware used to implement the shared address space.
 //
 //   - Shared memory is part of SM, so no communication limits.
 //
 //   - Amount of shared memory per SM (NOT per block) varies:
 //      48 kiB  CC 2.0 - CC 3.5
 //     112 kiB  CC 3.7
 //      64 kiB  CC 5.0, 5.3, 6.0, 6.2
 //      96 kiB  CC 5.2, 6.1
 //
 //   - Low latency (fast). As low as 12 cycles.
 //
 //   - High throughput.
 //
 //   - Banked organization.  Throughput depends on access patterns.
 //
 //
 /// Declaration and Use of Shared Memory
 //
 //   - Declare variables using __shared__ qualifier.
 //
 //   - Declaration can be at procedure or global scope.
 //
 //   - Any type can be shared, including arrays.
 //
 //   - Pointers to shared variables can be taken.
 //
 //  :Example: Declaration examples.

__shared__ int amount;
__shared__ float4 forces[12];


 /// Shared Memory Uses
 //
 //  Communication between threads.
 //    For example, to compute a block-wide sum.
 //
 //  Caching of global memory. 
 //    (Copying to a place where it can be accessed quickly.)


  /// Atomic Operations
  //
  //  :Def: Atomic Operation
  //        An operation that appears to be either ..
  //        .. completely finished or ..
  //        .. not yet started.
  //        An atomic operation NEVER appears to be partially done.

  //  :Example:
  //
  //  A the following "+=" operation is NOT atomic.
  //
  __shared__ int sum;

  if ( threadIdx.x == 0 )  sum = 0;
  if ( threadIdx.x == 40 ) sum += 40;
  if ( threadIdx.x == 70 ) sum += 70;
  //
  //  An we need an atomic operation to perform the additions above.

  /// CUDA C Atomic Operations
  //
  //  Reference: CUDA C Programming Guide B.12
  //
  /// oldval = atomicAdd( valAddress, amount )
  //  Atomically add amount to *valAddress, return old value.


 /// Other Stuff
 //
 //  Need to coordinate readers and writers.


 __syncthreads();

 atomicAdd(POINTER, AMT);

#endif


#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>

inline double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}


#if 0

__global__ void
cuda_thread_super_simple(int *output_data, int *input_data)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int my_element = input_data[tid];

  /// Reasonable use of shared memory.
  //
  //  Make thread 12's element available to all threads in the block.

  __shared__ int a;

  if ( threadIdx.x == 12 ) a = my_element;

  __syncthreads();

  output_data[tid] = my_element + a;


  /// Bad use of shared memory.
  //
  //  Everyone writes trouble.

  __shared__ int trouble;

  trouble = my_element;

  __syncthreads();

  // All threads read the same value, whoever got there last.

  output_data[tid] = my_element + trouble;


  /// Reasonable use of shared memory.
  //
  //  Share your array element with our neighbor.

  __shared__ int our_data[1024];

  our_data[threadIdx.x] = my_element;

  __syncthreads();

  output_data[tid] = my_element + our_data[threadIdx ^ 1];


}
#endif


struct Vector
{
  float a[4];
};

struct App
{
  int num_threads;
  int array_size;
  Vector *v_in;
  float *m_out;
  Vector *d_v_in;
  float *d_m_out;
  float *block_mag_sum;
  float *d_block_mag_sum;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

///
/// GPU Code (Kernel)
///

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Shared array, one element for each member of block (up to max bl size).
  __shared__ float our_mag_sums[1024];

  float my_mag_sum = 0;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      Vector p = d_app.d_v_in[h];
      float sos = 0;

      for ( int i=0; i<4; i++ ) sos += p.a[i] * p.a[i];

      const float mag = sqrtf( sos );

      // Write magnitude to global memory.
      d_app.d_m_out[h] = mag;

      // Compute this thread's magnitude sum.
      my_mag_sum += mag;
    }

  // Save this thread's magnitude sum in shared memory.
  //
  our_mag_sums[threadIdx.x] = my_mag_sum;

  // Wait for all threads to do this.
  //
  __syncthreads();

  // All but the first warp are finished.
  //
  if ( threadIdx.x >= 32 ) return;

  // Threads in first warp (first 32) each compute sum for their lane.
  //
  float lane_mag_sum = 0;
  for ( int i=threadIdx.x; i<blockDim.x; i+=32 )
    lane_mag_sum += our_mag_sums[i];

  // Save the sum for this lane in shared memory.
  //
  our_mag_sums[threadIdx.x] = lane_mag_sum;

  // Have just thread 0 finish up.
  //
  if ( threadIdx.x != 0 ) return;

  // Compute the sum of the last 32 elements.
  //
  float block_mag_sum = 0;
  for ( int i=0; i<32; i++ ) block_mag_sum += our_mag_sums[i];

  // Save this sum to global memory.  CPU will sum of blocks.
  //
  d_app.d_block_mag_sum[blockIdx.x] = block_mag_sum;
}


///
/// Collect Information About GPU and Code
///

void
cuda_init()
{
  GPU_Info gpu_info;

  gpu_info_print();

  // Choose GPU 0 because we don't have time to provide a way to let
  // the user choose.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  gpu_info.get_gpu_info(dev);

  gpu_info.GET_INFO(cuda_thread_start);

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");

  for ( int i=0; i<gpu_info.num_kernels; i++ )
    {
      printf("For %s:\n", gpu_info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             gpu_info.ki[i].cfa.sharedSizeBytes,
             gpu_info.ki[i].cfa.constSizeBytes,
             gpu_info.ki[i].cfa.localSizeBytes,
             gpu_info.ki[i].cfa.numRegs,
             gpu_info.ki[i].cfa.maxThreadsPerBlock);
    }

  printf("\n");
}

///
/// Main Routine
///

int
main(int argc, char **argv)
{
  const int threads_per_block = argc < 2 ? 1 : atoi(argv[1]);
  const int blocks_per_grid = argc < 3 ? 1 : atoi(argv[2]);
  app.num_threads = threads_per_block * blocks_per_grid;

  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);
  const int out_array_size_bytes = app.array_size * sizeof(app.m_out[0]);
  const int block_mag_sum_bytes =
    blocks_per_grid * sizeof(app.block_mag_sum[0]);

  if ( argc < 2 ) cuda_init();

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vector[app.array_size];
  app.m_out = new float[app.array_size];
  app.block_mag_sum = new float[blocks_per_grid];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes     ) );
  CE( hipMalloc( &app.d_m_out, out_array_size_bytes ) );
  CE( hipMalloc( &app.d_block_mag_sum, block_mag_sum_bytes ) );

  printf("Preparing for %d threads %d elements using %d blocks of size %d.\n",
         app.num_threads, app.array_size,
         blocks_per_grid, threads_per_block);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  const double time_start = time_fp();

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  /// Launch Kernel
  cuda_thread_start<<<blocks_per_grid,threads_per_block>>>();

  // Copy output arrays from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.m_out, app.d_m_out, out_array_size_bytes, hipMemcpyDeviceToHost) );
  CE( hipMemcpy
      ( app.block_mag_sum, app.d_block_mag_sum, block_mag_sum_bytes,
        hipMemcpyDeviceToHost) );

  float mag_sum = 0;
  for ( int i=0; i<blocks_per_grid; i++ )
    mag_sum += app.block_mag_sum[i];

  const double data_size = app.array_size * ( sizeof(Vector) + sizeof(float) );
  const double fp_op_count = app.array_size * 5;
  const double elapsed_time = time_fp() - time_start;

  float mag_sum_check = 0;
  for ( int i=0; i<app.array_size; i++ )
    mag_sum_check += app.m_out[i];

  const float mag_avg_check = mag_sum_check / app.array_size;
  const float mag_avg = mag_sum / app.array_size;

  if ( fabs(mag_avg_check-mag_avg) > 0.00001 )
    printf("** Averages don't check %.7f != %.7f (cpu)\n",
           mag_avg, mag_avg_check);

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);

  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);

}
