#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2015), GPU Programming
//

 /// Shared memory CUDA Example, without LSU ECE helper classes.

/// References
//
//

#if 0
/// Background

 /// Shared Memory
 //
 //  An address space that's shared amongst threads in a block.
 //    Members of a block can load values that other block members wrote.
 //
 //  The maximum amount of shared memory is 48 kiB per block in Kepler
 //  and Maxwell devices.
 //
 //  A variable is assigned to shared memory if it is declared using
 //  the __shared__ qualifier.
 //
 //  :Example: Declaration examples.

__shared__ int amount;
__shared__ float4 forces[12];

 /// Shared Memory Uses
 //
 //  Communication between threads.
 //  Caching of global memory. 
 //    (Copying to a place where it can be accessed quickly.)

 /// Other Stuff
 //
 //  Need to coordinate readers and writers.


 __syncthreads();

 atomicAdd(POINTER, AMT);

#endif


#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

struct Vector
{
  float a[4];
};

struct App
{
  int num_threads;
  int array_size;
  Vector *v_in;
  float *m_out;
  Vector *d_v_in;
  float *d_m_out;
  float *block_mag_sum;
  float *d_block_mag_sum;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

///
/// GPU Code (Kernel)
///


__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Shared array, one element for each member of block (up to max bl size).
  __shared__ float our_mag_sums[1024];

  float my_mag_sum = 0;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      Vector p = d_app.d_v_in[h];
      float sos = 0;

      for ( int i=0; i<4; i++ ) sos += p.a[i] * p.a[i];

      const float mag = sqrtf( sos );

      // Write magnitude to global memory.
      d_app.d_m_out[h] = mag;

      // Compute this thread's magnitude sum.
      my_mag_sum += mag;
    }

  // Save this thread's magnitude sum in shared memory.
  //
  our_mag_sums[threadIdx.x] = my_mag_sum;

  // Wait for all threads to do this.
  //
  __syncthreads();

  // All but the first warp are finished.
  //
  if ( threadIdx.x >= 32 ) return;

  // Threads in first warp (first 32) each compute sum for their lane.
  //
  float lane_mag_sum = 0;
  for ( int i=threadIdx.x; i<blockDim.x; i+=32 )
    lane_mag_sum += our_mag_sums[i];

  // Save the sum for this lane in shared memory.
  //
  our_mag_sums[threadIdx.x] = lane_mag_sum;

  // Have just thread 0 finish up.
  //
  if ( threadIdx.x != 0 ) return;

  // Compute the sum of the last 32 elements.
  //
  float block_mag_sum = 0;
  for ( int i=0; i<32; i++ ) block_mag_sum += our_mag_sums[i];

  // Save this sum to global memory.  CPU will sum of blocks.
  //
  d_app.d_block_mag_sum[blockIdx.x] = block_mag_sum;
}


///
/// Collect Information About GPU and Code
///

// Info about a specific kernel.
//
struct Kernel_Info {
  void (*func_ptr)();           // Pointer to kernel function.
  const char *name;             // ASCII version of kernel name.
  hipFuncAttributes cfa;       // Kernel attributes reported by CUDA.
};

// Info about GPU and each kernel.
//
struct GPU_Info {
  double bw_Bps;
  static const int num_kernels_max = 4;
  int num_kernels;
  Kernel_Info ki[num_kernels_max];
};

GPU_Info gpu_info;

void
cuda_init()
{
  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  hipGetDeviceCount(&device_count); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  hipDeviceProp_t cuda_prop;  // Properties of cuda device (GPU, cuda version).

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      CE(hipGetDeviceProperties(&cuda_prop,dev));
      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      const int cc_per_mp =
        cuda_prop.major == 1 ? 8 :
        cuda_prop.major == 2 ? ( cuda_prop.minor == 0 ? 32 : 48 ) :
        cuda_prop.major == 3 ? 192 : 0;

      const double chip_bw_Bps = gpu_info.bw_Bps =
        2 * cuda_prop.memoryClockRate * 1000.0
        * ( cuda_prop.memoryBusWidth >> 3 );
      const double chip_sp_flops =
        1000.0 * cc_per_mp * cuda_prop.clockRate
        * cuda_prop.multiProcessorCount;

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize >> 10,
         chip_bw_Bps * 1e-9,
         chip_sp_flops * 1e-9,
         4 * chip_sp_flops / chip_bw_Bps);

    }

  // Choose GPU 0 because we don't have time to provide a way to let
  // the user choose.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);

  gpu_info.num_kernels = 0;

#define GET_INFO(proc_name) {                                                 \
  const int idx = gpu_info.num_kernels++;                                     \
  if ( idx < gpu_info.num_kernels_max ) {                                     \
    gpu_info.ki[idx].name = #proc_name;                                       \
    gpu_info.ki[idx].func_ptr = (void(*)())proc_name;                         \
  }}

  GET_INFO(cuda_thread_start);

#undef GET_INFO

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");

  for ( int i=0; i<gpu_info.num_kernels; i++ )
    {
      printf("For %s:\n", gpu_info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             gpu_info.ki[i].cfa.sharedSizeBytes,
             gpu_info.ki[i].cfa.constSizeBytes,
             gpu_info.ki[i].cfa.localSizeBytes,
             gpu_info.ki[i].cfa.numRegs,
             gpu_info.ki[i].cfa.maxThreadsPerBlock);
    }

  printf("\n");
}

///
/// Main Routine
///

int
main(int argc, char **argv)
{
  const int threads_per_block = argc < 2 ? 1 : atoi(argv[1]);
  const int blocks_per_grid = argc < 3 ? 1 : atoi(argv[2]);
  app.num_threads = threads_per_block * blocks_per_grid;

  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);
  const int out_array_size_bytes = app.array_size * sizeof(app.m_out[0]);
  const int block_mag_sum_bytes =
    blocks_per_grid * sizeof(app.block_mag_sum[0]);

  if ( argc < 2 ) cuda_init();

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vector[app.array_size];
  app.m_out = new float[app.array_size];
  app.block_mag_sum = new float[blocks_per_grid];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes     ) );
  CE( hipMalloc( &app.d_m_out, out_array_size_bytes ) );
  CE( hipMalloc( &app.d_block_mag_sum, block_mag_sum_bytes ) );

  printf("Preparing for %d threads %d elements using %d blocks of size %d.\n",
         app.num_threads, app.array_size,
         blocks_per_grid, threads_per_block);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  const double time_start = time_fp();

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  /// Launch Kernel
  cuda_thread_start<<<blocks_per_grid,threads_per_block>>>();

  // Copy output arrays from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.m_out, app.d_m_out, out_array_size_bytes, hipMemcpyDeviceToHost) );
  CE( hipMemcpy
      ( app.block_mag_sum, app.d_block_mag_sum, block_mag_sum_bytes,
        hipMemcpyDeviceToHost) );

  float mag_sum = 0;
  for ( int i=0; i<blocks_per_grid; i++ )
    mag_sum += app.block_mag_sum[i];

  const double data_size = app.array_size * ( sizeof(Vector) + sizeof(float) );
  const double fp_op_count = app.array_size * 5;
  const double elapsed_time = time_fp() - time_start;

  float mag_sum_check = 0;
  for ( int i=0; i<app.array_size; i++ )
    mag_sum_check += app.m_out[i];

  const float mag_avg_check = mag_sum_check / app.array_size;
  const float mag_avg = mag_sum / app.array_size;

  if ( fabs(mag_avg_check-mag_avg) > 0.00001 )
    printf("** Averages don't check %.7f != %.7f (cpu)\n",
           mag_avg, mag_avg_check);

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);

  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);

}
