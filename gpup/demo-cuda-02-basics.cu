#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2016), GPU Programming
//

 /// Simple CUDA Example, without LSU ECE helper classes.

/// References
//
//

#if 0
/// Background

//  The following background describes the view of NVIDIA GPUs
//  provided by the CUDA system.  "View" means how the hardware
//  appears to the application programmer.  For CUDA, this should
//  be very close to the actual hardware.

 /// Compute Capability (CC)
//
//   An NVIDIA numbering system that identifies the approximate
//   capabilities of the hardware.
//
//   Compute Capabilities
//
//   -- CC 1.0, 1.1, 1.2, 1.3
//      Tesla. (Not to be confused with the Tesla board.)
//      Now considered obsolete.
//
//   -- CC 2.0, 2.1
//      Fermi
//
//   -- CC 3.0, 3.5
//      Kepler
//      Product cycle ending.
//      Currently the highest performance for scientific computation.
//
//   -- CC 5.2
//      Maxwell
//      Product cycle ending.
//
//   -- CC 6.X
//      Pascal
//      Product cycle starting.
//      Currently the highest performance for single-precision computation.



 /// Hardware Organization
//
//   Streaming Multiprocessor (SM, SMX)
//
//   Functional Unit
//     A piece of hardware that can perform a particular set of operations.
//     Examples:
//
//        CUDA Core:  Can perform most single-precision non-divide FP.


 /// CUDA Thread Organization
//
//   :Def: Kernel
//
//   :Def: Thread
//     Has one PC.
//      Sort of a brain, or maybe just a body that can be in one place
//      at one time.
//
//   :Def: Block
//     A grouping of threads.
//
//     The number of threads in a block is called the block size.


//     All threads in a block execute on the same multiprocessor.
//     Threads within a block share shared memory.
//
//   :Def: Grid
//     A collection of blocks.
//
//   Warp
//     A collection of 32 threads.
//     One day the size of warp may change but it's been 32 through CC 5.2.
//     Threads in a warp travel together.
//


 /// Kernel Launch
//
//   :Def: Kernel Launch
//         The initiation of execution of CUDA code.
//         Done by a CUDA API call.
//         Specify:
//           The name of the CUDA C procedure to start. (E.g., my_kernel();)
//           The grid size. (The number of blocks.)
//           The block size.
//
 ///  Launch Configuration Criteria
//
//    Number of blocks is a multiple of number of SMs.
//    Number of threads per block is a multiple of 32.


 /// 

#endif

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include "../cuda/intro-vtx-transform/util.h"

struct Vertex
{
  float a[4];
};

struct App
{
  int num_threads;
  int array_size;
  Vertex *v_in;
  float *m_out;
  float *m_out_check;

  float4 *d_v_in;
  float *d_m_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;


///
/// GPU Code (Kernels)
///


__global__ void
cuda_thread_start_simple()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread =
    ( d_app.array_size + d_app.num_threads - 1 ) / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      float4 p = d_app.d_v_in[h];

      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;

      d_app.d_m_out[h] = sos;
    }
}

__global__ void
cuda_thread_start_efficient()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      float4 p = d_app.d_v_in[h];

      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;

      d_app.d_m_out[h] = sos;
    }
}




GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(cuda_thread_start_simple);
  info.GET_INFO(cuda_thread_start_efficient);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = app.num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_bytes = app.array_size * sizeof( app.v_in[0] );
  const int out_size_bytes = app.array_size * sizeof( app.m_out[0] );
  const int overrun_size_bytes = 1024 * sizeof( app.v_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.m_out = new float[app.array_size];
  app.m_out_check = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_m_out, out_size_bytes + overrun_size_bytes ) );

  printf("Array size: %d  4-component vectors.\n", app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.array_size; i++ )
    {
      app.m_out_check[i] = 0;
      for ( int j=0; j<4; j++ )
        app.m_out_check[i] += app.v_in[i].a[j] * app.v_in[i].a[j];
    }

  const int64_t num_ops = 4 * app.array_size;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_v_in, app.v_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_m_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            typedef void (*KPtr)();
            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              ();

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac = 
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const int max_st_len = 52;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s %s\n",
                       num_wps, act_wps,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(comp_frac*max_st_len)]);

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.m_out, app.d_m_out, out_size_bytes, hipMemcpyDeviceToHost) );
            continue; // Battle Short.
            int err_count = 0;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( fabs( app.m_out_check[i] - app.m_out[i] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d: %.7f != %.7f (correct)\n",
                               i, app.m_out[i], app.m_out_check[i] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
