#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2015), GPU Programming
//

 /// Simple CUDA Example, without LSU ECE helper classes.

/// References
//
//

#if 0
/// Background

//  The following background describes the view of NVIDIA GPUs
//  provided by the CUDA system.  "View" means how the hardware
//  appears to the application programmer.  For CUDA, this should
//  be very close to the actual hardware.

 /// Compute Capability (CC)
//
//   An NVIDIA numbering system that identifies the approximate
//   capabilities of the hardware.
//
//   Compute Capabilities
//
//   -- CC 1.0, 1.1, 1.2, 1.3
//      Tesla. (Not to be confused with the Tesla board.)
//      Now considered obsolete.
//
//   -- CC 2.0, 2.1
//      Fermi
//
//   -- CC 3.0, 3.5
//      Kepler
//      Product cycle ending. (17 November 2014,  9:24:12 CST)
//      Currently the highest performance for scientific computation.
//
//   -- CC 5.2
//      Maxwell
//      Product cycle beginning. (17 November 2014,  9:24:16 CST)
//      Currently the highest performance for single-precision computation.


 /// Hardware Organization
//
//   Streaming Multiprocessor (SM, SMX)
//
//   Functional Unit
//     A piece of hardware that can perform a particular set of operations.
//     Examples:
//
//        CUDA Core:  Can perform most single-precision non-divide FP.


 /// Thread Organization
//
//
//   Thread
//     Has one PC.
//      Sort of a brain, or maybe just a body that can be in one place
//      at one time.
//
//   Warp
//     A collection of 32 threads.
//     One day the size of warp may change but it's been 32 through CC 5.2.
//     Threads in a warp travel together.
//
//   Block
//     A collection of threads, number determined by user.
//     All threads in a block execute on the same multiprocessor.
//     Threads within a block share shared memory.
//
//   Grid
//     A collection of blocks.

 /// Kernel Launch
//
//   :Def: Kernel Launch
//         The initiation of execution of CUDA code.
//         Done by a CUDA API call.
//         Specify:
//           The name of the CUDA C procedure to start. (E.g., my_kernel();)
//           The grid size. (The number of blocks.)
//           The block size.
//
 ///  Launch Configuration Criteria
//
//    Number of blocks is a multiple of number of SMs.
//    Number of threads per block is a multiple of 32.


 /// 

#endif

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

struct Vertex
{
  float a[4];
};

struct App
{
  int num_threads;
  int array_size;
  Vertex *v_in;
  float *m_out;
  float4 *d_v_in;
  float *d_m_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;


///
/// GPU Code (Kernels)
///


__global__ void
cuda_thread_start_simple()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread = d_app.array_size / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      float4 p = d_app.d_v_in[h];

      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;

      d_app.d_m_out[h] = sqrtf( sos );
    }
}

__global__ void
cuda_thread_start_efficient()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      float4 p = d_app.d_v_in[h];

      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;

      d_app.d_m_out[h] = sqrtf( sos );
    }
}


///
/// Collect Information About GPU and Code
///

// Info about a specific kernel.
//
struct Kernel_Info {
  void (*func_ptr)();           // Pointer to kernel function.
  const char *name;             // ASCII version of kernel name.
  hipFuncAttributes cfa;       // Kernel attributes reported by CUDA.
};

// Info about GPU and each kernel.
//
struct GPU_Info {
  double bw_Bps;
  static const int num_kernels_max = 4;
  int num_kernels;
  Kernel_Info ki[num_kernels_max];
};

GPU_Info gpu_info;

void
cuda_init()
{
  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  hipGetDeviceCount(&device_count); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  hipDeviceProp_t cuda_prop;  // Properties of cuda device (GPU, cuda version).

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      CE(hipGetDeviceProperties(&cuda_prop,dev));
      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      const int cc_per_mp =
        cuda_prop.major == 1 ? 8 :
        cuda_prop.major == 2 ? ( cuda_prop.minor == 0 ? 32 : 48 ) :
        cuda_prop.major == 3 ? 192 : 0;

      const double chip_bw_Bps = gpu_info.bw_Bps =
        2 * cuda_prop.memoryClockRate * 1000.0
        * ( cuda_prop.memoryBusWidth >> 3 );
      const double chip_sp_flops =
        1000.0 * cc_per_mp * cuda_prop.clockRate
        * cuda_prop.multiProcessorCount;

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize >> 10,
         chip_bw_Bps * 1e-9,
         chip_sp_flops * 1e-9,
         4 * chip_sp_flops / chip_bw_Bps);

    }

  // Choose GPU 0 because we don't have time to provide a way to let
  // the user choose.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);

  gpu_info.num_kernels = 0;

#define GET_INFO(proc_name) {                                                 \
  const int idx = gpu_info.num_kernels++;                                     \
  if ( idx < gpu_info.num_kernels_max ) {                                     \
    gpu_info.ki[idx].name = #proc_name;                                       \
    gpu_info.ki[idx].func_ptr = (void(*)())proc_name;                         \
    CE(hipFuncGetAttributes(&gpu_info.ki[idx].cfa,reinterpret_cast<const void*>(proc_name)));               \
  }}

  GET_INFO(cuda_thread_start_simple);
  GET_INFO(cuda_thread_start_efficient);

#undef GET_INFO

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");

  for ( int i=0; i<gpu_info.num_kernels; i++ )
    {
      printf("For %s:\n", gpu_info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             gpu_info.ki[i].cfa.sharedSizeBytes,
             gpu_info.ki[i].cfa.constSizeBytes,
             gpu_info.ki[i].cfa.localSizeBytes,
             gpu_info.ki[i].cfa.numRegs,
             gpu_info.ki[i].cfa.maxThreadsPerBlock);
    }

  printf("\n");
}

///
/// Main Routine
///

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  app.num_threads = abs(nt_raw);

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);
  const int out_array_size_bytes = app.array_size * sizeof(app.m_out[0]);

  const bool simple = argc < 4 ? 1 : atof(argv[3]);

  if ( argc < 2 ) cuda_init();

  const int threads_per_block = 256;
  const int blocks_per_grid =
    ( app.num_threads + threads_per_block-1 ) / threads_per_block;

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.m_out = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes     ) );
  CE( hipMalloc( &app.d_m_out, out_array_size_bytes ) );

  printf("Launching %d blocks * %d threads for %d elts using %s kernel.\n",
         blocks_per_grid, threads_per_block, app.array_size,
         simple ? "simple" : "efficient" );

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  const double time_start = time_fp();

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );



  /// Launch Kernel
  //
  if ( simple )

    cuda_thread_start_simple<<< blocks_per_grid, threads_per_block >>>();

  else

    cuda_thread_start_efficient<<< blocks_per_grid, threads_per_block >>>();

  // Copy output array from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.m_out, app.d_m_out, out_array_size_bytes, hipMemcpyDeviceToHost) );

  const double data_size = app.array_size * ( sizeof(Vertex) + sizeof(float) );
  const double fp_op_count = app.array_size * 5;
  const double elapsed_time = time_fp() - time_start;

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);
}
