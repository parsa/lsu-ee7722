#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2018), GPU Programming
//

 /// Simple CUDA Example, without LSU ECE helper classes.

/// References
//
//  :ccpg19: CUDA C Programming Guide Version 10
//           https://docs.nvidia.com/cuda/cuda-c-programming-guide


#if 0
/// Background

//  The following background describes the view of NVIDIA GPUs
//  provided by the CUDA system.  "View" means how the hardware
//  appears to the application programmer.  For CUDA, this should
//  be very close to the actual hardware.

 /// Compute Capability (CC)
//
//   An NVIDIA numbering system that identifies the approximate
//   capabilities of the hardware.
//
//   Compute Capabilities
//
//   -- CC 1.0, 1.1, 1.2, 1.3
//      Tesla. (Not to be confused with the Tesla board.)
//      Now considered obsolete.
//
//   -- CC 2.0, 2.1
//      Fermi
//
//   -- CC 3.0, 3.1, 3.5, 3.7
//      Kepler
//      Outdated.
//
//   -- CC 5.2
//      Maxwell
//      Outdated.
//
//   -- CC 6.0, 6.1, 6.2
//      Pascal
//      Product cycle ending.
//      If cost is no object, good double-precision and half-precision perf.
//      Some support for machine learning: 16-bit floats.
//
//   -- CC 7.0
//      Volta
//      Not used much for graphics.
//      If cost is no object, good double-precision and half-precision perf.
//      Machine learning support.
//
//   -- CC 7.5
//      Turing
//      Ray tracing support.
//      Machine learning support.


 /// CUDA Thread Organization
//
//   :Def: Kernel
//     A procedure that executes on the GPU.
//     "I launched a kernel to multiply two 1000 by 1000 matrices."
//
//   :Def: Thread
//     Similar to the definition of a thread on a CPU.
//     A path of execution through the kernel.
//     Each Thread:
//       Has its own id.
//       The id consists of a thread index, in variable threadIdx, ..
//       .. and a block index, in variable blockIdx.
//     "My kernel consists of 16384 threads."
//
//   :Def: Block
//     A grouping of threads.
//
//     The number of threads in a block is called the block size ..
//     .. its value is in variable blockDim.
//     "My kernel has a block size of 1024 threads."

//
//   :Def: Grid
//     A collection of blocks.
//     The grid size is specified in the kernel launch.
//     "My kernel consists of 16 blocks of 1024 threads each."
//
//   :Def: Warp
//     A group of threads that (usually) execute together.
//     For all NV GPUs so far warp size is 32 threads
//     One day the size of warp may change but it's been 32 through CC 7.5.
//     "I chose my block size to be a multiple of the warp size."
//


 /// Hardware Organization
//
//   :Def: Streaming Multiprocessor (SM, SMX, MP)
//     The hardware to execute a block of threads.
//     In class called a multiprocessor (the word streaming omitted) for short.
//     Roughly akin to a core in a CPU.
//     High-performance GPUs might have about 50 SMs.
//
//     Each block is assigned to a particular SM.
//     All threads in a block execute on the same multiprocessor.
//     Threads within a block share shared memory.
//
//      "Uh-oh, my new GPU has 50 SMs. I hope my code can launch enough
//       blocks to keep them all busy."
//         
//
//   :Def: Functional Unit
//     A piece of hardware that can perform a particular set of operations.
//     Typical, GPU and Non-GPU Examples:
//        Integer ALU: Can perform such as add, sub, AND, OR, etc.
//        Integer multiply.
//        FP add, mul, madd.
//        FP div, sqrt, trig.
//     NVIDIA GPU Units:
//        CUDA Core:  Can perform most single-precision non-divide FP.
//        Special Func Unit: division, reciprocal square root, approx trig.
//        Load / Store: Read and write from memory.


 /// Kernel Launch
//
//   :Def: Kernel Launch
//     The initiation of execution of CUDA code.
//     Done by a CUDA API call.
//     Specify:
//        The name of the CUDA C procedure to start. (E.g., my_kernel();)
//        The grid size. (The number of blocks.)
//        The block size.
//
//   :Def: Launch Configuration
//     The block size and grid sized used for a kernel launch.
//     Choosing the correct launch configuration is very important.
//         
//
 ///  Launch Configuration Criteria
//
//    - Number of blocks is a multiple of number of SMs.
//    - Number of threads per block is a multiple of warp size (32).


 /// Global Memory Access
//
//   :Sample:  mval = a[tid];
//
//   Important rule:
//
//     Consecutive threads should access consecutive data items.
//       As in:  mval = a[ tid ];         // Good. ☺
//       NOT:    mval = a[ tid * 1000 ];  // BAD.  ☹
//
//     Size of contiguous chunks (accessed by consecutive threads)     
//     should be a multiple of 32 bytes.
//
 /// Possible Locations of Global Data
//
//   - Off-Chip Global Memory
//     Requires about 400 cycles to obtain data.
//     Subject to off-chip BW limit. 
//       BW limit in Telsa K20c:  208 GB/s
//       BW limit in GTX 780:     288 GB/s
//
//   - Level 2 Cache  (Not caché, please).
//     Size varies, about 1 MiB.
//     Requires about 200 cycles to obtain data.
//     Much higher BW limit.
//
//   Note: L2 only has a small latency benefit.
//         Larger benefit is higher bandwidth.
//
 /// Memory Requests
//
//   How Things Work (CC 3.x to 6.x)
//
//     - Threads in a warp execute a load instruction. E.g., mval = a[tid];
//
//     - Hardware coalesces these loads based on address into
//        contiguous *requests* of size 32, 64, or 128 B.
//
//     - Requests are sent to L2 cache, and if necessary, off-chip storage.
//
//     - Dependent instructions can execute when requests return.
//
//   Implications
//
//     Bandwidth consumed determined by request size ..
//     .. not by how much data actually needed.
//
//     Possible slow down with a larger number of requests ..
//     .. so 10 128-B requests better than 40 32-B request ..
//     .. even though they are the same size.


 /// 

#endif

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>


struct App
{
  int num_threads;
  int array_size;

  float4 *h_in;         // Host address space, data in.
  float *h_out;         // Host address space, data out.
  float *h_out_check;   // Compute correct answer on CPU, to check GPU.

  float4 *d_in;         // Device address space, data in.
  float *d_out;         // Device address space, data out.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;


///
/// GPU Code (Kernels)
///


__global__ void
kmain_simple()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread =
    ( d_app.array_size + d_app.num_threads - 1 ) / d_app.num_threads;
  const int start = elt_per_thread * tid;  // Bad: Non-consecutive access.
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      float4 p = d_app.d_in[h];  // Bad: Non-consecutive access.
      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;
      d_app.d_out[h] = sos;
    }
}

__global__ void
kmain_efficient()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if ( tid >= d_app.num_threads ) return;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      float4 p = d_app.d_in[h];  // Good: Consecutive access.
      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;
      d_app.d_out[h] = sos;
    }
}


__global__ void
kmain_tuned()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if ( tid >= d_app.num_threads ) return;

  const int strip_len = 4;
  // Data "strip" is 32 threads wide and strip_len threads long.

  const int wp_sz = 32;         // Warp size.
  const int wp = tid / wp_sz;   // This thd's warp number within kernel. (0-)
  const int ln = tid % wp_sz;   // This thd's lane number within warp.  (0-31)
  const int start = wp * wp_sz * strip_len + ln;

  for ( int h=start; h<d_app.array_size; h += strip_len * d_app.num_threads )
    {
      float soses[strip_len];
      for ( int i=0; i<strip_len; i++ )
        {
          float4 p = d_app.d_in[ h + i * wp_sz ];
          soses[i] = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;
        }
      for ( size_t i=0; i<strip_len; i++ )
        d_app.d_out[ h + i * wp_sz ] = soses[i];
    }
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(kmain_simple);
  info.GET_INFO(kmain_efficient);
  info.GET_INFO(kmain_tuned);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_bytes = app.array_size * sizeof( app.h_in[0] );
  const int out_size_bytes = app.array_size * sizeof( app.h_out[0] );
  const int overrun_size_bytes = num_blocks * 1024 * sizeof( app.h_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new float4[app.array_size];
  app.h_out = new float[app.array_size];
  app.h_out_check = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Array size: %d  4-component vectors.\n", app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) ((float*)&app.h_in[i])[j] = drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.array_size; i++ )
    {
      float4 p = app.h_in[i];
      app.h_out_check[i] = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;
    }

  /// Compute Expected Computation and Communication
  //
  // Number of multiply/add operations. Ignore everything else.
  //
  const int64_t num_ops = 4 * app.array_size;  // Multiply-adds.
  //
  // Amount of data in and out of GPU chip.
  //
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 1;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            if ( vary_warps && wp_cnt > 4 && wp_cnt & 0x3 ) continue;

            app.num_threads = thd_per_block * num_blocks;

            // Copy App structure to GPU.
            //
            CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            typedef void (*KPtr)();

            /// Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr) <<< num_blocks, thd_per_block >>>();

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                const bool graph_bw = true;
                const double frac = graph_bw ? bw_frac : comp_frac;

                const int max_st_len = 43;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d %2d wp %6.0f µs %3.0f GF  %3.0f GB/s %s\n",
                       num_wps, act_wps,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(frac*max_st_len)]);

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDefault) );
            int err_count = 0;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( fabs( app.h_out_check[i] - app.h_out[i] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d: %.7f != %.7f (correct)\n",
                               i, app.h_out[i], app.h_out_check[i] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
