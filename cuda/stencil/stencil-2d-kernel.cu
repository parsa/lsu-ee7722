#include "hip/hip_runtime.h"
#include "stencil-2d.cuh"
#include <gp/cuda-util-kernel.h>


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ int row_stride, dim_size_lg, dim_block_lg;
__constant__ float* a;
__constant__ float* b;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__global__ void stencil();
__global__ void stencil_iter();
__global__ void stencil_blocked();
__global__ void stencil_shared();

__host__ hipError_t
kernels_get_attr(struct hipFuncAttributes *attr, char **names)
{
  hipError_t er;

  CU_SYM(a); CU_SYM(b);
  CU_SYM(array_size);
  CU_SYM(row_stride); CU_SYM(dim_size_lg); CU_SYM(dim_block_lg);
  CU_SYM(v0); CU_SYM(v1); CU_SYM(v2);

#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er ) return er;
  GETATTR(stencil);
  GETATTR(stencil_iter);
  GETATTR(stencil_blocked);
  GETATTR(stencil_shared);
  *names = NULL;
  return er;
#undef GETATTR
}

// This routine executes on the CPU.
//
__host__ void
stencil_launch(dim3 dg, dim3 db, bool blocked)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  //  stencil<<<dg,db>>>();
  if ( blocked )
    {
      stencil_shared<<<dg,db,db.x*3*sizeof(float)>>>();
    }
  else
    {
      stencil_iter<<<dg,db>>>();
    }
}

__global__ void
stencil()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;

  int col = idx & row_mask;
  int row = idx >> dim_size_lg;
  if ( row == 0 || row >= row_mask || col == 0 || col == row_mask ) return;

  int iu = idx - row_stride;
  int id = idx + row_stride;

  b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
    + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
}

__global__ void
stencil_iter()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;
  int thread_count = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      int col = idx & row_mask;
      int row = idx >> dim_size_lg;
      if ( row == 0 || row >= row_mask || col == 0 || col == row_mask )
        continue;

      int iu = idx - row_stride;
      int id = idx + row_stride;

      b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
        + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
    }
}

__global__ void
stencil_blocked()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tix = threadIdx.x;
  int tid = tix + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;

  int last_lrow = ( 1 << dim_block_lg ) - 1;
  int last_row = ( 1 << dim_size_lg ) - 1;

  int col_mask = ( 1 << dim_block_lg ) - 1;
  int array_row_stride = 1 << dim_size_lg;

  int thd_lrow = tix >> dim_block_lg;
  int thd_lcol = tix & col_mask;

  int block_col_shift = dim_block_lg << 1;
  int block_row_shift = dim_size_lg + dim_block_lg;
  int dim_umm_lg = dim_size_lg - dim_block_lg;
  int block_side_mask = ( 1 << dim_umm_lg ) - 1;

  int fat_row_stride = ( 1 << dim_block_lg ) + 2;

  int sidx = ( thd_lrow + 1 ) * fat_row_stride + ( thd_lcol + 1 );
  int siu = sidx - fat_row_stride;
  int sid = sidx + fat_row_stride;

  for ( int itid = tid; itid < array_size; itid += thread_count )
    {
      int block_col_start = ( itid >> block_col_shift ) & block_side_mask;
      int block_row_start = ( itid >> block_row_shift ) & block_side_mask;

      int thd_col = ( block_col_start << dim_block_lg ) + thd_lcol;
      int thd_row = ( block_row_start << dim_block_lg ) + thd_lrow;

      int idx = ( thd_row << dim_size_lg ) | thd_col;

      int iu = idx - array_row_stride;
      int id = idx + array_row_stride;
      s[sidx] = a[idx];

      bool edge = thd_row == 0 || thd_col == 0
        || thd_row == last_row || thd_col == last_row;

      if ( thd_lrow == 0 && thd_row > 0 )
        s[siu] = a[iu];
      if ( thd_lrow == last_lrow && thd_row < last_row )
        s[sid] = a[id];
      if ( thd_lcol == 0 && thd_col > 0 )
        s[sidx-1] = a[idx-1];
      if ( thd_lcol == last_lrow && thd_col < last_row )
        s[sidx+1] = a[idx+1];
      if ( thd_lrow == 0 && thd_lcol == 0 && !edge )
        s[siu-1] = a[iu-1];
      if ( thd_lrow == 0 && thd_lcol == last_lrow &&
           thd_row > 0 && thd_col < last_row )
        s[siu+1] = a[iu+1];
      if ( thd_lrow == last_lrow && thd_lcol == last_lrow
           && thd_row < last_row && thd_col < last_row )
        s[sid+1] = a[id+1];
      if ( thd_lrow == last_lrow && thd_lcol == 0
           && thd_row < last_row && thd_col > 0 )
        s[sid-1] = a[id-1];

      __syncthreads();

      if ( edge ) continue;

#if DEBUG_STENCIL
      b[idx] = v0 * s[sid+1];
#else
      b[idx] = v0 * s[sidx] + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
        + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
    }

}


__global__ void
stencil_shared()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  int block_compute_width = blockDim.x - 2;
  int blocks_per_row =
    int( ceilf( float(array_row_stride-2) / block_compute_width ) );

  int rows_per_block =
    float(blocks_per_row) * array_row_stride / gridDim.x + 0.999f;

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  int siu = threadIdx.x;
  int sidx = siu + blockDim.x;
  int sid = sidx + blockDim.x;

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 = 
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * block_compute_width;
      int col_9 = min( col_0 + blockDim.x, array_row_stride );

      int col = col_0 + threadIdx.x;
      if ( col >= array_row_stride ) return;
      int row_0s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      bool load_only =
        threadIdx.x == 0 || threadIdx.x == blockDim.x-1
        || col == array_row_stride - 1;

      int rows = row_0s;
      int idx = rows | col;
      int idx_stop = row_9s | col;
      s[siu] = a[idx];
      idx += array_row_stride;
      s[sidx] = a[ idx ];
      while ( idx < idx_stop )
        {
          int idx_next = idx + array_row_stride;
          s[sid] = a[idx_next];
          s[sid+1] = a[idx_next+1];
          __syncthreads();
          if ( !load_only )
            {
#if DEBUG_STENCIL
              b[idx] = v0 * s[sidx];
#else
              b[idx] = v0 * s[sidx]
                + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
                + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
            }
          //  __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          idx = idx_next;
        }

    }


}
