#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <gp/cuda-util-kernel.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1;
__constant__ int array_size;
__constant__ float* a;
__constant__ float* b;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__global__ void stencil();
__global__ void stencil_iter();
__global__ void stencil_shared();

__host__ hipError_t
kernels_get_attr(struct hipFuncAttributes *attr, char **names)
{
  // Return attributes of kernels, such as number of registers
  // used.

  hipError_t er;

  CU_SYM(a); CU_SYM(b);
  CU_SYM(array_size);
  CU_SYM(v0); CU_SYM(v1);

#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er ) return er;
  GETATTR(stencil);
  GETATTR(stencil_iter);
  GETATTR(stencil_shared);
  *names = NULL;
  return er;
#undef GETATTR
}

__host__ void
stencil_launch(dim3 dg, dim3 db, bool shared_kernel)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  // stencil<<<dg,db>>>();

  int shared_amt = (db.x + 2) * sizeof(float);

  if ( shared_kernel )
    stencil_shared<<<dg,db,shared_amt>>>();
  else
    stencil_iter<<<dg,db>>>();
}

__global__ void
stencil()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if ( idx >= array_size ) return;

  b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] );
}

__global__ void
stencil_iter()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] );
    }
}

__global__ void
stencil_shared()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;

  int bl_idx = blockDim.x - 1;
  int sidx = threadIdx.x + 1;

  __shared__ float s[1024];

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      // Load shared memory with a values.
      //
      float a_0 = a[idx];
      s[sidx] = a_0;

      // Two lucky threads per block get to preload a second element.
      //
      if ( threadIdx.x == 0 )
        s[0] = a[idx-1];
      if ( threadIdx.x == bl_idx )
        s[blockDim.x+1] = a[idx+1];

      // Wait for everyone to finish.
      //
      //  __syncthreads();

      // Note that values read from a register or shared memory.
      //
      b[idx] = v0 * a_0 + v1 * ( s[sidx-1] + s[sidx+1] );
    }
}

