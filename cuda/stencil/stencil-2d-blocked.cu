#include "hip/hip_runtime.h"
#include "stencil-2d.cuh"


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ int row_stride, dim_size_lg, dim_block_lg;
__constant__ float* a;
__constant__ float* b;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__global__ void stencil();
__global__ void stencil_iter();
__global__ void stencil_shared();

__host__ hipError_t
kernels_get_attr(struct hipFuncAttributes *attr, char **names)
{
  hipError_t er;

#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er ) return er;
  GETATTR(stencil);
  GETATTR(stencil_iter);
  GETATTR(stencil_shared_shared);
  *names = NULL;
  return er;
#undef GETATTR
}

// This routine executes on the CPU.
//
__host__ void
stencil_launch(dim3 dg, dim3 db, bool blocked)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  //  stencil<<<dg,db>>>();
  if ( blocked )
    {
      stencil_shared_shared<<<dg,db,db.x*3*sizeof(float)>>>();
    }
  else
    {
      stencil_iter<<<dg,db>>>();
    }
}

__global__ void
stencil()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;

  int col = idx & row_mask;
  int row = idx >> dim_size_lg;
  if ( row == 0 || row >= row_mask || col == 0 || col == row_mask ) return;

  int iu = idx - row_stride;
  int id = idx + row_stride;

  b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
    + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
}

__global__ void
stencil_iter()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;
  int thread_count = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      int col = idx & row_mask;
      int row = idx >> dim_size_lg;
      if ( row == 0 || row >= row_mask || col == 0 || col == row_mask )
        continue;

      int iu = idx - row_stride;
      int id = idx + row_stride;

      b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
        + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
    }
}

#if 1
__global__ void
stencil_shared_blocked()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tix = threadIdx.x;
  int tid = tix + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;

  const int shared_size = 2048;
  __shared__ float s[shared_size];

  int last_lrow = ( 1 << dim_block_lg ) - 1;
  int last_row = ( 1 << dim_size_lg ) - 1;

  int col_mask = ( 1 << dim_block_lg ) - 1;
  int array_row_stride = 1 << dim_size_lg;

  int thd_lrow = tix >> dim_block_lg;
  int thd_lcol = tix & col_mask;

  int block_col_shift = dim_block_lg << 1;
  int block_row_shift = dim_size_lg + dim_block_lg;
  int dim_umm_lg = dim_size_lg - dim_block_lg;
  int block_side_mask = ( 1 << dim_umm_lg ) - 1;

  int fat_row_stride = ( 1 << dim_block_lg ) + 2;

  int last_block_col_start = block_side_mask;
  int last_block_row_start = block_side_mask;
  int sidx_last = fat_row_stride * fat_row_stride - 1;

  int sidx = ( thd_lrow + 1 ) * fat_row_stride + ( thd_lcol + 1 );
  int siu = sidx - fat_row_stride;
  int sid = sidx + fat_row_stride;

#if 1
  int sidx_edge;
  enum Edge_Case { EC_top, EC_bottom, EC_left, EC_right,
                   EC_top_left, EC_top_right,
                   EC_bottom_left, EC_bottom_right,
                   EC_none } edge_case;

  if ( thd_lrow == 0 )
    { sidx_edge = siu;  edge_case = EC_top; }
  else if ( thd_lrow == last_lrow )
    {
      sidx_edge = sid;
      edge_case = EC_bottom;
    }
  else if ( thd_lrow == 2 )
    {
      sidx_edge = (thd_lcol+1)*fat_row_stride;
      edge_case = EC_left;
    }
  else if ( thd_lrow == 3 )
    {
      sidx_edge = (thd_lcol+1)*fat_row_stride+fat_row_stride-1;
      edge_case = EC_right;
    }
  else if ( thd_lrow == 1 && thd_lcol == 1 )
    { sidx_edge = 0; edge_case = EC_top_left; }
  else if ( thd_lrow == 1 && thd_lcol == 2 )
    { sidx_edge = fat_row_stride-1; edge_case = EC_top_right; }
  else if ( thd_lrow == 1 && thd_lcol == 3 )
    { sidx_edge = sidx_last; edge_case = EC_bottom_right; }
  else if ( thd_lrow == 1 && thd_lcol == 4 )
    { sidx_edge = sidx_last-fat_row_stride+1; edge_case = EC_bottom_left; }
  else
    { sidx_edge = -1; edge_case = EC_none; }

#endif

  //  int thd_local_row_col = ( thd_lrow << dim_size_lg ) | thd_rcol;
  //  int tc_incr = thread_count >> dim_block_lg;

  for ( int itid = tid; itid < array_size; itid += thread_count )
    {
      int block_col_start = ( itid >> block_col_shift ) & block_side_mask;
      int block_row_start = ( itid >> block_row_shift ) & block_side_mask;

      int thd_col = ( block_col_start << dim_block_lg ) + thd_lcol;
      int thd_row = ( block_row_start << dim_block_lg ) + thd_lrow;

      int idx = ( thd_row << dim_size_lg ) | thd_col;

      int iu = idx - array_row_stride;
      int id = idx + array_row_stride;
      s[sidx] = a[idx];

      bool edge = thd_row == 0 || thd_col == 0
        || thd_row == last_row || thd_col == last_row;

#if 1

#if 1
      int thd_col_0l = ( block_col_start << dim_block_lg ) - 1;
      int thd_col_9r = ( block_col_start + 1 ) << dim_block_lg;
      int thd_row_0 =  ( block_row_start << ( dim_block_lg + dim_size_lg ) );
      int thd_row_0u = ( block_row_start << ( dim_block_lg + dim_size_lg ) )
        - array_row_stride;
      int thd_row_9d = (block_row_start+1) << ( dim_block_lg + dim_size_lg );

      int idx_edge = -1; // Reassigned below.

      switch ( edge_case ){
      case EC_none:
        idx_edge = -1;
        break;
      case EC_top:
        if ( block_row_start > 0 ) idx_edge = iu;
        break;
      case EC_bottom:
        if ( block_row_start < last_block_row_start ) idx_edge = id;
        break;
      case EC_left:
        if ( block_col_start )
          idx_edge = thd_row_0|(thd_lcol<<dim_size_lg)|thd_col_0l;
        break;
      case EC_right:
        if ( block_col_start < last_block_col_start )
          idx_edge = thd_row_0|(thd_lcol<<dim_size_lg)|thd_col_9r;
        break;
      case EC_top_left:
        if ( block_row_start && block_col_start )
          idx_edge = thd_row_0u|thd_col_0l;
        break;
      case EC_top_right:
        if ( block_row_start && block_col_start < last_block_col_start )
          idx_edge = thd_row_0u|thd_col_9r;
        break;
      case EC_bottom_right:
        if ( block_col_start < last_block_col_start 
             && block_row_start < last_block_row_start )
          idx_edge = thd_row_9d|thd_col_9r;
        break;
      case EC_bottom_left:
        if ( block_col_start && block_row_start < last_block_row_start )
          idx_edge = thd_row_9d|thd_col_0l;
        break;
      }

      if ( idx_edge >= 0 )
        s[sidx_edge] = a[idx_edge];
#endif

#else
      if ( thd_lrow == 0 && thd_row > 0 )
        s[siu] = a[iu];
      if ( thd_lrow == last_lrow && thd_row < last_row )
        s[sid] = a[id];
      if ( thd_lcol == 0 && thd_col > 0 )
        s[sidx-1] = a[idx-1];
      if ( thd_lcol == last_lrow && thd_col < last_row )
        s[sidx+1] = a[idx+1];
      if ( thd_lrow == 0 && thd_lcol == 0 && !edge )
        s[siu-1] = a[iu-1];
      if ( thd_lrow == 0 && thd_lcol == last_lrow &&
           thd_row > 0 && thd_col < last_row )
        s[siu+1] = a[iu+1];
      if ( thd_lrow == last_lrow && thd_lcol == last_lrow
           && thd_row < last_row && thd_col < last_row )
        s[sid+1] = a[id+1];
      if ( thd_lrow == last_lrow && thd_lcol == 0
           && thd_row < last_row && thd_col > 0 )
        s[sid-1] = a[id-1];
#endif

      __syncthreads();

      if ( edge ) continue;

#if 0
      b[idx] = v0 * s[sid+1];
#else
      b[idx] = v0 * s[sidx] + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
        + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif

    }

}
#endif

__global__ void
stencil_shared_blocked2()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  int block_compute_width = blockDim.x - 2;
  int blocks_per_row =
    int( ceilf( float(array_row_stride-2) / block_compute_width ) );

  int rows_per_block =
    float(blocks_per_row) * array_row_stride / gridDim.x + 0.999f;

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  int siu = threadIdx.x;
  int sidx = siu + blockDim.x;
  int sid = sidx + blockDim.x;

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 = 
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * block_compute_width;
      int col_9 = min( col_0 + blockDim.x, array_row_stride );

      int col = col_0 + threadIdx.x;
      if ( col >= array_row_stride ) return;
      int row_0s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      bool load_only =
        threadIdx.x == 0 || threadIdx.x == blockDim.x-1
        || col == array_row_stride - 1;

      int rows = row_0s;
      int idx = rows | col;
      int idx_stop = row_9s | col;
      s[siu] = a[idx];
      idx += array_row_stride;
      s[sidx] = a[ idx ];
      while ( idx < idx_stop )
        {
          int idx_next = idx + array_row_stride;
          s[sid] = a[idx_next];
          __syncthreads();
          if ( !load_only )
            {
#if 0
              b[idx] = v0 * s[sidx];
#else
              b[idx] = v0 * s[sidx]
                + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
                + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
            }
          //  __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          idx = idx_next;
        }

    }
}
