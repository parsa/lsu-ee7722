#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Simple, Self-Contained, One-File CUDA Example

 /// How to Compile from the Command Line
//
//   nvcc -o addr-space addr-space.cu -O3 -Xcompiler -Wall

 /// Documentation
//
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//   C++:  http://en.cppreference.com/w/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
using namespace std;

#if 0
/// Address Spaces

 // - Global
 // - Constant
 // - Shared
 // - Local


/// Global Address Space

 /// Size:  32- or 64-bit address space.  Matches size used by host OS, usually 64 bits.

 /// Scope:
 //
 //   Single space shared by all threads.
 //   Readable and writeable by all threads and host.
 //   Difficult to use for sharing of data.

 /// Latency, Bandwidth
 //
 //  Cases
 //    - Off-Chip Access. (Most common.)
 //    - L2 Cache Hit.  (Fairly common. Possible through Volta [CC 7.x])
 //    - L1 Cache Hit.  (Depends on CC.  Possible in CC 2.x, 7.0)
 //    - Texture Cache Hit. (Depends on CC.)

 //  Off-Chip Access
 //    - Latency hundreds of cycles. Course default: 400 cycles.
 //    - Limited by off-chip bandwidth. 400 GB/s for high-end devices. (2018)
 //    

 /// Declaration
 //
 //  In global scope:
 //
 //  :Syntax:  __device__ TYPE VARNAME;
 //  :Sample:  __device__ double mydata[1000];
 //
 //  Note: Sample above shows *static* allocation of global space.
 //        In most cases dynamic allocation is used.


 /// Dynamic Allocation on Host
 //
 //  :Syntax: hipMalloc( PTR, AMT_CHARS );
 //  Allocates AMT_CHARS bytes of storage in global space,
 //  and puts address of that storage in PTR.
 //


 /// Transfer Between CPU and GPU (Either Direction)
 //
 //  :Syntax: hipMemcpy( DST_PTR, SRC_PTR, SZ_CHARS, hipMemcpyHostToDevice );
 //  Copies SZ_CHARS bytes 
 //  from the CPU (host) starting at address SRC_PTR
 //  to the GPU (device) starting at address DST_PTR, where
 //  SRC_PTR is an address in the CPU address space and
 //  DST_PTR is an address in the GPU global address space.
 //
 //  :Syntax: hipMemcpy( DST_PTR, SRC_PTR, SZ_CHARS, hipMemcpyDeviceToHost );
 //  Copies SZ_CHARS bytes 
 //  from the GPU (device) starting at address SRC_PTR
 //  to the CPU (host) starting at address DST_PTR, where
 //  SRC_PTR is an address in the GPU global address space and
 //  DST_PTR is an address in the CPU address space.

 /// Short Example

const int size = 1024 * 32;
__device__ float a[size];
__device__ float b[size];

__global__ void thread_main(float *x)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  a[idx] = idx + blockIdx.x;
  b[idx] = float(blockIdx.x) / (idx+1);
  x[idx] = a[idx] + b[idx];
}

__host__ int main(int argc, char** argv)
{
  void *x_dev;
  hipMalloc( &x_dev, size * sizeof(x_dev[0]) );

  thread_main<<<32,1024>>>(x_dev);

  // Copy data from GPU to CPU.
  //
  float x[size];
  hipMemcpy( x, x_dev, size * sizeof(x[0]), hipMemcpyDeviceToHost );
}


/// Constant Address Space

 /// Size 64 kiB ( 16-bit address space )

 /// Scope
 //
 //   Single space shared by all threads.
 //   Readable by all threads.
 //   Cannot be written by threads.
 //   Writeable from CPU.

 /// Declaration
 //
 //  In global scope:
 //
 //  :Syntax:  __constant__ TYPE VARNAME;
 //  :Sample:  __constant__ int my_int_var;

 /// Transfer Between CPU and GPU
 //
 // :Syntax: hipMemcpyToSymbol(HIP_SYMBOL(DST_SYM), SRC_PTR, SZ_CHARS, OFF, hipMemcpyHostToDevice );
 // Copy SZ_CHARS from CPU starting at address SRC_PTR + OFF
 // to GPU symbol DST_SYM, which can be in constant address space.


 /// Short Example

__constant__ int d_size;
__constant__ float *d_x;

__device__ float a[size];
__device__ float b[size];

__global__ void thread_main()
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if ( idx >= d_size ) return;

  a[idx] = idx + blockIdx.x;
  b[idx] = float(blockIdx.x) / (idx+1);
  d_x[idx] = a[idx] + b[idx];
}


__host__ int main(int argc, char** argv)
{
  int size = 1024 * 32;
  void *x_dev;
  hipMalloc( &x_dev, size * sizeof(x_dev[0]) );

  hipMemcpyToSymbol(HIP_SYMBOL( d_size), &size, sizeof(size), 0, hipMemcpyHostToDevice )
  hipMemcpyToSymbol(HIP_SYMBOL( d_x), &x_dev, sizeof(x_dev), 0, hipMemcpyHostToDevice )

  thread_main<<<32,1024>>>();

  // Copy data from GPU to CPU.
  //
  float x[size];
  hipMemcpy( x, x_dev, size * sizeof(x[0]), hipMemcpyDeviceToHost );
}


/// Shared Address Space

 /// Size 48 kiB  (Before CC 7.0)   

 /// Scope
 //
 //   Each block has its own shared address space.
 //   Shared address space shared by all threads within a block.
 //   Readable and writeable by threads.


 /// Declaration
 //
 //  In global or procedure scope
 //
 //  :Syntax:  __shared__ TYPE VARNAME;
 //  :Sample:  __shared__ int my_int_var;

 /// Transfer Between CPU and GPU
 //
 //  Not easily accomplished.


/// Local Address Space

 /// Size 512 kiB per thread.

 /// Scope
 //
 //  Each thread has its own local address space.

 /// Declaration
 //
 //  In procedure scope declared without a qualifier:
 //
 //  :Syntax:  TYPE VARNAME;
 //  :Sample:  float my_array[20];

 /// Implementation  <- IMPORTANT
 //
 //  Registers, If Possible
 //  Some L1 cache, depending on CC.
 //  L2, device memory.
 //




#endif


__device__ int size_d;
__constant__ int size_c;
__shared__ int size_s;


 /// Declaration of Kernel (Entry point for code running on GPU.)
//
// Note: the attribute __global__ indicates that the procedure is
// started by a kernel launch. A GPU-only procedure would use the
// attribute __device__ and a CPU-only procedure would use the
// attribute __host__.
//
__global__ void
thread_main(float *x, float *a, float *b)
{
  // Variables threadIdx, blockIdx, and blockDim are pre-set.
  //

  int size = size_c;

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Array size might not be a multiple of block size.
  //
  if ( idx >= size ) return;

  a[idx] = idx + blockIdx.x;
  b[idx] = float(blockIdx.x) / (idx+1);

  // Perform Computation
  //
  x[idx] = a[idx] + b[idx];
}


__host__ int
main(int argc, char** argv)
{
  const int SIZE = 100000000;

  // Declare host arrays for inputs and output.
  //
  vector<float> a(SIZE);
  vector<float> b(SIZE);
  vector<float> x(SIZE);

  // Compute size of each array.
  //
  const int array_size_chars = a.size() * sizeof(a[0]);

  // Allocate storage for GPU copy of data.
  //
  // The address of the allocated storage is returned in the first
  // argument, a_dev, etc. The addresses are in GPU global space and
  // so they are not necessarily valid on the CPU.
  //
  void *a_dev, *b_dev, *x_dev;
  hipMalloc( &a_dev, array_size_chars );
  hipMalloc( &b_dev, array_size_chars );
  hipMalloc( &x_dev, array_size_chars );

  // Specify Launch Configuration
  //
  const int db = 64;        // Number of threads per block.

  // Choose grid size so that there is at least one thread per array
  // element.
  //
  const int dg = (SIZE + db - 1 ) / db;

  // Launch Kernel
  //
  thread_main<<<dg,db>>>(SIZE, x.data(), a.data(), b.data());

  // Copy data from GPU to CPU.
  //
  hipMemcpy( x.data(), x_dev, array_size_chars, hipMemcpyDeviceToHost );

  printf("Finished with %d elements, element %d is %.5f\n",
         SIZE, argc, x[argc]);

  hipFree( a_dev );
  hipFree( b_dev );
  hipFree( x_dev );
}
