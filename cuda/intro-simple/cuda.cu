#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Simple CUDA Example, without LSU ECE helper classes.

// How to Compile From Emacs
//
//  Within Emacs, as set up for class, compile by pressing [F9].
//
// How to Compile from the Command Line
//
//   Simplest
//     nvcc cuda.cu
//
//   Reasonable
//     nvcc -o cuda cuda.cu  -O3 -g -Xcompiler -Wall

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

 /// Declaration of Kernel (Entry point for code running on GPU.)
//
__global__ void thread_main();
//
// Note: the attribute __global__ indicates that the procedure is
// started by a kernel launch. A GPU-only procedure would use the
// attribute __device__ and a CPU-only procedure would use the
// attribute __host__.


// This routine executes on the GPU.
//
__global__ void
thread_main(int size, float *x, float *a, float *b)
{
  // Variables threadIdx, blockIdx, and blockDim are pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= size ) return;

  // Perform Computation
  //
  x[idx] = a[idx] + b[idx];
}


 /// CUDA API Error-Checking Wrapper
///
#define CE(call) {                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess ) {                                                 \
     printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));  exit(1);}}


__host__ int
main(int argc, char** argv)
{
  srand48(1);                   // Seed random number generator.

  // Declare host arrays for inputs and output.
  //
  vector<float> a(SIZE);
  vector<float> b(SIZE);
  vector<float> x(SIZE);

  // Allocate storage for GPU copy of data.
  //
  // The returned addresses are in GPU global space. They are not
  // valid addresses on the CPU.
  //
  void *a_dev, b_dev, x_dev;
  CE(hipMalloc(&a_dev, host_array_size * sizeof(a[0]) ));
  CE(hipMalloc(&b_dev, host_array_size * sizeof(b[0]) ));
  CE(hipMalloc(&x_dev, host_array_size * sizeof(x[0]) ));

  // Initialize input array.
  //
  float xi = drand48(), yi = drand48();
  for ( int i=0; i<host_array_size; i++ )
    {
      a[i] = xi; b[i] = yi;
      xi += 0.1; yi += 0.1;
    }

  // Move input arrays to GPU.
  //
  CE(hipMemcpy(a_dev, a.data(), host_array_size * sizeof(a[0]),
                hipMemcpyHostToDevice));
  CE(hipMemcpy(b_dev, b.data(), host_array_size * sizeof(b[0]),
                hipMemcpyHostToDevice));

  // Specify Launch Configuration
  //
  dim3 db, dg;
  db.x = 64;          // Number of threads per block in x dimension.
  db.y = db.z = 1;    // Number of threads per block in y and z dimensions.

  // Choose grid size so that there is at least one thread per array
  // element.
  //
  dg.x = (SIZE + db.x - 1 ) / db.x;
  dg.y = dg.z = 1;

  // Launch Kernel
  //
  dots<<<dg,db>>>(SIZE, x.data(), a.data(), b.data());

  // Copy data from GPU to CPU.
  //
  CE(hipMemcpy(x.data(), x_dev, host_array_size * sizeof(x[0]),
                hipMemcpyDeviceToHost));

  printf("Finished with %d elements, element %d is %.5f\n",
         host_array_size, argc, x[argc]);
}

