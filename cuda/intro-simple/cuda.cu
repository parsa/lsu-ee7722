/// LSU EE 7722 GPU Microarchitecture
//
 /// Simple, Self-Contained, One-File CUDA Example

 /// How to Compile from the Command Line
//
//   nvcc -o cuda cuda.cu -O3 -Xcompiler -Wall

 /// Documentation
//
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//   C++:  http://en.cppreference.com/w/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
using namespace std;


 /// Declaration of Kernel (Entry point for code running on GPU.)
//
// Note: the attribute __global__ indicates that the procedure is
// started by a kernel launch. A GPU-only procedure would use the
// attribute __device__ and a CPU-only procedure would use the
// attribute __host__.
//
__global__ void
thread_main(int size, float *x, float *a, float *b)
{
  // Variables threadIdx, blockIdx, and blockDim are pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= size ) return;

  a[idx] = idx + blockIdx.x;
  b[idx] = float(blockIdx.x) / (idx+1);

  // Perform Computation
  //
  x[idx] = a[idx] + b[idx];
}


__host__ int
main(int argc, char** argv)
{
  const int SIZE = 100000000;

  // Declare host arrays for inputs and output.
  //
  vector<float> a(SIZE);
  vector<float> b(SIZE);
  vector<float> x(SIZE);

  // Compute size of each array.
  //
  const int array_size_chars = a.size() * sizeof(a[0]);

  // Allocate storage for GPU copy of data.
  //
  // The address of the allocated storage is returned in the first
  // argument, a_dev, etc. The addresses are in GPU global space and
  // so they are not necessarily valid on the CPU.
  //
  void *a_dev, *b_dev, *x_dev;
  hipMalloc( &a_dev, array_size_chars );
  hipMalloc( &b_dev, array_size_chars );
  hipMalloc( &x_dev, array_size_chars );

  // Specify Launch Configuration
  //
  const int db = 64;        // Number of threads per block.

  // Choose grid size so that there is at least one thread per array
  // element.
  //
  const int dg = (SIZE + db - 1 ) / db;

  // Launch Kernel
  //
  thread_main<<<dg,db>>>(SIZE, x.data(), a.data(), b.data());

  // Copy data from GPU to CPU.
  //
  hipMemcpy( x.data(), x_dev, array_size_chars, hipMemcpyDeviceToHost );

  printf("Finished with %d elements, element %d is %.5f\n",
         SIZE, argc, x[argc]);

  hipFree( a_dev );
  hipFree( b_dev );
  hipFree( x_dev );
}
