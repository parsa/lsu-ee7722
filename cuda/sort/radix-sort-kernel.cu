#include "hip/hip_runtime.h"
#include "radix-sort.cuh"
#include <gp/cuda-util-kernel.h>
#include <assert.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//

__constant__ Radix_Sort_GPU_Constants dapp;

__constant__ int *scan_out;
__constant__ int *scan_r2;

extern __shared__ Sort_Elt s[];

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__constant__ Sort_Elt *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

template <int BLOCK_LG, int RADIX_LG>
__global__ void radix_sort_1_pass_1(int digit_pos, bool first_iter);
template <int RADIX_LG>
__global__ void radix_sort_1_pass_2(int digit_pos, bool last_iter);

__host__ void
kernels_get_attr(GPU_Info *gpu_info)
{
  CU_SYM(dapp);
  CU_SYM(scan_out);
  CU_SYM(scan_r2);

  CU_SYM(sort_in); CU_SYM(sort_out); CU_SYM(sort_out_b);
  CU_SYM(sort_tile_histo);
  CU_SYM(sort_histo);

#define GETATTR(func) gpu_info->GET_INFO(func)
  GETATTR((radix_sort_1_pass_1<6,4>));
  GETATTR((radix_sort_1_pass_1<7,4>));
  GETATTR((radix_sort_1_pass_1<8,4>));
  GETATTR(radix_sort_1_pass_2<4>);
#undef GETATTR
}


// This routine executes on the CPU.
//
__host__ int
sort_launch(int dg, int db, int array_size, int array_size_lg)
{
  const int radix_lg = 4;
  const int radix = 1 << radix_lg;
  int elt_per_thread = 4;
  int size_per_elt_1 = 4 + 2; // Assuming sort_radix < block_size
  int shared_size_pass_1 = db * size_per_elt_1 * elt_per_thread
    + 4 * radix * 4;
  int shared_size_pass_2 = ( 3 * radix + 1 ) * 4;
  const int key_size_bits = 8 * sizeof(Sort_Elt);
  int ndigits = div_ceil( key_size_bits, radix_lg );
  int digit_pos_start = 0;
  int digit_pos_stop = ndigits;
  if ( !dg ) return shared_size_pass_1;
  for ( int digit_pos = digit_pos_start;
        digit_pos < digit_pos_stop;  digit_pos++ )
    {
      const bool first_iter = digit_pos == digit_pos_start;
      switch(db){
      case 64:
        radix_sort_1_pass_1<6,radix_lg><<<dg,db,shared_size_pass_1>>>
          (digit_pos,first_iter);
        break;
      case 128:
        radix_sort_1_pass_1
          <7,radix_lg><<<dg,db,shared_size_pass_1>>>
          (digit_pos,first_iter);
        break;
      case 256:
        radix_sort_1_pass_1
          <8,radix_lg><<<dg,db,shared_size_pass_1>>>
          (digit_pos,first_iter);
        break;
      }
      radix_sort_1_pass_2<radix_lg><<<dg,db,shared_size_pass_2>>>
        (digit_pos,digit_pos+1==digit_pos_stop);
    }
  return 0;
}

#ifdef DEBUG_SORT
const int debug_sort = true;
#else
const int debug_sort = false;
#endif

__device__ void
sort_block_1_bit_split(int bit_low, int bit_count, int block_lg);


template <int BLOCK_LG, int RADIX_LG>
__device__ void radix_sort_1_pass_1_tile
(int digit_pos, int tile_idx, bool first_iter);

#define SH_GLOBAL_HISTO(elt) s[ ghisto_sbase + (elt) ]
#define SH_TILE_HISTO(idx) s[ thisto_sbase + (idx) ]

template <int BLOCK_LG, int RADIX_LG> __global__ void
radix_sort_1_pass_1(int digit_pos, bool first_iter)
{
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);
  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;

  if ( threadIdx.x < radix ) SH_GLOBAL_HISTO( threadIdx.x ) = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_1_pass_1_tile<BLOCK_LG,RADIX_LG>(digit_pos,tile_idx,first_iter);

  if ( threadIdx.x >= radix ) return;

  int histo_idx = blockIdx.x * radix + threadIdx.x;
  sort_histo[ histo_idx ] = SH_GLOBAL_HISTO( threadIdx.x );
}

template <int BLOCK_LG, int RADIX_LG> __device__ void
radix_sort_1_pass_1_tile(int digit_pos, int tile_idx, bool first_iter)
{
  int start_bit = digit_pos * RADIX_LG;
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  const int digit_mask = radix - 1;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;

  int idx_block_stop = min( dapp.array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;
  int runend_sbase = ghisto_sbase + radix;
  int thisto_sbase = runend_sbase + radix;

  Sort_Elt *sort_src = first_iter ? sort_in : sort_out;

  // Load Element Keys
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_src[ idx_block_start + sidx ];

  // Sort based upon current digit position
  //
  sort_block_1_bit_split(start_bit,RADIX_LG,BLOCK_LG);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = s[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( s[sidx] >> start_bit ) & digit_mask;
      s[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) s[elt_per_tile] = radix;

  // Initialize histogram for this tile to zero.
  //
  if ( threadIdx.x < radix )
    SH_TILE_HISTO( threadIdx.x ) = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];      // Our digit.
      int digit_1 = s[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        s[ runend_sbase + digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];                             // Our digit.
      int digit_0 = sidx > 0 ? int(s[sidx-1]) : -1;    // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = s[ runend_sbase + digit ];
          int count = run_end_sidx - sidx + 1;
          SH_GLOBAL_HISTO( digit ) += count;     // Histogram for block.
          SH_TILE_HISTO( digit ) = count;      // Histogram for tile.
        }
    }

  __syncthreads();

  if ( threadIdx.x >= radix ) return;

  // Write out tile histogram.
  //
  int thisto_idx = tile_idx * radix + threadIdx.x;
  sort_tile_histo[ thisto_idx ] = SH_TILE_HISTO( threadIdx.x );
}

__device__ void
sort_block_1_bit_split(int bit_low, int bit_count, int block_lg)
{
  // Number of elements operated on per thread.
  //
  const int elt_per_thread = 4;

  int elt_per_block = elt_per_thread * blockDim.x;

  // Indices into shared memory for prefix sum.
  // pfe: Exclusive prefix. (Sum of smaller element values.)
  // pfi: Inclusive prefix. (Sum of this element and smaller element values.)
  //
  int pfe_base_rd = elt_per_block;
  int pfi_base_rd = elt_per_block + 1;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      Sort_Elt keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          //  const int sidx = threadIdx.x + i * blockDim.x;
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const Sort_Elt key = s[ sidx ];
          keys[i] = key;
          if ( key & bit_mask ) my_ones_write++;
        }

      s[ pfi_base_rd + threadIdx.x ] = my_ones_write;
      if ( threadIdx.x == 0 ) s[ pfe_base_rd ] = 0;

      uint my_prefix = my_ones_write;

      // Compute a prefix sum of vectors.
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_neighbor = threadIdx.x - dist;
          __syncthreads();
          uint neighbor_prefix =
            threadIdx.x >= dist ? s[ pfi_base_rd + idx_neighbor ] : 0;

          my_prefix += neighbor_prefix;
          __syncthreads();
          s[ pfi_base_rd + threadIdx.x ] = my_prefix;
        }

      // At this point my_prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = s[ pfe_base_rd + blockDim.x ];
      const int idx_one_tid_0 = elt_per_block - all_threads_num_ones;
      const int smaller_tids_num_ones = s[ pfe_base_rd + threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          s[ new_idx ] = key;
        }

    }
  __syncthreads();
}


__device__ void radix_sort_1_pass_2_tile
(int radix_lg, int digit_pos, int tile_idx, bool last_iter);

template <int RADIX_LG>
__global__ void
radix_sort_1_pass_2(int digit_pos, bool last_iter)
{
  int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  const int sort_radix = 1 << RADIX_LG;
  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_radix;
  int pfe_tile_sidx = pfe_tile_sbase + threadIdx.x;

  volatile int *sv = &((int*)s)[ 0 ];

  // Sum of all histogram bins for our digit value (threadIdx.x)
  //
  int global_bin_sum = 0;

  // Sum of histogram bins for our digit value for array indices
  // lower than ours.
  //
  int overhead_bin_sum = 0;

  if ( threadIdx.x == 0 ) sv[ pfe_tile_sbase ] = 0;

  if ( threadIdx.x < sort_radix )
    {
      // Code only correct for sort_radix <= 32.

      int idx_for_us = blockIdx.x * sort_radix + threadIdx.x;
      int pidx_stop = gridDim.x * sort_radix;

      // Compute Global Histogram
      //
      // Loop will set global_bin_sum to the total number of digits
      // of value threadIdx.x.
      //
      // Eg: Array:  123, 121, 373, 932, 863, 113
      //     Based on digit_pos, operating on least-significant digit.
      //     threadIdx.x = 3
      //     Then =>  global_bin_sum = 4
      //
      for ( int idx = threadIdx.x; idx < pidx_stop; idx += sort_radix )
        {
          if ( idx == idx_for_us ) overhead_bin_sum = global_bin_sum;
          global_bin_sum += sort_histo[idx];
        }

      // Compute Global Prefix Sum
      //
      //
      sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_sum;
      int global_bin_prefix = global_bin_sum;

      for ( int i=0; i< RADIX_LG; i++ )
        {
          int dist = 1 << i;
          int sum_0 = dist <= threadIdx.x
            ? sv[ pfi_tile_sbase + threadIdx.x - dist ] : 0;
          sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_prefix += sum_0;
        }

      // Initialize Tile Prefix Sum
      //
      sv[ pfe_tile_sidx ] += overhead_bin_sum;
    }

  __syncthreads();

  volatile int *tile_offsets = &((int*)s)[pf_offset_sbase_base];
  if ( threadIdx.x < sort_radix ) tile_offsets[threadIdx.x]=0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int count;
      if ( threadIdx.x < sort_radix )
        {
          int bo_idx = tile_idx * sort_radix + threadIdx.x;
          count = sort_tile_histo[ bo_idx ];
          int to_idx = sort_radix + threadIdx.x;
          int offset = count;
          tile_offsets[ to_idx ] = offset;
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 1 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 2 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 4 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 8 ];
          if ( debug_sort && last_iter )
            {
              scan_out[bo_idx] = sv[ pfe_tile_sidx ];
              scan_r2[bo_idx] = tile_offsets[ to_idx - 1];
            }
        }
      __syncthreads();
      radix_sort_1_pass_2_tile(RADIX_LG,digit_pos,tile_idx,last_iter);
      __syncthreads();
      if ( threadIdx.x < sort_radix ) sv[ pfe_tile_sidx ] += count;
    }
}

__device__ void
radix_sort_1_pass_2_tile
(int radix_lg, int digit_pos, int tile_idx, bool last_iter)
{
  const int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int idx_tile_start = tile_idx * elt_per_tile;

  const int sort_radix = 1 << radix_lg;
  const int digit_mask = sort_radix - 1;
  int start_bit = digit_pos * radix_lg;

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_radix;
  int pf_offset_sbase = pf_offset_sbase_base + sort_radix - 1;

  for ( int i=0; i<elt_per_thread; i++ )
    {
      int local_idx = threadIdx.x + i * blockDim.x;
      int idx = idx_tile_start + local_idx;
      uint key = sort_out_b[idx];
      uint digit = ( key >> start_bit ) & digit_mask;
      int local_offset = s[ pf_offset_sbase + digit ];
      int key_digit_rank = local_idx - local_offset;
      int idx_digit_index = s[ pfe_tile_sbase + digit ] + key_digit_rank;

      if ( debug_sort && last_iter )
        sort_out[idx] = ( idx_digit_index << 12 ) + local_offset;
      else
        sort_out[idx_digit_index] = key;

    }
}
