#include "hip/hip_runtime.h"
#include "radix-sort.cuh"
#include <gp/cuda-util-kernel.h>
#include <assert.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//

__constant__ Radix_Sort_GPU_Constants dapp;

__constant__ int *scan_out;
__constant__ int *scan_r2;

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__constant__ Sort_Elt *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

template <int BLOCK_LG, int RADIX_LG>
__global__ void radix_sort_1_pass_1(int digit_pos, bool first_iter);
template <int RADIX_LG>
__global__ void radix_sort_1_pass_2(int digit_pos, bool last_iter);

__host__ void
kernels_get_attr(GPU_Info *gpu_info)
{
  CU_SYM(dapp);
  CU_SYM(scan_out);
  CU_SYM(scan_r2);

  CU_SYM(sort_in); CU_SYM(sort_out); CU_SYM(sort_out_b);
  CU_SYM(sort_tile_histo);
  CU_SYM(sort_histo);

#define GETATTR(func) gpu_info->GET_INFO(func)
  GETATTR((radix_sort_1_pass_1<6,4>));
  GETATTR((radix_sort_1_pass_1<7,4>));
  GETATTR((radix_sort_1_pass_1<8,4>));
  GETATTR((radix_sort_1_pass_1<9,4>));
  GETATTR((radix_sort_1_pass_1<10,4>));
  GETATTR((radix_sort_1_pass_1<6,8>));
  GETATTR((radix_sort_1_pass_1<7,8>));
  GETATTR((radix_sort_1_pass_1<8,8>));
  GETATTR((radix_sort_1_pass_1<9,8>));
  GETATTR((radix_sort_1_pass_1<10,8>));
  GETATTR(radix_sort_1_pass_2<4>);
  GETATTR(radix_sort_1_pass_2<8>);
#undef GETATTR
}


// This routine executes on the CPU.
//
__host__ void
sort_launch_pass_1(int dg, int db, int radix_lg, int digit_pos, bool first_iter)
{
# define LAUNCH_RD(BLG,RD_LG) \
  case 1<<BLG: \
  radix_sort_1_pass_1<BLG,RD_LG><<<dg,db>>>(digit_pos,first_iter); \
  break;

#define LAUNCH_BLKS(RD_LG)                                                    \
  case RD_LG: switch ( db ){                                                  \
    LAUNCH_RD(6,RD_LG); LAUNCH_RD(7,RD_LG);                                   \
    LAUNCH_RD(8,RD_LG); LAUNCH_RD(9,RD_LG); LAUNCH_RD(10,RD_LG);              \
  default: assert( false );                                                   \
  } break;

  switch ( radix_lg ) {
    LAUNCH_BLKS(4);
    LAUNCH_BLKS(8);
    default: assert( false );
  }


# undef LAUNCH_RD
# undef LAUNCH_BLKS
}

__host__ void
sort_launch_pass_2
(int dg, int db, int radix_lg, int sm_bytes, int digit_pos, bool last_iter)
{
#define LAUNCH(RD_LG) \
  case RD_LG: \
    radix_sort_1_pass_2<RD_LG><<<dg,db,sm_bytes>>>(digit_pos,last_iter); \
    break;

  switch ( radix_lg ) {
    LAUNCH(4);
    LAUNCH(8);
    default: assert( false );
  }
#undef LAUNCH
}


#ifdef DEBUG_SORT
const int debug_sort = true;
#else
const int debug_sort = false;
#endif

template<int BLOCK_LG, int RADIX_LG>
struct Pass_1_Stuff
{
  Sort_Elt keys[elt_per_thread+(elt_per_thread<<BLOCK_LG)];
  int prefix[4 + (1<<BLOCK_LG)];
  int runend[1<<RADIX_LG];
  int thisto[1<<RADIX_LG];
  int ghisto[1<<RADIX_LG];
};

template <int BLOCK_LG, int RADIX_LG>
__device__ void
sort_block_1_bit_split
(int bit_low, int bit_count,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);

template <int BLOCK_LG, int RADIX_LG>
__device__ void radix_sort_1_pass_1_tile
(int digit_pos, int tile_idx, bool first_iter,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);

template <int BLOCK_LG, int RADIX_LG> __global__ void
radix_sort_1_pass_1(int digit_pos, bool first_iter)
{
  __shared__ Pass_1_Stuff<BLOCK_LG,RADIX_LG> p1s;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);

  if ( threadIdx.x < radix ) p1s.ghisto[ threadIdx.x ] = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_1_pass_1_tile<BLOCK_LG,RADIX_LG>
      (digit_pos,tile_idx,first_iter,p1s);

  if ( threadIdx.x >= radix ) return;

  int histo_idx = blockIdx.x * radix + threadIdx.x;
  sort_histo[ histo_idx ] = p1s.ghisto[ threadIdx.x ];
}

template <int BLOCK_LG, int RADIX_LG> __device__ void
radix_sort_1_pass_1_tile
(int digit_pos, int tile_idx,
 bool first_iter, Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s)
{
  int start_bit = digit_pos * RADIX_LG;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  const int digit_mask = radix - 1;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;

  int idx_block_stop = min( dapp.array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  Sort_Elt *sort_src = first_iter ? sort_in : sort_out;

  // Load Element Keys
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    p1s.keys[sidx] = sort_src[ idx_block_start + sidx ];

  // Sort based upon current digit position
  //
  sort_block_1_bit_split<BLOCK_LG,RADIX_LG>(start_bit,RADIX_LG,p1s);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = p1s.keys[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( p1s.keys[sidx] >> start_bit ) & digit_mask;
      p1s.keys[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) p1s.keys[elt_per_tile] = radix;

  // Initialize histogram for this tile to zero.
  //
  if ( threadIdx.x < radix )
    p1s.thisto[ threadIdx.x ] = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];      // Our digit.
      int digit_1 = p1s.keys[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        p1s.runend[ digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];                      // Our digit.
      int digit_0 = sidx > 0 ? int(p1s.keys[sidx-1]) : -1; // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = p1s.runend[ digit ];
          int count = run_end_sidx - sidx + 1;
          p1s.ghisto[ digit ] += count;     // Histogram for block.
          p1s.thisto[ digit ] = count;        // Histogram for tile.
        }
    }

  __syncthreads();

  if ( threadIdx.x >= radix ) return;

  // Write out tile histogram.
  //
  int thisto_idx = tile_idx * radix + threadIdx.x;
  sort_tile_histo[ thisto_idx ] = p1s.thisto[ threadIdx.x ];
}

template <int block_lg, int RADIX_LG>
__device__ void
sort_block_1_bit_split
(int bit_low, int bit_count, Pass_1_Stuff<block_lg,RADIX_LG>& p1s)
{
  const int block_size = 1 << block_lg;
  const int elt_per_tile = elt_per_thread * block_size;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      Sort_Elt keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const Sort_Elt key = p1s.keys[ sidx ];
          keys[i] = key;
          if ( key & bit_mask ) my_ones_write++;
        }

      p1s.prefix[ threadIdx.x + 1 ] = my_ones_write;
      if ( threadIdx.x == 0 ) p1s.prefix[ 0 ] = 0;

      uint my_prefix = my_ones_write;

      // Compute a prefix sum of vectors.
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_neighbor = threadIdx.x - dist;
          __syncthreads();
          uint neighbor_prefix =
            threadIdx.x >= dist ? p1s.prefix[ idx_neighbor + 1 ] : 0;

          my_prefix += neighbor_prefix;
          __syncthreads();
          p1s.prefix[ threadIdx.x + 1 ] = my_prefix;
        }

      // At this point my_prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = p1s.prefix[ block_size ];
      const int idx_one_tid_0 = elt_per_tile - all_threads_num_ones;
      const int smaller_tids_num_ones = p1s.prefix[ threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          p1s.keys[ new_idx ] = key;
        }

    }
  __syncthreads();
}


extern __shared__ Sort_Elt s[];

__device__ void radix_sort_1_pass_2_tile
(int radix_lg, int digit_pos, int tile_idx, bool last_iter);

template <int RADIX_LG>
__global__ void
radix_sort_1_pass_2(int digit_pos, bool last_iter)
{
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  const int sort_radix = 1 << RADIX_LG;
  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_radix;
  int pfe_tile_sidx = pfe_tile_sbase + threadIdx.x;

  volatile int *sv = &((int*)s)[ 0 ];

  // Sum of all histogram bins for our digit value (threadIdx.x)
  //
  int global_bin_sum = 0;

  // Sum of histogram bins for our digit value for array indices
  // lower than ours.
  //
  int overhead_bin_sum = 0;

  if ( threadIdx.x == 0 ) sv[ pfe_tile_sbase ] = 0;

  if ( threadIdx.x < sort_radix )
    {
      // Code only correct for sort_radix <= 32.

      int idx_for_us = blockIdx.x * sort_radix + threadIdx.x;
      int pidx_stop = gridDim.x * sort_radix;

      // Compute Global Histogram
      //
      // Loop will set global_bin_sum to the total number of digits
      // of value threadIdx.x.
      //
      // Eg: Array:  123, 121, 373, 932, 863, 113
      //     Based on digit_pos, operating on least-significant digit.
      //     threadIdx.x = 3
      //     Then =>  global_bin_sum = 4
      //
      for ( int idx = threadIdx.x; idx < pidx_stop; idx += sort_radix )
        {
          if ( idx == idx_for_us ) overhead_bin_sum = global_bin_sum;
          global_bin_sum += sort_histo[idx];
        }

      // Compute Global Prefix Sum
      //
      //
      sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_sum;
      int global_bin_prefix = global_bin_sum;

      for ( int i=0; i< RADIX_LG; i++ )
        {
          int dist = 1 << i;
          int sum_0 = dist <= threadIdx.x
            ? sv[ pfi_tile_sbase + threadIdx.x - dist ] : 0;
          sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_prefix += sum_0;
        }

      // Initialize Tile Prefix Sum
      //
      sv[ pfe_tile_sidx ] += overhead_bin_sum;
    }

  __syncthreads();

  volatile int *tile_offsets = &((int*)s)[pf_offset_sbase_base];
  if ( threadIdx.x < sort_radix ) tile_offsets[threadIdx.x]=0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int count;
      if ( threadIdx.x < sort_radix )
        {
          int bo_idx = tile_idx * sort_radix + threadIdx.x;
          count = sort_tile_histo[ bo_idx ];
          int to_idx = sort_radix + threadIdx.x;
          int offset = count;
          tile_offsets[ to_idx ] = offset;
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 1 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 2 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 4 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 8 ];
          if ( debug_sort && last_iter )
            {
              scan_out[bo_idx] = sv[ pfe_tile_sidx ];
              scan_r2[bo_idx] = tile_offsets[ to_idx - 1];
            }
        }
      __syncthreads();
      radix_sort_1_pass_2_tile(RADIX_LG,digit_pos,tile_idx,last_iter);
      __syncthreads();
      if ( threadIdx.x < sort_radix ) sv[ pfe_tile_sidx ] += count;
    }
}

__device__ void
radix_sort_1_pass_2_tile
(int radix_lg, int digit_pos, int tile_idx, bool last_iter)
{
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int idx_tile_start = tile_idx * elt_per_tile;

  const int sort_radix = 1 << radix_lg;
  const int digit_mask = sort_radix - 1;
  int start_bit = digit_pos * radix_lg;

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_radix;
  int pf_offset_sbase = pf_offset_sbase_base + sort_radix - 1;

  for ( int i=0; i<elt_per_thread; i++ )
    {
      int tile_elt_rank = threadIdx.x + i * blockDim.x;
      int idx = idx_tile_start + tile_elt_rank;
      uint key = sort_out_b[idx];
      uint digit = ( key >> start_bit ) & digit_mask;
      int tile_digit_rank = s[ pf_offset_sbase + digit ];
      int key_digit_rank = tile_elt_rank - tile_digit_rank;
      int idx_digit_index = s[ pfe_tile_sbase + digit ] + key_digit_rank;

      if ( debug_sort && last_iter )
        sort_out[idx] = ( idx_digit_index << 12 ) + tile_digit_rank;
      else
        sort_out[idx_digit_index] = key;

    }
}
