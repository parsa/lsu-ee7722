#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include "radix-sort.cuh"
#include <gp/cuda-util-kernel.h>
#include <assert.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//

__constant__ Radix_Sort_GPU_Constants dapp;

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__constant__ Sort_Elt *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

template <int BLOCK_LG, int RADIX_LG>
__global__ void radix_sort_pass_1(int digit_pos, bool first_iter);
template <int BLOCK_LG, int RADIX_LG>
__global__ void radix_sort_pass_2(int digit_pos, bool last_iter);

__host__ void
kernels_get_attr(GPU_Info *gpu_info)
{
  CU_SYM(dapp);

  CU_SYM(sort_in); CU_SYM(sort_out); CU_SYM(sort_out_b);
  CU_SYM(sort_tile_histo);
  CU_SYM(sort_histo);

#define GETATTR(func) gpu_info->GET_INFO(func)

#define GAPAIR(block_lg,radix_lg)                                             \
  GETATTR((radix_sort_pass_1<block_lg,radix_lg>));                          \
  GETATTR((radix_sort_pass_2<block_lg,radix_lg>));

#define GASET(radix_lg)                                                       \
  GAPAIR(5,radix_lg); GAPAIR(6,radix_lg); GAPAIR(7,radix_lg);                 \
  GAPAIR(8,radix_lg); GAPAIR(9,radix_lg); GAPAIR(10,radix_lg);

  GASET(4);
  GASET(5);
  GASET(6);
  GASET(8);
#undef GETATTR
#undef GASET
#undef GAPAIR
}

typedef void (*KPtr)(int,bool);

// This routine executes on the CPU.
//
__host__ Kernel_Info*
sort_launch_pass_1
(GPU_Info *gpu_info, int dg, int db,
 int radix_lg, int digit_pos, bool first_iter)
{

#define LAUNCH_RD(BLG,RD_LG)                                                  \
  case 1<<BLG: kfunc = radix_sort_pass_1<BLG,RD_LG>; break;

#define LAUNCH_BLKS(RD_LG)                                                    \
    case RD_LG: switch ( db ){                                                \
      LAUNCH_RD(5,RD_LG); LAUNCH_RD(6,RD_LG); LAUNCH_RD(7,RD_LG);             \
      LAUNCH_RD(8,RD_LG); LAUNCH_RD(9,RD_LG); LAUNCH_RD(10,RD_LG);            \
    default: assert( false );                                                 \
    } break;

    KPtr kfunc = NULL;

    switch ( radix_lg ) {
      LAUNCH_BLKS(4);
      LAUNCH_BLKS(5);
      LAUNCH_BLKS(6);
      LAUNCH_BLKS(8);
    default: assert( false );
    }

# undef LAUNCH_RD
# undef LAUNCH_BLKS

    if ( dg == 0 ) return &gpu_info->get_info(GPU_Info_Func(kfunc));

    kfunc<<<dg,db>>>(digit_pos,first_iter);

    return NULL;
}

__host__ Kernel_Info*
sort_launch_pass_2
(GPU_Info *gpu_info, int dg, int db,
 int radix_lg, int digit_pos, bool last_iter)
{
#define LAUNCH_RD(BLG,RD_LG)                                                  \
  case 1<<BLG: kfunc = radix_sort_pass_2<BLG,RD_LG>; break;

#define LAUNCH_BLKS(RD_LG)                                                    \
  case RD_LG: switch ( db ){                                                  \
    LAUNCH_RD(5,RD_LG); LAUNCH_RD(6,RD_LG); LAUNCH_RD(7,RD_LG);               \
    LAUNCH_RD(8,RD_LG); LAUNCH_RD(9,RD_LG); LAUNCH_RD(10,RD_LG);              \
  default: assert( false );                                                   \
  } break;

  KPtr kfunc = NULL;

  switch ( radix_lg ) {
    LAUNCH_BLKS(4);
    LAUNCH_BLKS(5);
    LAUNCH_BLKS(6);
    LAUNCH_BLKS(8);
    default: assert( false );
  }

  if ( dg == 0 ) return &gpu_info->get_info(GPU_Info_Func(kfunc));

  kfunc<<<dg,db>>>(digit_pos,last_iter);

  return NULL;

# undef LAUNCH_RD
# undef LAUNCH_BLKS
}


#ifdef DEBUG_SORT
const int debug_sort = true;
#else
const int debug_sort = false;
#endif

template<int BLOCK_LG, int RADIX_LG>
struct Pass_1_Stuff
{
  Sort_Elt keys[elt_per_thread+(elt_per_thread<<BLOCK_LG)];
  int prefix[4 + (1<<BLOCK_LG)];
  int runend[1<<RADIX_LG];
  int thisto[1<<RADIX_LG];
  int ghisto[1<<RADIX_LG];
};

template <int BLOCK_LG, int RADIX_LG>
__device__ void
sort_block_1_bit_split
(int bit_low, int bit_count,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);

template <int BLOCK_LG, int RADIX_LG>
__device__ void radix_sort_pass_1_tile
(int digit_pos, int tile_idx, bool first_iter,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);

template <int BLOCK_LG, int RADIX_LG> __global__ void
radix_sort_pass_1(int digit_pos, bool first_iter)
{
  __shared__ Pass_1_Stuff<BLOCK_LG,RADIX_LG> p1s;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);

  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : radix/block_size;
  const bool rad_participant = block_size <= radix || threadIdx.x < radix;
# define DIG(i) (threadIdx.x + (i) * block_size)

  if ( rad_participant )
    for ( int i = 0;  i < dig_per_thd;  i++ )
      p1s.ghisto[ DIG(i) ] = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_pass_1_tile<BLOCK_LG,RADIX_LG>
      (digit_pos,tile_idx,first_iter,p1s);

  if ( !rad_participant ) return;

  for ( int i = 0;  i < dig_per_thd;  i++ )
    {
      const int histo_idx = blockIdx.x * radix + DIG(i);
      sort_histo[ histo_idx ] = p1s.ghisto[ DIG(i) ];
    }
}

template <int BLOCK_LG, int RADIX_LG> __device__ void
radix_sort_pass_1_tile
(int digit_pos, int tile_idx,
 bool first_iter, Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s)
{
  int start_bit = digit_pos * RADIX_LG;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  const int digit_mask = radix - 1;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;
  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : radix/block_size;
  const bool rad_participant = block_size <= radix || threadIdx.x < radix;

  int idx_block_stop = min( dapp.array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  Sort_Elt *sort_src = first_iter ? sort_in : sort_out;

  // Load Element Keys
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    {
      Sort_Elt key = sort_src[ idx_block_start + sidx ];
      assert( key );
      p1s.keys[sidx] = key;
    }

  // Sort based upon current digit position
  //
  sort_block_1_bit_split<BLOCK_LG,RADIX_LG>(start_bit,RADIX_LG,p1s);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = p1s.keys[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( p1s.keys[sidx] >> start_bit ) & digit_mask;
      p1s.keys[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) p1s.keys[elt_per_tile] = radix;

  // Initialize histogram for this tile to zero.
  //
  if ( rad_participant )
    for ( int i = 0;  i < dig_per_thd;  i++ )
      p1s.thisto[ DIG(i) ] = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];      // Our digit.
      int digit_1 = p1s.keys[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        p1s.runend[ digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];                      // Our digit.
      int digit_0 = sidx > 0 ? int(p1s.keys[sidx-1]) : -1; // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = p1s.runend[ digit ];
          int count = run_end_sidx - sidx + 1;
          p1s.ghisto[ digit ] += count;     // Histogram for block.
          p1s.thisto[ digit ] = count;        // Histogram for tile.
        }
    }

  __syncthreads();

  if ( !rad_participant ) return;

  for ( int i = 0;  i < dig_per_thd;  i++ )
    {
      const int thisto_idx = tile_idx * radix + DIG(i);
      sort_tile_histo[ thisto_idx ] = p1s.thisto[ DIG(i) ];
    }
}

template <int block_lg, int RADIX_LG>
__device__ void
sort_block_1_bit_split
(int bit_low, int bit_count, Pass_1_Stuff<block_lg,RADIX_LG>& p1s)
{
  const int block_size = 1 << block_lg;
  const int elt_per_tile = elt_per_thread * block_size;

  if ( threadIdx.x == 0 ) p1s.prefix[0] = 0;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      Sort_Elt keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      const bool use_pop = false;

      const int wp_lg = 5;
      const int wp_sz = 1 << wp_lg;
      const int wp_mk = wp_sz - 1;
      const int lane = threadIdx.x & wp_mk;
      const int wp_idx = threadIdx.x >> wp_lg;
      const uint32_t msk = 0xffffffff;
      int my_pf_wp = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const Sort_Elt key = p1s.keys[ sidx ];
          keys[i] = key;
          const bool one = key & bit_mask;
          if ( one ) my_ones_write++;
          if ( !use_pop ) continue;

          // Compute intra-warp prefix sum for one set of 32 keys.

          // Get vector showing which lanes have a 1.
          //
          const uint32_t have_work_wp_v = __ballot_sync(msk,one);

          // Shift off bits corresponding to higher-numbered lanes.
          //
          const uint32_t have_work_pf_v = have_work_wp_v << ( 31 - lane );

          // Use popc (population count, which is number of bits = 1)
          // to compute prefix.
          //
          const uint32_t my_pf_wp_i = __popc(have_work_pf_v);

          my_pf_wp += my_pf_wp_i;
        }

      if ( !use_pop )
        {
          my_pf_wp = my_ones_write;

          // Compute intra-warp prefix sum. (Sum within warp.)
          //
          for ( int tree_level = 0; tree_level < wp_lg; tree_level++ )
            {
              int dist = 1 << tree_level;
              uint neighbor_prefix = __shfl_up_sync(msk,my_pf_wp,dist);
              if ( dist <= lane ) my_pf_wp += neighbor_prefix;
            }
        }

      // Write total number of 1's in warp to shared memory. This
      // will be used to compute prefix sum between warps.
      //
      if ( lane == wp_mk ) p1s.prefix[wp_idx+1] = my_pf_wp;

      __syncthreads();

      // Compute inter-warp prefix sum.  Only warp 0 does this.
      //
      if ( wp_idx == 0 )
        {
          uint wp_prefix = p1s.prefix[threadIdx.x+1];
          for ( int tree_level = 0; tree_level < block_lg - wp_lg;
                tree_level++ )
            {
              int dist = 1 << tree_level;
              uint neighbor_prefix = __shfl_up_sync(msk,wp_prefix,dist);
              if ( dist <= threadIdx.x ) wp_prefix += neighbor_prefix;
            }
          p1s.prefix[threadIdx.x+1] = wp_prefix;
        }
      __syncthreads();
      const uint wp_prefix = p1s.prefix[wp_idx];
      __syncthreads();

      // Combine inter-warp prefix (wp_prefix) with intra-warp prefix
      // (my_pf_wp) to get prefix sum within block.
      //
      p1s.prefix[threadIdx.x+1] = wp_prefix + my_pf_wp;

      // At this point p1s.prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = p1s.prefix[ block_size ];
      const int idx_one_tid_0 = elt_per_tile - all_threads_num_ones;
      const int smaller_tids_num_ones = p1s.prefix[ threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          p1s.keys[ new_idx ] = key;
        }

    }
  __syncthreads();
}


template <int BLOCK_LG, int RADIX_LG>
__global__ void
radix_sort_pass_2(int digit_pos, bool last_iter)
{
  const int block_size = 1 << BLOCK_LG;
  int elt_per_tile = elt_per_thread * block_size;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  const int sort_radix = 1 << RADIX_LG;
  const int digit_mask = sort_radix - 1;
  const int start_bit = digit_pos * RADIX_LG;

  volatile __shared__ int g_prefix[ sort_radix + 1 ];

  if ( threadIdx.x == 0 ) g_prefix[ 0 ] = 0;

  const int warp_sz = 32;
  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : sort_radix/block_size;
  const bool rad_participant =
    block_size <= sort_radix || threadIdx.x < sort_radix;
  const bool rad_attendee = rad_participant || sort_radix > warp_sz;
  volatile __shared__ int tile_offsets[ 2 * sort_radix ];

  if ( rad_attendee )
    {
      int overhead_bin_sum[dig_per_thd];
      int global_bin_sum[dig_per_thd];
      int global_bin_prefix[dig_per_thd];

      for ( auto& e: global_bin_sum ) e = 0;

      if ( rad_participant )
        {
          for ( int gh_idx = 0; gh_idx < gridDim.x; gh_idx++ )
            {
              for ( int i = 0;  i < dig_per_thd;  i++ )
                {
                  const int d = threadIdx.x + i * block_size;
                  const int gh_bin_idx = gh_idx * sort_radix + d;
                  if ( gh_idx == blockIdx.x )
                    overhead_bin_sum[i] = global_bin_sum[i];
                  global_bin_sum[i] += sort_histo[gh_bin_idx];
                }
            }

          //
          // Compute Global Prefix Sum
          //

          for ( int i = 0;  i < dig_per_thd;  i++ )
            {
              const int d = threadIdx.x + i * block_size;
              g_prefix[ 1 + d ] = global_bin_sum[i];
            }
          //
          // At this point g_prefix holds a global histogram.

          for ( int i=0; i<dig_per_thd; i++ )
            global_bin_prefix[i] = global_bin_sum[i];
        }

      for ( int lev=0; lev<RADIX_LG; lev++ )
        {
          const int dist = 1 << lev;
          int sum_0[dig_per_thd];

          if ( sort_radix > warp_sz ) __syncthreads();
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                sum_0[i] = dist <= d ? g_prefix[ 1 + d - dist ] : 0;
              }
          if ( sort_radix > warp_sz ) __syncthreads();
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                g_prefix[ 1 + d ] = global_bin_prefix[i] += sum_0[i];
              }
        }


      // Now, g_prefix holds a global prefix sum.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in the entire array is to be written. That key
      // is probably being handled by block 0.

      if ( sort_radix > warp_sz ) __syncthreads();
      if ( rad_participant )
        for ( int i = 0;  i < dig_per_thd;  i++ )
          {
            const int d = threadIdx.x + i * block_size;
            g_prefix[ d ] += overhead_bin_sum[i];
          }

      //
      // Now, g_prefix holds a prefix sum for this block.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in this block is to be written.

    }

  __syncthreads();

  if ( rad_participant )
    for ( int i=0;  i<dig_per_thd;  i++ )
      {
        const int d = threadIdx.x + i * block_size;
        tile_offsets[d] = 0;
      }

  __syncthreads();

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int counts[dig_per_thd];

      if ( rad_attendee )
        {
          int offsets[dig_per_thd];
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                const int bo_idx = tile_idx * sort_radix + d;
                const int to_idx = sort_radix + d;
                offsets[i] = counts[i] = tile_offsets[ to_idx ] =
                  sort_tile_histo[ bo_idx ];
              }

          for ( int lev=0; lev<RADIX_LG; lev++ )
            {
              const int dist = 1 << lev;
              if ( sort_radix > warp_sz ) __syncthreads();
              if ( rad_participant )
                for ( int i = 0;  i < dig_per_thd;  i++ )
                  {
                    const int d = threadIdx.x + i * block_size;
                    int to_idx = sort_radix + d;
                    offsets[i] += tile_offsets[ to_idx - dist ];
                  }
              if ( sort_radix > warp_sz ) __syncthreads();
              if ( rad_participant )
                for ( int i = 0;  i < dig_per_thd;  i++ )
                  {
                    const int d = threadIdx.x + i * block_size;
                    int to_idx = sort_radix + d;
                    tile_offsets[ to_idx ] = offsets[i];
                  }
            }
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                assert( counts[i] <= elt_per_tile );
                assert( tile_offsets[sort_radix + d - 1] <= elt_per_tile );
              }
        }

      __syncthreads();

      int idx_tile_start = tile_idx * elt_per_tile;

      for ( int i=0; i<elt_per_thread; i++ )
        {
          int tile_elt_rank = threadIdx.x + i * block_size;
          int idx = idx_tile_start + tile_elt_rank;
          Sort_Elt key = sort_out_b[idx];
          uint digit = ( key >> start_bit ) & digit_mask;
          int tile_digit_rank = tile_offsets[ sort_radix + digit - 1 ];
          int key_digit_rank = tile_elt_rank - tile_digit_rank;
          assert( key_digit_rank >= 0 );
          int idx_digit_index = g_prefix[ digit ] + key_digit_rank;

          if ( debug_sort && last_iter )
            sort_out[idx] = ( idx_digit_index << 12 ) + tile_digit_rank;
          else
            sort_out[idx_digit_index] = key;

        }

      __syncthreads();

      if ( rad_participant )
        for ( int i = 0;  i < dig_per_thd;  i++ )
          {
            const int d = threadIdx.x + i * block_size;
            g_prefix[ d ] += counts[i];
          }

      //
      // Now, g_prefix holds a prefix sum for the next tile.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in the next tile is to be written.
    }
}
