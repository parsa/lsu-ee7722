#include "hip/hip_runtime.h"
#include "sort.cuh"

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ int block_lg;
__constant__ int array_size, array_size_lg;
__constant__ int *scan_in, *scan_out;
__constant__ int *scan_r2;

extern __shared__ int s[];

__constant__ int *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

__constant__ int sort_bin_mask;
__constant__ int sort_bin_size, sort_bin_count;
__constant__ int sort_all_bin_count, sort_all_bin_lg;
__constant__ int sort_bin_lg;

__global__ void sort_segments_1_bit_split();
__global__ void sort_block_batcher();
__global__ void sort_block_batcher_1();
template <int BLOCK_LG> __global__ void sort_block_batcher_opt();
template <int BLOCK_LG, int BIN_LG>
__global__ void radix_sort_1_pass_1(int bin_idx, bool first_iter);
__global__ void radix_sort_1_pass_2(int bin_idx, bool last_iter);

static __host__ int
kernels_get_attr_(pCUDA_Func_Attributes *attr)
{
  int count = 0;

#define GETATTR(func)                                                         \
  count++;                                                                    \
  if ( attr ) {                                                               \
      attr->err = hipFuncGetAttributes(&attr->attr,reinterpret_cast<const void*>(func));                    \
      attr->name = #func;                                                     \
      attr++;                                                                 \
    }
  GETATTR(sort_segments_1_bit_split);
  GETATTR(sort_block_batcher);
  GETATTR(sort_block_batcher_1);
  GETATTR(sort_block_batcher_opt<8>);
  GETATTR((radix_sort_1_pass_1<6,4>));
  GETATTR(radix_sort_1_pass_2);
  return count;
#undef GETATTR
}

__host__ int
kernels_get_attr(pCUDA_Func_Attributes **attr)
{
  int count = kernels_get_attr_(NULL);
  *attr = (pCUDA_Func_Attributes*) calloc(count,sizeof(**attr));
  return kernels_get_attr_(*attr);
}

// This routine executes on the CPU.
//
__host__ void
sort_launch(dim3 dg, dim3 db, int version, int array_size, int array_size_lg)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  switch ( version ){
  case 0:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4 + 2;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_segments_1_bit_split<<<dg,db,shared_size>>>();
    }
    break;

  case 1:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_block_batcher<<<dg,db,shared_size>>>();
    }
    break;

  case 2:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_block_batcher_1<<<dg,db,shared_size>>>();
    }
    break;

  case 3:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      switch ( db.x ) {
      case 64: sort_block_batcher_opt<6><<<dg,db,shared_size>>>(); break;
      case 128: sort_block_batcher_opt<7><<<dg,db,shared_size>>>(); break;
      case 256: sort_block_batcher_opt<8><<<dg,db,shared_size>>>(); break;
      default:break;
      }
    }
    break;

  case 4:
    {
      const int bin_lg = 4;
      const int bin_size = 1 << bin_lg;
      int elt_per_thread = 4;
      int size_per_elt_1 = 4 + 2; // Assuming sort_bin_size < block_size
      int shared_size_pass_1 = db.x * size_per_elt_1 * elt_per_thread
        + 4 * bin_size * 4;
      int shared_size_pass_2 = ( 3 * bin_size + 1 ) * 4;
      int bins = int( ceil( 32.0 / bin_lg ) );
      int bin_idx_start = 0;
      int bin_idx_stop = bins;
      for ( int bin_idx = bin_idx_start; bin_idx < bin_idx_stop; bin_idx++ )
        {
          const bool first_iter = bin_idx == bin_idx_start;
          switch(db.x){
          case 64:
            radix_sort_1_pass_1<6,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          case 128:
            radix_sort_1_pass_1
              <7,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          case 256:
            radix_sort_1_pass_1
              <8,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          }
          radix_sort_1_pass_2<<<dg,db,shared_size_pass_2>>>
            (bin_idx,bin_idx+1==bin_idx_stop);
        }
    }
    break;

  default:
    break;
  }
}

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__device__ int div_ceil(int a, int b){return int( ceilf( ((float)a)/b ) );}

#ifdef DEBUG_SORT
const int debug_sort = true;
#else
const int debug_sort = false;
#endif


__device__ void sort_block_1_bit_split(int bit_low, int bit_count);

__global__ void
sort_segments_1_bit_split()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_block_stop = idx_block_start + elt_per_block;

  int idx_start = idx_block_start + threadIdx.x;
  int key_base_rd = 0;

  // Load Elements
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  sort_block_1_bit_split(0,32);

  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += blockDim.x, idx += blockDim.x )
        sort_out[idx] = s[ key_base_rd + sidx ];
}

__device__ void
sort_block_1_bit_split(int bit_low, int bit_count)
{
  // Number of elements operated on per thread.
  //
  const int elt_per_thread = 4;

  int elt_per_block = elt_per_thread * blockDim.x;

  // Indices into shared memory for prefix sum.
  // pfe: Exclusive prefix. (Sum of smaller element values.)
  // pfi: Inclusive prefix. (Sum of this element and smaller element values.)
  //
  int pfe_base_rd = elt_per_block;
  int pfi_base_rd = elt_per_block + 1;

  volatile __shared__ int col_total[16];
  if ( threadIdx.x < 16 )
    {
      s[ pfe_base_rd ] = 0;
      col_total[threadIdx.x] = 0;
    }

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      int bit_mask = 1 << bit_pos;
      int prefix_vector = 0;

      // Storage for thread's keys.
      //
      int keys[elt_per_thread];

      /// Strategy:
      //
      //  Pack "bit" from four elements' keys into one integer, the
      //  prefix vector, saving memory and time.
      //
      //  For example:
      //   Bit position: LSB
      //   Keys:            3,  4,  5,  7
      //   Bits:            1,  0,  1,  1
      //   Prefix Vector:   0x01000101

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      for ( int sidx = threadIdx.x, i = 0;
            i < elt_per_thread; i++, sidx += blockDim.x )
        {
          int positioned_bit = 1 << ( i * 8 );
          keys[i] = s[ sidx ];
          if ( keys[i] & bit_mask ) prefix_vector += positioned_bit;
        }

      // Store prefix vector for our for elements into shared memory.
      //
      s[ pfi_base_rd + threadIdx.x ] = prefix_vector;
      __syncthreads();

      // Operate on a neighbor's vector.
      //
      uint pfv_1 = s[ pfe_base_rd + threadIdx.x ];


      //
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_0 = threadIdx.x - dist;
          __syncthreads();
          uint pfv_0 = threadIdx.x >= dist ? s[ pfe_base_rd + idx_0 ] : 0;
          pfv_1 += pfv_0;
          __syncthreads();
          s[ pfe_base_rd + threadIdx.x ] = pfv_1;
        }

      // At this point pfv_1 contains exclusive prefix of each column.

      __syncthreads();

      const int ct_wbase = 8;
      if ( threadIdx.x < 8 )
        {
          int pfv = s[pfe_base_rd + blockDim.x - ( threadIdx.x & 0x1 ) ];
          int shift = ( threadIdx.x & 0x6 ) << 2;
          int my_val = ( pfv >> shift ) & 0xff;
          int sidx = ct_wbase + threadIdx.x;
          col_total[ sidx ] = my_val;
          col_total[ sidx ] = my_val += col_total[ sidx - 1 ] ;
          col_total[ sidx ] = my_val += col_total[ sidx - 2 ] ;
          col_total[ sidx ] = my_val += col_total[ sidx - 4 ] ;
        }

      __syncthreads();
      int ct_base = ct_wbase - 1;

#if 0
      if ( threadIdx.x == 0 && blockIdx.x == 0 )
        {
          for ( int i=0; i<5; i++ )
            scan_out[(bit_pos-bit_low)*16+i] = col_total[ct_base + 2*i];
        }
#endif

       int total_ones = col_total[ct_base+8];

      for ( int sidx = threadIdx.x, i = 0;
            i < elt_per_thread; i++, sidx += blockDim.x )
        {
          int count = ( pfv_1 & 0xff ) + col_total[ct_base+2*i];
          int key = keys[i];
          int new_idx = key & bit_mask
            ? elt_per_block - total_ones + count
            : sidx - count;
          s[ new_idx ] = key;
          pfv_1 >>= 8;
        }
    }
}

__global__ void
sort_block_batcher()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i++ )
            {
              int idx_0 = threadIdx.x + i * blockDim.x;
              int idx_1 = idx_0 | bit_vector;
              if ( idx_0 == idx_1 ) continue;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }

    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

__global__ void
sort_block_batcher_1()
{
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          uint bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * blockDim.x;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( ( key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

template <int BLOCK_LG>
__global__ void
sort_block_batcher_opt()
{
  int block_size = 1 << BLOCK_LG;
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * block_size;
  int chunk_lg = 2 + BLOCK_LG;
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<BLOCK_LG; m_lg++ )
    {
      int sort_dir_vector = 1 << m_lg;
      bool sort_dir = threadIdx.x & sort_dir_vector;
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          int idx_0_t = threadIdx.x + ( threadIdx.x & shift_mask );
          int idx_1_t = idx_0_t + bit_vector;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i += 2 )
            {
              int idx_0 = idx_0_t + ( i << BLOCK_LG );
              int idx_1 = idx_1_t + ( i << BLOCK_LG );
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }
  for ( int m_lg=BLOCK_LG; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * block_size;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    sort_out[ idx_block_start + sidx ] = s[sidx];

}

template <int BLOCK_LG, int BIN_LG>
__device__ void radix_sort_1_pass_1_tile
(int bin_idx, int tile_idx, bool first_iter);

template <int BLOCK_LG, int BIN_LG> __global__ void
radix_sort_1_pass_1(int bin_idx, bool first_iter)
{
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);
  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;

  if ( threadIdx.x < bin_size ) s[ ghisto_sbase + threadIdx.x ] = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_1_pass_1_tile<BLOCK_LG,BIN_LG>(bin_idx,tile_idx,first_iter);

  if ( threadIdx.x >= bin_size ) return;
  int histo_idx = blockIdx.x * bin_size + threadIdx.x;
  sort_histo[ histo_idx ] = s[ ghisto_sbase + threadIdx.x ];
}

template <int BLOCK_LG, int BIN_LG> __device__ void
radix_sort_1_pass_1_tile(int bin_idx, int tile_idx, bool first_iter)
{
  int start_bit = bin_idx * sort_bin_lg;
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;

  int idx_block_stop = min( array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;
  int runend_sbase = ghisto_sbase + bin_size;
  int thisto_sbase = runend_sbase + bin_size;

  int *sort_src = first_iter ? sort_in : sort_out;

  // Load Elements
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_src[ idx_block_start + sidx ];

  // Sort based upon current bin (digit position)
  //
  sort_block_1_bit_split(start_bit,sort_bin_lg);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = s[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( s[sidx] >> start_bit ) & sort_bin_mask;
      s[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) s[elt_per_tile] = bin_size;

  if ( threadIdx.x < bin_size )
    s[ thisto_sbase + threadIdx.x ] = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];      // Our digit.
      int digit_1 = s[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        s[ runend_sbase + digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];                        // Our digit.
      int digit_0 = sidx > 0 ? s[sidx-1] : -1;    // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = s[ runend_sbase + digit ];
          int count = run_end_sidx - sidx + 1;
          s[ ghisto_sbase + digit ] += count;     // Histogram for block.
          s[ thisto_sbase + digit ] = count;      // Histogram for tile.
        }
    }

  __syncthreads();

  if ( threadIdx.x >= bin_size ) return;

  // Write out tile histogram.
  //
  int thisto_idx = tile_idx * bin_size + threadIdx.x;
  sort_tile_histo[ thisto_idx ] = s[ thisto_sbase + threadIdx.x ];
}

__device__ void radix_sort_1_pass_2_tile
(int bin_idx, int tile_idx, bool last_iter);

__device__ void
radix_sort_1_pass_2(int bin_idx, bool last_iter)
{
  int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_bin_size;
  int pfe_tile_sidx = pfe_tile_sbase + threadIdx.x;

  volatile int *sv = &s[ 0 ]; // Volatile access.

  // Sum of all histogram bins for our digit value (threadIdx.x)
  //
  int global_bin_sum = 0;

  // Sum of histogram bins for our digit value for array indices
  // lower than ours.
  //
  int overhead_bin_sum = 0;

  if ( threadIdx.x == 0 ) sv[ pfe_tile_sbase ] = 0;

  if ( threadIdx.x < sort_bin_size )
    {
      // Code only correct for sort_bin_size <= 32.

      int idx_for_us = blockIdx.x * sort_bin_size + threadIdx.x;
      int pidx_stop = gridDim.x * sort_bin_size;

      // Compute Global Histogram
      //
      // Loop will set global_bin_sum to the total number of digits
      // of value threadIdx.x.
      //
      // Eg: Array:  123, 121, 373, 932, 863, 113
      //     Based on bin_idx, operating on least-significant digit.
      //     threadIdx.x = 3
      //     Then =>  global_bin_sum = 4
      //
      for ( int idx = threadIdx.x; idx < pidx_stop; idx += sort_bin_size )
        {
          if ( idx == idx_for_us ) overhead_bin_sum = global_bin_sum;
          global_bin_sum += sort_histo[idx];
        }
      
      // Compute Global Prefix Sum
      //
      //
      sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_sum;
      int global_bin_prefix = global_bin_sum;

      for ( int i=0; i<sort_bin_lg; i++ )
        {
          int dist = 1 << i;
          int sum_0 = dist <= threadIdx.x
            ? sv[ pfi_tile_sbase + threadIdx.x - dist ] : 0;
          sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_prefix += sum_0;
        }

      // Initialize Tile Prefix Sum
      //
      sv[ pfe_tile_sidx ] += overhead_bin_sum;
    }

  __syncthreads();

  volatile int *tile_offsets = &s[pf_offset_sbase_base];
  if ( threadIdx.x < sort_bin_size ) tile_offsets[threadIdx.x]=0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int count;
      if ( threadIdx.x < sort_bin_size )
        {
          int bo_idx = tile_idx * sort_bin_size + threadIdx.x;
          count = sort_tile_histo[ bo_idx ];
          int to_idx = sort_bin_size + threadIdx.x;
          int offset = count;
          tile_offsets[ to_idx ] = offset;
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 1 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 2 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 4 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 8 ];
          if ( debug_sort && last_iter )
            {
              scan_out[bo_idx] = sv[ pfe_tile_sidx ];
              scan_r2[bo_idx] = tile_offsets[ to_idx - 1];
            }
        }
      __syncthreads();
      radix_sort_1_pass_2_tile(bin_idx,tile_idx,last_iter);
      __syncthreads();
      if ( threadIdx.x < sort_bin_size ) sv[ pfe_tile_sidx ] += count;
    }
}

__device__ void
radix_sort_1_pass_2_tile
(int bin_idx, int tile_idx, bool last_iter)
{
  const int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int idx_tile_start = tile_idx * elt_per_tile;

  int start_bit = bin_idx * sort_bin_lg;

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_bin_size;
  int pf_offset_sbase = pf_offset_sbase_base + sort_bin_size - 1;

  for ( int i=0; i<elt_per_thread; i++ )
    {
      int local_idx = threadIdx.x + i * blockDim.x;
      int idx = idx_tile_start + local_idx;
      uint key = sort_out_b[idx];
      uint digit = ( key >> start_bit ) & sort_bin_mask;
      int local_offset = s[ pf_offset_sbase + digit ];
      int key_digit_rank = local_idx - local_offset;
      int idx_digit_index = s[ pfe_tile_sbase + digit ] + key_digit_rank;

      if ( debug_sort && last_iter )
        sort_out[idx] = ( idx_digit_index << 12 ) + local_offset;
      else
        sort_out[idx_digit_index] = key;

    }
}
