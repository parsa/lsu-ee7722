#include "hip/hip_runtime.h"
#include "sort.cuh"
#include <gp/cuda-util-kernel.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ int block_lg;
__constant__ int array_size, array_size_lg;
__constant__ int *scan_in, *scan_out;
__constant__ int *scan_r2;

extern __shared__ int s[];

__constant__ int *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

__constant__ int sort_bin_mask;
__constant__ int sort_bin_size, sort_bin_count;
__constant__ int sort_all_bin_count, sort_all_bin_lg;
__constant__ int sort_bin_lg;

__global__ void sort_segments_1_bit_split();
__global__ void sort_segments_1_bit_split_opt();
__global__ void sort_block_batcher();
__global__ void sort_block_batcher_1();
template <int BLOCK_LG> __global__ void sort_block_batcher_opt();
template <int BLOCK_LG, int BIN_LG>
__global__ void radix_sort_1_pass_1(int bin_idx, bool first_iter);
__global__ void radix_sort_1_pass_2(int bin_idx, bool last_iter);

__host__ void
kernels_get_attr(GPU_Info *gpu_info)
{
  CU_SYM(block_lg);
  CU_SYM(array_size); CU_SYM(array_size_lg);
  CU_SYM(scan_in); CU_SYM(scan_out);
  CU_SYM(scan_r2);

  CU_SYM(sort_in); CU_SYM(sort_out); CU_SYM(sort_out_b);
  CU_SYM(sort_tile_histo);
  CU_SYM(sort_histo);

  CU_SYM(sort_bin_mask);
  CU_SYM(sort_bin_size); CU_SYM(sort_bin_count);
  CU_SYM(sort_all_bin_count); CU_SYM(sort_all_bin_lg);
  CU_SYM(sort_bin_lg);

#define GETATTR(func) gpu_info->GET_INFO(func)
  GETATTR(sort_segments_1_bit_split);
  GETATTR(sort_segments_1_bit_split_opt);
  GETATTR(sort_block_batcher);
  GETATTR(sort_block_batcher_1);
  GETATTR(sort_block_batcher_opt<6>);
  GETATTR(sort_block_batcher_opt<8>);
  GETATTR(sort_block_batcher_opt<10>);
  GETATTR((radix_sort_1_pass_1<6,4>));
  GETATTR(radix_sort_1_pass_2);
#undef GETATTR
}


// This routine executes on the CPU.
//
__host__ int
sort_launch(dim3 dg, dim3 db, int version, int array_size, int array_size_lg)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  switch ( version ){
  case 0: case 1:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4 + 2;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      if ( !dg.x ) return shared_size;
      if ( version == 0 )
        sort_segments_1_bit_split<<<dg,db,shared_size>>>();
      else
        sort_segments_1_bit_split_opt<<<dg,db,shared_size>>>();
    }
    break;

  case 2:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      if ( !dg.x ) return shared_size;
      sort_block_batcher<<<dg,db,shared_size>>>();
    }
    break;

  case 3:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      if ( !dg.x ) return shared_size;
      sort_block_batcher_1<<<dg,db,shared_size>>>();
    }
    break;

  case 4:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      if ( !dg.x ) return shared_size;
      switch ( db.x ) {
      case 64: sort_block_batcher_opt<6><<<dg,db,shared_size>>>(); break;
      case 128: sort_block_batcher_opt<7><<<dg,db,shared_size>>>(); break;
      case 256: sort_block_batcher_opt<8><<<dg,db,shared_size>>>(); break;
      case 1024: sort_block_batcher_opt<10><<<dg,db,shared_size>>>(); break;
      default:break;
      }
    }
    break;

  case 5:
    {
      const int bin_lg = 4;
      const int bin_size = 1 << bin_lg;
      int elt_per_thread = 4;
      int size_per_elt_1 = 4 + 2; // Assuming sort_bin_size < block_size
      int shared_size_pass_1 = db.x * size_per_elt_1 * elt_per_thread
        + 4 * bin_size * 4;
      int shared_size_pass_2 = ( 3 * bin_size + 1 ) * 4;
      int bins = int( ceil( 32.0 / bin_lg ) );
      int bin_idx_start = 0;
      int bin_idx_stop = bins;
      if ( !dg.x ) return shared_size_pass_1;
      for ( int bin_idx = bin_idx_start; bin_idx < bin_idx_stop; bin_idx++ )
        {
          const bool first_iter = bin_idx == bin_idx_start;
          switch(db.x){
          case 64:
            radix_sort_1_pass_1<6,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          case 128:
            radix_sort_1_pass_1
              <7,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          case 256:
            radix_sort_1_pass_1
              <8,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx,first_iter);
          break;
          }
          radix_sort_1_pass_2<<<dg,db,shared_size_pass_2>>>
            (bin_idx,bin_idx+1==bin_idx_stop);
        }
    }
    break;

  default:
    assert( false );
    break;
  }
  return 0;
}

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__device__ int div_ceil(int a, int b){return (a+b-1)/b;}

#ifdef DEBUG_SORT
const int debug_sort = true;
#else
const int debug_sort = false;
#endif


__device__ void sort_block_1_bit_split(int bit_low, int bit_count);
__device__ void sort_block_1_bit_split_opt(int bit_low, int bit_count);

template<bool method_opt>
__device__ void
sort_segments_1_bit_split_method()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;

  int idx_start = idx_block_start + threadIdx.x;

  // Load Elements
  //
  for ( int i = 0;  i < elt_per_thread; i++ )
    {
      const int sidx = threadIdx.x + i * blockDim.x;
      s[ sidx ] = sort_in[ idx_block_start + sidx ];
    }

  if ( method_opt )
    sort_block_1_bit_split_opt(0,32);
  else
    sort_block_1_bit_split(0,32);

  for ( int i = 0;  i < elt_per_thread; i++ )
    {
      const int sidx = threadIdx.x + i * blockDim.x;
      const int idx = idx_start + i * blockDim.x;
      sort_out[idx] = s[ sidx ];
    }
}

__global__ void
sort_segments_1_bit_split()
{ sort_segments_1_bit_split_method<false>(); }
__global__ void
sort_segments_1_bit_split_opt()
{ sort_segments_1_bit_split_method<true>(); }



__device__ void
sort_block_1_bit_split(int bit_low, int bit_count)
{
  // Number of elements operated on per thread.
  //
  const int elt_per_thread = 4;

  int elt_per_block = elt_per_thread * blockDim.x;

  // Indices into shared memory for prefix sum.
  // pfe: Exclusive prefix. (Sum of smaller element values.)
  // pfi: Inclusive prefix. (Sum of this element and smaller element values.)
  //
  int pfe_base_rd = elt_per_block;
  int pfi_base_rd = elt_per_block + 1;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      int keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          //  const int sidx = threadIdx.x + i * blockDim.x;
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const int key = s[ sidx ];
          keys[i] = key;
          if ( key & bit_mask ) my_ones_write++;
        }

      s[ pfi_base_rd + threadIdx.x ] = my_ones_write;
      if ( threadIdx.x == 0 ) s[ pfe_base_rd ] = 0;

      uint my_prefix = my_ones_write;

      // Compute a prefix sum of vectors.
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_neighbor = threadIdx.x - dist;
          __syncthreads();
          uint neighbor_prefix =
            threadIdx.x >= dist ? s[ pfi_base_rd + idx_neighbor ] : 0;
          
          my_prefix += neighbor_prefix;
          __syncthreads();
          s[ pfi_base_rd + threadIdx.x ] = my_prefix;
        }

      // At this point my_prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = s[ pfe_base_rd + blockDim.x ];
      const int idx_one_tid_0 = elt_per_block - all_threads_num_ones;
      const int smaller_tids_num_ones = s[ pfe_base_rd + threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          s[ new_idx ] = key;
        }

    }
  __syncthreads();
}

__device__ void
sort_block_1_bit_split_opt(int bit_low, int bit_count)
{
  const int block_size = blockDim.x;

  // Number of elements operated on per thread.
  //
  const int elt_per_thread = 4;

  int elt_per_block = elt_per_thread * block_size;

  int* const prefix = &s[ elt_per_block ];

  if ( threadIdx.x == 0 ) prefix[ 0 ] = 0;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      int keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      const bool use_pop = true;

      const int wp_lg = 5;
      const int wp_sz = 1 << wp_lg;
      const int wp_mk = wp_sz - 1;
      const int lane = threadIdx.x & wp_mk;
      const int wp_idx = threadIdx.x >> wp_lg;
      const uint32_t msk = 0xffffffff;
      int my_pf_wp = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const int key = s[ sidx ];
          keys[i] = key;
          const bool one = key & bit_mask;
          if ( one ) my_ones_write++;
          if ( !use_pop ) continue;

          // Compute intra-warp prefix sum for one set of 32 keys.

          // Get vector showing which lanes have a 1.
          //
          const uint32_t have_work_wp_v = __ballot_sync(msk,one);

          // Shift off bits corresponding to higher-numbered lanes.
          //
          const uint32_t have_work_pf_v = have_work_wp_v << ( 31 - lane );

          // Use popc (population count, which is number of bits = 1)
          // to compute prefix.
          //
          const uint32_t my_pf_wp_i = __popc(have_work_pf_v);

          my_pf_wp += my_pf_wp_i;
        }

      if ( !use_pop )
        {
          my_pf_wp = my_ones_write;

          // Compute intra-warp prefix sum. (Sum within warp.)
          //
          for ( int tree_level = 0; tree_level < wp_lg; tree_level++ )
            {
              int dist = 1 << tree_level;
              uint neighbor_prefix = __shfl_up_sync(msk,my_pf_wp,dist);
              if ( dist <= lane ) my_pf_wp += neighbor_prefix;
            }
        }

      // Write total number of 1's in warp to shared memory. This
      // will be used to compute prefix sum between warps.
      //
      if ( lane == wp_mk ) prefix[wp_idx+1] = my_pf_wp;

      __syncthreads();

      // Compute inter-warp prefix sum.  Only warp 0 does this.
      //
      if ( wp_idx == 0 )
        {
          uint wp_prefix = prefix[threadIdx.x+1];
          for ( int tree_level = 0; tree_level < block_lg - wp_lg;
                tree_level++ )
            {
              int dist = 1 << tree_level;
              uint neighbor_prefix = __shfl_up_sync(msk,wp_prefix,dist);
              if ( dist <= threadIdx.x ) wp_prefix += neighbor_prefix;
            }
          prefix[threadIdx.x+1] = wp_prefix;
        }
      __syncthreads();
      const uint wp_prefix = prefix[wp_idx];
      __syncthreads();

      // Combine inter-warp prefix (wp_prefix) with intra-warp prefix
      // (my_pf_wp) to get prefix sum within block.
      //
      prefix[threadIdx.x+1] = wp_prefix + my_pf_wp;

      // At this point p1s.prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = prefix[ block_size ];
      const int idx_one_tid_0 = elt_per_block - all_threads_num_ones;
      const int smaller_tids_num_ones = prefix[ threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          s[ new_idx ] = key;
        }

    }
  __syncthreads();
}


__global__ void
sort_block_batcher()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i++ )
            {
              int idx_0 = threadIdx.x + i * blockDim.x;
              int idx_1 = idx_0 | bit_vector;
              if ( idx_0 == idx_1 ) continue;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }

    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

__global__ void
sort_block_batcher_1()
{
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          uint bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * blockDim.x;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( ( key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  __syncthreads();

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

template <int BLOCK_LG>
__global__ void
sort_block_batcher_opt()
{
  int block_size = 1 << BLOCK_LG;
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * block_size;
  int chunk_lg = 2 + BLOCK_LG;
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<BLOCK_LG; m_lg++ )
    {
      int sort_dir_vector = 1 << m_lg;
      bool sort_dir = threadIdx.x & sort_dir_vector;
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          int idx_0_t = threadIdx.x + ( threadIdx.x & shift_mask );
          int idx_1_t = idx_0_t + bit_vector;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i += 2 )
            {
              int idx_0 = idx_0_t + ( i << BLOCK_LG );
              int idx_1 = idx_1_t + ( i << BLOCK_LG );
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }
  for ( int m_lg=BLOCK_LG; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * block_size;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  __syncthreads();

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    sort_out[ idx_block_start + sidx ] = s[sidx];

}

template <int BLOCK_LG, int BIN_LG>
__device__ void radix_sort_1_pass_1_tile
(int bin_idx, int tile_idx, bool first_iter);

#define SH_GLOBAL_HISTO(elt) s[ ghisto_sbase + (elt) ]
#define SH_TILE_HISTO(idx) s[ thisto_sbase + (idx) ]

template <int BLOCK_LG, int BIN_LG> __global__ void
radix_sort_1_pass_1(int bin_idx, bool first_iter)
{
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);
  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;

  if ( threadIdx.x < bin_size ) SH_GLOBAL_HISTO( threadIdx.x ) = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_1_pass_1_tile<BLOCK_LG,BIN_LG>(bin_idx,tile_idx,first_iter);

  if ( threadIdx.x >= bin_size ) return;
  int histo_idx = blockIdx.x * bin_size + threadIdx.x;
  sort_histo[ histo_idx ] = SH_GLOBAL_HISTO( threadIdx.x );
}

template <int BLOCK_LG, int BIN_LG> __device__ void
radix_sort_1_pass_1_tile(int bin_idx, int tile_idx, bool first_iter)
{
  int start_bit = bin_idx * sort_bin_lg;
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;

  int idx_block_stop = min( array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;
  int runend_sbase = ghisto_sbase + bin_size;
  int thisto_sbase = runend_sbase + bin_size;

  int *sort_src = first_iter ? sort_in : sort_out;

  // Load Elements
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_src[ idx_block_start + sidx ];

  // Sort based upon current bin (digit position)
  //
  sort_block_1_bit_split(start_bit,sort_bin_lg);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = s[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( s[sidx] >> start_bit ) & sort_bin_mask;
      s[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) s[elt_per_tile] = bin_size;

  // Initialize histogram for this tile to zero.
  //
  if ( threadIdx.x < bin_size )
    SH_TILE_HISTO( threadIdx.x ) = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];      // Our digit.
      int digit_1 = s[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        s[ runend_sbase + digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];                        // Our digit.
      int digit_0 = sidx > 0 ? s[sidx-1] : -1;    // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = s[ runend_sbase + digit ];
          int count = run_end_sidx - sidx + 1;
          SH_GLOBAL_HISTO( digit ) += count;     // Histogram for block.
          SH_TILE_HISTO( digit ) = count;      // Histogram for tile.
        }
    }

  __syncthreads();

  if ( threadIdx.x >= bin_size ) return;

  // Write out tile histogram.
  //
  int thisto_idx = tile_idx * bin_size + threadIdx.x;
  sort_tile_histo[ thisto_idx ] = SH_TILE_HISTO( threadIdx.x );
}

__device__ void radix_sort_1_pass_2_tile
(int bin_idx, int tile_idx, bool last_iter);

__global__ void
radix_sort_1_pass_2(int bin_idx, bool last_iter)
{
  int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_bin_size;
  int pfe_tile_sidx = pfe_tile_sbase + threadIdx.x;

  volatile int *sv = &s[ 0 ]; // Volatile access.

  // Sum of all histogram bins for our digit value (threadIdx.x)
  //
  int global_bin_sum = 0;

  // Sum of histogram bins for our digit value for array indices
  // lower than ours.
  //
  int overhead_bin_sum = 0;

  if ( threadIdx.x == 0 ) sv[ pfe_tile_sbase ] = 0;

  if ( threadIdx.x < sort_bin_size )
    {
      // Code only correct for sort_bin_size <= 32.

      int idx_for_us = blockIdx.x * sort_bin_size + threadIdx.x;
      int pidx_stop = gridDim.x * sort_bin_size;

      // Compute Global Histogram
      //
      // Loop will set global_bin_sum to the total number of digits
      // of value threadIdx.x.
      //
      // Eg: Array:  123, 121, 373, 932, 863, 113
      //     Based on bin_idx, operating on least-significant digit.
      //     threadIdx.x = 3
      //     Then =>  global_bin_sum = 4
      //
      for ( int idx = threadIdx.x; idx < pidx_stop; idx += sort_bin_size )
        {
          if ( idx == idx_for_us ) overhead_bin_sum = global_bin_sum;
          global_bin_sum += sort_histo[idx];
        }
      
      // Compute Global Prefix Sum
      //
      //
      sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_sum;
      int global_bin_prefix = global_bin_sum;

      for ( int i=0; i<sort_bin_lg; i++ )
        {
          int dist = 1 << i;
          int sum_0 = dist <= threadIdx.x
            ? sv[ pfi_tile_sbase + threadIdx.x - dist ] : 0;
          sv[ pfi_tile_sbase + threadIdx.x ] = global_bin_prefix += sum_0;
        }

      // Initialize Tile Prefix Sum
      //
      sv[ pfe_tile_sidx ] += overhead_bin_sum;
    }

  __syncthreads();

  volatile int *tile_offsets = &s[pf_offset_sbase_base];
  if ( threadIdx.x < sort_bin_size ) tile_offsets[threadIdx.x]=0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int count;
      if ( threadIdx.x < sort_bin_size )
        {
          int bo_idx = tile_idx * sort_bin_size + threadIdx.x;
          count = sort_tile_histo[ bo_idx ];
          int to_idx = sort_bin_size + threadIdx.x;
          int offset = count;
          tile_offsets[ to_idx ] = offset;
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 1 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 2 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 4 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 8 ];
          if ( debug_sort && last_iter )
            {
              scan_out[bo_idx] = sv[ pfe_tile_sidx ];
              scan_r2[bo_idx] = tile_offsets[ to_idx - 1];
            }
        }
      __syncthreads();
      radix_sort_1_pass_2_tile(bin_idx,tile_idx,last_iter);
      __syncthreads();
      if ( threadIdx.x < sort_bin_size ) sv[ pfe_tile_sidx ] += count;
    }
}

__device__ void
radix_sort_1_pass_2_tile
(int bin_idx, int tile_idx, bool last_iter)
{
  const int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int idx_tile_start = tile_idx * elt_per_tile;

  int start_bit = bin_idx * sort_bin_lg;

  int pfe_tile_sbase = 0;
  int pfi_tile_sbase = 1;
  int pf_offset_sbase_base = pfi_tile_sbase + sort_bin_size;
  int pf_offset_sbase = pf_offset_sbase_base + sort_bin_size - 1;

  for ( int i=0; i<elt_per_thread; i++ )
    {
      int local_idx = threadIdx.x + i * blockDim.x;
      int idx = idx_tile_start + local_idx;
      uint key = sort_out_b[idx];
      uint digit = ( key >> start_bit ) & sort_bin_mask;
      int local_offset = s[ pf_offset_sbase + digit ];
      int key_digit_rank = local_idx - local_offset;
      int idx_digit_index = s[ pfe_tile_sbase + digit ] + key_digit_rank;

      if ( debug_sort && last_iter )
        sort_out[idx] = ( idx_digit_index << 12 ) + local_offset;
      else
        sort_out[idx_digit_index] = key;

    }
}
