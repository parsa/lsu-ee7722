#include "hip/hip_runtime.h"
#include "sort.cuh"

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ int block_lg;
__constant__ int array_size, array_size_lg;
__constant__ int *scan_in, *scan_out;
__constant__ int *scan_r2;

extern __shared__ int s[];

__constant__ int *sort_in, *sort_out, *sort_out_b;
__constant__ int *sort_tile_histo;
__constant__ int *sort_histo;

__constant__ int sort_bin_mask;
__constant__ int sort_bin_size, sort_bin_count;
__constant__ int sort_all_bin_count, sort_all_bin_lg;
__constant__ int sort_bin_lg;

__global__ void prefix_sum_0(int tree_level);
__global__ void prefix_sum_1_pass_1();
template <int BLOCK_LG> __global__ void prefix_sum_1_opt_pass_1();
__global__ void prefix_sum_1_pass_2();
__global__ void sort_segments_1_bit_split();
__global__ void sort_block_batcher();
__global__ void sort_block_batcher_1();
template <int BLOCK_LG> __global__ void sort_block_batcher_opt();
template <int BLOCK_LG, int BIN_LG>
__global__ void radix_sort_1_pass_1(int bin_idx);
__global__ void radix_sort_1_pass_2(int bin_idx);

static __host__ int
kernels_get_attr_(pCUDA_Func_Attributes *attr)
{
  int count = 0;

#define GETATTR(func)                                                         \
  count++;                                                                    \
  if ( attr ) {                                                               \
      attr->err = hipFuncGetAttributes(&attr->attr,reinterpret_cast<const void*>(func));                    \
      attr->name = #func;                                                     \
      attr++;                                                                 \
    }

  GETATTR(prefix_sum_0);
  GETATTR(prefix_sum_1_pass_1);
  GETATTR(prefix_sum_1_opt_pass_1<8>);
  GETATTR(prefix_sum_1_pass_2);
  GETATTR(sort_segments_1_bit_split);
  GETATTR(sort_block_batcher);
  GETATTR(sort_block_batcher_1);
  GETATTR(sort_block_batcher_opt<8>);
  GETATTR((radix_sort_1_pass_1<6,4>));
  GETATTR(radix_sort_1_pass_2);
  return count;
#undef GETATTR
}

__host__ int
kernels_get_attr(pCUDA_Func_Attributes **attr)
{
  int count = kernels_get_attr_(NULL);
  *attr = (pCUDA_Func_Attributes*) calloc(count,sizeof(**attr));
  return kernels_get_attr_(*attr);
}

// This routine executes on the CPU.
//
__host__ void
scan_launch(dim3 dg, dim3 db, int version, int array_size, int array_size_lg)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  switch ( version ){
  case 0:
    {
      for ( int tree_level = 0; tree_level < array_size_lg; tree_level++ )
        prefix_sum_0<<<dg,db>>>(tree_level);
    }
    break;
  case 1:
    {
      int size_per_elt = 4 * 2;
      int shared_size = db.x * size_per_elt;
      prefix_sum_1_pass_1<<<dg,db,shared_size>>>();
      prefix_sum_1_pass_2<<<dg,db>>>();
    }
    break;
  case 2:
    {
      int size_per_elt = 4 * 4;
      int shared_size = db.x * size_per_elt;
      switch ( db.x ) {
      case 64:
        prefix_sum_1_opt_pass_1<6><<<dg,db,shared_size>>>();
        break;
      case 256:
        prefix_sum_1_opt_pass_1<8><<<dg,db,shared_size>>>();
        break;
      case 1024:
        prefix_sum_1_opt_pass_1<10><<<dg,db,shared_size>>>();
        break;
      }
      prefix_sum_1_pass_2<<<dg,db>>>();
    }
    break;

  case 4:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4 + 2;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_segments_1_bit_split<<<dg,db,shared_size>>>();
    }
    break;

  case 5:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_block_batcher<<<dg,db,shared_size>>>();
    }
    break;

  case 6:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      sort_block_batcher_1<<<dg,db,shared_size>>>();
    }
    break;

  case 7:
    {
      int elt_per_thread = 4;
      int size_per_elt = 4;
      int shared_size = db.x * size_per_elt * elt_per_thread;
      switch ( db.x ) {
      case 64: sort_block_batcher_opt<6><<<dg,db,shared_size>>>(); break;
      case 128: sort_block_batcher_opt<7><<<dg,db,shared_size>>>(); break;
      case 256: sort_block_batcher_opt<8><<<dg,db,shared_size>>>(); break;
      default:break;
      }
    }
    break;

  case 8:
    {
      const int bin_lg = 4;
      const int bin_size = 1 << bin_lg;
      int elt_per_thread = 4;
      int size_per_elt_1 = 4 + 2; // Assuming sort_bin_size < block_size
      int shared_size_pass_1 = db.x * size_per_elt_1 * elt_per_thread
        + 4 * bin_size * 4;
      int shared_size_pass_2 = (bin_size+1) * 4 * 3;
      int bins = int( ceil( 32.0 / bin_lg ) );
      for ( int bin_idx = 0; bin_idx < bins; bin_idx++ )
        {
          switch(db.x){
          case 64:
            radix_sort_1_pass_1<6,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx);
          break;
          case 128:
            radix_sort_1_pass_1<7,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx);
          break;
          case 256:
            radix_sort_1_pass_1<8,bin_lg><<<dg,db,shared_size_pass_1>>>
              (bin_idx);
          break;
          }

          radix_sort_1_pass_2<<<dg,db,shared_size_pass_2>>>(bin_idx);

        }
    }
    break;

  default:
    break;
  }
}

__device__ int lg_ceil(uint n)
{
  if ( n == 0 ) return 0;
  return 32 - __clz(n-1);
}

__device__ int div_ceil(int a, int b){return int( ceilf( ((float)a)/b ) );}


__global__ void
prefix_sum_0(int tree_level)
{
  /// Simple Prefix Sum Kernel
  //
  // Limited to one thread per element.

  // Does not work because writes can occur before reads.

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if ( tid >= array_size ) return;

  int idx = tid;
  int key_1;

  if ( tree_level == 0 )
    {
      key_1 = scan_in[ idx ];
      scan_out[idx] = key_1;
    }
  else
    {
      key_1 = scan_out[ idx ];
    }

  int dist = 1 << tree_level;
  if ( dist > idx ) return;
  int idx_0 = idx - dist;
  int key_0 = scan_out[ idx_0 ];
  scan_out[ idx ] = key_0 + key_1;
}


__device__ int
prefix_sum_1_pass_1_chunk(int idx_block_start, int sum_in);

__global__ void
prefix_sum_1_pass_1()
{
  int chunks_per_array = int( ceilf( float(array_size) / blockDim.x ));
  int chunks_per_block = int( ceilf( float(chunks_per_array) / gridDim.x ));
  int elt_per_block = chunks_per_block * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_stop = min( array_size, idx_block_start + elt_per_block );
  int sum = 0;
  for ( int idx_block = idx_block_start; idx_block < idx_stop;
        idx_block += blockDim.x )
    sum = prefix_sum_1_pass_1_chunk(idx_block, sum);

  scan_r2[blockIdx.x] = sum;
}

__device__ int
prefix_sum_1_pass_1_chunk(int idx_block_start, int sum_in)
{
  int idx = idx_block_start + threadIdx.x;

  // Copy input array to local memory.
  //
  s[ threadIdx.x ] = scan_in[idx];

  // Pointers to Read- and Write- Shared Memory Areas.
  //
  // Code will read one area of shared memory and write a different
  // one.
  //
  int sidx_rd = 0;
  int sidx_wr = blockDim.x;

  for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
    {
      // Compute the distance to the neighbor to be read.
      //
      int dist = 1 << tree_level;

      __syncthreads();

      // Location of "our" element in read area of shared memory.
      //
      int sidx_1r = sidx_rd + threadIdx.x;

      // Location of our element in write area.
      //
      int sidx_1w = sidx_wr + threadIdx.x;

      // Read neighbor's element, or zero if there is no neighbor at that
      // distance.
      //
      int key_0 = threadIdx.x >= dist ? s[sidx_1r-dist] : 0;

      // Write sum.
      //
      s[sidx_1w] = key_0 + s[sidx_1r];

      // Swap indexes to read and write areas of shared memory.
      //
      int old_rd = sidx_rd; sidx_rd = sidx_wr; sidx_wr = old_rd;
    }

  // Write prefix to global memory, adding on value from previous chunk.
  //
  scan_out[idx] = s[ sidx_rd + threadIdx.x ] + sum_in;

  __syncthreads();

  // Return maximum value found in this chunk + prior chunks.
  //
  return s[ sidx_rd + blockDim.x-1] + sum_in;
}

template <int BLOCK_LG> __device__ int
prefix_sum_1_opt_pass_1_chunk(int idx_block_start, int sum_in);


template <int BLOCK_LG> __global__ void
prefix_sum_1_opt_pass_1()
{
  int chunks_per_array = int( ceilf( float(array_size) / blockDim.x ));
  int chunks_per_block = int( ceilf( float(chunks_per_array) / gridDim.x ));
  int elt_per_block = chunks_per_block * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_stop = min( array_size, idx_block_start + elt_per_block );
  int sum = 0;
  s[ threadIdx.x ] = 0;
  s[ threadIdx.x + 2 * blockDim.x ] = 0;
  for ( int idx_block = idx_block_start; idx_block < idx_stop;
        idx_block += blockDim.x )
    sum = prefix_sum_1_opt_pass_1_chunk<BLOCK_LG>(idx_block, sum);

  scan_r2[blockIdx.x] = sum;
}

template <int BLOCK_LG> __device__ int
prefix_sum_1_opt_pass_1_chunk(int idx_block_start, int sum_in)
{
  /// Optimized Prefix Sum Code
  //
  //  Routine is custom compiled for each needed block size using templates.


  int idx = idx_block_start + threadIdx.x;
  int block_size = 1 << BLOCK_LG;
  int sidx_rd = block_size;
  int sidx_wr = 3 * block_size;

  s[ sidx_rd + threadIdx.x ] = scan_in[idx];

  for ( int tree_level = 0; tree_level < BLOCK_LG; tree_level++ )
    {
      int dist = 1 << tree_level;

      // No need to sync if 32 (or fewer) threads.
      //
      if ( BLOCK_LG > 5 ) __syncthreads();

      int sidx_1r = sidx_rd + threadIdx.x;
      int sidx_1w = sidx_wr + threadIdx.x;

      // Shared memory has been padded with zeros so there is
      // no need to check whether dist <= threadIdx.x.
      //
      int key_0 = s[sidx_1r-dist];

      s[sidx_1w] = key_0 + s[sidx_1r];

      // Because loop is unrollable these should not generate
      // any code.
      //
      int old_rd = sidx_rd; sidx_rd = sidx_wr; sidx_wr = old_rd;
    }

  scan_out[idx] = s[ sidx_rd + threadIdx.x ] + sum_in;

  if ( BLOCK_LG > 5 ) __syncthreads();
  return s[ sidx_rd + block_size - 1 ] + sum_in;
}

__global__ void
prefix_sum_1_pass_2()
{
  __shared__ int sum_in;
  if ( threadIdx.x == 0 )
    {
      int sum = 0;
      for ( int i=0; i<((int)blockIdx.x); i++ )
        sum += scan_r2[i];
      sum_in = sum;
    }
  __syncthreads();
  int chunks_per_array = int( ceilf( float(array_size) / blockDim.x ));
  int chunks_per_block = int( ceilf( float(chunks_per_array) / gridDim.x ));
  int elt_per_block = chunks_per_block * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_stop = min( array_size, idx_block_start + elt_per_block );
  int idx_thread_start = idx_block_start + threadIdx.x;
  for ( int idx = idx_thread_start; idx < idx_stop; idx += blockDim.x )
    scan_out[idx] += sum_in;
}

__device__ void sort_block_1_bit_split(int bit_low, int bit_count);


__global__ void
sort_segments_1_bit_split()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_block_stop = idx_block_start + elt_per_block;

  int idx_start = idx_block_start + threadIdx.x;
  int key_base_rd = 0;

  // Load Elements
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  sort_block_1_bit_split(0,32);

  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += blockDim.x, idx += blockDim.x )
        sort_out[idx] = s[ key_base_rd + sidx ];
}

__device__ void
sort_block_1_bit_split(int bit_low, int bit_count)
{
  const int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int pfe_base_rd = elt_per_block;
  int pfi_base_rd = elt_per_block + 1;

  volatile __shared__ int col_total[16];
  if ( threadIdx.x < 16 )
    {
      s[ pfe_base_rd ] = 0;
      col_total[threadIdx.x] = 0;
    }

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      int bit_mask = 1 << bit_pos;
      int prefix_vector = 0;
      int count_bit = 1;

      int keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      for ( int sidx = threadIdx.x, i = 0;
            i < elt_per_thread; i++, sidx += blockDim.x )
        {
          keys[i] = s[ sidx ];
          if ( keys[i] & bit_mask ) prefix_vector += count_bit;
          count_bit = count_bit << 8;
        }

      s[ pfi_base_rd + threadIdx.x ] = prefix_vector;
      __syncthreads();
      uint pfv_1 = s[ pfe_base_rd + threadIdx.x ];

      // Compute Prefix Sum of Columns
      //
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_0 = threadIdx.x - dist;
          __syncthreads();
          uint pfv_0 = threadIdx.x >= dist ? s[ pfe_base_rd + idx_0 ] : 0;
          pfv_1 += pfv_0;
          s[ pfe_base_rd + threadIdx.x ] = pfv_1;
        }

      __syncthreads();

      const int ct_wbase = 8;
      if ( threadIdx.x < 8 )
        {
          int pfv = s[pfe_base_rd + blockDim.x - ( threadIdx.x & 0x1 ) ];
          int shift = ( threadIdx.x & 0x6 ) << 2;
          int my_val = ( pfv >> shift ) & 0xff;
          int sidx = ct_wbase + threadIdx.x;
          col_total[ sidx ] = my_val;
          col_total[ sidx ] = my_val += col_total[ sidx - 1 ] ;
          col_total[ sidx ] = my_val += col_total[ sidx - 2 ] ;
          col_total[ sidx ] = my_val += col_total[ sidx - 4 ] ;
        }

      __syncthreads();
      int ct_base = ct_wbase - 1;

#if 0
      if ( threadIdx.x == 0 && blockIdx.x == 0 )
        {
          for ( int i=0; i<5; i++ )
            scan_out[(bit_pos-bit_low)*16+i] = col_total[ct_base + 2*i];
        }
#endif

       int total_ones = col_total[ct_base+8];

      for ( int sidx = threadIdx.x, i = 0;
            i < elt_per_thread; i++, sidx += blockDim.x )
        {
          int count = ( pfv_1 & 0xff ) + col_total[ct_base+2*i];
          int key = keys[i];
          int new_idx = key & bit_mask
            ? elt_per_block - total_ones + count
            : sidx - count;
          s[ new_idx ] = key;
          pfv_1 >>= 8;
        }
    }
}

__global__ void
sort_block_batcher()
{
  int elt_per_thread = 4;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i++ )
            {
              int idx_0 = threadIdx.x + i * blockDim.x;
              int idx_1 = idx_0 | bit_vector;
              if ( idx_0 == idx_1 ) continue;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }

    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

__global__ void
sort_block_batcher_1()
{
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * blockDim.x;
  int chunk_lg = lg_ceil(elt_per_block);
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          uint bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * blockDim.x;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( ( key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += blockDim.x )
    sort_out[ idx_block_start + sidx ] = s[sidx];
}

template <int BLOCK_LG>
__global__ void
sort_block_batcher_opt()
{
  int block_size = 1 << BLOCK_LG;
  int elt_per_thread = 4;
  int elt_per_thread_half = elt_per_thread >> 1;
  int elt_per_block = elt_per_thread * block_size;
  int chunk_lg = 2 + BLOCK_LG;
  int idx_block_start = elt_per_block * blockIdx.x;

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_in[ idx_block_start + sidx ];

  for ( int m_lg=0; m_lg<BLOCK_LG; m_lg++ )
    {
      int sort_dir_vector = 1 << m_lg;
      bool sort_dir = threadIdx.x & sort_dir_vector;
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          int idx_0_t = threadIdx.x + ( threadIdx.x & shift_mask );
          int idx_1_t = idx_0_t + bit_vector;
          __syncthreads();
          for ( int i=0; i<elt_per_thread; i += 2 )
            {
              int idx_0 = idx_0_t + ( i << BLOCK_LG );
              int idx_1 = idx_1_t + ( i << BLOCK_LG );
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }
  for ( int m_lg=BLOCK_LG; m_lg<chunk_lg; m_lg++ )
    {
      int sort_dir_vector = 1 << ( m_lg + 1 );
      for ( int bit = m_lg; bit >= 0; bit-- )
        {
          int bit_vector = 1 << bit;
          uint shift_mask = ~( bit_vector - 1 );
          __syncthreads();
          for ( int i=0; i<elt_per_thread_half; i++ )
            {
              int idx_ref = threadIdx.x + i * block_size;
              int idx_0 = idx_ref + ( idx_ref & shift_mask );
              int idx_1 = idx_0 + bit_vector;
              bool sort_dir = idx_0 & sort_dir_vector;
              uint key_0 = s[idx_0];
              uint key_1 = s[idx_1];
              if ( (key_0 < key_1 ) == sort_dir )
                { s[idx_0] = key_1;  s[idx_1] = key_0; }
            }
        }
    }

  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    sort_out[ idx_block_start + sidx ] = s[sidx];

}

template <int BLOCK_LG, int BIN_LG>
__device__ void radix_sort_1_pass_1_tile(int bin_idx, int tile_idx);

template <int BLOCK_LG, int BIN_LG> __global__ void
radix_sort_1_pass_1(int bin_idx)
{
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);
  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;

  if ( threadIdx.x < bin_size ) s[ ghisto_sbase + threadIdx.x ] = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx ++ )
    radix_sort_1_pass_1_tile<BLOCK_LG,BIN_LG>(bin_idx,tile_idx);

  if ( threadIdx.x >= bin_size ) return;
  int histo_idx = blockIdx.x * bin_size + threadIdx.x;
  sort_histo[ histo_idx ] = s[ ghisto_sbase + threadIdx.x ];
}

template <int BLOCK_LG, int BIN_LG> __device__ void
radix_sort_1_pass_1_tile(int bin_idx, int tile_idx)
{
  int start_bit = bin_idx * sort_bin_lg;
  int elt_per_thread = 4;
  int block_size = 1 << BLOCK_LG;
  int bin_size = 1 << BIN_LG;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;

  int idx_block_stop = min( array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  int sbase_1_bit_split_end = elt_per_tile + block_size + 1;
  int ghisto_sbase = sbase_1_bit_split_end;
  int runend_sbase = ghisto_sbase + bin_size;
  int thisto_sbase = runend_sbase + bin_size;

  int *sort_src = bin_idx == 0 ? sort_in : sort_out;

  // Load Elements
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    s[sidx] = sort_src[ idx_block_start + sidx ];

  sort_block_1_bit_split(start_bit,sort_bin_lg);

  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      sort_out_b[idx] = s[sidx];
      int digit = ( s[sidx] >> start_bit ) & sort_bin_mask;
      s[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) s[elt_per_tile] = bin_size;

  if ( threadIdx.x < bin_size )
    s[ thisto_sbase + threadIdx.x ] = 0;

  __syncthreads();

  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];
      int digit_1 = s[sidx+1];
      if ( digit != digit_1 )
        s[ runend_sbase + digit ] = sidx;
    }

  __syncthreads();

  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = s[sidx];
      int digit_0 = sidx > 0 ? s[sidx-1] : -1;
      if ( digit != digit_0 )
        {
          int run_end_sidx = s[ runend_sbase + digit ];
          int count = run_end_sidx - sidx + 1;
          s[ ghisto_sbase + digit ] += count;
          s[ thisto_sbase + digit ] = count;
        }
    }

  __syncthreads();

  if ( threadIdx.x >= bin_size ) return;

  int offset_idx = tile_idx * bin_size + threadIdx.x;
  sort_tile_histo[ offset_idx ] = s[ thisto_sbase + threadIdx.x ];
}

__device__ void radix_sort_1_pass_2_tile(int bin_idx, int tile_idx);

__device__ void
radix_sort_1_pass_2(int bin_idx)
{
  int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  int pf_prefix_sbase = 0;
  int pf_global_sbase = 1;
  int pf_offset_sbase_base = pf_global_sbase + sort_bin_size;
  int pf_prefix_sidx = pf_prefix_sbase + threadIdx.x;

  if ( threadIdx.x < sort_bin_size )
    {
      // Code only correct for sort_bin_size <= 32.

      int idx_for_us = blockIdx.x * sort_bin_size + threadIdx.x;
      int pidx_stop = gridDim.x * sort_bin_size;

      int global_bin_sum = 0;
      int overhead_bin_sum = 0;
      for ( int idx = threadIdx.x; idx < pidx_stop; idx += sort_bin_size )
        {
          if ( idx == idx_for_us ) overhead_bin_sum = global_bin_sum;
          global_bin_sum += sort_histo[idx];
        }

      int * volatile global_prefix = &s[ pf_global_sbase ];
      global_prefix[ threadIdx.x ] = global_bin_sum;

      int global_bin_prefix = global_bin_sum;

      for ( int i=0; i<sort_bin_lg; i++ )
        {
          int dist = 1 << i;
          int sum_0 = dist <= threadIdx.x
            ? global_prefix[ threadIdx.x - dist ] : 0;
          global_prefix[ threadIdx.x ] = global_bin_prefix += sum_0;
        }

      if ( threadIdx.x == 0 ) s[ pf_prefix_sbase ] = 0;
      s[ pf_prefix_sidx ] += overhead_bin_sum;
    }

  __syncthreads();

  int * volatile tile_offsets = &s[pf_offset_sbase_base];
  if ( threadIdx.x < sort_bin_size ) tile_offsets[threadIdx.x]=0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int count;
      if ( threadIdx.x < sort_bin_size )
        {
          int bo_idx = tile_idx * sort_bin_size + threadIdx.x;
          count = sort_tile_histo[ bo_idx ];
          int to_idx = sort_bin_size + threadIdx.x;
          int offset = count;
          tile_offsets[ to_idx ] = offset;
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 1 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 2 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 4 ];
          tile_offsets[ to_idx ] = offset += tile_offsets[ to_idx - 8 ];
          
          // int so_idx = tile_idx * sort_bin_size + threadIdx.x;
          //  if ( bin_idx == 1 ) scan_out[so_idx] = offset;
        }
      __syncthreads();
      radix_sort_1_pass_2_tile(bin_idx,tile_idx);
      __syncthreads();
      if ( threadIdx.x < sort_bin_size ) s[ pf_prefix_sidx ] += count;
    }
}

__device__ void
radix_sort_1_pass_2_tile(int bin_idx, int tile_idx)
{
  const int elt_per_thread = 4;
  int elt_per_tile = elt_per_thread * blockDim.x;
  int tiles_per_array = div_ceil(array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int idx_tile_start = tile_idx * elt_per_tile;

  int start_bit = bin_idx * sort_bin_lg;

  int pf_prefix_sbase = 0;
  int pf_global_sbase = 1;
  int pf_offset_sbase_base = pf_global_sbase + sort_bin_size;
  int pf_offset_sbase = pf_offset_sbase_base + sort_bin_size - 1;

  for ( int i=0; i<elt_per_thread; i++ )
    {
      int local_idx = threadIdx.x + i * blockDim.x;
      int idx = idx_tile_start + local_idx;
      uint key = sort_out_b[idx];
      uint digit = ( key >> start_bit ) & sort_bin_mask;
      int local_offset = s[ pf_offset_sbase + digit ];
      int key_digit_rank = local_idx - local_offset;
      int idx_digit_index = s[ pf_prefix_sbase + digit ] + key_digit_rank;
      sort_out[idx_digit_index] = key;
      //sort_out[idx] = idx_digit_index;
    }
}

#if 0
__global__ void
radix_sort_0_pass_1()
{
  int elt_per_block = int( ceilf( float(array_size) / blockDim.x ) );
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_block_stop = idx_block_start + elt_per_block;

  int idx_start = idx_block_start + threadIdx.x;

  for ( int idx = idx_start; idx < idx_block_stop; idx += blockDim.x )
    {
      int key = sort_in[idx];
      int digit = ( key >> sort_key_shift ) & sort_bin_mask;
      s[ threadIdx.x + ( digit << block_lg ) ]++;
    }

  __syncthreads();

  for ( int tree_level = 1; tree_level < block_lg; tree_level++ )
    {
      int dist = 1 << tree_level;
      int participation_mask = dist - 1;
      if ( threadIdx.x & participation_mask != participation_mask ) continue;
      int bin_neighbor = threadIdx.x - ( dist >> 1 );
      for ( int digit = 0; digit < sort_bin_size; digit++ )
        {
          int h_base = digit << block_lg;
          s[ h_base + threadIdx.x ] += s[ h_base + bin_neighbor ];
        }
    }

  if ( threadIdx.x >= sort_bin_size ) return;
  int so_base = sort_bin_count * blockIdx.x;
  scan_in[ so_base + threadIdx.x ] =
    s[ ( threadIdx.x << blockDim.x ) + blockDim.x - 1 ];
}

__global__ void
radix_sort_0_pass_2()
{
  int total_bin_count = sort_bin_count * gridDim.x;

  for ( int idx = threadIdx.x; idx < sort_all_bin_count; idx += blockDim.x )
    {
      int digit = idx & sort_bin_mask;
      int chunk = idx >> sort_bin_lg;
      int sidx = chunk + digit * gridDim.x;
      s[sidx] = scan_in[idx];
    }

  int base_rd = 0;
  int base_wr = total_bin_count;

  for ( int tree_level = 0; tree_level < sort_all_bin_lg; tree_level++ )
    {
      int dist = 1 << tree_level;
      for ( int idx = threadIdx.x; idx < total_bin_count; idx += blockDim.x )
        {
          int idx_0 = idx - dist;
          int key_0 = idx_0 >= 0 ? s[ base_rd + idx_0 ] : 0;
          s[ base_wr + idx ] = s[ base_rd + idx ] + key_0;
        }
      int base_rd_old = base_rd; base_rd = base_wr; base_wr = base_rd_old;
      __syncthreads();
    }

  if ( threadIdx.x < sort_bin_count )
    {
      int sidx = threadIdx.x * gridDim.x + blockIdx.x - 1;
      int my_bin = sidx >= 0 ? s[ base_rd + sidx ] : 0;
      __syncthreads();
      s[ threadIdx.x ] = my_bin;
    }

  int elt_per_block = int( ceilf( float(array_size) / blockDim.x ) );
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_block_stop = idx_block_start + elt_per_block;

  int idx_start = idx_block_start + threadIdx.x;

  for ( int idx = idx_start; idx < idx_block_stop; idx += blockDim.x )
    {
      int key = sort_in[idx];
      int digit = ( key >> sort_key_shift ) & sort_bin_mask;
      //  int idx_out = atomicAdd( &s[ digit ], 1 );
      int idx_out = s[ digit ]++;
      sort_out[idx_out] = key;
      __syncthreads();
    }
}
#endif
