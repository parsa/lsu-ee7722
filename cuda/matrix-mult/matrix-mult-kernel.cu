#include "hip/hip_runtime.h"
#include "matrix-mult.cuh"
#include <gp/cuda-util-kernel.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ int array_size, array_size_lg;
__constant__ int row_stride, row_stride_lg, dim_block_lg;
__constant__ float *a, *b, *c;
__constant__ int32_t *t_compute, *t_all;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__constant__ int cs_itid_stride;

texture<float> a_tex, b_tex;

__global__ void mm();
__global__ void mm_iter();
__global__ void mm_blk_cache_ab();
__global__ void mm_blk_cache_ab_opt();
__global__ void mm_blk_cache_ab_tc();
__global__ void mm_blk_cache_a_local();

template <int dim_block> __global__ void
mm_blk_cache_a_local_t();


static __host__ void
collect_symbols()
{
  CU_SYM(array_size); CU_SYM(array_size_lg);
  CU_SYM(row_stride); CU_SYM(row_stride_lg); CU_SYM(dim_block_lg);
  CU_SYM(a); CU_SYM(b); CU_SYM(c);
  CU_SYM(t_compute); CU_SYM(t_all);
  CU_SYM(cs_itid_stride);
}


static __host__ int
kernels_get_attr_(pCUDA_Func_Attributes *attr)
{
  collect_symbols();

  int count = 0;

#define GETATTR(func)                                                         \
  count++;                                                                    \
  if ( attr ) {                                                               \
      attr->err = hipFuncGetAttributes(&attr->attr,reinterpret_cast<const void*>(func));                    \
      attr->name = #func;                                                     \
      attr++;                                                                 \
    }

  GETATTR(mm);
  GETATTR(mm_iter);
  GETATTR(mm_blk_cache_ab);
  GETATTR(mm_blk_cache_ab_opt);
  GETATTR(mm_blk_cache_a_local_t<3>);
  GETATTR(mm_blk_cache_a_local_t<4>);
  GETATTR(mm_blk_cache_ab_tc);
  return count;
#undef GETATTR
}

__host__ int
kernels_get_attr(pCUDA_Func_Attributes **attr)
{
  int count = kernels_get_attr_(NULL);
  *attr = (pCUDA_Func_Attributes*) calloc(count,sizeof(**attr));
  return kernels_get_attr_(*attr);
}

// This routine executes on the CPU.
//
__host__ void
mmult_launch(dim3 dg, dim3 db, int version,
             void *a_dev, void *b_dev, int array_size)
{
  static bool tex_bound = false;

  const bool need_tex = version == 6;

  if ( need_tex && !tex_bound )
    {
      const size_t size = array_size * sizeof(float);
      size_t offset;
      const hipChannelFormatDesc fd =
        hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
      hipBindTexture(&offset, a_tex, a_dev, fd, size);
      hipBindTexture(&offset, b_tex, b_dev, fd, size);
      tex_bound = true;
    }
  else if ( !need_tex && tex_bound )
    {
      hipUnbindTexture(a_tex);
      hipUnbindTexture(b_tex);
      tex_bound = false;
    }

  // Launch the kernel, using the provided configuration (block size, etc).
  //
  switch ( version ){
  case 1: mm_iter<<<dg,db>>>(); break;
  case 2:
    {
      int shared_size = 2 * 4 * db.x;
      mm_blk_cache_ab<<<dg,db,shared_size>>>();
    }
    break;
  case 3:
    {
      int shared_size = 2 * 4 * db.x;
      mm_blk_cache_ab_opt<<<dg,db,shared_size>>>();
    }
    break;
  case 4:
    {
      int shared_size = 4 * db.x;
      if ( db.x == 64 ) mm_blk_cache_a_local_t<3><<<dg,db,shared_size>>>();
    }
    break;
  case 5:
    {
      int shared_size = 4 * db.x;
      if ( db.x == 256 ) mm_blk_cache_a_local_t<4><<<dg,db,shared_size>>>();
    }
    break;
  case 6:
    {
      mm_blk_cache_ab_tc<<<dg,db>>>();
    }
    break;
  }
}

__global__ void
mm()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if ( idx > array_size ) return;

  int row_mask = row_stride - 1;
  int col = idx & row_mask;
  int row = idx >> row_stride_lg;
  int idx_base = row << row_stride_lg;

  float c_value = 0;

  for ( int k=0; k<row_stride; k++ )
    {
      int a_idx = idx_base | k;
      int b_idx = ( k << row_stride_lg ) | col;
      c_value += a[a_idx] * b[b_idx];
    }

  c[idx] = c_value;
}

__global__ void
mm_iter()
{
  bool lane_0 = ( threadIdx.x & 0x1f ) == 0;
  int time_start = clock();

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;
  int row_mask = row_stride - 1;

  ///
  /// Iteration Strategy
  //
  // Value of c_idx in first iteration of loop below assuming:
  //   Array size is 1024 x 1024
  //   Block size is 256 threads.
  //
  //    9 8 7 6 5 4 3 2 1 0 9 8 7 6 5 4 3 2 1 0   <- Bit position.
  //   |              blockIdx |     threadIdx |  <- Relationship to threads.
  //   |   row             |     col           |  <- Array element.
  //
  // Value of c_idx in the second iteration of the loop below:
  //
  //    9 8 7 6 5 4 3 2 1 0 9 8 7 6 5 4 3 2 1 0   <- Bit position.
  //   |    gridDim + blockIdx |     threadIdx |  <- Relationship to threads.
  //   |   row             |     col           |  <- Array element.
  //
  // Value of c_idx in the third iteration of the loop below:
  //
  //    9 8 7 6 5 4 3 2 1 0 9 8 7 6 5 4 3 2 1 0   <- Bit position.
  //   |    2 * gridDim + blockIdx | threadIdx |  <- Relationship to threads.
  //   |   row             |     col           |  <- Array element.


  for ( int c_idx = tid; c_idx < array_size; c_idx += thread_count )
    {
      int col = c_idx & row_mask;
      int row = c_idx >> row_stride_lg;
      int a_idx_base = row << row_stride_lg;

      float c_value = 0;

      for ( int k=0; k<row_stride; k++ )
        {
          int a_idx = a_idx_base | k;
          int b_idx = ( k << row_stride_lg ) | col;
          c_value += a[a_idx] * b[b_idx];
        }

      c[c_idx] = c_value;
    }

  if ( !lane_0 ) return;
  t_all[tid>>5] = clock() - time_start;
  t_compute[tid>>5] = 0;
}

__global__ void
mm_blk_cache_ab()
{
  int group_count_lg = row_stride_lg - dim_block_lg;
  int group_count = 1 << group_count_lg;
  int dim_block = 1 << dim_block_lg;
  int local_row_mask = dim_block - 1;
  int block_lg = dim_block_lg << 1;

  int local_col = threadIdx.x & local_row_mask;
  int local_row = threadIdx.x >> dim_block_lg;
  int b_sidx_base = blockDim.x;
  int a_sidx = threadIdx.x;
  int b_sidx = threadIdx.x + b_sidx_base;

  int row_idx_pos = row_stride_lg;
  int row_sidx_pos = dim_block_lg;

  int c_col = ( blockIdx.x << dim_block_lg ) + local_col;
  int c_row = local_row << row_stride_lg;
  int itid_stride = gridDim.x << dim_block_lg;

  for ( ;; c_col += itid_stride )
    {
      int col_overflow = c_col >> row_stride_lg;
      c_row += col_overflow << ( dim_block_lg + row_stride_lg );
      c_col &= row_stride - 1;

      int c_idx = c_row + c_col;

      if ( c_row >= array_size ) break;

      float c_value = 0;

      int a_idx = c_row + local_col;
      int b_idx = ( local_row << row_idx_pos ) | c_col;

      for ( int k_group = 0; k_group < group_count; k_group++ )
        {
          s[a_sidx] = a[a_idx];
          s[b_sidx] = b[b_idx];

          a_idx += 1 << dim_block_lg;
          b_idx += 1 << ( block_lg + group_count_lg );

          __syncthreads();

          for ( int kk = 0; kk < dim_block; kk++ )
            {
              int a_sidx_k = ( local_row << row_sidx_pos ) | kk;
              int b_sidx_k =
                b_sidx_base + ( ( kk << row_sidx_pos ) | local_col );

              c_value += s[a_sidx_k] * s[b_sidx_k];
            }

          __syncthreads();
        }

      c[c_idx] = c_value;
    }
}

__global__ void
mm_blk_cache_ab_tc()
{
  int group_count_lg = row_stride_lg - dim_block_lg;
  int group_count = 1 << group_count_lg;
  int dim_block = 1 << dim_block_lg;
  int local_row_mask = dim_block - 1;

  int local_col = threadIdx.x & local_row_mask;
  int local_row = threadIdx.x >> dim_block_lg;

  int c_col = ( blockIdx.x << dim_block_lg ) + local_col;
  int c_row = local_row << row_stride_lg;
  int itid_stride = gridDim.x << dim_block_lg;

  for ( ;; c_col += itid_stride )
    {
      int col_overflow = c_col >> row_stride_lg;
      c_row += col_overflow << ( dim_block_lg + row_stride_lg );
      c_col &= row_stride - 1;

      int c_idx = c_row + c_col;

      if ( c_row >= array_size ) break;

      float c_value = 0;

      int a_idx = c_row;
      int b_idx = c_col;

      for ( int k_group = 0; k_group < group_count; k_group++ )
        {
          __syncthreads();


#define LOOP(DIM_BLOCK)                                                       \
          for ( int kk = 0; kk < DIM_BLOCK; kk++ )                            \
            {                                                                 \
              c_value += tex1Dfetch(a_tex,a_idx) * tex1Dfetch(b_tex,b_idx);   \
              a_idx++; b_idx += row_stride;                                   \
            }

          if ( dim_block == 16 ) { LOOP(16); }
          else if ( dim_block == 8 ) { LOOP(8); }
          else { LOOP(dim_block); }
#undef LOOP

        }

      c[c_idx] = c_value;
    }
}

__global__ void
mm_blk_cache_ab_opt()
{
  bool lane_0 = ( threadIdx.x & 0x1f ) == 0;
  int time_start = clock();
  int time_compute = 0;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;
  int group_count_lg = row_stride_lg - dim_block_lg;
  int group_count = 1 << group_count_lg;
  int dim_block = 1 << dim_block_lg;
  int local_row_mask = dim_block - 1;
  int block_lg = dim_block_lg << 1;

  int col_group_itid_pos = block_lg;
  int row_group_itid_pos = block_lg + group_count_lg;
  int col_group_mask = group_count - 1;
  int row_group_idx_pos = row_group_itid_pos;
  int col_group_idx_pos = dim_block_lg;
  int row_idx_pos = row_stride_lg;
  int row_sidx_pos = dim_block_lg;

  int local_col = threadIdx.x & local_row_mask;
  int local_row = threadIdx.x >> row_sidx_pos;
  int b_sidx_base = blockDim.x;
  int a_sidx = threadIdx.x;
  int b_sidx = threadIdx.x + b_sidx_base;

  for ( int itid = tid; itid < array_size; itid += thread_count )
    {
      int row_group = itid >> row_group_itid_pos;
      int col_group = itid >> col_group_itid_pos & col_group_mask;
      if ( row_group > group_count ) break;

      int c_idx =
        ( row_group << row_group_idx_pos )
        | ( local_row << row_idx_pos )
        | ( col_group << col_group_idx_pos )
        | local_col;

      float c_value = 0;

      int a_idx =
        ( row_group << row_group_idx_pos )
        | ( local_row << row_idx_pos )
        | local_col;
      int a_idx_stride = 1 << col_group_idx_pos;
      int b_idx =
        ( local_row << row_idx_pos )
        | ( col_group << col_group_idx_pos )
        | local_col;
      int b_idx_stride = 1 << row_group_idx_pos;


      for ( int k_group = 0; k_group < group_count; k_group++ )
        {
          s[a_sidx] = a[a_idx];
          s[b_sidx] = b[b_idx];
          a_idx += a_idx_stride;
          b_idx += b_idx_stride;

          __syncthreads();
          int compute_start = clock();

          int a_sidx_k = ( local_row << row_sidx_pos );
          int b_sidx_k = b_sidx_base | local_col;

#define LOOP                                                                  \
          for ( int kk = 0; kk < dim_block; kk++ )                            \
            {                                                                 \
              c_value += s[a_sidx_k] * s[b_sidx_k];                           \
              a_sidx_k++;                                                     \
              b_sidx_k += row_sidx_stride;                                    \
            }

          if ( row_sidx_pos == 3 )
            {
              int row_sidx_stride = 8;
              #pragma unroll 8
              LOOP;
            }
          else if ( row_sidx_pos == 4 )
            {
              int row_sidx_stride = 16;
              #pragma unroll 16
              LOOP;
            }
          else
            {
              int row_sidx_stride = 1 << row_sidx_pos;
              LOOP;
            }

          __syncthreads();
          time_compute += clock() - compute_start;
        }

      c[c_idx] = c_value;
    }
  if ( !lane_0 ) return;
  t_all[tid>>5] = clock() - time_start;
  t_compute[tid>>5] = time_compute;
}


template <int DIM_BLOCK_LG> __global__ void
mm_blk_cache_a_local_t()
{
  int dim_block = 1 << DIM_BLOCK_LG;
  int block_lg = DIM_BLOCK_LG << 1;

  int local_a_col = threadIdx.x & ( dim_block - 1 );
  int local_a_row = threadIdx.x >> DIM_BLOCK_LG;
  int c_idx_col = threadIdx.x;
  int b_sidx_copy = threadIdx.x;

  int itid_stride = gridDim.x << ( DIM_BLOCK_LG + row_stride_lg );
  int c_idx_row = blockIdx.x << ( DIM_BLOCK_LG + row_stride_lg );

  /// C Block Loop: Each iteration computes blk values of C.
  //
  for ( ;; c_idx_row += itid_stride )
    {
      int row_overflow = c_idx_row >> array_size_lg << block_lg;
      c_idx_col += row_overflow;
      c_idx_row &= array_size - 1;
      if ( c_idx_col >= row_stride ) break;

      int c_idx = c_idx_row + c_idx_col;

      int a_idx = ( local_a_row << row_stride_lg ) + c_idx_row + local_a_col;
      int b_idx = c_idx_col;

      float cloc[1 << DIM_BLOCK_LG];

      for ( int kk=0; kk<dim_block; kk++ ) cloc[kk] = 0;

      /// A Block Loop:  Each iteration uses a blk x blk submatrix of A.
      //  The iterations move across columns.
      //
      while ( b_idx < array_size )
        {
          __syncthreads();
          s[b_sidx_copy] = a[a_idx];
          a_idx += dim_block;
          __syncthreads();

          /// B Value Loop: Each iteration uses one value of B.
#         pragma unroll
          for ( int kk = 0; kk < dim_block; kk++, b_idx += row_stride )
            {
              float b_val = b[b_idx];
              for ( int ii = 0; ii < dim_block; ii++ )
                cloc[ii] +=  s[ kk + dim_block * ii ] * b_val;
            }
        }

      for ( int ii = 0 ; ii < dim_block; ii++ )
        c[ c_idx + ii * row_stride ] = cloc[ ii ];
    }
}
