#include "hip/hip_runtime.h"
#include "stream-2.cuh"

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ Vector2* a;
__constant__ float* b;

__global__ void dots();
__global__ void dots_iterate();

// This routine executes on the CPU.
//
__host__ void
dots_launch(dim3 dg, dim3 db)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  dots<<<dg,db>>>();
}

// This routine executes on the GPU.
//
__global__ void
dots()
{
  // Variable threadIdx, blockIdx, and blockDim pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

   //  b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
  b[idx] = v0 + v1 * ax[idx] + v2 * ay[idx];
}

// This routine executes on the CPU.
//
__host__ void
dots_iterate_launch(dim3 dg, dim3 db)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  dots_iterate<<<dg,db>>>();
}

// This routine also executes on the GPU.
//
__global__ void
dots_iterate()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int thread_count = blockDim.x * gridDim.x;
  int elt_per_thread = array_size / thread_count;

  int idx_start = threadIdx.x + blockIdx.x * blockDim.x;

  int idx_start = ( threadIdx.x + blockIdx.x * blockDim.x ) * elt_per_thread;
  int idx_stop = idx_start + elt_per_thread;


  for ( int idx = idx_start; idx < idx_stop; idx++ )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;


  for ( int idx = idx_start; idx < array_size; idx += thread_count )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}

//
// Code for collecting information about kernels (routines above).
//

__host__ hipError_t
kernels_get_attr(struct hipFuncAttributes *attr, char **names, int max_count)
{
  struct hipFuncAttributes *attr_stop = attr + max_count;
  hipError_t er = hipSuccess; // Tentative.
#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er || attr == attr_stop ) { *names = NULL; return er; }
  GETATTR(dots);
  GETATTR(dots_iterate);
  *names = NULL;
  return er;
#undef GETATTR
}

