#include "hip/hip_runtime.h"
#include "stream-2.cuh"
#include <gp/cuda-util-kernel.h>

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ Vector2* a;
__constant__ float* b;

extern "C" __global__ void dots();
extern "C" __global__ void dots_iterate1();
extern "C" __global__ void dots_iterate2();
extern "C" __global__ void dots_iterate3();

// This routine executes on the CPU.
//
__host__ void
dots_launch(dim3 dg, dim3 db)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  dots<<<dg,db>>>();
}

// This routine executes on the GPU.
//
extern "C" __global__ void
dots()
{
  // Variable threadIdx, blockIdx, and blockDim pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}

// This routine executes on the CPU.
//
__host__ void
dots_iterate_launch(dim3 dg, dim3 db, int kernel)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  switch ( kernel ) {
  case 0: dots_iterate1<<<dg,db>>>(); break;
  case 1: dots_iterate2<<<dg,db>>>(); break;
  case 2: dots_iterate3<<<dg,db>>>(); break;
  }
}

// This routine also executes on the GPU.
//
extern "C" __global__ void
dots_iterate1()
{
  // This is the preferred way of iterating over array elements
  // because consecutive threads access consecutive elements.

  int thread_count = blockDim.x * gridDim.x;
  int idx_start = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int idx = idx_start; idx < array_size; idx += thread_count )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}

extern "C" __global__ void
dots_iterate2()
{
  // This method is less efficient.

  int thread_count = blockDim.x * gridDim.x;
  int elt_per_thread = array_size / thread_count;
  int idx_start = ( threadIdx.x + blockIdx.x * blockDim.x ) * elt_per_thread;
  int idx_stop = idx_start + elt_per_thread;

  for ( int idx = idx_start; idx < idx_stop; idx++ )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;

}


// This routine also executes on the GPU.
//
extern "C" __global__ void
dots_iterate3()
{
#define chunk 32
#define degree 4

  const int thread_count = blockDim.x * gridDim.x;
  int idx_start = threadIdx.x + degree * blockIdx.x * blockDim.x;

  for ( int idx = idx_start; idx < array_size; idx += degree * thread_count )
    {
      for ( int i=0; i<degree; i++ )
        b[idx+i*chunk] = v0 + v1 * a[idx+i*chunk].x + v2 * a[idx+i*chunk].y;
    }
}


//
// Code for collecting information about kernels (routines above).
//

__host__ hipError_t
kernels_get_attr(struct hipFuncAttributes *attr, char **names, int max_count)
{
  struct hipFuncAttributes *attr_stop = attr + max_count;
  hipError_t er = hipSuccess; // Tentative.

  CU_SYM(a); CU_SYM(b);
  CU_SYM(array_size);
  CU_SYM(v0); CU_SYM(v1); CU_SYM(v2);

#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er || attr == attr_stop ) { *names = NULL; return er; }
  GETATTR(dots);
  GETATTR(dots_iterate1);
  GETATTR(dots_iterate2);
  GETATTR(dots_iterate3);
  *names = NULL;
  return er;
#undef GETATTR
}

