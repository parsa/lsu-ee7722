
// Simple CUDA Example, without LSU ECE helper classes.

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ float *ax, *ay;
__constant__ float* b;

__global__ void dots();

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }


__host__ int
main(int argc, char** argv)
{
  srand48(1);                   // Seed random number generator.
  const int size_lg = argc > 1 ? atoi(argv[1]) : 10;
  const int host_array_size = 1 << size_lg;

  // Instantiate data arrays.
  //
  float *host_ax = (float*) malloc( host_array_size * sizeof(host_ax[0]) );
  float *host_ay = (float*) malloc( host_array_size * sizeof(host_ay[0]) );
  float *host_b = (float*) malloc( host_array_size * sizeof(host_b[0]) );

  // Allocate on CUDA.

  void *ax_dev, *ay_dev, *b_dev;
  CE(hipMalloc(&ax_dev, host_array_size * sizeof(host_ax[0]) ));
  CE(hipMalloc(&ay_dev, host_array_size * sizeof(host_ay[0]) ));
  CE(hipMalloc(&b_dev, host_array_size * sizeof(host_b[0]) ));

  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ax), &ax_dev, sizeof(ax_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ay), &ay_dev, sizeof(ay_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(b), &b_dev, sizeof(b_dev), 0, hipMemcpyHostToDevice));

  // Initialize input array.
  //
  float xi = drand48(), yi = drand48();
  for ( int i=0; i<host_array_size; i++ )
    {
      host_ax[i] = xi; host_ay[i] = yi;
      xi += 0.1; yi += 0.1;
    }

  // Move input arrays to CUDA.
  //
  CE(hipMemcpy(ax_dev, host_ax, host_array_size * sizeof(host_ax[0]),
                hipMemcpyHostToDevice));
  CE(hipMemcpy(ay_dev, host_ay, host_array_size * sizeof(host_ay[0]),
                hipMemcpyHostToDevice));

  // Initialize coefficients and send them to CUDA.
  //
  const float host_v0 = drand48(), host_v1 = drand48(), host_v2 = drand48();

  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v0), &host_v0, sizeof(host_v0), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v1), &host_v1, sizeof(host_v1), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v2), &host_v2, sizeof(host_v2), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(array_size), &host_array_size, sizeof(host_array_size), 0,
      hipMemcpyHostToDevice));

  // Specify Launch Configuration
  //
  dim3 db, dg;
  db.x = 64;
  db.y = db.z = 1;
  dg.x = int(ceil(double(host_array_size) / db.x));
  dg.y = dg.z = 1;

  // Launch Kernel
  //
  dots<<<dg,db>>>();

  // Retrieve data from CUDA.
  //
  CE(hipMemcpy(host_b, b_dev, host_array_size * sizeof(host_b[0]),
                hipMemcpyDeviceToHost));

  printf("Finished with %d elements, element %d is %.5f\n",
         host_array_size, argc, host_b[argc]);
}

// This routine executes on the GPU.
//
__global__ void
dots()
{
  // Variable threadIdx, blockIdx, and blockDim pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  // Note: this will be improved.
  //
  b[idx] = v0 + v1 * ax[idx] + v2 * ay[idx];
}
