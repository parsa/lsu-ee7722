
// Simple CUDA Example, without LSU ECE helper classes.

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ float *ax, *ay;
__constant__ float* b;

__global__ void dots();

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }


__host__ int
main(int argc, char** argv)
{
  srand48(1);                   // Seed random number generator.
  const int size_lg = argc > 1 ? atoi(argv[1]) : 10;
  const int host_array_size = 1 << size_lg;

  // Allocate storage for CPU copy of data.
  //
  float *host_ax = (float*) malloc( host_array_size * sizeof(host_ax[0]) );
  float *host_ay = (float*) malloc( host_array_size * sizeof(host_ay[0]) );
  float *host_b = (float*) malloc( host_array_size * sizeof(host_b[0]) );

  // Allocate storage for GPU copy of data.
  //
  void *ax_dev, *ay_dev, *b_dev;
  CE(hipMalloc(&ax_dev, host_array_size * sizeof(host_ax[0]) ));
  CE(hipMalloc(&ay_dev, host_array_size * sizeof(host_ay[0]) ));
  CE(hipMalloc(&b_dev, host_array_size * sizeof(host_b[0]) ));

  // Write GPU Scalar Variables
  //
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ax), &ax_dev, sizeof(ax_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ay), &ay_dev, sizeof(ay_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(b), &b_dev, sizeof(b_dev), 0, hipMemcpyHostToDevice));

  // Initialize input array.
  //
  float xi = drand48(), yi = drand48();
  for ( int i=0; i<host_array_size; i++ )
    {
      host_ax[i] = xi; host_ay[i] = yi;
      xi += 0.1; yi += 0.1;
    }

  // Move input arrays to GPU.
  //
  CE(hipMemcpy(ax_dev, host_ax, host_array_size * sizeof(host_ax[0]),
                hipMemcpyHostToDevice));
  CE(hipMemcpy(ay_dev, host_ay, host_array_size * sizeof(host_ay[0]),
                hipMemcpyHostToDevice));

  // Initialize some more variables ...
  //
  const float host_v0 = drand48(), host_v1 = drand48(), host_v2 = drand48();

  // ... and send their values to GPU.
  //
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v0), &host_v0, sizeof(host_v0), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v1), &host_v1, sizeof(host_v1), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v2), &host_v2, sizeof(host_v2), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(array_size), &host_array_size, sizeof(host_array_size), 0,
      hipMemcpyHostToDevice));

  // Specify Launch Configuration
  //
  dim3 db, dg;
  db.x = 64;          // Number of threads per block.
  db.y = db.z = 1;

  dg.x =              // Number of blocks.
    int(ceil(double(host_array_size) / db.x));
  dg.y = dg.z = 1;

  // Launch Kernel
  //
  dots<<<dg,db>>>();

  // Copy data from GPU to CPU.
  //
  CE(hipMemcpy(host_b, b_dev, host_array_size * sizeof(host_b[0]),
                hipMemcpyDeviceToHost));

  printf("Finished with %d elements, element %d is %.5f\n",
         host_array_size, argc, host_b[argc]);
}

// This routine executes on the GPU.
//
__global__ void
dots()
{
  // Variable threadIdx, blockIdx, and blockDim pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  // Note: this will be improved.
  //
  b[idx] = v0 + v1 * ax[idx] + v2 * ay[idx];
}



__global__ void
dots2()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = 
    threadIdx.x
    + threadIdx.y * blockDim.x
    + threadIdx.z * blockDim.x * blockDim.y
    + blockIdx.x * blockDim.x * blockDim.y * blockDim.z
    + blockIdx.y * blockDim.x * blockDim.y * blockDim.z * gridDim.x
    + blockIdx.z * blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y;

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  // Note: this will be improved.
  //
  b[idx] = v0 + v1 * ax[idx] + v2 * ay[idx];
}
