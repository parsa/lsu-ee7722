/// LSU EE 7722 GPU Microarchitecture
//
 /// Simple CUDA Example, without LSU ECE helper classes.

// How to Compile From Emacs
//
//  Within Emacs, as set up for class, compile by pressing [F9].
//
// How to Compile from the Command Line
//
//   Simplest
//     nvcc stream-full.cu
//
//   Reasonable
//     nvcc -o stream-full stream-full.cu  -O3 -g -Xcompiler -Wall

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


 /// Constants holding array sizes, pointers, and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ float *ax, *ay, *b;

 /// Declaration of Kernel (Entry point for code running on GPU.)
//
__global__ void dots();
//
// Note: the attribute __global__ indicates that the procedure is
// started by a kernel launch. A GPU-only procedure would use the
// attribute __device__ and a CPU-only procedure would use the
// attribute __host__.


 /// CUDA API Error-Checking Wrapper
///
#define CE(call) {                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess ) {                                                 \
     printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));  exit(1);}}


__host__ int
main(int argc, char** argv)
{
  srand48(1);                   // Seed random number generator.
  const int size_lg = argc > 1 ? atoi(argv[1]) : 10;  // Arg 1: size of array.
  const int host_array_size = 1 << size_lg;

  // Allocate storage for CPU copy of data.
  //
  float *host_ax = (float*) malloc( host_array_size * sizeof(host_ax[0]) );
  float *host_ay = (float*) malloc( host_array_size * sizeof(host_ay[0]) );
  float *host_b = (float*) malloc( host_array_size * sizeof(host_b[0]) );

  // Allocate storage for GPU copy of data.
  //
  void *ax_dev, *ay_dev, *b_dev;
  CE(hipMalloc(&ax_dev, host_array_size * sizeof(host_ax[0]) ));
  CE(hipMalloc(&ay_dev, host_array_size * sizeof(host_ay[0]) ));
  CE(hipMalloc(&b_dev, host_array_size * sizeof(host_b[0]) ));

  // Write GPU Scalar Variables
  //
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ax), &ax_dev, sizeof(ax_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(ay), &ay_dev, sizeof(ay_dev), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(b), &b_dev, sizeof(b_dev), 0, hipMemcpyHostToDevice));

  // Initialize input array.
  //
  float xi = drand48(), yi = drand48();
  for ( int i=0; i<host_array_size; i++ )
    {
      host_ax[i] = xi; host_ay[i] = yi;
      xi += 0.1; yi += 0.1;
    }

  // Move input arrays to GPU.
  //
  CE(hipMemcpy(ax_dev, host_ax, host_array_size * sizeof(host_ax[0]),
                hipMemcpyHostToDevice));
  CE(hipMemcpy(ay_dev, host_ay, host_array_size * sizeof(host_ay[0]),
                hipMemcpyHostToDevice));

  // Initialize some more variables ...
  //
  const float host_v0 = drand48(), host_v1 = drand48(), host_v2 = drand48();

  // ... and send their values to GPU.
  //
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v0), &host_v0, sizeof(host_v0), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v1), &host_v1, sizeof(host_v1), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(v2), &host_v2, sizeof(host_v2), 0, hipMemcpyHostToDevice));
  CE(hipMemcpyToSymbol
     (HIP_SYMBOL(array_size), &host_array_size, sizeof(host_array_size), 0,
      hipMemcpyHostToDevice));

  // Specify Launch Configuration
  //
  dim3 db, dg;
  db.x = 64;          // Number of threads per block in x dimension.
  db.y = db.z = 1;    // Number of threads per block in y and z dimensions.

  // Choose grid size so that there is at least one thread per array
  // element.
  //
  dg.x = (host_array_size + db.x - 1 ) / db.x;
  dg.y = dg.z = 1;

  // Launch Kernel
  //
  dots<<<dg,db>>>();

  // Copy data from GPU to CPU.
  //
  CE(hipMemcpy(host_b, b_dev, host_array_size * sizeof(host_b[0]),
                hipMemcpyDeviceToHost));

  printf("Finished with %d elements, element %d is %.5f\n",
         host_array_size, argc, host_b[argc]);
}

// This routine executes on the GPU.
//
__global__ void
dots()
{
  // Variables threadIdx, blockIdx, and blockDim are pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  // Perform Computation
  //
  b[idx] = v0 + v1 * ax[idx] + v2 * ay[idx];
}
