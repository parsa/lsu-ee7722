#include "hip/hip_runtime.h"
#include "stream.cuh"


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ Vector2* a;
__constant__ float* b;

__global__ void dots();

// This routine executes on the CPU.
//
__host__ void
dots_launch(dim3 dg, dim3 db)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  dots<<<dg,db>>>();
}

// This routine executes on the GPU.
//
__global__ void
dots()
{
  // Variable threadIdx, blockIdx, and blockDim pre-set.
  //

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  // Note: this will be improved.
  //
  b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}
