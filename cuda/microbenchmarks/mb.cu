#include "hip/hip_runtime.h"
/// LSU EE 7722 - GPU Microarchitecture
//
 /// Microbenchmark Code
 //
 //  Load latency
 //  MADD latency.

#include <stdio.h>
#include <assert.h>
#include <ptable.h>
#include "util.h"
#include "misc.h"

const int unroll_deg = 10;
const int unroll_deg_op_lat = 20;
const int unroll_deg_op_lat2 = 10;

union MultiTypeU
{
  double d;
  int64_t l;
  float f;
  int32_t i;
};


class MultiType
{
public:
  template<typename T>
  __device__ __host__ MultiType(T x, int64_t z){ v.l = z; *this = x; }
  __device__ __host__ MultiType(int64_t l){v.l = l;}
  __device__ __host__ MultiType(){};
  MultiTypeU v;
  __device__ __host__ operator double () { return v.d; }
  __device__ __host__ operator float () { return v.f; }
  __device__ __host__ inline float operator = (float f) { return v.f = f; }
  __device__ __host__ inline double operator = (double d) { return v.d = d; }
  __device__ __host__ operator MultiTypeU () { return v; }
};

struct App
{
  uint array_n_elts;
  uint n_iters;
  MultiType *insn_in_d;
  MultiType *insn_out_d;
  void **a;
  uint *result_d;
  clock_t *wp_time_d, *thd_time_d;
  double madd_coef;
};

App app;
__constant__ App app_c;

#define div_up(a,b) ( b ? ((a)+(b)-1)/(b) : 0 )

string
fmt_Po2(uint64_t val, uint64_t scaled_max = 1023)
{
  const int size_lg = fl1(val);
  vector<const char*> mult( {"  ","ki","Mi","Gi","Ti","Pi","Ei","Zi","Yi"} );
  int tens = max(0,size_lg-1) / 10;
  while ( tens && ( val >> 10 * ( tens - 1 ) ) < scaled_max ) tens--;
  pStringF val_str("%lu %s", val >> 10*tens, mult[tens]);
  return val_str.ss();
}


__global__ void
array_init(uint stride)
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  const uint n_thds = blockDim.x * gridDim.x;
  for ( uint i = tid;  i < app_c.array_n_elts;  i += n_thds )
    app_c.a[i] = &app_c.a[( i + stride ) % app_c.array_n_elts ];
}

typedef void (*KPtr)(void **a);

template<bool ro> __device__ void
mb(void **a)
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int wp_lg = 5;
  const int lane = threadIdx.x & ( (1<<wp_lg) - 1 );
  void** idx = &a[tid];
  clock_t start = clock();
  for ( int i=0; i<app_c.n_iters; i += unroll_deg )
    for ( int j=0; j<unroll_deg; j++ )
      idx = ro ? (void**)__ldg((uint64_t*)idx) : (void**)*idx;
  app_c.result_d[tid] = idx - a;
  if ( lane == 0 )
    app_c.wp_time_d[tid>>wp_lg] = clock() - start;
}

__global__ void mb_g(void **a) {mb<false>(a);}
__global__ void mb_ro(void **a) {mb<true>(a);}

template <typename T>
__global__ void
mb_op_latency()
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int wp_lg = 5;
  const int lane = threadIdx.x & ( (1<<wp_lg) - 1 );
  T accum = app_c.insn_in_d[tid];
  T coef = app_c.madd_coef;

  clock_t delta = 0;

  auto body = [&](int j) { accum += accum * coef; };

#pragma unroll 1
  for ( int i=0; i<app_c.n_iters; i += unroll_deg_op_lat )
    {
      clock_t start = clock();
#pragma unroll unroll_deg_op_lat
      for ( int j=0; j<unroll_deg_op_lat; j++ ) body(j);
      clock_t mid = clock();
#pragma unroll unroll_deg_op_lat2
      for ( int j=0; j<unroll_deg_op_lat2; j++ ) accum += accum * coef;
      clock_t finish = clock();
      delta = mid - start - ( finish - mid );
    }

  if ( lane == 0 ) app_c.wp_time_d[tid>>wp_lg] = delta;
  app_c.insn_out_d[tid] = accum;
}

template<typename T>
MultiType op_latency_body(T accum)
{
  for ( uint i=0; i<app.n_iters; i++ ) accum += accum * T(app.madd_coef);
  return MultiType(accum,0);
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mb_g);
  info.GET_INFO(mb_ro);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}

class MB_Main
{
public:
  MB_Main() { inited = false; };
  bool inited;
  GPU_Info gpu_info;
  int gpu_dev_num;

  GPU_Info& gpu_init()
  {
    if ( !inited )
      {
        inited = true;
        // Determine which GPU to use. (For starters, if there's more than
        // one, choose the one connected to the display.)
        //
        gpu_dev_num = gpu_choose_index();
        CE(hipSetDevice(gpu_dev_num));
        gpu_info_print();
        printf("Using GPU %d\n",gpu_dev_num);
        gpu_info.get_gpu_info(gpu_dev_num);
      }
    return gpu_info;
  }

  void run_op_latency();
  void run_mem_latency(int argc, char **argv);
};


void
MB_Main::run_op_latency()
{
  GPU_Info& info = gpu_init();

  struct FTy { GPU_Info_Func f; char ty; };

  vector<FTy> kernels_op_lat;

  kernels_op_lat.push_back({info.GET_INFO(mb_op_latency<double>).func_ptr,'d'});
  kernels_op_lat.push_back({info.GET_INFO(mb_op_latency<float>).func_ptr,'f'});

  const int num_mp = info.cuda_prop.multiProcessorCount;

  const uint wp_sz = 32;
  const uint max_thds_per_blk = 32 * wp_sz;
  const uint n_blks_max = num_mp;

  const uint max_num_threads = max_thds_per_blk * n_blks_max;
  const uint max_num_wps = max_num_threads / wp_sz;

  const size_t max_out_size_elts = max_num_threads;
  const size_t max_out_size_bytes =
    max_out_size_elts * sizeof(app.insn_out_d[0]);
  const size_t max_wp_time_size_bytes = max_num_wps * sizeof(app.wp_time_d[0]);

  const size_t max_in_size_elts = max_num_threads;
  const size_t max_in_size_bytes = max_in_size_elts * sizeof(app.insn_in_d[0]);

  // Allocate storage for GPU copy of array.
  //
  CE( hipMalloc( &app.insn_in_d,  max_in_size_bytes ) );
  CE( hipMalloc( &app.insn_out_d,  max_out_size_bytes ) );
  CE( hipMalloc( &app.wp_time_d,  max_wp_time_size_bytes ) );

  vector<MultiType> insn_in_h(max_in_size_elts);
  vector<MultiType> insn_out_h(max_out_size_elts);
  vector<clock_t> wp_time_h(max_num_wps);

  const int tot_unroll = unroll_deg_op_lat + unroll_deg_op_lat2;
  const int delta_unroll = unroll_deg_op_lat - unroll_deg_op_lat2;

  app.n_iters = 10 * tot_unroll;
  app.madd_coef = 1.125;

  const int output_width = stdout_width_get();
  const double clock_period_ns = 1e6 / info.cuda_prop.clockRate;

  for ( FTy fty: kernels_op_lat )
    {
      Kernel_Info* const ki = &info.get_info(fty.f);
      pTable table(stdout);

      const double scale_max_latency_multiple_cyc = 10;
      const double preferred_scale_max_latency_cyc = 20;
      double scale_max_latency_cyc = 0;
      printf("Kernel %s:\n", ki->name);
      for ( auto& e: insn_in_h )
        if ( fty.ty == 'f' ) e.v.f = drand48(); else e.v.d = drand48();

      CE( hipMemcpy
          (app.insn_in_d, insn_in_h.data(),
           max_in_size_bytes, hipMemcpyHostToDevice) );

      for ( uint thds_per_blk: {32, 64, 128} )
        {
          const uint n_blks = num_mp;
          const uint n_thds = n_blks * thds_per_blk;
          assert( n_thds <= max_num_threads );

          const size_t out_size_elts = n_thds;
          const size_t out_size_bytes =
            out_size_elts * sizeof(app.insn_out_d[0]);
          const size_t wp_time_size_bytes =
            n_thds / 32 * sizeof(app.wp_time_d[0]);

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( app_c), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          CE(hipMemset(app.insn_out_d,0,out_size_bytes));
          CE(hipMemset(app.wp_time_d,0,wp_time_size_bytes));

          KPtr(ki->func_ptr)<<<n_blks,thds_per_blk>>>(app.a);

          CE( hipDeviceSynchronize() );

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (insn_out_h.data(), app.insn_out_d,
               out_size_bytes, hipMemcpyDeviceToHost) );
          CE( hipMemcpy
              (wp_time_h.data(), app.wp_time_d,
               wp_time_size_bytes, hipMemcpyDeviceToHost) );

          clock_t min_cyc = wp_time_h[0];
          for ( uint i=1; i<n_blks; i++ ) set_min(min_cyc,wp_time_h[i]);
          double latency_cyc = double(min_cyc) / delta_unroll;
          double latency_ns = latency_cyc * clock_period_ns;

          if ( scale_max_latency_cyc == 0 )
            scale_max_latency_cyc =
              max( preferred_scale_max_latency_cyc,
                   scale_max_latency_multiple_cyc *
                   ceil( latency_cyc / scale_max_latency_multiple_cyc ) );

          double frac = latency_cyc / scale_max_latency_cyc;

          pTable_Row row(table);
          table.entry("wps","%3d", thds_per_blk / wp_sz);
          table.entry("nbl","%3d", n_blks);
          table.entry("iter","%7d", app.n_iters);
          table.header_span_start("Latency");
          table.entry("ns", "%4.1f", latency_ns);
          table.entry("cyc", "%4.1f", latency_cyc);

          const int max_st_len =
            max(5, output_width - 1 - table.row_len_get() );
          pStringF fmt("%%-%ds",max_st_len);
          pStringF max_lat_txt("%.0f ns",scale_max_latency_cyc);
          const int pad_total = max_st_len - 4 - max_lat_txt.length();
          const int pad_left = max(0,pad_total/2);
          const int pad_right = max(0,pad_total-pad_left);
          // Note: "<","|" as first character indicates left just, centering.
          string lat_hdr = "||<" + string(pad_left,'-') + max_lat_txt
            + string(pad_right,'-') + ">|";
          table.entry
            (lat_hdr,fmt,
             string( size_t(max(0.0,frac*max_st_len)), '*' ),
             pTable::pT_Left);

          table.header_span_end();

          int err_count = 0;

          for ( uint i=0; i<out_size_elts; i++ )
            {
              MultiTypeU acc = insn_in_h[i];
              MultiTypeU res = insn_out_h[i];
              MultiTypeU expect =
                fty.ty == 'f' ? op_latency_body(acc.f) : op_latency_body(acc.d);
              int64_t mask = ~0x3l;

              if ( expect.l & mask != res.l & mask )
                if ( ++err_count < 5 )
                  printf("Error at elt %u: %#lx != %#lx (correct)\n",
                         i, res.l, expect.l );
            }

          if ( err_count ) printf("Total errors %d\n", err_count);
        }
    }
}

void
MB_Main::run_mem_latency(int argc, char **argv)
{
  GPU_Info& info = gpu_init();

  vector<GPU_Info_Func> kernels_mem_lat;
  kernels_mem_lat.push_back(info.GET_INFO(mb_g).func_ptr);
  kernels_mem_lat.push_back(info.GET_INFO(mb_ro).func_ptr);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( auto f: kernels_mem_lat )
    {
      Kernel_Info& ki = info.get_info(f);
      printf("For %s:\n", ki.name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             ki.cfa.sharedSizeBytes,
             ki.cfa.constSizeBytes,
             ki.cfa.localSizeBytes,
             ki.cfa.numRegs,
             ki.cfa.maxThreadsPerBlock);
    }

  const int num_mp = info.cuda_prop.multiProcessorCount;
  const uint num_mp_po2 = floor2(num_mp);

  const size_t array_bytes = ceil2( 4 * info.cuda_prop.l2CacheSize );
  uint& n_elts = app.array_n_elts = array_bytes / sizeof(app.a[0]);

  const uint min_n_iters = 10000;

  const uint wp_sz = 32;
  const uint wp_per_block = 1;
  const uint n_blks_max = num_mp_po2 * 8;
  const uint thds_per_blk = wp_per_block * wp_sz;

  const uint max_num_threads = thds_per_blk * n_blks_max;
  const uint max_num_wps = max_num_threads / wp_sz;

  const size_t max_out_size_elts = max_num_threads;
  const size_t max_out_size_bytes = max_out_size_elts * sizeof(app.result_d[0]);
  const size_t [[gnu::unused]] max_wp_time_size_bytes =
    max_num_wps * sizeof(app.wp_time_d[0]);
  const size_t max_thd_time_size_bytes =
    max_num_threads * sizeof(app.thd_time_d[0]);

  const size_t in_size_elts = size_t(app.array_n_elts);
  const size_t in_size_bytes = in_size_elts * sizeof( app.a[0] );

  // Allocate storage for GPU copy of array.
  //
  CE( hipMalloc( &app.a,  in_size_bytes ) );
  CE( hipMalloc( &app.result_d,  max_out_size_bytes ) );
  CE( hipMalloc( &app.thd_time_d,  max_thd_time_size_bytes ) );
  app.wp_time_d = app.thd_time_d;

  vector<uint> result_h(max_out_size_elts);
  vector<clock_t> wp_time_h(max_num_wps);

  printf("Array size: %u elts. Block size %u thds.\n",
         app.array_n_elts, thds_per_blk);

  const int output_width = stdout_width_get();

  const int init_num_blocks = num_mp * 2;
  const int init_thds_per_blk = 1024;

  const double clock_period_ns = 1e6 / info.cuda_prop.clockRate;

  const size_t bytes_per_block = thds_per_blk * sizeof(app.a[0]);
  const size_t min_bytes = 1 << 9;

  for ( GPU_Info_Func f: kernels_mem_lat )
    {
      Kernel_Info* const ki = &info.get_info(f);
      pTable table(stdout);

      const double scale_max_latency_multiple_ns = 100;
      const double preferred_scale_max_latency_ns = 500;
      double scale_max_latency_ns = 0;
      printf("Kernel %s:\n", ki->name);

      for ( size_t tot_bytes = array_bytes; tot_bytes >= min_bytes;
            tot_bytes >>= 1 )
        {
          const size_t elt_per_thd_target = 1000;
          const uint n_blks_raw =
            tot_bytes / ( elt_per_thd_target * bytes_per_block );
          const uint n_blks_f = floor2(n_blks_raw);
          const uint n_blks_c = ceil2(n_blks_raw);
          const uint n_blks_raw_r =
            n_blks_raw - n_blks_f < n_blks_c - n_blks_raw
            ? n_blks_f : n_blks_c;
          const uint n_blks = min(max(1,n_blks_raw_r),n_blks_max);
          const uint n_thds = n_blks * thds_per_blk;

          // In units of elements.
          const size_t stride = n_thds * array_bytes / tot_bytes;
          if ( stride < n_thds ) break;
          assert( n_elts % stride == 0 );

          const int elt_per_thd = n_elts / stride;
          const int elt_per_blk = elt_per_thd * thds_per_blk;
          const size_t touched_total_bytes =
            n_blks * elt_per_blk * sizeof(app.a[0]);
          assert( touched_total_bytes == tot_bytes );

          app.n_iters = max(min_n_iters,elt_per_thd*unroll_deg);

          const size_t out_size_elts = n_thds;
          const size_t out_size_bytes = out_size_elts * sizeof(app.result_d[0]);
          const size_t wp_time_size_bytes =
            n_thds / 32 * sizeof(app.wp_time_d[0]);

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( app_c), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          CE(hipMemset(app.result_d,0,out_size_bytes));
          CE(hipMemset(app.wp_time_d,0,wp_time_size_bytes));

          array_init<<<init_num_blocks,init_thds_per_blk>>>(stride);

          KPtr(ki->func_ptr)<<<n_blks,thds_per_blk>>>(app.a);

          CE( hipDeviceSynchronize() );

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (result_h.data(), app.result_d,
               out_size_bytes, hipMemcpyDeviceToHost) );
          CE( hipMemcpy
              (wp_time_h.data(), app.wp_time_d,
               wp_time_size_bytes, hipMemcpyDeviceToHost) );

          clock_t min_cyc = wp_time_h[0];
          for ( uint i=1; i<n_blks; i++ ) set_min(min_cyc,wp_time_h[i]);
          double self_time_ns = min_cyc * clock_period_ns;
          double latency_ns = self_time_ns / app.n_iters;
          double latency_cyc = double(min_cyc) / app.n_iters;

          if ( scale_max_latency_ns == 0 )
            scale_max_latency_ns =
              max( preferred_scale_max_latency_ns,
                   scale_max_latency_multiple_ns *
                   ceil( latency_ns / scale_max_latency_multiple_ns ) );

          double frac = latency_ns / scale_max_latency_ns;

          pTable_Row row(table);
          table.entry("nbl","%3d", n_blks);
          table.entry("iter","%7d", app.n_iters);
          table.header_span_start("Data Touched");
          table.entry
            ( "Block", "%7sB",
              fmt_Po2(elt_per_blk*sizeof(app.a[0])), pTable::pT_Right );
          table.entry
            ( "Total", "%8sB",
              fmt_Po2( touched_total_bytes), pTable::pT_Right );
          table.header_span_end();
          table.header_span_start("Latency");
          table.entry("ns", "%4.0f", latency_ns);
          table.entry("cyc", "%4.0f", latency_cyc);

          const int max_st_len =
            max(5, output_width - 1 - table.row_len_get() );
          pStringF fmt("%%-%ds",max_st_len);
          pStringF max_lat_txt("%.0f ns",scale_max_latency_ns);
          const int pad_total = max_st_len - 4 - max_lat_txt.length();
          const int pad_left = max(0,pad_total/2);
          const int pad_right = max(0,pad_total-pad_left);
          // Note: "<","|" as first character indicates left just, centering.
          string lat_hdr = "||<" + string(pad_left,'-') + max_lat_txt
            + string(pad_right,'-') + ">|";
          table.entry
            (lat_hdr,fmt,
             string( size_t(max(0.0,frac*max_st_len)), '*' ),
             pTable::pT_Left);

          table.header_span_end();

          int err_count = 0;
          const uint n_elt_mask = n_elts - 1;
          for ( uint i=0; i<out_size_elts; i++ )
            {
              if ( i % 32 ) continue;
              const uint expect = ( i + app.n_iters * stride ) & n_elt_mask;

              if ( expect != result_h[i] )
                {
                  err_count++;
                  if ( err_count < 5 )
                    printf("Error at elt %u: %u != %u (correct)\n",
                           i, result_h[i], expect );
                }
            }
          if ( err_count ) printf("Total errors %d\n", err_count);
        }
    }
}

int
main(int argc, char **argv)
{
  MB_Main mb_main;

  mb_main.run_op_latency();
  mb_main.run_mem_latency(argc,argv);

  return 0;
}
