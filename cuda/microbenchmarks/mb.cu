#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <ptable.h>
#include "util.h"
#include "misc.h"

/// LSU EE 7722 - GPU Microarchitecture
//
 /// Microbenchmark Code
 //
 //  Load latency


const int unroll_deg = 10;

struct App
{
  uint array_n_elts;
  uint n_iters;
  void **a;
  uint *result_d;
  clock_t *wp_time_d;
};

App app;
__constant__ App app_c;

#define div_up(a,b) ( b ? ((a)+(b)-1)/(b) : 0 )

string
fmt_Po2(uint64_t val, uint64_t scaled_max = 1023)
{
  const int size_lg = fl1(val);
  vector<const char*> mult( {"  ","ki","Mi","Gi","Ti","Pi","Ei","Zi","Yi"} );
  int tens = max(0,size_lg-1) / 10;
  while ( tens && ( val >> 10 * ( tens - 1 ) ) < scaled_max ) tens--;
  pStringF val_str("%lu %s", val >> 10*tens, mult[tens]);
  return val_str.ss();
}


__global__ void
array_init(uint stride)
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  const uint n_thds = blockDim.x * gridDim.x;
  for ( uint i = tid;  i < app_c.array_n_elts;  i += n_thds )
    app_c.a[i] = &app_c.a[( i + stride ) % app_c.array_n_elts ];
}

typedef void (*KPtr)(void **a);

template<bool ro> __device__ void
mb(void **a)
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int wp_lg = 5;
  const int lane = threadIdx.x & ( (1<<wp_lg) - 1 );
  void** idx = &a[tid];
  clock_t start = clock();
  for ( int i=0; i<app_c.n_iters; i += unroll_deg )
    for ( int j=0; j<unroll_deg; j++ )
      idx = ro ? (void**)__ldg((uint64_t*)idx) : (void**)*idx;
  app_c.result_d[tid] = idx - a;
  if ( lane == 0 )
    app_c.wp_time_d[tid>>wp_lg] = clock() - start;
}

__global__ void mb_g(void **a) {mb<false>(a);}
__global__ void mb_ro(void **a) {mb<true>(a);}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mb_g);
  info.GET_INFO(mb_ro);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;
  const uint num_mp_po2 = floor2(num_mp);

  const size_t array_bytes = ceil2( 4 * info.cuda_prop.l2CacheSize );
  uint& n_elts = app.array_n_elts = array_bytes / sizeof(app.a[0]);

  const uint min_n_iters = 10000;

  const uint wp_sz = 32;
  const uint wp_per_block = 1;
  const uint n_blks_max = num_mp_po2 * 8;
  const uint thds_per_blk = wp_per_block * wp_sz;

  const uint max_num_threads = thds_per_blk * n_blks_max;
  const uint max_num_wps = max_num_threads / wp_sz;

  const size_t max_out_size_elts = max_num_threads;
  const size_t max_out_size_bytes = max_out_size_elts * sizeof(app.result_d[0]);
  const size_t max_wp_time_size_bytes = max_num_wps * sizeof(app.wp_time_d[0]);

  const size_t in_size_elts = size_t(app.array_n_elts);
  const size_t in_size_bytes = in_size_elts * sizeof( app.a[0] );

  // Allocate storage for GPU copy of array.
  //
  CE( hipMalloc( &app.a,  in_size_bytes ) );
  CE( hipMalloc( &app.result_d,  max_out_size_bytes ) );
  CE( hipMalloc( &app.wp_time_d,  max_wp_time_size_bytes ) );

  vector<uint> result_h(max_out_size_elts);
  vector<clock_t> wp_time_h(max_num_wps);

  printf("Array size: %u elts. Block size %u thds.\n",
         app.array_n_elts, thds_per_blk);

  const int output_width = stdout_width_get();

  const int init_num_blocks = num_mp * 2;
  const int init_thds_per_blk = 1024;

  const double clock_period_ns = 1e6 / info.cuda_prop.clockRate;

  const size_t bytes_per_block = thds_per_blk * sizeof(app.a[0]);
  const size_t min_bytes = 1 << 9;

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      hipFuncAttributes& cfa = info.ki[kernel].cfa;

      pTable table;
      table.stream = stdout;

      const double scale_max_latency_multiple_ns = 100;
      const double preferred_scale_max_latency_ns = 500;
      double scale_max_latency_ns = 0;
      printf("Kernel %s:\n", info.ki[kernel].name);

      for ( size_t tot_bytes = array_bytes; tot_bytes >= min_bytes;
            tot_bytes >>= 1 )
        {
          const size_t elt_per_thd_target = 1000;
          const uint n_blks_raw =
            tot_bytes / ( elt_per_thd_target * bytes_per_block );
          const uint n_blks_f = floor2(n_blks_raw);
          const uint n_blks_c = ceil2(n_blks_raw);
          const uint n_blks_raw_r =
            n_blks_raw - n_blks_f < n_blks_c - n_blks_raw
            ? n_blks_f : n_blks_c;
          const uint n_blks = min(max(1,n_blks_raw_r),n_blks_max);
          const uint n_thds = n_blks * thds_per_blk;

          // In units of elements.
          const size_t stride = n_thds * array_bytes / tot_bytes;
          if ( stride < n_thds ) break;
          assert( n_elts % stride == 0 );

          const int elt_per_thd = n_elts / stride;
          const int elt_per_blk = elt_per_thd * thds_per_blk;
          const size_t touched_total_bytes =
            n_blks * elt_per_blk * sizeof(app.a[0]);
          assert( touched_total_bytes == tot_bytes );

          app.n_iters = max(min_n_iters,elt_per_thd*unroll_deg);

          const size_t out_size_elts = n_thds;
          const size_t out_size_bytes = out_size_elts * sizeof(app.result_d[0]);
          const size_t wp_time_size_bytes =
            n_thds / 32 * sizeof(app.wp_time_d[0]);

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( app_c), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          CE(hipMemset(app.result_d,0,out_size_bytes));
          CE(hipMemset(app.wp_time_d,0,wp_time_size_bytes));

          array_init<<<init_num_blocks,init_thds_per_blk>>>(stride);

          KPtr(info.ki[kernel].func_ptr)<<<n_blks,thds_per_blk>>>(app.a);

          CE( hipDeviceSynchronize() );

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (result_h.data(), app.result_d,
               out_size_bytes, hipMemcpyDeviceToHost) );
          CE( hipMemcpy
              (wp_time_h.data(), app.wp_time_d,
               wp_time_size_bytes, hipMemcpyDeviceToHost) );

          clock_t min_cyc = wp_time_h[0];
          for ( uint i=1; i<n_blks; i++ ) set_min(min_cyc,wp_time_h[i]);
          double self_time_ns = min_cyc * clock_period_ns;
          double latency_ns = self_time_ns / app.n_iters;
          double latency_cyc = double(min_cyc) / app.n_iters;

          if ( scale_max_latency_ns == 0 )
            scale_max_latency_ns =
              max( preferred_scale_max_latency_ns,
                   scale_max_latency_multiple_ns *
                   ceil( latency_ns / scale_max_latency_multiple_ns ) );

          double frac = latency_ns / scale_max_latency_ns;

          table.row_start();
          table.entry("nbl","%3d", n_blks);
          table.entry("iter","%7d", app.n_iters);
          table.header_span_start("Data Touched");
          table.entry
            ( "Block", "%7sB",
              fmt_Po2(elt_per_blk*sizeof(app.a[0])), pTable::pT_Right );
          table.entry
            ( "Total", "%8sB",
              fmt_Po2( touched_total_bytes), pTable::pT_Right );
          table.header_span_end();
          table.header_span_start("Latency");
          table.entry("ns", "%4.0f", latency_ns);
          table.entry("cyc", "%4.0f", latency_cyc);

          const int max_st_len =
            max(5, output_width - 1 - table.row_len_get() );
          pStringF fmt("%%-%ds",max_st_len);
          pStringF max_lat_txt("%.0f ns",scale_max_latency_ns);
          const int pad_total = max_st_len - 4 - max_lat_txt.length();
          const int pad_left = max(0,pad_total/2);
          const int pad_right = max(0,pad_total-pad_left);
          // Note: "<","|" as first character indicates left just, centering.
          string lat_hdr = "||<" + string(pad_left,'-') + max_lat_txt
            + string(pad_right,'-') + ">|";
          table.entry
            (lat_hdr,fmt,
             string( size_t(max(0.0,frac*max_st_len)), '*' ),
             pTable::pT_Left);

          table.header_span_end();

          int err_count = 0;
          const uint n_elt_mask = n_elts - 1;
          for ( uint i=0; i<out_size_elts; i++ )
            {
              const uint expect = ( i + app.n_iters * stride ) & n_elt_mask;

              if ( expect != result_h[i] )
                {
                  err_count++;
                  if ( err_count < 5 )
                    printf("Error at elt %u: %u != %u (correct)\n",
                           i, result_h[i], expect );
                }
            }
          if ( err_count )
            printf("Total errors %d\n", err_count);
        }
    }
}
