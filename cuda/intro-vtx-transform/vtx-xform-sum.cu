#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include "util.h"

#define N 4

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(16) Vertex
{
  Elt_Type __align__(16) a[N];
};

struct __align__(16) V4 {
  Elt_Type x, y, z, w;
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
  V4 * d_v4_in;

  Elt_Type *thd_sum;
  Elt_Type *d_thd_sum;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

#define BLOCK_SIZE_MAX 1024
__shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];
__shared__ Elt_Type wshared_sum[32];

extern "C" __global__ void reduce_method_0();
extern "C" __global__ void reduce_method_1();
extern "C" __global__ void reduce_method_2();
extern "C" __global__ void reduce_method_3();
extern "C" __global__ void reduce_method_4();

__device__ Elt_Type d_sum_sum_of_sq;

__device__ Elt_Type
cuda_vtx_xform()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Compute element number to start at.
  //
  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = d_app.num_threads;

  Elt_Type sum_sum_of_sq = 0;

  for ( int h=start; h<stop; h += inc )
    {
      V4 p2 = d_app.d_v4_in[h];
      Vertex p;
      p.a[0] = p2.x; p.a[1] = p2.y; p.a[2] = p2.z; p.a[3] = p2.w;
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;

      //  Compute the magnitude squared of q and update "sum" variable.
      //
      Elt_Type sum_of_sq = 0;
      for ( int i=0; i<N; i++ ) sum_of_sq += q.a[i] * q.a[i];

      sum_sum_of_sq += sum_of_sq;
    }

  return sum_sum_of_sq;
}


extern "C" __global__ void
reduce_method_0()
{
  // No reduction performed by GPU.
  //
  // Each thread computes its own sum and writes it to global memory.
  // CPU computes sum of BG elements.
  //
  // Where: B is threads per block (blockDim.x)
  //        G is number of blocks (gridDim.x)
  //        Total number of threads is BG.

  const Elt_Type thd_sum = cuda_vtx_xform();
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  d_app.d_thd_sum[tid] = thd_sum;
}

extern "C" __global__ void
reduce_method_1()
{
  // Linear reduction by one thread per block.
  //
  // Each thread computes its own sum and writes it to shared memory.
  // One thread per block computes a block-wide sum and write it to global mem.
  //
  // G CPU threads each compute sum of B elements.
  // CPU computes sum of G elements.

  const Elt_Type thd_sum = cuda_vtx_xform();

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to thread 0.
  __syncthreads();

  if ( threadIdx.x != 0 ) return;

  Elt_Type our_sum = thd_sum;

  for ( int i=1; i<blockDim.x; i++ )
    our_sum += shared_sum[i];

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_atomic_sum_block()
{
  // Use an atomic add operating on a shared variable.

  const Elt_Type thd_sum = cuda_vtx_xform();
  __shared__ Elt_Type our_sum;

  if ( threadIdx.x == 0 ) our_sum = 0;

  __syncthreads();

  atomicAdd( &our_sum, thd_sum );

  __syncthreads();

  if ( threadIdx.x == 0 ) d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_atomic_sum_grid()
{
  // Use an atomic add operating on a global variable.

  const Elt_Type thd_sum = cuda_vtx_xform();

  atomicAdd( &d_app.d_thd_sum[0], thd_sum );

}

extern "C" __global__ void
reduce_method_2()
{
  // Use a reduction tree.
  //
  // By using a complete tree reduction we perform the minimum
  // number of adds. That's the good news. The bad news is that
  // we need to synchronize each iteration.
  //

  const Elt_Type thd_sum = cuda_vtx_xform();

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  for ( int dist = blockDim.x >> 1;  dist;  dist >>= 1 )
    {
      __syncthreads();  // Wait for other threads to finish.
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }
    }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_method_3()
{
  // A mixture of a linear sum and a tree reduction, chosen so
  // that only a single warp of threads participates.  Since only
  // a single warp is executing, no synchronizations are necessary
  // other than the one after the thread's initial sum is written.

  const Elt_Type thd_sum = cuda_vtx_xform();

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  __syncthreads();

  // In the code below, only the first 32 threads do something useful.

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  // Perform a linear sum.
  //
  // The first 32 threads each compute their own sum.
  //
  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      our_sum += shared_sum[i];

  shared_sum[threadIdx.x] = our_sum;

  // Perform a tree reduction.
  //
  // The first 32 threads perform a tree reduction of the linear sums
  // found in the previous step.
  //
  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_method_4()
{
  const Elt_Type thd_sum = cuda_vtx_xform();

  shared_sum[threadIdx.x] = thd_sum;

  Elt_Type our_sum = thd_sum;

  const int warp_lg = 5;
  const int warp_size = 1 << warp_lg;
  const int half_warp_size = warp_size >> 1;
  const int lane = threadIdx.x & ( warp_size - 1 );

  for ( int dist = half_warp_size;  dist;  dist >>= 1 )
    if ( lane < dist )
      {
        our_sum += shared_sum[ threadIdx.x + dist ];
        shared_sum[ threadIdx.x ] = our_sum;
      }

#if 1

  const int warp_num = threadIdx.x >> warp_lg;

  if ( !lane )
    wshared_sum[ warp_num ] = our_sum;

  __syncthreads();

  Elt_Type our_osum = 0;

  if ( threadIdx.x < half_warp_size )
    {
      our_osum = wshared_sum[ threadIdx.x ];

      for ( int dist = half_warp_size;  dist;  dist >>= 1 )
        if ( lane < dist )
          {
            our_osum += wshared_sum[ threadIdx.x + dist ];
            if ( dist > 1 )
              wshared_sum[ threadIdx.x ] = our_osum;
          }

    }

#else

  __syncthreads();

  const int our_elt = threadIdx.x << warp_lg;
  const int block_size = blockDim.x;
  Elt_Type our_osum = threadIdx.x < warp_size ? shared_sum[ our_elt ] : 0;

  for ( int dist = block_size >> 1;  dist >= warp_size;  dist >>= 1 )
      if ( our_elt < dist )
        {
          our_osum += shared_sum[ our_elt + dist ];
          shared_sum[ our_elt ] = our_osum;
        }

#endif

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_osum;
}

extern "C" __global__ void
reduce_method_5()
{
  const Elt_Type thd_sum = cuda_vtx_xform();

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  __syncthreads();

  // In the code below, only the first 32 threads do something useful.

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  // Perform a linear sum.
  //
  // The first 32 threads each compute their own sum.
  //
  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      our_sum += shared_sum[i];

  __all ( our_sum > 1.23 );

  // Perform a tree reduction.
  //
  // The first 32 threads perform a tree reduction of the linear sums
  // found in the previous step.
  //
  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(reduce_method_0);
  info.GET_INFO(reduce_method_1);
  info.GET_INFO(reduce_atomic_sum_block);
  info.GET_INFO(reduce_atomic_sum_grid);
  info.GET_INFO(reduce_method_2);
  info.GET_INFO(reduce_method_3);
  info.GET_INFO(reduce_method_4);
  //  info.GET_INFO(reduce_method_5);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Examine argument 1, block count.
  //
  const int arg1_int = argc < 2 ? 3 * 16 : atoi(argv[1]);
  const int num_blocks = abs(arg1_int);

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 1024 : atoi(argv[2]);
  app.num_threads = num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  const int sum_array_size = app.num_threads;

  if ( app.num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );
  app.d_v4_in = (V4*) app.d_v_in;

  //  Allocate storage on CPU and GPU for the minimum magnitude (sq) and
  //  its index.
  //
  app.thd_sum = new Elt_Type[sum_array_size];
  CE( hipMalloc( &app.d_thd_sum, sum_array_size * sizeof(Elt_Type) ) );

  // Initialize device memory to zeros. Helps catch bugs.
  //
  CE( hipMemset( app.d_thd_sum, 0, sum_array_size*sizeof(Elt_Type) ) );

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  // Compute correct answer.
  double cpu_grand_sum = 0;

  for ( int h=0; h<app.array_size; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      Elt_Type sos = 0; for(int i=0; i<N; i++ ) sos+= q.a[i]*q.a[i];
      cpu_grand_sum += sos;
    }

    {
      // Prepare events used for timing.
      //
      hipEvent_t gpu_start_ce, gpu_stop_ce;
      CE(hipEventCreate(&gpu_start_ce));
      CE(hipEventCreate(&gpu_stop_ce));

      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      // Launch kernel multiple times and keep track of the best time.
      printf("Launching with %d blocks of %d threads ...\n",
             num_blocks, thd_per_block);

      for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
        {
          const int samples = 10;
          double elapsed_time_s = 86400; // Reassigned to minimum run time.
          double elapsed_time_sum = 0;

          for ( int s=0; s<samples; s++ )
            {

              CE(hipMemset(app.d_thd_sum,0,sizeof(Elt_Type)*sum_array_size));

              // Measure execution time starting "now", which is after data
              // set to GPU.
              //
              CE(hipEventRecord(gpu_start_ce,0));

              // Launch Kernel
              //
              info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

              // Stop measuring execution time now, which is before is data
              // returned from GPU.
              //
              CE(hipEventRecord(gpu_stop_ce,0));
              CE(hipEventSynchronize(gpu_stop_ce));
              float cuda_time_ms = -1.1;
              CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

              const double this_elapsed_time_s = cuda_time_ms * 0.001;
              if ( s ) elapsed_time_sum += this_elapsed_time_s;
              elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);
            }

          printf("%-26s   %11.3f µs min, %11.3f µs avg.\n",
                 info.ki[kernel].name,
                 elapsed_time_s * 1e6,
                 elapsed_time_sum * 1e6 / (samples-1));

          //  Copy back per-thread sums.
          //
          CE( hipMemcpy
              ( app.thd_sum, app.d_thd_sum,
                sizeof(Elt_Type) * sum_array_size, hipMemcpyDeviceToHost) );

          const int sum_array_used = kernel == 0 ? app.num_threads : num_blocks;

          // Find the sum of each thread or block's sum.
          //
          Elt_Type grand_sum = 0;
          for ( int i=0; i<sum_array_used; i++ )
            grand_sum += app.thd_sum[i];

          Elt_Type diff = fabs(grand_sum-cpu_grand_sum) / app.array_size;
          const bool correct = diff < 1e-5;
          if ( !correct )
          printf
            ("Sum is %s,  %.1f %s %.1f (correct)\n",
             correct ? "correct" : "**wrong**",
             grand_sum,
             grand_sum == cpu_grand_sum ? "==" : diff < 1e-5 ? "~" : "!=",
             cpu_grand_sum
             );
        }
    }

  // Copy output array from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );
  
}
