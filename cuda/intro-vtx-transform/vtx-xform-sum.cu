#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "util.h"
using namespace std;

// Make it easy to switch between different types for array elements.
// Note: Using a float can cause rounding errors in reduce_iter_atomic_grid.
//
typedef int Elt_Type;


struct App
{
  int array_size;
  Elt_Type *v_in;
  Elt_Type *d_v_in;

  Elt_Type *thd_sum;
  Elt_Type *d_thd_sum;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

#define BLOCK_SIZE_MAX 1024


__device__ Elt_Type
reduce_thread()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  // Unroll degree.
  const int deg = 8;

  // Compute element number to start at.
  //
  const int wp_lg = 5;
  const int wp_msk = ( 1 << wp_lg ) - 1;
  const int lane = tid & wp_msk;

  const int start = ( tid - lane ) * deg + lane;
  const int stop = d_app.array_size;
  const int inc = num_threads * deg;

  Elt_Type thd_sum = 0;

  for ( int h=start; h<stop; h += inc )
    for ( int i=0; i<deg; i++ )
      thd_sum += d_app.d_v_in[h+(i<<wp_lg)];

  return thd_sum;
}

extern "C" __global__ void
reduce_iter_atomic_grid()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  // Unroll degree.
  const int deg = 8;

  // Compute element number to start at.
  //
  const int wp_lg = 5;
  const int wp_msk = ( 1 << wp_lg ) - 1;
  const int lane = tid & wp_msk;

  const int start = ( tid - lane ) * deg + lane;
  const int stop = d_app.array_size;
  const int inc = num_threads * deg;

  for ( int h=start; h<stop; h += inc )
    for ( int i=0; i<deg; i++ )
      atomicAdd( &d_app.d_thd_sum[blockIdx.x], d_app.d_v_in[h+(i<<wp_lg)] );
}


extern "C" __global__ void
reduce_per_thd()
{
  // No reduction performed by GPU.
  //
  // Each thread computes its own sum and writes it to global memory.
  // CPU computes sum of BG elements.
  //
  // Where: B is threads per block (blockDim.x)
  //        G is number of blocks (gridDim.x)
  //        Total number of threads is BG.

  const Elt_Type thd_sum = reduce_thread();
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  d_app.d_thd_sum[tid] = thd_sum;
}

extern "C" __global__ void
reduce_per_blk_1thd()
{
  // Linear reduction by one thread per block.
  //
  // Each thread computes its own sum and writes it to shared memory.
  // One thread per block computes a block-wide sum and write it to global mem.
  //
  // G CPU threads each compute sum of B elements.
  // CPU computes sum of G elements.

  const Elt_Type thd_sum = reduce_thread();
  __shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to thread 0.
  __syncthreads();

  if ( threadIdx.x != 0 ) return;

  Elt_Type our_sum = thd_sum;

  for ( int i=1; i<blockDim.x; i++ )
    our_sum += shared_sum[i];

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_thd_atomic_blk()
{
  // Use an atomic add operating on a shared variable.

  const Elt_Type thd_sum = reduce_thread();
  __shared__ Elt_Type our_sum;

  if ( threadIdx.x == 0 ) our_sum = 0;

  __syncthreads();

  atomicAdd( &our_sum, thd_sum );

  __syncthreads();

  if ( threadIdx.x == 0 ) d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_thd_atomic_grd()
{
  // Use an atomic add operating on a global variable.

  const Elt_Type thd_sum = reduce_thread();

  atomicAdd( &d_app.d_thd_sum[0], thd_sum );

}

extern "C" __global__ void
reduce_thd_tree_blk()
{
  // Use a reduction tree.
  //
  // By using a complete tree reduction we perform the minimum
  // number of adds. That's the good news. The bad news is that
  // we need to synchronize each iteration.
  //

  const Elt_Type thd_sum = reduce_thread();
  __shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  for ( int dist = blockDim.x >> 1;  dist;  dist >>= 1 )
    {
      __syncthreads();  // Wait for other threads to finish.
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }
    }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_wp_lin_tree_blk()
{
  // A mixture of a linear sum and a tree reduction, chosen so
  // that only a single warp of threads participates.  Since only
  // a single warp is executing, no synchronizations are necessary
  // other than the one after the thread's initial sum is written.

  const Elt_Type thd_sum = reduce_thread();

  volatile __shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  __syncthreads();

  // In the code below, only the first 32 threads do something useful.

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  // Perform a linear sum.
  //
  // The first 32 threads each compute their own sum.
  //
  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      our_sum += shared_sum[i];

  shared_sum[threadIdx.x] = our_sum;

  // Perform a tree reduction.
  //
  // The first 32 threads perform a tree reduction of the linear sums
  // found in the previous step.
  //
  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

extern "C" __global__ void
reduce_thd_tree_wp_tree_blk()
{
  const Elt_Type thd_sum = reduce_thread();

  volatile __shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];

  shared_sum[threadIdx.x] = thd_sum;

  Elt_Type our_sum = thd_sum;

  const int warp_lg = 5;
  const int warp_size = 1 << warp_lg;
  const int half_warp_size = warp_size >> 1;
  const int lane = threadIdx.x & ( warp_size - 1 );

  // Each warp computes a sum of its elements using a reduction tree.
  //
  for ( int dist = half_warp_size;  dist;  dist >>= 1 )
    if ( lane < dist )
      {
        our_sum += shared_sum[ threadIdx.x + dist ];
        shared_sum[ threadIdx.x ] = our_sum;
      }

#if 1

  const int warp_num = threadIdx.x >> warp_lg;
  volatile __shared__ Elt_Type wshared_sum[32];

  // Note: Only needed if blockDim.x < 1024.
  if ( warp_num == 0 ) wshared_sum[ threadIdx.x ] = 0;
  __syncthreads();

  // Store per-warp sum in shared memory.
  //
  if ( !lane ) wshared_sum[ warp_num ] = our_sum;

  __syncthreads();

  Elt_Type our_osum = 0;

  if ( threadIdx.x < half_warp_size )
    {
      // Use a reduction tree for the per-warp sums.

      our_osum = wshared_sum[ threadIdx.x ];

      for ( int dist = half_warp_size;  dist;  dist >>= 1 )
        if ( lane < dist )
          {
            our_osum += wshared_sum[ threadIdx.x + dist ];
            wshared_sum[ threadIdx.x ] = our_osum;
          }
    }

#else

  __syncthreads();

  const int our_elt = threadIdx.x << warp_lg;
  const int block_size = blockDim.x;
  Elt_Type our_osum = threadIdx.x < warp_size ? shared_sum[ our_elt ] : 0;

  for ( int dist = block_size >> 1;  dist >= warp_size;  dist >>= 1 )
      if ( our_elt < dist )
        {
          our_osum += shared_sum[ our_elt + dist ];
          shared_sum[ our_elt ] = our_osum;
        }

#endif

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_osum;
}

extern "C" __global__ void
reduce_method_5()
{
  const Elt_Type thd_sum = reduce_thread();
  __shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  __syncthreads();

  // In the code below, only the first 32 threads do something useful.

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  // Perform a linear sum.
  //
  // The first 32 threads each compute their own sum.
  //
  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      our_sum += shared_sum[i];

  __all ( our_sum > 1.23 );

  // Perform a tree reduction.
  //
  // The first 32 threads perform a tree reduction of the linear sums
  // found in the previous step.
  //
  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(reduce_per_thd);
  info.GET_INFO(reduce_per_blk_1thd);
  info.GET_INFO(reduce_iter_atomic_grid);
  info.GET_INFO(reduce_thd_atomic_blk);
  info.GET_INFO(reduce_thd_atomic_grd);
  info.GET_INFO(reduce_thd_tree_blk);
  info.GET_INFO(reduce_wp_lin_tree_blk);
  info.GET_INFO(reduce_thd_tree_wp_tree_blk);
  //  info.GET_INFO(reduce_method_5);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 1024 : atoi(argv[2]);
  const int num_threads = num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 24 : int( atof(argv[3]) * (1<<24) );

  const int sum_array_size = num_threads;

  if ( num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int max_unroll_deg = 32;
  const int overrun_elts = 32 * max_unroll_deg;
  const int array_size_elts = app.array_size + overrun_elts;
  const int array_size_bytes = array_size_elts * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Elt_Type[array_size_elts];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  array_size_bytes ) );

  //  Allocate storage on CPU and GPU for the sums. The number of elements
  //  used for the sum can vary.
  //
  app.thd_sum = new Elt_Type[sum_array_size];
  CE( hipMalloc( &app.d_thd_sum, sum_array_size * sizeof(app.d_thd_sum[0]) ) );

  // Initialize input array. Set overrun area to zero.
  //
  //  for ( int i=0; i<app.array_size; i++ ) app.v_in[i] = 2 * drand48();
  for ( int i=0; i<app.array_size; i++ ) app.v_in[i] = 1;
  for ( int i=app.array_size+1; i<array_size_elts; i++ ) app.v_in[i] = 0;

  // Compute correct answer.
  double cpu_grand_sum = 0;

  for ( int h=0; h<app.array_size; h++ ) cpu_grand_sum += app.v_in[h];

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of %d threads for %d elts ...\n",
           num_blocks, thd_per_block, app.array_size);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        const int sum_array_used = kernel == 0 ? num_threads : num_blocks;
        const int samples = 20;
        const int hsamp = samples/2;
        vector<double> elapsed_times;

        for ( int s=0; s<samples; s++ )
          {
            CE(hipMemset(app.d_thd_sum,0,sizeof(Elt_Type)*sum_array_used));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;
            elapsed_times.push_back( this_elapsed_time_s );
          }

        sort(elapsed_times.begin(),elapsed_times.end());
        const double elapsed_time_s = elapsed_times[hsamp];
        const double err_s = elapsed_times[hsamp+1]-elapsed_time_s;
        const double err = err_s/elapsed_time_s;

        printf("%-27s   %8.0f µs min, err %5.1f µs %3.1f%%.\n",
               info.ki[kernel].name,
               elapsed_time_s * 1e6, err_s * 1e6, 100 * err);

        //  Copy back per-thread sums.
        //
        CE( hipMemcpy
            ( app.thd_sum, app.d_thd_sum,
              sizeof(Elt_Type) * sum_array_used, hipMemcpyDeviceToHost) );

        // Find the sum of each thread or block's sum.
        //
        double grand_sum = 0;
        for ( int i=0; i<sum_array_used; i++ )
          grand_sum += app.thd_sum[i];

        double diff = fabs(grand_sum-cpu_grand_sum) / app.array_size;
        const bool correct = diff <= 1e-5;
        if ( !correct )
          printf
            ("Kernel above sum is %s,  %.1f %s %.1f (correct)\n",
             correct ? "correct" : "**wrong**",
             grand_sum,
             grand_sum == cpu_grand_sum ? "==" : diff < 1e-5 ? "~" : "!=",
             cpu_grand_sum);
      }
  }
}
