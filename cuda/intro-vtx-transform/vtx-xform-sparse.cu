#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

#define N 16
#define M 16

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef double Elt_Type;

typedef uint32_t Op_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check, *h_out_check_n;
  Op_Type norm_threshold;
  Op_Type *h_op;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;
  Op_Type *d_op;

};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);

__device__ __host__ bool skip(Elt_Type elt0) { return elt0 < 0.5; }

extern "C" __global__ void
mxv()
{
  // Compute element number to start at.
  //

  const int CS = 32/sizeof(Elt_Type);
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const bool skip = d_app.d_op[hb + thd_v_offset] > d_app.norm_threshold;
      if ( __all_sync(0xffffffff, skip ) ) continue;

      Elt_Type vout[M];
      for ( auto& e: vout ) e = 0;

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[ ( hb + thd_g_offset + g ) * N + c + thd_c_offset ];

          if ( skip ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      if(1)
#pragma unroll
      for ( int rr=0; rr<M; rr += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[thd_c_offset][thd_g_offset+g] = vout[rr+g];
          for ( int g=0; g<CS; g++ )
            d_app.d_out[ ( hb + thd_g_offset + g ) * M + rr + thd_c_offset ]
              = vxfer[g][threadIdx.x];
        }
      else
#pragma unroll
      for ( int r=0; r<M; r++ )
        d_app.d_out[ ( hb + threadIdx.x ) * M + r ] = vout[r];

    }
}

extern "C" __global__ void
mxv_wq()
{
  // Compute element number to start at.
  //

  const int CS = 32/sizeof(Elt_Type);
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  __shared__ int workq[MAX_BLOCK_SIZE];
  workq[threadIdx.x] = threadIdx.x;
  __shared__ unsigned int work_pos;

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      __syncthreads();
      if ( threadIdx.x == 0 ) work_pos = 0;
      __syncthreads();
      const bool work = d_app.d_op[hb + thd_v_offset] <= d_app.norm_threshold;
      if ( work ) workq[atomicAdd(&work_pos,1)] = threadIdx.x;
      __syncthreads();
      const int work_pos_rnd_up = ( work_pos + CS - 1 ) & ~ ( CS - 1 );

      if ( thd_v_offset >= work_pos_rnd_up ) continue;
      const bool skip = thd_v_offset >= work_pos;
      const int work_v_offset = workq[threadIdx.x];

      Elt_Type vout[M];
      for ( auto& e: vout ) e = 0;

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[ ( hb + workq[thd_g_offset + g] ) * N
                          + c + thd_c_offset ];

          if ( skip ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      if ( skip ) continue;

#pragma unroll
      for ( int r=0; r<M; r++ )
        d_app.d_out[ ( hb + work_v_offset ) * M + r ] = vout[r];

    }
}

struct CData { int thd; int amt_work; };

__device__ CData
compress(bool active)
{
  const int MAX_BLOCK_SIZE = 1024;
  __volatile__ __shared__ short prefix[MAX_BLOCK_SIZE];
  prefix[threadIdx.x] = active;
  short my_val = active;
  for ( int dist = 1; dist <= blockDim.x; dist <<= 1 )
    {
      __syncthreads();
      prefix[threadIdx.x] = my_val;
      __syncthreads();
      if ( dist <= threadIdx.x ) my_val += prefix[threadIdx.x-dist];
    }
  __shared__ short num_active;
  if ( threadIdx.x == blockDim.x - 1 ) num_active = my_val;
  __syncthreads();
  if ( active ) prefix[my_val-1] = threadIdx.x;
  __syncthreads();
  int thd_num = threadIdx.x < num_active ? prefix[threadIdx.x] : threadIdx.x;
  CData rv; rv.thd = thd_num; rv.amt_work = num_active;
  return rv;
}

__device__ CData
compress2(bool active)
{
  const int MAX_BLOCK_SIZE = 1024;
  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;
  const int wp_mk = wp_sz - 1;
  __shared__ int num_act_blk;
  if ( threadIdx.x == 0 ) num_act_blk = 0;
  __shared__ short prefix[MAX_BLOCK_SIZE];
  const int lane = threadIdx.x & wp_mk;
  const uint32_t msk = 0xffffffff;

  const uint32_t active_wp_v = __ballot_sync(msk,active);
  const uint32_t active_pf_v = active_wp_v << ( 31 - lane );
  const uint32_t my_pf = __popc(active_pf_v);

  int pfx_wp = 0;
  __syncthreads();
  if ( lane == wp_mk ) pfx_wp = atomicAdd( &num_act_blk, my_pf );
  pfx_wp = __shfl_sync(msk,pfx_wp,wp_mk);
  const int pfx_me = pfx_wp + my_pf;
  if ( active ) prefix[pfx_me-1] = threadIdx.x;
  __syncthreads();
  int thd_num = threadIdx.x < num_act_blk ? prefix[threadIdx.x] : threadIdx.x;
  CData rv; rv.thd = thd_num; rv.amt_work = num_act_blk;
  return rv;
}

extern "C" __global__ void
mxv_prefix()
{
  // Compute element number to start at.
  //

  const int CS = 32/sizeof(Elt_Type);
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  __shared__ short workq[MAX_BLOCK_SIZE];

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const bool work = d_app.d_op[hb + thd_v_offset] <= d_app.norm_threshold;
      CData planb = compress(work);
      const int work_pos = planb.amt_work;
      const int work_pos_rnd_up = ( work_pos + CS - 1 ) & ~ ( CS - 1 );

      if ( thd_v_offset >= work_pos_rnd_up ) continue;
      workq[threadIdx.x] = planb.thd;
      const bool skip = thd_v_offset >= work_pos;
      const int work_v_offset = planb.thd;

      Elt_Type vout[M];
      for ( auto& e: vout ) e = 0;

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[ ( hb + workq[thd_g_offset + g] ) * N
                          + c + thd_c_offset ];

          if ( skip ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      if ( skip ) continue;

#pragma unroll
      for ( int r=0; r<M; r++ )
        d_app.d_out[ ( hb + work_v_offset ) * M + r ] = vout[r];

    }
}
extern "C" __global__ void
mxv_prefix2()
{
  // Compute element number to start at.
  //

  const int CS = 32/sizeof(Elt_Type);
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  __shared__ short workq[MAX_BLOCK_SIZE];

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const bool work = d_app.d_op[hb + thd_v_offset] <= d_app.norm_threshold;
      CData planb = compress2(work);
      const int work_pos = planb.amt_work;
      const int work_pos_rnd_up = ( work_pos + CS - 1 ) & ~ ( CS - 1 );

      if ( thd_v_offset >= work_pos_rnd_up ) continue;
      workq[threadIdx.x] = planb.thd;
      const bool skip = thd_v_offset >= work_pos;
      const int work_v_offset = planb.thd;

      Elt_Type vout[M];
      for ( auto& e: vout ) e = 0;

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[ ( hb + workq[thd_g_offset + g] ) * N
                          + c + thd_c_offset ];

          if ( skip ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      if ( skip ) continue;

#pragma unroll
      for ( int r=0; r<M; r++ )
        d_app.d_out[ ( hb + work_v_offset ) * M + r ] = vout[r];

    }
}



#if 0
template <int block_lg, int RADIX_LG>
__device__ void
sort_block_1_bit_split
(int bit_low, int bit_count, Pass_1_Stuff<block_lg,RADIX_LG>& p1s)
{
  const int block_size = 1 << block_lg;
  const int elt_per_tile = elt_per_thread * block_size;

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      const int sidx = threadIdx.x;

      // Make a copy of key.
      //
      const Sort_Elt key = p1s.keys[ sidx ];

      p1s.prefix[ threadIdx.x + 1 ] = bool(key);
      if ( threadIdx.x == 0 ) p1s.prefix[ 0 ] = 0;

      uint my_prefix = my_ones_write;

      // Compute a prefix sum of vectors.
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_neighbor = threadIdx.x - dist;
          __syncthreads();
          uint neighbor_prefix =
            threadIdx.x >= dist ? p1s.prefix[ idx_neighbor + 1 ] : 0;

          my_prefix += neighbor_prefix;
          __syncthreads();
          p1s.prefix[ threadIdx.x + 1 ] = my_prefix;
        }

      // At this point my_prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = p1s.prefix[ block_size ];
      const int idx_one_tid_0 = elt_per_tile - all_threads_num_ones;
      const int smaller_tids_num_ones = p1s.prefix[ threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          p1s.keys[ new_idx ] = key;
        }

  __syncthreads();
}
#endif



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

#if 0
  info.GET_INFO(mxv_prefix);
#else
  info.GET_INFO(mxv);
  info.GET_INFO(mxv_wq);
  info.GET_INFO(mxv_prefix);
  info.GET_INFO(mxv_prefix2);
#endif

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("gld_efficiency");
  if ( opt_p )
    {
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( false )
    NPerf_metrics_off();

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t op_size_bytes = in_size_elts * sizeof( app.h_op[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * M;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_op = new Op_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];
  app.h_out_check_n = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_op,  op_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  const Op_Type norm_threshold_max = ( 1 << 20 );

  for ( int i=0; i<app.num_vecs; i++ )
    app.h_op[i] = random() % norm_threshold_max;

#if 0
  const int CS = 8;
  for ( int i=0; i<app.num_vecs; i+=CS )
    {
      Op_Type th = random() % norm_threshold_max;
      for ( int j=i; j<i+CS; j++ ) app.h_op[j] = th;
    }
#endif

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    {
      vector<Elt_Type> vo(M);
      for ( int r=0; r<M; r++ )
        for ( int c=0; c<N; c++ )
          vo[ r ] += app.h_in[ i * N + c ] * app.matrix[r][c];
      Elt_Type *cptr = &app.h_out_check[ i * M ];
      for ( auto e: vo ) *cptr++ = e;
      {
        Elt_Type sos = 0;
        for ( auto e: vo ) sos += e*e;
        Elt_Type nf = sos ? powf(sos,-0.5) : 1;
        for ( auto& e: vo ) e *= nf;
      }
      Elt_Type *cptrn = &app.h_out_check_n[ i * M ];
      for ( auto e: vo ) *cptrn++ = e;
    }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int64_t amt_data_bytes = in_size_bytes + op_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  const int output_width = stdout_width_get();

  // Compute number of blocks available per MP based only on
  // the number of blocks.  This may be larger than the
  // number of blocks that can run.
  //
  const int bl_per_mp_available = ( num_blocks + num_mp - 1 ) / num_mp;

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_op, app.h_op, op_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    double tscale = 0;

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = min( 4, wp_limit );

        vector<int> best_num_wp(bl_per_mp_available+1);
        for ( int num_wp = wp_start; num_wp <= wp_limit; num_wp++ )
          best_num_wp[info.get_max_active_blocks_per_mp(kernel,num_wp<<5)]
            = num_wp;
        int num_wp = 0;
        for ( auto w: best_num_wp ) if ( w ) num_wp = w;
        assert( num_wp );

        pTable table;
        table.stream = stdout;

        for ( double work_frac: { 1.0, .75, .5, .25, 0.0 } )
          {
            Op_Type norm_threshold = work_frac * norm_threshold_max;
            int wp_cnt = num_wp;
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            app.norm_threshold = norm_threshold;

            // Copy App structure to GPU.
            //
            CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              work_frac * num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  1e9 * thpt_compute_gflops
                  / ( sizeof(Elt_Type) == 4 ? info.chip_sp_flops :
                      sizeof(Elt_Type) == 8 ? info.chip_dp_flops : 1 );
                const double comm_frac =
                  min(2.0,1e9 * thpt_data_gbps / info.chip_bw_Bps);

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( table.num_lines < 1 )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("work","%5.3f", work_frac);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry
                  ("I/op","%4.1f",
                   NPerf_metric_value_get("inst_executed") * 32.0 / num_ops );
                if ( opt_p )
                  {
                    table.entry
                      ("Ld eff","%5.1f%%",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("L2rθ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("L2wθ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.entry
                      ("DRrθ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("DRwθ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                  }

                const bool plot_bandwidth = true;

                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                if ( tscale == 0 ) tscale = this_elapsed_time_s * 2;

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);

                const bool ref_time = true;

                string util_hdr =
                  ref_time ? "Reference Time" :
                  plot_bandwidth ? "Data BW Util" : "FP Utilization";
                const double frac =
                  ref_time ? this_elapsed_time_s / tscale :
                  plot_bandwidth ? comm_frac : comp_frac;
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);
                table.row_end();

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes,
                  hipMemcpyDeviceToHost ) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              {
                const bool norm = app.h_op[i] > norm_threshold;
                if ( norm ) continue;
                Elt_Type* const check =
                  norm ? app.h_in : app.h_out_check;
                for ( int r=0; r<M; r++ )
                  {
                    const int idx = i * M + r;

                    if ( fabs( check[idx] - app.h_out[idx] ) > 1e-5 )
                      {
                        err_count++;
                        if ( err_count < 5 )
                          printf
                            ("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                             i, r, app.h_out[idx], check[idx] );
                      }
                  }
              }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }
}
