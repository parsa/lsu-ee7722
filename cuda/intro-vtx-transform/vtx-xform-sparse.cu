#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>
#include <misc.h>

#define N 16
#define M 16

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

typedef uint32_t Op_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check, *h_out_check_n;
  Op_Type norm_threshold;
  Op_Type *h_op;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;
  Op_Type *d_op;

};


// Declare application data structure.
//
// .. in host address space ..
App app;
//
// .. and in device constant address space.
__constant__ App d_app;


// Pointer to kernel main (global) functions.
typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);

const int chunk_size = 8;

extern "C" __global__ void
mxv_ld_all()
{
  // Group (chunk) size: The number of threads that cooperate to read
  // and write vectors.
  //
  const int CS = chunk_size;
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const int h = hb + thd_v_offset;
      const bool have_work = d_app.d_op[h] <= d_app.norm_threshold;

      Elt_Type vout[M]{};

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[( hb+thd_g_offset + g )*N + c + thd_c_offset ];

          if ( !have_work ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

#pragma unroll
      for ( int rr=0; rr<M; rr += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[thd_c_offset][thd_g_offset+g] = vout[rr+g];
          for ( int g=0; g<CS; g++ )
              d_app.d_out[ ( hb + thd_g_offset + g ) * M + rr + thd_c_offset ]
                = vxfer[g][threadIdx.x];
        }
    }
}

extern "C" __global__ void
mxv_ld_chunk()
{
  // Group (chunk) size: The number of threads that cooperate to read
  // and write vectors.
  //
  const int CS = chunk_size;
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;
  const int lane = threadIdx.x % wp_sz;
  const int thd_c_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );
  const int thd_wp_c0 = lane & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  assert( threadIdx.x || blockDim.x <= MAX_BLOCK_SIZE );
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const int h = hb + thd_v_offset;
      const bool have_work = d_app.d_op[h] <= d_app.norm_threshold;
      const uint32_t have_work_wp_v = __ballot_sync(~0,have_work);
      if ( !have_work_wp_v ) continue;
      const uint32_t chunk_wk_v = have_work_wp_v >> thd_wp_c0;

      Elt_Type vout[M]{};

      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              chunk_wk_v  &  1 << g
              ? d_app.d_in[( hb+thd_g_offset + g )*N + c + thd_c_offset ]
              : 0;

          if ( !have_work ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

#pragma unroll
      for ( int rr=0; rr<M; rr += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[thd_c_offset][thd_g_offset+g] = vout[rr+g];
          for ( int g=0; g<CS; g++ )
            if ( chunk_wk_v  &  1 << g )
              d_app.d_out[ ( hb + thd_g_offset + g ) * M + rr + thd_c_offset ]
                = vxfer[g][threadIdx.x];
        }
    }
}

typedef int worka_t;

// Declare a type for the compression function. This is needed for the
// templates.
typedef int (*Compress_Func)(bool have_work, worka_t* const &work_assignments);

__device__ int
compress_atomic(bool have_work, worka_t* const &worka)
{
  __shared__ int amt_work_block; // Number of threads with work.
  __syncthreads();
  if ( threadIdx.x == 0 ) amt_work_block = 0;
  __syncthreads();
  if ( have_work ) worka[atomicAdd(&amt_work_block,1)] = threadIdx.x;
  __syncthreads();
  return amt_work_block;
}

__device__ int
compress_prefix_shared(bool have_work, worka_t* const &worka)
{
  const int MAX_BLOCK_SIZE = 1024;
  worka_t* const prefix_array = worka;
  worka_t my_val = have_work;
  for ( int dist = 1; dist < MAX_BLOCK_SIZE; dist <<= 1 )
    {
      __syncthreads();
      prefix_array[threadIdx.x] = my_val;
      __syncthreads();
      if ( dist <= threadIdx.x ) my_val += prefix_array[threadIdx.x-dist];
      if ( dist >= blockDim.x ) break;
    }
  __shared__ int amt_work_block;
  if ( threadIdx.x == blockDim.x - 1 ) amt_work_block = my_val;
  __syncthreads();
  if ( have_work ) worka[my_val-1] = threadIdx.x;
  __syncthreads();
  return amt_work_block;
}

__device__ int
compress_prefix_ballot(bool have_work, worka_t* const &worka)
{
  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;
  const int lane_last = wp_sz - 1;  // The highest lane number.
  const int lane = threadIdx.x & ( wp_sz - 1 );
  const uint32_t msk = ~0;  // A thread mask.

  __shared__ int amt_work_block;
  if ( threadIdx.x == 0 ) amt_work_block = 0;

  // Bit vector indicating which threads in this warp have work.
  const uint32_t have_work_wp_v = __ballot_sync(msk,have_work);

  // Shift off bits corresponding to higher-numbered lanes.
  const uint32_t have_work_pf_v = have_work_wp_v << ( lane_last - lane );

  // Use population count function to compute the number of threads
  // that have work in this warp at this and lower-numbered lanes.
  // Note: pf is an abbreviation for prefix sum.
  //
  const uint32_t my_pf = __popc(have_work_pf_v);
  //
  // Note: __popc compiles to a single machine instruction in Kepler to
  // Pascal GPUs (and hopefully future models too).

  int pfx_wp = 0;
  __syncthreads();
  if ( lane == lane_last ) pfx_wp = atomicAdd( &amt_work_block, my_pf );
  pfx_wp = __shfl_sync(msk,pfx_wp,lane_last);

  const int pfx_me = pfx_wp + my_pf;
  if ( have_work ) worka[pfx_me-1] = threadIdx.x;
  __syncthreads();
  return amt_work_block;
}

template <Compress_Func compress> __device__ void
mxv_compress()
{
  // Group (chunk) size: The number of threads that cooperate to read
  // and write vectors.
  //
  const int CS = chunk_size;
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  __shared__ int worka[MAX_BLOCK_SIZE]; // Work assignment.
  worka[threadIdx.x] = threadIdx.x;

  for ( int hb = bl_start; hb<stop; hb += num_threads )
    {
      const bool work = d_app.d_op[hb + threadIdx.x] <= d_app.norm_threshold;

      const int work_pos = compress(work,worka);

      // If true, no vector assigned to this thread.
      const bool skip = threadIdx.x >= work_pos;

      // If true, this thread is in a group in which at least one thread
      // has a vector and at least one thread does not have a vector.
      //
      const bool tail_chunk =
        thd_g_offset < work_pos && thd_g_offset + CS >= work_pos;

      if ( skip && !tail_chunk ) continue;

      const int work_v_offset = worka[threadIdx.x];
      //
      // This thread, threadIdx.x, will operate on the same element
      // as thread work_v_offset would have.

      Elt_Type vout[M]{};

      for ( int c=0; c<N; c += CS )
        {
          Elt_Type vx[CS];
          for ( int g=0; g<CS; g++ )
            vx[g] = d_app.d_in[ ( hb + worka[thd_g_offset + g] ) * N
                          + c + thd_c_offset ];

          for ( int g=0; g<CS; g++ ) vxfer[g][threadIdx.x] = vx[g];

          if ( skip ) continue;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      if ( skip && tail_chunk ) continue;

      if ( tail_chunk )
#pragma unroll
        for ( int r=0; r<M; r++ )
          d_app.d_out[ ( hb + work_v_offset ) * M + r ] = vout[r];
      else
#pragma unroll
        for ( int rr=0; rr<M; rr += CS )
          {
            for ( int g=0; g<CS; g++ )
              vxfer[thd_c_offset][thd_g_offset+g] = vout[rr+g];
            for ( int g=0; g<CS; g++ )
              d_app.d_out
                [ ( hb + worka[thd_g_offset + g] ) * M + rr + thd_c_offset ]
                = vxfer[g][threadIdx.x];
          }
    }
}

__global__ void
mxv_compress_wps()
{
  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;
  const int wp_mk = wp_sz - 1;
  const int lane_last = wp_sz - 1;
  const int lane = threadIdx.x & wp_mk;
  const uint32_t msk = ~0;

  // Group (chunk) size: The number of threads that cooperate to read
  // and write vectors.
  //
  const int CS = chunk_size;
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_g_offset = threadIdx.x & ~ ( CS - 1 );
  const int thd_wp_c0 = lane & ~ ( CS - 1 );

  constexpr int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[CS][MAX_BLOCK_SIZE+1];

  __shared__ int workb[2 * MAX_BLOCK_SIZE];
  int *wwork = &workb[ 2 * ( threadIdx.x & ~wp_mk ) ];

  wwork[lane] = wwork[wp_sz + lane] = -1;

  int amt_wk_buf = 0;

  for ( int hb = bl_start; hb < stop || amt_wk_buf > 0; hb += num_threads )
    {
      const bool last_iter = hb + num_threads >= stop;
      const int h = hb + threadIdx.x;
      const bool work = hb < stop && d_app.d_op[h] <= d_app.norm_threshold;

      // Bit vector indicating which threads in this warp have work.
      const uint32_t have_work_wp_v = __ballot_sync(msk,work);

      // If there's nothing new and it's not the last iteration, continue.
      if ( !have_work_wp_v && !last_iter ) continue;

      // Shift off bits corresponding to higher-numbered lanes.
      const int amt_wk_here = __popc(have_work_wp_v);
      const uint32_t have_work_pf_v = have_work_wp_v << ( lane_last - lane );
      const uint32_t my_pf = __popc(have_work_pf_v);

      if ( work ) wwork[ amt_wk_buf + my_pf - 1] = h;
      amt_wk_buf += amt_wk_here;
      if ( amt_wk_buf < wp_sz && !last_iter ) continue;

      if ( lane >= amt_wk_buf )
        {
          if ( thd_wp_c0 >= amt_wk_buf ) break;
          wwork[lane] = wwork[thd_wp_c0];
        }

      Elt_Type vout[M]{};

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[g][threadIdx.x] =
              d_app.d_in[ ( wwork[thd_wp_c0 + g] ) * N + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_c_offset ][ thd_g_offset + cc ];

          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

#pragma unroll
      for ( int rr=0; rr<M; rr += CS )
        {
          for ( int g=0; g<CS; g++ )
            vxfer[thd_c_offset][thd_g_offset+g] = vout[rr+g];
          for ( int g=0; g<CS; g++ )
            d_app.d_out[ ( wwork[thd_wp_c0 + g] ) * M + rr + thd_c_offset ]
              = vxfer[g][threadIdx.x];
        }

      wwork[lane] = wwork[lane+wp_sz];
      amt_wk_buf -= wp_sz;

    }
}


extern "C" __global__ void mxv_atomic() { mxv_compress<compress_atomic>(); }
extern "C" __global__ void mxv_pfx_shared()
{ mxv_compress<compress_prefix_shared>(); }
extern "C" __global__ void mxv_pfx_ballot()
{ mxv_compress<compress_prefix_ballot>(); }

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_ld_all);
  info.GET_INFO(mxv_ld_chunk);
#if 1
  info.GET_INFO(mxv_atomic);
  info.GET_INFO(mxv_pfx_shared);
  info.GET_INFO(mxv_pfx_ballot);
  info.GET_INFO(mxv_compress_wps);
#endif
  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;
  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;

  // Examine argument 1, number of blocks per MP.
  //
  // 0: Largest number that will fit.
  // n: Number of blocks per MP.

  const int arg1_int = argc < 2 ? 1 : atoi(argv[1]);
  const bool choose_blocks_per_mp = arg1_int == 0;
  const int blocks_per_mp = abs(arg1_int);

  // Examine argument 2, number of warps per block.
  //
  // 0: If arg1 is != 0, maximize number of warps per block.
  //    If arg1 is == 0, maximize number of warps per MP.
  // n: Exact size.

  const bool opt_p = argc >= 3 && argv[2][0] == 'p';
  const int arg2_int = argc < 3 ? 0 : atoi(argv[2]+opt_p);
  const int thd_per_block_goal = arg2_int ? arg2_int << wp_lg: 1024;
  const bool choose_wps_per_block = arg2_int == 0;

  // Examine argument 3, size of array in elts per MP. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? num_mp << 20 : int( atof(argv[3]) * (num_mp<<20) );

  if ( thd_per_block_goal <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS | 0 ] [[p][ 0 | WP_PER_BLOCK]] "
             "[NUM_M_ELTs_PER_SM]\n",
             argv[0]);
      exit(1);
    }

  const bool vary_work = true;

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  if ( opt_p )
    {
      NPerf_metric_collect("gld_efficiency");
      NPerf_metric_collect("gst_efficiency");
      NPerf_metric_collect("shared_efficiency");
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( false )
    NPerf_metrics_off();

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t op_size_bytes = app.num_vecs * sizeof( app.h_op[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * M;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_op = new Op_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];
  app.h_out_check_n = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_op,  op_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Matrix size: %d x %d.  Vectors: %d.\n",
         N, M, app.num_vecs );

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  const Op_Type norm_threshold_max = ( 1 << 20 );

  for ( int i=0; i<app.num_vecs; i++ )
    app.h_op[i] = random() % norm_threshold_max;

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    {
      vector<Elt_Type> vo(M);
      for ( int r=0; r<M; r++ )
        for ( int c=0; c<N; c++ )
          vo[ r ] += app.h_in[ i * N + c ] * app.matrix[r][c];
      Elt_Type *cptr = &app.h_out_check[ i * M ];
      for ( auto e: vo ) *cptr++ = e;
      {
        Elt_Type sos = 0;
        for ( auto e: vo ) sos += e*e;
        Elt_Type nf = sos ? powf(sos,-0.5) : 1;
        for ( auto& e: vo ) e *= nf;
      }
      Elt_Type *cptrn = &app.h_out_check_n[ i * M ];
      for ( auto e: vo ) *cptrn++ = e;
    }

  const int64_t num_ops_max_mxv = ( int64_t(M) * N + N + M ) * app.num_vecs;
  const int64_t num_ops_max_iter = 7 * app.num_vecs;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_op, app.h_op, op_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    double tscale = 0;

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> wp_lg;
        const int thd_limit = wp_limit << wp_lg;
        // gl: Goal and limit.
        const int thd_per_block_gl = min(thd_per_block_goal,thd_limit);

        int block_sz = 0;
        int bl_p_mp = 0;

        if ( choose_wps_per_block && choose_blocks_per_mp )
          {
            int num_wp_per_mp = 0;
            int num_wp = 0;
            for ( int nwpi = 1; nwpi <= wp_limit; nwpi++ )
              if ( set_max
                   ( num_wp_per_mp,
                     nwpi * info.get_max_active_blocks_per_mp(kernel,nwpi<<5) )
                   )
                num_wp = nwpi;
            bl_p_mp = num_wp_per_mp / num_wp;
            block_sz = num_wp * wp_sz;
          }
        else if ( !choose_wps_per_block && choose_blocks_per_mp )
          {
            block_sz = thd_per_block_gl;
            bl_p_mp = info.get_max_active_blocks_per_mp(kernel,block_sz);
          }
        else if ( choose_wps_per_block && !choose_blocks_per_mp )
          {
            for ( int nwpi = wp_limit; nwpi >= 1; nwpi-- )
              {
                const int bl_sz = nwpi << wp_lg;
                if ( !set_max
                     ( bl_p_mp,
                       info.get_max_active_blocks_per_mp(kernel,bl_sz) ) )
                  continue;
                block_sz = bl_sz;
                set_min( bl_p_mp, blocks_per_mp );
                if ( bl_p_mp >= blocks_per_mp ) break;
              }
          }
        else if ( !choose_wps_per_block && !choose_blocks_per_mp )
          {
            block_sz = thd_per_block_gl;
            bl_p_mp = blocks_per_mp;
          }
        else { assert( false ); }

        const int n_blocks = bl_p_mp * num_mp;

        assert( block_sz && n_blocks );

        pTable table(stdout);

        const double wp_1_2 = 1.0 - pow(0.5,1.0/wp_sz);
        const double wp_1_8 = 1.0 - pow(0.875,1.0/wp_sz);

        for ( double work_frac: { 1.0, .75, .5, .25, wp_1_2, wp_1_8, 0.0 } )
          {
            Op_Type norm_threshold = work_frac * norm_threshold_max;

            app.norm_threshold = norm_threshold;

            // Copy App structure to GPU.
            //
            CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<n_blocks,block_sz>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const int64_t num_ops =
              work_frac * num_ops_max_mxv + num_ops_max_iter;

            const int64_t amt_data_bytes =
              op_size_bytes + work_frac * ( in_size_bytes + out_size_bytes );

            const double thpt_compute_gflops =
              work_frac * num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_work )
              {
                const double comp_frac =
                  1e9 * thpt_compute_gflops
                  / ( sizeof(Elt_Type) == 4 ? info.chip_sp_flops :
                      sizeof(Elt_Type) == 8 ? info.chip_dp_flops : 1 );
                const double comm_frac =
                  min(2.0,1e9 * thpt_data_gbps / info.chip_bw_Bps);

                // Number of warps, rounded up.
                //
                const int num_wps = ( block_sz + 31 ) >> wp_lg;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,block_sz);

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( max_bl_per_mp, ( n_blocks + num_mp - 1 )/ num_mp );

                // Based on the number of blocks, compute the number of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( table.num_lines < 1 )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("work","%5.3f", work_frac);
                table.entry("t/µs","%5.0f", this_elapsed_time_s * 1e6);
                table.entry
                  ("I/op","%4.1f",
                   NPerf_metric_value_get("inst_executed")
                   * 32.0 / max(int64_t(1),num_ops) );
                if ( opt_p )
                  {
                    table.entry
                      ("SM eff","%5.1f%%",
                       NPerf_metric_value_get("shared_efficiency"));
                    table.header_span_start("R-Eff-%");
                    table.entry
                      ("Ld","%3.0f",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("St","%3.0f",
                       NPerf_metric_value_get("gst_efficiency"));
                    table.header_span_end();
                    table.header_span_start("L2-Cache");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.header_span_end();
                    table.header_span_start("DRAM");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                    table.header_span_end();
                  }

                const bool plot_bandwidth = true;
                if ( !plot_bandwidth )
                  table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                if ( tscale == 0 ) tscale = this_elapsed_time_s * 2;

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);

                const bool ref_time = false;

                string util_hdr =
                  ref_time ? "Reference Time" :
                  plot_bandwidth ? "Data BW Util" : "FP Utilization";
                const double frac =
                  ref_time ? this_elapsed_time_s / tscale :
                  plot_bandwidth ? comm_frac : comp_frac;
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);
                table.row_end();

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (block_sz + wp_sz - 1 ) >> wp_lg,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes,
                  hipMemcpyDeviceToHost ) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              {
                const bool norm = app.h_op[i] > norm_threshold;
                for ( int r=0; r<M; r++ )
                  {
                    const int idx = i * M + r;
                    Elt_Type cval = norm ? 0 : app.h_out_check[idx];

                    if ( fabs( cval - app.h_out[idx] ) > 1e-5 )
                      {
                        err_count++;
                        if ( err_count < 5 )
                          printf
                            ("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                             i, r, app.h_out[idx], cval );
                      }
                  }
              }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }
}
