#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2016), GPU Microarchitecture
//
// Simple CUDA Example, without LSU ECE helper classes.

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

#include "util.h"

typedef float Elt_Type;

struct Vertex
{
  Elt_Type a[4];
};

struct App
{
  int num_threads;
  Elt_Type matrix[4][4];
  int array_size;
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<4; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<4; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread = d_app.array_size / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = d_app.d_v_in[h];
      Vertex q;
      for ( int i=0; i<4; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<4; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;
    }
}

void
print_gpu_and_kernel_info()
{
  print_gpu_info();

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  const bool use_pthreads = nt_raw < 0;
  app.num_threads = abs(nt_raw);

  if ( app.num_threads == 1 )
    print_gpu_and_kernel_info();

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( & app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( & app.d_v_out, app.array_size * sizeof(Vertex) ) );

  printf("Preparing for %d threads %d elements.\n",
         app.num_threads, app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<4; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<4; i++ )
    for ( int j=0; j<4; j++ )
      app.matrix[i][j] = drand48();

  const double time_start = time_fp();

  if ( use_pthreads )
    {
      pthread_t* const ptid = new pthread_t[app.num_threads];

      pthread_attr_t attr;
      pthread_attr_init(&attr);
      pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

      for ( int i=0; i<app.num_threads; i++ )
        {
          pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);
        }

      for ( int i=0; i<app.num_threads; i++ )
        {
          pthread_join( ptid[i], NULL );
        }
    }
  else
    {
      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      const int threads_per_block = 256;
      const int blocks_per_grid =
        ( app.num_threads + threads_per_block-1 ) / threads_per_block;

      /// Kernel Launch
      cuda_thread_start <<< blocks_per_grid, threads_per_block >>>();

      // Copy output array from GPU to CPU.
      //
      CE( hipMemcpy
          ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );

    }

  const double data_size = app.array_size * sizeof(Vertex) * 2;
  const double fp_op_count = app.array_size * ( 16 + 12 );
  const double elapsed_time = time_fp() - time_start;

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time,
         1e-9 * data_size / elapsed_time);

}
