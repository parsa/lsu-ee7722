#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2016), GPU Microarchitecture
//
// Simple CUDA Example, without LSU ECE helper classes.

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>

#include "util.h"

typedef float Elt_Type;

#define N 4

struct Vector
{
  Elt_Type a[N];
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;
  Vector *v_in, *v_out;
  Vector *d_v_in, *d_v_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

void
omp_start()
{
#pragma omp parallel for num_threads(app.num_threads)
  for ( int h=0; h<app.num_threads; h++ )
    {
      Vector p = app.v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  //  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vector p = app.v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread = d_app.array_size / d_app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vector p = d_app.d_v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;
    }
}

void
print_gpu_and_kernel_info()
{
  gpu_info_print();

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  const bool use_pthreads = nt_raw < 0;
  app.num_threads = abs(nt_raw);

  if ( app.num_threads == 1 )
    print_gpu_and_kernel_info();

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vector[app.array_size];
  app.v_out = new Vector[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( & app.d_v_in,  app.array_size * sizeof(Vector) ) );
  CE( hipMalloc( & app.d_v_out, app.array_size * sizeof(Vector) ) );

  printf("Preparing for %d threads %d elements.\n",
         app.num_threads, app.array_size);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  const double time_start = time_fp();
  const bool use_omp = true;
  printf("For %d elements...\n", app.array_size);

  for ( int ntl = 0; ntl < 5; ntl++ )
    {
      app.num_threads = 1 << ntl;
      for ( int i=0; i<3; i++ )
        {
          const char *version = NULL;
          switch ( i ) {

          case 0:
            version = "omp";
            omp_start();
            break;

          case 1: // pthreads
            {
              version = "pth";
              pthread_t* const ptid = new pthread_t[app.num_threads];

              pthread_attr_t attr;
              pthread_attr_init(&attr);
              pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

              for ( int i=0; i<app.num_threads; i++ )
                {
                  pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);
                }

              for ( int i=0; i<app.num_threads; i++ )
                {
                  pthread_join( ptid[i], NULL );
                }
            }
            break;
          case 2: // CUDA
            {
              version = "cuda";
              // Copy input array from CPU to GPU.
              //
              CE( hipMemcpy
                  ( app.d_v_in, app.v_in, array_size_bytes,
                    hipMemcpyHostToDevice ) );

              // Copy App structure to GPU.
              //
              CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

              const int threads_per_block = 256;
              const int blocks_per_grid =
                ( app.num_threads + threads_per_block-1 ) / threads_per_block;

              /// Kernel Launch
              cuda_thread_start <<< blocks_per_grid, threads_per_block >>>();

              // Copy output array from GPU to CPU.
              //
              CE( hipMemcpy
                  ( app.v_out, app.d_v_out, array_size_bytes,
                    hipMemcpyDeviceToHost) );
            }
            break;
          }

          const double data_size = app.array_size * sizeof(Vector) * 2;
          const double fp_op_count = app.array_size * ( 16 + 12 );
          const double elapsed_time = time_fp() - time_start;

          printf("%4s  %4d thds  %7.3f us  Rate %.3f GFLOPS,  %.3f GB/s\n",
                 version,
                 app.num_threads,
                 1e6 * elapsed_time,
                 1e-9 * fp_op_count / elapsed_time,
                 1e-9 * data_size / elapsed_time);

        }
    }


}
