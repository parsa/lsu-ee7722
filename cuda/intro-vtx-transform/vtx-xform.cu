#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2017), GPU Microarchitecture
//
// Simple CUDA Example, without LSU ECE helper classes.

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <vector>
using namespace std;

#include <hip/hip_runtime.h>

#include "util.h"

typedef float Elt_Type;

#define N 4

struct __builtin_align__(N*sizeof(Elt_Type)) __align__(N*sizeof(Elt_Type))
Vector
{
  Elt_Type a[N];
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  size_t array_size;
  Vector *v_in, *v_out;
  Vector *d_v_in, *d_v_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

void
omp_start()
{
#pragma omp parallel for num_threads(app.num_threads)
  for ( unsigned int h=0; h<app.array_size; h++ )
    {
      Vector p = app.v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  //  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vector p = app.v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

__global__ void
cuda_thread_start()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  for ( int h=tid; h<d_app.array_size; h += d_app.num_threads )
    {
      Vector p = d_app.d_v_in[h];
      Vector q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;
    }
}

void
print_gpu_and_kernel_info()
{
  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

int
main(int argc, char **argv)
{
  const int nt_raw = argc < 2 ? 1 : atoi(argv[1]);
  app.num_threads = abs(nt_raw);

  print_gpu_and_kernel_info();

  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );
  const size_t array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vector[app.array_size];
  app.v_out = new Vector[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( & app.d_v_in,  app.array_size * sizeof(Vector) ) );
  CE( hipMalloc( & app.d_v_out, app.array_size * sizeof(Vector) ) );

  printf("Using  %zd elements.\n", app.array_size);

  // Initialize input array.
  //
  for ( size_t i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  printf("For %zd elements...\n", app.array_size);

  const char* const names[] = { "OMP", "PTH", "CUDA" };

  vector<pthread_t> ptid;
  pthread_attr_t attr;
  pthread_attr_init(&attr);
  pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  for ( int i=0; i<3; i++ )
    {
      printf("Running %s...\n", names[i]);
      for ( int ntl = 0; ntl < 7; ntl++ )
        {
          const double time_start = time_fp();

          app.num_threads = i < 2 ? 1 << ntl : 32 << ntl;
          switch ( i ) {

          case 0:
            omp_start();
            break;

          case 1: // pthreads
            {
              ptid.resize(app.num_threads);
              for ( ptrdiff_t i=0; i<app.num_threads; i++ )
                {
                  pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);
                }

              for ( int i=0; i<app.num_threads; i++ )
                {
                  pthread_join( ptid[i], NULL );
                }
            }
            break;
          case 2: // CUDA
            {
              // Copy App structure to GPU.
              //
              CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

              const int threads_per_block = min(app.num_threads,32);
              const int blocks_per_grid =
                ( app.num_threads + threads_per_block-1 ) / threads_per_block;

              /// Kernel Launch
              cuda_thread_start <<< blocks_per_grid, threads_per_block >>>();

              CE( hipStreamSynchronize( 0 ) );

            }
            break;
          }

          const double data_size = app.array_size * sizeof(Vector) * 2;
          const double fp_op_count = app.array_size * ( 16 + 12 );
          const double elapsed_time = time_fp() - time_start;

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              ( app.v_out, app.d_v_out, array_size_bytes,
                hipMemcpyDeviceToHost) );

          printf(" %4d thds  %10.3f µs  Rate %7.3f GFLOPS,  %7.3f GB/s\n",
                 app.num_threads,
                 1e6 * elapsed_time,
                 1e-9 * fp_op_count / elapsed_time,
                 1e-9 * data_size / elapsed_time);

        }
    }


}
