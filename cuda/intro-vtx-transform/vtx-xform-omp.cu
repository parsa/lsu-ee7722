#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2015), GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "util.h"

#define N 4

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(16) Vertex
{
  Elt_Type __align__(16) a[N];
};

struct __align__(16) V4 {
  Elt_Type x, y, z, w;
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
  V4 * d_v4_in;

  int reduction_method;

  Elt_Type *thd_sum;
  Elt_Type *d_thd_sum;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

#define BLOCK_SIZE_MAX 1024
__shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];
__shared__ Elt_Type wshared_sum[32];

__device__ void reduce_method_1(float mag_sq);
__device__ void reduce_method_2(float mag_sq);
__device__ void reduce_method_3(float mag_sq);
__device__ void reduce_method_4(float mag_sq);


// The entry point for the GPU code.
//
__global__ void
cuda_thread_start()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Compute element number to start at.
  //
  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = d_app.num_threads;

  Elt_Type sum_sum_of_sq = 0;

  for ( int h=start; h<stop; h += inc )
    {
      V4 p2 = d_app.d_v4_in[h];
      Vertex p;
      p.a[0] = p2.x; p.a[1] = p2.y; p.a[2] = p2.z; p.a[3] = p2.w;
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;

      //  Compute the magnitude squared of q and update "sum" variable.
      //
      Elt_Type sum_of_sq = 0;
      for ( int i=0; i<N; i++ ) sum_of_sq += q.a[i] * q.a[i];
      sum_sum_of_sq += sum_of_sq;
    }

  // Use the desired reduction method routine.
  //
  // The reduction routine finds the sum of sum_sum_of_sq over all
  // the threads in a block and writes it to global memory.
  //
  switch ( d_app.reduction_method ) {
  case 0:
    // Have the CPU perform reduction.
    d_app.d_thd_sum[tid] = sum_sum_of_sq;
    break;

  case 1: // One thread computes the sum.
    reduce_method_1(sum_sum_of_sq);
    break;

  case 2: // Use a reduction tree with a __syncthreads each iteration.
    reduce_method_2(sum_sum_of_sq);
    break;

  case 3: // To avoid syncs, use one block.
    reduce_method_3(sum_sum_of_sq);
    break;

  case 4: // Homework goes solution in this routine.
    reduce_method_4(sum_sum_of_sq); break;

    // Force an error if reduction method unknown.
  default:  d_app.d_thd_sum[threadIdx.x] = 0; break;
  }

}

__device__ void
reduce_method_1(float thd_sum)
{
  // One thread computes the sum.

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to thread 0.
  __syncthreads();

  if ( threadIdx.x != 0 ) return;

  Elt_Type our_sum = thd_sum;

  for ( int i=1; i<blockDim.x; i++ )
    our_sum += shared_sum[i];

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

__device__ void
reduce_method_2(float thd_sum)
{
  // Use a reduction tree.
  //
  // By using a complete tree reduction we perform the minimum
  // number of adds. That's the good news. The bad news is that
  // we need to synchronize each iteration.

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  for ( int dist = blockDim.x >> 1;  dist;  dist >>= 1 )
    {
      __syncthreads();  // Wait for other threads to finish.
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }
    }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

__device__ void
reduce_method_3(float thd_sum)
{
  // A mixture of a linear sum and a tree reduction, chosen so
  // that only a single warp of threads participates.  Since only
  // a single warp is executing, no synchronizations are necessary
  // other than the one after the thread's initial sum is written.

  shared_sum[threadIdx.x] = thd_sum;  // Make sum available to other threads.
  Elt_Type our_sum = thd_sum;

  __syncthreads();

  // In the code below, only the first 32 threads do something useful.

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  // Perform a linear sum.
  //
  // The first 32 threads each compute their own sum.
  //
  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      our_sum += shared_sum[i];

  shared_sum[threadIdx.x] = our_sum;

  // Perform a tree reduction.
  //
  // The first 32 threads perform a tree reduction of the linear sums
  // found in the previous step.
  //
  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist )
        {
          our_sum += shared_sum[ threadIdx.x + dist ];
          shared_sum[ threadIdx.x ] = our_sum;
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_sum;
}

__device__ void
reduce_method_4(float thd_sum)
{
  shared_sum[threadIdx.x] = thd_sum;

  Elt_Type our_sum = thd_sum;

  const int warp_lg = 5;
  const int warp_size = 1 << warp_lg;
  const int half_warp_size = warp_size >> 1;
  const int lane = threadIdx.x & ( warp_size - 1 );

  for ( int dist = half_warp_size;  dist;  dist >>= 1 )
    if ( lane < dist )
      {
        our_sum += shared_sum[ threadIdx.x + dist ];
        shared_sum[ threadIdx.x ] = our_sum;
      }

#if 1

  const int warp_num = threadIdx.x >> warp_lg;

  if ( !lane )
    wshared_sum[ warp_num ] = our_sum;

  __syncthreads();

  Elt_Type our_osum = 0;

  if ( threadIdx.x < half_warp_size )
    {
      our_osum = wshared_sum[ threadIdx.x ];

      for ( int dist = half_warp_size;  dist;  dist >>= 1 )
        if ( lane < dist )
          {
            our_osum += wshared_sum[ threadIdx.x + dist ];
            if ( dist > 1 )
              wshared_sum[ threadIdx.x ] = our_osum;
          }

    }

#else

  __syncthreads();

  const int our_elt = threadIdx.x << warp_lg;
  const int block_size = blockDim.x;
  Elt_Type our_osum = threadIdx.x < warp_size ? shared_sum[ our_elt ] : 0;

  for ( int dist = block_size >> 1;  dist >= warp_size;  dist >>= 1 )
      if ( our_elt < dist )
        {
          our_osum += shared_sum[ our_elt + dist ];
          shared_sum[ our_elt ] = our_osum;
        }

#endif

  if ( threadIdx.x ) return;

  d_app.d_thd_sum[blockIdx.x] = our_osum;
}

void
print_gpu_and_kernel_info()
{
  print_gpu_info();

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about GPU kernel routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

Elt_Type
omp_start()
{
  Elt_Type sum_sum_of_sq = 0;

#pragma omp parallel for reduction(+:sum_sum_of_sq) num_threads(app.num_threads)
  for ( int h=0; h<app.array_size; h++ )
    {
      const int tid = omp_get_thread_num();
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;

      //  Compute the magnitude squared of q and update "sum" variable.
      //
      Elt_Type sum_of_sq = 0;
      for ( int i=0; i<N; i++ ) sum_of_sq += q.a[i] * q.a[i];
      sum_sum_of_sq += sum_of_sq;
    }

  return sum_sum_of_sq;
}

int
main(int argc, char **argv)
{
  // Examine argument 1, block count, if negative, use pthreads.
  //
  const int arg1_int = argc < 2 ? 32 : atoi(argv[1]);
  const bool use_omp = arg1_int < 0;
  const int num_blocks = abs(arg1_int);
  const bool use_pthreads = false;
  const bool use_cuda = !( use_pthreads || use_omp );

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 256 : atoi(argv[2]);
  app.num_threads = !use_cuda ? -arg1_int : num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  app.reduction_method = argc < 5 ? 0 : atoi(argv[4]);
  const int sum_array_size =
    app.reduction_method ? num_blocks : app.num_threads;

  if ( app.num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ -NUM_PTHREADS | NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  if ( use_cuda )
    print_gpu_and_kernel_info();

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );
  app.d_v4_in = (V4*) app.d_v_in;

  //  Allocate storage on CPU and GPU for the minimum magnitude (sq) and
  //  its index.
  //
  app.thd_sum = new Elt_Type[sum_array_size];
  CE( hipMalloc( &app.d_thd_sum, sum_array_size * sizeof(Elt_Type) ) );

  // Initialize device memory to zeros. Helps catch bugs.
  //
  CE( hipMemset( app.d_thd_sum, 0, sum_array_size*sizeof(Elt_Type) ) );

  printf
    ("\nPreparing for %d %s threads for %d vectors.  "
     "Reduction method %d.\n",
         app.num_threads,
         use_cuda ? "GPU" : "CPU",
         app.array_size, app.reduction_method);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  double grand_sum = 0;

  if ( use_omp )
    {
      const double time_start = time_fp();
      grand_sum = omp_start();
      elapsed_time_s = time_fp() - time_start;
    }
  else if ( use_pthreads )
    {
      const double time_start = time_fp();

      // Allocate a structure to hold pthread thread ids.
      //
      pthread_t* const ptid = new pthread_t[app.num_threads];

      // Set up a pthread attribute, used for specifying options.
      //
      pthread_attr_t attr;
      pthread_attr_init(&attr);
      pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

      // Launch the threads.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);

      // Wait for each thread to finish.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_join( ptid[i], NULL );

      elapsed_time_s = time_fp() - time_start;
    }
  else
    {
      // Prepare events used for timing.
      //
      hipEvent_t gpu_start_ce, gpu_stop_ce;
      CE(hipEventCreate(&gpu_start_ce));
      CE(hipEventCreate(&gpu_stop_ce));

      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      // Launch kernel multiple times and keep track of the best time.

      const int num_reps = 5;
      for ( int r=0; r<num_reps; r++ )
        {
          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          printf("Launching with %d blocks of %d threads ... ",
                 num_blocks, thd_per_block);

          // Tell CUDA to start our threads on the GPU.
          //
          cuda_thread_start<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

          const double this_elapsed_time_s = cuda_time_ms * 0.001;
          printf(" %11.3f µs\n", this_elapsed_time_s * 1e6 );

          elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);
        }

      // Copy output array from GPU to CPU.
      //
      CE( hipMemcpy
          ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );

      //  Copy back per-thread sums.
      //
      CE( hipMemcpy
          ( app.thd_sum, app.d_thd_sum,
            sizeof(Elt_Type) * sum_array_size, hipMemcpyDeviceToHost) );

      // Find the sum of each thread or block's sum.
      //
      grand_sum = app.thd_sum[0];
      for ( int i=1; i<sum_array_size; i++ )
        grand_sum += app.thd_sum[i];
    }

  const double data_size = app.array_size * sizeof(Vertex) * 2;
  const double fp_op_count = app.array_size * ( 2 * N * N - N  );

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time_s);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time_s,
         1e-9 * data_size / elapsed_time_s);

  {
    // Compute correct answer.
    double cpu_grand_sum = 0;

    for ( int h=0; h<app.array_size; h++ )
      {
        Vertex p = app.v_in[h];
        Vertex q;
        for ( int i=0; i<N; i++ )
          {
            q.a[i] = 0;
            for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
          }
        Elt_Type sos = 0; for(int i=0; i<N; i++ ) sos+= q.a[i]*q.a[i];
        cpu_grand_sum += sos;
      }
    Elt_Type diff = fabs(grand_sum-cpu_grand_sum) / app.array_size;
    printf
      ("\nSum is %s,  %.1f %s %.1f (correct)\n",
       diff < 1e-5 ? "correct" : "**wrong**",
       grand_sum,
       grand_sum == cpu_grand_sum ? "==" : diff < 1e-5 ? "~" : "!=",
       cpu_grand_sum
       );

  }
}
