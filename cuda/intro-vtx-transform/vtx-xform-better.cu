#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2015), GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>

#include "util.h"

#define N 4

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(16) Vertex
{
  Elt_Type __align__(16) a[N];
};

struct __align__(16) V4 {
  Elt_Type x, y, z, w;
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
  V4 * d_v4_in;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

// The entry point for the GPU code.
//
__global__ void
cuda_thread_start()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Compute element number to start at.
  //
#if 1
  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = d_app.num_threads;
#else
  const int start = tid * elt_per_thread;
  const int stop = start + elt_per_thread;
  const int inc = 1;
#endif

  for ( int h=start; h<stop; h += inc )
    {
#if 1
      Vertex p = d_app.d_v_in[h];
#else
      V4 p2 = d_app.d_v4_in[h];
      Vertex p; 
      p.a[0] = p2.x; p.a[1] = p2.y; p.a[2] = p2.z; p.a[3] = p2.w;
#endif
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;
    }
}

void
print_gpu_and_kernel_info()
{
  gpu_info_print();

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

int
main(int argc, char **argv)
{
  // Examine argument 1, block count, if negative, use pthreads.
  //
  const int arg1_int = argc < 2 ? 1 : atoi(argv[1]);
  const bool use_pthreads = arg1_int < 0;
  const int num_blocks = abs(arg1_int);

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 1024 : atoi(argv[2]);
  app.num_threads = use_pthreads ? -arg1_int : num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 22 : int( atof(argv[3]) * (1<<20) );

  if ( app.num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ -NUM_PTHREADS | NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  if ( !use_pthreads && argc == 1 )
    print_gpu_and_kernel_info();

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );
  app.d_v4_in = (V4*) app.d_v_in;

  printf
    ("\nPreparing for %d %s threads operating on %d vectors of %d elements.\n",
         app.num_threads,
         use_pthreads ? "CPU" : "GPU",
         app.array_size, N);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  if ( use_pthreads )
    {
      const double time_start = time_fp();

      // Allocate a structure to hold pthread thread ids.
      //
      pthread_t* const ptid = new pthread_t[app.num_threads];

      // Set up a pthread attribute, used for specifying options.
      //
      pthread_attr_t attr;
      pthread_attr_init(&attr);
      pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

      // Launch the threads.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_create(&ptid[i], &attr, pt_thread_start, (void*)ptrdiff_t(i));

      // Wait for each thread to finish.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_join( ptid[i], NULL );

      elapsed_time_s = time_fp() - time_start;
    }
  else
    {
      // Prepare events used for timing.
      //
      hipEvent_t gpu_start_ce, gpu_stop_ce;
      CE(hipEventCreate(&gpu_start_ce));
      CE(hipEventCreate(&gpu_stop_ce));

      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      // Launch kernel multiple times and keep track of the best time.

      const int num_reps = 5;
      for ( int r=0; r<num_reps; r++ )
        {
          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          printf("Launching with %d blocks of %d threads ... ",
                 num_blocks, thd_per_block);

          // Tell CUDA to start our threads on the GPU.
          //
          cuda_thread_start<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

          const double this_elapsed_time_s = cuda_time_ms * 0.001;
          printf(" %11.3f µs\n", this_elapsed_time_s * 1e6 );

          elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);
        }

      // Copy output array from GPU to CPU.
      //
      CE( hipMemcpy
          ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );

    }

  const double data_size = app.array_size * sizeof(Vertex) * 2;
  const double fp_op_count = app.array_size * ( 2 * N * N - N  );

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time_s);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time_s,
         1e-9 * data_size / elapsed_time_s);
}
