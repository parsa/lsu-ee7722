#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2015), GPU Microarchitecture
//
// Simple CUDA Example, without LSU ECE helper classes.

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include "util.h"

#define N 4

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(16) Vertex
{
  Elt_Type __align__(16) a[N];
};

struct __align__(16) V4 {
  Elt_Type x, y, z, w;
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
  V4 * d_v4_in;

  int reduction_method;

  Elt_Type *thd_sum;
  Elt_Type *thd_min_mag_sq;
  int *thd_min_mag_sq_idx;
  Elt_Type *d_thd_sum;
  Elt_Type *d_thd_min_mag_sq;
  int *d_thd_min_mag_sq_idx;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

const int warp_size = 32;

#define BLOCK_SIZE_MAX 1024
__shared__ Elt_Type shared_sum[BLOCK_SIZE_MAX];
__shared__ int shared_mag_sq_idx[BLOCK_SIZE_MAX];
__shared__ Elt_Type wshared_mag_sq[warp_size];
__shared__ int wshared_mag_sq_idx[warp_size];

__device__ void find_min_method_1(float mag_sq, int idx);
__device__ void find_min_method_2(float mag_sq, int idx);
__device__ void find_min_method_3(float mag_sq, int idx);
__device__ void find_min_method_4(float mag_sq, int idx);


// The entry point for the GPU code.
//
__global__ void
cuda_thread_start()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Compute element number to start at.
  //
  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = d_app.num_threads;

  int min_mag_idx = -1;
  Elt_Type min_sum_of_sq = 0;

  for ( int h=start; h<stop; h += inc )
    {
      V4 p2 = d_app.d_v4_in[h];
      Vertex p; 
      p.a[0] = p2.x; p.a[1] = p2.y; p.a[2] = p2.z; p.a[3] = p2.w;
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;

      //  Compute the magnitude squared of q and update "min" variables
      //  it it's the smallest yet.
      //
      Elt_Type sum_of_sq = 0;
      for ( int i=0; i<N; i++ ) sum_of_sq += q.a[i] * q.a[i];
      if ( h == start || sum_of_sq < min_sum_of_sq )
        {
          min_sum_of_sq = sum_of_sq;
          min_mag_idx = h;
        }
    }

  switch ( d_app.reduction_method ) {
  case 0: 
    d_app.d_thd_min_mag_sq[tid] = min_sum_of_sq;
    d_app.d_thd_min_mag_sq_idx[tid] = min_mag_idx;
    break;
  case 1: find_min_method_1(min_sum_of_sq,min_mag_idx); break;
  case 2: find_min_method_2(min_sum_of_sq,min_mag_idx); break;
  case 3: find_min_method_3(min_sum_of_sq,min_mag_idx); break;
  case 4: find_min_method_4(min_sum_of_sq,min_mag_idx); break;

  default:  d_app.d_thd_min_mag_sq[threadIdx.x] = 0; break;
  }

}

__device__ void
find_min_method_1(float thd_min_mag_sq, int thd_min_mag_sq_idx)
{
  shared_mag_sq[threadIdx.x] = thd_min_mag_sq;
  shared_mag_sq_idx[threadIdx.x] = thd_min_mag_sq_idx;
  __syncthreads();

  if ( threadIdx.x != 0 ) return;

  Elt_Type our_min = thd_min_mag_sq;
  int our_idx = thd_min_mag_sq_idx;

  for ( int i=1; i<blockDim.x; i++ )
    if ( shared_mag_sq[i] < our_min )
      {
        our_min = shared_mag_sq[i];
        our_idx = shared_mag_sq_idx[i];
      }

  d_app.d_thd_min_mag_sq[blockIdx.x] = our_min;
  d_app.d_thd_min_mag_sq_idx[blockIdx.x] = our_idx;
}

__device__ void
find_min_method_2(float thd_min_mag_sq, int thd_min_mag_sq_idx)
{
  shared_mag_sq[threadIdx.x] = thd_min_mag_sq;
  shared_mag_sq_idx[threadIdx.x] = thd_min_mag_sq_idx;

  Elt_Type our_min = thd_min_mag_sq;
  int our_idx = thd_min_mag_sq_idx;

  // Note: Block size must be a power of 2.

  for ( int dist = blockDim.x >> 1;  dist;  dist >>= 1 )
    {
      __syncthreads();
      if ( threadIdx.x < dist
           && shared_mag_sq[ threadIdx.x + dist ] < our_min )
        {
          our_min = shared_mag_sq[ threadIdx.x ] =
            shared_mag_sq[ threadIdx.x + dist ];
          our_idx = shared_mag_sq_idx[ threadIdx.x ] =
            shared_mag_sq_idx[ threadIdx.x + dist ];
        }
    }

  if ( threadIdx.x ) return;

  d_app.d_thd_min_mag_sq[blockIdx.x] = our_min;
  d_app.d_thd_min_mag_sq_idx[blockIdx.x] = our_idx;
}

__device__ void
find_min_method_3(float thd_min_mag_sq, int thd_min_mag_sq_idx)
{
  shared_mag_sq[threadIdx.x] = thd_min_mag_sq;
  shared_mag_sq_idx[threadIdx.x] = thd_min_mag_sq_idx;
  __syncthreads();

  Elt_Type our_min = thd_min_mag_sq;
  int our_idx = thd_min_mag_sq_idx;

  const int warp_size = 32;
  const int half_warp_size = warp_size >> 1;

  if ( threadIdx.x < warp_size )
    for ( int i = threadIdx.x + warp_size;  i < blockDim.x;  i += warp_size )
      if ( shared_mag_sq[ i ] < our_min )
        {
          our_min = shared_mag_sq[ i ];
          our_idx = shared_mag_sq_idx[ i ];
        }

  shared_mag_sq[threadIdx.x] = our_min;
  shared_mag_sq_idx[threadIdx.x] = our_idx;

  if ( threadIdx.x < half_warp_size )
    for ( int dist = half_warp_size;  dist;  dist >>= 1 )
      if ( threadIdx.x < dist
           && shared_mag_sq[ threadIdx.x + dist ] < our_min )
        {
          our_min = shared_mag_sq[ threadIdx.x ] =
            shared_mag_sq[ threadIdx.x + dist ];
          our_idx = shared_mag_sq_idx[ threadIdx.x ] =
            shared_mag_sq_idx[ threadIdx.x + dist ];
        }

  if ( threadIdx.x ) return;

  d_app.d_thd_min_mag_sq[blockIdx.x] = our_min;
  d_app.d_thd_min_mag_sq_idx[blockIdx.x] = our_idx;
}

__device__ void
find_min_method_4(float thd_min_mag_sq, int thd_min_mag_sq_idx)
{
  shared_mag_sq[threadIdx.x] = thd_min_mag_sq;
  shared_mag_sq_idx[threadIdx.x] = thd_min_mag_sq_idx;

  Elt_Type our_min = thd_min_mag_sq;
  int our_idx = thd_min_mag_sq_idx;

  const int warp_lg = 5;
  const int warp_size = 1 << warp_lg;
  const int half_warp_size = warp_size >> 1;
  const int lane = threadIdx.x & ( warp_size - 1 );

  for ( int dist = half_warp_size;  dist;  dist >>= 1 )
    if ( lane < dist && shared_mag_sq[ threadIdx.x + dist ] < our_min )
      {
        our_min = shared_mag_sq[ threadIdx.x ] =
          shared_mag_sq[ threadIdx.x + dist ];
        our_idx = shared_mag_sq_idx[ threadIdx.x ] =
          shared_mag_sq_idx[ threadIdx.x + dist ];
      }


#if 1

  const int warp_num = threadIdx.x >> warp_lg;

  if ( !lane )
    {
      wshared_mag_sq[ warp_num ] = our_min;
      wshared_mag_sq_idx[ warp_num ] = our_idx;
    }

  __syncthreads();

  if ( threadIdx.x < half_warp_size )
  for ( int dist = half_warp_size;  dist;  dist >>= 1 )
    if ( lane < dist && wshared_mag_sq[ threadIdx.x + dist ] < our_min )
      {
        our_min = wshared_mag_sq[ threadIdx.x ] =
          wshared_mag_sq[ threadIdx.x + dist ];
        our_idx = wshared_mag_sq_idx[ threadIdx.x ] =
          wshared_mag_sq_idx[ threadIdx.x + dist ];
      }

#else

  __syncthreads();

  const int our_elt = threadIdx.x << warp_lg;
  const int block_size = blockDim.x;

  if ( threadIdx.x < warp_size )
  for ( int dist = block_size >> 1;  dist >= warp_size;  dist >>= 1 )
      if ( our_elt < dist
           && shared_mag_sq[ our_elt + dist ] < our_min )
        {
          our_min = shared_mag_sq[ our_elt ] = shared_mag_sq[ our_elt + dist ];
          our_idx = shared_mag_sq_idx[ our_elt ]
            = shared_mag_sq_idx[ our_elt + dist ];
        }

#endif

  if ( threadIdx.x ) return;

  d_app.d_thd_min_mag_sq[blockIdx.x] = our_min;
  d_app.d_thd_min_mag_sq_idx[blockIdx.x] = our_idx;
}

void
print_gpu_and_kernel_info()
{
  print_gpu_info();

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

int
main(int argc, char **argv)
{
  // Examine argument 1, block count, if negative, use pthreads.
  //
  const int arg1_int = argc < 2 ? 32 : atoi(argv[1]);
  const bool use_pthreads = arg1_int < 0;
  const int num_blocks = abs(arg1_int);

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 256 : atoi(argv[2]);
  app.num_threads = use_pthreads ? -arg1_int : num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  app.reduction_method = argc < 5 ? 0 : atoi(argv[4]);
  const int mag_array_size =
    app.reduction_method ? num_blocks : app.num_threads;

  if ( app.num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ -NUM_PTHREADS | NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  if ( !use_pthreads )
    print_gpu_and_kernel_info();

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );
  app.d_v4_in = (V4*) app.d_v_in;

  //  Allocate storage on CPU and GPU for the minimum magnitude (sq) and
  //  its index.
  //
  app.thd_min_mag_sq_idx = new int[mag_array_size];
  app.thd_min_mag_sq = new Elt_Type[mag_array_size];
  CE( hipMalloc( &app.d_thd_min_mag_sq_idx, mag_array_size * sizeof(int) ) );
  CE( hipMalloc( &app.d_thd_min_mag_sq, mag_array_size * sizeof(Elt_Type) ) );

  // Initialize device memory to zeros. Helps catch bugs.
  //
  CE( hipMemset( app.d_thd_min_mag_sq_idx, 0, mag_array_size*sizeof(int) ) );
  CE( hipMemset( app.d_thd_min_mag_sq, 0, mag_array_size*sizeof(Elt_Type) ) );

  printf
    ("\nPreparing for %d %s threads for %d vectors.  "
     "Reduction method %d.\n",
         app.num_threads,
         use_pthreads ? "CPU" : "GPU",
         app.array_size, app.reduction_method);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  int minimum_mag_index = 0;
  Elt_Type minimum_mag_val = 0;

  if ( use_pthreads )
    {
      const double time_start = time_fp();

      // Allocate a structure to hold pthread thread ids.
      //
      pthread_t* const ptid = new pthread_t[app.num_threads];

      // Set up a pthread attribute, used for specifying options.
      //
      pthread_attr_t attr;
      pthread_attr_init(&attr);
      pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

      // Launch the threads.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);

      // Wait for each thread to finish.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_join( ptid[i], NULL );

      elapsed_time_s = time_fp() - time_start;
    }
  else
    {
      // Prepare events used for timing.
      //
      hipEvent_t gpu_start_ce, gpu_stop_ce;
      CE(hipEventCreate(&gpu_start_ce));
      CE(hipEventCreate(&gpu_stop_ce));

      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      // Launch kernel multiple times and keep track of the best time.

      const int num_reps = 5;
      for ( int r=0; r<num_reps; r++ )
        {
          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          printf("Launching with %d blocks of %d threads ... ",
                 num_blocks, thd_per_block);

          // Tell CUDA to start our threads on the GPU.
          //
          cuda_thread_start<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

          const double this_elapsed_time_s = cuda_time_ms * 0.001;
          printf(" %11.3f µs\n", this_elapsed_time_s * 1e6 );

          elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);
        }
    }

  // Copy output array from GPU to CPU.
  //
  CE( hipMemcpy
      ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );

  //  Copy back per-thread minimum magnitudes (squared) and their indices.
  //
  CE( hipMemcpy
      ( app.thd_min_mag_sq, app.d_thd_min_mag_sq,
        sizeof(Elt_Type) * mag_array_size, hipMemcpyDeviceToHost) );
  CE( hipMemcpy
      ( app.thd_min_mag_sq_idx, app.d_thd_min_mag_sq_idx,
        sizeof(int) * mag_array_size, hipMemcpyDeviceToHost) );

  // Find the minimum magnitude squared and its index.
  //
  Elt_Type min_sos = app.thd_min_mag_sq[0];
  minimum_mag_index = app.thd_min_mag_sq_idx[0];
  for ( int i=1; i<mag_array_size; i++ )
    if ( app.thd_min_mag_sq[i] < min_sos )
      {
        min_sos = app.thd_min_mag_sq[i];
        minimum_mag_index = app.thd_min_mag_sq_idx[i];
      }

  // Take the square root to get the magnitude.
  //
  minimum_mag_val = sqrt(min_sos);

  const double data_size = app.array_size * sizeof(Vertex) * 2;
  const double fp_op_count = app.array_size * ( 2 * N * N - N  );

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time_s);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time_s,
         1e-9 * data_size / elapsed_time_s);

  {
    // Compute correct answer.
    Elt_Type min_val = 0;
    int min_idx = -1;

    for ( int h=0; h<app.array_size; h++ )
      {
        Vertex p = app.v_in[h];
        Vertex q;
        for ( int i=0; i<N; i++ )
          {
            q.a[i] = 0;
            for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
          }
        Elt_Type sos = 0; for(int i=0; i<N; i++ ) sos+= q.a[i]*q.a[i];
        Elt_Type mag = sqrt(sos);
        if ( min_idx < 0 || mag < min_val ) { min_val = mag; min_idx = h; }
      }
    Elt_Type diff = fabs(min_val-minimum_mag_val);
    printf
      ("\nMinimum mag is %s,  %d %s %d (correct)  %.4f %s %.4f (correct)\n",
       diff < 1e-5 ? "correct" : "**wrong**",
       minimum_mag_index,
       min_idx == minimum_mag_index ? "==" : "!=",
       min_idx,
       minimum_mag_val,
       min_val == minimum_mag_val ? "==" : diff < 1e-5 ? "~" : "!=",
       min_val
       );

  }

}
