#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 2 - Spring 2015
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2015/hw02.pdf

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "util.h"

#define N 8
#define M 8


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  // No local memory.
  //
  // In the inner loop use global memory accesses to access the input
  // vector elements. Hope that the compiler recognizes the repeated
  // accesses and so keeps each input vector element in a register
  // rather than reading global memory M times per input element.
  //
  // The compiler will avoid the repeated reads if it is convinced
  // that the input and output arrays don't overlap.  For the NVIDIA
  // compiler (CUDA 7.0) that seems to require declaring the array
  // pointers with the __restrict__ attributes as kernel arguments.
  //
  // Note that dout and d_app.d_out hold the same address, as do din
  // and d_app.d_in.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
tryout(Elt_Type *dout, const Elt_Type *din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  for ( int h = tid; h<d_app.num_vecs; h += num_threads )
    dout[h] = din[h] + din[h+0x20] + 1234;
}

extern "C" __global__ void
mxv_i_lbuf()
{
  // Local memory for input vector.
  //
  // Use local memory to buffer entire input vector, and write
  // each output vector element as soon as its computed.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      Elt_Type vin[N];
      for ( int c=0; c<N; c++ ) vin[c] = d_app.d_in[ h * N + c ];

      for ( int r=0; r<M; r++ )
        {
          Elt_Type elt = 0;
          for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * vin[c];
          d_app.d_out[ h * M + r ] = elt;
        }
    }
}

extern "C" __global__ void
mxv_o_per_thd()
{
  // Assign one vector to M threads, each thread computes one element.
  //
  // This arrangement avoids the need for any local memory buffering,
  // results in efficient global memory writes. Global memory reads
  // are still inefficient.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid / M;  // First vector number computed by this thread.
  const int r = tid % M;      // Vector element computed by this thread.

  const int stop = d_app.num_vecs;
  const int inc = num_threads / M;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h, compute output vector element r.

      Elt_Type vout = 0;

      for ( int c=0; c<N; c+=4 )
        {
          // float4 v4 = d_app.d_in_f4[ ( h * N + c ) / 4 ];  // Plan A
          float4 v4 = d_app.d_in_f4[ ( h * N ) / 4 + c / 4 ]; // Plan B
          vout +=
            d_app.matrix[r][c] * v4.x
            + d_app.matrix[r][c+1] * v4.y
            + d_app.matrix[r][c+2] * v4.z
            + d_app.matrix[r][c+3] * v4.w;
        }

      d_app.d_out[ h * M + r ] = vout;
    }
}


extern "C" __global__ void
mxv_sh_ochunk()
{
  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

      for ( int c=0; c<N; c += CS )
        {
          vxfer[threadIdx.x] =
            c + CS < N || c + thd_c_offset < N
            ? d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ]
            : 0;

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }

      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < M )
            d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }

    }
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_i_lbuf);
  info.GET_INFO(mxv_o_per_thd);

  info.GET_INFO(mxv_sh_ochunk);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int =
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks =
           arg1_int == 0 ? info.cuda_prop.multiProcessorCount : abs(arg1_int);

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.num_vecs * N;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int out_size_elts = app.num_vecs * M;
  const int out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const int max_st_len = 52;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d wp  %6.0f µs  %5.0f GF  %5.0f GB/s %s\n",
                       (thd_per_block + 31 ) >> 5,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(comp_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
