#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 1 - Spring 2015
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2015/hw01.pdf

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "util.h"

#define N 16
#define M 16


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  // No local memory.
  //
  // In the inner loop use global memory accesses to access the input
  // vector elements. Hope that the compiler recognizes the repeated
  // accesses and so keeps each input vector element in a register
  // rather than reading global memory M times per input element.
  //
  // The compiler will avoid the repeated reads if it is convinced
  // that the input and output arrays don't overlap.  For the NVIDIA
  // compiler (CUDA 7.0) that seems to require declaring the array
  // pointers with the __restrict__ attributes as kernel arguments.
  //
  // Note that dout and d_app.d_out hold the same address, as do din
  // and d_app.d_in.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.
#pragma unroll
    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_i_lbuf()
{
  // Local memory for input vector.
  //
  // Use local memory to buffer entire input vector, and write
  // each output vector element as soon as its computed.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      Elt_Type vin[N];
      for ( int c=0; c<N; c++ ) vin[c] = d_app.d_in[ h * N + c ];

      for ( int r=0; r<M; r++ )
        {
          Elt_Type elt = 0;
          for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * vin[c];
          d_app.d_out[ h * M + r ] = elt;
        }
    }
}


extern "C" __global__ void
mxv_o_lbuf()
{
  // Local memory for output vector.
  //
  // Use local memory to buffer entire output vector.  Right after each
  // input vector element is read use it to partially compute each
  // of the M output vector elements.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h.

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ ) vout[r] = 0;

      for ( int c=0; c<N; c++ )
        {
          const Elt_Type vin = d_app.d_in[ h * N + c ];
          for ( int r=0; r<M; r++ ) vout[r] += d_app.matrix[r][c] * vin;
        }
      for ( int r=0; r<M; r++ ) d_app.d_out[ h * M + r ] = vout[ r ];
    }
}


extern "C" __global__ void
mxv_o_per_thd()
{
  // Assign one vector to M threads, each thread computes one element.
  //
  // This arrangement avoids the need for any local memory buffering,
  // results in efficient global memory writes. Global memory reads
  // are still inefficient.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid / M;  // First vector number computed by this thread.
  const int r = tid % M;      // Vector element computed by this thread.

  const int stop = d_app.num_vecs;
  const int inc = num_threads / M;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h, compute output vector element r.

      Elt_Type vout = 0;

      for ( int c=0; c<N; c++ )
        vout += d_app.matrix[r][c] * d_app.d_in[ h * N + c ];

      d_app.d_out[ h * M + r ] = vout;
    }
}


extern "C" __global__ void
mxv_sh()
{
  // Local memory for output vector, use shared memory to redistribute
  // input- and output-vector elements so that global memory reads and
  // writes are fully utilized.

  // Chunk Size: Number of vector elements to redistribute at a time.
  const int CS = 8;

  const int num_threads = blockDim.x * gridDim.x;

  // The vector number operated on by threadIdx.x 0 in the first iteration.
  //
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  // Used to compute the vector element number to read or write from
  // global memory.
  //
  const int thd_x_offset = threadIdx.x % CS;

  // Used to compute the vector number to read or write from global memory.
  //
  const int thd_x_idx_st = threadIdx.x / CS;

  const int64_t BLOCK_SIZE = blockDim.x;
  const int64_t MAX_BLOCK_SIZE = 1024;

  // Shared memory used to redistribute vector elements.
  //
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE][CS + 1];

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      // Compute matrix-vector product for vector number  hb + threadIdx.x.

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ ) vout[r] = 0;

      for ( int c=0; c<N; c += CS )
        {
          // Read and redistribute input vector elements c, c+1,
          // ... c+CS-1, then use those to compute part of the
          // matrix-vector product.

          // Read input vector elements sequentially and place them in
          // shared memory.
          //
          // The entire g loop reads CS elements of each thread's
          // input vector. Each iteration of the g loop reads CS
          // elements from BLOCK_SIZE / CS vectors.
          //
          __syncthreads();
          for ( int g=0; g<CS; g++ )
            {
              const int idx = g * BLOCK_SIZE / CS + thd_x_idx_st;
              vxfer[idx][thd_x_offset] =
                d_app.d_in[ hb * N + idx * N + c + thd_x_offset ];
            }

          // Copy the portion of the input vector just read to local
          // memory (the vin array). We expect that the compiler will
          // use registers for all values of vin.
          //
          __syncthreads();
          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ ) vin[cc] = vxfer[threadIdx.x][cc];

          // Perform the matrix-vector multiplication for the parts of
          // the input vector just read.
          //
          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              if ( c+cc < N ) vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      // Use shared memory to redistribute the output vector elements to
      // threads so that the write to global memory will be efficient.
      //
      for ( int r=0; r<M; r += CS )
        {
          __syncthreads();
          for ( int rr=0; rr<CS; rr++ ) vxfer[threadIdx.x][rr] = vout[r+rr];

          __syncthreads();
          for ( int g=0; g<CS; g++ )
            {
              const int idx = g * BLOCK_SIZE / CS + thd_x_idx_st;

              // The if statement is needed of M is not a multiple of CS.
              if ( thd_x_offset + r < M )
                d_app.d_out[ hb * M + idx * M + r + thd_x_offset ] =
                  vxfer[idx][thd_x_offset];
            }
        }
    }
}



extern "C" __global__ void
mxv_sh_ochunk()
{
  // Compute element number to start at.
  //

  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                if ( c+cc < N )
                  vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < M )
            d_app.d_out[ hb * M + ( threadIdx.x / CS ) * M + r ] = vout[rr];
        }

    }
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_i_lbuf);
  info.GET_INFO(mxv_o_lbuf);
  info.GET_INFO(mxv_o_per_thd);

  info.GET_INFO(mxv_sh);
  info.GET_INFO(mxv_sh_ochunk);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = 
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks = 
           arg1_int == 0 ? info.cuda_prop.multiProcessorCount : abs(arg1_int);

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.num_vecs * N;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int out_size_elts = app.num_vecs * M;
  const int out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac = 
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const int max_st_len = 52;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d wp  %6.0f µs  %5.0f GF  %5.0f GB/s %s\n",
                       (thd_per_block + 31 ) >> 5,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(comp_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
