#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2021
 /// Homework 2 -- SOLUTION
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2021/hw02.pdf
 //

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <ptable.h>
#include <nperf.h>

typedef float wht_t;
typedef float acc_t;

struct Layer_Shape
{
  int nn, nc, ni, no, nm;
};

//                               n    c   i   o   m
constexpr Layer_Shape ls[] = { { 800, 20, 32, 32, 20 },
                               { 800, 52, 48, 48, 52 } };

constexpr int n_shapes = sizeof(ls)/sizeof(ls[0]);

struct Layer
{
  wht_t *w_h, *w_d;
  wht_t *w2_h, *w2_d;
  acc_t *ai_h, *ai_d;
  acc_t *ao_h, *ao_d, *ao_check_h;

  Layer_Shape s;

  size_t sz_ai_elts, sz_ao_elts, sz_w_elts;
  size_t sz_ai_bytes, sz_ao_bytes, sz_w_bytes;

};


template<int tpnn=0, int tpnc=0, int tpni=0>
__global__ void
dnn_base(Layer l)
{
  Layer_Shape& s = l.s;

  const int [[gnu::unused]] tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int [[gnu::unused]] num_threads = blockDim.x * gridDim.x;

  // If template parameters are non-zero verify and use them.
  const int nn = tpnn ? tpnn : s.nn;  // Number of batches.
  const int ni = tpni ? tpni : s.ni;  // Number of input neurons per channel.
  const int nc = tpnc ? tpnc : s.nc;  // Number of input channels.
  const int no = ni;                  // Number of output neurons per channel.
  const int nm = nc;                  // Number of output channels.
  assert( ni == s.ni );
  assert( nc == s.nc );
  assert( nn == s.nn );

  acc_t* const ai = l.ai_d;
  acc_t* const ao = l.ao_d;
  wht_t [[gnu::unused]] * const w = l.w_d;
  wht_t [[gnu::unused]] * const w2 = l.w2_d;

  const int nnmo = nn * nm * no;

  for ( int inmo = tid; inmo < nnmo; inmo += num_threads )
    {
      // o n m
      const int im = inmo % nm;
      const int ino = inmo / nm;
      const int in = ino % nn;
      const int io = ino / nn;

      acc_t ac = 0;
      for ( int ic = 0; ic < nc; ic++ )
        for ( int ii = 0; ii < ni; ii++ )
          ac +=
            ai[ ii + ni * ( ic + nc * in ) ]
            * w[ im + nm * ( ii + ni * ( ic + nc * io ) ) ];

      ao[ io + no * ( im + nm * in ) ] = ac;
    }
}

template<int tpnn=0, int tpnc=0, int tpni=0>
__global__ void
dnn_sol_a(Layer l)
{
  Layer_Shape& s = l.s;

  const int [[gnu::unused]] tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int [[gnu::unused]] num_threads = blockDim.x * gridDim.x;

  // If template parameters are non-zero verify and use them.
  const int nn = tpnn ? tpnn : s.nn;  // Number of batches.
  const int ni = tpni ? tpni : s.ni;  // Number of input neurons per channel.
  const int nc = tpnc ? tpnc : s.nc;  // Number of input channels.
  const int no = ni;                  // Number of output neurons per channel.
  const int nm = nc;                  // Number of output channels.
  assert( ni == s.ni );
  assert( nc == s.nc );
  assert( nn == s.nn );


  /// SOLUTION 
  //  Wrap l.ai_d and other pointers in __builtin_assume_aligned with
  //  alignment set to 16 bytes so that compiler can use vector loads.
  //
  acc_t* const ai = (acc_t*) __builtin_assume_aligned(l.ai_d,16);
  acc_t* const ao = (acc_t*) __builtin_assume_aligned(l.ao_d,16);
  wht_t [[gnu::unused]] * const w = (wht_t*) __builtin_assume_aligned(l.w_d,16);
  wht_t [[gnu::unused]] * const w2 =
    (wht_t*) __builtin_assume_aligned(l.w2_d,16);

  /// SOLUTION -- Partial (See dnn_sol_b for full solution)
  //
  //  Load a weight once and use it 8 (or value of bn) times.
  //
  //  To do this a new loop, i_bn has been added ..
  //  .. and the outer inmo loop iterates for 1/8 the number of iterations.
  //
  //  With the code in this file each load for a weight will be used 8
  //  (bn) times, but each load of ai will be used just once. See
  //  dnn_sol_b for a solution in which loads of both w and ai are
  //  re-used.

  // Blocking factor for n.
  constexpr int bn = 8;

  constexpr int ab = bn;

  // The number of outer-loop iterations without blocking.
  const int nnmo = nn * nm * no;

  // The number of outer-loop iterations with blocking. 
  const int nnmo_ab = nnmo / ab;

  // Number of values of in iterated over by outer loop.
  const int nn_bn = nn / bn;

  for ( int inmo = tid; inmo < nnmo_ab; inmo += num_threads )
    {
      // o n m
      const int im = inmo % nm;
      const int ino = inmo / nm;

      // The first of 8 (bn) values of in0.
      const int in0 = ino % nn_bn * bn;
      //
      // That is, the loop below will compute outputs for
      //    in =  in0, in0+1, in0+2, ..., in0+bn-1.

      const int io = ino / nn_bn;

      // Declare storage for the bn different values being computed.
      //
      acc_t ac[bn]{};

      for ( int ic = 0; ic < nc; ic++ )
        /// SOLUTION
        //  Unroll by 4 so that can use vector loads for ai.
#pragma unroll 4
        for ( int ii = 0; ii < ni; ii++ )
          {
            wht_t wht = w[ im + nm * ( ii + ni * ( ic + nc * io ) ) ];
            for ( int i_bn = 0; i_bn < bn; i_bn++ )
              {
                const int in = in0 + i_bn;
                ac[i_bn] += ai[ ii + ni * ( ic + nc * in ) ] * wht;
              }
          }
      for ( int i_bn = 0; i_bn < bn; i_bn++ )
        {
          const int in = in0 + i_bn;
          ao[ io + no * ( im + nm * in ) ] = ac[i_bn];
        }
    }
}

template<int tpnn=0, int tpnc=0, int tpni=0>
__global__ void
dnn_sol_b(Layer l)
{
  Layer_Shape& s = l.s;

  const int [[gnu::unused]] tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int [[gnu::unused]] num_threads = blockDim.x * gridDim.x;

  // If template parameters are non-zero verify and use them.
  const int nn = tpnn ? tpnn : s.nn;  // Number of batches.
  const int ni = tpni ? tpni : s.ni;  // Number of input neurons per channel.
  const int nc = tpnc ? tpnc : s.nc;  // Number of input channels.
  const int no = ni;                  // Number of output neurons per channel.
  const int nm = nc;                  // Number of output channels.
  assert( ni == s.ni );
  assert( nc == s.nc );
  assert( nn == s.nn );

  acc_t* const ai = (acc_t*) __builtin_assume_aligned(l.ai_d,16);
  acc_t* const ao = (acc_t*) __builtin_assume_aligned(l.ao_d,16);
  wht_t [[gnu::unused]] * const w = (wht_t*) __builtin_assume_aligned(l.w_d,16);
  wht_t [[gnu::unused]] * const w2 =
    (wht_t*) __builtin_assume_aligned(l.w2_d,16);


  /// SOLUTION -- Better
  //
  //  To re-use loads of ai, we need to compute the output for
  //  different values of io or im or both. That is because neither io
  //  or im are used to compute the address of ai.
  //
  //  The code below blocks the inmo loop into three new loops, i_bm,
  //  i_bo, and i_bn. Each loaded weight is used bn=8 times and each
  //  loaded input is used bo * bm = 2*4=8 times. (Based on the values
  //  of bn, bm, and bo AOTW.)

  constexpr int bo = 2;
  constexpr int bm = 4;
  constexpr int bn = 8;
  constexpr int ab = bo * bn * bm;

  const int nnmo = nn * nm * no;
  const int nnmo_ab = nnmo / ab;
  const int nn_bn = nn / bn;
  const int nm_bm = nm / bm;

  for ( int inmo = tid; inmo < nnmo_ab; inmo += num_threads )
    {
      // o n m
      const int im_bm = inmo % nm_bm;
      const int im0 = im_bm * bm;
      const int ino = inmo / nm_bm;
      const int in_bn = ino % nn_bn;
      const int in0 = in_bn * bn;
      const int io_bo = ino / nn_bn;
      const int io0 = io_bo * bo;

      acc_t ac[bo][bm][bn]{};
      for ( int ic = 0; ic < nc; ic++ )
#pragma unroll 4
        for ( int ii = 0; ii < ni; ii++ )
          {

            // Pre-load the inputs (ai) that will be needed, one input
            // for each of bn batches. Each input will be used bm * bo
            // times.
            //
            acc_t ain[bn];
            for ( int i_bn = 0; i_bn < bn; i_bn++ )
              {
                const int in = in0 + i_bn;
                ain[i_bn] = ai[ ii + ni * ( ic + nc * in ) ];
              }

            for ( int i_bm = 0; i_bm < bm; i_bm++ )
              for ( int i_bo = 0; i_bo < bo; i_bo++ )
                {
                  const int im = im0 + i_bm;
                  const int io = io0 + i_bo;

                  // Load a weight. The weight will be used bn times.
                  //
                  wht_t wht = w[ im + nm * ( ii + ni * ( ic + nc * io ) ) ];

                  for ( int i_bn = 0; i_bn < bn; i_bn++ )
                    ac[i_bo][i_bm][i_bn] += ain[i_bn] * wht;
                }
          }

      for ( int i_bn = 0; i_bn < bn; i_bn++ )
        for ( int i_bm = 0; i_bm < bm; i_bm++ )
          for ( int i_bo = 0; i_bo < bo; i_bo++ )
            {
              const int io = io0 + i_bo;
              const int in = in0 + i_bn;
              const int im = im0 + i_bm;
              ao[ io + no * ( im + nm * in ) ] = ac[i_bo][i_bm][i_bn];
            }
    }
}


template<int tpnn=0, int tpnc=0, int tpni=0>
__global__ void
dnn_sol_b_alt(Layer l)
{
  Layer_Shape& s = l.s;

  const int [[gnu::unused]] tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int [[gnu::unused]] num_threads = blockDim.x * gridDim.x;

  // If template parameters are non-zero verify and use them.
  const int nn = tpnn ? tpnn : s.nn;  // Number of batches.
  const int ni = tpni ? tpni : s.ni;  // Number of input neurons per channel.
  const int nc = tpnc ? tpnc : s.nc;  // Number of input channels.
  const int no = ni;                  // Number of output neurons per channel.
  const int nm = nc;                  // Number of output channels.
  assert( ni == s.ni );
  assert( nc == s.nc );
  assert( nn == s.nn );

  acc_t* const ai = (acc_t*) __builtin_assume_aligned(l.ai_d,16);
  acc_t* const ao = (acc_t*) __builtin_assume_aligned(l.ao_d,16);
  wht_t [[gnu::unused]] * const w = (wht_t*) __builtin_assume_aligned(l.w_d,16);
  wht_t [[gnu::unused]] * const w2 =
    (wht_t*) __builtin_assume_aligned(l.w2_d,16);


  /// SOLUTION -- Better, Simpler
  //
  //  This solution blocks the loop in the same way as dnn_sol_b,
  //  but it relies on the compiler to buffer w and ai. That is,
  //  in dnn_sol_b a value of w was copied into variable wht, and
  //  bn values of ai were copied into an array. The code below relies
  //  on the compiler to recognize that values of w and ai are reused
  //  and to load them just once. (For the compiler to do this the
  //  loop must be properly blocked, otherwise re-used values would
  //  not be assigned to the same thread.)

  constexpr int bo = 2;
  constexpr int bm = 4;
  constexpr int bn = 8;
  constexpr int ab = bo * bn * bm;

  const int nnmo = nn * nm * no;
  const int nnmo_ab = nnmo / ab;
  const int nn_bn = nn / bn;
  const int nm_bm = nm / bm;

  for ( int inmo = tid; inmo < nnmo_ab; inmo += num_threads )
    {
      // o n m
      const int im_bm = inmo % nm_bm;
      const int im0 = im_bm * bm;
      const int ino = inmo / nm_bm;
      const int in_bn = ino % nn_bn;
      const int in0 = in_bn * bn;
      const int io_bo = ino / nn_bn;
      const int io0 = io_bo * bo;

      acc_t ac[bo][bm][bn]{};
      for ( int ic = 0; ic < nc; ic++ )
#pragma unroll 4
        for ( int ii = 0; ii < ni; ii++ )
          for ( int i_bm = 0; i_bm < bm; i_bm++ )
            for ( int i_bo = 0; i_bo < bo; i_bo++ )
              for ( int i_bn = 0; i_bn < bn; i_bn++ )
                {
                  const int in = in0 + i_bn;
                  const int im = im0 + i_bm;
                  const int io = io0 + i_bo;

                  ac[i_bo][i_bm][i_bn] +=
                    ai[ ii + ni * ( ic + nc * in ) ]
                    * w[ im + nm * ( ii + ni * ( ic + nc * io ) ) ];
                }

      for ( int i_bn = 0; i_bn < bn; i_bn++ )
        for ( int i_bm = 0; i_bm < bm; i_bm++ )
          for ( int i_bo = 0; i_bo < bo; i_bo++ )
            {
              const int io = io0 + i_bo;
              const int in = in0 + i_bn;
              const int im = im0 + i_bm;
              ao[ io + no * ( im + nm * in ) ] = ac[i_bo][i_bm][i_bn];
            }
    }
}









void
layer_init(Layer &l)
{
  Layer_Shape& s = l.s;
  int &ni = s.ni; // Number of input neurons.
  int &no = s.no; // Number of output neurons.
  int &nc = s.nc; // Number of input channels.
  int &nm = s.nm; // Number of output channels.
  int &nn = s.nn; // Number of batches.

  // Seed random number generator. Will generate the same sequence
  // of random numbers each time.
  srand48(2735);

  l.sz_ai_elts = ni * nc * nn;
  l.sz_ao_elts = no * nm * nn;
  l.sz_w_elts = nc * nm * ni * no;

  l.ai_h = new acc_t[l.sz_ai_elts];
  l.ao_h = new acc_t[l.sz_ao_elts];
  l.ao_check_h = new acc_t[l.sz_ao_elts];
  l.w_h = new wht_t[l.sz_w_elts];
  l.w2_h = new wht_t[l.sz_w_elts];

  l.sz_ai_bytes = l.sz_ai_elts * sizeof(*l.ai_d);
  l.sz_w_bytes = l.sz_w_elts * sizeof(*l.w_d);
  l.sz_ao_bytes = l.sz_ao_elts * sizeof(*l.ao_d);

  CE( hipMalloc( &l.ai_d, l.sz_ai_bytes ) );
  CE( hipMalloc( &l.w_d, l.sz_w_bytes ) );
  CE( hipMalloc( &l.w2_d, l.sz_w_bytes ) );
  CE( hipMalloc( &l.ao_d, l.sz_ao_bytes ) );

  acc_t* const ai = l.ai_h;
  acc_t* const ao = l.ao_check_h;
  wht_t* const w = l.w_h;

  const bool debug = false;
  const bool debug_w = false;

  // Initialize ai array.
  //
  for ( int in = 0; in < nn; in++ )
    for ( int ic = 0; ic < nc; ic++ )
      for ( int ii = 0; ii < ni; ii++ )
        {
          size_t idx_ai = ii + ni * ( ic + nc * in );
          ai[idx_ai] = debug ? ic : drand48();
        }

  // Initialize w array.
  //
  for ( size_t i=0; i<l.sz_w_elts; i++ ) w[i] = debug_w ? 1.0 : drand48()-0.5;

# pragma omp parallel for
  for ( size_t i=0; i<l.sz_ao_elts; i++ ) ao[i] = -1;

  // Compute correct result.
  //
# pragma omp parallel for
  for ( int in = 0; in < nn; in++ )
    for ( int im = 0; im < nm; im++ )
      for ( int io = 0; io < no; io++ )
        {
          acc_t ac = 0;
          for ( int ic = 0; ic < nc; ic++ )
            for ( int ii = 0; ii < ni; ii++ )
              {
                size_t idx_ai = ii + ni * ( ic + nc * in );
                size_t idx_w = im + nm * ( ii + ni * ( ic + nc * io ) );
                ac += ai[ idx_ai ] * w[ idx_w ];
              }
          ao[ io + no * ( im + nm * in ) ] = ac;
        }


  // Initialize alternative (w2) array.
  //
# pragma omp parallel for
  for ( int im = 0; im < nm; im++ )
    for ( int io = 0; io < no; io++ )
      for ( int ic = 0; ic < nc; ic++ )
        for ( int ii = 0; ii < ni; ii++ )
          {
            size_t idx_w = im + nm * ( ii + ni * ( ic + nc * io ) );
            assert( idx_w < l.sz_w_elts );

            size_t idx_w2 = im + nm * ( ii + ni * ( ic + nc * io ) );

            assert( idx_w2 < l.sz_w_elts );
            l.w2_h[idx_w2] = w[idx_w];
          }
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  NPerf_init();
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("l2_global_load_bytes");
  NPerf_metric_collect("l2_write_transactions");
  NPerf_metric_collect("dram_read_bytes");
  NPerf_metric_collect("dram_write_bytes");

  constexpr int wp_sz = 32;

  struct App_Kernel_Info {
    App_Kernel_Info(Kernel_Info& k,const char *name, int i):
      k_ptr(k.func_ptr),name_base(name),shape_idx{i}{}
    GPU_Info_Func k_ptr;
    const char *name_base;
    const int shape_idx;
  };

  vector<App_Kernel_Info> kernels;

  #define EXAMINE_KERNEL(k,sidx) \
    { const int idx = kernels.size(); \
      kernels.emplace_back(info.GET_INFO((k)),#k,sidx); }

  #define SPECIFY_KERNEL(k,sidx) \
    EXAMINE_KERNEL((k<ls[sidx].nn,ls[sidx].nc,ls[sidx].ni>),sidx);

  #define SPECIALIZE_KERNEL(sidx) \
    SPECIFY_KERNEL(dnn_base,sidx); \
    SPECIFY_KERNEL(dnn_sol_a,sidx); \
    SPECIFY_KERNEL(dnn_sol_b,sidx);

  SPECIALIZE_KERNEL(0);
  SPECIALIZE_KERNEL(1);

  #undef SPECIALIZE_KERNEL

  const bool want_kernel_info = true;
  if ( want_kernel_info )
    {
      // Print information about kernel.
      //
      printf("\nCUDA Kernel Resource Usage:\n");

      for ( int i=0; i<info.num_kernels; i++ )
        {
          printf("For %s:\n", info.ki[i].name);
          printf("  %6zd shared, %zd const, %zd loc, %d regs; "
                 "%d max threads per block.\n",
                 info.ki[i].cfa.sharedSizeBytes,
                 info.ki[i].cfa.constSizeBytes,
                 info.ki[i].cfa.localSizeBytes,
                 info.ki[i].cfa.numRegs,
                 info.ki[i].cfa.maxThreadsPerBlock);
        }
    }


  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of warps per block.
  //
  const int wp_per_block_arg = argc < 3 ? 0 : atoi(argv[2]);
  const int wp_per_block_goal =
   wp_per_block_arg == 0 ? 32 : wp_per_block_arg;
  const int n_threads = num_blocks * wp_per_block_goal * wp_sz;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = wp_per_block_arg == 0;


  Layer layers[n_shapes];
  for ( int i=0; i<n_shapes; i++ )
    {
      Layer& l = layers[i];
      Layer_Shape& s = l.s;
      s = ls[i];
      layer_init(l);
      const size_t act_one_bytes =
        ( s.ni*s.nc + s.no*s.nm ) * sizeof(l.ai_h[0]);
      const size_t act_all_bytes = act_one_bytes * s.nn;
      printf("Layer shape %d: ni=no=%d.  nc=nm=%d. nn=%d.\n",
             i, s.ni, s.nc, s.nn);
      printf("  Number elts: activations %zd, weights %zd\n",
             l.sz_ai_elts + l.sz_ao_elts, l.sz_w_elts);
      printf("  Weights size: %zu kiB   L2 cache units: %.3f\n",
             l.sz_w_bytes >> 10,
             double(l.sz_w_bytes) / info.cuda_prop.l2CacheSize);
      printf("  Act size one batch  : %zu B   L2 cache units: %.3f\n",
             act_one_bytes,
             double(act_one_bytes) / info.cuda_prop.l2CacheSize);
      printf("  Act size all batches: %zu B   L2 cache units: %.3f\n",
             act_all_bytes,
             double(act_all_bytes) / info.cuda_prop.l2CacheSize);
    }

  if ( n_threads <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [WARPS_PER_BLOCK] "
             "[COL PER MP]\n",
             argv[0]);
      exit(1);
    }


  // The width, in characters, of the output to which we are printing.
  //
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    for ( auto &l: layers )
      {
        CE( hipMemcpy
            ( l.ai_d, l.ai_h, l.sz_ai_bytes, hipMemcpyHostToDevice ) );
        CE( hipMemcpy
            ( l.w_d, l.w_h, l.sz_w_bytes, hipMemcpyHostToDevice ) );
        CE( hipMemcpy
            ( l.w2_d, l.w2_h, l.sz_w_bytes, hipMemcpyHostToDevice ) );
      }

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d warps. \n",
           num_blocks, wp_per_block_goal);

    for ( auto& aki: kernels )
      {
        const char* kname = aki.name_base;
        const int sidx = aki.shape_idx;
        Layer& l = layers[sidx];
        const Layer_Shape s = l.s;

        printf("Kernel %s:\n", kname);
        pTable table(stdout);

        Kernel_Info* const ki = &info.get_info(aki.k_ptr);

        const int wp_limit = ki->cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary =
          min(wp_per_block_goal*wp_sz,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 1;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            if ( vary_warps && wp_cnt > 4 && wp_cnt & 0x3 ) continue;


            /// Compute Expected Computation and Communication
            //
            // Number of multiply/add operations. Ignore everything else.
            //
            const int64_t num_ops_fp = l.sz_w_elts * s.nn;
            // Load and store instructions.
            const int64_t num_ops_ls =
              s.nn * l.sz_w_elts
              + s.no * s.nm * l.sz_ai_elts + l.sz_ao_elts;

            //
            // Amount of data in and out of GPU chip --- if perfect.
            //
            const int64_t amt_data_bytes =
              l.sz_w_bytes + l.sz_ai_bytes + l.sz_ao_bytes;

            {
              // Zero the output array.
              //
              CE(hipMemset(l.ao_d,0,l.sz_ao_bytes));

              // Measure execution time starting "now", which is after data
              // set to GPU.
              //
              CE(hipEventRecord(gpu_start_ce,0));

              typedef void (*KPtr)(Layer);

              /// Launch Kernel
              //
              for ( NPerf_data_reset(); NPerf_need_run_get(); )
                KPtr(ki->func_ptr) <<< num_blocks, thd_per_block >>>(l);

              // Stop measuring execution time now, which is before is data
              // returned from GPU.
              //
              CE(hipEventRecord(gpu_stop_ce,0));
              CE(hipEventSynchronize(gpu_stop_ce));
              float cuda_time_ms = -1.1;
              CE( hipEventElapsedTime
                  (&cuda_time_ms,gpu_start_ce,gpu_stop_ce) );

              const double this_elapsed_time_s =
                NPerf_metrics_collection_get()
                ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

              const double thpt_compute_gflops =
                num_ops_fp / this_elapsed_time_s * 1e-9;
              const double thpt_data_gbps =
                amt_data_bytes / this_elapsed_time_s * 1e-9;

              // Number of load/store operations per second.
              const double chip_ls_ops = info.chip_sp_flops / 4;

              const double t_bound_fp = num_ops_fp / info.chip_sp_flops;
              const double t_bound_ls = num_ops_ls / chip_ls_ops;
              const double t_bound_insn = t_bound_fp + t_bound_ls;

              {
                const double comp_frac = t_bound_insn / this_elapsed_time_s;
                //  1e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;
                const double fp_frac = t_bound_fp / this_elapsed_time_s;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  ki->get_max_active_blocks_per_mp(thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute number of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                pTable_Row row(table);
                table.entry("nn","%2d",s.nn);
                table.entry("nc","%2d",s.nc);
                table.entry("ni","%2d",s.ni);
                table.entry("wp",num_wps);
                if ( num_blocks > num_mp )
                  table.entry("ac",act_wps);
                if ( NPerf_metrics_collection_get() )
                  {
                    const double transaction_sz_bytes = 32;
                    double dram_rd_bytes =
                      NPerf_metric_value_get("dram_read_bytes");
                    double dram_wr_bytes =
                      NPerf_metric_value_get("dram_write_bytes");

                    double l2_rd_bytes =
                      NPerf_metric_value_get("l2_global_load_bytes");
                    double l2_wr_bytes =
                      NPerf_metric_value_get("l2_write_transactions")
                      * transaction_sz_bytes;

                    table.entry
                      ("I/op","%4.1f",
                       NPerf_metric_value_get("inst_executed")
                       * 32.0 / num_ops_fp );
                    table.entry
                      ("DUse","%4.1f",
                       ( dram_rd_bytes + dram_wr_bytes ) / amt_data_bytes);
                    if ( false )
                    table.entry("DW","%4.1f", dram_wr_bytes / l.sz_ao_bytes );
                    table.entry
                      ("2Use","%5.1f",
                       ( l2_rd_bytes + l2_wr_bytes ) / amt_data_bytes);
                    if ( false )
                      table.entry("2W","%4.1f", l2_wr_bytes / l.sz_ao_bytes);
                  }
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                if ( false )
                  table.entry("GB/s","%4.0f", thpt_data_gbps);

                const size_t max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%zds",max_st_len);
                string util_hdr =
                  "=== Util: FP++  Insn-- Data**  ";
                if ( max_st_len > util_hdr.length() )
                  util_hdr += string(max_st_len - util_hdr.length(),'=');

                typedef struct { double f; char c; } Elt;
                vector<Elt> segments =
                  { { fp_frac, '+' }, { comp_frac, '-' }, { bw_frac, '*' } };

                sort( segments.begin(), segments.end(),
                      [](Elt& a, Elt& b){ return a.f < b.f; } );

                string bar;
                for ( Elt& e: segments )
                  if ( size_t p = e.f * max_st_len + 0.5; p > bar.length() )
                    bar += string( p - bar.length(), e.c );

                if ( bar.length() > max_st_len )
                  {
                    bar.resize(max_st_len);
                    bar[max_st_len-1] = '>';
                  }

                table.entry(util_hdr,fmt, bar, pTable::pT_Left);
              }

              // Copy output array from GPU to CPU.
              //
              CE( hipMemcpy
                  ( l.ao_h, l.ao_d, l.sz_ao_bytes, hipMemcpyDefault) );
              {
                int err_count = 0;
                Layer_Shape& s = l.s;

                const int no = s.no; // Number of output neurons.
                const int nm = s.nm; // Number of output channels.
                const int nn = s.nn; // Number of batches.

                for ( int in = 0; in < nn; in++ )
                  for ( int im = 0; im < nm; im++ )
                    for ( int io = 0; io < no; io++ )
                      {
                        size_t idx = io + no * ( im + nm * in );
                        if ( fabs( l.ao_check_h[idx] - l.ao_h[idx] ) > 1e-4 )
                          {
                            err_count++;
                            if ( err_count < 5 )
                              printf
                                ("Error at %d, %d, %d: "
                                 "%.7f != %.7f (correct)\n",
                                 in, im, io, l.ao_h[idx],
                                 l.ao_check_h[idx]);
                          }}
                if ( err_count )
                  printf("Total errors %d\n", err_count);
              }
            }
          }
      }
  }
}

