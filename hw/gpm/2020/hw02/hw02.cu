#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2020
 /// Homework 2
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2020/hw02.pdf
 //
 //  Note: The solution to Homework 2 is "paper" only. This file
 //  will not be collected.

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <ptable.h>

constexpr int Dj = 512; // Maximum

struct App
{
  int array_size;

  float *h_in;         // Host address space, data in.
  float *h_w;          // Host address space, weight array.
  float *h_out;        // Host address space, data out.

  float *d_in;         // Device address space, data in.
  float *d_w;          // Device address space, weight array.
  float *d_out;        // Device address space, data out.

  float w[Dj];         // Same address space as App object.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App dapp;


///
/// GPU Code (Kernels)
///

template <int n_wps, int Dj>
__global__ void
conv_wbuf()
{
  constexpr int wp_sz = 32;
  constexpr int block_dim = n_wps * wp_sz;
  const int n_threads = block_dim * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * block_dim;
  assert( block_dim == blockDim.x );

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}

template <int n_wps, int Dj>
__global__ void
conv_inbuf_block()
{
  constexpr int wp_sz = 32;
  constexpr int block_dim = n_wps * wp_sz;
  assert( block_dim == blockDim.x );
  assert( Dj <= block_dim );

  const int tx = threadIdx.x;  // Space-saving abbreviation.
  const int elt_per_block = ( dapp.array_size + gridDim.x - 1 ) / gridDim.x;
  const int bl_start = elt_per_block * blockIdx.x;
  const int bl_end = min( bl_start + elt_per_block, int(dapp.array_size) );

  __shared__ float in_buffer[block_dim*2];
  in_buffer[tx+block_dim] = dapp.d_in[bl_start+tx];

  for ( int hb=bl_start; hb<bl_end; hb += block_dim )
    {
      const int h = hb + tx;
      __syncthreads();
      in_buffer[tx] = in_buffer[tx+block_dim];
      in_buffer[tx+block_dim] = dapp.d_in[h+block_dim];
      __syncthreads();

      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += in_buffer[tx+j] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  constexpr int wp_sz = 32;

  struct App_Kernel_Info {
    App_Kernel_Info(Kernel_Info& k,const char *name, int n_wpsp, int djp):
      k_ptr(k.func_ptr),name_base(name),n_wps(n_wpsp),dj(djp){};
    GPU_Info_Func k_ptr;
    const char *name_base;
    int n_wps, dj;
  };

  map<string,vector<App_Kernel_Info>> kernels;
  map<int,int> w_sizesm;
  map<int,int> n_wpsm;

  #define EXAMINE_KERNEL(k,b,n) \
   kernels[#k].emplace_back(info.GET_INFO((k<b,n>)),#k,b,n); \
   w_sizesm[n]++;  n_wpsm[n]++;

  #define SPECIALIZE_KERNEL_2(b,n) \
    EXAMINE_KERNEL(conv_wbuf,b,n); \
    EXAMINE_KERNEL(conv_inbuf_block,b,n);

  #define SPECIALIZE_KERNEL(n) \
    SPECIALIZE_KERNEL_2(8,n) \
    SPECIALIZE_KERNEL_2(16,n) \
    SPECIALIZE_KERNEL_2(32,n)

  SPECIALIZE_KERNEL(4);
  SPECIALIZE_KERNEL(8);
  SPECIALIZE_KERNEL(16);
  SPECIALIZE_KERNEL(18);
  SPECIALIZE_KERNEL(24);
  SPECIALIZE_KERNEL(32);
  SPECIALIZE_KERNEL(64);
  SPECIALIZE_KERNEL(128);
  SPECIALIZE_KERNEL(256);
  SPECIALIZE_KERNEL(512);

  #undef SPECIALIZE_KERNEL


  vector<int> w_sizes;
  for ( auto& e: w_sizesm ) w_sizes.push_back(e.first);

  const bool want_kernel_info = false;
  if ( want_kernel_info )
    {
      // Print information about kernel.
      //
      printf("\nCUDA Kernel Resource Usage:\n");

      for ( int i=0; i<info.num_kernels; i++ )
        {
          printf("For %s:\n", info.ki[i].name);
          printf("  %6zd shared, %zd const, %zd loc, %d regs; "
                 "%d max threads per block.\n",
                 info.ki[i].cfa.sharedSizeBytes,
                 info.ki[i].cfa.constSizeBytes,
                 info.ki[i].cfa.localSizeBytes,
                 info.ki[i].cfa.numRegs,
                 info.ki[i].cfa.maxThreadsPerBlock);
        }
    }


  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of warps per block.
  //
  const int wp_per_block_arg = argc < 3 ? 32 : atoi(argv[2]);
  const int wp_per_block_goal =
   wp_per_block_arg == 0 ? 32 : wp_per_block_arg;
  const int n_threads = num_blocks * wp_per_block_goal * wp_sz;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = wp_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 16 << 20 : int( atof(argv[3]) * (1<<20) );
  const int64_t array_size_64 = app.array_size; // Need 64 bits in places.

  if ( n_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // The following are maximum values. The value for a particular run
  // depends on the number of weights, dj, rather than the maximum, Dj.
  //
  const size_t in_size_elts = array_size_64 + Dj;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t w_size_bytes = Dj * sizeof( app.h_w[0] );
  const size_t out_size_bytes = array_size_64 * sizeof( app.h_out[0] );
  const size_t overrun_size_bytes = num_blocks * 1024 * sizeof( app.h_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new float[in_size_elts];
  app.h_w = new float[Dj];
  app.h_out = new float[array_size_64];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_w,  w_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Array size: %zd elements, num weights %d\n", array_size_64, Dj);

  // Initialize input arrays.
  //
#pragma omp parallel for
  for ( size_t i=0; i<in_size_elts; i++ ) app.h_in[i] = drand48();
  for ( int i=0; i<Dj; i++ ) app.w[i] = app.h_w[i] = drand48();

  map<int,vector<float>> h_outs_check;
  for ( int dj: w_sizes ) h_outs_check[dj].resize(in_size_elts);

  // Compute correct answer.
  //
#pragma omp parallel for
  for ( int i=0; i<app.array_size; i++ )
    {
      float s = 0;
      int j = 0;
      for ( int dj: w_sizes )
        {
          for ( ; j<dj; j++ ) s += app.h_in[i+j] * app.h_w[j];
          h_outs_check[dj][i] = s;
        }
    }

  // The width, in characters, of the output to which we are printing.
  //
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_w, app.h_w, w_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d warps. \n",
           num_blocks, wp_per_block_goal);

    for ( auto& elt: kernels )
      {
        auto& klist = elt.second;
        const char* kname = klist[0].name_base;

        printf("Kernel %s:\n", kname);
        pTable table(stdout);

        for ( auto& aki: klist )
          {
            Kernel_Info* const ki = &info.get_info(aki.k_ptr);
            const int wp_per_block = aki.n_wps;
            const int dj = aki.dj;

            if ( !vary_warps && wp_per_block != wp_per_block_goal ) continue;

            /// Compute Expected Computation and Communication
            //
            // Number of multiply/add operations. Ignore everything else.
            //
            const int64_t num_ops_fp = dj * array_size_64;  // Multiply-adds.
            const int64_t num_ops_ls = 0; // Load and store instructions.

            //
            // Amount of data in and out of GPU chip --- if perfectly-written.
            //
            const int64_t amt_data_elts = 2 * array_size_64 + dj;
            const int64_t amt_data_bytes = sizeof(app.h_in[0]) * amt_data_elts;

            {
              const int thd_per_block = wp_per_block * wp_sz;

              // Zero the output array.
              //
              CE(hipMemset(app.d_out,0,out_size_bytes));

              // Measure execution time starting "now", which is after data
              // set to GPU.
              //
              CE(hipEventRecord(gpu_start_ce,0));

              typedef void (*KPtr)();

              /// Launch Kernel
              //
              KPtr(ki->func_ptr) <<< num_blocks, thd_per_block >>>();

              // Stop measuring execution time now, which is before is data
              // returned from GPU.
              //
              CE(hipEventRecord(gpu_stop_ce,0));
              CE(hipEventSynchronize(gpu_stop_ce));
              float cuda_time_ms = -1.1;
              CE( hipEventElapsedTime
                  (&cuda_time_ms,gpu_start_ce,gpu_stop_ce) );

              const double this_elapsed_time_s = cuda_time_ms * 0.001;

              const double thpt_compute_gflops =
                num_ops_fp / this_elapsed_time_s * 1e-9;
              const double thpt_data_gbps =
                amt_data_bytes / this_elapsed_time_s * 1e-9;

              // Number of load/store operations per second.
              const double chip_ls_ops = info.chip_sp_flops / 4;

              const double t_bound_fp = num_ops_fp / info.chip_sp_flops;
              const double t_bound_ls = num_ops_ls / chip_ls_ops;
              const double t_bound_insn = t_bound_fp + t_bound_ls;

              {
                const double comp_frac = t_bound_insn / this_elapsed_time_s;
                  //  1e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  ki->get_max_active_blocks_per_mp(thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute number of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                pTable_Row row(table);
                table.entry("dj","%3d",dj);
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);
                string util_hdr =
                  "--- Utilization: ++Compute++   **Data**  ";
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                const bool bw_more = bw_frac > comp_frac;
                const char* sym = bw_more ? "+*" : "*+";
                const double frac_min =  bw_more ? comp_frac : bw_frac;
                const double frac_max = !bw_more ? comp_frac : bw_frac;
                const size_t len_min = frac_min * max_st_len + 0.5;
                const size_t len_max = frac_max * max_st_len + 0.5;
                string bar = string( len_min, sym[0] ) +
                  string( len_max - len_min, sym[1] );

                table.entry(util_hdr,fmt, bar, pTable::pT_Left);
              }

              // Copy output array from GPU to CPU.
              //
              CE( hipMemcpy
                  ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDefault) );
              int err_count = 0;
              for ( int i=0; i<app.array_size; i++ )
                {
                  if ( fabs( h_outs_check[dj][i] - app.h_out[i] ) > 1e-4 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d: %.7f != %.7f (correct)\n",
                               i, app.h_out[i], h_outs_check[dj][i] );
                    }
                }
              if ( err_count )
                printf("Total errors %d\n", err_count);
            }
          }
      }

  }

}
