#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2020
 /// Homework 1 -- SOLUTION
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2020/hw01.pdf
 //
 //  Modify this file only.

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <ptable.h>

constexpr int Dj = 32;

struct App
{
  int array_size;

  float *h_in;         // Host address space, data in.
  float *h_w;          // Host address space, weight array.
  float *h_out;        // Host address space, data out.
  float *h_out_check;  // Compute correct answer on CPU, to check GPU.

  float *d_in;         // Device address space, data in.
  float *d_w;          // Device address space, weight array.
  float *d_out;        // Device address space, data out.

  float w[Dj];         // Same address space as App object.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App dapp;


///
/// GPU Code (Kernels)
///

__global__ void
conv_simple()
{
  // Each thread operates on its own contiguous section of the array.

  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int elt_per_thread = ( dapp.array_size + n_threads - 1 ) / n_threads;
  const int start = elt_per_thread * tid;  // Bad: Non-consecutive access.
  const int stop = min( start + elt_per_thread, dapp.array_size );

  for ( int h=start; h<stop; h++ )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.d_w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_efficient()
{
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.d_w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_wbuf()
{
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;


  // Copy dapp.d_w into an array (shared or local address space)
  // and use that array inside the loop.
  //
  // DO NOT use dapp.w[j] in this routine.

  /// SOLUTION -- Problem 1
  //
  //  Declare a local array and fill it with the weights (dapp.d_w).
  //
  float w[Dj];
  for ( int j=0; j<Dj; j++ ) w[j] = dapp.d_w[j];

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      // SOLUTION - Use w instead of dapp.d_w.
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * w[j];
      dapp.d_out[h] = s;
    }

}

__global__ void
conv_inbuf_wp_a()
{
  constexpr int wp_lg = 5;
  constexpr int wp_sz = 1 << wp_lg;
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_wps = n_threads >> wp_lg;
  assert( n_wps << wp_lg == n_threads );

  const int [[gnu::unused]] wp_idx = tid >> wp_lg;
  const int [[gnu::unused]] wp_ldx = threadIdx.x >> wp_lg;
  const int [[gnu::unused]] lane = threadIdx.x % wp_sz;

  /// SOLUTION -- Problem 2 -- Using a barrier (syncwarp).

  // First, find number of array elements per warp, rounding up.
  //
  const int elt_per_wp_raw = ( dapp.array_size + n_wps - 1 ) / n_wps;

  // Next, round up to a multiple of warp size. This will avoid overlap.
  //
  const int elt_per_wp = ( elt_per_wp_raw + wp_sz - 1 ) & ~( wp_sz - 1 );
  //
  // Note: wp_sz - 1 is 31 or 0x1f.
  // The ~ is bitwise NOT, so  ~0x1f  is  0xffffffe0.
  // The & is bitwise AND, so   x & ~0x1f  
  //   sets the least significant 5 bits of x to zero,
  //   making it a multiple of 32.

  // Find the starting and ending element for this warp.
  //
  const int wp_start = elt_per_wp * wp_idx;
  const int wp_end = min( wp_start + elt_per_wp, dapp.array_size );

  // This solution won't work if Dj > wp_sz.
  assert( wp_sz >= Dj );

  // Declare an array to hold input elements.
  // Each warp uses 64 elements of this array.
  //
  __shared__ float in_buffer[2048];

  // Declare a pointer to this warp's part of the shared array.
  //
  float* const wp_buf = &in_buffer[ wp_ldx * 2 * wp_sz ];

  // Load the first set of values.
  //
  wp_buf[ wp_sz + lane ] = dapp.d_in[ wp_start + lane ];

  for ( int hw=wp_start; hw<wp_end; hw += wp_sz )
    {
      const int h = hw + lane;

      // Move data loaded in the previous iteration ..
      //
      wp_buf[ lane ] = wp_buf[ wp_sz + lane ];
      //
      // .. and load the next set of values.
      //
      wp_buf[ wp_sz + lane ] = dapp.d_in[ wp_sz + h ];

      // Make sure that the assignment to wp_buf above is done before
      // the code below.
      //
      __syncwarp();
      //
      // The __syncwarp call is being used to tell the compiler to not
      // move the  "wp_buf[ wp_sz + lane ] = dapp.d_in[ wp_sz + h ];"
      // line past the __syncwarp.

      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += wp_buf[ lane + j ] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_inbuf_wp_b()
{
  constexpr int wp_lg = 5;
  constexpr int wp_sz = 1 << wp_lg;
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_wps = n_threads >> wp_lg;
  assert( n_wps << wp_lg == n_threads );

  const int [[gnu::unused]] wp_idx = tid >> wp_lg;
  const int [[gnu::unused]] wp_ldx = threadIdx.x >> wp_lg;
  const int [[gnu::unused]] lane = threadIdx.x % wp_sz;

  /// SOLUTION -- Problem 2 -- Using a volatile declaration.

  // First, find number of array elements per warp, rounding up.
  //
  const int elt_per_wp_raw = ( dapp.array_size + n_wps - 1 ) / n_wps;

  // Next, round up to a multiple of warp size. This will avoid overlap.
  //
  const int elt_per_wp = ( elt_per_wp_raw + wp_sz - 1 ) & ~( wp_sz - 1 );
  //
  // Note: wp_sz - 1 is 31 or 0x1f.
  // The ~ is bitwise NOT, so  ~0x1f  is  0xffffffe0.
  // The & is bitwise AND, so   x & ~0x1f  
  //   sets the least significant 5 bits of x to zero,
  //   making it a multiple of 32.

  // Find the starting and ending element for this warp.
  //
  const int wp_start = elt_per_wp * wp_idx;
  const int wp_end = min( wp_start + elt_per_wp, dapp.array_size );

  // This solution won't work if Dj > wp_sz.
  assert( wp_sz >= Dj );

  // Declare an array to hold input elements.
  // Each warp uses 64 elements of this array.
  //
  __shared__ volatile float in_buffer[2048];
  //
  // Note: the volatile attribute tells the compiler that values in
  // the array (in this case) can be changed by other threads (or in
  // other cases, by hardware). This forces the compiler to load a
  // value each time it is needed, and to store values in to the
  // variable at the place in the code an assignment like
  // in_buffer[x]=y appears.
  //
  // For example, suppose in_buffer[x] appears in two places in the
  // code and x is the same in both places. Without the volatile
  // qualifier the compile would load in_buffer[x] into a register,
  // and use that register at each place in the code in_buffer[x]
  // appears. But, if in_buffer is declared with the volatile
  // qualifier the compiler will load in_buffer[x] from shared memory
  // twice, one each place it appears in the code.

  // Declare a pointer to this warp's part of the shared array.
  //
  volatile float* const wp_buf = &in_buffer[ wp_ldx * 2 * wp_sz ];

  // Load the first set of values.
  //
  wp_buf[ wp_sz + lane ] = dapp.d_in[ wp_start + lane ];

  for ( int hw=wp_start; hw<wp_end; hw += wp_sz )
    {
      const int h = hw + lane;

      // Move data loaded in the previous iteration ..
      //
      wp_buf[ lane ] = wp_buf[ wp_sz + lane ];
      //
      // .. and load the next set of values.
      //
      wp_buf[ wp_sz + lane ] = dapp.d_in[ wp_sz + h ];

      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += wp_buf[ lane + j ] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_inbuf_class()
{
  constexpr int wp_lg = 5;
  constexpr int wp_sz = 1 << wp_lg;
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_wps = n_threads >> wp_lg;
  assert( n_wps << wp_lg == n_threads );

  const int [[gnu::unused]] wp_idx = tid >> wp_lg;
  const int [[gnu::unused]] wp_ldx = threadIdx.x >> wp_lg;
  const int [[gnu::unused]] lane = threadIdx.x % wp_sz;

  /// SOLUTION -- Written during class on 13 March.

  // First, find number of array elements per warp, rounding up.
  //
  const int elt_per_wp_raw = ( dapp.array_size + n_wps - 1 ) / n_wps;

  // Next, round up to a multiple of warp size. This will avoid overlap.
  //
  const int elt_per_wp = ( elt_per_wp_raw + wp_sz - 1 ) & ~( wp_sz - 1 );

  // Find the starting and ending element for this warp.
  //
  const int wp_start = elt_per_wp * wp_idx;
  const int wp_end = min( wp_start + elt_per_wp, dapp.array_size );

  const int mask = 63;
  __shared__ float in_buffer[2048];

  float* const wp_buf = &in_buffer[ wp_ldx * 2 * wp_sz ];

  int idx1 = 0;

  wp_buf[ idx1 + lane ] = dapp.d_in[ wp_start + lane ];

  for ( int h_wp=wp_start; h_wp<wp_end; h_wp += wp_sz )
    {
      const int h = h_wp + lane;
      wp_buf[ ( idx1 + wp_sz + lane ) & mask ] = dapp.d_in[ wp_sz + h ];

      __syncwarp();

      float s = 0;
      for ( int j=0; j<Dj; j++ )
        s += wp_buf[ ( idx1 + lane + j ) & mask ] * dapp.w[j];
      dapp.d_out[h] = s;

      idx1 += wp_sz;
    }
}

__global__ void
conv_inbuf_block()
{
  /// Alternative Approach -- Divide array between blocks.

  const int tx = threadIdx.x;  // Space-saving abbreviation.
  const int elt_per_block = ( dapp.array_size + gridDim.x - 1 ) / gridDim.x;
  const int bl_start = elt_per_block * blockIdx.x;
  const int bl_end = min( bl_start + elt_per_block, dapp.array_size );

  __shared__ float in_buffer[2048];
  in_buffer[tx+blockDim.x] = dapp.d_in[bl_start+tx];

  for ( int hb=bl_start; hb<bl_end; hb += blockDim.x )
    {
      const int h = hb + tx;
      __syncthreads();
      in_buffer[tx] = in_buffer[tx+blockDim.x];
      in_buffer[tx+blockDim.x] = dapp.d_in[h+blockDim.x];
      __syncthreads();

      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += in_buffer[tx+j] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  const bool show_all = false;
  if ( show_all )
    {
      info.GET_INFO(conv_simple);
      info.GET_INFO(conv_efficient);
    }
  info.GET_INFO(conv_wbuf);
  info.GET_INFO(conv_inbuf_wp_a);
  info.GET_INFO(conv_inbuf_wp_b);
  info.GET_INFO(conv_inbuf_block);
  info.GET_INFO(conv_inbuf_class);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int n_threads = num_blocks * thd_per_block_goal;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 16 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( n_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.array_size + Dj;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int w_size_bytes = Dj * sizeof( app.h_w[0] );
  const int out_size_bytes = app.array_size * sizeof( app.h_out[0] );
  const int overrun_size_bytes = num_blocks * 1024 * sizeof( app.h_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new float[in_size_elts];
  app.h_w = new float[Dj];
  app.h_out = new float[app.array_size];
  app.h_out_check = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_w,  w_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Array size: %d elements, num weights %d\n", app.array_size,Dj);

  // Initialize input arrays.
  //
#pragma omp parallel for
  for ( int i=0; i<in_size_elts; i++ ) app.h_in[i] = drand48();
  for ( int i=0; i<Dj; i++ ) app.w[i] = app.h_w[i] = drand48();

  // Compute correct answer.
  //
#pragma omp parallel for
  for ( int i=0; i<app.array_size; i++ )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += app.h_in[i+j] * app.h_w[j];
      app.h_out_check[i] = s;
    }

  /// Compute Expected Computation and Communication
  //
  // Number of multiply/add operations. Ignore everything else.
  //
  const int64_t num_ops = Dj * app.array_size;  // Multiply-adds.
  //
  // Amount of data in and out of GPU chip --- if code well-written.
  //
  const int amt_data_bytes = in_size_bytes + w_size_bytes + out_size_bytes;

  // The width, in characters, of the output to which we are printing.
  //
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_w, app.h_w, w_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 1;

        pTable table(stdout);

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            if ( vary_warps && wp_cnt > 4 && wp_cnt & 0x3 ) continue;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            typedef void (*KPtr)();

            /// Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr) <<< num_blocks, thd_per_block >>>();

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                const bool graph_bw = true;
                const double frac = graph_bw ? bw_frac : comp_frac;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                pTable_Row row(table);
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);
                string util_hdr =
                  graph_bw ? "Data BW Util" : "FP Utilization";
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDefault) );
            int err_count = 0;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( fabs( app.h_out_check[i] - app.h_out[i] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d: %.7f != %.7f (correct)\n",
                               i, app.h_out[i], app.h_out_check[i] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
