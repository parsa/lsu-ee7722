#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2020
 /// Homework 1
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2020/hw01.pdf
 //
 //  Modify this file only.

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <ptable.h>

constexpr int Dj = 32;

struct App
{
  int array_size;

  float *h_in;         // Host address space, data in.
  float *h_w;          // Host address space, weight array.
  float *h_out;        // Host address space, data out.
  float *h_out_check;  // Compute correct answer on CPU, to check GPU.

  float *d_in;         // Device address space, data in.
  float *d_w;          // Device address space, weight array.
  float *d_out;        // Device address space, data out.

  float w[Dj];         // Same address space as App object.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App dapp;


///
/// GPU Code (Kernels)
///

__global__ void
conv_simple()
{
  // Each thread operates on its own contiguous section of the array.

  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int elt_per_thread = ( dapp.array_size + n_threads - 1 ) / n_threads;
  const int start = elt_per_thread * tid;  // Bad: Non-consecutive access.
  const int stop = min( start + elt_per_thread, dapp.array_size );

  for ( int h=start; h<stop; h++ )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.d_w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_efficient()
{
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.d_w[j];
      dapp.d_out[h] = s;
    }
}

__global__ void
conv_wbuf()
{
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;


  // Copy dapp.d_w into an array (shared or local address space)
  // and use that array inside the loop.
  //
  // DO NOT use dapp.w[j] in this routine.

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.d_w[j];
      dapp.d_out[h] = s;
    }

}

__global__ void
conv_inbuf_a()
{
  constexpr int wp_lg = 5;
  constexpr int wp_sz = 1 << wp_lg;
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_wps = n_threads >> wp_lg;
  assert( n_wps << wp_lg == n_threads );

  const int [[gnu::unused]] wp_idx = tid >> wp_lg;
  const int [[gnu::unused]] wp_ldx = threadIdx.x >> wp_lg;
  const int [[gnu::unused]] lane = threadIdx.x % wp_sz;

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.w[j];
      dapp.d_out[h] = s;
    }

}

__global__ void
conv_inbuf_b()
{
  constexpr int wp_lg = 5;
  constexpr int wp_sz = 1 << wp_lg;
  const int n_threads = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_wps = n_threads >> wp_lg;
  assert( n_wps << wp_lg == n_threads );

  const int [[gnu::unused]] wp_idx = tid >> wp_lg;
  const int [[gnu::unused]] wp_ldx = threadIdx.x >> wp_lg;
  const int [[gnu::unused]] lane = threadIdx.x % wp_sz;

  for ( int h=tid; h<dapp.array_size; h += n_threads )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += dapp.d_in[h+j] * dapp.w[j];
      dapp.d_out[h] = s;
    }
}





GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(conv_simple);
  info.GET_INFO(conv_efficient);
  info.GET_INFO(conv_wbuf);
  info.GET_INFO(conv_inbuf_a);
  info.GET_INFO(conv_inbuf_b);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int n_threads = num_blocks * thd_per_block_goal;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 16 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( n_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.array_size + Dj;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int w_size_bytes = Dj * sizeof( app.h_w[0] );
  const int out_size_bytes = app.array_size * sizeof( app.h_out[0] );
  const int overrun_size_bytes = num_blocks * 1024 * sizeof( app.h_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new float[in_size_elts];
  app.h_w = new float[Dj];
  app.h_out = new float[app.array_size];
  app.h_out_check = new float[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_w,  w_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Array size: %d elements, num weights %d\n", app.array_size,Dj);

  // Initialize input arrays.
  //
#pragma omp parallel for
  for ( int i=0; i<in_size_elts; i++ ) app.h_in[i] = drand48();
  for ( int i=0; i<Dj; i++ ) app.w[i] = app.h_w[i] = drand48();

  // Compute correct answer.
  //
#pragma omp parallel for
  for ( int i=0; i<app.array_size; i++ )
    {
      float s = 0;
      for ( int j=0; j<Dj; j++ ) s += app.h_in[i+j] * app.h_w[j];
      app.h_out_check[i] = s;
    }

  /// Compute Expected Computation and Communication
  //
  // Number of multiply/add operations. Ignore everything else.
  //
  const int64_t num_ops = Dj * app.array_size;  // Multiply-adds.
  //
  // Amount of data in and out of GPU chip --- if code well-written.
  //
  const int amt_data_bytes = in_size_bytes + w_size_bytes + out_size_bytes;

  // The width, in characters, of the output to which we are printing.
  //
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_w, app.h_w, w_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 1;

        pTable table(stdout);

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            if ( vary_warps && wp_cnt > 4 && wp_cnt & 0x3 ) continue;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            typedef void (*KPtr)();

            /// Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr) <<< num_blocks, thd_per_block >>>();

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                const bool graph_bw = true;
                const double frac = graph_bw ? bw_frac : comp_frac;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                pTable_Row row(table);
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);
                string util_hdr =
                  graph_bw ? "Data BW Util" : "FP Utilization";
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDefault) );
            int err_count = 0;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( fabs( app.h_out_check[i] - app.h_out[i] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d: %.7f != %.7f (correct)\n",
                               i, app.h_out[i], app.h_out_check[i] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
