#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2020
 /// Homework 3 -- Partial SOLUTION  (Problem 3 solution omitted.)
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2020/hw03.pdf
 //

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <ptable.h>

inline __device__ int
fl1(u_int32_t n) { return 8 * sizeof(n) - __clz(n);}

template<class T> __device__
T floor2(T x) { return x <= 0 ? 0 : T(1) << fl1(x) - 1; }
template<class T> __device__
T ceil2(T x) { return x <= 0 ? 0 : T(1) << fl1(x-1); }


constexpr int wc_max = 16;
constexpr int wr_max = 16;
constexpr int w_elts_max = wc_max * wr_max;

constexpr bool use_ro = false;


struct App
{
  int in_nr, in_nc;   // Input:  Number rows, number columns.
  int out_nr, out_nc; // Output: Number rows, number columns.
  int w_nr, w_nc;     // Weights:Number rows, number columns.

  float *h_in;         // Host address space, data in.
  float *h_w;          // Host address space, weight array.
  float *h_out;        // Host address space, data out.

  float *d_in;         // Device address space, data in.
  float *d_w;          // Device address space, weight array.
  float *d_out;        // Device address space, data out.

  float w[w_elts_max]; // Constant address space, weights.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App dapp;


///
/// GPU Code (Kernels)
///

template <int w_nr, int w_nc>
__global__ void
conv_wbuf()
{
  const int block_dim = blockDim.x;
  const int n_threads = block_dim * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * block_dim;
  const int n_elt = dapp.out_nr * dapp.out_nc;

  assert( w_nr == dapp.w_nr );
  assert( w_nc == dapp.w_nc );

  for ( int h=tid; h<n_elt; h += n_threads )
    {
      const int ro = h / dapp.out_nc;
      const int co = h % dapp.out_nc;

      float s = 0;

#pragma unroll
      for ( int rw=0; rw<w_nr; rw++ )
#pragma unroll
        for ( int cw=0; cw<w_nc; cw++ )
          {
            const int ri = ro + rw;
            const int ci = co + cw;
            const int iidx = ri * dapp.in_nc + ci;
            const int widx = rw * w_nc + cw;
            const float din = dapp.d_in[iidx];
            s += din * dapp.w[widx];
          }
      dapp.d_out[h] = s;
    }
}

template <int w_nr, int w_nc>
__global__ void
conv_prob1()
{
  const int block_dim = blockDim.x;
  const int tid = threadIdx.x + blockIdx.x * block_dim;

  // Make sure that template parameters match actual weight array size.
  assert( w_nr == dapp.w_nr && w_nc == dapp.w_nc );

  /// SOLUTION -- Problem 1.
  //
  //  Assign a group of columns to each block. Choose number of columns
  //  to largest amount for which data fits in cache.

  // Number of floats that will fit in the L1 or texture cache.
  const int cache_elts = use_ro ? 4 << 10 : 16 << 10;

  // Find the maximum number of columns that a block can handle without
  // exceeding the assumed cache capacity, cache_elts.
  //
  const float qa = w_nr;  // Quadratic formula coefficient.
  const float qb = block_dim + w_nr * w_nc - cache_elts;
  const float qc = w_nc;
  const float bwid_raw = ( -qb + sqrt( qb*qb - 4 * qa * qc ) ) / ( 2 * qa );
  assert( bwid_raw >= 0 );

  // Round number of columns down to a multiple of 8 (for request
  // efficiency) and clamp between 8 and the block size.
  //
  const int block_wid =
    min( block_dim, max(8, int(bwid_raw) & ~0x7 ) );
  const int block_ht = block_dim / block_wid;

  // Number of columns handled by all blocks.
  const int grid_calc_wid = block_wid * gridDim.x;

  // The initial row and column handled by this thread.
  const int thread_c0 = blockIdx.x * block_wid + tid % block_wid;
  const int thread_r0 = threadIdx.x / block_wid;

  for ( int cc = 0; cc < dapp.out_nc; cc += grid_calc_wid )
    {
      const int co = cc + thread_c0;
      if ( co >= dapp.out_nc ) break;

      for ( int rr = 0; rr < dapp.out_nr; rr += block_ht )
        {
          const int ro = rr + thread_r0;
          if ( ro >= dapp.out_nr ) break;
          const int oidx = ro * dapp.out_nc + co;

          float s = 0;

#pragma unroll
          for ( int rw=0; rw<w_nr; rw++ )
#pragma unroll
            for ( int cw=0; cw<w_nc; cw++ )
              {
                const int ri = ro + rw;
                const int ci = co + cw;
                const int iidx = ri * dapp.in_nc + ci;
                const int widx = rw * w_nc + cw;
                const float din = dapp.d_in[iidx];
                s += din * dapp.w[widx];
              }
          dapp.d_out[oidx] = s;
        }
    }
}

template <int w_nr, int w_nc>
__global__ void
conv_prob2_inefficient()
{
  constexpr int n_per_thd = 8;
  const int block_dim = blockDim.x;
  const int n_threads = block_dim * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * block_dim;

  // Inefficient solution to Problem 2.
  // To be used for a final exam question.

  const int h_0 = tid * n_per_thd;
  const int n_elt = dapp.out_nr * dapp.out_nc;

  for ( int h=h_0; h<n_elt; h += n_threads*n_per_thd )
    {
      const int ro = h / dapp.out_nc;
      const int co = h % dapp.out_nc;

      float s[n_per_thd];
      for ( auto& e: s ) e = 0;

#pragma unroll
      for ( int rw=0; rw<w_nr; rw++ )
#pragma unroll
        for ( int cw=0; cw<w_nc+n_per_thd-1; cw++ )
          {
            const int ri = ro + rw,  ci = co + cw;
            const int iidx = ri * dapp.in_nc + ci;
            const float din = dapp.d_in[iidx];

            // Compute output for each of n_per_thd rows.
            //
            for ( int k=0; k<n_per_thd; k++ )
              {
                // Find column number of weight array to use for output k.
                const int cwk = cw - k;
                const int widxk = rw * w_nc + cwk;
                if (  cwk >= 0  &&  cwk < w_nc  )
                  s[k] += din * dapp.w[ widxk ];
              }
          }

      // Write output for n_per_thd rows.
      for ( int k=0; k<n_per_thd; k++ ) dapp.d_out[h+k] = s[k];
    }
}

template <int w_nr, int w_nc>
__global__ void
conv_probfutil()
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

  /// Another possible final exam question.

  for ( int co = tid;  co < dapp.out_nc;  co += n_threads )
    {
      for ( int ro = 0;  ro < dapp.out_nr;  ro++ )
        {
          const int oidx = ro * dapp.out_nc + co;
          float s = 0;

#pragma unroll
          for ( int rw=0; rw<w_nr; rw++ )
            for ( int cw=0; cw<w_nc; cw++ )
              {
                const int ri = ro + rw;
                const int ci = co + cw;
                const int iidx = ri * dapp.in_nc + ci;
                const int widx = rw * w_nc + cw;
                s += dapp.d_in[iidx] * dapp.w[widx];
              }
          dapp.d_out[oidx] = s;

        }
    }
}

template <int w_nr, int w_nc, int n_per_thd>
__device__ void
conv_prob2t()
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

  /// SOLUTION -- Problem 2.

  //  Each thread operates on n_per_thd outputs at a time.
  //  The n_per_thd outputs are in consecutive rows and the same column.
  //  Each thread operates on an entire column.

  for ( int co = tid;  co < dapp.out_nc;  co += n_threads )
    for ( int ro = 0;  ro < dapp.out_nr;  ro += n_per_thd )
      {
        const int oidx = ro * dapp.out_nc + co;

        // Initialize sum for the n_per_thd outputs.
        //
        float s[n_per_thd];
        for ( auto& e: s ) e = 0;

        // The unroll pragmata are important, if the loops aren't unrolled
        // the values of widx-k*w_nc, used to index dapp.s, will not
        // be compile-time constants and so whatever instruction throughput
        // benefit that was gained by performing n_per_thd FMA's per global
        // load will be 

#pragma unroll
        for ( int rw=0; rw<w_nr+n_per_thd-1; rw++ )
#pragma unroll
          for ( int cw=0; cw<w_nc; cw++ )
            {
              const int ri = ro + rw,  ci = co + cw;
              const int iidx = ri * dapp.in_nc + ci;
              const float din = dapp.d_in[iidx];

              // Compute output for each of n_per_thd rows.
              //
              for ( int k=0; k<n_per_thd; k++ )
                {
                  // Find row number of weight array to use for output k.
                  const int rwk = rw - k;
                  const int widxk = rwk * w_nc + cw;
                  if (  rwk >= 0  &&  rwk < w_nr )
                    s[k] += din * dapp.w[ widxk ];
                }
            }

        // Write output for n_per_thd rows.
        for ( int k=0; k<n_per_thd; k++ )
          dapp.d_out[ oidx + k * dapp.out_nc ] = s[k];
      }
}


// Instantiate Problem 2 modules at different n_per_thd values.

template <int w_nr, int w_nc>
__global__ void conv_prob21() {conv_prob2t<w_nr,w_nc,1>();}
template <int w_nr, int w_nc>
__global__ void conv_prob22() {conv_prob2t<w_nr,w_nc,2>();}
template <int w_nr, int w_nc>
__global__ void conv_prob24() {conv_prob2t<w_nr,w_nc,4>();}
template <int w_nr, int w_nc>
__global__ void conv_prob28() {conv_prob2t<w_nr,w_nc,8>();}
template <int w_nr, int w_nc>
__global__ void conv_prob216() {conv_prob2t<w_nr,w_nc,16>();}




GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  return info;
}


///
/// Main Routine
///


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  constexpr int wp_sz = 32;

  struct App_Kernel_Info {
    App_Kernel_Info(Kernel_Info& k,const char *name, int w_nrp, int w_ncp):
      k_ptr(k.func_ptr),name_base(name),rc{w_nrp,w_ncp}{}
    GPU_Info_Func k_ptr;
    const char *name_base;
    pair<int,int> rc;
  };

  map<string,vector<int>> kernel_map;
  map<pair<int,int>,int> rc_uses;
  vector<pair<int,int>> rc_list;
  vector<App_Kernel_Info> kernels;

  #define EXAMINE_KERNEL(k,r,c) \
    { const int idx = kernels.size(); \
      kernels.emplace_back(info.GET_INFO((k<r,c>)),#k,r,c);   \
      kernel_map[#k].push_back(idx); \
      if ( !rc_uses[make_pair(r,c)]++ ) rc_list.emplace_back(r,c); }


  #define SPECIALIZE_KERNEL(r,c) \
    EXAMINE_KERNEL(conv_wbuf,r,c); \
    EXAMINE_KERNEL(conv_prob1,r,c); \
    EXAMINE_KERNEL(conv_prob2_inefficient,r,c); \
    EXAMINE_KERNEL(conv_prob21,r,c); \
    EXAMINE_KERNEL(conv_prob22,r,c); \
    EXAMINE_KERNEL(conv_prob24,r,c);\
    EXAMINE_KERNEL(conv_prob28,r,c);


  SPECIALIZE_KERNEL(8,8);
  SPECIALIZE_KERNEL(16,16);

  #undef SPECIALIZE_KERNEL

  const bool want_kernel_info = false;
  if ( want_kernel_info )
    {
      // Print information about kernel.
      //
      printf("\nCUDA Kernel Resource Usage:\n");

      for ( int i=0; i<info.num_kernels; i++ )
        {
          printf("For %s:\n", info.ki[i].name);
          printf("  %6zd shared, %zd const, %zd loc, %d regs; "
                 "%d max threads per block.\n",
                 info.ki[i].cfa.sharedSizeBytes,
                 info.ki[i].cfa.constSizeBytes,
                 info.ki[i].cfa.localSizeBytes,
                 info.ki[i].cfa.numRegs,
                 info.ki[i].cfa.maxThreadsPerBlock);
        }
    }


  // Get number of multiprocessors. (A.k.a. streaming multiprocessors or SMs)
  //
  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of warps per block.
  //
  const int wp_per_block_arg = argc < 3 ? 32 : atoi(argv[2]);
  const int wp_per_block_goal =
   wp_per_block_arg == 0 ? 32 : wp_per_block_arg;
  const int n_threads = num_blocks * wp_per_block_goal * wp_sz;

  // If true, run kernels at multiple block sizes.
  //
  const bool vary_warps = wp_per_block_arg == 0;

  // Examine argument 3, number of array columns per MP.
  //
  const double col_per_mp = argc < 4 ? 1024 : atof(argv[3]);

  // Want array size to be larger than L2 cache.
  //
  const size_t target_array_sz_bytes = 1 << 25;
  const size_t target_array_sz_elts =
    target_array_sz_bytes / sizeof(app.h_in[0]);

  app.out_nc = num_mp * col_per_mp;
  app.out_nr = max( 32ul, target_array_sz_elts / app.out_nc );

  // Maximum values.
  app.w_nr = wr_max;
  app.w_nc = wc_max;
  app.in_nr = app.out_nr + app.w_nr;
  app.in_nc = app.out_nc + app.w_nc;

  if ( n_threads <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[COL PER MP]\n",
             argv[0]);
      exit(1);
    }

  const int n_per_thd_max = 32;
  const size_t in_msize_elts = app.in_nr * app.in_nc;
  const size_t in_msize_bytes = in_msize_elts * sizeof( app.h_in[0] );
  const size_t w_msize_elts = app.w_nr * app.w_nc;
  const size_t w_msize_bytes = w_msize_elts * sizeof( app.h_w[0] );
  const size_t out_size_elts = app.out_nr * app.out_nc;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );
  const size_t overrun_size_bytes =
    app.out_nc * n_per_thd_max * sizeof( app.h_in[0] ) +
    num_blocks * 1024 * sizeof( app.h_in[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new float[in_msize_elts];
  app.h_w = new float[w_msize_elts];
  app.h_out = new float[out_size_elts];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_msize_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_w,  w_msize_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Out array size: %d * %d elements, num weights max %d * %d\n",
         app.out_nr, app.out_nc, app.w_nr, app.w_nc );

  // Initialize input arrays.
  //
#pragma omp parallel for
  for ( size_t i=0; i<in_msize_elts; i++ ) app.h_in[i] = drand48();
  for ( size_t i=0; i<w_msize_elts; i++ ) app.w[i] = app.h_w[i] = drand48();

  map<pair<int,int>,vector<float>> h_outs_check;
  for ( auto rc: rc_list ) h_outs_check[rc].resize(out_size_elts);

  // Compute correct answer.
  //
#pragma omp parallel for
  for ( int ro=0; ro<app.out_nr; ro++ )
    for ( int co=0; co<app.out_nc; co++ )
      {
        const int oidx = ro * app.out_nc + co;
        for ( auto rc: rc_list )
          {
            const int w_nr = rc.first, w_nc = rc.second;
            const int in_nc = app.out_nc + w_nc;
            float s = 0;
            for ( int rw=0; rw<w_nr; rw++ )
              for ( int cw=0; cw<w_nc; cw++ )
                {
                  const int widx = rw * w_nc + cw;
                  const int iidx = (ro+rw) * in_nc + (co+cw);
                  s += app.h_in[iidx] * app.h_w[widx];
                }
            h_outs_check[rc][oidx] = s;
          }
      }

  // The width, in characters, of the output to which we are printing.
  //
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_msize_bytes, hipMemcpyHostToDevice ) );
    CE( hipMemcpy
        ( app.d_w, app.h_w, w_msize_bytes, hipMemcpyHostToDevice ) );


    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d warps. \n",
           num_blocks, wp_per_block_goal);

    for ( auto& aki: kernels )
      {
        const char* kname = aki.name_base;

        printf("Kernel %s:\n", kname);
        pTable table(stdout);

        Kernel_Info* const ki = &info.get_info(aki.k_ptr);
        auto rc = aki.rc;
        const int wr = rc.first, wc = rc.second;

        app.w_nr = wr;
        app.w_nc = wc;
        app.in_nr = app.out_nr + wr;
        app.in_nc = app.out_nc + wc;

        const size_t w_size_elts = wr * wc;
        const size_t in_size_elts = ( app.out_nc + wc ) * ( app.out_nr + wr );

        // Copy App structure to GPU.
        //
        CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

        const int wp_limit = ki->cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary =
          min(wp_per_block_goal*wp_sz,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 1;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            if ( vary_warps && wp_cnt > 4 && wp_cnt & 0x3 ) continue;


            /// Compute Expected Computation and Communication
            //
            // Number of multiply/add operations. Ignore everything else.
            //
            const int64_t num_ops_fp = out_size_elts * w_size_elts;
            const int64_t num_ops_ls = 0; // Load and store instructions.

            //
            // Amount of data in and out of GPU chip --- if perfect.
            //
            const int64_t amt_data_elts =
              in_size_elts + out_size_elts + w_size_elts;
            const int64_t amt_data_bytes =
              sizeof(app.h_in[0]) * amt_data_elts;

            {
              // Zero the output array.
              //
              CE(hipMemset(app.d_out,0,out_size_bytes));

              // Measure execution time starting "now", which is after data
              // set to GPU.
              //
              CE(hipEventRecord(gpu_start_ce,0));

              typedef void (*KPtr)();

              /// Launch Kernel
              //
              KPtr(ki->func_ptr) <<< num_blocks, thd_per_block >>>();

              // Stop measuring execution time now, which is before is data
              // returned from GPU.
              //
              CE(hipEventRecord(gpu_stop_ce,0));
              CE(hipEventSynchronize(gpu_stop_ce));
              float cuda_time_ms = -1.1;
              CE( hipEventElapsedTime
                  (&cuda_time_ms,gpu_start_ce,gpu_stop_ce) );

              const double this_elapsed_time_s = cuda_time_ms * 0.001;

              const double thpt_compute_gflops =
                num_ops_fp / this_elapsed_time_s * 1e-9;
              const double thpt_data_gbps =
                amt_data_bytes / this_elapsed_time_s * 1e-9;

              // Number of load/store operations per second.
              const double chip_ls_ops = info.chip_sp_flops / 4;

              const double t_bound_fp = num_ops_fp / info.chip_sp_flops;
              const double t_bound_ls = num_ops_ls / chip_ls_ops;
              const double t_bound_insn = t_bound_fp + t_bound_ls;

              {
                const double comp_frac = t_bound_insn / this_elapsed_time_s;
                //  1e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  ki->get_max_active_blocks_per_mp(thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute number of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                pTable_Row row(table);
                table.entry("r","%2d",wr);
                table.entry("c","%2d",wc);
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);
                string util_hdr =
                  "--- Utilization: ++Compute++   **Data**  ";
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                const bool bw_more = bw_frac > comp_frac;
                const char* sym = bw_more ? "+*" : "*+";
                const double frac_min =  bw_more ? comp_frac : bw_frac;
                const double frac_max = !bw_more ? comp_frac : bw_frac;
                const size_t len_min = frac_min * max_st_len + 0.5;
                const size_t len_max = frac_max * max_st_len + 0.5;
                string bar = string( len_min, sym[0] ) +
                  string( len_max - len_min, sym[1] );

                table.entry(util_hdr,fmt, bar, pTable::pT_Left);
              }

              // Copy output array from GPU to CPU.
              //
              CE( hipMemcpy
                  ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDefault) );
              int err_count = 0;
              for ( size_t i=0; i<out_size_elts; i++ )
                {
                  if ( fabs( h_outs_check[rc][i] - app.h_out[i] ) > 1e-4 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf
                          ("Error at rc %zd %zd: %.7f != %.7f (correct)\n",
                           i / app.out_nc, i % app.out_nc,
                           app.h_out[i], h_outs_check[rc][i] );
                    }
                }
              if ( err_count )
                printf("Total errors %d\n", err_count);
            }
          }
      }

  }

}
