#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 1 - Spring 2017
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2017/hw02.pdf

 /// Documentation
//
//   c++:  http://en.cppreference.com
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <new>
#include <string>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <unistd.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>

using namespace std;

inline double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

// Matrix dimension. (Matrix size will be N by N.)
//
const int N = 16;

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

const char* elt_type_str(float f){ return "float"; }
const char* elt_type_str(double f){ return "double"; }
const char* elt_type_str(int f){ return "int"; }

struct App
{
  // Number of input and output matrices.
  //
  int n_mats;

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding n_mats N by N-element matrices
  //       and so the total size of h_in is n_mats * N * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;
};


// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxm_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.n_mats;
  const int inc = num_threads;

  auto idx = [](int i,int r,int c) { return i * N*N + r * N + c; };

  for ( int h=start; h<stop; h += inc )
    {
      for ( int r=0; r<N; r++ )
        {
          for ( int c=0; c<N; c++ )
            {
              Elt_Type elt_rc = 0;
              for ( int k=0; k<N; k++ )
                elt_rc += din[ idx( h, r, k ) ] * din[ idx( h, k, c ) ];
              dout[ idx( h, r, c ) ] = elt_rc;
            }
        }
    }
}


template<int thds_per_mat>
__global__ void
mxm_g_split(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// HOMEWORK 2 Problem 2 solution in this routine.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  auto idx = [](int i,int r,int c) { return i * N*N + r * N + c; };

  const int start = tid;
  const int stop = d_app.n_mats;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      for ( int r=0; r<N; r++ )
        {
          for ( int c=0; c<N; c++ )
            {
              Elt_Type elt_rc = 0;
              for ( int k=0; k<N; k++ )
                elt_rc += din[ idx( h, r, k ) ] * din[ idx( h, k, c ) ];
              dout[ idx( h, r, c ) ] = elt_rc;
            }
        }
    }
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxm_g_only);

  /// Homework 2: Add mxm_g_split specializations as needed.
  //
  info.GET_INFO(mxm_g_split<1>);
  info.GET_INFO(mxm_g_split<8>);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  const int nsq = N * N;

  // Examine argument 1, block count, default is number of MPs.
  // Negative value is a multiple of the number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
    arg1_int == 0 ? num_mp :
    arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  // Zero means vary the number of threads in multiples of 32.
  //
  const int thd_per_block_arg = argc < 3 ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
    thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  const int in_size_bytes_targ =
    argc < 4 ? 1 << 24 : int( atof(argv[3]) * (1<<20) );
  app.n_mats = in_size_bytes_targ / ( nsq * sizeof(app.h_in[0]) );

  if ( num_threads <= 0 || app.n_mats <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const size_t in_size_elts = size_t(app.n_mats) * nsq;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t out_size_elts = size_t(app.n_mats) * nsq;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * nsq;
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf
    ("Input is %d  %d x %d matrices of %s, total size %zd bytes (%.1f MiB).\n",
     app.n_mats, N, N,
     elt_type_str(Elt_Type(1)), in_size_bytes,
     double(in_size_bytes)/(size_t(1)<<20));

#define IDX(i,r,c) ((i) * nsq + (r) * N + (c))

  // Initialize input array.
  //
  for ( int i=0; i<app.n_mats; i++ )
    for ( int r=0; r<N; r++ )
      for ( int c=0; c<N; c++ )
        app.h_in[ IDX(i,r,c) ] = debug ? Elt_Type(c) : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.n_mats; i++ )
    for ( int r=0; r<N; r++ )
      for ( int c=0; c<N; c++ )
        {
          app.h_out_check[IDX(i,r,c)] = 0;
          for ( int k=0; k<N; k++ )
            app.h_out_check[IDX(i,r,c)] +=
              app.h_in[IDX(i,r,k)] * app.h_in[IDX(i,k,c)];
        }

  const int64_t num_ops = int64_t(N) * nsq * app.n_mats;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int64_t amt_data_bytes = in_size_bytes + out_size_bytes;


  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  // Launch kernel multiple times and keep track of the best time.
  printf("Launching with %d blocks of up to %d threads. \n",
         num_blocks, thd_per_block_goal);

  auto check = [&]()
    {
      int err_count = 0;
      for ( int i=0; i<app.n_mats; i++ )
        for ( int r=0; r<N; r++ )
          for ( int c=0; c<N; c++ )
            {
              const int idx = IDX(i,r,c);

              if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                {
                  err_count++;
                  if ( err_count < 5 )
                    printf
                      ("Error at mat %d elt %d,%d: "
                       "%.7f != %.7f (correct)\n",
                       i, r, c, app.h_out[idx], app.h_out_check[idx] );
                }
            }
      if ( err_count )
        printf("Total errors %d\n", err_count);
    };

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      Kernel_Info& k = info.ki[kernel];

      hipFuncAttributes& cfa = k.cfa;
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      const int thd_limit = wp_limit << 5;
      const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

      const int wp_start = 4;
      const int wp_stop = vary_warps ? wp_limit : wp_start;
      const int wp_inc = 4;

      for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
        {
          const int thd_per_block =
            vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

          double this_elapsed_time_s = 0;

          // Zero the output array.
          //
          CE(hipMemset(app.d_out,0,out_size_bytes));

          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
            (app.d_out,app.d_in);

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime
             (&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
          this_elapsed_time_s = cuda_time_ms * 0.001;

          // Should count fmadd as 1.
          const double thpt_compute_gflops =
            num_ops / this_elapsed_time_s * 1e-9;
          const double thpt_data_gbps =
            amt_data_bytes / this_elapsed_time_s * 1e-9;

          if ( vary_warps )
            {
              const int rate =
                sizeof(Elt_Type) == sizeof(float)
                ? info.chip_sp_flops : info.chip_dp_flops;
              const double comp_frac __attribute__((unused))
                = 1e9 * thpt_compute_gflops / rate;
              const double comm_frac =
                1e9 * thpt_data_gbps / info.chip_bw_Bps;
              const int max_st_len = 52;

              // Number of warps, rounded up.
              //
              const int num_wps = ( thd_per_block + 31 ) >> 5;

              // The maximum number of active blocks per MP for this
              // kernel when launched with a block size of thd_per_block.
              //
              const int max_bl_per_mp =
                info.get_max_active_blocks_per_mp(kernel,thd_per_block);

              // Compute number of blocks available per MP based only on
              // the number of blocks.  This may be larger than the
              // number of blocks that can run.
              //
              const int bl_per_mp_available =
                0.999 + double(num_blocks) / num_mp;

              // The number of active blocks is the minimum of what
              // can fit and how many are available.
              //
              const int bl_per_mp =
                min( bl_per_mp_available, max_bl_per_mp );

              // Based on the number of blocks, compute the num ber of warps.
              //
              const int act_wps = num_wps * bl_per_mp;

              if ( wp_cnt == wp_start )
                printf("Kernel %s:\n", info.ki[kernel].name);

              printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s %s\n",
                     num_wps, act_wps,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps,
                     string(int(comm_frac*max_st_len),'=').c_str());

            } else {

            printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                   info.ki[kernel].name,
                   (thd_per_block + 31 ) >> 5,
                   this_elapsed_time_s * 1e6,
                   thpt_compute_gflops, thpt_data_gbps);

          }

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              ( app.h_out, app.d_out, out_size_bytes,
                hipMemcpyDeviceToHost) );

          check();

        }
    }
}
