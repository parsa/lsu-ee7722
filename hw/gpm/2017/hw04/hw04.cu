#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <nperf.h>

using namespace std;

struct App
{

  float elt_min;
  float bin_width_inv;
  int nbins;
  int nelts;

  float *d_data;
  int *d_histo;

};

App app;
__constant__ App d_app;

const int NBINS = 20;

typedef void (*KPtr)();

extern "C" __global__ void
h_basic()
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_threads = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < d_app.nelts; idx += num_threads )
    {
      float elt = d_app.d_data[idx];
      int bin_raw = ( elt - d_app.elt_min ) * d_app.bin_width_inv;
      int bin_pos = max( 0, bin_raw );
      int bin = min( bin_pos, d_app.nbins-1 );
      atomicAdd(&d_app.d_histo[bin],1);
    }
}

extern "C" __global__ void
h_priv_blk()
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_threads = blockDim.x * gridDim.x;

  __shared__ int histo[NBINS];

  for ( int i = threadIdx.x; i < NBINS; i += blockDim.x )
    histo[i] = 0;

  __syncthreads();

  for ( int idx = tid;  idx < d_app.nelts; idx += num_threads )
    {
      float elt = d_app.d_data[idx];
      int bin_raw = ( elt - d_app.elt_min ) * d_app.bin_width_inv;
      int bin_pos = max( 0, bin_raw );
      int bin = min( bin_pos, NBINS-1 );
      atomicAdd(&histo[bin],1);
    }
  __syncthreads();
  for ( int i = threadIdx.x; i < NBINS; i += blockDim.x )
    atomicAdd( &d_app.d_histo[i], histo[i] );
}

extern "C" __global__ void
h_priv_blk2()
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int DEG = 256;
  __shared__ int histo[NBINS][DEG];
  int *hlinear = &histo[0][0];
  const int dupidx = threadIdx.x % DEG;

  for ( int i = threadIdx.x; i < NBINS*DEG; i += blockDim.x )
    hlinear[i] = 0;

  __syncthreads();

  for ( int idx = tid;  idx < d_app.nelts; idx += num_threads )
    {
      float elt = d_app.d_data[idx];
      int bin_raw = ( elt - d_app.elt_min ) * d_app.bin_width_inv;
      int bin_pos = max( 0, bin_raw );
      int bin = min( bin_pos, NBINS-1 );
      if ( blockDim.x > DEG )
        atomicAdd(&histo[bin][dupidx],1);
      else
        histo[bin][dupidx]++;
    }
  __syncthreads();

  if ( threadIdx.x >= NBINS ) return;

  int val = 0;
  for ( int i=0; i<DEG; i++ )
    val += histo[threadIdx.x][i];
  atomicAdd( &d_app.d_histo[threadIdx.x], val );
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);


  /// Add kernels to the list of kernels to run.
  //

  info.GET_INFO(h_basic);
  info.GET_INFO(h_priv_blk);
  info.GET_INFO(h_priv_blk2);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}

int
main(int argc, char **argv)
{
  // Initialize the collection of metrics if GPU debugging is
  // off. Metrics collected using the NPerf library which itself uses
  // the NVIDIA CUPTI API to collect data from GPU performance (event)
  // counters and to compute performance metrics in terms of the event
  // data.
  //
# ifdef __P_CUDA_DEBUG__
  NPerf_init(false);
# else
  NPerf_init(true);
# endif

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  /// Indicate which metrics to collect.
  //
  //  See the CUPTI User's Guide for a list of metrics, Section 1.6
  //   for the February 2016 guide.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("eligible_warps_per_cycle");
  NPerf_metric_collect("gld_efficiency");
  NPerf_metric_collect("gst_efficiency");
  NPerf_metric_collect("gld_throughput");
  NPerf_metric_collect("gst_throughput");
  NPerf_metric_collect("shared_load_transactions_per_request");
  NPerf_metric_collect("shared_store_transactions_per_request");
  //
  // Note: The more metrics that are collected, the more times a
  // kernel will need to be run and the longer you'll have to wait for
  // an answer. 


  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  // Negative value is a multiple of the number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
    arg1_int == 0 ? num_mp :
    arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  // Zero means vary the number of threads in multiples of 32.
  //
  const int thd_per_block_arg = argc < 3 ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
    thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  double arg_3_raw = argc < 4 ? 1 : atof(argv[3]);
  const int in_size_bytes_targ =
    argc < 4 ? 1 << 24 : int( atof(argv[3]) * (1<<20) );
  app.nelts = in_size_bytes_targ / ( sizeof(app.d_data[0]) );

  app.nbins = NBINS;

  if ( num_threads <= 0 || app.nbins <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  app.elt_min = -0.5 / app.nbins;
  const float elt_max = 1.0 + 0.5 / app.nbins;
  app.bin_width_inv = app.nbins / ( elt_max - app.elt_min );

  vector<float> din(app.nelts);
  for ( auto& e: din ) e = drand48();
  vector<int> histo_check(app.nbins);
  vector<int> histo(app.nbins);

  for ( auto& e: din )
    histo_check
      [min(app.nbins-1,max(0,int((e-app.elt_min)*app.bin_width_inv)))]++;

  const size_t data_size_bytes = app.nelts * sizeof(app.d_data[0]);
  const size_t histo_size_bytes = app.nbins * sizeof(app.d_histo[0]);

  printf("Running with %d bins and %.1f MiB array, %.1f elt/thd\n",
         app.nbins, double(data_size_bytes)/(1<<20) ,
         double(app.nelts) / ( num_blocks * num_threads ));


  CE( hipMalloc( &app.d_data, data_size_bytes ) );
  CE( hipMalloc( &app.d_histo, histo_size_bytes ) );

  CE( hipMemcpy
      ( app.d_data, din.data(), data_size_bytes, hipMemcpyHostToDevice ) );

  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  const size_t amt_data_bytes = data_size_bytes + num_mp * histo_size_bytes;
  const size_t num_madds = app.nelts;

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      Kernel_Info& k = info.ki[kernel];
      bool heading_printed = false;

      hipFuncAttributes& cfa = k.cfa;
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      const int thd_limit = wp_limit << 5;
      const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

      const int wp_start = 4;
      const int wp_stop = vary_warps ? 32 : wp_start;
      const int wp_inc = 4;

      for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
        {
          const int thd_per_block =
            vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

          if ( ! k.block_size_okay(thd_per_block) ) continue;

          // Zero the output array.
          //
          CE( hipMemset( app.d_histo, 0, histo_size_bytes ) );

          // Measure execution time starting "now", which is after data
          // set to GPU.  This is only used when NPerf is not active.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          for ( NPerf_data_reset(); NPerf_need_run_get(); )
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>();

          const int nlaunches = NPerf_kernel_nlaunches_get();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime
             (&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

          const double this_elapsed_time_s =
            NPerf_metrics_collection_get()
            ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

          const double thpt_compute_gflops =
            num_madds / this_elapsed_time_s * 1e-9;
          const double thpt_data_gbps =
            amt_data_bytes / this_elapsed_time_s * 1e-9;

          if ( vary_warps )
            {
              const int rate =
                sizeof(app.d_data[0]) == sizeof(float)
                ? info.chip_sp_flops : info.chip_dp_flops;
              const double comp_frac __attribute__((unused)) =
                1e9 * thpt_compute_gflops / rate;
              const double comm_frac =
                1e9 * thpt_data_gbps / info.chip_bw_Bps;
              const int max_st_len = 40;

              // Number of warps, rounded up.
              //
              const int num_wps = ( thd_per_block + 31 ) >> 5;

              // The maximum number of active blocks per MP for this
              // kernel when launched with a block size of thd_per_block.
              //
              const int max_bl_per_mp =
                info.get_max_active_blocks_per_mp(kernel,thd_per_block);

              // Compute number of blocks available per MP based only on
              // the number of blocks.  This may be larger than the
              // number of blocks that can run.
              //
              const int bl_per_mp_available =
                0.999 + double(num_blocks) / num_mp;

              // The number of active blocks is the minimum of what
              // can fit and how many are available.
              //
              const int bl_per_mp =
                min( bl_per_mp_available, max_bl_per_mp );

              // Based on the number of blocks, compute the num ber of warps.
              //
              const int act_wps = num_wps * bl_per_mp;

              if ( !heading_printed )
                printf("Kernel %s:\n", info.ki[kernel].name);
              heading_printed = true;

              printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s %s\n",
                     num_wps, act_wps,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps,
                     string(int(comm_frac*max_st_len),'=').c_str());

              printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s "
                     "%5.1f I/F  %4.1f wp/c  %3.0f%%  %4.1f\n",
                     num_wps, act_wps,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps,
                     NPerf_metric_value_get("inst_executed") * 32 / num_madds,
                     NPerf_metric_value_get("eligible_warps_per_cycle"),
                     NPerf_metric_value_get("gld_efficiency"),
                     NPerf_metric_value_get("shared_load_transactions_per_request"));


            } else {

              printf
                ("%-10s %2d wp  %7.0f µs  %4.0f GF  %4.0f (%4.0f) GB/s  "
                 "%5.2f I/F  %5.1f%% %4.1f\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 ( NPerf_metric_value_get("gld_throughput") +
                   NPerf_metric_value_get("gst_throughput") ) * 1e-9,
                 NPerf_metric_value_get("inst_executed") * 32 / num_madds,
                 //  NPerf_metric_value_get("gld_efficiency"),
                 //  NPerf_metric_value_get("gst_efficiency"),
                 NPerf_metric_value_get("eligible_warps_per_cycle"),
                 //  NPerf_metric_value_get("shared_load_transactions_per_request"),
                 NPerf_metric_value_get("shared_store_transactions_per_request")
                 );

              if (0 )
            printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                   info.ki[kernel].name,
                   (thd_per_block + 31 ) >> 5,
                   this_elapsed_time_s * 1e6,
                   thpt_compute_gflops, thpt_data_gbps);

          }

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              ( histo.data(), app.d_histo,
                histo_size_bytes, hipMemcpyDeviceToHost ) );

          int errs = 0;
          for ( int i=0; i<app.nbins; i++ )
            {
              const int bin_scaled = histo[i] / nlaunches;
              if ( bin_scaled == histo_check[i] ) continue;
              errs++;
              if ( errs < 5 ) printf("Error at %d  %d != %d (correct)\n",
                                     i, bin_scaled, histo_check[i]);
            }

        }
    }


  return 0;
}
