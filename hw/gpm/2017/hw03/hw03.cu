#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 3 - Spring 2017
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2017/hw03.pdf

 /// Documentation
//
//   c++:  http://en.cppreference.com
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <new>
#include <string>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <unistd.h>
#include <stdlib.h>
#include <functional>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>
#include <nperf.h>

using namespace std;

inline double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

// Matrix dimension. (Matrix size will be N by N.)
//
const int N = 8;

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

const char* elt_type_str(float f){ return "float"; }
const char* elt_type_str(double f){ return "double"; }
const char* elt_type_str(int f){ return "int"; }

struct App
{
  // Number of input and output matrices.
  //
  int n_mats;

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_a, *h_b, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding n_mats N by N-element matrices
  //       and so the total size of h_in is n_mats * N * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_a, *d_b, *d_out;
};


// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout);




extern "C" __global__ void
mxm_volk(Elt_Type* __restrict__ dout)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  // Convenience function for finding index of the element at row r,
  // column c, in matrix i.
  //
  auto idx = [](int i,int r,int c) { return i * N*N + r * N + c; };

  const int start = tid / N;
  const int stop = d_app.n_mats;
  const int inc = num_threads / N;

  // Chunk Size: Number of columns of matrix A to load into shared memory.
  //
  const int CS = 32 / sizeof(Elt_Type);

  // Column in matrix B assigned to this thread.
  //
  const int cb = threadIdx.x % N;

  // Column offset to load when populating shared memory.
  //
  const int c0 = threadIdx.x % CS;

  // First row to load when populating shared memory.
  //
  const int r0 = ( threadIdx.x % N ) / CS;

  // 
  //
  const int h0 = threadIdx.x / N;

  // Number of times per matrix that shared memory will have to be loaded.
  //
  const int RS = N / CS;

  const int MpB = 32 * 32 / N;    // Matrices per block.

  // Storage for buffering N by CS submatrix of matrix A.
  //
  __shared__ Elt_Type mat_a[MpB][N][CS];

  for ( int h=start; h<stop; h += inc )
    {
      // Storage for column of output matrix.
      //
      Elt_Type elt[N];
      for ( auto& e: elt ) e = 0;

      for ( int cc=0; cc<N; cc += CS )
        {
          // Write shared memory with an N by CS submatrix of A.
          //
          for ( int rr = 0; rr<N; rr += RS )
            mat_a[h0][rr + r0][c0] =
              d_app.d_a[ idx( h, rr + r0, cc + c0 ) ];

          if ( N > 32 ) __syncthreads();

          for ( int rb=0; rb<CS; rb++ )
            {
              const int r = cc + rb;  // Row in matrix B, column in mat A.
              Elt_Type elt_rb_cb = d_app.d_b[ idx( h, r, cb ) ];
              for ( int ra=0; ra<N; ra++ )
                elt[ra] += mat_a[h0][ra][rb] * elt_rb_cb;
            }

          if ( N > 32 ) __syncthreads();

        }
      for ( int r=0; r<N; r++ )
        dout[ idx( h, r, cb ) ] = elt[r];
    }
}

template<int thd_p_col = 2 >
__global__ void
mxm_tpc(Elt_Type* __restrict__ dout)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  // Convenience function for finding index of the element at row r,
  // column c, in matrix i.
  //
  auto idx = [](int i,int r,int c) { return i * N*N + r * N + c; };

  const int start = tid / N;
  const int stop = d_app.n_mats;
  const int inc = num_threads / N;

  // Chunk Size: Number of columns of matrix A to load into shared memory.
  //
  const int CS = 32 / sizeof(Elt_Type);

  // Column in matrix B assigned to this thread.
  //
  const int cb = threadIdx.x % N;

  // Column offset to load when populating shared memory.
  //
  const int c0 = threadIdx.x % CS;

  // First row to load when populating shared memory.
  //
  const int r0 = ( threadIdx.x % N ) / CS;

  // 
  //
  const int h0 = threadIdx.x / N;

  // Number of times per matrix that shared memory will have to be loaded.
  //
  const int RS = N / CS;

  const int MpB = 32 * 32 / N;    // Matrices per block.

  // Storage for buffering N by CS submatrix of matrix A.
  //
  __shared__ Elt_Type mat_a[MpB][N][CS];

  for ( int h=start; h<stop; h += inc )
    {
      // Storage for column of output matrix.
      //
      Elt_Type elt[N];
      for ( auto& e: elt ) e = 0;

      for ( int cc=0; cc<N; cc += CS )
        {
          // Write shared memory with an N by CS submatrix of A.
          //
          for ( int rr = 0; rr<N; rr += RS )
            mat_a[h0][rr + r0][c0] =
              d_app.d_a[ idx( h, rr + r0, cc + c0 ) ];

          if ( N > 32 ) __syncthreads();

          for ( int rb=0; rb<CS; rb++ )
            {
              const int r = cc + rb;  // Row in matrix B, column in mat A.
              Elt_Type elt_rb_cb = d_app.d_b[ idx( h, r, cb ) ];
              for ( int ra=0; ra<N; ra++ )
                elt[ra] += mat_a[h0][ra][rb] * elt_rb_cb;
            }

          if ( N > 32 ) __syncthreads();

        }
      for ( int r=0; r<N; r++ )
        dout[ idx( h, r, cb ) ] = elt[r];
    }
}


template<int tpc> bool mxm_tpc_block_size_okay(int bsize)
{
  // Return true if mxm_tpc can run for a block size of bsize threads.
  //
  const int thd_p_mat = N * tpc;
  return thd_p_mat <= bsize && bsize % thd_p_mat == 0;
};

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);


  /// Add kernels to the list of kernels to run.
  //

  info.GET_INFO(mxm_volk);

  info.GET_INFO(mxm_tpc<1>)
    .block_size_okay_user_func = mxm_tpc_block_size_okay<1>;
  info.GET_INFO(mxm_tpc<2>)
    .block_size_okay_user_func = mxm_tpc_block_size_okay<2>;
  info.GET_INFO(mxm_tpc<4>)
    .block_size_okay_user_func = mxm_tpc_block_size_okay<4>;
  info.GET_INFO(mxm_tpc<8>)
    .block_size_okay_user_func = mxm_tpc_block_size_okay<8>;


  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Initialize the collection of metrics if GPU debugging is
  // off. Metrics collected using the NPerf library which itself uses
  // the NVIDIA CUPTI API to collect data from GPU performance (event)
  // counters and to compute performance metrics in terms of the event
  // data.
  //
# ifdef __P_CUDA_DEBUG__
  NPerf_init(false);   // 
# else
  NPerf_init(true);
# endif

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  /// Indicate which metrics to collect.
  //
  //  See the CUPTI User's Guide for a list of metrics, Section 1.6
  //   for the February 2016 guide.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("eligible_warps_per_cycle");
  NPerf_metric_collect("gld_efficiency");
  NPerf_metric_collect("gst_efficiency");
  NPerf_metric_collect("gld_throughput");
  NPerf_metric_collect("gst_throughput");
  NPerf_metric_collect("shared_load_transactions_per_request");
  NPerf_metric_collect("shared_store_transactions_per_request");
  //
  // Note: The more metrics that are collected, the more times a
  // kernel will need to be run and the longer you'll have to wait for
  // an answer. 


  const int num_mp = info.cuda_prop.multiProcessorCount;

  const int nsq = N * N;

  // Examine argument 1, block count, default is number of MPs.
  // Negative value is a multiple of the number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
    arg1_int == 0 ? num_mp :
    arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  // Zero means vary the number of threads in multiples of 32.
  //
  const int thd_per_block_arg = argc < 3 ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
    thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  const int in_size_bytes_targ =
    argc < 4 ? 1 << 24 : int( atof(argv[3]) * (1<<20) );
  app.n_mats = in_size_bytes_targ / ( 2 * nsq * sizeof(app.h_a[0]) );

  if ( num_threads <= 0 || app.n_mats <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const size_t a_size_elts = size_t(app.n_mats) * nsq;
  const size_t a_size_bytes = a_size_elts * sizeof( app.h_a[0] );
  const size_t out_size_elts = size_t(app.n_mats) * nsq;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  // Amount of extra storage to allocate so that kernels can safely
  // access data beyond the end of the input, avoiding the need for
  // if statements.
  //
  const int overrun_size_elts = thd_per_block_goal * nsq;
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_a = new Elt_Type[ a_size_elts ];
  app.h_b = new Elt_Type[ a_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_a,  a_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_b,  a_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  const size_t in_size_bytes = 2 * a_size_bytes;

  printf
    ("Input is %d pairs of %d x %d matrices of %s,\n",
     app.n_mats, N, N, elt_type_str(Elt_Type(1)));
  printf
    ("  total size %zd bytes (%.1f MiB).\n",
     in_size_bytes, double(in_size_bytes)/(size_t(1)<<20));

  // Define a convenience function that computes the index for the
  // element at row r, column c in matrix i.
  //
  auto idx = [&](int i,int r,int c) { return i * nsq + r * N + c; };

  // Initialize input array.
  //
  for ( int i=0; i<app.n_mats; i++ )
    for ( int r=0; r<N; r++ )
      for ( int c=0; c<N; c++ )
        {
          app.h_a[ idx(i,r,c) ] = debug ? Elt_Type(c) : drand48();
          app.h_b[ idx(i,r,c) ] = debug ? Elt_Type(r) : drand48();
        }

  // Compute correct answer.
  //
  for ( int i=0; i<app.n_mats; i++ )
    for ( int r=0; r<N; r++ )
      for ( int c=0; c<N; c++ )
        {
          app.h_out_check[idx(i,r,c)] = 0;
          for ( int k=0; k<N; k++ )
            app.h_out_check[idx(i,r,c)] +=
              app.h_a[idx(i,r,k)] * app.h_b[idx(i,k,c)];
        }

  // Compute the total number of MADD operations.
  //
  const int64_t num_madds = int64_t(N) * nsq * app.n_mats;

  // Amount of data in and out of GPU chip.
  //
  const int64_t amt_data_bytes = in_size_bytes + out_size_bytes;

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_a, app.h_a, a_size_bytes, hipMemcpyHostToDevice ) );
  CE( hipMemcpy
      ( app.d_b, app.h_b, a_size_bytes, hipMemcpyHostToDevice ) );

  // Copy App structure to GPU.
  //
  CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

  // Launch kernel multiple times and keep track of the best time.
  printf("Launching %d blocks of %d threads with %.2f matrices per thread.\n",
         num_blocks, thd_per_block_goal,
         double(app.n_mats) * N / ( num_blocks * thd_per_block_goal ));
#ifdef __P_CUDA_DEBUG__
  printf("*** THIS VERSION compiled with CUDA debugging on. Will be slow.\n");
#endif

  auto check = [&]()
    {
      int err_count = 0;
      for ( int i=0; i<app.n_mats; i++ )
        for ( int r=0; r<N; r++ )
          for ( int c=0; c<N; c++ )
            {
              const int ei = idx(i,r,c);

              if ( fabs( app.h_out_check[ei] - app.h_out[ei] ) > 1e-5 )
                {
                  err_count++;
                  if ( err_count < 5 )
                    printf
                      ("Error at mat %d elt %d,%d: "
                       "%.7f != %.7f (correct)\n",
                       i, r, c, app.h_out[ei], app.h_out_check[ei] );
                }
            }
      if ( err_count )
        printf("Total errors %d\n", err_count);
    };

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      Kernel_Info& k = info.ki[kernel];
      bool heading_printed = false;

      hipFuncAttributes& cfa = k.cfa;
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      const int thd_limit = wp_limit << 5;
      const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

      const int wp_start = 4;
      const int wp_stop = vary_warps ? 32 : wp_start;
      const int wp_inc = 4;

      for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
        {
          const int thd_per_block =
            vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

          if ( ! k.block_size_okay(thd_per_block) ) continue;

          // Zero the output array.
          //
          CE(hipMemset(app.d_out,0,out_size_bytes));

          // Measure execution time starting "now", which is after data
          // set to GPU.  This is only used when NPerf is not active.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          for ( NPerf_data_reset(); NPerf_need_run_get(); )
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out);

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime
             (&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

          const double this_elapsed_time_s =
            NPerf_metrics_collection_get()
            ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

          const double thpt_compute_gflops =
            num_madds / this_elapsed_time_s * 1e-9;
          const double thpt_data_gbps =
            amt_data_bytes / this_elapsed_time_s * 1e-9;

          if ( vary_warps )
            {
              const int rate =
                sizeof(Elt_Type) == sizeof(float)
                ? info.chip_sp_flops : info.chip_dp_flops;
              const double comp_frac __attribute__((unused)) =
                1e9 * thpt_compute_gflops / rate;
              const double comm_frac =
                1e9 * thpt_data_gbps / info.chip_bw_Bps;
              const int max_st_len = 52;

              // Number of warps, rounded up.
              //
              const int num_wps = ( thd_per_block + 31 ) >> 5;

              // The maximum number of active blocks per MP for this
              // kernel when launched with a block size of thd_per_block.
              //
              const int max_bl_per_mp =
                info.get_max_active_blocks_per_mp(kernel,thd_per_block);

              // Compute number of blocks available per MP based only on
              // the number of blocks.  This may be larger than the
              // number of blocks that can run.
              //
              const int bl_per_mp_available =
                0.999 + double(num_blocks) / num_mp;

              // The number of active blocks is the minimum of what
              // can fit and how many are available.
              //
              const int bl_per_mp =
                min( bl_per_mp_available, max_bl_per_mp );

              // Based on the number of blocks, compute the num ber of warps.
              //
              const int act_wps = num_wps * bl_per_mp;

              if ( !heading_printed )
                printf("Kernel %s:\n", info.ki[kernel].name);
              heading_printed = true;

              printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s %s\n",
                     num_wps, act_wps,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps,
                     string(int(comm_frac*max_st_len),'=').c_str());

              printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s "
                     "%5.2f I/F  %4.1f wp/c  %3.0f%%  %4.1f\n",
                     num_wps, act_wps,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps,
                     NPerf_metric_value_get("inst_executed") * 32 / num_madds,
                     NPerf_metric_value_get("eligible_warps_per_cycle"),
                     NPerf_metric_value_get("gld_efficiency"),
                     NPerf_metric_value_get("shared_load_transactions_per_request"));


            } else {

              printf
                ("%-10s %2d wp  %7.0f µs  %4.0f GF  %4.0f (%4.0f) GB/s  "
                 "%5.2f I/F  %5.1f%% %4.1f\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 ( NPerf_metric_value_get("gld_throughput") +
                   NPerf_metric_value_get("gst_throughput") ) * 1e-9,
                 NPerf_metric_value_get("inst_executed") * 32 / num_madds,
                 //  NPerf_metric_value_get("gld_efficiency"),
                 //  NPerf_metric_value_get("gst_efficiency"),
                 NPerf_metric_value_get("eligible_warps_per_cycle"),
                 //  NPerf_metric_value_get("shared_load_transactions_per_request"),
                 NPerf_metric_value_get("shared_store_transactions_per_request")
                 );

              if (0 )
            printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                   info.ki[kernel].name,
                   (thd_per_block + 31 ) >> 5,
                   this_elapsed_time_s * 1e6,
                   thpt_compute_gflops, thpt_data_gbps);

          }

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              ( app.h_out, app.d_out, out_size_bytes,
                hipMemcpyDeviceToHost) );

          check();

        }
    }
}
