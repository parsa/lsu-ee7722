#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Sp 2011), GPU Microarchitecture
//
/// Homework 1 -- SOLUTION

 /// Search for SOLUTION to see parts that were changed.

 /// See stream-2.cc for instructions.

#include "solution.cuh"


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ Vector2* a;
__constant__ float* b;


// Declare Kernels
//
__global__ void dots_loopless();
__global__ void dots_stride_large();
__global__ void dots_stride_small();


__host__ hipError_t
kernels_get_attr
(struct hipFuncAttributes *attr, char **names, int max_count)
{
  // Return information about kernels.
  //
  // Information includes number of registers used.

  struct hipFuncAttributes *attr_stop = attr + max_count;
  hipError_t er = hipSuccess; // Tentative.
#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er || attr == attr_stop ) { *names = NULL; return er; }
  GETATTR(dots_loopless);
  GETATTR(dots_stride_large);
  GETATTR(dots_stride_small);
  *names = NULL;
  return er;
#undef GETATTR
}


 /// Homework 1: The routine below will have to be modfied for several problems.
 // 
__host__ void
dots_launch(dim3 dg, dim3 db, char version, int array_size)
{
  /// SOLUTION -- Problems 3 and 4
  //
  //  This routine modified so that it can call either dots_loopless,
  //  dots_stride_large, and dots_stride_small.  The routine that's
  //  called depends on the new argument version.

#if ALTERNATE_SOL
  /// SOLUTION -- Problem 3
  //
  //  This could be a solution to Problem 3, but this code doesn't use
  //  it. Instead, version is set to 'n' by the caller of dots_launch.
  
  int thread_count = dg.x * db.x;
  if ( thread_count >= array_size ) version = 'n';
#endif

  switch ( version ) {
  case 'n': dots_loopless<<<dg,db>>>(); break;
  case 'l': dots_stride_large<<<dg,db>>>(); break;
  case 's': dots_stride_small<<<dg,db>>>(); break;
  }
}

// This routine executes on the GPU.
//
__global__ void
dots_loopless()
{
  // Run the kernel for one iteration.  This only works
  // if the total number of threads is equal to the array size.

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}


// This routine also executes on the GPU.
//
__global__ void
dots_stride_large()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx_start = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;
  int stride = thread_count;

  for ( int idx = idx_start; idx < array_size; idx += stride )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}


 /// Homework 1: Modfiy this routine for one of the problems.
//
__global__ void
dots_stride_small()
{
  /// SOLUTION -- Problem 4
  //
  //  First, determine how many array elements each block should
  //  access:
  //
  int elt_per_block = ceilf( float(array_size) / gridDim.x );

  //  Determine the first and last+1 element to be accessed by this
  //  block (blockIdx.x).
  //
  int idx_block_start = elt_per_block * blockIdx.x;
  int idx_block_stop = min(array_size, idx_block_start + elt_per_block);

  //  Determine the first element to be accessed by this thread.
  //
  int idx_start = idx_block_start + threadIdx.x;
  //
  //  Note that because idx_start includes a "+ threadIdx.x" term
  //  consecutive threads will access consecutive array elements, which is
  //  necessary to construct fully occupied memory transactions.

  //  Determine how far ahead to skip each iteration.
  //
  int stride = blockDim.x;
  //
  // Note that the largest value of threadIdx.x is blockDim.x-1.

  for ( int idx = idx_start; idx < idx_block_stop; idx += stride )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}
