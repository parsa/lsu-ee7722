#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 1 - Spring 2016
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2016/hw01.pdf

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "util.h"


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

// Number of iterations used in scheduler_bm.
//
const int iters = 2;

struct App
{
  int array_size;

  // Host pointers to the input and output arrays, and to CPU-computed
  // output arrays used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  Elt_Type *h_out_sc_check;

  int mask;

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

extern "C" __global__ void
lane_aligned()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = num_threads;

  const int lane = threadIdx.x & 0x1f;
  const Elt_Type lane_label = 1e-5 * lane;

  for ( int i = start; i < stop; i += inc )
    d_app.d_out[i] = d_app.d_in[i] + lane_label;

}

extern "C" __global__ void
scheduler_bm()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = num_threads;

  for ( int i = start; i < stop; i += inc )
    {
      Elt_Type accum = d_app.d_in[i];

      for ( int j=0; j<iters; j++ ) accum = __sinf(accum);
      d_app.d_out[i] = accum;
    }
}


typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(lane_aligned);
  info.GET_INFO(scheduler_bm);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  app.mask = 0x0;

  const int in_size_bytes = app.array_size * sizeof( app.h_in[0] );
  const int out_size_bytes = app.array_size * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal;
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ app.array_size ];
  app.h_out = new Elt_Type[ app.array_size ];
  app.h_out_check = new Elt_Type[ app.array_size ];
  app.h_out_sc_check = new Elt_Type[ app.array_size ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  // Initialize input array and correct answer.
  //
  for ( int i=0; i<app.array_size; i++ )
    {
      app.h_in[i] = i + 0.01 * ( i & 0x1f );
      app.h_out[i] = 0;
      app.h_out_check[i] = app.h_in[i] + 0.00001 * ( i & 0x1f );
      Elt_Type accum = app.h_in[i];
      for ( int j=0; j<iters; j++ ) accum = sin(accum);
      app.h_out_sc_check[i] = i < 100000 ? accum : -1;
    }

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;
                const int max_st_len = 52;  // Maximum stars length.

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                const int wps = ( thd_per_block + 31 ) >> 5;

                /// Problem 2: Assign appropriate value.
                const int act_wps = 0;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d wp  %2d acwp  %6.0f µs  %5.0f GB/s %s\n",
                       wps, act_wps,
                       this_elapsed_time_s * 1e6,
                       thpt_data_gbps,
                       &stars[stars_len-int(bw_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes,
                  hipMemcpyDeviceToHost) );
            int err_count = 0;
            Elt_Type* const out_check =
              kernel == 0 ? app.h_out_check : app.h_out_sc_check;
            const double tolerance = kernel == 0 ? 1e-5 : 1e-2;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( out_check[i] < 0 ) continue; // Don't check.

                /// Problem 3:
                //
                // Skip correctness check if thread was not supposed
                // to compute.

                if ( fabs( out_check[i] - app.h_out[i] ) > tolerance )
                  {
                    err_count++;
                    if ( err_count < 5 )
                      printf("Error at elt %#x: %.7f != %.7f (correct)\n",
                             i, app.h_out[i], out_check[i] );
                  }
              }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
