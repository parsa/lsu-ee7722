#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 1 SOLUTION - Spring 2016
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2016/hw01.pdf

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "util.h"


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  int array_size;

  // Host pointers to the input and output arrays, and to CPU-computed
  // output arrays used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  Elt_Type *h_out_sc_check;

  /// Problem 3 -- This might come in handy.
  int mask;

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

extern "C" __global__ void
lane_aligned()
{
  /// Problem 1 -- Put solution in this routine.

  /// SOLUTION Problem 1
  //
  //  Round the block size down to the largest multiple of 32, and
  //  use that "useful" block size instead of the real one.

  // SOLUTION: Round block dim to largest multiple of 32 <= blockDim.x
  //
  const int useful_block_dim = blockDim.x & ~ 0x1f;

  // SOLUTION: Use useful_block_dim to compute number of threads
  //
  const int useful_num_threads = useful_block_dim * gridDim.x;

  // SOLUTION: Just return if we are beyond the useful part.
  //
  if ( threadIdx.x >= useful_block_dim ) return;

  const int tid = threadIdx.x + blockIdx.x * useful_block_dim;
  const int lane = threadIdx.x & 0x1f;

  const int start = tid;
  const int stop = d_app.array_size;

  // Encode lane number so that it can be written to output array
  // elements. The CPU code will use this to check whether any
  // partial warps executed. Please don't change the line below,
  // that would be lying to the CPU code.
  //
  const Elt_Type lane_label = 1e-5 * lane;

  for ( int i = start; i < stop; i += useful_num_threads )
    d_app.d_out[i] = d_app.d_in[i] + lane_label;

}



 /// SOLUTION -- Problem 3
//
//   Increase number of iterations from 2 to 16 so that computation
//   time is a significant portion of execution time.

// Number of iterations used in scheduler_bm.
//
const int iters = 16;
//
// It is important that this value is a compile-time constant.


extern "C" __global__ void
scheduler_bm()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.array_size;
  const int inc = num_threads;

  for ( int i = start; i < stop; i += inc )
    {
      Elt_Type accum = d_app.d_in[i];

      // SOLUTION -- Problem 3
      //
      // Use mask to suppress execution of certain threads.
      //
      if ( ( tid & d_app.mask ) == 0 )
        for ( int j=0; j<iters; j++ ) accum = __sinf(accum);

      d_app.d_out[i] = accum;
    }
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(lane_aligned);
  info.GET_INFO(scheduler_bm);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count. Default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS | -BLOCKS_PER_MP ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  /// SOLUTION -- Problem 3. Get mask value from the command line.
  //
  app.mask = argc < 5 ? 0 : strtol(argv[4],NULL,0);

  const int in_size_bytes = app.array_size * sizeof( app.h_in[0] );
  const int out_size_bytes = app.array_size * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ app.array_size ];
  app.h_out = new Elt_Type[ app.array_size ];
  app.h_out_check = new Elt_Type[ app.array_size ];
  app.h_out_sc_check = new Elt_Type[ app.array_size ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes ) );

  // Initialize input array and arrays holding correct answers.
  //
  for ( int i=0; i<app.array_size; i++ )
    {
      app.h_in[i] = i + 0.01 * ( i & 0x1f );
      app.h_out[i] = 0;
      app.h_out_check[i] = app.h_in[i] + 0.00001 * ( i & 0x1f );
      Elt_Type accum = app.h_in[i];
      for ( int j=0; j<iters; j++ ) accum = sin(accum);

      // The NVIDIA hardware sin is not accurate for larger values,
      // so use a -2 to indicate that GPU output should not be checked
      // at this element.
      //
      app.h_out_sc_check[i] = i < 100000 ? accum : -2;
    }

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("\nLaunching with %d blocks of up to %d threads for %d elts "
           "and mask %#x.\n",
           num_blocks, thd_per_block_goal, app.array_size, app.mask);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE( hipMemset(app.d_out,0,out_size_bytes) );

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            info.ki[kernel].func_ptr <<< num_blocks, thd_per_block >>>();
            //
            // Confused?
            //
            //  info.ki[kernel].func_ptr holds a pointer to the kernel
            //    routine.

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double bw_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;
                const int max_st_len = 52;  // Maximum stars length.

                // Number of warps, rounded up.
                //
                const int wps = ( thd_per_block + 31 ) >> 5;

                /// Problem 2 Solution Goes Around Here

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                /// Problem 2: Assign appropriate value.

                /// SOLUTION - Problem 2
                //

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d wp  %2d acwp  %6.0f µs  %5.0f GB/s %s\n",
                       (thd_per_block + 31 ) >> 5,
                       act_wps,
                       this_elapsed_time_s * 1e6,
                       thpt_data_gbps,
                       &stars[stars_len-int(bw_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes,
                  hipMemcpyDeviceToHost) );
            int err_count = 0;
            Elt_Type* const out_check =
              kernel == 0 ? app.h_out_check : app.h_out_sc_check;
            const double tolerance = kernel == 0 ? 1e-5 : 1e-2;
            for ( int i=0; i<app.array_size; i++ )
              {
                if ( out_check[i] == -2 ) continue; // Don't check.

                /// SOLUTION -- Problem 3
                //
                // Skip correctness check if sine execution was
                // suppressed by the mask.
                // 
                if ( kernel == 1 && i & app.mask ) continue;

                if ( fabs( out_check[i] - app.h_out[i] ) > tolerance )
                  {
                    err_count++;
                    if ( err_count < 5 )
                      printf("Error at elt %#x: %.7f != %.7f (correct)\n",
                             i, app.h_out[i], out_check[i] );
                  }
              }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
