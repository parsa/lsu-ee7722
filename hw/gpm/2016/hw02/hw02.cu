#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 2 - Spring 2016
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2016/hw02.pdf
//

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "util.h"

#define N 16
#define M 16


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// Homework 2: Don't modify this routine.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_g_only_g(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_g_only_s(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}


extern "C" __global__ void
mxv_g_only_prob2
(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_g_only_g);
  info.GET_INFO(mxv_g_only_s);
  info.GET_INFO(mxv_g_only_prob2);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int =
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks =
           arg1_int == 0 ? info.cuda_prop.multiProcessorCount : abs(arg1_int);

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.num_vecs * N;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int out_size_elts = app.num_vecs * M;
  const int out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const int max_st_len = 52;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d wp  %6.0f µs  %5.0f GF  %5.0f GB/s %s\n",
                       (thd_per_block + 31 ) >> 5,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(comp_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
