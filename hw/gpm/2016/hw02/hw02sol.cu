#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 2 - Spring 2016 -- SOLUTION
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2016/hw02.pdf
//  Solution writeup: http://www.ece.lsu.edu/koppel/gp/2016/hw02_sol.pdf
//

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <string>
#include "util.h"

#define N 16
#define M 16

 /// SOLUTION -- Problem 3
//
//   Increase interleave factor from 4 to 16 to improve performance.
//
#define IF 16


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;

  /// SOLUTION -- Problem 3
  //
  //  Declare types for interleaved input and output. Note that there
  //  is no device interleaved output, the kernel always writes d_out,
  //  and host code copies that to either h_out_interleaved when
  //  necessary.
  //
  Elt_Type *h_in_interleaved;
  Elt_Type *d_in_interleaved;
  Elt_Type *h_out_interleaved;

  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

 /// SOLUTION - Problem 1
//
//   Declare a global (device)-space variable for the matrix.
//
__device__ App d_app_g;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// Homework 2: Don't modify this routine.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_g_only_g(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// Problem 1 Solution here, and other places.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;

        /// SOLUTION -- Problem 1
        //
        //  Read the matrix that was placed in the global address space.
        //
        for ( int c=0; c<N; c++ )
          elt += d_app_g.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_g_only_s(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// Problem 2 Solution here, and other places.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  /// SOLUTION -- Problem 2
  //
  //  Copy matrix to shared memory.
  //
  __shared__ Elt_Type matrix[M][N];
  Elt_Type* const m0 = &matrix[0][0];
  Elt_Type* const m0_c = &d_app.matrix[0][0];
  const int elts = M * N;
  for ( int idx = threadIdx.x; idx < elts; idx += blockDim.x )
    m0[idx] = m0_c[idx];
  __syncthreads();

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;

        /// SOLUTION -- Problem 2
        //
        // Use shared matrix instead of constant-space matrix.

        for ( int c=0; c<N; c++ ) elt += matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}


extern "C" __global__ void
mxv_g_only_interleave
(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  /// Problem 3 Solution here, and other places.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {

      /// SOLUTION -- Problem 3
      //
      //  Split index for vector h into two indices h = h0 + hh. Index
      //  h0 is the index of the first vector of the IF group we need,
      //  and hh is where within the IF group our vector is. For
      //  example, suppose IF = 4 and h = 17. Vectors 16, 17, 18, and
      //  19, form a group with h0 = 16, and we need the second vector
      //  so hh = 1.
      //
      const int64_t h0 = ( h / IF ) * IF;
      const int hh = h - h0;
      //
      // The type for h0 is set to 64 bits so that the compiler can
      // emit load instructions with constant offsets. Consider "h0*M
      // + r * IF + hh". If the compiler evaluates that using 32-bit
      // arithmetic then there might be 32-bit overflow, which
      // according to the C language must be discarded. Since the
      // compiler doesn't know whether there would be overflow (there
      // won't unless the array has more than 2^31 elements) it must
      // recompute the expression for each r iteration. By setting h0
      // to a 64-bit type there can't be overflow so the compiler can
      // put "h0*M" in a register and use "r*IF+hh" as a constant
      // offset (after adjusting for data type size).

      for ( int r=0; r<M; r++ )
        {
          Elt_Type elt = 0;

          /// SOLUTION -- Problem 3
          //
          //  Use group plus offset indexing for both din and dout.
          //
          for ( int c=0; c<N; c++ )
            elt += d_app.matrix[r][c] * din[ h0 * N + c * IF + hh ];
          dout[ h0 * M + r * IF + hh ] = elt;
        }
    }
}

GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  print_gpu_info();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_g_only_g);
  info.GET_INFO(mxv_g_only_s);
  info.GET_INFO(mxv_g_only_interleave);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

    // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block_arg = argc < 3 ? 1024 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int in_size_elts = app.num_vecs * N;
  const int in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const int out_size_elts = app.num_vecs * M;
  const int out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  /// SOLUTION -- Problem 3
  //
  //  Allocate host and device storage for interleaved arrays.
  //
  app.h_in_interleaved = new Elt_Type[ in_size_elts ];
  app.h_out_interleaved = new Elt_Type[ out_size_elts ];
  CE( hipMalloc( &app.d_in_interleaved, in_size_bytes + overrun_size_bytes ) );


  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  /// SOLUTION -- Problem 3
  //
  //  Copy data from the "flat" array to the interleaved array.
  //
  for ( int i0=0; i0<app.num_vecs; i0 += IF )
    for ( int c=0; c<N; c++ )
      for ( int ii = 0;  ii < IF;  ii ++ )
        {
          // The i0 loop iterates over groups of vectors containing
          // IF vectors each.  The ii loop iterates over vectors within
          // a group. Below, compute the vector index used by din.
          //
          const int i = i0 + ii;

          // Compute the index of component c for the regular array.
          //
          const int idx = i * N + c;

          // Compute the index for component c in the interleaved
          // array.
          //
          const int inter_idx = i0 * N + ii + c * IF;

          app.h_in_interleaved[inter_idx] = app.h_in[ idx ];
        }

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );


    /// SOLUTION -- Problem 3
    //
    //  Copy interleaved data to GPU.
    //
    CE( hipMemcpy
        ( app.d_in_interleaved, app.h_in_interleaved,
          in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    /// SOLUTION -- Problem 1
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app_g), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        /// SOLUTION -- Problem 3
        //
        const bool uses_interleaved_data =
          std::string(info.ki[kernel].name) == "mxv_g_only_interleave";

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            /// SOLUTION -- Problem 3
            //
            //  Use pointer to appropriate type.
            //
            Elt_Type* const d_in =
              uses_interleaved_data ? app.d_in_interleaved : app.d_in;

            // Launch Kernel
            //
            KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
              (app.d_out,d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s = cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac =
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const int max_st_len = 50;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                printf("%2d %2d wp  %6.0f µs  %4.0f GF  %4.0f GB/s %s\n",
                       num_wps, act_wps,
                       this_elapsed_time_s * 1e6,
                       thpt_compute_gflops, thpt_data_gbps,
                       &stars[stars_len-int(comp_frac*max_st_len)]
                       );

              } else {

              printf("K %-15s %2d wp  %11.3f µs  %8.3f GFLOPS  %8.3f GB/s\n",
                     info.ki[kernel].name,
                     (thd_per_block + 31 ) >> 5,
                     this_elapsed_time_s * 1e6,
                     thpt_compute_gflops, thpt_data_gbps);

            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            /// SOLUTION -- Problem 3
            //
            //  If interleaved data is being used, copy GPU data
            //  into app.h_out_interleaved and then restore it
            //  back to its original order.
            //
            Elt_Type* const h_out =
              uses_interleaved_data ? app.h_out_interleaved : app.h_out;

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );

            /// SOLUTION -- Problem 3
            //
            //  Uninterleave data.
            //
            if ( uses_interleaved_data )
              for ( int i0=0; i0<app.num_vecs; i0 += IF )
                for ( int r=0; r<M; r++ )
                  for ( int ii = 0;  ii < IF;  ii ++ )
                    {
                      const int reg_idx = ( i0 + ii ) * M + r;
                      const int inter_idx = i0 * M + ii + r * IF;
                      app.h_out[reg_idx] = h_out[inter_idx];
                    }

            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
