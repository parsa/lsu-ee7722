#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2019
 /// Homework 2
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2019/hw02.pdf
 //
 //  Modify this file only.


#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

#define N 24
#define M 24


// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;


// Vector of Four Elt_Types. Used instead of a float4.
//
struct __builtin_align__(4*sizeof(Elt_Type)) __align__(4*sizeof(Elt_Type))
  Elt_Type4
{
  // Elements can be accessed using homogeneous vector component names.
  Elt_Type x, y, z, w;

  // Elements can also be accessed as a 4-element array.
  __host__ __device__ Elt_Type& operator [] (int i)
  { return i==0?x:i==1?y:i==2?z:w; }
};

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to Elt_Type4's.
  //
  Elt_Type4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to Elt_Type4 elements than to four consecutive floats.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  // Relay on the compiler to use the read-only cache for din.
  //
  // Note that dout and d_app.d_out hold the same address, as do din
  // and d_app.d_in.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_vec_ld()
{
  // Read and write vector components as Elt_Type4.
  // This should use more, but not all, of a 32-B request for float elements.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;
  const int nq = N/4;  // One quarter of N.

  for ( int h=tid; h<d_app.num_vecs; h += num_threads )
    {
      // Read elements as Elt_Type4.
      Elt_Type4 vin[nq];
      for ( int i=0; i<nq; i++ ) vin[i] = d_app.d_in_f4[ h * ( N >> 2 ) + i ];

      Elt_Type vout[M];

      // Note: Unrolling this loop avoids local memory access.
#     pragma unroll
      for ( int r=0; r<M; r++ )
        {
          vout[r] = 0;
          for ( int c=0; c<N; c += 4 )
            for ( int i=0; i<4; i++ )
              vout[r] += d_app.matrix[r][c+i] * vin[c/4][i];
        }
      for ( int r=0; r<M; r+=4 )
        d_app.d_out_f4[ ( h * M + r ) >> 2 ] =
          Elt_Type4( { vout[r], vout[r+1], vout[r+2], vout[r+3] } );
    }
}



extern "C" __global__ void
mxv_o_per_thd()
{
  // Assign one vector to M threads, each thread computes one element.
  //
  // This arrangement avoids the need for any local memory buffering
  // and results in efficient global memory writes. Global memory
  // reads are still inefficient.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid / M;  // First vector number computed by this thread.
  const int r = tid % M;      // Vector element computed by this thread.

  const int stop = d_app.num_vecs;
  const int inc = num_threads / M;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h, compute output vector element r.

      Elt_Type vout = 0;
      for ( int c=0; c<N; c++ )
        vout += d_app.matrix[r][c] * d_app.d_in[ h * N + c ];

      d_app.d_out[ h * M + r ] = vout;
    }
}

extern "C" __global__ void
mxv_o_per_thd_sol()
{
  // Assign one vector to M threads, each thread computes one element.
  //
  // This arrangement avoids the need for any local memory buffering and
  // results in efficient global memory writes. 

  /// Homework 2:
  //  Modify so that input vectors read as Elt_Type4's.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid / M;  // First vector number computed by this thread.
  const int r = tid % M;      // Vector element computed by this thread.

  const int stop = d_app.num_vecs;
  const int inc = num_threads / M;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h, compute output vector element r.

      Elt_Type vout = 0;
      for ( int c=0; c<N; c++ )
        vout += d_app.matrix[r][c] * d_app.d_in[ h * N + c ];

      d_app.d_out[ h * M + r ] = vout;
    }
}


const int mxv_sh_ochunk_CS = 8;

extern "C" __global__ void
mxv_sh_ochunk()
{
  const int CS = mxv_sh_ochunk_CS;

  // Have CS threads perform one multiplication.
  // Each thread writes M/CS elements. 

  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          Elt_Type vin[CS];

          // Read in one vector component, and put it in shared memory.
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          // Read CS components, one we loaded, the others by our neighbors.
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }

    }
}

extern "C" __global__ void
mxv_sh_ochunk_sol()
{
  const int CS = mxv_sh_ochunk_CS;

  // Have CS threads perform one multiplication.
  // Each thread writes M/CS elements. 

  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          Elt_Type vin[CS];

          // Read in one vector component, and put it in shared memory.
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          // Read CS components, one we loaded, the others by our neighbors.
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }
    }
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_vec_ld);

  info.GET_INFO(mxv_o_per_thd);
  info.GET_INFO(mxv_o_per_thd_sol);

  info.GET_INFO(mxv_sh_ochunk);
  info.GET_INFO(mxv_sh_ochunk_sol);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("gld_efficiency");
  if ( opt_p )
    {
      NPerf_metric_collect("gst_efficiency");
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( false )
    NPerf_metrics_off();

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * M;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (Elt_Type4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (Elt_Type4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops_fp = int64_t(M) * N * app.num_vecs;  // Multiply-adds.
  const int64_t insns_addr = 4, insns_loop = 3;
  const int64_t insns_ld_st = N + M;
  const int64_t num_ops = num_ops_fp + insns_ld_st + insns_addr + insns_loop;

  // Amount of data in and out of GPU chip.
  const int64_t amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  const int output_width = stdout_width_get();
#if 0
  // Analysis of mxv_o_lbuf:
  const double lat_mem_cyc = 345;
  const double lat_iter_cyc = lat_mem_cyc + M * N + M;
  const double lat_iter_s = lat_iter_cyc / ( info.cuda_prop.clockRate * 1e3 );
  const double data_iter_B = sizeof(Elt_Type) * ( M + N );
  const double p = info.chip_bw_Bps * lat_iter_s / data_iter_B;
  const int sm_thpt_ls = 64;
  const double ni_fp = N * M;
  const double ni_mem = N + M;
  const double t_issue_1 = ni_fp / (num_mp*info.cc_per_mp)
    + ni_mem / ( num_mp * sm_thpt_ls );
  const double q = lat_iter_cyc / t_issue_1;
  printf("Analysis for mxv_o_lbuf: L = %.1f ns, "
         "p = %.1f wp/sm,  q= %.1f wp/sm\n",
         lat_iter_s * 1e9, p/32/num_mp, q/32/num_mp );
#endif
  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const auto func_ptr = info.ki[kernel].func_ptr;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 1;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        const int thd_per_vec =
          func_ptr == mxv_o_per_thd || func_ptr == mxv_o_per_thd_sol ? M :
          func_ptr == mxv_sh_ochunk || func_ptr == mxv_sh_ochunk_sol
          ? mxv_sh_ochunk_CS : 1;

        pTable table(stdout);

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop;
              wp_cnt += ( wp_cnt < 4 ? 1 : wp_inc ) )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  1e9 * thpt_compute_gflops
                  / ( sizeof(Elt_Type) == 4 ? info.chip_sp_flops :
                      sizeof(Elt_Type) == 8 ? info.chip_dp_flops : 1 );
                const double comm_frac =
                  min(2.0,1e9 * thpt_data_gbps / info.chip_bw_Bps);

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;
                const int act_thds_gpu =
                  min( num_mp * act_wps * 32, num_blocks * thd_per_block );

                const double iter_per_thd =
                  thd_per_vec * app.num_vecs / act_thds_gpu;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                pTable_Row row(table);
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("Lw/µs","%6.1f",
                            this_elapsed_time_s*1e6 / iter_per_thd );
                table.entry
                  ("I/op","%4.1f",
                   NPerf_metric_value_get("inst_executed") * 32.0 / num_ops );
                if ( opt_p )
                  {
                    table.header_span_start("R-Eff-%");
                    table.entry
                      ("Ld","%3.0f",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("St","%3.0f",
                       NPerf_metric_value_get("gst_efficiency"));
                    table.header_span_end();
                    table.header_span_start("L2-Cache");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.header_span_end();
                    table.header_span_start("DRAM");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                    table.header_span_end();
                  }

                const bool plot_bandwidth = true;

                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);

                string util_hdr =
                  plot_bandwidth ? "Data BW Util" : "FP Utilization";
                const double frac = plot_bandwidth ? comm_frac : comp_frac;
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }

}
