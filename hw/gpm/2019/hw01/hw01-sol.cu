#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 /// Spring 2019
 /// Homework 1 -- SOLUTION
 //
 //  Assignment: https://www.ece.lsu.edu/koppel/gp/2019/hw01.pdf
 //
 //  Modify this file only.


#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

const int slen = 24;
struct __align__(32) Some_Struct
{
  float f0, f1;
  char str[slen];
};


struct App
{
  int n_elts;
  Some_Struct *ss_in_d, *ss_in_h, *ss_out_d, *ss_out_h, *ss_check_h;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(const Some_Struct *dout, const Some_Struct *din);

extern "C" __global__ void
ss_l1ro
(Some_Struct* __restrict__ ss_out, const Some_Struct* __restrict__ ss_in)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  for ( int h=tid; h<d_app.n_elts; h += num_threads )
    {
      Some_Struct elt = ss_in[h];
      Some_Struct elt_out;
      bool ord = elt.f0 <= elt.f1;
      elt_out.f0 = ord ? elt.f0 : elt.f1;
      elt_out.f1 = ord ? elt.f1 : elt.f0;
      int delta = elt.f0 == elt.f1 ? 0 : ord ? -1 : 1;
      for ( int i=0; i<slen; i++ ) elt_out.str[i] = elt.str[i] + delta;
      ss_out[h] = elt_out;
    }
}

extern "C" __global__ void
ss_g_only(Some_Struct* ss_out, const Some_Struct* ss_in)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  for ( int h=tid; h<d_app.n_elts; h += num_threads )
    {
      Some_Struct elt = ss_in[h];
      Some_Struct elt_out;
      bool ord = elt.f0 <= elt.f1;
      elt_out.f0 = ord ? elt.f0 : elt.f1;
      elt_out.f1 = ord ? elt.f1 : elt.f0;
      int delta = elt.f0 == elt.f1 ? 0 : ord ? -1 : 1;
      for ( int i=0; i<slen; i++ ) elt_out.str[i] = elt.str[i] + delta;
      ss_out[h] = elt_out;
    }
}

typedef int4 my_type;
// typedef Some_Struct my_type;

extern "C" __global__ void
ss_sh(Some_Struct* ss_out, const Some_Struct* ss_in)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int ss_size_words = sizeof(Some_Struct) / sizeof(my_type);

  __shared__ Some_Struct ss_blk[1024];

  // Cast pointers to Some_Struct to pointers to integers so that
  // Some_Struct data can be moved around as a simple array of integers.
  //
  my_type* const ss_blk_wds = (my_type*) &ss_blk[0];
  my_type* const ss_in_wds = (my_type*) &ss_in[0];
  my_type* const ss_out_wds = (my_type*) &ss_out[0];

  /// SOLUTION

  // Prepare some common warp-related constants values.
  //
  const int wp_lg = 5;
  const int wp_sz = 1 << wp_lg;
  assert( wp_sz == warpSize );
  const int wp_mk = wp_sz - 1;  // Warp mask.
  const int lane = tid & wp_mk;
  const int wp_num = threadIdx.x >> wp_lg;

  for ( int h=tid; h<d_app.n_elts; h += num_threads )
    {
      // Compute element number accessed by first thread in this warp.
      const int h_wp = h - lane;
      // Compute word number accessed by first thread in this warp.
      const int h_wds = h_wp * ss_size_words + lane;

      // Load elements (structures) operated on by this warp.
      for ( int i=0; i<ss_size_words; i++ )
        ss_blk_wds[     wp_num * wp_sz * ss_size_words + lane + i * wp_sz    ]
          = ss_in_wds[h_wds + i * wp_sz];

      Some_Struct  elt = ss_blk[  threadIdx.x   ];
      Some_Struct& elt_out = ss_blk[threadIdx.x];

      bool ord = elt.f0 <= elt.f1;
      elt_out.f0 = ord ? elt.f0 : elt.f1;
      elt_out.f1 = ord ? elt.f1 : elt.f0;
      int delta = elt.f0 == elt.f1 ? 0 : ord ? -1 : 1;
      for ( int i=0; i<slen; i++ ) elt_out.str[i] += delta;

      // Store elements (structures) operated on by this warp.
      for ( int i=0; i<ss_size_words; i++ )
        ss_out_wds[h_wds + i * wp_sz]
          = ss_blk_wds[ wp_num * wp_sz * ss_size_words + lane + i * wp_sz];
    }
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(ss_g_only);
  info.GET_INFO(ss_l1ro);
  info.GET_INFO(ss_sh);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array. Fractional values okay.
  //
  app.n_elts = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.n_elts <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("gld_efficiency");
  if ( opt_p )
    {
      NPerf_metric_collect("gst_efficiency");
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
      NPerf_metric_collect("shared_load_transactions_per_request");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( false )
    NPerf_metrics_off();

  const size_t in_size_elts = app.n_elts;
  const size_t in_size_bytes = in_size_elts * sizeof( app.ss_in_h[0] );
  const size_t out_size_elts = app.n_elts;
  const size_t out_size_bytes = out_size_elts * sizeof( app.ss_out_h[0] );

  const int overrun_size_elts = thd_per_block_goal * 32;
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.ss_out_h[0] );

  // Allocate storage for CPU copy of data.
  //
  vector<Some_Struct> ss_in_v(in_size_elts);
  vector<Some_Struct> ss_out_v(in_size_elts);
  vector<Some_Struct> ss_check_v(in_size_elts);
  app.ss_in_h = ss_in_v.data();
  app.ss_out_h = ss_out_v.data();
  app.ss_check_h = ss_check_v.data();

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.ss_in_d,   in_size_bytes + overrun_size_bytes ) );
  CE( hipMalloc( &app.ss_out_d,  in_size_bytes + overrun_size_bytes ) );

  printf("Array size: %d.  Grid: %d blocks of %d thds.\n",
         app.n_elts, num_blocks, thd_per_block_goal);
  printf("Structure Size: %zd B,  slen = %d\n",
         sizeof(Some_Struct), slen);

  // Initialize input array.
  //
  for ( int i=0; i<app.n_elts; i++ )
    {
      Some_Struct& elt = ss_in_v[i];
      elt.f0 = drand48();
      elt.f1 = drand48();
      for ( int j=0; j<slen; j++ ) elt.str[j] = random();
      Some_Struct& elt_out = ss_check_v[i];
      bool ord = elt.f0 <= elt.f1;
      elt_out.f0 = ord ? elt.f0 : elt.f1;
      elt_out.f1 = ord ? elt.f1 : elt.f0;
      int delta = elt.f0 == elt.f1 ? 0 : ord ? -1 : 1;
      for ( int i=0; i<slen; i++ ) elt_out.str[i] = elt.str[i] + delta;
    }

  const int64_t num_ops = app.n_elts * ( 3 + slen );

  // Amount of data in and out of GPU chip.
  const int64_t amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  const int output_width = stdout_width_get();
  int n_errors_all = 0;

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.ss_in_d, app.ss_in_h, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        pTable table;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.ss_out_d,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.ss_out_d,app.ss_in_d);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  1e9 * thpt_compute_gflops
                  / ( sizeof(Some_Struct::f0) == 4 ? info.chip_sp_flops :
                      sizeof(Some_Struct::f0) == 8 ? info.chip_dp_flops : 1 );
                const double comm_frac =
                  min(2.0,1e9 * thpt_data_gbps / info.chip_bw_Bps);

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%4.0f", this_elapsed_time_s * 1e6);
                table.entry
                  ("I/op","%4.1f",
                   NPerf_metric_value_get("inst_executed") * 32.0 / num_ops );
                if ( opt_p )
                  {
                    table.header_span_start("Req U %");
                    table.entry
                      ("Ld","%3.0f",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("St","%3.0f",
                       NPerf_metric_value_get("gst_efficiency"));
                    table.header_span_end();
                    table.entry
                      ("Shx","%4.1f",
                       NPerf_metric_value_get
                       ("shared_load_transactions_per_request"));
                    table.header_span_start("L2 Cache");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.header_span_end();
                    table.header_span_start("DRAM");
                    table.entry
                      ("Rd θ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("Wr θ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                    table.header_span_end();
                  }

                const bool plot_bandwidth = true;

                if ( ! plot_bandwidth )
                  table.entry("FP θ","%4.0f", thpt_compute_gflops);
                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);

                string util_hdr =
                  plot_bandwidth ? "Data BW Util" : "FP Utilization";
                const double frac = plot_bandwidth ? comm_frac : comp_frac;
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.ss_out_h, app.ss_out_d,
                  out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.n_elts; i++ )
                {
                  Some_Struct eo = app.ss_out_h[i];
                  Some_Struct ec = app.ss_check_h[i];
                  int nd = 0;
                  for ( int j=0; j<slen; j++ ) nd += eo.str[j] != ec.str[j];
                  if ( eo.f0 != ec.f0 || eo.f1 != ec.f1 || nd )
                    {
                      err_count++;
                      n_errors_all++;
                      if ( err_count < 5 )
                        printf("Error at elt %d: "
                               "(%.2f,%.2f) != (%.2f,%.2f) (correct) %d nd\n",
                               i, eo.f0,eo.f1, ec.f0,ec.f1,nd);
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
        printf("%s",table.body_get());
      }
  }
  printf("Run completed, total errors: %d\n",n_errors_all);
}
