#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 4 - Spring 2018
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2018/hw04.pdf

 /// PUT SOLUTION IN THIS FILE. 


 /// Documentation
//
//   c++:  http://en.cppreference.com
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//   CUDA debugger: https://docs.nvidia.com/cuda/cuda-gdb/index.html


template <int BLOCK_LG, int RADIX_LG>
__device__ void
sort_block_sol_1_bit_split
(int bit_low, int bit_count,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);

template <int BLOCK_LG, int RADIX_LG>
__device__ void radix_sort_sol_pass_1_tile
(int digit_pos, int tile_idx, bool first_iter,
 Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s);


template <int BLOCK_LG, int RADIX_LG> __global__ void
radix_sort_sol_pass_1(int digit_pos, bool first_iter)
{
  __shared__ Pass_1_Stuff<BLOCK_LG,RADIX_LG> p1s;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  int elt_per_tile = block_size * elt_per_thread;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);
  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block);

  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : radix/block_size;
  const bool rad_participant = block_size <= radix || threadIdx.x < radix;
# define DIG(i) (threadIdx.x + (i) * block_size)

  if ( rad_participant )
    for ( int i = 0;  i < dig_per_thd;  i++ )
      p1s.ghisto[ DIG(i) ] = 0;

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    radix_sort_sol_pass_1_tile<BLOCK_LG,RADIX_LG>
      (digit_pos,tile_idx,first_iter,p1s);

  if ( !rad_participant ) return;

  for ( int i = 0;  i < dig_per_thd;  i++ )
    {
      const int histo_idx = blockIdx.x * radix + DIG(i);
      sort_histo[ histo_idx ] = p1s.ghisto[ DIG(i) ];
    }
}

template <int BLOCK_LG, int RADIX_LG> __device__ void
radix_sort_sol_pass_1_tile
(int digit_pos, int tile_idx,
 bool first_iter, Pass_1_Stuff<BLOCK_LG,RADIX_LG>& p1s)
{
  int start_bit = digit_pos * RADIX_LG;
  int block_size = 1 << BLOCK_LG;
  int radix = 1 << RADIX_LG;
  const int digit_mask = radix - 1;
  int elt_per_tile = elt_per_thread * block_size;
  int idx_block_start = elt_per_tile * tile_idx;
  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : radix/block_size;
  const bool rad_participant = block_size <= radix || threadIdx.x < radix;

  int idx_block_stop = min( dapp.array_size, idx_block_start + elt_per_tile );
  int idx_start = idx_block_start + threadIdx.x;

  Sort_Elt *sort_src = first_iter ? sort_in : sort_out;

  // Load Element Keys
  //
  for ( int sidx = threadIdx.x, i = 0;
        i < elt_per_thread; i++, sidx += block_size )
    {
      Sort_Elt key = sort_src[ idx_block_start + sidx ];
      assert( key );
      p1s.keys[sidx] = key;
    }

  // Sort based upon current digit position
  //
  sort_block_sol_1_bit_split<BLOCK_LG,RADIX_LG>(start_bit,RADIX_LG,p1s);

  // Write sorted elements to global memory and prepare for histogram.
  //
  for ( int idx = idx_start, sidx = threadIdx.x;
        idx < idx_block_stop; sidx += block_size, idx += block_size )
    {
      // Write element.
      //
      sort_out_b[idx] = p1s.keys[sidx];

      // Extract digit and write to shared memory.
      //
      int digit = ( p1s.keys[sidx] >> start_bit ) & digit_mask;
      p1s.keys[sidx] = digit;
    }

  if ( threadIdx.x == 0 ) p1s.keys[elt_per_tile] = radix;

  // Initialize histogram for this tile to zero.
  //
  if ( rad_participant )
    for ( int i = 0;  i < dig_per_thd;  i++ )
      p1s.thisto[ DIG(i) ] = 0;

  __syncthreads();

  // Find highest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];      // Our digit.
      int digit_1 = p1s.keys[sidx+1];  // Next guy's digit.

      // If "next guy's" digit is different then sidx is highest index
      // for digit.
      //
      if ( digit != digit_1 )
        p1s.runend[ digit ] = sidx;
    }

  __syncthreads();

  // Finish histogram by looking for smallest index for each digit.
  //
  for ( int i = 0; i < elt_per_thread; i++ )
    {
      int sidx = threadIdx.x + i * block_size;
      int digit = p1s.keys[sidx];                      // Our digit.
      int digit_0 = sidx > 0 ? int(p1s.keys[sidx-1]) : -1; // Previous guy's digit.
      if ( digit != digit_0 )
        {
          int run_end_sidx = p1s.runend[ digit ];
          int count = run_end_sidx - sidx + 1;
          p1s.ghisto[ digit ] += count;     // Histogram for block.
          p1s.thisto[ digit ] = count;        // Histogram for tile.
        }
    }

  __syncthreads();

  if ( !rad_participant ) return;

  for ( int i = 0;  i < dig_per_thd;  i++ )
    {
      const int thisto_idx = tile_idx * radix + DIG(i);
      sort_tile_histo[ thisto_idx ] = p1s.thisto[ DIG(i) ];
    }
}

template <int block_lg, int RADIX_LG>
__device__ void
sort_block_sol_1_bit_split
(int bit_low, int bit_count, Pass_1_Stuff<block_lg,RADIX_LG>& p1s)
{
  const int block_size = 1 << block_lg;
  const int elt_per_tile = elt_per_thread * block_size;

  // Sort Elements From LSB to MSB.
  //
  for ( int bit_pos=bit_low; bit_pos<bit_low+bit_count; bit_pos++ )
    {
      const uint bit_mask = 1 << bit_pos;

      // Storage for thread's keys.
      //
      Sort_Elt keys[elt_per_thread];

      __syncthreads();

      // Initialize data for prefix sum of bit bit_pos, and make copy of key.
      //
      int my_ones_write = 0;

      for ( int i = 0; i < elt_per_thread; i++ )
        {
          const int sidx = threadIdx.x * elt_per_thread + i;

          // Make a copy of key.
          //
          const Sort_Elt key = p1s.keys[ sidx ];
          keys[i] = key;
          if ( key & bit_mask ) my_ones_write++;
        }

      p1s.prefix[ threadIdx.x + 1 ] = my_ones_write;
      if ( threadIdx.x == 0 ) p1s.prefix[ 0 ] = 0;

      uint my_prefix = my_ones_write;

      // Compute a prefix sum of vectors.
      for ( int tree_level = 0; tree_level < block_lg; tree_level++ )
        {
          int dist = 1 << tree_level;
          int idx_neighbor = threadIdx.x - dist;
          __syncthreads();
          uint neighbor_prefix =
            threadIdx.x >= dist ? p1s.prefix[ idx_neighbor + 1 ] : 0;

          my_prefix += neighbor_prefix;
          __syncthreads();
          p1s.prefix[ threadIdx.x + 1 ] = my_prefix;
        }

      // At this point p1s.prefix contains exclusive prefix of each group.

      __syncthreads();

      const int all_threads_num_ones = p1s.prefix[ block_size ];
      const int idx_one_tid_0 = elt_per_tile - all_threads_num_ones;
      const int smaller_tids_num_ones = p1s.prefix[ threadIdx.x ];

      int idx_zero_me = threadIdx.x * elt_per_thread - smaller_tids_num_ones;
      int idx_one_me = idx_one_tid_0 + smaller_tids_num_ones;

      for ( int i = 0;  i < elt_per_thread;  i++ )
        {
          const int key = keys[i];
          const int new_idx = key & bit_mask ? idx_one_me++ : idx_zero_me++;
          p1s.keys[ new_idx ] = key;
        }

    }
  __syncthreads();
}


template <int BLOCK_LG, int RADIX_LG>
__global__ void
radix_sort_sol_pass_2(int digit_pos, bool last_iter)
{
  const int block_size = 1 << BLOCK_LG;
  int elt_per_tile = elt_per_thread * block_size;
  int tiles_per_array = div_ceil(dapp.array_size,elt_per_tile);
  int tiles_per_block = div_ceil(tiles_per_array,gridDim.x);

  int tile_start = tiles_per_block * blockIdx.x;
  int tile_stop = min( tiles_per_array, tile_start + tiles_per_block );

  const int sort_radix = 1 << RADIX_LG;
  const int digit_mask = sort_radix - 1;
  const int start_bit = digit_pos * RADIX_LG;

  volatile __shared__ int g_prefix[ sort_radix + 1 ];

  if ( threadIdx.x == 0 ) g_prefix[ 0 ] = 0;

  const int warp_sz = 32;
  const int dig_per_thd = BLOCK_LG >= RADIX_LG ? 1 : sort_radix/block_size;
  const bool rad_participant =
    block_size <= sort_radix || threadIdx.x < sort_radix;
  const bool rad_attendee = rad_participant || sort_radix > warp_sz;
  volatile __shared__ int tile_offsets[ 2 * sort_radix ];

  if ( rad_attendee )
    {
      int overhead_bin_sum[dig_per_thd];
      int global_bin_sum[dig_per_thd];
      int global_bin_prefix[dig_per_thd];

      for ( auto& e: global_bin_sum ) e = 0;

      if ( rad_participant )
        {
          for ( int gh_idx = 0; gh_idx < gridDim.x; gh_idx++ )
            {
              for ( int i = 0;  i < dig_per_thd;  i++ )
                {
                  const int d = threadIdx.x + i * block_size;
                  const int gh_bin_idx = gh_idx * sort_radix + d;
                  if ( gh_idx == blockIdx.x )
                    overhead_bin_sum[i] = global_bin_sum[i];
                  global_bin_sum[i] += sort_histo[gh_bin_idx];
                }
            }

          //
          // Compute Global Prefix Sum
          //

          for ( int i = 0;  i < dig_per_thd;  i++ )
            {
              const int d = threadIdx.x + i * block_size;
              g_prefix[ 1 + d ] = global_bin_sum[i];
            }
          //
          // At this point g_prefix holds a global histogram.

          for ( int i=0; i<dig_per_thd; i++ )
            global_bin_prefix[i] = global_bin_sum[i];
        }

      for ( int lev=0; lev<RADIX_LG; lev++ )
        {
          const int dist = 1 << lev;
          int sum_0[dig_per_thd];

          if ( sort_radix > warp_sz ) __syncthreads();
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                sum_0[i] = dist <= d ? g_prefix[ 1 + d - dist ] : 0;
              }
          if ( sort_radix > warp_sz ) __syncthreads();
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                g_prefix[ 1 + d ] = global_bin_prefix[i] += sum_0[i];
              }
        }


      // Now, g_prefix holds a global prefix sum.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in the entire array is to be written. That key
      // is probably being handled by block 0.

      if ( sort_radix > warp_sz ) __syncthreads();
      if ( rad_participant )
        for ( int i = 0;  i < dig_per_thd;  i++ )
          {
            const int d = threadIdx.x + i * block_size;
            g_prefix[ d ] += overhead_bin_sum[i];
          }

      //
      // Now, g_prefix holds a prefix sum for this block.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in this block is to be written.

    }

  __syncthreads();

  if ( rad_participant )
    for ( int i=0;  i<dig_per_thd;  i++ )
      {
        const int d = threadIdx.x + i * block_size;
        tile_offsets[d] = 0;
      }

  __syncthreads();

  for ( int tile_idx = tile_start; tile_idx < tile_stop; tile_idx++ )
    {
      int counts[dig_per_thd];

      if ( rad_attendee )
        {
          int offsets[dig_per_thd];
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                const int bo_idx = tile_idx * sort_radix + d;
                const int to_idx = sort_radix + d;
                offsets[i] = counts[i] = tile_offsets[ to_idx ] =
                  sort_tile_histo[ bo_idx ];
              }

          for ( int lev=0; lev<RADIX_LG; lev++ )
            {
              const int dist = 1 << lev;
              if ( sort_radix > warp_sz ) __syncthreads();
              if ( rad_participant )
                for ( int i = 0;  i < dig_per_thd;  i++ )
                  {
                    const int d = threadIdx.x + i * block_size;
                    int to_idx = sort_radix + d;
                    offsets[i] += tile_offsets[ to_idx - dist ];
                  }
              if ( sort_radix > warp_sz ) __syncthreads();
              if ( rad_participant )
                for ( int i = 0;  i < dig_per_thd;  i++ )
                  {
                    const int d = threadIdx.x + i * block_size;
                    int to_idx = sort_radix + d;
                    tile_offsets[ to_idx ] = offsets[i];
                  }
            }
          if ( rad_participant )
            for ( int i = 0;  i < dig_per_thd;  i++ )
              {
                const int d = threadIdx.x + i * block_size;
                assert( counts[i] <= elt_per_tile );
                assert( tile_offsets[sort_radix + d - 1] <= elt_per_tile );
              }
        }

      __syncthreads();

      int idx_tile_start = tile_idx * elt_per_tile;

      for ( int i=0; i<elt_per_thread; i++ )
        {
          int tile_elt_rank = threadIdx.x + i * block_size;
          int idx = idx_tile_start + tile_elt_rank;
          Sort_Elt key = sort_out_b[idx];
          uint digit = ( key >> start_bit ) & digit_mask;
          int tile_digit_rank = tile_offsets[ sort_radix + digit - 1 ];
          int key_digit_rank = tile_elt_rank - tile_digit_rank;
          assert( key_digit_rank >= 0 );
          int idx_digit_index = g_prefix[ digit ] + key_digit_rank;

          if ( debug_sort && last_iter )
            sort_out[idx] = ( idx_digit_index << 12 ) + tile_digit_rank;
          else
            sort_out[idx_digit_index] = key;

        }

      __syncthreads();

      if ( rad_participant )
        for ( int i = 0;  i < dig_per_thd;  i++ )
          {
            const int d = threadIdx.x + i * block_size;
            g_prefix[ d ] += counts[i];
          }

      //
      // Now, g_prefix holds a prefix sum for the next tile.
      //
      // E.g., g_prefix[3] is the location where the first key having
      // digit value 3 in the next tile is to be written.
    }
}
