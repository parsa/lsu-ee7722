#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//
 ///  Homework 3 - Spring 2018
//
//  Assignment: http://www.ece.lsu.edu/koppel/gp/2018/hw03.pdf

 /// Documentation
//
//   c++:  http://en.cppreference.com
//   CUDA: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//   CUDA debugger: https://docs.nvidia.com/cuda/cuda-gdb/index.html

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

// Maximum size of matrix.
#define N 64

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[N][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.

  // Size of matrix to use.
  int n;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);

template <int eN> __device__ void
mxv_sh_ochunk()
{
  /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
  //  Instead, modify mxv_sh_ochunk_sol_mn.

  // Matrix size is eN by eN.
  // Since size in this case is a compile-time constant code will
  // be of higher quality.

  // First things, first. Make sure that the problem size
  // matches the specialization.

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  assert( tid != 0 || eN == d_app.n );

  const int CS = 8;  // Chunk Size: Number of input vector components to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  // Maximum number of output vector components written per thread,
  // based on vector size, eN.
  const int NL = eN / CS;

  /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
  //  Instead, modify mxv_sh_ochunk_sol_mn.

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      // Initialize output vector components to zero.
      Elt_Type vout[NL];
      for ( auto& v: vout ) v = 0;

#pragma unroll
      for ( int c=0; c<eN; c += CS )
        {
          // This thread along with CS-1 of its neighbors load CS
          // components of an input vector.
          //
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * eN + c + thd_c_offset ];

          // Copy input vector components from shared to local address space.
          //
          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          // Using the CS input vector components, compute part
          // of our NL output vector components.
          for ( int rr=0; rr<NL; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }

#pragma unroll
      // Write output vector components to global memory.
      for ( int rr=0; rr<NL; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          d_app.d_out[ ( hb + thd_v_offset ) * eN + r ] = vout[rr];
        }
    }
}

// Instantiate versions for 16 by 16 and 32 by 32 matrices.
extern "C" __global__ void mxv_sh_ochunk_64(){ mxv_sh_ochunk<64>(); }
extern "C" __global__ void mxv_sh_ochunk_32(){ mxv_sh_ochunk<32>(); }
extern "C" __global__ void mxv_sh_ochunk_16(){ mxv_sh_ochunk<16>(); }


extern "C" __global__ void
mxv_sh_ochunk_mn()
{
  /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
  //  Instead, modify mxv_sh_ochunk_sol_mn.

  // Matrix size is in d_app structure. d_app.n by d_app.n.

  const int CS = 8;  // Chunk Size: Number of input vector components to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = thd_c_offset;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int n = d_app.n;

  // Maximum number of output vector components written per thread,
  // based on maximum vector size, N.
  const int NL = N / CS;
  // Number of output vector components written per thread based on
  // vector size, n.
  const int nl =  n / CS;

  /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
  //  Instead, modify mxv_sh_ochunk_sol_mn.

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      // Initialize output vector components to zero. Note that the
      // code initializes for the maximum number of components
      // assigned to this thread, NL, not the actual number, nl or
      // nl-1.
      Elt_Type vout[NL];
      for ( auto& v: vout ) v = 0;

      for ( int c=0; c<n; c += CS )
        {
          // This thread along with CS-1 of its neighbors load CS
          // components of an input vector.
          //
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * n + c + thd_c_offset ];

          // Copy input vector components from shared to local address space.
          //
          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          // Using the CS input vector components, compute part
          // of our nl output vector components.
          for ( int rr=0; rr<nl; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }

      /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
      //  Instead, modify mxv_sh_ochunk_sol_mn.

      // Write output vector components to global memory.
      for ( int rr=0; rr<nl; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          d_app.d_out[ ( hb + thd_v_offset ) * n + r ] = vout[rr];
        }
    }
  /// DO NOT MODIFY THIS ROUTINE. USE IT FOR COMPARISON.
  //  Instead, modify mxv_sh_ochunk_sol_mn.
}


extern "C" __global__ void
mxv_sh_ochunk_sol_mn()
{
  /// PUT SOLUTION IN ↓↓ THIS ROUTINE ↓↓.

  // Matrix size is in d_app structure. d_app.n by d_app.n.

  const int CS = 8;  // Chunk Size: Number of input vector components to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = thd_c_offset;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int n = d_app.n;

  // Maximum number of output vector components written per thread,
  // based on maximum vector size, N.
  const int NL = N / CS;
  // Number of output vector components written per thread based on
  // vector size, n.
  const int nl =  n / CS;

  /// PUT SOLUTION IN ↕↕ THIS ROUTINE ↕↕.

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      // Initialize output vector components to zero. Note that the
      // code initializes for the maximum number of components
      // assigned to this thread, NL, not the actual number, nl or
      // nl-1.
      Elt_Type vout[NL];
      for ( auto& v: vout ) v = 0;

      for ( int c=0; c<n; c += CS )
        {
          // This thread along with CS-1 of its neighbors load CS
          // components of an input vector.
          //
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * n + c + thd_c_offset ];

          // Copy input vector components from shared to local address space.
          //
          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          // Using the CS input vector components, compute part
          // of our nl output vector components.
          for ( int rr=0; rr<nl; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }

      // Write output vector components to global memory.
      for ( int rr=0; rr<nl; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          d_app.d_out[ ( hb + thd_v_offset ) * n + r ] = vout[rr];
        }
    }

  /// PUT SOLUTION IN ↑↑ THIS ROUTINE ↑↑.
}



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);
#if 0
  info.GET_INFO(mxv_sh_ochunk_sol_mn);
#else
  info.GET_INFO(mxv_sh_ochunk_16);
  info.GET_INFO(mxv_sh_ochunk_32);
  info.GET_INFO(mxv_sh_ochunk_64);
  info.GET_INFO(mxv_sh_ochunk_sol_mn);
  info.GET_INFO(mxv_sh_ochunk_mn);
#endif

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  // When debug true: matrix is identity and for each vector v_i=i
  // (component i is set to value i).
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  if ( opt_p )
    {
      NPerf_metric_collect("gld_efficiency");
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("flop_sp_efficiency");
      NPerf_metric_collect("shared_efficiency");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  if ( false ) NPerf_metrics_off();

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * N;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * N * 32;
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = NULL;

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf
    ("Max matrix: %d x %d.  Num vectors (S): %d.   Grid size: %d blocks.\n",
         N, N, app.num_vecs, num_blocks);
  printf("Elements per thread: %.1f (4 wp) - %.1f (32 wp)\n",
         double(app.num_vecs) / ( num_blocks * 4 * 32 ),
         double(app.num_vecs) / ( num_blocks * 32 * 32 ));

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize matrix.
  //
  for ( int r=0; r<N; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  struct Shape {
    Shape(int np):n(np),h_out_check(app.num_vecs*np){};
    Shape():n(0){};
    int n;
    vector<Elt_Type> h_out_check;
  };

  vector<int> sizes = { 64, 32, 16 };
  map<int,Shape> shapes;
  for ( auto n: sizes ) shapes.emplace(n,n);

  // Compute correct answers.
  //
  for ( auto& sh: shapes )
    {
      Shape& s = sh.second;
      assert( s.n <= N );
      for ( int i=0; i<app.num_vecs; i++ )
        for ( int r=0; r<s.n; r++ )
          {
            s.h_out_check[ i * s.n + r ] = 0;
            for ( int c=0; c<s.n; c++ )
              s.h_out_check[ i * s.n + r ] +=
                app.h_in[ i * s.n + c ] * app.matrix[r][c];
          }
    }

  app.h_out_check = shapes[N].h_out_check.data();

  struct KShape {
    KShape(int knop, Shape *sp, Kernel_Info *kip):kno(knop),s(sp),ki(kip){};
    const int kno;
    Shape* const s;
    Kernel_Info* const ki;
  };
  vector<KShape> kshapes;

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      Kernel_Info* const ki = &info.ki[kernel];
      string kn = ki->name;
      // Eagerly awaiting c++20 ends_with. RHEL should have it by 2035.
      string suffix = kn.substr(kn.size()-3,3);
      if ( suffix == "_mn" )
        for ( auto& sh: shapes ) kshapes.emplace_back(kernel,&sh.second,ki);
      else if ( suffix == "_64" )
        kshapes.emplace_back(kernel,&shapes[64],ki);
      else if ( suffix == "_32" )
        kshapes.emplace_back(kernel,&shapes[32],ki);
      else if ( suffix == "_16" )
        kshapes.emplace_back(kernel,&shapes[16],ki);
    }

  double elapsed_time_s = 86400; // Reassigned to minimum run time.
  const int output_width = stdout_width_get();

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( KShape ks: kshapes )
      {
        const int kernel = ks.kno;
        Kernel_Info* const ki = ks.ki;
        hipFuncAttributes& cfa = ki->cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        app.n = ks.s ? ks.s->n : N;
        assert( app.n > 0 );

        const int64_t num_ops =   // Multiply-adds.
          int64_t(app.n) * app.n * app.num_vecs;

        // Amount of data in and out of GPU chip.
        const int64_t amt_data_bytes =
          sizeof(app.h_in[0]) * app.num_vecs * ( app.n + app.n );

        // Copy App structure to GPU.
        //
        CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

        pString msize = pStringF("(%d,%d)",app.n,app.n);

        pTable table;
        table.stream = stdout;

        for ( int wp_cnt = 0, wp_iter = wp_start;
              wp_cnt < wp_stop && ( wp_cnt = min(wp_iter,wp_stop) );
              wp_iter += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const double comp_frac =
                  1e9 * thpt_compute_gflops
                  / ( sizeof(Elt_Type) == 4 ? info.chip_sp_flops :
                      sizeof(Elt_Type) == 8 ? info.chip_dp_flops : 1 );
                const double comm_frac =
                  min(2.0,1e9 * thpt_data_gbps / info.chip_bw_Bps);

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s%s:\n", info.ki[kernel].name, msize.s);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry
                  ("I/op","%4.1f",
                   NPerf_metric_value_get("inst_executed") * 32.0 / num_ops );
                if ( opt_p )
                  {
                    table.entry
                      ("Ld eff","%5.1f%%",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("SM eff","%5.1f%%",
                       NPerf_metric_value_get("shared_efficiency"));
                    table.entry
                      ("L2rθ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("L2wθ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.entry
                      ("FP%","%5.1f%%",
                       NPerf_metric_value_get("flop_sp_efficiency"));
                  }

                const bool plot_bandwidth = false;

                table.entry("GB/s","%4.0f", thpt_data_gbps);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);

                const int max_st_len =
                  max(5, output_width - 1 - table.row_len_get() );
                pStringF fmt("%%-%ds",max_st_len);

                string util_hdr =
                  plot_bandwidth ? "Bandwidth Util" : "FP Utilization";
                const double frac = plot_bandwidth ? comm_frac : comp_frac;
                util_hdr += string(max_st_len - util_hdr.length(),'-');
                table.entry
                  (util_hdr,fmt,
                   string( size_t(max(0.0,frac*max_st_len)), '*' ),
                   pTable::pT_Left);

              } else {

              printf
                ("%-20s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops
                 );
            }

            table.row_end();

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes,
                  hipMemcpyDeviceToHost) );
            int err_count = 0;
            Elt_Type* const h_out_check =
              ks.s ? ks.s->h_out_check.data() : app.h_out_check;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<app.n; r++ )
                {
                  const int idx = i * app.n + r;

                  if ( fabs( h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf
                          ("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
      }
  }
}
