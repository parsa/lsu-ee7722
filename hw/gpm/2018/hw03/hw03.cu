#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

#define N 16
#define M 16

#if N * 1024 * 4 < 32769
#define SMALL
#endif

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.

  int m, n;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  // No local memory.
  //
  // In the inner loop use global memory accesses to access the input
  // vector elements. Hope that the compiler recognizes the repeated
  // accesses and so keeps each input vector element in a register
  // rather than reading global memory M times per input element.
  //
  // The compiler will avoid the repeated reads if it is convinced
  // that the input and output arrays don't overlap.  For the NVIDIA
  // compiler (CUDA 7.0) that seems to require declaring the array
  // pointers with the __restrict__ attributes as kernel arguments.
  //
  // Note that dout and d_app.d_out hold the same address, as do din
  // and d_app.d_in.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_i_lbuf()
{
  // Local address space for input vector.
  //
  // Use local address space to buffer entire input vector, and write
  // each output vector element as soon as its computed.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      Elt_Type vin[N];
      for ( int c=0; c<N; c++ ) vin[c] = d_app.d_in[ h * N + c ];

      for ( int r=0; r<M; r++ )
        {
          Elt_Type elt = 0;
          for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * vin[c];
          d_app.d_out[ h * M + r ] = elt;
        }
    }
}


extern "C" __global__ void
mxv_o_lbuf()
{
  // Local memory for output vector.
  //
  // Use local memory to buffer entire output vector.  Right after each
  // input vector element is read use it to partially compute each
  // of the M output vector elements.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h.

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ ) vout[r] = 0;

      for ( int c=0; c<N; c++ )
        {
          const Elt_Type vin = d_app.d_in[ h * N + c ];
          for ( int r=0; r<M; r++ ) vout[r] += d_app.matrix[r][c] * vin;
        }
      for ( int r=0; r<M; r++ ) d_app.d_out[ h * M + r ] = vout[ r ];
    }
}


extern "C" __global__ void
mxv_o_per_thd()
{
  // Assign one vector to M threads, each thread computes one element.
  //
  // This arrangement avoids the need for any local memory buffering,
  // results in efficient global memory writes. Global memory reads
  // are still inefficient.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid / M;  // First vector number computed by this thread.
  const int r = tid % M;      // Vector element computed by this thread.

  const int stop = d_app.num_vecs;
  const int inc = num_threads / M;

  for ( int h=start; h<stop; h += inc )
    {
      // Operate on vector number h, compute output vector element r.

      Elt_Type vout = 0;

      for ( int c=0; c<N; c++ )
        vout += d_app.matrix[r][c] * d_app.d_in[ h * N + c ];

      d_app.d_out[ h * M + r ] = vout;
    }
}


extern "C" __global__ void
mxv_sh()
{
  // Local memory for output vector, use shared memory to redistribute
  // input- and output-vector elements so that global memory reads and
  // writes are fully utilized.

  // Chunk Size: Number of vector elements to redistribute at a time.
  const int CS = 8;

  const int num_threads = blockDim.x * gridDim.x;

  // The vector number operated on by threadIdx.x 0 in the first iteration.
  //
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  // Used to compute the vector element number to read or write from
  // global memory.
  //
  const int thd_x_offset = threadIdx.x % CS;

  // Used to compute the vector number to read or write from global memory.
  //
  const int thd_x_idx_st = threadIdx.x / CS;

  const int64_t BLOCK_SIZE = blockDim.x;
  const int64_t MAX_BLOCK_SIZE = 1024;

  // Shared memory used to redistribute vector elements.
  //
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE][CS + 1];

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      // Compute matrix-vector product for vector number  hb + threadIdx.x.

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ ) vout[r] = 0;

      for ( int c=0; c<N; c += CS )
        {
          // Read and redistribute input vector elements c, c+1,
          // ... c+CS-1, then use those to compute part of the
          // matrix-vector product.

          // Read input vector elements sequentially and place them in
          // shared memory.
          //
          // The entire g loop reads CS elements of each thread's
          // input vector. Each iteration of the g loop reads CS
          // elements from BLOCK_SIZE / CS vectors.
          //
          __syncthreads();

          for ( int v=thd_x_idx_st;  v<BLOCK_SIZE; v += BLOCK_SIZE/CS )
            vxfer[v][thd_x_offset] =
              d_app.d_in[ hb * N + v * N + c + thd_x_offset ];

          // Copy the portion of the input vector just read to local
          // memory (the vin array). We hope that the compiler will
          // use registers for all values of vin.
          //
          __syncthreads();
          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ ) vin[cc] = vxfer[threadIdx.x][cc];

          // Perform the matrix-vector multiplication for the parts of
          // the input vector just read.
          //
          for ( int r=0; r<M; r++ )
            for ( int cc=0; cc<CS; cc++ )
              if ( c+cc < N ) vout[r] += d_app.matrix[r][c+cc] * vin[cc];
        }

      // Use shared memory to redistribute the output vector elements to
      // threads so that the write to global memory will be efficient.
      //
      for ( int r=0; r<M; r += CS )
        {
          __syncthreads();
          for ( int rr=0; rr<CS; rr++ ) vxfer[threadIdx.x][rr] = vout[r+rr];

          __syncthreads();
          for ( int g=0; g<CS; g++ )
            {
              const int v = g * BLOCK_SIZE / CS + thd_x_idx_st;

              // The if statement is needed of M is not a multiple of CS.
              if ( thd_x_offset + r < M )
                d_app.d_out[ hb * M + v * M + r + thd_x_offset ] =
                  vxfer[v][thd_x_offset];
            }
        }
    }
}

extern "C" __global__ void
mxv_sh_ochunk()
{
  // Compute element number to start at.
  //

  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  //  const int M = d_app.m;
  //  const int N = d_app.n;

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                if ( c+cc < N )
                  vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < M )
            d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }

    }
}

extern "C" __global__ void
mxv_sh_ochunk_orig()
{
  // Compute element number to start at.
  //

  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                if ( c+cc < N )
                  vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < M )
            d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }

    }
}

extern "C" __global__ void
mxv_vec_ld()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  // Compute element number to start at.
  //
  const int start = tid;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )
    {
      Elt_Type vin[N];
      for ( int c=0; c<N; c += 4 )
        {
          // float4 f4 = d_app.d_in_f4[ ( h * N + c ) >> 2 ];
          float4 f4 = d_app.d_in_f4[ h * ( N >> 2 ) + ( c >> 2 )];
          vin[c] = f4.x;
          vin[c+1] = f4.y;
          vin[c+2] = f4.z;
          vin[c+3] = f4.w;
        }

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ )
        {
          vout[r] = 0;
          for ( int c=0; c<N; c++ )
            vout[r] += d_app.matrix[r][c] * vin[c];
        }
      for ( int r=0; r<M; r+=4 )
        {
          float4 f4 = { vout[r], vout[r+1], vout[r+2], vout[r+3] };
          d_app.d_out_f4[ ( h * M + r ) >> 2 ] = f4;
        }
    }
}

typedef union { float f; int i; } float_int;

struct fi4 {
  union { float x; int i; };
  union { float y; int j; };
  union { float z; int k; };
  union { float w; int l; };
};

extern "C" __global__ void
mxv_vls()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int offset = threadIdx.x & 1;

  // Compute element number to start at.
  //
  const int start = tid;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 300
#undef USE_SHARED
#else
#define USE_SHARED
#endif
  __shared__ float4 v0[1024];

  for ( int h=start; h<stop; h += inc )
    {
      const int h0 = h - offset;

      __syncthreads();


      Elt_Type vin[N];
      for ( int cc=0; cc<N; cc += 8 )
        {
          const int c = cc + 4 * offset;

#if 1
          const int64_t addr =
            (int64_t) &d_app.d_in_f4[ ( h0 * N + c ) >> 2 ];

          float4 v0_o, v1_o;
          asm( "ld.global.v4.f32 {%0,%1,%2,%3}, [%4]; // Hello, world!"
               : "=f"(v0_o.x), "=f"(v0_o.y), "=f"(v0_o.z),"=f"(v0_o.w)
               : "l" ( addr ) );

          asm( "ld.global.v4.f32 {%0,%1,%2,%3}, [%4];"
               : "=f"(v1_o.x), "=f"(v1_o.y), "=f"(v1_o.z),"=f"(v1_o.w)
               : "l" ( addr + N * sizeof(v1_o.x) ) );

#else
          float4 v0_o = d_app.d_in_f4[ ( h0 * N + c ) >> 2 ];
          float4 v1_o = d_app.d_in_f4[ ( ( h0 + 1 ) * N + c ) >> 2 ];
#endif

#ifdef USE_SHARED

          v0[threadIdx.x] = offset ? v0_o : v1_o;

          float4 v_xfer = v0[threadIdx.x ^ 1];
          float4 v_03 = offset ? v_xfer : v0_o;
          float4 v_47 = offset ? v1_o : v_xfer;

#else

          union { float4 fl4; fi4 fi4; } f40x, f41x;
          f40x.fl4 = v0_o;
          f41x.fl4 = v1_o;
          fi4 f40 = f40x.fi4;
          fi4 f41 = f41x.fi4;

          fi4 fswap = offset ? f40 : f41;
          const unsigned mask = ~0;
          fswap.i = __shfl_xor_sync(mask,fswap.i,1);
          fswap.j = __shfl_xor_sync(mask,fswap.j,1);
          fswap.k = __shfl_xor_sync(mask,fswap.k,1);
          fswap.l = __shfl_xor_sync(mask,fswap.l,1);

          fi4 v_03 = offset ? fswap : f40;
          fi4 v_47 = offset ? f41   : fswap;
#endif

          vin[cc] = v_03.x;
          vin[cc+1] = v_03.y;
          vin[cc+2] = v_03.z;
          vin[cc+3] = v_03.w;
          if ( cc + 4 < N )
            {
              vin[cc+4] = v_47.x;
              vin[cc+5] = v_47.y;
              vin[cc+6] = v_47.z;
              vin[cc+7] = v_47.w;
            }
        }

      Elt_Type vbuf[8];
      const int M8 = ( M + 7 ) & ~7;
#pragma unroll
      for ( int64_t r=0; r<M8; r++ )
        {
          const int bpos = r % 8;
          Elt_Type elt = 0;
          for ( int c=0; c<N; c++ )
            elt += d_app.matrix[r][c] * vin[c];
          vbuf[bpos] = elt;
          if ( bpos == 7 )
            {
              float4 vo_0 = { vbuf[0], vbuf[1], vbuf[2], vbuf[3] };
              float4 vo_1 = { vbuf[4], vbuf[5], vbuf[6], vbuf[7] };
              v0[threadIdx.x] = offset ? vo_0 : vo_1;
              float4 v_xfer = v0[threadIdx.x^1];
              float4 v0_o = offset ? v_xfer : vo_0;
              float4 v1_o = offset ? vo_1 : v_xfer;

              const int rr = r - 7 + offset * 4;

              if ( rr < M )
                {
                  d_app.d_out_f4[ ( h0 * M + rr ) >> 2 ] = v0_o;
                  d_app.d_out_f4[ ( (h0+1) * M + rr ) >> 2 ] = v1_o;
                }

            }
        }

    }
}



#ifdef SMALL

extern "C" __global__ void
mxv_sh_easy()
{
  // Compute element number to start at.
  //
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  const int64_t BLOCK_SIZE = 1024;
#if N > M
  const int sm_stride = N;
#else
  const int sm_stride = M;
#endif
  __shared__ Elt_Type vins[BLOCK_SIZE][sm_stride];

  const int offset = threadIdx.x % N;
  const int idx_start = threadIdx.x / N;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      for ( int g=0; g<N; g++ )
        vins[ idx_start + g * BLOCK_SIZE/N ][ offset ] =
          d_app.d_in[ g * BLOCK_SIZE + hb * N + threadIdx.x ];

      __syncthreads();
      Elt_Type vin[N];
      for ( int c=0; c<N; c++ ) vin[c] = vins[ threadIdx.x ][ c ];

      Elt_Type vout[M];
      for ( int r=0; r<M; r++ )
        {
          vout[r] = 0;
          for ( int c=0; c<N; c++ )
            vout[r] += d_app.matrix[r][c] * vin[c];
        }

      __syncthreads();

      for ( int r=0; r<M; r++ ) vins[ threadIdx.x ][ r ] = vout[ r ];

      __syncthreads();

      for ( int r=0; r<M; r++ )
        d_app.d_out[ hb * M + r * BLOCK_SIZE + threadIdx.x ] =
          vins[ threadIdx.x / M + r * BLOCK_SIZE/M ] [ threadIdx.x % M ];
    }
}
#endif



GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_i_lbuf);
  info.GET_INFO(mxv_o_lbuf);
  info.GET_INFO(mxv_o_per_thd);

#if N / 4 == (N+3)/4
  info.GET_INFO(mxv_vec_ld);
#endif
#if N / 4 == (N+3)/4 && M / 4 == (M+3)/4
  info.GET_INFO(mxv_vls);
#endif
  info.GET_INFO(mxv_sh);
  info.GET_INFO(mxv_sh_ochunk);

#ifdef SMALL
  info.GET_INFO(mxv_sh_easy);
#endif

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("gld_efficiency");
  if ( opt_p )
    {
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( !opt_p && vary_warps )
    NPerf_metrics_off();

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * M;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = new Elt_Type[ out_size_elts ];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize transformation matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  // Compute correct answer.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int r=0; r<M; r++ )
      {
        app.h_out_check[ i * M + r ] = 0;
        for ( int c=0; c<N; c++ )
          app.h_out_check[ i * M + r ] +=
            app.h_in[ i * N + c ] * app.matrix[r][c];
      }

  const int64_t num_ops = int64_t(M) * N * app.num_vecs;  // Multiply-adds.

  // Amount of data in and out of GPU chip.
  const int64_t amt_data_bytes = in_size_bytes + out_size_bytes;

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Copy App structure to GPU.
    //
    CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
      {
        hipFuncAttributes& cfa = info.ki[kernel].cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        pTable table;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double comp_frac = 
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double comm_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s:\n", info.ki[kernel].name);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                if ( opt_p )
                  {
                    table.entry
                      ("Ld eff","%5.1f%%",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("L2rθ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("L2wθ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.entry
                      ("DRrθ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("DRwθ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                  }

                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len = 79 - table.row_len_get();
                pStringF fmt("%%-%ds",max_st_len);

                string bw_util_hdr = "Bandwidth Util";
                bw_util_hdr += string(max_st_len - bw_util_hdr.length(),'-');
                table.entry
                  (bw_util_hdr,fmt,
                   &stars[stars_len-int(comm_frac*max_st_len)],
                   pTable::pT_Left);

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<M; r++ )
                {
                  const int idx = i * M + r;

                  if ( fabs( app.h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], app.h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
        printf("%s",table.body_get());
      }
  }

}
