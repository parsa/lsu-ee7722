#include "hip/hip_runtime.h"
/// LSU EE 7722 GPU Microarchitecture
//

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <nperf.h>
#include "util.h"
#include <ptable.h>

#define N 16
#define M 16

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct App
{
  // Number of input and output vectors, that is, the size of the
  // input and output arrays.
  //
  int num_vecs;

  Elt_Type matrix[M][N];

  // Host pointers to the input and output arrays, and to a CPU-computed
  // output array used for checking results.
  //
  Elt_Type *h_in, *h_out, *h_out_check;
  //
  // Note: h_in points to an array holding num_vecs N-element vectors,
  //       and so the total size of h_in is num_vects * N elements.

  // GPU pointers to the input and output arrays.
  //
  Elt_Type *d_in, *d_out;

  // GPU pointers to the input and output arrays, cast to float4s.
  //
  float4 *d_in_f4, *d_out_f4;
  //
  // The compiler can emit more efficient load and store instructions
  // to float4 elements than to four consecutive floats.
  //
  // Note: These "_f4" pointers only work when Elt_Type is a float.

  int m, n;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

typedef void (*KPtr)(Elt_Type *dout, const Elt_Type *din);


extern "C" __global__ void
mxv_g_only(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  // No local memory.
  //
  // In the inner loop use global memory accesses to access the input
  // vector elements. Hope that the compiler recognizes the repeated
  // accesses and so keeps each input vector element in a register
  // rather than reading global memory M times per input element.
  //
  // The compiler will avoid the repeated reads if it is convinced
  // that the input and output arrays don't overlap.  For the NVIDIA
  // compiler (CUDA 7.0) that seems to require declaring the array
  // pointers with the __restrict__ attributes as kernel arguments.
  //
  // Note that dout and d_app.d_out hold the same address, as do din
  // and d_app.d_in.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;        // First vector number computed by this thread.
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  for ( int h=start; h<stop; h += inc )

    // Operate on vector number h.

    for ( int r=0; r<M; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<N; c++ ) elt += d_app.matrix[r][c] * din[ h * N + c ];
        dout[ h * M + r ] = elt;
      }
}

extern "C" __global__ void
mxv_g_only_mn(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  const int m = d_app.m;
  const int n = d_app.n;

  for ( int h=start; h<stop; h += inc )

    for ( int r=0; r<m; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<n; c++ ) elt += d_app.matrix[r][c] * din[ h * n + c ];
        dout[ h * m + r ] = elt;
      }
}


extern "C" __global__ void
mxv_g_only_sol_mn
(Elt_Type* __restrict__ dout, const Elt_Type* __restrict__ din)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = blockDim.x * gridDim.x;

  const int start = tid;
  const int stop = d_app.num_vecs;
  const int inc = num_threads;

  const int m = d_app.m;
  const int n = d_app.n;

  for ( int h=start; h<stop; h += inc )

    for ( int r=0; r<m; r++ )
      {
        Elt_Type elt = 0;
        for ( int c=0; c<n; c++ ) elt += d_app.matrix[r][c] * din[ h * n + c ];
        dout[ h * m + r ] = elt;
      }
}


extern "C" __global__ void
mxv_sh_ochunk_mn()
{
  // Compute element number to start at.
  //

  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int m = d_app.m;
  const int n = d_app.n;

  const int ML = ( M + CS - 1 ) / CS;
  const int ml = ( m + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ml; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<n; c += CS )
        {
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * n + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ml; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                if ( c+cc < n )
                  vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ml; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < m )
            d_app.d_out[ ( hb + thd_v_offset ) * m + r ] = vout[rr];
        }

    }
}

extern "C" __global__ void
mxv_sh_ochunk()
{
  // Compute element number to start at.
  //

  const int CS = 8;  // Chunk Size: Number of input vector elts to read.
  const int num_threads = blockDim.x * gridDim.x;

  // First element used by this block.
  const int bl_start = blockIdx.x * blockDim.x / CS;
  const int stop = d_app.num_vecs;
  const int inc = num_threads / CS;

  const int thd_c_offset = threadIdx.x % CS;
  const int thd_r_offset = threadIdx.x % CS;
  const int thd_v_offset = threadIdx.x / CS;

  const int MAX_BLOCK_SIZE = 1024;
  __shared__ Elt_Type vxfer[MAX_BLOCK_SIZE];

  const int ML = ( M + CS - 1 ) / CS;

  for ( int hb = bl_start; hb<stop; hb += inc )
    {
      Elt_Type vout[ML];
      for ( int rl=0; rl<ML; rl++ ) vout[rl] = 0;

#pragma unroll
      for ( int c=0; c<N; c += CS )
        {
          vxfer[threadIdx.x] =
            d_app.d_in[ ( hb + thd_v_offset ) * N + c + thd_c_offset ];

          Elt_Type vin[CS];
          for ( int cc=0; cc<CS; cc++ )
            vin[cc] = vxfer[ thd_v_offset * CS + cc ];

          for ( int rr=0; rr<ML; rr++ )
            {
              const int r = rr * CS + thd_r_offset;
              for ( int cc=0; cc<CS; cc++ )
                if ( c+cc < N )
                  vout[rr] += d_app.matrix[r][c+cc] * vin[cc];
            }
        }
#pragma unroll
      for ( int rr=0; rr<ML; rr++ )
        {
          const int r = rr * CS + thd_r_offset;
          if ( r < M )
            d_app.d_out[ ( hb + thd_v_offset ) * M + r ] = vout[rr];
        }

    }
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Determine which GPU to use. (For starters, if there's more than
  // one, choose the one connected to the display.)
  //
  int dev = gpu_choose_index();
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(mxv_g_only);
  info.GET_INFO(mxv_g_only_mn);
  info.GET_INFO(mxv_g_only_sol_mn);

  info.GET_INFO(mxv_sh_ochunk);
  info.GET_INFO(mxv_sh_ochunk_mn);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


int
main(int argc, char **argv)
{
  const bool debug = false;

  // Must be called before any CUDA API calls.
  NPerf_init();

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_and_kernel_info();

  const int num_mp = info.cuda_prop.multiProcessorCount;

  // Examine argument 1, block count, default is number of MPs.
  //
  const int arg1_int = argc < 2 ? num_mp : atoi(argv[1]);
  const int num_blocks =
     arg1_int == 0 ? num_mp :
     arg1_int < 0  ? -arg1_int * num_mp : arg1_int;

  // Examine argument 2, number of threads per block.
  //
  const bool opt_p = argc >= 3 && string(argv[2]) == "p";
  const int thd_per_block_arg = argc < 3 ? 1024 : opt_p ? 0 : atoi(argv[2]);
  const int thd_per_block_goal =
   thd_per_block_arg == 0 ? 1024 : thd_per_block_arg;
  const int num_threads = num_blocks * thd_per_block_goal;

  const bool vary_warps = thd_per_block_arg == 0;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.num_vecs = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( num_threads <= 0 || app.num_vecs <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [THD_PER_BLOCK|p] "
             "[DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Collect performance data using a wrapper to NVIDIA CUPTI event
  // counter API.
  //
  NPerf_metric_collect("inst_executed");
  NPerf_metric_collect("gld_efficiency");
  if ( opt_p )
    {
      NPerf_metric_collect("l2_read_throughput");
      NPerf_metric_collect("l2_write_throughput");
      NPerf_metric_collect("dram_read_throughput");
      NPerf_metric_collect("dram_write_throughput");
    }
  //
  // Note: The more metrics that are collected, the more times a kernel
  // will need to be run.

  // Don't collect performance data if we are varying warps. Why?
  // Because it takes too long.
  if ( !opt_p && vary_warps )
    NPerf_metrics_off();

  app.m = M;
  app.n = N;

  const size_t in_size_elts = size_t(app.num_vecs) * N;
  const size_t in_size_bytes = in_size_elts * sizeof( app.h_in[0] );
  const size_t out_size_elts = size_t(app.num_vecs) * M;
  const size_t out_size_bytes = out_size_elts * sizeof( app.h_out[0] );

  const int overrun_size_elts = thd_per_block_goal * max(N,M);
  const int overrun_size_bytes = overrun_size_elts * sizeof( app.h_out[0] );

  // Allocate storage for CPU copy of data.
  //
  app.h_in = new Elt_Type[ in_size_elts ];
  app.h_out = new Elt_Type[ out_size_elts ];
  app.h_out_check = NULL;

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_in,  in_size_bytes + overrun_size_bytes ) );
  app.d_in_f4 = (float4*) app.d_in;
  CE( hipMalloc( &app.d_out, out_size_bytes + overrun_size_bytes ) );
  app.d_out_f4 = (float4*) app.d_out;

  printf("Matrix size: %d x %d.  Vectors: %d.   %d blocks of %d thds.\n",
         N, M, app.num_vecs, num_blocks, thd_per_block_goal);

  // Initialize input array.
  //
  for ( int i=0; i<app.num_vecs; i++ )
    for ( int c=0; c<N; c++ )
      app.h_in[ i * N + c ] = debug ? Elt_Type(c) : drand48();

  // Initialize matrix.
  //
  for ( int r=0; r<M; r++ )
    for ( int c=0; c<N; c++ )
      app.matrix[r][c] = debug ? r == c : drand48();

  struct Shape {
    Shape(int np, int mp):n(np),m(mp),h_out_check(app.num_vecs*mp){};
    int n, m;
    vector<Elt_Type> h_out_check;
  };

  vector<Shape> sizes = { {N,M}, {N/2,M} };

  // Compute correct answers.
  //
  for ( Shape& s: sizes )
    for ( int i=0; i<app.num_vecs; i++ )
      for ( int r=0; r<s.m; r++ )
        {
          s.h_out_check[ i * s.m + r ] = 0;
          for ( int c=0; c<s.n; c++ )
            s.h_out_check[ i * s.m + r ] +=
              app.h_in[ i * s.n + c ] * app.matrix[r][c];
        }

  app.h_out_check = sizes[0].h_out_check.data();

  struct KShape {
    KShape(int knop, Shape *sp, Kernel_Info *kip):kno(knop),s(sp),ki(kip){};
    const int kno;
    Shape* const s;
    Kernel_Info* const ki;
  };
  vector<KShape> kshapes;

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      Kernel_Info* const ki = &info.ki[kernel];
      string kn = ki->name;
      // Eagerly awaiting c++20 ends_with. RHEL should have it by 2035.
      if ( kn.substr(kn.size()-3,3) == "_mn" )
        for ( auto& s: sizes ) kshapes.emplace_back(kernel,&s,ki);
      else
        kshapes.emplace_back(kernel,(Shape*)NULL,ki);
    }

  double elapsed_time_s = 86400; // Reassigned to minimum run time.

  {
    // Prepare events used for timing.
    //
    hipEvent_t gpu_start_ce, gpu_stop_ce;
    CE(hipEventCreate(&gpu_start_ce));
    CE(hipEventCreate(&gpu_stop_ce));

    // Copy input array from CPU to GPU.
    //
    CE( hipMemcpy
        ( app.d_in, app.h_in, in_size_bytes, hipMemcpyHostToDevice ) );

    // Launch kernel multiple times and keep track of the best time.
    printf("Launching with %d blocks of up to %d threads. \n",
           num_blocks, thd_per_block_goal);

    for ( KShape ks: kshapes )
      {
        const int kernel = ks.kno;
        Kernel_Info* const ki = ks.ki;
        hipFuncAttributes& cfa = ki->cfa;
        const int wp_limit = cfa.maxThreadsPerBlock >> 5;

        const int thd_limit = wp_limit << 5;
        const int thd_per_block_no_vary = min(thd_per_block_goal,thd_limit);

        const int wp_start = 4;
        const int wp_stop = vary_warps ? wp_limit : wp_start;
        const int wp_inc = 4;

        if ( ks.s ) { app.m = ks.s->m; app.n = ks.s->n; }
        else        { app.m = M;       app.n = N;       }

        const int64_t num_ops =   // Multiply-adds.
          int64_t(app.m) * app.n * app.num_vecs;

        // Amount of data in and out of GPU chip.
        const int64_t amt_data_bytes =
          sizeof(app.h_in[0]) * app.num_vecs * ( app.m + app.n );

        // Copy App structure to GPU.
        //
        CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

        pString msize = pStringF("(%d,%d)",app.m,app.n);

        pTable table;

        for ( int wp_cnt = wp_start; wp_cnt <= wp_stop; wp_cnt += wp_inc )
          {
            const int thd_per_block =
              vary_warps ? wp_cnt << 5 : thd_per_block_no_vary;

            // Zero the output array.
            //
            CE(hipMemset(app.d_out,0,out_size_bytes));

            // Measure execution time starting "now", which is after data
            // set to GPU.
            //
            CE(hipEventRecord(gpu_start_ce,0));

            // Launch Kernel
            //
            for ( NPerf_data_reset(); NPerf_need_run_get(); )
              KPtr(info.ki[kernel].func_ptr)<<<num_blocks,thd_per_block>>>
                (app.d_out,app.d_in);

            // Stop measuring execution time now, which is before is data
            // returned from GPU.
            //
            CE(hipEventRecord(gpu_stop_ce,0));
            CE(hipEventSynchronize(gpu_stop_ce));
            float cuda_time_ms = -1.1;
            CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));

            const double this_elapsed_time_s =
              NPerf_metrics_collection_get()
              ? NPerf_kernel_et_get() : cuda_time_ms * 0.001;

            const double thpt_compute_gflops =
              num_ops / this_elapsed_time_s * 1e-9;
            const double thpt_data_gbps =
              amt_data_bytes / this_elapsed_time_s * 1e-9;

            if ( vary_warps )
              {
                const char* const stars = "********************************************************************************";
                const int stars_len = 80;
                const double __attribute__((unused)) comp_frac = 
                  4e9 * thpt_compute_gflops / info.chip_sp_flops;
                const double comm_frac =
                  1e9 * thpt_data_gbps / info.chip_bw_Bps;

                // Number of warps, rounded up.
                //
                const int num_wps = ( thd_per_block + 31 ) >> 5;

                // The maximum number of active blocks per MP for this
                // kernel when launched with a block size of thd_per_block.
                //
                const int max_bl_per_mp =
                  info.get_max_active_blocks_per_mp(kernel,thd_per_block);

                // Compute number of blocks available per MP based only on
                // the number of blocks.  This may be larger than the
                // number of blocks that can run.
                //
                const int bl_per_mp_available =
                  0.999 + double(num_blocks) / num_mp;

                // The number of active blocks is the minimum of what
                // can fit and how many are available.
                //
                const int bl_per_mp =
                  min( bl_per_mp_available, max_bl_per_mp );

                // Based on the number of blocks, compute the num ber of warps.
                //
                const int act_wps = num_wps * bl_per_mp;

                if ( wp_cnt == wp_start )
                  printf("Kernel %s%s:\n", info.ki[kernel].name, msize.s);

                table.row_start();
                table.entry("wp",num_wps);
                table.entry("ac",act_wps);
                table.entry("t/µs","%6.0f", this_elapsed_time_s * 1e6);
                table.entry("FP θ","%4.0f", thpt_compute_gflops);
                if ( opt_p )
                  {
                    table.entry
                      ("Ld eff","%5.1f%%",
                       NPerf_metric_value_get("gld_efficiency"));
                    table.entry
                      ("L2rθ","%5.1f",
                       NPerf_metric_value_get("l2_read_throughput") * 1e-9 );
                    table.entry
                      ("L2wθ","%5.1f",
                       NPerf_metric_value_get("l2_write_throughput") * 1e-9 );
                    table.entry
                      ("DRrθ","%5.1f",
                       NPerf_metric_value_get("dram_read_throughput") * 1e-9 );
                    table.entry
                      ("DRwθ","%5.1f",
                       NPerf_metric_value_get("dram_write_throughput") * 1e-9 );
                  }

                table.entry("GB/s","%4.0f", thpt_data_gbps);

                const int max_st_len = 79 - table.row_len_get();
                pStringF fmt("%%-%ds",max_st_len);

                string bw_util_hdr = "Bandwidth Util";
                bw_util_hdr += string(max_st_len - bw_util_hdr.length(),'-');
                table.entry
                  (bw_util_hdr,fmt,
                   &stars[max(0,stars_len-int(comm_frac*max_st_len))],
                   pTable::pT_Left);

              } else {

              printf
                ("%-15s %2d wp  %7.0f µs  %8.3f GF  %8.3f GB/s  "
                 "%5.2f I/F  %5.1f%%\n",
                 info.ki[kernel].name,
                 (thd_per_block + 31 ) >> 5,
                 this_elapsed_time_s * 1e6,
                 thpt_compute_gflops, thpt_data_gbps,
                 NPerf_metric_value_get("inst_executed") * 32 / num_ops,
                 NPerf_metric_value_get("gld_efficiency")
                 );
            }

            elapsed_time_s = min(this_elapsed_time_s,elapsed_time_s);

            // Copy output array from GPU to CPU.
            //
            CE( hipMemcpy
                ( app.h_out, app.d_out, out_size_bytes, hipMemcpyDeviceToHost) );
            int err_count = 0;
            Elt_Type* const h_out_check =
              ks.s ? ks.s->h_out_check.data() : app.h_out_check;
            for ( int i=0; i<app.num_vecs; i++ )
              for ( int r=0; r<app.m; r++ )
                {
                  const int idx = i * app.m + r;

                  if ( fabs( h_out_check[idx] - app.h_out[idx] ) > 1e-5 )
                    {
                      err_count++;
                      if ( err_count < 5 )
                        printf("Error at vec %d elt %d: %.7f != %.7f (correct)\n",
                               i, r, app.h_out[idx], h_out_check[idx] );
                    }
                }
            if ( err_count )
              printf("Total errors %d\n", err_count);
          }
        printf("%s",table.body_get());
      }
  }
}
