#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2013), GPU Programming
//
 /// Homework 3
 //

 /// Most of the solution should be in this file.


 /// Instructions
 //
 //  Read the assignment: http://www.ece.lsu.edu/koppel/gpup/2013/hw03.pdf



__host__ void
launch_data_pack(int grid_size, int block_size)
{
  data_pack<<<grid_size,block_size>>>();
}

__global__ void
data_pack()
{
  const int bi = threadIdx.x + blockIdx.x * blockDim.x;
  if ( bi >= dc.chain_length ) return;
  if ( !dc.d_pos ) return;

  /// Put Solution Code Here.

}

__host__ void
launch_data_unpack(int grid_size, int block_size)
{
  data_unpack<<<grid_size,block_size>>>();
}

__global__ void
data_unpack()
{
  const int bi = threadIdx.x + blockIdx.x * blockDim.x;
  if ( bi >= dc.chain_length ) return;
  if ( !dc.d_pos ) return;

  /// Put Solution Code Here.


}


__host__ void
launch_time_step_gpu_sol_part_1
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_sol_part_1<<<grid_size,block_size>>>(delta_t);
}

__global__ void
time_step_gpu_sol_part_1(float delta_t)
{
  /// Modify this routine.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int bi = tid; // Ball Index

  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  // Convenient zero vector.
  const float3 zero_vec = make_float3(0,0,0);

  if ( dc.opt_head_lock && bi == 0
       || dc.opt_tail_lock && bi == dc.chain_length - 1 )
    {
      ball->velocity = zero_vec;
      return;
    }

  // Load the ball position and radius from memory. They will be used
  // several times. (They are loaded into registers.)
  //
  const float4 ball_position = ball->position;
  const float ball_radius = ball->radius;

  float3 force = zero_vec;

  // Gravitational Force
  //
  force += ball->mass * dc.gravity_accel;

  // Spring Force from Neighbor Balls
  //
  for ( int j = -3;  j < 4;  j++ )
    {
      const int n_idx = bi + j;
      if ( j == 0 ) continue;
      if ( n_idx < 0 ) continue;
      if ( n_idx >= dc.chain_length ) continue;
      const int idx_dist = abs(j);
      const bool rep_only = idx_dist > 1;

      Ball* const neighbor_ball = &dc.d_balls[n_idx];

      // Construct a normalized (Unit) Vector from ball to neighbor.
      //
      pNorm ball_to_neighbor = mn(ball_position,neighbor_ball->position);

      // Compute the speed of ball towards neighbor_ball.
      //
      pVect delta_v = neighbor_ball->velocity - ball->velocity;
      float delta_s = dot( delta_v, ball_to_neighbor );

      // Compute distance between us and neighbor at which "spring"
      // will be relaxed (neither stretched nor compressed).
      //
      const float distance_relaxed =
        rep_only ? idx_dist * dc.distance_relaxed : dc.distance_relaxed;

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch =
        ball_to_neighbor.magnitude - distance_relaxed;

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0f ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? dc.opt_spring_constant : dc.opt_spring_constant * 0.7f;

      if ( !rep_only || spring_stretch < 0 )
        force += spring_constant * spring_stretch * ball_to_neighbor;
    }

  // Check for contact between us and other balls, except our immediate
  // neighbors.
  //
  for ( int j=0; j<dc.chain_length; j++ )
    if ( abs(j-bi)>1 )
      {
        Ball* const ball2 = &dc.d_balls[j];
        pNorm ball_to_2 = mn(ball_position,ball2->position);
        const float rsum = ball_radius + ball2->radius;
        if ( rsum * rsum > ball_to_2.mag_sq )
          {
            const float compression = rsum - ball_to_2.magnitude;
            const float f = -compression * 1000;
            force += f * ball_to_2;
          }
      }

  // Update Velocity
  //
  // This code assumes that force on ball is constant over time
  // step. This is clearly wrong when balls are moving with
  // respect to each other because the springs are changing
  // length. This inaccuracy will make the simulation unstable
  // when spring constant is large for the time step.
  //
  ball->velocity += delta_t * ball->mass_inv * force;

  // Air Resistance
  //
  const double fs = powf(1+dc.opt_air_resistance,-delta_t);
  ball->velocity *= fs;
}


__host__ void
launch_time_step_gpu_sol_part_2
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_sol_part_2<<<grid_size,block_size>>>(delta_t);
}

__global__ void
time_step_gpu_sol_part_2(float delta_t)
{
  /// Modify this routine.

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int bi = tid; // Ball Index

  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  ///
  /// Update Position of Each Ball
  ///

  // Update Position
  //
  // Assume that velocity is constant.
  //
  ball->position += ball->velocity * delta_t;

  // Possible Collision with Platform
  //

  // Skip if collision impossible.
  //
  if ( !platform_collision_possible(ball->position) ) return;
  if ( ball->position.y >= ball->radius ) return;
  if ( ball->position.y < 0 ) return;

  // Apply spring forces due to collision with platform.
  //
  const float spr_const = ball->velocity.y > 0 ? 500000 : 1000000;
  ball->velocity.y += ( ball->radius - ball->position.y ) * delta_t * spr_const;
}
