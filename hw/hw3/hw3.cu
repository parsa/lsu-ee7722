#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Spring 2013), GPU Microarchitecture
//

 /// Homework 3
 //
 // Assignment in: http://www.ece.lsu.edu/koppel/gp/2013/hw03.pdf
 //
 /// Your Name:

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>

#define N 4

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(16) Vertex
{
  Elt_Type a[N];
};

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  bool find_minimum_magnitude; // For problem 2.
  Vertex *v_in, *v_out;
  Vertex *d_v_in, *d_v_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;

// The entry point for the GPU code.
//
__global__ void
cuda_thread_start()
{
  // Compute an id number that will be in the range from 0 to num_threads-1.
  //
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Number of elements that each thread should work on.  We are ignoring
  // rounding errors.
  //
  const int elt_per_thread = d_app.array_size / d_app.num_threads;

  /// WARNING:
  //
  // The order in which the threads examine elements here is poorly
  // chosen and will unnecessarily make inefficient use of the memory
  // system.

  // Compute element number to start at.
  //
  const int start = elt_per_thread * tid;

  // Compute element number to stop at.
  //
  const int stop = start + elt_per_thread;

  // WARNING: This code accesses elements in an inefficient order.
  for ( int h=start; h<stop; h++ )
    {
      Vertex p = d_app.d_v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
        }
      d_app.d_v_out[h] = q;
    }
}

void
print_gpu_info()
{
  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  hipGetDeviceCount(&device_count); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  int dev = 0;
  CE(hipGetDevice(&dev));
  printf("Using GPU %d\n",dev);

  hipDeviceProp_t cuda_prop;  // Properties of cuda device (GPU, cuda version).

  /// Print information about the available GPUs.
  //
  {
    CE(hipGetDeviceProperties(&cuda_prop,dev));
    printf
      ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
       dev, cuda_prop.name, cuda_prop.clockRate/1e6,
       int(cuda_prop.totalGlobalMem >> 20));

    printf
      ("GPU %d: CC: %d.%d  MP: %2d  TH/WP: %3d  TH/BL: %4d\n",
       dev, cuda_prop.major, cuda_prop.minor,
       cuda_prop.multiProcessorCount,
       cuda_prop.warpSize,
       cuda_prop.maxThreadsPerBlock
       );

    printf
      ("GPU %d: SHARED: %5d  CONST: %5d  # REGS: %5d\n",
       dev,
       int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
       cuda_prop.regsPerBlock
       );
  }

  hipFuncAttributes cfa_prob1; // Properties of code to run on device.
  CE( hipFuncGetAttributes(&cfa_prob1,reinterpret_cast<const void*>(cuda_thread_start)) );

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");
  printf(" Our CUDA Thread: %6zd shared, %zd const, %zd loc, %d regs; "
         "%d max threads per block.\n",
         cfa_prob1.sharedSizeBytes,
         cfa_prob1.constSizeBytes,
         cfa_prob1.localSizeBytes,
         cfa_prob1.numRegs,
         cfa_prob1.maxThreadsPerBlock);
}

void*
pt_thread_start(void *arg)
{
  const int tid = (ptrdiff_t) arg;
  printf("Hello from %d\n",tid);
  const int elt_per_thread = app.array_size / app.num_threads;
  const int start = elt_per_thread * tid;
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      Vertex p = app.v_in[h];
      Vertex q;
      for ( int i=0; i<N; i++ )
        {
          q.a[i] = 0;
          for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
        }
      app.v_out[h] = q;
    }

  return NULL;
}

int
main(int argc, char **argv)
{
  // Examine argument 1, block size, if negative, find minimum magnitude.
  //
  const int arg1_int = argc < 2 ? 1 : atoi(argv[1]);
  const bool find_mag = arg1_int < 0;
  const int num_blocks = abs(arg1_int);

  const bool use_pthreads = false;

  // For Problem 2.
  app.find_minimum_magnitude = find_mag;

  // Examine argument 2, number of threads per block.
  //
  const int thd_per_block = argc < 3 ? 1 : atoi(argv[2]);
  app.num_threads = use_pthreads ? -arg1_int : num_blocks * thd_per_block;

  // Examine argument 3, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 4 ? 1 << 20 : int( atof(argv[3]) * (1<<20) );

  if ( app.num_threads <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_PTHREADS | - NUM_CUDA_BLOCKS ] [THD_PER_BLOCK] [DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  if ( !use_pthreads )
    print_gpu_info();

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );

  printf
    ("\nPreparing for %d %s threads operating on %d vectors of %d elements.\n",
         app.num_threads,
         use_pthreads ? "CPU" : "GPU",
         app.array_size, N);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  double elapsed_time_s;
  int minimum_mag_index = 0;     // For Problem 2.
  Elt_Type minimum_mag_val = 0;  // For Problem 2.

  if ( use_pthreads )
    {
      const double time_start = time_fp();

      // Allocate a structure to hold pthread thread ids.
      //
      pthread_t* const ptid = new pthread_t[app.num_threads];

      // Set up a pthread attribute, used for specifying options.
      //
      pthread_attr_t attr;
      pthread_attr_init(&attr);
      pthread_attr_setscope(&attr, PTHREAD_SCOPE_SYSTEM);

      // Launch the threads.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_create(&ptid[i], &attr, pt_thread_start, (void*)i);

      // Wait for each thread to finish.
      //
      for ( int i=0; i<app.num_threads; i++ )
        pthread_join( ptid[i], NULL );

      elapsed_time_s = time_fp() - time_start;
    }
  else
    {
      // Prepare events used for timing.
      //
      hipEvent_t gpu_start_ce, gpu_stop_ce;
      CE(hipEventCreate(&gpu_start_ce));
      CE(hipEventCreate(&gpu_stop_ce));

      // Copy input array from CPU to GPU.
      //
      CE( hipMemcpy
          ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

      // Copy App structure to GPU.
      //
      CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

      // Measure execution time starting "now", which is after data
      // set to GPU.
      //
      CE(hipEventRecord(gpu_start_ce,0));

      printf("Launching with %d blocks of %d threads.\n",
             num_blocks, thd_per_block);

      // Tell CUDA to start our threads on the GPU.
      //
      cuda_thread_start<<<num_blocks,thd_per_block>>>();

      // Stop measuring execution time now, which is before is data
      // returned from GPU.
      //
      CE(hipEventRecord(gpu_stop_ce,0));
      CE(hipEventSynchronize(gpu_stop_ce));
      float cuda_time_ms = -1.1;
      CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
      elapsed_time_s = cuda_time_ms * 0.001;

      // Copy output array from GPU to CPU.
      //
      CE( hipMemcpy
          ( app.v_out, app.d_v_out, array_size_bytes, hipMemcpyDeviceToHost) );

      // PROBLEM 2
      //
      // Insert code for reading magnitude information and having
      // CPU finish up finding the minimum.

      minimum_mag_index = -1;  // PROBLEM 2 - change this line.
      minimum_mag_val = 0.0;   // PROBLEM 2 - change this line too.
    }

  const double data_size = app.array_size * sizeof(Vertex) * 2;
  const double fp_op_count = app.array_size * ( 2 * N * N - N  );

  printf("Elapsed time for %d threads and %d elements is %.3f µs\n",
         app.num_threads, app.array_size, 1e6 * elapsed_time_s);
  printf("Rate %.3f GFLOPS,  %.3f GB/s\n",
         1e-9 * fp_op_count / elapsed_time_s,
         1e-9 * data_size / elapsed_time_s);

  if ( app.find_minimum_magnitude )
    {
      // Compute correct answer.
      Elt_Type min_val = 0;
      int min_idx = -1;

      for ( int h=0; h<app.array_size; h++ )
        {
          Vertex p = app.v_in[h];
          Vertex q;
          for ( int i=0; i<N; i++ )
            {
              q.a[i] = 0;
              for ( int j=0; j<N; j++ ) q.a[i] += app.matrix[i][j] * p.a[j];
            }
          Elt_Type sos = 0; for(int i=0; i<N; i++ ) sos+= q.a[i]*q.a[i];
          Elt_Type mag = sqrt(sos);
          if ( min_idx < 0 || mag < min_val ) { min_val = mag; min_idx = h; }
        }
      Elt_Type diff = fabs(min_val-minimum_mag_val);
      printf
        ("\nMinimum mag is %s,  %d %s %d (correct)  %.4f %s %.4f (correct)\n",
         diff < 1e-5 ? "correct" : "**wrong**",
         minimum_mag_index,
         min_idx == minimum_mag_index ? "==" : "!=",
         min_idx,
         minimum_mag_val,
         min_val == minimum_mag_val ? "==" : diff < 1e-5 ? "~" : "!=",
         min_val
         );

    }

}
