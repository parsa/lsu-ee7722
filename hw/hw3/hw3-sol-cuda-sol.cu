#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2013), GPU Programming
//
 /// Homework 3  -- SOLUTION
 //

 /// Most of the solution is in this file.

 //  See the assignment: http://www.ece.lsu.edu/koppel/gpup/2013/hw03.pdf


__host__ void
launch_data_pack(int grid_size, int block_size)
{
  data_pack<<<grid_size,block_size>>>();
}

__global__ void
data_pack()
{
  const int bi = threadIdx.x + blockIdx.x * blockDim.x;
  if ( bi >= dc.chain_length ) return;
  if ( !dc.d_pos ) return;

  /// SOLUTION

  Ball* const ball = &dc.d_balls[bi];

  // Put ball position and radius into d_pos. An element of d_pos is a
  // float4. If space were the only concern it would have been a
  // float3. But GPUs can load float4 values more efficiently, and so
  // a float4 was used. The fourth element is used to hold the ball
  // radius.
  //
  dc.d_pos[bi] = ball->position;
  dc.d_pos[bi].w = ball->radius;
}

__host__ void
launch_data_unpack(int grid_size, int block_size)
{
  data_unpack<<<grid_size,block_size>>>();
}

__global__ void
data_unpack()
{
  const int bi = threadIdx.x + blockIdx.x * blockDim.x;
  if ( bi >= dc.chain_length ) return;
  if ( !dc.d_pos ) return;

  /// SOLUTION

  // Put the position data back into the ball structure. Note that the
  // pack and unpack routines are called less frequently than the time
  // step routines.
  //
  Ball* const ball = &dc.d_balls[bi];
  ball->position = dc.d_pos[bi];
  ball->position.w = 1;
}


__host__ void
launch_time_step_gpu_sol_part_1
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_sol_part_1<<<grid_size,block_size>>>(delta_t);
}

__global__ void
time_step_gpu_sol_part_1(float delta_t)
{
  /// SOLUTION

  // Based on the launch configuration (number of threads per block
  // [blockDim.x] and number of blocks [gridDim.x]) determine how many
  // balls each block will handle and the number of threads per ball.
  //
  const int balls_per_block = ( gridDim.x - 1 + dc.chain_length ) / gridDim.x;
  const int thds_per_ball = blockDim.x / balls_per_block;

  const int bi_block = blockIdx.x * balls_per_block;

  const int b_local = threadIdx.x % balls_per_block;
  const int bi = bi_block + b_local;
  const int ai_first = threadIdx.x / balls_per_block;

  if ( ai_first >= thds_per_ball ) return;
  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  // Convenient zero vector.
  const float3 zero_vec = make_float3(0,0,0);

  if ( dc.opt_head_lock && bi == 0
       || dc.opt_tail_lock && bi == dc.chain_length - 1 )
    {
      ball->velocity = zero_vec;
      return;
    }

  // Load the ball position and radius from memory. They will be used
  // several times. (They are loaded into registers.)
  //

  /// SOLUTION
  //  Load from new d_pos array.
  const float4 ball_position = dc.d_pos[bi];

  const float ball_radius = ball->radius;
  const float3 ball_velocity = ball->velocity;

  float3 force = zero_vec;

  // Gravitational Force
  //
  /// SOLUTION
  //  Only one thread per ball applies gravitational force.
  if ( ai_first == 0 )
    force += ball->mass * dc.gravity_accel;

  // Spring Force from Neighbor Balls
  //

  /// SOLUTION
  //  Divide work of applying a ball's neighbor forces between the
  //  different threads assigned to that ball.
  //
  for ( int j = ai_first - 3;  j < 4;  j += thds_per_ball )
    {
      const int n_idx = bi + j;
      if ( j == 0 ) continue;
      if ( n_idx < 0 ) continue;
      if ( n_idx >= dc.chain_length ) continue;
      const int idx_dist = abs(j);
      const bool rep_only = idx_dist > 1;

      Ball* const neighbor_ball = &dc.d_balls[n_idx];

      // Construct a normalized (Unit) Vector from ball to neighbor.
      //
      /// SOLUTION
      //  Use d_pos array.
      pNorm ball_to_neighbor = mn(ball_position,dc.d_pos[n_idx]);

      // Compute the speed of ball towards neighbor_ball.
      //
      pVect delta_v = neighbor_ball->velocity - ball_velocity;
      float delta_s = dot( delta_v, ball_to_neighbor );

      // Compute distance between us and neighbor at which "spring"
      // will be relaxed (neither stretched nor compressed).
      //
      const float distance_relaxed =
        rep_only ? idx_dist * dc.distance_relaxed : dc.distance_relaxed;

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch =
        ball_to_neighbor.magnitude - distance_relaxed;

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0f ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? dc.opt_spring_constant : dc.opt_spring_constant * 0.7f;

      if ( !rep_only || spring_stretch < 0 )
        force += spring_constant * spring_stretch * ball_to_neighbor;
    }

  /// SOLUTION

  __shared__ float3 forces[1024];
  forces[threadIdx.x] = force;
  
  __syncthreads();

  // At this point the values of each ball's forces are spread over
  // several threads. Here one volunteer thread per ball (the thread
  // with ai_first==0) will add up these forces.
  //
  if ( ai_first == 0 )
    {
      // Add up forces.  Because only 7 values are added a linear
      // reduction (the loop) is acceptable. If more values needed
      // to be added a tree reduction would be used.
      //
      const int iters = min(7,thds_per_ball);
      for ( int i=1; i<iters; i++ ) 
        force += forces[threadIdx.x + i * balls_per_block];
      forces[threadIdx.x] = force;
    }

  __syncthreads();

  // Check for contact between us and other balls, except our immediate
  // neighbors.  If opt_test1 is true then skip these tests, as a way
  // of demonstrating how much time they take.
  //
  if ( !dc.opt_test1 )
    for ( int j=ai_first; j<dc.chain_length; j += thds_per_ball )
      if ( abs(j-bi)>1 )
        {
          Ball* const ball2 = &dc.d_balls[j];
          pNorm ball_to_2 = 
            mn( ball_position,
                dc.opt_test2 ? ball2->position : dc.d_pos[j]);
          const float rsum = ball_radius +
            ( dc.opt_test2 ? ball2->radius : dc.d_pos[j].w );
          if ( rsum * rsum > ball_to_2.mag_sq )
            {
              const float compression = rsum - ball_to_2.magnitude;
              const float f = -compression * 1000;
              const float3 sep_f = f * ball_to_2;
              atomicAdd(&forces[b_local].x,sep_f.x);
              atomicAdd(&forces[b_local].y,sep_f.y);
              atomicAdd(&forces[b_local].z,sep_f.z);
            }
        }

  __syncthreads();

  if ( ai_first ) return;

  float3 velocity = ball_velocity;

  velocity += delta_t * ball->mass_inv * forces[b_local];

  // Air Resistance
  //
  const double fs = powf(1+dc.opt_air_resistance,-delta_t);
  velocity *= fs;

  ball->velocity = velocity;
}


__host__ void
launch_time_step_gpu_sol_part_2
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_sol_part_2<<<grid_size,block_size>>>(delta_t); 
}

__global__ void
time_step_gpu_sol_part_2(float delta_t)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int bi = tid; // Ball Index

  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  // Update Position
  //
  // Assume that velocity is constant.
  //
  /// SOLUTION
  //  Update d_pos array.
  float4 pos = dc.d_pos[bi] += ball->velocity * delta_t;

  /// SOLUTION
  //  Also update ball structure if test2 is true, to be able to
  //  see the affect of less efficient memory access.
  //
  if ( dc.opt_test2 )
    ball->position = pos;

  // Possible Collision with Platform
  //

  // Skip if collision impossible.
  //
  if ( !platform_collision_possible(pos) ) return;
  if ( pos.y < 0 ) return;
  if ( pos.y >= ball->radius ) return;

  const float spr_const = ball->velocity.y > 0 ? 500000 : 1000000;

  ball->velocity.y += ( ball->radius - pos.y ) * delta_t * spr_const;
}
