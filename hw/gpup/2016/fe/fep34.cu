#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2016), GPU Programming
//

 /// Code used in Final Exam problems 3 and 4.
 //
 //  See  http://www.ece.lsu.edu/koppel/gpup/2016/fe.pdf
 //
 //  Based on demo-cuda-02-basics.cu
 //
 //  Not runnable.

/// References
//
//  :ccpg8: CUDA C Programming Guide Version 8
//          https://docs.nvidia.com/cuda/cuda-c-programming-guide

#endif

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>

#include <hip/hip_runtime.h>
#include <gp/cuda-gpuinfo.h>


struct App
{
  int num_threads;
  int array_size;

  float4 *h_in;         // Host address space, data in.
  float *h_out;         // Host address space, data out.
  float *h_out_check;   // Compute correct answer on CPU, to check GPU.

  float4 *d_in;         // Device address space, data in.
  float *d_out;         // Device address space, data out.
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;


///
/// GPU Code (Kernels)
///


__global__ void
kmain_simple()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid >= d_app.num_threads ) return;

  const int elt_per_thread =
    ( d_app.array_size + d_app.num_threads - 1 ) / d_app.num_threads;
  const int start = elt_per_thread * tid;  // Bad: Non-consecutive access.
  const int stop = start + elt_per_thread;

  for ( int h=start; h<stop; h++ )
    {
      float4 p = d_app.d_in[h];  // Bad: Non-consecutive access.
      float sos = p.x * p.x + p.y * p.y + p.z * p.z + p.w * p.w;
      d_app.d_out[h] = sos;
    }
}

float dot(float4 p, float4 q) { return p.x*q.x+p.y*q.y+p.z*q.z+p.w*q.w;}
float dot(float3 p, float3 q) { return p.x*q.x+p.y*q.y+p.z*q.z;}
float dot(float2 p, float2 q) { return p.x*q.x+p.y*q.y;}

__global__ void
kmain_efficient()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_threads = blockDim.x * gridDim.x;

  for ( int h=tid; h<d_app.array_size; h += n_threads )
    {
      float4 p = d_app.d_in[h];
      d_app.d_out[h] = dot(p,p);
    }
}

__global__ void
prob_x(float2 *d_in, float *d_out)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_threads = blockDim.x * gridDim.x;

  for ( int h=tid; h<d_app.array_size-1; h += n_threads )
    {
      float2 p = d_in[h];
      float2 q = d_in[h+1];
      float dp = dot(p,q);
      d_out[h] = dp;
    }
}

__global__ void
prob_x2_sol(float2 *d_in, float *d_out)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_threads = blockDim.x * gridDim.x;

  __shared__ float3 sm[1024];

  for ( int h=tid; h<d_app.array_size; h += n_threads )
    {
      float2 p = d_in[h];
      __syncthreads();
      sm[threadIdx.x] = p;
      __syncthreads();
      float2 q = sm[threadIdx.x+1];
      float dp = dot(p,q);
      d_out[h] = dp;
    }
}

__global__ void
prob_x2(float2 *d_in, float *d_out)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int n_threads = blockDim.x * gridDim.x;

  __shared__ float3 sm[1024];

  for ( int h=tid; h<d_app.array_size; h += n_threads )
    {
      float2 p = d_in[h];


      float2 q = d_in[h+1];

      float dp = dot(p,q);
      d_out[h] = dp;
    }
}

__global__ void
prob_x3(float3 *d_in, float *d_out)
{
  const int tid = threadIdx.x + blockIdx.x * ( blockDim.x - 1 );
  const int n_threads = blockDim.x * gridDim.x;
  const bool last_thd = threadIdx.x == blockDim.x - 1;

  __shared__ float3 sm[1024];

  for ( int h=tid; h<d_app.array_size-1; h += n_threads )
    {
      float3 p = d_in[h];
      __syncthreads();
      sm[threadIdx.x] = p;
      __syncthreads();
      if ( last_thd ) continue;
      float3 q = sm[threadIdx.x+1];
      float dp = dot(p,q);
      d_out[h] = dp;
    }
}


GPU_Info
print_gpu_and_kernel_info()
{
  GPU_Info info;

  gpu_info_print();

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(kmain_simple);
  info.GET_INFO(kmain_efficient);
  info.GET_INFO(kmain_tuned);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }
  return info;
}


