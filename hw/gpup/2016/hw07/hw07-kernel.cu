#include "hip/hip_runtime.h"

#include "cuda-coord.cu"
#include "hw07.cuh"
#include <gp/cuda-util-kernel.h>

// opt_air_resistance

// ball:: mass, mass_min, fdt_to_do
// ball:: force, torque, omatrix, position, velocity, omega, orientation
// link:: ball1, ball2, cb1, cb2, distance_relaxed
// link:: torque1, torque2, spring_force_12


__global__ void
time_step_gpu(double delta_t)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const int n_threads = blockDim.x * gridDim.x;

  const int n_balls = balls;
  const int n_links = links;

  for ( int i=0; i<n_links; i++ )
    {
      Link* const link = links[i];
      if ( !link->is_simulatable ) continue;

      // Spring Force from Neighbor Balls
      //
      Ball* const ball1 = link->ball1;
      Ball* const ball2 = link->ball2;

      // Find position and velocity of the point where the link touches
      // the surface of ball 1 ...
      //
      pVect dir1 = ball1->omatrix * link->cb1;
      pCoor pos1 = ball1->position + dir1;
      pVect vel1 = ball1->velocity + ball1->point_rot_vel(dir1);

      // ... and ball 2.
      //
      pVect dir2 = ball2->omatrix * link->cb2;
      pCoor pos2 = ball2->position + dir2;
      pVect vel2 = ball2->velocity + ball2->point_rot_vel(dir2);

      // Construct a normalized (Unit) Vector from ball to neighbor
      // based on link connection points and ball centers.
      //
      pNorm link_dir(pos1,pos2);
      pNorm c_to_c(ball1->position,ball2->position);

      const float link_length = link_dir.magnitude;

      // Compute the speed of ball's end of link towards neighbor's end of link.
      //
      pVect delta_v = vel2 - vel1;
      float delta_s = dot( delta_v, link_dir );

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch = link_length - link->distance_relaxed;

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0 ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? opt_spring_constant : opt_spring_constant * 0.7;

      const float force_mag = spring_constant * spring_stretch;
      pVect spring_force_12 = force_mag * link_dir;

      // Apply forces affecting linear momentum.
      //
      link->spring_force_12 = spring_force_12;

      if ( ! link->is_surface_connection ) continue;

      // Apply torque.
      //
      link->torque1 = cross(pNorm(dir1), spring_force_12);
      link->torque2 = cross(pNorm(dir2), spring_force_12);
    }

  for ( int bi=tid; bi<n_balls; bi += n_threads )
    {
      Ball* const ball = balls[bi];
      ball->force = ball->mass * gravity_accel;
      ball->torque = pVect(0,0,0);
    }

  // Note: Because two links can reference the same ball this should
  // not be done in parallel.
  for ( Link *link: links )
    {
      if ( !link->is_simulatable ) continue;
      Ball* const ball1 = link->ball1;
      Ball* const ball2 = link->ball2;
      ball1->force += link->spring_force_12;
      ball2->force -= link->spring_force_12;
      ball1->torque += link->torque1;
      ball2->torque -= link->torque2;
    }

  ///
  /// Update Position of Each Ball
  ///

  for ( int bi=0; bi<n_balls; bi++ )
    {
      Ball* const ball = balls[bi];
      if ( ball->locked )
        {
          ball->velocity = pVect(0,0,0);
          ball->omega = pVect(0,0,0);
          continue;
        }

      // Update Velocity
      //
      // This code assumes that force on ball is constant over time
      // step. This is clearly wrong when balls are moving with
      // respect to each other because the springs are changing
      // length. This inaccuracy will make the simulation unstable
      // when spring constant is large for the time step.
      //
      const float mass = max( ball->mass, ball->mass_min );

      pVect delta_v = ( ball->force / mass ) * delta_t;

      if ( platform_collision_possible(ball->position) && ball->position.y < 0 )
        {
          const float spring_constant_plat =
            ball->velocity.y < 0 ? 100000 : 50000;
          const float fric_coefficient = 0.1;
          const float force_up = -ball->position.y * spring_constant_plat;
          const float delta_v_up = force_up / mass * delta_t;
          const float fric_force_mag = fric_coefficient * force_up;
          pNorm surface_v(ball->velocity.x,0,ball->velocity.z);
          const float delta_v_surf = fric_force_mag / mass * delta_t;

          if ( delta_v_surf > surface_v.magnitude )
            {
              // Ignoring other forces?
              delta_v =
                pVect(-ball->velocity.x,delta_v.y,-ball->velocity.z);
            }
          else
            {
              delta_v -= delta_v_surf * surface_v;
            }
          delta_v.y += delta_v_up;
        }

      ball->velocity += delta_v;

      // Air Resistance
      //
      const double fs = pow(1+opt_air_resistance,-delta_t);
      ball->velocity *= fs;

      // Update Position
      //
      // Assume that velocity is constant.
      //
      ball->position += ball->velocity * delta_t;

      ball->omega += delta_t * ball->fdt_to_do * ball->torque;

      pNorm axis(ball->omega);

      // Update Orientation
      //
      // If ball isn't spinning fast skip expensive rotation.
      //
      if ( axis.mag_sq < 0.000001 ) continue;

      // Update ball orientation.
      //
      ball->orientation_set
        ( pQuat(axis,delta_t * axis.magnitude) * ball->orientation );
    }
}
