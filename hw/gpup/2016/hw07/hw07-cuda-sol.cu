#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2016), GPU Programming
//
 /// Homework 7 -- SOLUTION
 //
 //  See http://www.ece.lsu.edu/koppel/gpup/2016/hw07_sol.pdf

 /// Use this file for your solution.

#include "cuda-coord.cu"
#include "hw07.cuh"
#include <gp/cuda-util-kernel.h>

// Physical State Variables
//
__constant__ float4 *helix_position;
__constant__ float3 *helix_velocity;     // Note: float4 would be faster.
__constant__ float4 *helix_orientation;
__constant__ float3 *helix_omega;        // Note: float4 would be faster.

__constant__ Timing_Data *timing_data;   // Measure execution time of intersect.
__constant__ Helix_Info hi;  // Scalar Constants

__global__ void
time_step_intersect_1()
{
  /// Homework 7  SOLUTION IN THIS ROUTINE

  // Find intersections of one helix segment with some other
  // segments. Each block handles several "a" segments, the threads in
  // the block check for intersection with other segments, called "b"
  // segments.

  __shared__ clock_t time_start;
  if ( !threadIdx.x ) time_start = clock64();


  // Note: The size of the helix_position array is hi.phys_helix_segments.

  // Compute how many "a" elements will be handled by each block.
  //
  const int a_per_block = hi.phys_helix_segments / gridDim.x;

  // Compute how many threads handle each "a" element.
  //
  const int thd_per_a = blockDim.x / a_per_block;

  // Compute the smallest "a" element index that this block will handle.
  //
  const int a_idx_block = blockIdx.x * a_per_block;

  /// Assignment of "a" and "b" Values to Threads
  //
  //  The table below is an example of how this routine
  //  assigns "a" and "b" elements to threads.  The table
  //  is based upon the following values:
  //
  //    blockDim = 8,       blockIdx = 4,     hi.phys_helix_segments = 1024
  //    m:a_per_block = 4,  d:thd_per_a = 2,  a_idx_block = 16
  //
  // tIx     al   a      b ---> 
  //   0     0    16     0  2  4 ... 1022
  //   1     1    17     0  2  4 ... 1022
  //   2     2    18     0  2  4 ... 1022
  //   3     3    19     0  2  4 ... 1022
  //   4     0    16     1  3  5 ... 1023
  //   5     1    17     1  3  5 ... 1023
  //   6     2    18     1  3  5 ... 1023
  //   7     3    19     1  3  5 ... 1023
  //   |     |     |     |
  //   |     |     |     |
  //   |     |     |     |--------> b_idx_start
  //   |     |     |--------------> a_idx
  //   |     |--------------------> a_local_idx
  //   |--------------------------> threadIdx.x

  // Compute a_idx and b_idx_start to realize ordering above.
  //
  const int a_local_idx = threadIdx.x % a_per_block;
  const int a_idx = a_idx_block + a_local_idx;
  const int b_idx_start = threadIdx.x / a_per_block;

  const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
  const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;

  // Declare dynamically allocated shared memory. Will be split
  // between array for forces, force, and position cache, pos_cache.
  //
  extern __shared__ float3 shared[];

  pVect* const force = shared;
  float3* const pos_cache = &shared[a_per_block];

  if ( threadIdx.x < a_per_block ) force[threadIdx.x] = mv(0,0,0);

  __syncthreads();

  const float3 a_position = m3(helix_position[a_idx]);

  /// SOLUTION -- Problem 3
  //
  //  The next element of pos_cache to use. Its value should be
  //  between 0 and blockDim.x (block size) -1. It is intentionally
  //  initialized to an out-of-range value so that the cache will be
  //  loaded.
  //
  int cache_idx_next = b_idx_start + blockDim.x;
  //
  //  The next element of helix to put into the cache.
  //
  int b_idx_next = threadIdx.x;

  for ( int j=b_idx_start; j<hi.phys_helix_segments; j += thd_per_a )
    {
      if ( hi.opt_sm_option == SMO_one_iteration )
        {
          __syncthreads();
          if ( threadIdx.x < thd_per_a )
            pos_cache[threadIdx.x] =
              m3(helix_position[ j - b_idx_start + threadIdx.x ] );
          __syncthreads();
        }
      else if ( hi.opt_sm_option == SMO_sync_experiment )
        {
          /// SOLUTION -- Problem 2
          //
          //  See if just executing __syncthreads slows things down.
          //
          __syncthreads();
          __syncthreads();
        }
      else if ( hi.opt_sm_option == SMO_multiple_iterations )
        {
          /// SOLUTION -- Problem 3
          //
          // If the next pos_cache element to use is out of range, then
          // load pos_cache with a new batch of data.
          //
          if ( cache_idx_next >= blockDim.x )
            {
              __syncthreads();
              cache_idx_next = b_idx_start;
              pos_cache[ threadIdx.x ] = m3(helix_position[ b_idx_next ] );
              b_idx_next += blockDim.x;
              __syncthreads();
            }
        }

      /// SOLUTION -- Problem 3
      //
      //  For the multiple iteration case the index to pos_cache
      //  is a function of j.
      //
      float3 b_position =
        hi.opt_sm_option == SMO_one_iteration
        ? pos_cache[ b_idx_start ] :
        hi.opt_sm_option == SMO_multiple_iterations
        ? pos_cache[ cache_idx_next ]
        : m3( helix_position[j] );

      /// SOLUTION -- Problem 3
      //
      cache_idx_next += thd_per_a;

      pVect ab = mv(a_position,b_position);

      // Skip if segment is too close.
      if ( abs(a_idx-j) < min_idx_dist ) continue;

      // Skip if no chance of intersection.
      if ( mag_sq(ab) >= four_wire_radius_sq ) continue;

      // Compute intersection force based on bounding sphere, an
      // admittedly crude approximation.
      //
      pNorm dist = mn(ab);
      const float pen = 2 * hi.wire_radius - dist.magnitude;
      float3 f = pen * hi.opt_spring_constant * dist;

      // Add force to shared variable. This is time consuming
      // (especially in CC 3.x and older GPUs) but done
      // infrequently. (A segment can normally only intersect a a few
      // other segments.)
      //
      atomicAdd(&force[a_local_idx].x,f.x);
      atomicAdd(&force[a_local_idx].y,f.y);
      atomicAdd(&force[a_local_idx].z,f.z);
      //
      // Optimization Note: Could acquire a lock and then update
      // all three components.
    }

  // Wait for all threads to finish.
  __syncthreads();

  // Leave it to thread 0 to update velocity.
  if ( threadIdx.x >= a_per_block ) return;

  // Update velocity and write it.
  //
  float3 velocity = helix_velocity[a_idx];
  velocity -= hi.delta_t_mass_inv * force[a_local_idx];
  if ( hi.opt_end_fixed && a_idx + 1 == hi.phys_helix_segments )
    velocity = mv(0,0,0);
  helix_velocity[a_idx] = velocity;

  if ( !threadIdx.x )
    {
      timing_data[blockIdx.x].intersect_time += clock64() - time_start;
      timing_data[blockIdx.x].intersect_count++;
    }
}

__global__ void
time_step_intersect_2()
{
  /// DO NOT MODIFY THIS ROUTINE.

  // Find intersections of one helix segment with some other
  // segments. Each block handles several "a" segments, the threads in the
  // block check for intersection with other segments, called "b"
  // segments.

  __shared__ clock_t time_start;
  if ( !threadIdx.x ) time_start = clock64();

  // Note: The size of the helix_position array is hi.phys_helix_segments.

  // Compute how many "a" elements will be handled by each block.
  //
  const int a_per_block = hi.phys_helix_segments / gridDim.x;

  // Compute how many threads handle each "a" element.
  //
  const int thd_per_a = blockDim.x / a_per_block;

  // Compute the smallest "a" element index that this block will handle.
  //
  const int a_idx_block = blockIdx.x * a_per_block;

  /// Assignment of "a" and "b" Values to Threads
  //
  //  The table below is an example of how this routine
  //  assigns "a" and "b" elements to threads.  The table
  //  is based upon the following values:
  //
  //    blockDim = 8,     blockIdx = 4,   hi.phys_helix_segments = 1024
  //    a_per_block = 4,  thd_per_a = 2,  a_idx_block = 16
  //
  // tIx     al   a      b ---> 
  //   0     0    16     0  2  4 ...
  //   1     0    16     1  3  5
  //   2     1    17     0  2  4
  //   3     1    17     1  3  5
  //   4     2    18     0  2  4
  //   5     2    18     1  3  5
  //   6     3    19     0  2  4
  //   7     3    19     1  3  5 
  //   |     |     |     |
  //   |     |     |     |
  //   |     |     |     |--------> b_idx_start
  //   |     |     |--------------> a_idx
  //   |     |--------------------> a_local_idx
  //   |--------------------------> threadIdx.x

  // Compute a_idx and b_idx_start to realize ordering above.
  //
  const int a_local_idx = threadIdx.x / thd_per_a;
  const int a_idx = a_idx_block + a_local_idx;
  const int b_idx_start = threadIdx.x % thd_per_a;

  const float3 a_position = m3(helix_position[a_idx]);
  const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
  const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;

  // Declare dynamically allocated shared memory. Will be split
  // between array for forces, force, and position cache, pos_cache.
  //
  extern __shared__ float3 shared[];

  pVect* const force = shared;
  if ( threadIdx.x < a_per_block ) force[threadIdx.x] = mv(0,0,0);

  // Wait for thread 0 to initialize force.
  __syncthreads();

  const bool use_shared =
    hi.opt_sm_option == SMO_one_iteration
    || hi.opt_sm_option == SMO_multiple_iterations;

  float3* const pos_cache = &shared[a_per_block];

  int cache_elt_remaining = use_shared ? 0 : -1;
  int cache_num_refills = 0;
  int cache_idx_start = 0;

  for ( int j=b_idx_start; j<hi.phys_helix_segments; j += thd_per_a )
    {
      if ( hi.opt_sm_option == SMO_one_iteration )
        {
          /// SOLUTION
          cache_idx_start = j - b_idx_start;
          __syncthreads();
          if ( threadIdx.x < thd_per_a )
            pos_cache[threadIdx.x] = m3(helix_position[j]);
          __syncthreads();
        }
      else if ( hi.opt_sm_option == SMO_multiple_iterations
                && cache_elt_remaining == 0 )
        {
          __syncthreads();
          cache_idx_start = cache_num_refills * blockDim.x;
          pos_cache[threadIdx.x] =
            m3(helix_position[ cache_idx_start + threadIdx.x ] );
          __syncthreads();
          cache_num_refills++;
          cache_elt_remaining = blockDim.x;
        }
      cache_elt_remaining -= thd_per_a;

      float3 b_position =
        use_shared ? pos_cache[j-cache_idx_start] : m3(helix_position[j]);

      pVect ab = mv(a_position,b_position);

      // Skip if segment is too close.
      if ( abs(a_idx-j) < min_idx_dist ) continue;

      // Skip if no chance of intersection.
      if ( mag_sq(ab) >= four_wire_radius_sq ) continue;

      // Compute intersection force based on bounding sphere, an
      // admittedly crude approximation.
      //
      pNorm dist = mn(ab);
      const float pen = 2 * hi.wire_radius - dist.magnitude;
      float3 f = pen * hi.opt_spring_constant * dist;

      // Add force to shared variable. This is time consuming but
      // done infrequently. (A segment can normally only intersect a
      // a few other segments.)
      //
      atomicAdd(&force[a_local_idx].x,f.x);
      atomicAdd(&force[a_local_idx].y,f.y);
      atomicAdd(&force[a_local_idx].z,f.z);
      //
      // Optimization Note: Could acquire a lock and then update
      // all three components.
    }

  // Wait for all threads to finish.
  __syncthreads();

  // Leave it to thread 0 to update velocity.
  if ( threadIdx.x >= a_per_block ) return;

  {
    // Re-compute a_idx so that first a_per_block threads can write
    // velocities.

    const int a_local_idx = threadIdx.x;
    const int a_idx = a_idx_block + a_local_idx;

    // Update velocity and write it.
    //
    float3 velocity = helix_velocity[a_idx];
    velocity -= hi.delta_t_mass_inv * force[a_local_idx];
    if ( hi.opt_end_fixed && a_idx + 1 == hi.phys_helix_segments )
      velocity = mv(0,0,0);
    helix_velocity[a_idx] = velocity;

    if ( !threadIdx.x )
      {
        timing_data[blockIdx.x].intersect_time += clock64() - time_start;
        timing_data[blockIdx.x].intersect_count++;
      }
  }
}


__global__ void time_step();
__global__ void time_step_intersect_1();
__global__ void time_step_intersect_2();
__global__ void time_step_update_pos();


__host__ hipError_t
cuda_setup(GPU_Info *gpu_info)
{
  // Pass the device address to host code. (See gp/cuda-util-kernel.h ).
  CU_SYM(helix_position);
  CU_SYM(helix_velocity);
  CU_SYM(helix_orientation);
  CU_SYM(helix_omega);
  CU_SYM(timing_data);
  CU_SYM(hi);

  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.

  hipError_t e1 = hipSuccess;

  gpu_info->GET_INFO(time_step);
  gpu_info->GET_INFO(time_step_intersect_1);
  gpu_info->GET_INFO(time_step_intersect_2);
  gpu_info->GET_INFO(time_step_update_pos);


  return e1;
}

__host__ void time_step_launch(int grid_size, int block_size)
{
  time_step<<<grid_size,block_size>>>();
}

__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude);


__global__ void
time_step()
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Use tid for helix segment number.

  if ( tid + 1 > hi.phys_helix_segments ) return;

  // The position of segment 0 is fixed, so don't evolve it.
  if ( tid == 0 ) return;

  pVect vZero = mv(0,0,0);
  pVect gravity_force = hi.helix_seg_mass_inv * hi.gravity_accel;

  pQuat c_orientation = cq(helix_orientation[tid]);
  float3 c_position = m3(helix_position[tid]);

  pMatrix3x3 c_rot;
  // Initialize c_rot to a rotation matrix based on quaternion c_orientation.
  pMatrix_set_rotation(c_rot,c_orientation);

  float3 c_u = c_rot * mv(0,0,1);  // mv: Make Vector.
  float3 c_v = c_rot * mv(0,1,0);
  float3 c_ctr_to_right_dir = c_rot * mv(1,0,0);
  pVect c_ctr_to_right = hi.helix_seg_hlength * c_ctr_to_right_dir;
  float3 c_pos_right = c_position + c_ctr_to_right;
  float3 c_pos_left = c_position - c_ctr_to_right;

  float3 force = hi.opt_gravity ? gravity_force : vZero;
  float3 torque = vZero;

  const int pieces = 3;
  const float delta_theta = 2 * M_PI / pieces;

  /// Compute forces due to right neighbor.
  //
  if ( tid + 1 < hi.phys_helix_segments )
    {
      pQuat r_orientation = cq(helix_orientation[tid+1]);
      float3 r_position = m3(helix_position[tid+1]);
      pMatrix3x3 r_rot;
      pMatrix_set_rotation(r_rot,r_orientation);
      float3 r_u = r_rot * mv(0,0,1);
      float3 r_v = r_rot * mv(0,1,0);
      float3 r_ctr_to_right_dir = r_rot * mv(1,0,0);
      pVect r_ctr_to_right = hi.helix_seg_hlength * r_ctr_to_right_dir;
      float3 r_pos_left = r_position - r_ctr_to_right;

      pQuat cn_rot_q = c_orientation * hi.helix_rn_trans;
      pMatrix3x3 cn_rot;
      pMatrix_set_rotation(cn_rot,cn_rot_q);
      pVect n_ru = cn_rot * mv(0,0,1);
      pVect n_rv = cn_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_right + cosf(theta) * n_ru + sinf(theta) * n_rv;
          pCoor r_pt = r_pos_left + cosf(theta) * r_u + sinf(theta) * r_v;
          pNorm dist = mn(c_pt,r_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// Compute forces due to left neighbor.
  //
  if ( tid > 0 )
    {
      pQuat l_orientation = cq(helix_orientation[tid-1]);
      float3 l_position = m3(helix_position[tid-1]);
      pMatrix3x3 l_rot;
      pMatrix_set_rotation(l_rot,l_orientation);
      float3 l_u = l_rot * mv(0,0,1);
      float3 l_v = l_rot * mv(0,1,0);
      float3 l_ctr_to_right_dir = l_rot * mv(1,0,0);
      pVect l_ctr_to_right = hi.helix_seg_hlength * l_ctr_to_right_dir;
      float3 l_pos_right = l_position + l_ctr_to_right;

      pQuat ln_rot_q = l_orientation * hi.helix_rn_trans;
      pMatrix3x3 ln_rot;
      pMatrix_set_rotation(ln_rot,ln_rot_q);
      pVect n_cu = ln_rot * mv(0,0,1);
      pVect n_cv = ln_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_left + cosf(theta) * c_u + sinf(theta) * c_v;
          pCoor l_pt = l_pos_right + cosf(theta) * n_cu + sinf(theta) * n_cv;
          pNorm dist = mn(c_pt,l_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  float3 velocity = helix_velocity[tid];
  velocity *= 0.99999f;
  float3 omega = helix_omega[tid];
  omega *= 0.99999f;
  velocity += hi.delta_t_mass_inv * force;
  const float torque_axial_mag = dot( torque, c_ctr_to_right_dir );
  pVect torque_axial = torque_axial_mag * c_ctr_to_right_dir;
  pVect do_axial = hi.delta_t_ma_axis * torque_axial;
  pVect torque_other = torque - torque_axial;
  pVect do_other = hi.delta_t_ma_perp_axis * torque_other;
  omega += do_axial + do_other;

  // Update velocity and omega. Don't update position or orientation
  // because we don't want threads in this kernel to accidentally read
  // the updated values.

  helix_omega[tid] = omega;
  helix_velocity[tid] = velocity;
}


__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude)
{
  // Update force and torque of segment for a force acting on FORCE_POS
  // pointing in direction DIR of magnitude MAGNITUDE.
  //
  force += magnitude * dir;
  pVect arm = mv(position,force_pos);
  pVect axis = cross( arm, dir );
  pVect amt = magnitude * axis;
  torque += amt;
}



__host__ void
time_step_intersect_launch
(int grid_size, int block_size, int version, int dynamic_sm_amt)
{
  switch ( version ) {
    case 1: time_step_intersect_1<<<grid_size,block_size,dynamic_sm_amt>>>();
    case 2: time_step_intersect_2<<<grid_size,block_size,dynamic_sm_amt>>>();
  }
}

__global__ void
time_step_update_pos()
{
  // Update position and orientation of spring segments.

  // Use tid for helix segment number.
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Skip out-of-range segments.
  if ( tid >= hi.phys_helix_segments ) return;
  if ( tid == 0 ) return;

  // Update Orientation
  //
  pQuat orientation = cq(helix_orientation[tid]);
  float3 omega = helix_omega[tid];
  pNorm axis = mn(omega);
  helix_orientation[tid] =
    c4( quat_normalize
    ( quat_mult ( mq( axis, hi.delta_t * axis.magnitude ), orientation)));

  // Return if at last segment and it is fixed. Note that even
  // if the segment's position is fixed, it can still rotate.
  //
  if ( hi.opt_end_fixed && tid + 1 == hi.phys_helix_segments ) return;

  // Update Velocity
  //
  float3 position = m3(helix_position[tid]);
  float3 velocity = helix_velocity[tid];
  helix_position[tid] = m4(position + hi.delta_t * velocity,1);
}

__host__ void
time_step_update_pos_launch
(int grid_size, int block_size)
{
  time_step_update_pos<<<grid_size,block_size>>>();
}
