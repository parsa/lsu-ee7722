#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2013), GPU Programming
//
 /// Homework 3
 //

 /// Instructions
 //
 //  Read the assignment: http://www.ece.lsu.edu/koppel/gpup/2013/hw03.pdf

/// DO NOT MODIFY THIS FILE.
//  Instead, put your CUDA changes in hw3-sol-cuda.cu


#include "cuda-coord.cu"
#include "hw3.cuh"
#include <gp/cuda-util-kernel.h>

__constant__ CPU_GPU_Common dc;

__global__ void time_step_gpu_base_part_1(float delta_t);
__global__ void time_step_gpu_base_part_2(float delta_t);
__global__ void time_step_gpu_sol_part_1(float delta_t);
__global__ void time_step_gpu_sol_part_2(float delta_t);
__global__ void data_pack();
__global__ void data_unpack();
__device__ bool platform_collision_possible(pCoor pos);


__host__ hipError_t
cuda_setup(GPU_Info *gpu_info)
{
  CU_SYM(dc);
  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.

  hipError_t e1 = hipSuccess;

#define GET_INFO(proc_name) {                                                 \
  const int idx = gpu_info->num_kernels++;                                    \
  if ( idx >= gpu_info->num_kernels_max ) return e1;                          \
  gpu_info->ki[idx].name = #proc_name;                                        \
  gpu_info->ki[idx].func_ptr = (void(*)())proc_name;                          \
  e1 = hipFuncGetAttributes(&gpu_info->ki[idx].cfa,reinterpret_cast<const void*>(proc_name));               \
  if ( e1 != hipSuccess ) return e1; }

  GET_INFO(time_step_gpu_base_part_1);
  GET_INFO(time_step_gpu_base_part_2);
  GET_INFO(time_step_gpu_sol_part_1);
  GET_INFO(time_step_gpu_sol_part_2);
  GET_INFO(data_pack);
  GET_INFO(data_unpack);

#undef GET_INFO

  return e1;
}

#ifdef HW_SOL
#include "hw3-sol-cuda-sol.cu"
#else
#include "hw3-sol-cuda.cu"
#endif

__host__ void
launch_time_step_gpu_base_part_1
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_base_part_1<<<grid_size,block_size>>>(delta_t);
}

/// DO NOT MODIFY THIS FILE.
//  Instead, put your CUDA changes in hw3-sol-cuda.cu

__global__ void
time_step_gpu_base_part_1(float delta_t)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int bi = tid; // Ball Index

  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  // Convenient zero vector.
  const float3 zero_vec = make_float3(0,0,0);

  if ( dc.opt_head_lock && bi == 0
       || dc.opt_tail_lock && bi == dc.chain_length - 1 )
    {
      ball->velocity = zero_vec;
      return;
    }

  // Load the ball position and radius from memory. They will be used
  // several times. (They are loaded into registers.)
  //
  const float4 ball_position = ball->position;
  const float ball_radius = ball->radius;

  float3 force = zero_vec;

  // Gravitational Force
  //
  force += ball->mass * dc.gravity_accel;

  // Spring Force from Neighbor Balls
  //
  for ( int j = -3;  j < 4;  j++ )
    {
      const int n_idx = bi + j;
      if ( j == 0 ) continue;
      if ( n_idx < 0 ) continue;
      if ( n_idx >= dc.chain_length ) continue;
      const int idx_dist = abs(j);
      const bool rep_only = idx_dist > 1;

      Ball* const neighbor_ball = &dc.d_balls[n_idx];

      // Construct a normalized (Unit) Vector from ball to neighbor.
      //
      pNorm ball_to_neighbor = mn(ball_position,neighbor_ball->position);

      // Compute the speed of ball towards neighbor_ball.
      //
      pVect delta_v = neighbor_ball->velocity - ball->velocity;
      float delta_s = dot( delta_v, ball_to_neighbor );

      // Compute distance between us and neighbor at which "spring"
      // will be relaxed (neither stretched nor compressed).
      //
      const float distance_relaxed =
        rep_only ? idx_dist * dc.distance_relaxed : dc.distance_relaxed;

      // Compute by how much the spring is stretched (positive value)
      // or compressed (negative value).
      //
      const float spring_stretch =
        ball_to_neighbor.magnitude - distance_relaxed;

      // Determine whether spring is gaining energy (whether its length
      // is getting further from its relaxed length).
      //
      const bool gaining_e = ( delta_s > 0.0f ) == ( spring_stretch > 0 );

      // Use a smaller spring constant when spring is loosing energy,
      // a quick and dirty way of simulating energy loss due to spring
      // friction.
      //
      const float spring_constant =
        gaining_e ? dc.opt_spring_constant : dc.opt_spring_constant * 0.7f;

      if ( !rep_only || spring_stretch < 0 )
        force += spring_constant * spring_stretch * ball_to_neighbor;
    }

  // Check for contact between us and other balls, except our immediate
  // neighbors.
  //
  for ( int j=0; j<dc.chain_length; j++ )
    if ( abs(j-bi)>1 )
      {
        Ball* const ball2 = &dc.d_balls[j];
        pNorm ball_to_2 = mn(ball_position,ball2->position);
        const float rsum = ball_radius + ball2->radius;
        if ( rsum * rsum > ball_to_2.mag_sq )
          {
            const float compression = rsum - ball_to_2.magnitude;
            const float f = -compression * 1000;
            force += f * ball_to_2;
          }
      }

  // Update Velocity
  //
  // This code assumes that force on ball is constant over time
  // step. This is clearly wrong when balls are moving with
  // respect to each other because the springs are changing
  // length. This inaccuracy will make the simulation unstable
  // when spring constant is large for the time step.
  //
  ball->velocity += delta_t * ball->mass_inv * force;

  // Air Resistance
  //
  const double fs = powf(1+dc.opt_air_resistance,-delta_t);
  ball->velocity *= fs;
}

__host__ void
launch_time_step_gpu_base_part_2
(int grid_size, int block_size, float delta_t)
{
 time_step_gpu_base_part_2<<<grid_size,block_size>>>(delta_t);
}

/// DO NOT MODIFY THIS FILE.
//  Instead, put your CUDA changes in hw3-sol-cuda.cu

__global__ void
time_step_gpu_base_part_2(float delta_t)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int bi = tid; // Ball Index

  if ( bi >= dc.chain_length ) return;

  Ball* const ball = &dc.d_balls[bi];

  ///
  /// Update Position of Each Ball
  ///

  // Update Position
  //
  // Assume that velocity is constant.
  //
  ball->position += ball->velocity * delta_t;

  // Possible Collision with Platform
  //

  // Skip if collision impossible.
  //
  if ( !platform_collision_possible(ball->position) ) return;
  if ( ball->position.y >= ball->radius ) return;
  if ( ball->position.y < 0 ) return;

  // Apply spring forces due to collision with platform.
  //
  const float spr_const = ball->velocity.y > 0 ? 500000 : 1000000;
  ball->velocity.y += ( ball->radius - ball->position.y ) * delta_t * spr_const;
}

/// DO NOT MODIFY THIS FILE.
//  Instead, put your CUDA changes in hw3-sol-cuda.cu

__device__ bool
platform_collision_possible(pCoor pos)
{
  // Assuming no motion in x or z axes.
  //
  return pos.x >= dc.platform_xmin && pos.x <= dc.platform_xmax
    && pos.z >= dc.platform_zmin && pos.z <= dc.platform_zmax;
}
