#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2014), GPU Microarchitecture
//

 /// Homework 1 and 2  (Spring 2014)
 //
 // Assignment in: http://www.ece.lsu.edu/koppel/gp/2014/hw01.pdf
 //                and http://www.ece.lsu.edu/koppel/gp/2014/hw02.pdf
 //

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include "util.h"


// The unroll degree is used by kernels dots_iterate1 and dots_iterate2.
//
const int unroll_degree = 8;

struct App_Common
{
  float2 *a;
  float *b;
  float *b_check;

  float2 *d_a;
  float *d_b;

  float v0, v1, v2;
  int array_size;
  int num_threads;
};


// In host address space.
App_Common app;

// In device constant address space.
__constant__ App_Common dapp;



extern "C" __global__ void
dots_iterate0()
{
  const int thread_count = blockDim.x * gridDim.x;
  const int idx_start = threadIdx.x + blockIdx.x * blockDim.x;

  // For convenience, assign to local variables.
  float2* const a = dapp.d_a;
  float* const b = dapp.d_b;

  for ( int idx = idx_start; idx < dapp.array_size; idx += thread_count )
    b[idx] = dapp.v0 + dapp.v1 * a[idx].x + dapp.v2 * a[idx].y;

}


extern "C" __global__ void
dots_iterate1()
{
  const int thread_count = blockDim.x * gridDim.x;
  const int idx_start = threadIdx.x + blockIdx.x * blockDim.x;

  // For convenience, assign to local variables.
  float2* const a = dapp.d_a;
  float* const b = dapp.d_b;

  for ( int idx = idx_start; idx < dapp.array_size;
        idx += unroll_degree * thread_count )
    {
      float keep[unroll_degree];
      for ( int i=0; i<unroll_degree; i++ )
        {
          const int idx2 = idx + i * thread_count;
          if ( idx2 < dapp.array_size )
            keep[i] = dapp.v0 + dapp.v1 * a[idx2].x + dapp.v2 * a[idx2].y;
        }
      for ( int i=0; i<unroll_degree; i++ )
        {
          const int idx2 = idx + i * thread_count;
          if ( idx2 < dapp.array_size )
            b[idx2] = keep[i];
        }
    }
}

extern "C" __global__ void
dots_iterate2()
{
  /// Modify this routine for Homework 2.
  //
  //  Start your solution by removing the return statement below.

  return;

  const int thread_count = blockDim.x * gridDim.x;
  const int idx_start = threadIdx.x + blockIdx.x * blockDim.x;

  // For convenience, assign to local variables.
  float2* const a = dapp.d_a;
  float* const b = dapp.d_b;

  for ( int idx = idx_start; idx < dapp.array_size;
        idx += unroll_degree * thread_count )
    {
      float keep[unroll_degree];
      for ( int i=0; i<unroll_degree; i++ )
        {
          const int idx2 = idx + i * thread_count;
          if ( idx2 < dapp.array_size )
            keep[i] = dapp.v0 + dapp.v1 * a[idx2].x + dapp.v2 * a[idx2].y;
        }
      for ( int i=0; i<unroll_degree; i++ )
        {
          const int idx2 = idx + i * thread_count;
          if ( idx2 < dapp.array_size )
            b[idx2] = keep[i];
        }
    }

}


GPU_Info
print_gpu_info()
{
  GPU_Info info;

  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  CE( hipGetDeviceCount(&device_count) ); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      info.get_gpu_info(dev);  // Look in file util.h.
      hipDeviceProp_t& cuda_prop = info.cuda_prop;

      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         info.cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize >> 10,
         info.chip_bw_Bps * 1e-9,
         info.chip_sp_flops * 1e-9,
         4 * info.chip_sp_flops / info.chip_bw_Bps);
    }

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(dots_iterate0);
  info.GET_INFO(dots_iterate1);
  info.GET_INFO(dots_iterate2);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }

  return info;
}

int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_info();

  // Examine argument 1, grid size.
  //
  const int arg1_int =
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks = abs(arg1_int);

   // Examine argument 2, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );

  if ( num_blocks <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  const int a_size_bytes = app.array_size * sizeof(app.a[0]);
  const int b_size_bytes = app.array_size * sizeof(app.b[0]);

  // Allocate storage for CPU copy of data.
  //
  app.a = new float2[app.array_size];
  app.b = new float[app.array_size];
  app.b_check = new float[app.array_size];
  const int overrun_size = 256;

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_a, (app.array_size+overrun_size)*sizeof(app.a[0]) ) );
  CE( hipMalloc( &app.d_b, (app.array_size+overrun_size)*sizeof(app.b[0]) ) );

  printf
    ("\nPreparing for %d blocks operating on %d elements. Unroll degree: %d.\n",
     num_blocks, app.array_size, unroll_degree);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    {
      app.a[i].x = drand48();
      app.a[i].y = drand48();
    }

  // Initialize Coefficients
  //
  app.v0 = drand48();
  app.v1 = drand48();
  app.v2 = drand48();

  // Compute correct answer (for checking).
  //
  for ( int e=0; e<app.array_size; e++ )
    app.b_check[e] = app.v0 + app.v1 * app.a[e].x + app.v2 * app.a[e].y;

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy( app.d_a, app.a, a_size_bytes, hipMemcpyHostToDevice ) );

  const double data_size =
    app.array_size * ( sizeof(app.a[0]) + sizeof(app.b[0]) );

  // Stuff needed to print bar graph.
  //
  const int full_width = 49;  // Width of longest bar.
  char **stars = (char**) alloca( info.num_kernels * sizeof(void*) );
  for ( int j=0; j< info.num_kernels; j++ )
    {
      stars[j] = (char*) alloca(full_width+1);
      for ( int i=0; i<full_width; i++ ) stars[j][i] = '0' + j;
      stars[j][full_width] = 0;
    }

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      hipFuncAttributes& cfa = info.ki[kernel].cfa;

      if ( kernel == 2 && cfa.numRegs < 3 )
        {
          printf("Not running kernel %s because code probably not present.\n",
                info.ki[kernel].name);
          continue;
        }

      // Maximum number of warps per block that this GPU can handle
      // for this kernel.
      //
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      printf("\nRunning kernel %s which uses %d regs on %d blocks.",
             info.ki[kernel].name, cfa.numRegs, num_blocks);
      if ( kernel )
        printf("  Unroll degree %d\n", unroll_degree);
      else
        printf("\n");

      printf(" %3s %9s  %6s  %5s\n", "Num", "Time", "Data", "" );
      printf(" %3s %9s  %6s  %5s\n", "Wps", "µs", " GB/s", "Pct" );

      for ( int warp_cnt = 1; warp_cnt <= wp_limit; warp_cnt++ )
        {
          const int thd_per_block = warp_cnt << 5;
          app.num_threads = num_blocks * thd_per_block;

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          // Zero result array (to catch errors when kernel skips elements).
          //
          CE( hipMemset( app.d_b, 0, b_size_bytes ) );

          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
          const double elapsed_time_s = cuda_time_ms * 0.001;

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (app.b, app.d_b, b_size_bytes, hipMemcpyDeviceToHost) );

          int err = 0;
          for ( int e=0; e<app.array_size; e++ )
            {
              const float diff = fabs( app.b[e] - app.b_check[e] );
              if ( diff > 0.00001 )
                {
                  err++;
                  if ( err < 2 )
                    printf
                      ("Error at array idx %d: %f != %f (correct)\n",
                       e, app.b[e], app.b_check[e]);
                }
            }

          const double frac = data_size / elapsed_time_s / info.chip_bw_Bps;
          const int s_idx = max(0.0,full_width * ( 1 - frac ));

          printf(" %2d  %9.3f  %6.2f %5.1f%%  %s\n",
                 warp_cnt, 1e6 * elapsed_time_s,
                 1e-9 * data_size / elapsed_time_s,
                 100 * frac, &stars[kernel][s_idx] );
        }
    }
}
