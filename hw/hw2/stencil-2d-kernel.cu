#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Sp 2011), GPU Microarchitecture
//
/// Homework 2

 /// See stencil-2d.cc for instructions.

#include "stencil-2d.cuh"

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ int row_stride, dim_size_lg, dim_block_lg;
__constant__ float* a;
__constant__ float* b;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__global__ void stencil();
__global__ void stencil_iter();
__global__ void stencil_shared();
__global__ void stencil_shared_2();

static __host__ int
kernels_get_attr_(pCUDA_Func_Attributes *attr)
{
  int count = 0;

#define GETATTR(func,version)                                                 \
  count++;                                                                    \
  if ( attr ) {                                                               \
      attr->err = hipFuncGetAttributes(&attr->attr,reinterpret_cast<const void*>(func));                    \
      attr->name = #func;                                                     \
      attr->abbrev = version;                                                 \
      attr++;                                                                 \
    }

  GETATTR(stencil,'1');
  GETATTR(stencil_iter,'i');
  GETATTR(stencil_shared,'s');
  GETATTR(stencil_shared_2,'2');
  return count;
#undef GETATTR
}

__host__ int
kernels_get_attr(pCUDA_Func_Attributes **attr)
{
  int count = kernels_get_attr_(NULL);
  *attr = (pCUDA_Func_Attributes*) calloc(count,sizeof(**attr));
  return kernels_get_attr_(*attr);
}


// This routine executes on the CPU.
//
__host__ void
stencil_launch(dim3 dg, dim3 db, int shared_bytes, char version)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
   switch ( version ) {
  case 'i': stencil_iter<<<dg,db>>>(); break;
  case 's': stencil_shared<<<dg,db,shared_bytes>>>(); break;
  case '2': stencil_shared_2<<<dg,db,shared_bytes>>>(); break;
  }
}

__global__ void
stencil()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;

  int col = idx & row_mask;
  int row = idx >> dim_size_lg;
  if ( row == 0 || row >= row_mask || col == 0 || col == row_mask ) return;

  int iu = idx - row_stride;
  int id = idx + row_stride;

  b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
    + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
}

__global__ void
stencil_iter()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;
  int thread_count = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      int col = idx & row_mask;
      int row = idx >> dim_size_lg;
      if ( row == 0 || row >= row_mask || col == 0 || col == row_mask )
        continue;

      int iu = idx - row_stride;
      int id = idx + row_stride;

#ifdef DEBUG_STENCIL
      b[idx] = v0 * a[idx];
#else
      b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
        + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
#endif
    }
}

__global__ void
stencil_shared()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  int block_compute_width = blockDim.x - 2;
  int blocks_per_row =
    ceilf( float(array_row_stride-2) / block_compute_width );

  int rows_per_block =
    ceilf( float(blocks_per_row) * array_row_stride / gridDim.x );

  //  int cols_per_block = ceilf( float(array_row_stride) / blocks_per_row );
  int cols_per_block = blockDim.x - 2;
  if ( threadIdx.x >= cols_per_block + 2 ) return;

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  int siu = threadIdx.x;
  int sidx = siu + blockDim.x;
  int sid = sidx + blockDim.x;

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 =
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * cols_per_block;
      int col_9 = min( col_0 + cols_per_block + 2, array_row_stride );

      int col = col_0 + threadIdx.x;
      if ( col >= array_row_stride ) return;
      int row_0s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      bool load_only =
        threadIdx.x == 0 || threadIdx.x == cols_per_block + 1
        || col == array_row_stride - 1;

      int rows = row_0s;
      int idx = rows | col;
      int idx_stop = row_9s | col;
      s[siu] = a[idx];
      idx += array_row_stride;
      s[sidx] = a[idx];
      while ( idx < idx_stop )
        {
          int idx_next = idx + array_row_stride;
          s[sid] = a[idx_next];
          __syncthreads();
          if ( !load_only )
            {
#if DEBUG_STENCIL
              b[idx] = v0 * s[sidx];
#else
              b[idx] = v0 * s[sidx]
                + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
                + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
            }
          __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          idx = idx_next;
        }

    }
}


__global__ void
stencil_shared_2()
{
  /// Modify this code.

  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  int block_compute_width = blockDim.x - 2;
  int blocks_per_row =
    ceilf( float(array_row_stride-2) / block_compute_width );

  int rows_per_block =
    ceilf( float(blocks_per_row) * array_row_stride / gridDim.x );

  //  int cols_per_block = ceilf( float(array_row_stride) / blocks_per_row );
  int cols_per_block = blockDim.x - 2;
  if ( threadIdx.x >= cols_per_block + 2 ) return;

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  int siu = threadIdx.x;
  int sidx = siu + blockDim.x;
  int sid = sidx + blockDim.x;

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 =
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * cols_per_block;
      int col_9 = min( col_0 + cols_per_block + 2, array_row_stride );

      int col = col_0 + threadIdx.x;
      if ( col >= array_row_stride ) return;
      int row_0s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      bool load_only =
        threadIdx.x == 0 || threadIdx.x == cols_per_block + 1
        || col == array_row_stride - 1;

      int rows = row_0s;
      int idx = rows | col;
      int idx_stop = row_9s | col;
      s[siu] = a[idx];
      idx += array_row_stride;
      s[sidx] = a[idx];
      while ( idx < idx_stop )
        {
          int idx_next = idx + array_row_stride;
          s[sid] = a[idx_next];
          __syncthreads();
          if ( !load_only )
            {
#if DEBUG_STENCIL
              b[idx] = v0 * s[sidx];
#else
              b[idx] = v0 * s[sidx]
                + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
                + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
            }
          __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          idx = idx_next;
        }

    }
}
