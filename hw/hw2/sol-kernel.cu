#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Sp 2011), GPU Microarchitecture
//
/// Homework 2 -- PARTIAL SOLUTION
/// Homework 3 -- Assignment (Edit this file.)

 // This file has a solution to Homework 2 Problem 3(a).
 // Edit this file to solve Homework 3.


#include "sol.cuh"

// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ int row_stride, dim_size_lg, dim_block_lg;
__constant__ float* a;
__constant__ float* b;
__constant__ int homework_R;
extern __shared__ float s[]; // Shared memory for buffering a elements.

__global__ void stencil();
__global__ void stencil_iter();
__global__ void stencil_shared();
__global__ void stencil_shared_2();
__global__ void stencil_shared_3();

static __host__ int
kernels_get_attr_(pCUDA_Func_Attributes *attr)
{
  int count = 0;

#define GETATTR(func,version)                                                 \
  count++;                                                                    \
  if ( attr ) {                                                               \
      attr->err = hipFuncGetAttributes(&attr->attr,reinterpret_cast<const void*>(func));                    \
      attr->name = #func;                                                     \
      attr->abbrev = version;                                                 \
      attr++;                                                                 \
    }

  GETATTR(stencil,'1');
  GETATTR(stencil_iter,'i');
  GETATTR(stencil_shared,'s');
  GETATTR(stencil_shared_2,'2');
  GETATTR(stencil_shared_3,'3');
  return count;
#undef GETATTR
}

__host__ int
kernels_get_attr(pCUDA_Func_Attributes **attr)
{
  int count = kernels_get_attr_(NULL);
  *attr = (pCUDA_Func_Attributes*) calloc(count,sizeof(**attr));
  return kernels_get_attr_(*attr);
}


// This routine executes on the CPU.
//
__host__ void
stencil_launch(dim3 dg, dim3 db, int shared_bytes, char version)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
   switch ( version ) {
  case 'i': stencil_iter<<<dg,db>>>(); break;
  case 's': stencil_shared<<<dg,db,shared_bytes>>>(); break;
  case '2': stencil_shared_2<<<dg,db,shared_bytes>>>(); break;
  case '3': stencil_shared_3<<<dg,db,shared_bytes>>>(); break;
  }
}

__global__ void
stencil()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;

  int col = idx & row_mask;
  int row = idx >> dim_size_lg;
  if ( row == 0 || row >= row_mask || col == 0 || col == row_mask ) return;

  int iu = idx - row_stride;
  int id = idx + row_stride;

  b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
    + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
}

__global__ void
stencil_iter()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row_mask = row_stride - 1;
  int thread_count = blockDim.x * gridDim.x;

  for ( int idx = tid;  idx < array_size;  idx += thread_count )
    {
      int col = idx & row_mask;
      int row = idx >> dim_size_lg;
      if ( row == 0 || row >= row_mask || col == 0 || col == row_mask )
        continue;

      int iu = idx - row_stride;
      int id = idx + row_stride;

#ifdef DEBUG_STENCIL
      b[idx] = v0 * a[idx];
#else
      b[idx] = v0 * a[idx] + v1 * ( a[idx-1] + a[idx+1] + a[iu] + a[id] )
        + v2 * ( a[iu-1] + a[iu+1] + a[id-1] + a[id+1] );
#endif
    }
}

__global__ void
stencil_shared()
{
  // This code operates on a square array of pixels.

  // Compute the array_row_stride, which for this code is also equal
  // to the number of rows and the number of columns.
  //
  int array_row_stride = 1 << dim_size_lg;

  int array_row_mask = array_row_stride - 1;

  // Determine how many pixels each block will compute. The number is
  // based on the fact that the first and last thread of each block
  // never compute a pixel, the other threads compute at most one
  // pixel each.
  //
  int block_compute_width = blockDim.x - 2;

  // Determine how many blocks are needed to compute all of the
  // pixels in one row.  The computation is based on the fact that
  // the first and last pixels in a row should be left untouched.
  //
  int blocks_per_row =
    ceilf( float(array_row_stride-2) / block_compute_width );

  // Determine how many rows each block computes.
  //
  int rows_per_block =
    ceilf( float(blocks_per_row) * array_row_stride / gridDim.x );

  int cols_per_block = block_compute_width;

  // If this thread is past the last column that a block is supposed
  // to compute or load, then return.
  //
  if ( threadIdx.x >= cols_per_block + 2 ) return;

  // Compute a "large" row number for this block. These large
  // row numbers can exceed the number of rows in the array.
  // The actual row number to use is row_0_large mod blockDim.x,
  // the column number is 
  //  ( row_0_large / blockDim.x ) * cols_per_block + threadIdx.x.
  //
  int row_0_large = rows_per_block * blockIdx.x;       // Starting row.
  int row_9_large = row_0_large + rows_per_block + 2;  // Ending row.

  // A column group is a set of columns handled by one block. If
  // blocks_per_row < gridDim.x (the number of blocks) then a block
  // will have to cover more than one sets of columns (put another
  // way, a thread will have to cover more than one column).
  //
  int col_group_0 = row_0_large >> dim_size_lg;        // Starting column.
  int col_group_9 = row_9_large >> dim_size_lg;        // Ending column.

  // Compute indices into shared memory. At any one time shared memory
  // will hold three rows of pixels.  Index siu holds the upper row,
  // sidx holds the middle row (corresponding to the pixel to be written),
  // and sid is the lower (down) row.  A single shared memory array
  // is used to hold all three rows.
  //
  int siu = threadIdx.x;
  int sidx = siu + blockDim.x;
  int sid = sidx + blockDim.x;

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      // The starting row will be zero if this isn't the first column
      // group (because computation reached the bottom of one column
      // and is now wrapping around to the top [row 0] of another column).
      //
      int row_0 =
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;

      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      // Compute the first column number for the block.
      //
      int col_0 = col_group * cols_per_block;

      // Compute the column number for this thread.
      //
      int col = col_0 + threadIdx.x;
      if ( col >= array_row_stride ) return;

      // Shift the row numbers over so that they can easily be used
      // to compute the array idx.
      //
      int row_0s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      // Check whether we should just load a value to shared memory.
      // If load_only is false then we both load the value and compute
      // a pixel.
      //
      bool load_only =
        threadIdx.x == 0 || threadIdx.x == cols_per_block + 1
        || col == array_row_stride - 1;

      // Compute the array index for the "up" row. This will be loaded
      // to shared memory but nothing will be computed for it.
      //
      int idx = row_0s | col;
      s[siu] = a[idx];

      // Increment by the row stride to obtain the address of the middle
      // row. We will compute a value for this in the first iteration
      // of the while loop below.
      //
      idx += array_row_stride;
      s[sidx] = a[idx];

      // Compute the array index at which we should stop.
      //
      int idx_stop = row_9s | col;

      // Compute pixels for column col, starting from row_0s +
      // array_row_stride and ending at row_9s.
      //
      while ( idx < idx_stop )
        {
          // Compute the address of the pixel in the row below us,
          // idx_next, and load it into shared memory.
          //
          int idx_next = idx + array_row_stride;
          s[sid] = a[idx_next];

          // Wait for other threads in this block to finish writing
          // shared memory.
          //
          __syncthreads();

          if ( !load_only )
            {
              // Compute the pixel value and write it to b.
#ifdef DEBUG_STENCIL
              b[idx] = v0 * s[sidx];
#else
              b[idx] = v0 * s[sidx]
                + v1 * ( s[sidx-1] + s[sidx+1] + s[siu] + s[sid] )
                + v2 * ( s[siu-1] + s[siu+1] + s[sid-1] + s[sid+1] );
#endif
            }


          // Wait for other threads in this block to finish reading
          // shared memory.
          //
          __syncthreads();

          // Rotate indices so that what is currently the middle row,
          // sidx, becomes the up row, and what is currently the down
          // row, sid, becomes the current row, and siu will be the sid.
          //
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;

          // Move the idx down one row.
          //
          idx = idx_next;
        }

    }
}


__global__ void
stencil_shared_2()
{
  /// SOLUTION - Homework 2

  /// DO NOT edit this routine for Homework 3, instead modify stencil_shared_3.

  /// NOTE: This solution is inefficient.
  //
  //  The code here will run more slowly than stencil_shared, at least
  //  on CC 1.x devices, due to inefficient global and shared memory
  //  access patterns.
  //
  //  This code does not include unrolling.  That will be added
  //  later to another routine.

  //
  // For the solution to Problem 3a the code here is adjusted so that
  // each thread can operate on R pixels. Look for the "Times R"
  // comments.
  //

  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  // Adjust the number of columns that each block computes.
  //
  int cols_per_block = blockDim.x * homework_R - 2;          //  Times R
  int blocks_per_row =
    ceilf( float(array_row_stride-2) / cols_per_block );

  int rows_per_block =
    ceilf( float(blocks_per_row) * array_row_stride / gridDim.x );

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  // Adjust the indices into shared memory.
  //
  int siu = threadIdx.x * homework_R;                        // Times R
  int sidx = siu + blockDim.x * homework_R;                  // Times R
  int sid = sidx + blockDim.x * homework_R;                  // Times R

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 =
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * cols_per_block;
      int col_9 = min( col_0 + cols_per_block, array_row_stride - 2 );

      // Since each thread handles R pixels need to multiply by R to
      // find the starting column number for a thread.
      //
      int col = col_0 + threadIdx.x * homework_R;            // Times R
      if ( col >= array_row_stride ) return;

      int row_s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      // Use loops to load data.
      //
      for ( int i=0; i<homework_R; i++ )
        {
          // Compute column number for this element.
          //
          int coli = col + i;

          // If column out of range skip this element.
          //
          if ( coli >= array_row_stride ) continue;

          // Compute array index, and cache elements.
          //
          int idx = row_s + coli;
          s[siu+i] = a[ idx ];
          s[sidx+i] = a[ idx + array_row_stride ];
        }

      row_s += array_row_stride;

      while ( row_s < row_9s )
        {
          int row_next = row_s + array_row_stride;
          
          for ( int i=0; i<homework_R; i++ )
            {
              // Cache the next row of elements.
              //
              int coli = col + i;
              if ( coli < array_row_stride ) s[sid + i] = a[ row_next + coli ];
            }

          __syncthreads();

          // Use a loop for computation.
          //
          for ( int i=0; i<homework_R; i++ )
            {
              // Compute column and if out of range, skip this element.
              //
              int coli = col + i;
              if ( coli == col_0 || coli > col_9 ) continue;
              int idx = row_s + coli;
#ifdef DEBUG_STENCIL
              b[idx] = v0 * s[sidx+i];
#else
              b[idx] = v0 * s[sidx+i]
                + v1 * ( s[sidx+i-1] + s[sidx+i+1] + s[siu+i] + s[sid+i] )
                + v2 * ( s[siu+i-1] + s[siu+i+1] + s[sid+i-1] + s[sid+i+1] );
#endif
            }
          __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          row_s = row_next;
        }

    }
}

__global__ void
stencil_shared_3()
{
  /// SOLVE HOMEWORK 3 HERE

  /// NOTE: This code is inefficient.
  //
  //  The code here will run more slowly than stencil_shared, at least
  //  on CC 1.x devices, due to inefficient global and shared memory
  //  access patterns.
  //
  //  Fix it.

  int array_row_stride = 1 << dim_size_lg;
  int array_row_mask = array_row_stride - 1;

  // Adjust the number of columns that each block computes.
  //
  int cols_per_block = blockDim.x * homework_R - 2;          //  Times R
  int blocks_per_row =
    ceilf( float(array_row_stride-2) / cols_per_block );

  int rows_per_block =
    ceilf( float(blocks_per_row) * array_row_stride / gridDim.x );

  int row_0_large = rows_per_block * blockIdx.x;
  int row_9_large = row_0_large + rows_per_block + 2;
  int col_group_0 = row_0_large >> dim_size_lg;
  int col_group_9 = row_9_large >> dim_size_lg;

  // Adjust the indices into shared memory.
  //
  int siu = threadIdx.x * homework_R;                        // Times R
  int sidx = siu + blockDim.x * homework_R;                  // Times R
  int sid = sidx + blockDim.x * homework_R;                  // Times R

  for ( int col_group = col_group_0; col_group <= col_group_9; col_group++ )
    {
      int row_0 =
        col_group == col_group_0 ? row_0_large & array_row_mask : 0;
      int row_9 = col_group == col_group_9
        ? ( row_9_large & array_row_mask ) : array_row_mask;

      int col_0 = col_group * cols_per_block;
      int col_9 = min( col_0 + cols_per_block, array_row_stride - 2 );

      // Since each thread handles R pixels need to multiply by R to
      // find the starting column number for a thread.
      //
      int col = col_0 + threadIdx.x * homework_R;            // Times R
      if ( col >= array_row_stride ) return;

      int row_s = row_0 << dim_size_lg;
      int row_9s = row_9 << dim_size_lg;

      // Use loops to load data.
      //
      for ( int i=0; i<homework_R; i++ )
        {
          // Compute column number for this element.
          //
          int coli = col + i;

          // If column out of range skip this element.
          //
          if ( coli >= array_row_stride ) continue;

          // Compute array index, and cache elements.
          //
          int idx = row_s + coli;
          s[siu+i] = a[ idx ];
          s[sidx+i] = a[ idx + array_row_stride ];
        }

      row_s += array_row_stride;

      while ( row_s < row_9s )
        {
          int row_next = row_s + array_row_stride;
          
          for ( int i=0; i<homework_R; i++ )
            {
              // Cache the next row of elements.
              //
              int coli = col + i;
              if ( coli < array_row_stride ) s[sid + i] = a[ row_next + coli ];
            }

          __syncthreads();

          // Use a loop for computation.
          //
          for ( int i=0; i<homework_R; i++ )
            {
              // Compute column and if out of range, skip this element.
              //
              int coli = col + i;
              if ( coli == col_0 || coli > col_9 ) continue;

              int idx = row_s + coli;
#ifdef DEBUG_STENCIL
              b[idx] = v0 * s[sidx+i];
#else
              b[idx] = v0 * s[sidx+i]
                + v1 * ( s[sidx+i-1] + s[sidx+i+1] + s[siu+i] + s[sid+i] )
                + v2 * ( s[siu+i-1] + s[siu+i+1] + s[sid+i-1] + s[sid+i+1] );
#endif
            }
          __syncthreads();
          int sid_new = siu; siu = sidx; sidx = sid; sid = sid_new;
          row_s = row_next;
        }

    }
}
