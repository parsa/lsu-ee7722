#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2012), GPU Programming
//

 /// Homework 5
//
//   See hw5.cc for details.

#include "cuda-util.cu"
#include "hw5-sol.cuh"
#include <gp/cuda-util-kernel.h>

// Physical State Variables
//
__constant__ float3 *helix_position;
__constant__ float3 *helix_velocity;
__constant__ pQuat *helix_orientation;
__constant__ float3 *helix_omega;

// Scalar Constants (Placed in a struct for convenience.)
//
__constant__ Helix_Info hi;

__global__ void time_step();

__host__ hipError_t
cuda_setup(struct hipFuncAttributes *attr_helix)
{
  // Pass the device address to host code. (See gp/cuda-util-kernel.h ).
  CU_SYM(helix_position);
  CU_SYM(helix_velocity);
  CU_SYM(helix_orientation);
  CU_SYM(helix_omega);
  CU_SYM(hi);

  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.
  hipError_t e1 = hipFuncGetAttributes(attr_helix,reinterpret_cast<const void*>(time_step));
  if ( e1 ) return e1;
  return e1;
}

__host__ void time_step_launch(int grid_size, int block_size)
{
  time_step<<<grid_size,block_size>>>();
}

__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude);


__global__ void
time_step()
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Use tid for helix segment number.

  if ( tid + 1 > hi.phys_helix_segments ) return;

  // The position of segment 0 is fixed, so don't evolve it.
  if ( tid == 0 ) return;

  pVect vZero = mv(0,0,0);
  pVect gravity_force = hi.helix_seg_mass_inv * hi.gravity_accel;

  pQuat c_orientation = helix_orientation[tid];
  float3 c_position = helix_position[tid];

  pMatrix3x3 c_rot;
  // Initialize c_rot to a rotation matrix based on quaternion c_orientation.
  pMatrix_set_rotation(c_rot,c_orientation);

  float3 c_u = c_rot * mv(0,0,1);  // mv: Make Vector.
  float3 c_v = c_rot * mv(0,1,0);
  float3 c_ctr_to_right_dir = c_rot * mv(1,0,0);
  pVect c_ctr_to_right = hi.helix_seg_hlength * c_ctr_to_right_dir;
  float3 c_pos_right = c_position + c_ctr_to_right;
  float3 c_pos_left = c_position - c_ctr_to_right;

  float3 force = hi.opt_gravity ? gravity_force : vZero;
  float3 torque = vZero;

  const int pieces = 3;
  const float delta_theta = 2 * M_PI / pieces;

  /// Compute forces due to right neighbor.
  //
  if ( tid + 1 < hi.phys_helix_segments )
    {
      pQuat r_orientation = helix_orientation[tid+1];
      float3 r_position = helix_position[tid+1];
      pMatrix3x3 r_rot;
      pMatrix_set_rotation(r_rot,r_orientation);
      float3 r_u = r_rot * mv(0,0,1);
      float3 r_v = r_rot * mv(0,1,0);
      float3 r_ctr_to_right_dir = r_rot * mv(1,0,0);
      pVect r_ctr_to_right = hi.helix_seg_hlength * r_ctr_to_right_dir;
      float3 r_pos_left = r_position - r_ctr_to_right;

      pQuat cn_rot_q = c_orientation * hi.helix_rn_trans;
      pMatrix3x3 cn_rot;
      pMatrix_set_rotation(cn_rot,cn_rot_q);
      pVect n_ru = cn_rot * mv(0,0,1);
      pVect n_rv = cn_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_right + cosf(theta) * n_ru + sinf(theta) * n_rv;
          pCoor r_pt = r_pos_left + cosf(theta) * r_u + sinf(theta) * r_v;
          pNorm dist = mn(c_pt,r_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// Compute forces due to left neighbor.
  //
  if ( tid > 0 )
    {
      pQuat l_orientation = helix_orientation[tid-1];
      float3 l_position = helix_position[tid-1];
      pMatrix3x3 l_rot;
      pMatrix_set_rotation(l_rot,l_orientation);
      float3 l_u = l_rot * mv(0,0,1);
      float3 l_v = l_rot * mv(0,1,0);
      float3 l_ctr_to_right_dir = l_rot * mv(1,0,0);
      pVect l_ctr_to_right = hi.helix_seg_hlength * l_ctr_to_right_dir;
      float3 l_pos_right = l_position + l_ctr_to_right;

      pQuat ln_rot_q = l_orientation * hi.helix_rn_trans;
      pMatrix3x3 ln_rot;
      pMatrix_set_rotation(ln_rot,ln_rot_q);
      pVect n_cu = ln_rot * mv(0,0,1);
      pVect n_cv = ln_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_left + cosf(theta) * c_u + sinf(theta) * c_v;
          pCoor l_pt = l_pos_right + cosf(theta) * n_cu + sinf(theta) * n_cv;
          pNorm dist = mn(c_pt,l_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// SOLUTION - Problem 1
  if ( hi.opt_interpen_method == 1 )
    {
      const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
      const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;
      for ( int j=0; j<hi.phys_helix_segments; j++ )
        {
          float3 b_position = helix_position[j];
          pVect ab = mv(c_position,b_position);
          if ( abs(tid-j) < min_idx_dist ) continue;
          if ( mag_sq(ab) < four_wire_radius_sq )
            {
              pNorm dist = mn(ab);
              const float pen = 2 * hi.wire_radius - dist.magnitude;
              force -= pen * hi.opt_spring_constant * dist;
            }
        }
    }

  float3 velocity = helix_velocity[tid];
  velocity *= 0.9999f;
  float3 omega = helix_omega[tid];
  omega *= 0.9999f;
  velocity += hi.delta_t_mass_inv * force;
  const float torque_axial_mag = dot( torque, c_ctr_to_right_dir );
  pVect torque_axial = torque_axial_mag * c_ctr_to_right_dir;
  pVect do_axial = hi.delta_t_ma_axis * torque_axial;
  pVect torque_other = torque - torque_axial;
  pVect do_other = hi.delta_t_ma_perp_axis * torque_other;
  omega += do_axial + do_other;

  // Update velocity and omega. Don't update position or orientation
  // because we don't want threads in this kernel to accidentally read
  // the updated values.

  helix_omega[tid] = omega;
  helix_velocity[tid] = velocity;
}


__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude)
{
  // Update force and torque of segment for a force acting on FORCE_POS
  // pointing in direction DIR of magnitude MAGNITUDE.
  //
  force += magnitude * dir;
  pVect arm = mv(position,force_pos);
  pVect axis = cross( arm, dir );
  pVect amt = magnitude * axis;
  torque += amt;
}

 /// SOLUTION -- Problem 2
//
__global__ void
time_step_intersect()
{
  // Find intersections of one helix segment with some other segments.
  // Each block handles one "a" segment, the threads in the block
  //  check for intersection with other segments.
  // The kernel is launched with one block per segment.

  int a_idx = blockIdx.x;
  int b_idx_start = threadIdx.x;

  if ( a_idx == 0 ) return;

  float3 a_position = helix_position[a_idx];

  const int min_idx_dist = 0.999f + hi.wire_radius / hi.helix_seg_hlength;
  const float four_wire_radius_sq = 4 * hi.wire_radius * hi.wire_radius;
  __shared__ pVect force;

  if ( threadIdx.x == 0 ) force = mv(0,0,0);

  // Wait for thread 0 to initialize force.
  __syncthreads();

  for ( int j=b_idx_start; j<hi.phys_helix_segments; j += blockDim.x )
    {
      float3 b_position = helix_position[j];
      pVect ab = mv(a_position,b_position);

      // Skip if segment is too close.
      if ( abs(a_idx-j) < min_idx_dist ) continue;

      // Skip if no chance of intersection.
      if ( mag_sq(ab) >= four_wire_radius_sq ) continue;

      // Compute intersection force based on bounding sphere, an
      // admittedly crude approximation.
      //
      pNorm dist = mn(ab);
      const float pen = 2 * hi.wire_radius - dist.magnitude;
      float3 f = pen * hi.opt_spring_constant * dist;

      // Add force to shared variable. This is time consuming but
      // done infrequently. (A segment can normally only intersect a
      // a few other segments.
      //
      atomicAdd(&force.x,f.x);
      atomicAdd(&force.y,f.y);
      atomicAdd(&force.z,f.z);
      //
      // Optimization Note: Could acquire a lock and then update
      // all three components.
    }

  // Wait for all threads to finish.
  __syncthreads();

  // Leave it to thread 0 to update velocity.
  if ( threadIdx.x != 0 ) return;

  // Update velocity and write it.
  //
  float3 velocity = helix_velocity[a_idx];
  velocity -= hi.delta_t_mass_inv * force;
  if ( hi.opt_end_fixed && a_idx + 1 == hi.phys_helix_segments )
    velocity = mv(0,0,0);
  helix_velocity[a_idx] = velocity;
}

__host__ void
time_step_intersect_launch(int grid_size, int block_size)
{
  time_step_intersect<<<grid_size,block_size>>>();
}

__global__ void
time_step_update_pos()
{
  // Update position and orientation of spring segments.

  // Use tid for helix segment number.
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Skip out-of-range segments.
  if ( tid >= hi.phys_helix_segments ) return;
  if ( tid == 0 ) return;

  // Update Orientation
  //
  pQuat orientation = helix_orientation[tid];
  float3 omega = helix_omega[tid];
  pNorm axis = mn(omega);
  helix_orientation[tid] =
    quat_normalize
    ( quat_mult ( mq( axis, hi.delta_t * axis.magnitude ), orientation));

  // Return if at last segment and it is fixed. Note that even
  // if the segment's position is fixed, it can still rotate.
  //
  if ( hi.opt_end_fixed && tid + 1 == hi.phys_helix_segments ) return;

  // Update Velocity
  //
  float3 position = helix_position[tid];
  float3 velocity = helix_velocity[tid];
  helix_position[tid] = position + hi.delta_t * velocity;
}

__host__ void 
time_step_update_pos_launch
(int grid_size, int block_size)
{
  time_step_update_pos<<<grid_size,block_size>>>();
}
