#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2012), GPU Programming
//

 /// Homework 5
//
//   See hw5.cc for details.

#include "cuda-util.cu"
#include "hw5.cuh"
#include <gp/cuda-util-kernel.h>

// Physical State Variables
//
__constant__ float3 *helix_position;
__constant__ float3 *helix_velocity;
__constant__ pQuat *helix_orientation;
__constant__ float3 *helix_omega;

// Scalar Constants (Placed in a struct for convenience.)
//
__constant__ Helix_Info hi;

__global__ void time_step();

__host__ hipError_t
cuda_setup(struct hipFuncAttributes *attr_helix)
{
  // Pass the device address to host code. (See gp/cuda-util-kernel.h ).
  CU_SYM(helix_position);
  CU_SYM(helix_velocity);
  CU_SYM(helix_orientation);
  CU_SYM(helix_omega);
  CU_SYM(hi);

  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.
  hipError_t e1 = hipFuncGetAttributes(attr_helix,reinterpret_cast<const void*>(time_step));
  if ( e1 ) return e1;
  return e1;
}

__host__ void time_step_launch(int grid_size, int block_size)
{
  time_step<<<grid_size,block_size>>>();
}

__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude);


__global__ void
time_step()
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Use tid for helix segment number.

  if ( tid + 1 > hi.phys_helix_segments ) return;

  // The position of segment 0 is fixed, so don't evolve it.
  if ( tid == 0 ) return;

  pVect vZero = mv(0,0,0);
  pVect gravity_force = hi.helix_seg_mass_inv * hi.gravity_accel;

  pQuat c_orientation = helix_orientation[tid];
  float3 c_position = helix_position[tid];

  pMatrix3x3 c_rot;
  // Initialize c_rot to a rotation matrix based on quaternion c_orientation.
  pMatrix_set_rotation(c_rot,c_orientation);

  float3 c_u = c_rot * mv(0,0,1);  // mv: Make Vector.
  float3 c_v = c_rot * mv(0,1,0);
  float3 c_ctr_to_right_dir = c_rot * mv(1,0,0);
  pVect c_ctr_to_right = hi.helix_seg_hlength * c_ctr_to_right_dir;
  float3 c_pos_right = c_position + c_ctr_to_right;
  float3 c_pos_left = c_position - c_ctr_to_right;

  float3 force = hi.opt_gravity ? gravity_force : vZero;
  float3 torque = vZero;

  const int pieces = 3;
  const float delta_theta = 2 * M_PI / pieces;

  /// Compute forces due to right neighbor.
  //
  if ( tid + 1 < hi.phys_helix_segments )
    {
      pQuat r_orientation = helix_orientation[tid+1];
      float3 r_position = helix_position[tid+1];
      pMatrix3x3 r_rot;
      pMatrix_set_rotation(r_rot,r_orientation);
      float3 r_u = r_rot * mv(0,0,1);
      float3 r_v = r_rot * mv(0,1,0);
      float3 r_ctr_to_right_dir = r_rot * mv(1,0,0);
      pVect r_ctr_to_right = hi.helix_seg_hlength * r_ctr_to_right_dir;
      float3 r_pos_left = r_position - r_ctr_to_right;

      pQuat cn_rot_q = c_orientation * hi.helix_rn_trans;
      pMatrix3x3 cn_rot;
      pMatrix_set_rotation(cn_rot,cn_rot_q);
      pVect n_ru = cn_rot * mv(0,0,1);
      pVect n_rv = cn_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_right + cosf(theta) * n_ru + sinf(theta) * n_rv;
          pCoor r_pt = r_pos_left + cosf(theta) * r_u + sinf(theta) * r_v;
          pNorm dist = mn(c_pt,r_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// Compute forces due to left neighbor.
  //
  if ( tid > 0 )
    {
      pQuat l_orientation = helix_orientation[tid-1];
      float3 l_position = helix_position[tid-1];
      pMatrix3x3 l_rot;
      pMatrix_set_rotation(l_rot,l_orientation);
      float3 l_u = l_rot * mv(0,0,1);
      float3 l_v = l_rot * mv(0,1,0);
      float3 l_ctr_to_right_dir = l_rot * mv(1,0,0);
      pVect l_ctr_to_right = hi.helix_seg_hlength * l_ctr_to_right_dir;
      float3 l_pos_right = l_position + l_ctr_to_right;

      pQuat ln_rot_q = l_orientation * hi.helix_rn_trans;
      pMatrix3x3 ln_rot;
      pMatrix_set_rotation(ln_rot,ln_rot_q);
      pVect n_cu = ln_rot * mv(0,0,1);
      pVect n_cv = ln_rot * mv(0,1,0);

      for ( int j=0; j<pieces; j++ )
        {
          const float theta = delta_theta * j;
          pCoor c_pt = c_pos_left + cosf(theta) * c_u + sinf(theta) * c_v;
          pCoor l_pt = l_pos_right + cosf(theta) * n_cu + sinf(theta) * n_cv;
          pNorm dist = mn(c_pt,l_pt);
          const float force_mag = dist.magnitude * hi.opt_spring_constant;
          helix_apply_force_at(c_position,force,torque,c_pt,dist.v,force_mag);
        }
    }

  /// Use forces to update velocity, omega, position, and orientation.

  float3 velocity = helix_velocity[tid];
  velocity *= 0.9999f;
  float3 omega = helix_omega[tid];
  omega *= 0.9999f;
  velocity += hi.delta_t_mass_inv * force;
  const float torque_axial_mag = dot( torque, c_ctr_to_right_dir );
  pVect torque_axial = torque_axial_mag * c_ctr_to_right_dir;
  pVect do_axial = hi.delta_t_ma_axis * torque_axial;
  pVect torque_other = torque - torque_axial;
  pVect do_other = hi.delta_t_ma_perp_axis * torque_other;
  omega += do_axial + do_other;

  // Update velocity and omega. Don't update position or orientation
  // because we don't want threads in this kernel to accidentally read
  // the updated values.

  helix_omega[tid] = omega;
  helix_velocity[tid] = velocity;
}


__device__ void
helix_apply_force_at
(float3 position, float3& force, float3& torque,
 float3 force_pos, pVect dir, float magnitude)
{
  // Update force and torque of segment for a force acting on FORCE_POS
  // pointing in direction DIR of magnitude MAGNITUDE.
  //
  force += magnitude * dir;
  pVect arm = mv(position,force_pos);
  pVect axis = cross( arm, dir );
  pVect amt = magnitude * axis;
  torque += amt;
}


__global__ void
time_step_update_pos()
{
  // Update position and orientation of spring segments.

  // Use tid for helix segment number.
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Skip out-of-range segments.
  if ( tid >= hi.phys_helix_segments ) return;
  if ( tid == 0 ) return;

  // Update Orientation
  //
  pQuat orientation = helix_orientation[tid];
  float3 omega = helix_omega[tid];
  pNorm axis = mn(omega);
  helix_orientation[tid] =
    quat_normalize
    ( quat_mult ( mq( axis, hi.delta_t * axis.magnitude ), orientation));

  // Return if at last segment and it is fixed. Note that even
  // if the segment's position is fixed, it can still rotate.
  //
  if ( hi.opt_end_fixed && tid + 1 == hi.phys_helix_segments ) return;

  // Update Velocity
  //
  float3 position = helix_position[tid];
  float3 velocity = helix_velocity[tid];
  helix_position[tid] = position + hi.delta_t * velocity;
}

__host__ void 
time_step_update_pos_launch
(int grid_size, int block_size)
{
  time_step_update_pos<<<grid_size,block_size>>>();
}
