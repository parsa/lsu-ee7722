#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2014), GPU Microarchitecture
//

 /// Homework 3 (Spring 2014)  SOLUTION
 //
 // Assignment in: http://www.ece.lsu.edu/koppel/gp/2014/hw03.pdf
 //

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include "util.h"


struct App_Common
{
  int length;
  int num_pieces;
  int idx_array_elts;
  int data_array_elts;
  int *idx_array;
  float *data_array;
  float *sum_array;
  float *sum_array_check;

  int *d_idx_array;
  int4 *d_idx_array4;   /// SOLUTION Problem 2
  float *d_data_array;
  float *d_sum_array;
};


// In host address space.
App_Common app;

// In device constant address space.
__constant__ App_Common dapp;


const int clength = 8;

extern "C" __global__ void
sums_0()
{
  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
    {
      const int idx_piece_start = piece * clength;
      float sum = 0;
      for ( int i=0; i<clength; i++ )
        sum += dapp.d_data_array[dapp.d_idx_array[idx_piece_start+i]];

      dapp.d_sum_array[piece] = sum;
    }
}

extern "C" __global__ void
sums_1()
{
  /// PROBLEM 2 SOLUTION
  //  Use type punning so that idx_array accessed using vector loads.

  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
    {
      const int idx_piece4_start = piece * clength / 4;
      float sum = 0;
      for ( int i=0; i<clength/4; i++ )
        {
          int4 indices = dapp.d_idx_array4[idx_piece4_start+i];
          sum += dapp.d_data_array[indices.x];
          sum += dapp.d_data_array[indices.y];
          sum += dapp.d_data_array[indices.z];
          sum += dapp.d_data_array[indices.w];
        }

      dapp.d_sum_array[piece] = sum;
    }
}

extern "C" __global__ void
sums_2()
{
  /// PROBLEM 3 SOLUTION
  //  Reduce noncontiguous access to dapp.d_data_array using shared memory.

  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int dcache_elts = ( 1 << 15 ) >> 2;
  const int nrounds = ( dapp.data_array_elts + dcache_elts -1 ) / dcache_elts;

  for ( int round = 0; round < nrounds; round++ )
    {
      __shared__ float dcache[dcache_elts];
      const int chunk_start = round * dcache_elts;
      if ( round != 0 ) __syncthreads();
      for ( int sdidx = threadIdx.x; sdidx < dcache_elts; sdidx += blockDim.x )
        dcache[sdidx] = dapp.d_data_array[chunk_start + sdidx];
      __syncthreads();

      for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
        {
          const int idx_piece_start = piece * clength;
          float sum = 0;
          for ( int i=0; i<clength; i++ )
            {
              const int didx = dapp.d_idx_array[idx_piece_start+i];
              const unsigned int sidx = didx - chunk_start;
              if ( sidx < dcache_elts ) sum += dcache[sidx];
            }
          if ( round == 0 )
            dapp.d_sum_array[piece] = sum;
          else
            dapp.d_sum_array[piece] += sum;
        }
    }
}

extern "C" __global__ void
sums_3()
{
  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int dcache_elts = ( 1 << 15 ) >> 2;
  const int nrounds = ( dapp.data_array_elts + dcache_elts -1 ) / dcache_elts;
  __shared__ float dcache[dcache_elts];

  const int ppi = 4;
  const int work_per_iter = thread_count * ppi;
  const int max_iters = ( dapp.num_pieces + work_per_iter - 1 ) / work_per_iter;

  const int lidx_size = ppi * clength;

  for ( int p = 0;  p < max_iters;  p++ )
    {
      const int piece_start = tid * ppi + p * work_per_iter;
      const int idx_piece_start = piece_start * clength;

      int4 l4[ lidx_size / 4 ];
      float sums[ ppi ];
      for ( int i=0; i<lidx_size; i += 4 )
        l4[ i/4 ] = *(int4*)&dapp.d_idx_array[idx_piece_start+i];
      int *lidx = (int*)l4;

      for ( int i=0; i<ppi; i++ ) sums[i] = 0;

      for ( int round = 0; round <nrounds; round++ )
        {
          const int chunk_start = round * dcache_elts;
          __syncthreads();
          for ( int sdidx = threadIdx.x; sdidx < dcache_elts;
                sdidx += blockDim.x )
            dcache[sdidx] = dapp.d_data_array[chunk_start + sdidx];
          __syncthreads();

          for ( int j=0; j<ppi; j++ )
            {
              float sum = 0;
              for ( int i=0; i<clength; i++ )
                {
                  const int didx = lidx[ j*clength + i ];
                  const unsigned int sidx = didx - chunk_start;
                  if ( sidx < dcache_elts ) sum += dcache[sidx];
                }
              sums[j] += sum;
            }
        }
      for ( int j=0; j<ppi; j++ )
        {
          const int piece = piece_start + j;
          if ( piece < dapp.num_pieces ) dapp.d_sum_array[piece] = sums[j];
        }
    }
}


GPU_Info
print_gpu_info()
{
  GPU_Info info;

  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  CE( hipGetDeviceCount(&device_count) ); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      info.get_gpu_info(dev);  // Look in file util.h.
      hipDeviceProp_t& cuda_prop = info.cuda_prop;

      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         info.cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize >> 10,
         info.chip_bw_Bps * 1e-9,
         info.chip_sp_flops * 1e-9,
         4 * info.chip_sp_flops / info.chip_bw_Bps);
    }

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(sums_0);
  info.GET_INFO(sums_1);
  info.GET_INFO(sums_2);
  info.GET_INFO(sums_3);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }

  return info;
}

int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_info();

  // Examine argument 1, grid size.
  //
  const int arg1_int =
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks = abs(arg1_int);

  // Examine argument 2, number of pieces. Fractional values okay.
  //
  app.num_pieces = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );

  if ( num_blocks <= 0 || app.num_pieces <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [NUMBER OF PIECES]\n",
             argv[0]);
      exit(1);
    }

  app.length = clength;
  app.idx_array_elts = app.num_pieces * app.length;
  app.data_array_elts = 1 << 14;

  const int idx_array_bytes = app.idx_array_elts * sizeof(app.idx_array[0]);
  const int data_array_bytes = app.data_array_elts * sizeof(app.data_array[0]);
  const int sum_array_bytes = app.num_pieces * sizeof(app.sum_array[0]);

  // Allocate storage for CPU copy of data.
  //
  app.idx_array = new int[idx_array_bytes];
  app.data_array = new float[data_array_bytes];
  app.sum_array = new float[sum_array_bytes];
  app.sum_array_check = new float[sum_array_bytes];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_idx_array, idx_array_bytes ) );
  CE( hipMalloc( &app.d_data_array, data_array_bytes ) );
  CE( hipMalloc( &app.d_sum_array, sum_array_bytes ) );

  /// SOLUTION  -  Problem 2
  app.d_idx_array4 = (int4*)app.d_idx_array;

  printf("\nData array size %d elements, index array size %d elements.\n",
         app.data_array_elts, app.idx_array_elts);

  // Initialize input array.
  //
  for ( int i=0; i<app.data_array_elts; i++ )
    {
      app.data_array[i] = drand48();
    }

  for ( int i=0; i<app.idx_array_elts; i++ )
    {
      app.idx_array[i] = random() % app.data_array_elts;
    }

  // Compute correct answer (for checking).
  //
  for ( int p=0; p<app.num_pieces; p++ )
    {
      float sum = 0;
      int idx_base = p * app.length;
      for ( int i=0; i<app.length; i++ )
        sum += app.data_array[app.idx_array[idx_base+i]];
      app.sum_array_check[p] = sum;
    }

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy( app.d_idx_array, app.idx_array, idx_array_bytes,
                  hipMemcpyHostToDevice ) );
  CE( hipMemcpy( app.d_data_array, app.data_array, data_array_bytes,
                  hipMemcpyHostToDevice ) );

  /// SOLUTION Problem 1

  const int idx_array_elts = app.idx_array_elts;

  // Amount of data transferred if all elts in a request are used.
  // For sum_0 and sum_1 only some elts are used, so this ideal
  // amount is not the answer to Problem 1.
  //
  const double __attribute__((unused)) data_size_ideal =
    idx_array_bytes + idx_array_elts * sizeof(app.data_array[0])
    + sum_array_bytes;

  const bool have_l1_cache = info.cuda_prop.major == 2;
  const int l1_cache_size = 3 << 14;

  // Compute the probability that an access to a data array element
  // hits the cache (after the cache has warmed up). This is easy to
  // compute because elements are accessed randomly.
  //
  const double warm_hit_ratio = 
    min( 1.0, double(l1_cache_size) / data_array_bytes );

  const int req_size = 32;
  const int line_size = 128;

  // Compute the amount of data accessed assuming that idx_array_elts
  // is large and that data_array_elts is much larger than 32 (the
  // warp size). Each index array element is read once (this is
  // certainly true for sums_1 and is true for sums_0 if the cache
  // cooperates). Each sum_array element is written once. The data
  // array is accessed idx_array_elts times. Using the cache hit ratio
  // and the line size one can find the amount of data transferred. If
  // there is no L1 cache then assume that every accesses brings in a
  // 32-byte request.
  //
  const double data_size_sums_0 =
    have_l1_cache 
    ? ( idx_array_bytes 
        + idx_array_elts * line_size * ( 1 - warm_hit_ratio )
        + sum_array_bytes
        )
    : ( idx_array_bytes + idx_array_elts * req_size + sum_array_bytes );

  // Amount of data when data_array cached.
  //
  const int elt_per_round = ( 1 << 15 ) / sizeof(app.data_array[0]);
  const int rounds = ( app.data_array_elts + elt_per_round -1 ) / elt_per_round;
  const double data_size_sums_2 =
    rounds * idx_array_bytes + data_array_bytes + sum_array_bytes;

  // Stuff needed to print bar graph.
  //
  const int full_width = 49;  // Width of longest bar.
  char **stars = (char**) alloca( info.num_kernels * sizeof(void*) );
  for ( int j=0; j< info.num_kernels; j++ )
    {
      stars[j] = (char*) alloca(full_width+1);
      for ( int i=0; i<full_width; i++ ) stars[j][i] = '0' + j;
      stars[j][full_width] = 0;
    }

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      hipFuncAttributes& cfa = info.ki[kernel].cfa;

      if ( kernel > 0 && cfa.numRegs < 3 )
        {
          printf("Not running kernel %s because code probably not present.\n",
                info.ki[kernel].name);
          continue;
        }

      // Maximum number of warps per block that this GPU can handle
      // for this kernel.
      //
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      printf("\nRunning kernel %s which uses %d regs on %d blocks.\n",
             info.ki[kernel].name, cfa.numRegs, num_blocks);

      printf(" %3s %9s  %6s  %5s\n", "Num", "Time", "Data", "" );
      printf(" %3s %9s  %6s  %5s\n", "Wps", "µs", " GB/s", "Pct" );

      for ( int warp_cnt = 2; warp_cnt <= wp_limit; warp_cnt += 2 )
        {
          const int thd_per_block = warp_cnt << 5;

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          // Zero result array (to catch errors when kernel skips elements).
          //
          CE( hipMemset( app.d_sum_array, 0, sum_array_bytes ) );

          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
          const double elapsed_time_s = cuda_time_ms * 0.001;

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (app.sum_array, app.d_sum_array, sum_array_bytes,
               hipMemcpyDeviceToHost) );

          int err = 0;
          for ( int e=0; e<app.num_pieces; e++ )
            {
              const float diff =
                fabs( app.sum_array[e] - app.sum_array_check[e] );
              if ( diff > 0.00001 )
                {
                  err++;
                  if ( err < 2 )
                    printf
                      ("Error at array idx %d: %f != %f (correct)\n",
                       e, app.sum_array[e], app.sum_array_check[e]);
                }
            }

          const double data_size = 
            kernel < 2 ? data_size_sums_0 : data_size_sums_2;

          const double frac = data_size / elapsed_time_s / info.chip_bw_Bps;
          const int s_idx = max(0.0,full_width * ( 1 - frac ));

          printf(" %2d  %9.3f  %6.2f %5.1f%%  %s\n",
                 warp_cnt, 1e6 * elapsed_time_s,
                 1e-9 * data_size / elapsed_time_s,
                 100 * frac, &stars[kernel][s_idx] );
        }
    }
}
