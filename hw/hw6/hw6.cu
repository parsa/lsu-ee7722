#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Spring 2013), GPU Microarchitecture
//

 /// Homework 6
 //
 // Assignment in: http://www.ece.lsu.edu/koppel/gp/2013/hw06.pdf
 //
 /// Your Name:

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>

// Size of vectors.
#define N 2

 /// CUDA API Error-Checking Wrapper
///
#define CE(call)                                                              \
 {                                                                            \
   const hipError_t rv = call;                                               \
   if ( rv != hipSuccess )                                                   \
     {                                                                        \
       printf("CUDA error %d, %s\n",rv,hipGetErrorString(rv));               \
       exit(1);                                                               \
     }                                                                        \
 }

double
time_fp()
{
  struct timespec tp;
  clock_gettime(CLOCK_REALTIME,&tp);
  return ((double)tp.tv_sec)+((double)tp.tv_nsec) * 0.000000001;
}

// Make it easy to switch between float and double for vertex and matrix
// elements.
//
typedef float Elt_Type;

struct __align__(4 * N) Vertex
{
  Elt_Type a[N];
};

#if N == 4
__device__ Vertex make_vertex(float4 f4)
{
  Vertex p;
  p.a[0]=f4.x; p.a[1]=f4.y; p.a[2]=f4.z; p.a[3]=f4.w;
  return p;
}
#endif

__device__ Vertex make_vertex(float2 f)
{
  Vertex p;
  p.a[0]=f.x; p.a[1]=f.y;
  return p;
}

struct App
{
  int num_threads;
  Elt_Type matrix[N][N];
  int array_size;  // Number of vertices.
  Vertex *v_in, *v_out, *v_out_cpu;
#if N == 4
  float4 *d_v_in_f4;
#endif
#if N == 2
  float2 *d_v_in_f4;
#endif
  // Note: Undeclared variable error if N not 2 or 4.
  Vertex *d_v_in;
  Vertex *d_v_out;
};

// In host address space.
App app;

// In device constant address space.
__constant__ App d_app;


//
// Perform vertex transformation assuming lots of threads per block.
//
extern "C" __global__ void
kernel_many_threads()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const int num_threads = d_app.num_threads;
  const int array_size = d_app.array_size;

  // Prepare a vertex holding the constant zero.
  Vertex zero; for ( int i=0; i<N; i++ ) zero.a[i] = 0;

  for ( int h=tid;  h<array_size;  h += num_threads )
    {
      // Load a vertex from global memory. Loaded as a float2 so that
      // the compiler uses a vector load, operated on as a Vertex for
      // coding convenience.
      //
      Vertex p = make_vertex(d_app.d_v_in_f4[h]);

      Vertex q = zero;

      // Perform the transformation.
      //
      for ( int i=0; i<N; i++ )
        for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];

      // And write the result.
      //
      d_app.d_v_out[h] = q;
    }
}


//
// Perform vertex transformation assuming fewer threads.
//
// Degree is number of times that loop unrolled.
//
template<int DEGREE>
__device__ void
kernel_few_threads_d()
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  const int num_threads = d_app.num_threads;
  const int array_size = d_app.array_size;
  Vertex zero; for ( int i=0; i<N; i++ ) zero.a[i] = 0;

  const int safe_limit = array_size - DEGREE * num_threads;
  int h = tid;

  // Make sure compiler doesn't try to unroll it further.
# pragma unroll 1

  // Main loop, each iteration operates on DEGREE vertices.
  //
  for ( ; h<safe_limit;  h += DEGREE * num_threads )
    {
      // Load the vertices.
      //
      Vertex p[DEGREE];
      for ( int i=0; i<DEGREE; i++ )
        p[i] = make_vertex(d_app.d_v_in_f4[h + i * num_threads]);

      // Transform and store them.
      //
      for ( int l=0; l<DEGREE; l++ )
        {
          Vertex q = zero;
          for ( int i=0; i<N; i++ )
            for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p[l].a[j];
          d_app.d_v_out[h+l*num_threads] = q;
        }
    }

  // Perform the last few iterations.
  //
  for ( ; h < array_size; h += num_threads )
    {
      Vertex p = make_vertex(d_app.d_v_in_f4[h]);
      Vertex q = zero;
      for ( int i=0; i<N; i++ )
        for ( int j=0; j<N; j++ ) q.a[i] += d_app.matrix[i][j] * p.a[j];
      d_app.d_v_out[h] = q;
    }
}

// Declare Kernels with varying unroll degrees.
//
extern "C" __global__ void kernel_few_threads_d2()
{ kernel_few_threads_d<2>(); }
extern "C" __global__ void kernel_few_threads_d4()
{ kernel_few_threads_d<4>(); }
extern "C" __global__ void kernel_few_threads_d8()
{ kernel_few_threads_d<8>(); }


//
// Collect GPU and Kernel Info
//

// Info about a specific kernel.
//
struct Kernel_Info {
  void (*func_ptr)();           // Pointer to kernel function.
  char *name;                   // ASCII version of kernel name.
  hipFuncAttributes cfa;       // Kernel attributes reported by CUDA.
};

// Info about GPU and each kernel.
//
struct GPU_Info {
  double bw_Bps;
  static const int num_kernels = 4;
  Kernel_Info ki[num_kernels];
};

GPU_Info
print_gpu_info()
{
  GPU_Info info;

  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  hipGetDeviceCount(&device_count); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  hipDeviceProp_t cuda_prop;  // Properties of cuda device (GPU, cuda version).

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      CE(hipGetDeviceProperties(&cuda_prop,dev));
      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      const int cc_per_mp =
        cuda_prop.major == 1 ? 8 :
        cuda_prop.major == 2 ? ( cuda_prop.minor == 0 ? 32 : 48 ) :
        cuda_prop.major == 3 ? 192 : 0;

      const double chip_bw_Bps = info.bw_Bps =
        2 * cuda_prop.memoryClockRate * 1000.0
        * ( cuda_prop.memoryBusWidth >> 3 );
      const double chip_sp_flops =
        1000.0 * cc_per_mp * cuda_prop.clockRate
        * cuda_prop.multiProcessorCount;

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize,
         chip_bw_Bps * 1e-9,
         chip_sp_flops * 1e-9,
         4 * chip_sp_flops / chip_bw_Bps);

    }

  // Choose GPU 0 because we don't have time to provide a way to let
  // the user choose.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);

#define GET_INFO(idx,proc_name)                                               \
  info.ki[idx].name = #proc_name;                                             \
  info.ki[idx].func_ptr = proc_name;                                          \
  CE( hipFuncGetAttributes(&info.ki[idx].cfa,reinterpret_cast<const void*>(proc_name)) );

  GET_INFO(0,kernel_many_threads);
  GET_INFO(1,kernel_few_threads_d2);
  GET_INFO(2,kernel_few_threads_d4);
  GET_INFO(3,kernel_few_threads_d8);

#undef GET_INFO

  // Print information about time_step routine.
  //
  printf("\nCUDA Routine Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }

  return info;
}

int
main(int argc, char **argv)
{
  // Examine argument 1, grid size.
  //
  const int arg1_int = argc < 2 ? 4 : atoi(argv[1]);
  const int num_blocks = abs(arg1_int);

   // Examine argument 2, size of array in MiB. Fractional values okay.
  //
  app.array_size = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );

  if ( num_blocks <= 0 || app.array_size <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [DATA_SIZE_MiB]\n",
             argv[0]);
      exit(1);
    }

  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_info();

  const int array_size_bytes = app.array_size * sizeof(app.v_in[0]);

  // Allocate storage for CPU copy of data.
  //
  app.v_in = new Vertex[app.array_size];
  app.v_out = new Vertex[app.array_size];
  app.v_out_cpu = new Vertex[app.array_size];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_v_in,  app.array_size * sizeof(Vertex) ) );
  CE( hipMalloc( &app.d_v_out, app.array_size * sizeof(Vertex) ) );

  // Cast input data pointer to a type that the CUDA compiler handles better.
  //
  app.d_v_in_f4 = (typeof app.d_v_in_f4) app.d_v_in;

  printf
    ("\nPreparing for %d blocks operating on %d vectors of %d elements.\n",
     num_blocks, app.array_size, N);

  // Initialize input array.
  //
  for ( int i=0; i<app.array_size; i++ )
    for ( int j=0; j<N; j++ ) app.v_in[i].a[j] = drand48();

  // Initialize transformation matrix.
  //
  for ( int i=0; i<N; i++ )
    for ( int j=0; j<N; j++ )
      app.matrix[i][j] = drand48();

  // Compute correct answer (for checking).
  //
  for ( int e=0; e<app.array_size; e++ )
    for ( int i=0; i<N; i++ )
      {
        app.v_out_cpu[e].a[i] = 0;
        for ( int j=0; j<N; j++ )
          app.v_out_cpu[e].a[i] += app.v_in[e].a[j] * app.matrix[i][j];
      }

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy
      ( app.d_v_in, app.v_in, array_size_bytes, hipMemcpyHostToDevice ) );

  const double data_size = app.array_size * sizeof(Vertex) * 2;

  // Stuff needed to print bar graph.
  //
  const int full_width = 49;  // Width of longest bar.
  char * const stars = (char*) alloca(full_width+1);
  for ( int i=0; i<full_width; i++ ) stars[i] = '*';
  stars[full_width] = 0;

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      hipFuncAttributes& cfa = info.ki[kernel].cfa;

      // Maximum number of warps per block that this GPU can handle
      // for this kernel.
      //
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      printf("Running kernel %s which uses %d regs.\n",
             info.ki[kernel].name, cfa.numRegs);

      printf(" %3s %9s  %6s  %5s\n", "Num", "Time", "Data", "" );
      printf(" %3s %9s  %6s  %5s\n", "Wps", "µs", " GB/s", "Pct" );

      for ( int warp_cnt = 1; warp_cnt <= wp_limit; warp_cnt++ )
        {
          const int thd_per_block = warp_cnt << 5;
          app.num_threads = num_blocks * thd_per_block;

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( d_app), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          // Zero result array (to catch errors when kernel skips elements).
          //
          CE( hipMemset( app.d_v_out, 0, array_size_bytes ) );

          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Tell CUDA to start our threads on the GPU.
          //
          info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
          const double elapsed_time_s = cuda_time_ms * 0.001;

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              ( app.v_out, app.d_v_out, array_size_bytes,
                hipMemcpyDeviceToHost) );

          int err = 0;
          for ( int e=0; e<app.array_size; e++ )
            for ( int i=0; i<N; i++ )
              {
                const double diff =
                  fabs( app.v_out[e].a[i] - app.v_out_cpu[e].a[i] );
                if ( diff > 0.00001 )
                  {
                    err++;
                    if ( err < 2 )
                      printf
                        ("Error at array idx %d, "
                         "vector elt %d: %f != %f (correct)\n",
                         e, i, app.v_out[e].a[i], app.v_out_cpu[e].a[i]);
                  }
              }

          const double frac = data_size / elapsed_time_s / info.bw_Bps;
          const int s_idx = full_width * ( 1 - frac );

          printf(" %2d  %9.3f  %6.2f %5.1f%%  %s\n",
                 warp_cnt, 1e6 * elapsed_time_s,
                 1e-9 * data_size / elapsed_time_s,
                 100 * frac, &stars[s_idx] );

        }
    }
}
