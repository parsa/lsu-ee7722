#include "hip/hip_runtime.h"
/// LSU EE X70X-X (Sp 2010), GPU Microarchitecture
//
 /// Demo of Dynamic Simulation, Multiple Balls on Curved Platform

// $Id:$

/// Purpose
//
//   Demonstrate Several Graphical and Simulation Techniques.
//   This file contains GPU/cuda code.
//   See demo-x-collide.cc for main program.

#include "demo-x.cuh"


// Emulation Code
//
// The code below is only included when the kernel is compiled to
// run on the CPU, for debugging.
//
#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#define ASSERTS(expr) { if ( !(expr) ) abort();}
#endif

///
/// Variables Read or Written By With Host Code
///

 /// Ball Information Structure
//
// This is in soa (structure of arrays) form, rather than
// in the programmer-friendly aos (array of structure) form.
// In soa form it is easier for multiple thread to read contiguous
// blocks of data.
//
__constant__ CUDA_Ball_X balls_x;

///
 /// Ball Contact (tact) Pair Information
///

 /// Balls needed by block.
//
// This array identifies those balls that will be used by each block
// during each contact pass. When a thread starts balls are placed in
// shared memory, then contact between a pair of balls is tested for
// and resolved.
//
__constant__ int *block_balls_needed;

 /// Pairs of Balls to Check
//
__constant__ SM_Idx2 *tacts_schedule;


__constant__ float3 gravity_accel_dt;
__constant__ float opt_bounce_loss;
__constant__ float opt_friction_coeff, opt_friction_roll;
__constant__ float platform_xmin, platform_xmax;
__constant__ float platform_zmin, platform_zmax;
__constant__ float platform_xmid, platform_xrad;
__constant__ float delta_t;
__constant__ float elasticity_inv_dt;


///
/// Usefull Functions and Types
///

typedef float3 pCoor;
typedef float3 pVect;

__device__ float3 make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}
__device__ float3 m3(float4 a){ return make_float3(a); }
__device__ float3 xyz(float4 a){ return m3(a); }
__device__ float4 m4(float3 v, float w) { return make_float4(v.x,v.y,v.z,w); }

__device__ pVect operator +(pVect a,pVect b)
{ return make_float3(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pVect operator -(pVect a,pVect b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(float4 a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator *(float s, pVect v)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator -(pVect v) { return make_float3(-v.x,-v.y,-v.z); }
__device__ float3 operator -=(float3& a, pVect b) {a = a - b; return a;}
__device__ float3 operator +=(float3& a, pVect b) {a = a + b; return a;}

struct pNorm {
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

// Make a Coordinate
__device__ pCoor 
mc(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pCoor mc(float4 c){ return make_float3(c.x,c.y,c.z); }

__device__ void set_f3(float3& a, float4 b){a.x = b.x; a.y = b.y; a.z = b.z;}
__device__ void set_f4(float4& a, float3 b)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = 1.0;}

// Make a Vector
__device__ pVect
mv(float x, float y, float z){ return make_float3(x,y,z); }

__device__ float dot(pVect a, pVect b){ return a.x*b.x + a.y*b.y + a.z*b.z;}
__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }
__device__ float dot3(float4 a, float4 b){ return dot(m3(a),m3(b)); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

// Make a Normal (a structure containing a normalized vector and length)
__device__ pNorm mn(pVect v)
{
  pNorm n;
  n.mag_sq = mag_sq(v);
  if ( n.mag_sq == 0 )
    {
      n.magnitude = 0;
      n.v.x = n.v.y = n.v.z = 0;
    }
  else
    {
      n.magnitude = sqrtf(n.mag_sq);
      n.v = (1.0/n.magnitude) * v;
    }
  return n;
}
__device__ pNorm mn(float4 a, float4 b) {return mn(b-a);}
__device__ pNorm mn(pCoor a, pCoor b) {return mn(b-a);}

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

struct pQuat {
  float w;
  pVect v;
};

// Make Quaternion
__device__ float4 mq(pNorm axis, float angle)
{
  return m4( sin(angle/2) * axis.v, cos(angle/2) );
}

// Make float4
__device__ float4 m4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }
__device__ float4 m4(pNorm v, float w) { return m4(v.v,w); }


__device__ int
div_p2_ceil(int num, int den_lg)
{
  const int quot = num >> den_lg;
  return quot << den_lg == num ? quot : quot + 1;
}

// Cross Product of Two Vectors
__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}
__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }
__device__ pVect crossf3(float4 a, float4 b) { return cross(m3(a),m3(b)); }

// Cross Product of Vectors Between Coordinates
__device__ float3
 cross3(float3 a, float3 b, float3 c)
{
  float3 ab = a - b;
  float3 cb = c - b;
  return cross(ab,cb);
}
__device__ pVect cross3(pVect a, pVect b, pNorm c) { return cross3(a,b,c.v); }

__device__ float4 quat_mult(float4 a, float4 b)
{
  float w = a.w * b.w - dot3(a,b);
  float3 v = a.w * m3(b) + b.w * m3(a) + crossf3(a,b);
  return make_float4(v.x,v.y,v.z,w);
};


//
 /// Ball Physics Functions
//
// See demo-x-collide.cc for details.

__device__ pVect
point_rot_vel(CUDA_Ball_W& ball, pNorm direction)
{
  /// Return velocity of point on surface of ball.
  //
  return ball.radius * cross( ball.omega, direction );
}

__device__ float
get_fdt_to_do(CUDA_Ball_W& ball)
{
  return 2.5 * ball.mass_inv / ball.radius;
}

__device__ void
apply_tan_do(CUDA_Ball_W& ball, pNorm tact_dir, pVect tan_delta_omega)
{
  /// Change rotation rate based on..
  //
  pVect delta_omega_axis = cross(tact_dir, tan_delta_omega);
  ball.omega += delta_omega_axis;
}

__device__ void
apply_tan_force_dt
(CUDA_Ball_W& ball, pNorm tact_dir, pNorm force_dir, double force_dt)
{
  /// Change rotation rate due to force_dt at tact_dir in direction force_dir.
  //
  pVect delta_omega_tan = get_fdt_to_do(ball) * force_dt * force_dir;
  apply_tan_do(ball, tact_dir, delta_omega_tan);
}

__device__ void
apply_tan_force_dt(CUDA_Ball_W& ball, pNorm tact_dir, pVect force_dt)
{
  apply_tan_do(ball, tact_dir, get_fdt_to_do(ball) * force_dt );
}


///
/// Major Ball Physics Routines
///

// A time step is computed using two kernels, pass_pairs and
// pass_platform. The pass_pairs kernel, which might be launched
// several times, handles collisions between balls.  The pass_platform
// kernel handles collision between balls and the platform, and also
// updates position and orientation.


///
/// Collision (Penetration) Detection and Resolution Routine
///

// Used in both passes.

__device__ void
penetration_balls_resolve
(CUDA_Ball_W& ball1_r, CUDA_Ball_W& ball2_r, bool b2_real)
{
  /// Update velocity and angular momentum for a pair of balls in contact.

  CUDA_Ball_W *ball1 = &ball1_r;
  CUDA_Ball_W *ball2 = &ball2_r;
  pVect zero_vec = mv(0,0,0);
  pNorm dist = mn(ball1->position,ball2->position);

  // For Debugging
  //
  if ( b2_real ) { ball1->debug_pair_calls++;  ball2->debug_pair_calls++; }

  const float radii_sum = ball1->radius + ball2->radius;

  if ( dist.magnitude >= radii_sum ) return;

  // Update counters used for optimization (contact_count) and
  // to decide when to release new balls (collision_count).
  //
  ball1->collision_count++; ball1->contact_count++; 
  if ( b2_real ) {ball2->collision_count++; ball2->contact_count++;}

  /// WARNING:  This doesn't work: somefunc(-dist); 
  pNorm ndist = -dist;

  // Compute relative (approach) velocity.
  //
  pVect prev_appr_vel = ball1->prev_velocity - ball2->prev_velocity;
  const float prev_approach_speed = dot( prev_appr_vel, dist );

  const float loss_factor = 1 - opt_bounce_loss;

  // Compute change in speed based on how close balls touching, ignoring
  // energy loss.
  //
  const float appr_force_dt_no_loss =
    ( radii_sum - dist.magnitude ) * elasticity_inv_dt;

  // Change in speed accounting for energy loss. Only applied when
  // balls separating.
  //
  const float appr_force_dt =
    prev_approach_speed > 0
    ? appr_force_dt_no_loss : loss_factor * appr_force_dt_no_loss;

  const float appr_deltas_1 = appr_force_dt * ball1->mass_inv;

  /// Update Linear Velocity
  //
  ball1->velocity -= appr_deltas_1 * dist;
  if ( b2_real ) ball2->velocity += appr_force_dt * ball2->mass_inv * dist;

  // Find speed on surface of balls at point of contact.
  //
  pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
  pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);

  // Find relative velocity of surfaces at point of contact
  // in the plane formed by their surfaces.
  //
  pVect tan_vel = prev_appr_vel - prev_approach_speed * dist;
  pNorm tact_vel_dir = mn(tact1_rot_vel - tact2_rot_vel + tan_vel);

  // Find change in velocity due to friction.
  //
  const double fric_force_dt_potential =
    appr_force_dt_no_loss * opt_friction_coeff;

  const double mass_inv_sum =
    b2_real ? ball1->mass_inv + ball2->mass_inv : ball1->mass_inv;

  const double force_dt_limit =
    tact_vel_dir.magnitude / ( 3.5 * mass_inv_sum );

  // If true, surfaces are not sliding or will stop sliding after
  // frictional forces applied. (If a ball surface isn't sliding
  // against another surface than it must be rolling.)
  //
  const bool will_roll = force_dt_limit <= fric_force_dt_potential;

  const double sliding_fric_force_dt =
    will_roll ? force_dt_limit : fric_force_dt_potential;

  const double dv_tolerance = 0.000001;

  const double sliding_fric_dv_1 = sliding_fric_force_dt * ball1->mass_inv;

  if ( sliding_fric_dv_1 > dv_tolerance )
    {
      // Apply tangential force (resulting in angular momentum change) and
      // linear force (resulting in velocity change).
      //
      apply_tan_force_dt(ball1_r,dist,tact_vel_dir,-sliding_fric_force_dt);
      ball1->velocity -= sliding_fric_dv_1 * tact_vel_dir;
    }

  const double sliding_fric_dv_2 = sliding_fric_force_dt * ball2->mass_inv;

  if ( b2_real && sliding_fric_dv_2 > dv_tolerance )
    {
      // Apply frictional forces for ball 2.
      //
      apply_tan_force_dt(ball2_r,ndist,tact_vel_dir,sliding_fric_force_dt);
      ball2->velocity += sliding_fric_dv_2 * tact_vel_dir;;
    }

  const float fdt_to_do_1 = get_fdt_to_do(ball1_r);
  const float fdt_to_do_2 = get_fdt_to_do(ball2_r);

  {
    /// Torque
    //
    //
    // Account for forces of surfaces twisting against each
    // other. (For example, if one ball is spinning on top of
    // another.)
    //
    const double appr_omega = dot(ball2->omega,dist) - dot(ball1->omega,dist);
    const double fdt_to_do_sum =
      b2_real ? fdt_to_do_1 + fdt_to_do_2 : fdt_to_do_1;
    const double fdt_limit = fabs(appr_omega) / fdt_to_do_sum;
    const bool rev = appr_omega < 0;
    const double fdt_raw = min(fdt_limit,fric_force_dt_potential);
    const pVect fdt_v = ( rev ? -fdt_raw : fdt_raw ) * dist;
    ball1->omega += fdt_to_do_1 * fdt_v;
    if ( b2_real ) ball2->omega -= fdt_to_do_2 * fdt_v;
  }

  return;

  {
    /// Rolling Friction
    //
    // The rolling friction model used here is ad-hoc.

    pVect tan_b12_vel = b2_real ? 0.5 * tan_vel : zero_vec;
    const double torque_limit_sort_of = appr_force_dt_no_loss
      * sqrt( radii_sum - dist.mag_sq / radii_sum );
      //  * sqrt( ball1->radius - 0.25 * dist.mag_sq * r_inv );

    pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
    pVect tact1_roll_vel = tact1_rot_vel + tan_b12_vel;
    pNorm tact1_roll_vel_dir = mn(tact1_roll_vel);
    pVect lost_vel = zero_vec;

    const double rfric_loss_dv_1 =
      torque_limit_sort_of * 2.5 * ball1->mass_inv *
      ( tact1_roll_vel_dir.magnitude * opt_friction_roll /
        ( 1 + tact1_roll_vel_dir.magnitude * opt_friction_roll ) );
    
    pVect lost_vel1 =
      min(tact1_roll_vel_dir.magnitude, rfric_loss_dv_1) * tact1_roll_vel_dir;

    lost_vel = -lost_vel1;
    
    if ( b2_real )
      {
        pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);
        pVect tact2_roll_vel = tact2_rot_vel - tan_b12_vel;
        pNorm tact2_roll_vel_dir = mn(tact2_roll_vel);
        const double rfric_loss_dv_2 =
          torque_limit_sort_of * 2.5 * ball2->mass_inv *
          ( tact2_roll_vel_dir.magnitude * opt_friction_roll /
            ( 1 + tact2_roll_vel_dir.magnitude * opt_friction_roll ) );
        pVect lost_vel2 =
          min(tact2_roll_vel_dir.magnitude, rfric_loss_dv_2 )
          * tact2_roll_vel_dir;

        lost_vel += lost_vel2;
      }

    apply_tan_force_dt(ball1_r,dist,0.4 / ball1->mass_inv * lost_vel);
    if ( b2_real )
      apply_tan_force_dt(ball2_r,dist, 0.4/ ball2->mass_inv * lost_vel);
  }
}

///
/// Pairs Pass
///
//
// Resolve ball collisions with each other.

__global__ void pass_pairs
(int prefetch_offset, int schedule_offset, int round_cnt);

__host__ void 
pass_pairs_launch
(dim3 dg, dim3 db, int prefetch_offset, int schedule_offset, int round_cnt)
{
  pass_pairs<<<dg,db>>>(prefetch_offset,schedule_offset,round_cnt);
}

__device__ void
pass_pairs(int prefetch_offset, int schedule_offset, int round_cnt)
{
  const int tid = threadIdx.x;
  const int max_balls_per_thread =
    int( ceil( float(BALLS_PER_BLOCK) / blockDim.x ) );

  // Initialized variables used to access balls_needed and tacts_schedule
  // arrays.
  //
  const int si_block_size = blockIdx.x * max_balls_per_thread * blockDim.x;
  const int si_block_base = prefetch_offset + si_block_size + tid;
  const int sp_block_size = blockIdx.x * round_cnt * blockDim.x;
  const int sp_block_base = schedule_offset + sp_block_size + tid;

  /// Shared memory array holding balls updated by this block.
  //
  __shared__ CUDA_Ball_W sm_balls[BALLS_PER_BLOCK];

  /// Prefetch balls to shared memory.
  //
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      const int m_idx = block_balls_needed[ si_block_base + i * blockDim.x ];
      CUDA_Ball_W& ball = sm_balls[idx];
      ball.m_idx = m_idx;
      if ( m_idx < 0 ) continue;
      ball.velocity = xyz(balls_x.velocity[m_idx]);
      ball.prev_velocity = xyz(balls_x.prev_velocity[m_idx]);
      ball.position = xyz(balls_x.position[m_idx]);
      ball.omega = xyz(balls_x.omega[m_idx]);
      float4 ball_props = balls_x.ball_props[m_idx];
      ball.radius = ball_props.x;
      ball.mass_inv = ball_props.y;

      int4 tact_counts = balls_x.tact_counts[m_idx];
      ball.collision_count = tact_counts.x;
      ball.contact_count = tact_counts.y;
      ball.debug_pair_calls = tact_counts.z;
    }

  __syncthreads();

  /// Resolve Collisions
  //
  for ( int round=0; round<round_cnt; round++ )
    {
      SM_Idx2 indices = tacts_schedule[ sp_block_base + round * blockDim.x ];

      // Wait for all threads to reach this point (to avoid having
      // two threads operate on the same ball simultaneously).
      //
      __syncthreads();

      if ( indices.x == indices.y ) continue;
      penetration_balls_resolve(sm_balls[indices.x],sm_balls[indices.y],true);
    }

  __syncthreads();

  /// Copy Ball Data to Memory
  //
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      CUDA_Ball_W& ball = sm_balls[idx];
      const int m_idx = ball.m_idx;
      if ( m_idx < 0 ) continue;

      set_f4(balls_x.velocity[m_idx], ball.velocity);
      set_f4(balls_x.omega[m_idx], ball.omega);

      int4 tact_counts;
      tact_counts.x = ball.collision_count;
      tact_counts.y = ball.contact_count;
      tact_counts.z = ball.debug_pair_calls;
      balls_x.tact_counts[m_idx] = tact_counts;
    }
}


///
/// Platform Pass
///
//
// Resolve ball collisions with platform, also update ball position
// and orientation.

__device__ void platform_collision(CUDA_Ball_W& ball);
__global__ void pass_platform(int ball_count);

__host__ hipError_t
cuda_get_attr_plat_pairs
(struct hipFuncAttributes *attr_platform,
 struct hipFuncAttributes *attr_pairs)
{
  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.
  hipError_t e1 = hipFuncGetAttributes(attr_platform,reinterpret_cast<const void*>(pass_platform));
  if ( e1 ) return e1;
  hipError_t e2 = hipFuncGetAttributes(attr_pairs,reinterpret_cast<const void*>(pass_pairs));
  return e2;
}

__host__ void 
pass_platform_launch
(dim3 dg, dim3 db, int ball_count)
{
  pass_platform<<<dg,db>>>(ball_count);
}

__global__ void
pass_platform(int ball_count)
{
  /// Main CUDA routine for resolving collisions with platform and
  /// updating ball position and orientation.

  // One ball per thread.

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= ball_count ) return;

  CUDA_Ball_W ball;

  /// Copy ball data from memory to local variables.
  //
  //  Local variables hopefully will be in GPU registers, not
  //  slow local memory.
  //
  ball.prev_velocity = xyz(balls_x.prev_velocity[idx]);
  ball.velocity = xyz(balls_x.velocity[idx]) + gravity_accel_dt;
  set_f3(ball.position,balls_x.position[idx]);
  set_f3(ball.omega, balls_x.omega[idx]);
  int4 tact_counts = balls_x.tact_counts[idx];
  ball.collision_count = tact_counts.x;
  ball.contact_count = tact_counts.y;
  float4 ball_props = balls_x.ball_props[idx];
  ball.radius = ball_props.x;
  ball.mass_inv = ball_props.y;

  /// Handle Ball/Platform Collision
  //
  platform_collision(ball);

  /// Update Position and Orientation
  //
  ball.position += delta_t * ball.velocity;
  pNorm axis = mn(ball.omega);
  balls_x.orientation[idx] =
    quat_mult( mq( axis, delta_t * axis.magnitude ),
               balls_x.orientation[idx] );

  /// Copy other updated data to memory.
  //
  set_f4(balls_x.velocity[idx], ball.velocity);
  set_f4(balls_x.prev_velocity[idx], ball.velocity);
  set_f4(balls_x.omega[idx], ball.omega);
  set_f4(balls_x.position[idx],ball.position);
  tact_counts.x = ball.collision_count;
  tact_counts.y = ball.contact_count << 8;
  tact_counts.z = tact_counts.z << 16;
  balls_x.tact_counts[idx] = tact_counts;
}

__device__ void
platform_collision(CUDA_Ball_W& ball)
{
  /// Check if ball in contact with platform, if so apply forces.

  pCoor pos = ball.position;
  const float r = ball.radius;
  bool collision_possible =
    pos.y < r
    && pos.x >= platform_xmin - r && pos.x <= platform_xmax + r
    && pos.z >= platform_zmin - r && pos.z <= platform_zmax + r;

  if ( !collision_possible ) return;

  CUDA_Ball_W pball;

  pCoor axis = mc(platform_xmid,0,pos.z);
  const float short_xrad = platform_xrad - r;
  const float short_xrad_sq = short_xrad * short_xrad;

  // Test for different ways ball can touch platform. If contact
  // is found find position of an artificial platform ball (pball)
  // that touches the real ball at the same place and angle as
  // the platform. This pball will be used for the ball-ball penetration
  // routine, penetration_balls_resolve.

  if ( pos.y > 0 )
    {
      // Possible contact with upper edge of platform.
      //
      pCoor tact
        = mc(pos.x > platform_xmid ? platform_xmax : platform_xmin, 0, pos.z);
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= r * r ) return;
      pball.position = tact + r * tact_dir;
    }
  else if ( pos.z > platform_zmax || pos.z < platform_zmin )
    {
      // Possible contact with side (curved) edges of platform.
      //
      pNorm ball_dir = mn(axis,pos);
      if ( ball_dir.mag_sq <= short_xrad_sq ) return;
      const float zedge =
        pos.z > platform_zmax ? platform_zmax : platform_zmin;
      pCoor axis_edge = mc(platform_xmid,0,zedge);
      pCoor tact = axis_edge + platform_xrad * ball_dir;
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= r * r ) return;
      pball.position = tact + r * tact_dir;
    }
  else
    {
      // Possible contact with surface of platform.
      //
      pNorm tact_dir = mn(axis,pos);
      if ( tact_dir.mag_sq <= short_xrad_sq ) return;
      pball.position = axis + (r+platform_xrad) * tact_dir;
    }

  // Finish initializing platform ball, and call routine to
  // resolve penetration.
  //
  pVect zero_vec = mv(0,0,0);
  pball.omega = zero_vec;
  pball.prev_velocity = pball.velocity = zero_vec;
  pball.radius = ball.radius;
  pball.mass_inv = ball.mass_inv;
  penetration_balls_resolve(ball,pball,false);
}
