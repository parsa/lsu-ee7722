#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2009), GPU Programming
//
 /// Demo of Dynamic Simulation, Multiple Balls on Curved Platform

// $Id:$

/// Purpose
//
//   Demonstrate Several Graphical and Simulation Techniques.
//   This file contains GPU/cuda code.
//   See demo-x-collide.cc for main program.

#include "demo-x.cuh"

#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#define ASSERTS(expr) { if ( !(expr) ) abort();}
#endif


__constant__ CUDA_Ball_X balls_x_0, balls_x_1;

__constant__ int *schedule_inputs;
__constant__ SM_Idx2 *schedule;

__constant__ float3 gravity_accel_dt;
__constant__ float opt_ball_radius, opt_bounce_loss;
__constant__ float opt_friction_coeff, opt_friction_roll;
__constant__ bool opt_debug;
__constant__ float platform_xmin, platform_xmax;
__constant__ float platform_zmin, platform_zmax;
__constant__ float platform_xmin_mr, platform_xmax_pr;
__constant__ float platform_zmin_mr, platform_zmax_pr;
__constant__ float platform_xmid, platform_xrad;
__constant__ float delta_t;
__constant__ float short_xrad_sq;
__constant__ float r_inv, two_r, two_r_sq;
__constant__ float elasticity_inv_dt, ball_mass_inv;
__constant__ float mo_vel_factor, v_to_do;

typedef float3 pCoor;
typedef float3 pVect;

__device__ pVect operator +(pVect a,pVect b)
{ return make_float3(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pVect operator -(pVect a,pVect b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(float4 a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator *(float s, pVect v)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator -(pVect v) { return make_float3(-v.x,-v.y,-v.z); }
__device__ float3 operator -=(float3& a, pVect b) {a = a - b; return a;}
__device__ float3 operator +=(float3& a, pVect b) {a = a + b; return a;}

struct pNorm {
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

__device__ pCoor 
mc(float x, float y, float z){ return make_float3(x,y,z); }

__device__ pCoor mc(float4 c){ return make_float3(c.x,c.y,c.z); }

__device__ void set_f3(float3& a, float4 b){a.x = b.x; a.y = b.y; a.z = b.z;}
__device__ void set_f4(float4& a, float3 b){a.x = b.x; a.y = b.y; a.z = b.z;}

__device__ pVect
mv(float x, float y, float z){ return make_float3(x,y,z); }

__device__ float
dot(pVect a, pVect b){return a.x*b.x + a.y*b.y + a.z*b.z;}

__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

__device__ pNorm mn(pVect v)
{
  pNorm n;
  n.mag_sq = mag_sq(v);
  if ( n.mag_sq == 0 )
    {
      n.magnitude = 0;
      n.v.x = n.v.y = n.v.z = 0;
    }
  else
    {
      n.magnitude = sqrtf(n.mag_sq);
      n.v = (1.0/n.magnitude) * v;
    }
  return n;
}

__device__ pNorm mn(float4 a, float4 b) {return mn(b-a);}
__device__ pNorm mn(pCoor a, pCoor b) {return mn(b-a);}

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

struct pQuat {
  float w;
  pVect v;
};

__device__ pQuat mq(pNorm axis, float angle)
{
  pQuat q;
  q.w = cos(angle/2);
  q.v = sin(angle/2) * axis;
  return q;
}

__device__ pQuat mq(float4 a)
{
  pQuat q;
  q.w = a.w; q.v.x = a.x; q.v.y = a.y; q.v.z = a.z;
  return q;
}

__device__ float4 m4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }



__device__ float3 make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}

__device__ int
div_p2_ceil(int num, int den_lg)
{
  const int quot = num >> den_lg;
  return quot << den_lg == num ? quot : quot + 1;
}


__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}

__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }


__device__ float3
cross3(float3 a, float3 b, float3 c)
{
  float3 ab = a - b;
  float3 cb = c - b;
  return cross(ab,cb);
}

__device__ pVect
cross3(pVect a, pVect b, pNorm c) { return cross3(a,b,c.v); }

__device__ pQuat operator *(pQuat a, pQuat b)
{
  pQuat p;
  p.w = a.w * b.w - dot(a.v,b.v);
  p.v = a.w * b.v + b.w * a.v + cross(a.v,b.v);
  return p;
};


__device__ pVect
point_rot_vel(CUDA_Ball_W ball, pNorm direction)
{
  return opt_ball_radius * cross( ball.angular_momentum, direction );
}

__device__ void
apply_tan_do(CUDA_Ball_W& ball, pNorm tact_dir, pVect force)
{
  pVect axis_torque = cross( tact_dir, force );
  ball.angular_momentum += axis_torque;
}

__device__ void
apply_deltao(CUDA_Ball_W& ball, pNorm tact_dir, pNorm force_dir, double deltao)
{
  apply_tan_do(ball, tact_dir, deltao * force_dir );
}

__device__ void
apply_tan_dv(CUDA_Ball_W& ball, pNorm tact_dir, pVect force)
{
  apply_tan_do(ball, tact_dir, r_inv * force);
}


__device__ void
fly_to_pc(CUDA_Ball_W& ball, double ts_t)
{
  ball.position += ts_t * ball.velocity;
}



__global__ void pass_something(int read_side, int ball_count);

__host__ void 
pass_something_launch
(dim3 dg, dim3 db, int read_side, int ball_count)
{
  pass_something<<<dg,db>>>(read_side,ball_count);
}

__device__ void
penetration_balls_resolve
(CUDA_Ball_W& ball1_r, CUDA_Ball_W& ball2_r, bool b2_real)
{
  CUDA_Ball_W *ball1 = &ball1_r;
  CUDA_Ball_W *ball2 = &ball2_r;
  pVect zero_vec = mv(0,0,0);
  pNorm dist = mn(ball1->position,ball2->position);

  if ( b2_real ) {
    ball1->debug_pair_calls++;  ball2->debug_pair_calls++; }
  if ( dist.mag_sq >= two_r_sq ) return;

  ball1->collision_count++;
  ball1->contact_count++;
  if ( b2_real ) { ball2->collision_count++; ball2->contact_count++; }


  /// WARNING:  This doesn't work: somefunc(-dist); 
  pNorm ndist = -dist;

  pVect appr_vel = ball1->velocity - ball2->velocity;
  pVect prev_appr_vel = ball1->prev_velocity - ball2->prev_velocity;
  const double approach_speed = dot( appr_vel, dist );
  const double prev_approach_speed = dot( prev_appr_vel, dist );

  const double loss_factor = 1 - opt_bounce_loss;

  const double appr_deltas_no_loss =
    ( two_r - dist.magnitude ) * elasticity_inv_dt * ball_mass_inv;

  const double appr_deltas =
    approach_speed > 0
    ? appr_deltas_no_loss : loss_factor * appr_deltas_no_loss;

  ball1->velocity -= appr_deltas * dist;
  if ( b2_real ) ball2->velocity += appr_deltas * dist;

  pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
  pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);

  const bool use_prev_vel = true;

  pVect tan_vel_prev = prev_appr_vel - prev_approach_speed * dist;
  pVect tan_vel_curr = appr_vel - approach_speed * dist;
  pVect tan_vel = use_prev_vel ? tan_vel_prev : tan_vel_curr;
  pNorm tact_vel_dir = mn(tact1_rot_vel - tact2_rot_vel + tan_vel);

  const double fric_dv_potential =
    fabs(appr_deltas_no_loss) * opt_friction_coeff;

  const double dv_limit_raw = tact_vel_dir.magnitude * mo_vel_factor;
  const double dv_limit = b2_real ? dv_limit_raw : 2 * dv_limit_raw;
  const bool will_roll = dv_limit <= fric_dv_potential;
  const double sliding_fric_deltav =
    will_roll ? dv_limit : fric_dv_potential;

  const double dv_tolerance = 0.000001;

  if ( sliding_fric_deltav > dv_tolerance )
    {
      const double fric_deltao = sliding_fric_deltav * v_to_do;

      apply_deltao(ball1_r,dist,tact_vel_dir,-fric_deltao);
      ball1->velocity -= sliding_fric_deltav * tact_vel_dir;

      if ( b2_real )
        {
          apply_deltao(ball2_r,dist,tact_vel_dir,-fric_deltao);
          ball2->velocity += sliding_fric_deltav * tact_vel_dir;
        }
    }

#if 0
  if ( !use_prev_vel && opt_verify && b2_real && will_roll )
    {
      pVect appr_vel2 = ball1->velocity - ball2->velocity;
      const double approach_speed2 = dot( appr_vel2, dist );

      pVect tact1_rot_vel2 = point_rot_vel(ball1_r,dist);
      pVect tact2_rot_vel2 = point_rot_vel(ball2_r,ndist);

      pVect tan_vel2 = appr_vel2 - approach_speed2 * dist;
      pNorm tan_vel_dir2 = tact1_rot_vel2 - tact2_rot_vel2 + tan_vel2;
      ASSERTS( tan_vel_dir2.magnitude <= 0.0001 + 100 * dv_tolerance );
      ball1->color_event = ball2->color_event = pColor(1,1,1);
    }
#endif

  {
    /// Torque
    //
    const double appr_omega =
      dot(ball1->angular_momentum,dist) - dot(ball2->angular_momentum,dist);
    const double fric_deltao_pot = fric_dv_potential * v_to_do;
    const bool rev = appr_omega < 0;
    const double fric_deltao = min(fabs(appr_omega),fric_deltao_pot);
    pVect delta_am = rev ? -fric_deltao * dist : fric_deltao * dist;
    ball1->angular_momentum -= delta_am;
    if ( b2_real ) ball2->angular_momentum += delta_am;
  }

  if ( opt_debug ) return;

  {
    /// Rolling Friction
    //
    pVect tan_b12_vel = b2_real ? 0.5 * tan_vel : zero_vec;
    const double torque_limit_sort_of = appr_deltas_no_loss
      * sqrt( opt_ball_radius - 0.25 * dist.mag_sq * r_inv );

    pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
    pVect tact1_roll_vel = tact1_rot_vel + tan_b12_vel;
    pNorm tact1_roll_vel_dir = mn(tact1_roll_vel);
    pVect lost_vel = zero_vec;

    const double rfric_loss1 =
      torque_limit_sort_of *
      ( tact1_roll_vel_dir.magnitude * opt_friction_roll /
        ( 1 + tact1_roll_vel_dir.magnitude * opt_friction_roll ) );
    
    pVect lost_vel1 =
      min(tact1_roll_vel_dir.magnitude, rfric_loss1) * tact1_roll_vel_dir;

    lost_vel = -lost_vel1;
    
    if ( b2_real )
      {
        pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);
        pVect tact2_roll_vel = tact2_rot_vel - tan_b12_vel;
        pNorm tact2_roll_vel_dir = mn(tact2_roll_vel);
        const double rfric_loss2 =
          torque_limit_sort_of *
          ( tact2_roll_vel_dir.magnitude * opt_friction_roll /
            ( 1 + tact2_roll_vel_dir.magnitude * opt_friction_roll ) );
        pVect lost_vel2 =
          min(tact2_roll_vel_dir.magnitude, rfric_loss2 ) * tact2_roll_vel_dir;

        lost_vel += lost_vel2;
      }

    apply_tan_dv(ball1_r,dist,lost_vel);
    if ( b2_real ) apply_tan_dv(ball2_r,dist,lost_vel);

#if 0
    if ( opt_verify )
      {
        pVect ch_tact1_rot_vel = point_rot_vel(ball1_r,dist);
        pVect ch_tact1_roll_vel = ch_tact1_rot_vel + tan_b12_vel;
        const double magloss = tact1_roll_vel.mag() - ch_tact1_roll_vel.mag();
        ASSERTS( magloss >= -10.0 );
      }
#endif
  }
}


__device__ void
platform_collision(CUDA_Ball_W& ball)
{
  const float ts_mov_max = 0.0;
  pVect zero_vec = mv(0,0,0);

  pCoor pos = ball.position;

  bool collision_possible =
    pos.y < opt_ball_radius + ts_mov_max
    && pos.x - ts_mov_max >= platform_xmin_mr
    && pos.x + ts_mov_max <= platform_xmax_pr
    && pos.z - ts_mov_max >= platform_zmin_mr
    && pos.z + ts_mov_max <= platform_zmax_pr;

  if ( !collision_possible ) return;

  CUDA_Ball_W pball;

  pCoor axis = mc(platform_xmid,0,pos.z);

  if ( pos.y > 0 )
    {
      pCoor tact
        = mc(pos.x > platform_xmid ? platform_xmax : platform_xmin, 0, pos.z);
      pVect pos_tact = tact - pos;
      float tact_dir_mag_sq = mag_sq(pos_tact);
      if ( tact_dir_mag_sq >= two_r_sq ) return;
      pball.position =
        tact + opt_ball_radius / sqrt(tact_dir_mag_sq) * pos_tact;
    }
  else if ( pos.z > platform_zmax || pos.z < platform_zmin )
    {
      pNorm ball_dir = mn(axis,pos);
      if ( ball_dir.mag_sq <= short_xrad_sq ) return;
      const float zedge =
        pos.z > platform_zmax ? platform_zmax : platform_zmin;
      pCoor axis_edge = mc(platform_xmid,0,zedge);
      pCoor tact = axis_edge + platform_xrad * ball_dir;
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= two_r_sq ) return;
      pball.position = tact + opt_ball_radius * tact_dir;
    }
  else
    {
      pNorm tact_dir = mn(axis,pos);
      if ( tact_dir.mag_sq <= short_xrad_sq ) return;
      pball.position = axis + (opt_ball_radius+platform_xrad) * tact_dir;
    }

  pball.angular_momentum = zero_vec;
  pball.prev_velocity = pball.velocity = zero_vec;
  penetration_balls_resolve(ball,pball,false);
}

__device__ void
pass_something(int read_side, int ball_count)
{
  //  const int tid = threadIdx.x;
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= ball_count ) return;

  CUDA_Ball_X bi = read_side ? balls_x_1 : balls_x_0;
  CUDA_Ball_X bo = read_side ? balls_x_1 : balls_x_0;


  CUDA_Ball_W ball;

  ball.prev_velocity = bi.prev_velocity[idx];
  ball.velocity = bi.velocity[idx] + gravity_accel_dt;
  set_f3(ball.position,bi.position[idx]);
  ball.angular_momentum = bi.angular_momentum[idx];
  ball.collision_count = bi.collision_count[idx];
  ball.contact_count = bi.contact_count[idx];

  platform_collision(ball);

  ball.position += delta_t * ball.velocity;
  pNorm axis = mn(ball.angular_momentum);
  bo.orientation[idx] =
    m4( mq(axis,delta_t * axis.magnitude) * mq(bi.orientation[idx]) );
  bo.velocity[idx] = ball.velocity;
  bo.prev_velocity[idx] = ball.velocity;
  bo.angular_momentum[idx] = ball.angular_momentum;
  set_f4(bo.position[idx],ball.position);
  bo.collision_count[idx] = ball.collision_count;
  bo.contact_count[idx] = ball.contact_count << 8;
  bo.debug_pair_calls[idx] = bi.debug_pair_calls[idx] << 16;
}

__global__ void pass_pairs
(int read_side, int prefetch_offset, int schedule_offset, int round_cnt);

__host__ void 
pass_pairs_launch
(dim3 dg, dim3 db, int read_side, int prefetch_offset,
 int schedule_offset, int round_cnt)
{
  pass_pairs<<<dg,db>>>(read_side,prefetch_offset,schedule_offset,round_cnt);
}

__device__ void
pass_pairs
(int read_side, int prefetch_offset, int schedule_offset, int round_cnt)
{
  const int tid = threadIdx.x;
  const int max_balls_per_thread =
    int( ceil( float(BALLS_PER_BLOCK) / blockDim.x ) );

  const int si_block_size = blockIdx.x * max_balls_per_thread * blockDim.x;
  const int si_block_base = prefetch_offset + si_block_size + tid;
  const int sp_block_size = blockIdx.x * round_cnt * blockDim.x;
  const int sp_block_base = schedule_offset + sp_block_size + tid;

  __shared__ CUDA_Ball_W sm_balls[BALLS_PER_BLOCK];

  CUDA_Ball_X bi = read_side ? balls_x_1 : balls_x_0;
  CUDA_Ball_X bo = read_side ? balls_x_1 : balls_x_0;

   for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      const int m_idx = schedule_inputs[ si_block_base + i * blockDim.x ];
      CUDA_Ball_W& ball = sm_balls[idx];
      ball.m_idx = m_idx;
      if ( m_idx < 0 ) continue;
      ball.velocity = bi.velocity[m_idx];
      ball.prev_velocity = bi.prev_velocity[m_idx];
      set_f3(ball.position,bi.position[m_idx]);
      ball.angular_momentum = bi.angular_momentum[m_idx];
      ball.collision_count = bi.collision_count[m_idx];
      ball.contact_count = bi.contact_count[m_idx];
      ball.debug_pair_calls = bi.debug_pair_calls[m_idx];
    }

  for ( int round=0; round<round_cnt; round++ )
    {
      const SM_Idx2 indices = schedule[ sp_block_base + round * blockDim.x ];
      __syncthreads();
      if ( indices.x == indices.y ) continue;
      penetration_balls_resolve(sm_balls[indices.x],sm_balls[indices.y],true);
    }

  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      CUDA_Ball_W& ball = sm_balls[idx];
      const int m_idx = ball.m_idx;
      if ( m_idx < 0 ) continue;

      bo.velocity[m_idx] = ball.velocity;
      bo.angular_momentum[m_idx] = ball.angular_momentum;
      bo.collision_count[m_idx] = ball.collision_count;
      bo.contact_count[m_idx] = ball.contact_count;
      bo.debug_pair_calls[m_idx] = ball.debug_pair_calls;
    }
}
