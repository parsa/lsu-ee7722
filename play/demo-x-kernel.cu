#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2009), GPU Programming
//
 /// Demo of Dynamic Simulation, Multiple Balls on Curved Platform

// $Id:$

/// Purpose
//
//   Demonstrate Several Graphical and Simulation Techniques.
//   This file contains GPU/cuda code.
//   See demo-x-collide.cc for main program.

#include "demo-x.cuh"


// Emulation Code
//
// The code below is only included when the kernel is compiled to
// run on the CPU, for debugging.
//
#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#define ASSERTS(expr) { if ( !(expr) ) abort();}
#endif

///
/// Variables Read or Written By With Host Code
///

 /// Ball Information Structure
//
// This is in soa (structure of arrays) form, rather than
// in the programmer-friendly aos (array of structure) form.
// In soa form it is easier for multiple thread to read contiguous
// blocks of data.
//
__constant__ CUDA_Ball_X balls_x;

///
 /// Ball Contact (tact) Pair Information
///

 /// Balls needed by block.
//
// This array identifies those balls that will be used by each block
// during each contact pass. When a thread starts balls are placed in
// shared memory, then contact between a pair of balls is tested for
// and resolved.
//
__constant__ int *block_balls_needed;

 /// Pairs of Balls to Check
//
__constant__ SM_Idx2 *tacts_schedule;


__constant__ float3 gravity_accel_dt;
__constant__ float opt_ball_radius, opt_bounce_loss;
__constant__ float opt_friction_coeff, opt_friction_roll;
__constant__ float platform_xmin, platform_xmax;
__constant__ float platform_zmin, platform_zmax;
__constant__ float platform_xmin_mr, platform_xmax_pr;
__constant__ float platform_zmin_mr, platform_zmax_pr;
__constant__ float platform_xmid, platform_xrad;
__constant__ float delta_t;
__constant__ float short_xrad_sq;
__constant__ float r_inv, two_r, two_r_sq;
__constant__ float elasticity_inv_dt, ball_mass_inv;
__constant__ float mo_vel_factor, v_to_do;


///
/// Usefull Functions and Types
///

typedef float3 pCoor;
typedef float3 pVect;

__device__ float3 make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}
__device__ float3 m3(float4 a){ return make_float3(a); }
__device__ float3 xyz(float4 a){ return m3(a); }
__device__ float4 m4(float3 v, float w) { return make_float4(v.x,v.y,v.z,w); }

__device__ pVect operator +(pVect a,pVect b)
{ return make_float3(a.x+b.x,a.y+b.y,a.z+b.z); }
__device__ pVect operator -(pVect a,pVect b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(float4 a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator -(pCoor a,float4 b)
{ return make_float3(a.x-b.x,a.y-b.y,a.z-b.z); }
__device__ pVect operator *(float s, pVect v)
{return make_float3(s*v.x,s*v.y,s*v.z);}
__device__ pVect operator -(pVect v) { return make_float3(-v.x,-v.y,-v.z); }
__device__ float3 operator -=(float3& a, pVect b) {a = a - b; return a;}
__device__ float3 operator +=(float3& a, pVect b) {a = a + b; return a;}

struct pNorm {
  pVect v;
  float mag_sq, magnitude;
};

__device__ pVect operator *(float s, pNorm n) { return s * n.v;}

// Make a Coordinate
__device__ pCoor 
mc(float x, float y, float z){ return make_float3(x,y,z); }
__device__ pCoor mc(float4 c){ return make_float3(c.x,c.y,c.z); }

__device__ void set_f3(float3& a, float4 b){a.x = b.x; a.y = b.y; a.z = b.z;}
__device__ void set_f4(float4& a, float3 b)
{a.x = b.x; a.y = b.y; a.z = b.z; a.w = 1.0;}

// Make a Vector
__device__ pVect
mv(float x, float y, float z){ return make_float3(x,y,z); }

__device__ float dot(pVect a, pVect b){ return a.x*b.x + a.y*b.y + a.z*b.z;}
__device__ float dot(pVect a, pNorm b){ return dot(a,b.v); }
__device__ float dot3(float4 a, float4 b){ return dot(m3(a),m3(b)); }

__device__ float mag_sq(pVect v){ return dot(v,v); }
__device__ float length(pVect a) {return sqrtf(mag_sq(a));}
__device__ pVect normalize(pVect a) { return rsqrtf(mag_sq(a))*a; }

// Make a Normal (a structure containing a normalized vector and length)
__device__ pNorm mn(pVect v)
{
  pNorm n;
  n.mag_sq = mag_sq(v);
  if ( n.mag_sq == 0 )
    {
      n.magnitude = 0;
      n.v.x = n.v.y = n.v.z = 0;
    }
  else
    {
      n.magnitude = sqrtf(n.mag_sq);
      n.v = (1.0/n.magnitude) * v;
    }
  return n;
}
__device__ pNorm mn(float4 a, float4 b) {return mn(b-a);}
__device__ pNorm mn(pCoor a, pCoor b) {return mn(b-a);}

// The unary - operator doesn't seem to work when used in an argument.
__device__ pNorm operator -(pNorm n)
{
  pNorm m;
  m.magnitude = n.magnitude;
  m.mag_sq = n.mag_sq;
  m.v = -n.v;
  return m;
}

struct pQuat {
  float w;
  pVect v;
};

// Make Quaternion
__device__ float4 mq(pNorm axis, float angle)
{
  return m4( sin(angle/2) * axis, cos(angle/2) );
}

// Make float4
__device__ float4 m4(pQuat q){ return make_float4(q.v.x,q.v.y,q.v.z,q.w); }
__device__ float4 m4(pNorm v, float w) { return m4(v.v,w); }


__device__ int
div_p2_ceil(int num, int den_lg)
{
  const int quot = num >> den_lg;
  return quot << den_lg == num ? quot : quot + 1;
}

// Cross Product of Two Vectors
__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}
__device__ pVect cross(pVect a, pNorm b){ return cross(a,b.v); }
__device__ pVect cross(pNorm a, pVect b){ return cross(a.v,b); }
__device__ pVect crossf3(float4 a, float4 b) { return cross(m3(a),m3(b)); }

// Cross Product of Vectors Between Coordinates
__device__ float3
 cross3(float3 a, float3 b, float3 c)
{
  float3 ab = a - b;
  float3 cb = c - b;
  return cross(ab,cb);
}
__device__ pVect cross3(pVect a, pVect b, pNorm c) { return cross3(a,b,c.v); }

__device__ float4 quat_mult(float4 a, float4 b)
{
  float w = a.w * b.w - dot3(a,b);
  float3 v = a.w * m3(b) + b.w * m3(a) + crossf3(a,b);
  return make_float4(v.x,v.y,v.z,w);
};


//
 /// Ball Physics Functions
//
// See demo-x-collide.cc for details.

__device__ pVect
point_rot_vel(CUDA_Ball_W ball, pNorm direction)
{
  return opt_ball_radius * cross( ball.angular_momentum, direction );
}

__device__ void
apply_tan_do(CUDA_Ball_W& ball, pNorm tact_dir, pVect force)
{
  pVect axis_torque = cross( tact_dir, force );
  ball.angular_momentum += axis_torque;
}

__device__ void
apply_deltao(CUDA_Ball_W& ball, pNorm tact_dir, pNorm force_dir, double deltao)
{
  apply_tan_do(ball, tact_dir, deltao * force_dir );
}

__device__ void
apply_tan_dv(CUDA_Ball_W& ball, pNorm tact_dir, pVect force)
{
  apply_tan_do(ball, tact_dir, r_inv * force);
}


///
/// Major Ball Physics Routines
///

// A time step is computed using two kernels, pass_pairs and
// pass_platform. The pass_pairs kernel, which might be launched
// several times, handles collisions between balls.  The pass_platform
// kernel handles collision between balls and the platform, and also
// updates position and orientation.


///
/// Collision (Penetration) Detection and Resolution Routine
///

// Used in both passes.

__device__ void
penetration_balls_resolve
(CUDA_Ball_W& ball1_r, CUDA_Ball_W& ball2_r, bool b2_real)
{
  /// Update velocity and angular momentum for a pair of balls in contact.

  CUDA_Ball_W *ball1 = &ball1_r;
  CUDA_Ball_W *ball2 = &ball2_r;
  pVect zero_vec = mv(0,0,0);
  pNorm dist = mn(ball1->position,ball2->position);

  // For Debugging
  //
  if ( b2_real ) { ball1->debug_pair_calls++;  ball2->debug_pair_calls++; }

  // Return if balls aren't touching.  Note avoidance of square root.
  //
  if ( dist.mag_sq >= two_r_sq ) return;

  // Update counters used for optimization (contact_count) and
  // to decide when to release new balls (collision_count).
  //
  ball1->collision_count++;  ball1->contact_count++;
  if ( b2_real ) { ball2->collision_count++; ball2->contact_count++; }


  /// WARNING:  This doesn't work: somefunc(-dist); 
  pNorm ndist = -dist;

  // Compute relative (approach) velocity.
  //
  pVect prev_appr_vel = ball1->prev_velocity - ball2->prev_velocity;
  const double prev_approach_speed = dot( prev_appr_vel, dist );

  const double loss_factor = 1 - opt_bounce_loss;

  // Compute change in speed based on how close balls touching, ignoring
  // energy loss.
  //
  const double appr_deltas_no_loss =
    ( two_r - dist.magnitude ) * elasticity_inv_dt * ball_mass_inv;

  // Change in speed accounting for energy loss. Only applied when
  // balls separating.
  //
  const double appr_deltas =
    prev_approach_speed > 0
    ? appr_deltas_no_loss : loss_factor * appr_deltas_no_loss;

  /// Update Linear Velocity
  //
  ball1->velocity -= appr_deltas * dist;
  if ( b2_real ) ball2->velocity += appr_deltas * dist;

  // Find speed on surface of balls at point of contact.
  //
  pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
  pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);

  // Find relative velocity of surfaces at point of contact
  // in the plane formed by their surfaces.
  //
  pVect tan_vel = prev_appr_vel - prev_approach_speed * dist;
  pNorm tact_vel_dir = mn(tact1_rot_vel - tact2_rot_vel + tan_vel);

  // Find change in velocity due to friction.
  //
  const double fric_dv_potential =
    fabs(appr_deltas_no_loss) * opt_friction_coeff;
  const double dv_limit_raw = tact_vel_dir.magnitude * mo_vel_factor;
  const double dv_limit = b2_real ? dv_limit_raw : 2 * dv_limit_raw;

  // If true, surfaces are not sliding or will stop sliding after
  // frictional forces applied. (If a ball surface isn't sliding
  // against another surface than it must be rolling.)
  //
  const bool will_roll = dv_limit <= fric_dv_potential;
  const double sliding_fric_deltav =
    will_roll ? dv_limit : fric_dv_potential;

  const double dv_tolerance = 0.000001;

  if ( sliding_fric_deltav > dv_tolerance )
    {
      // Apply frictional force.

      // Compute change in angular momentum due to friction.
      //
      const double fric_deltao = sliding_fric_deltav * v_to_do;

      // Apply torque (resulting in angular momentum change) and
      // linear force (resulting in velocity change).
      //
      apply_deltao(ball1_r,dist,tact_vel_dir,-fric_deltao);
      ball1->velocity -= sliding_fric_deltav * tact_vel_dir;

      // Ditto for the other ball, if it's real.
      if ( b2_real )
        {
          apply_deltao(ball2_r,dist,tact_vel_dir,-fric_deltao);
          ball2->velocity += sliding_fric_deltav * tact_vel_dir;
        }
    }

  {
    /// Torque
    //
    //
    // Account for forces of surfaces twisting against each
    // other. (For example, if one ball is spinning on top of
    // another.)
    //
    const double appr_omega =
      dot(ball1->angular_momentum,dist) - dot(ball2->angular_momentum,dist);
    const double fric_deltao_pot = fric_dv_potential * v_to_do;
    const bool rev = appr_omega < 0;
    const double fric_deltao = min(fabs(appr_omega),fric_deltao_pot);
    pVect delta_am = rev ? -fric_deltao * dist : fric_deltao * dist;
    ball1->angular_momentum -= delta_am;
    if ( b2_real ) ball2->angular_momentum += delta_am;
  }

  {
    /// Rolling Friction
    //
    // The rolling friction model used here is ad-hoc.

    pVect tan_b12_vel = b2_real ? 0.5 * tan_vel : zero_vec;
    const double torque_limit_sort_of = appr_deltas_no_loss
      * sqrt( opt_ball_radius - 0.25 * dist.mag_sq * r_inv );

    pVect tact1_rot_vel = point_rot_vel(ball1_r,dist);
    pVect tact1_roll_vel = tact1_rot_vel + tan_b12_vel;
    pNorm tact1_roll_vel_dir = mn(tact1_roll_vel);
    pVect lost_vel = zero_vec;

    const double rfric_loss1 =
      torque_limit_sort_of *
      ( tact1_roll_vel_dir.magnitude * opt_friction_roll /
        ( 1 + tact1_roll_vel_dir.magnitude * opt_friction_roll ) );
    
    pVect lost_vel1 =
      min(tact1_roll_vel_dir.magnitude, rfric_loss1) * tact1_roll_vel_dir;

    lost_vel = -lost_vel1;
    
    if ( b2_real )
      {
        pVect tact2_rot_vel = point_rot_vel(ball2_r,ndist);
        pVect tact2_roll_vel = tact2_rot_vel - tan_b12_vel;
        pNorm tact2_roll_vel_dir = mn(tact2_roll_vel);
        const double rfric_loss2 =
          torque_limit_sort_of *
          ( tact2_roll_vel_dir.magnitude * opt_friction_roll /
            ( 1 + tact2_roll_vel_dir.magnitude * opt_friction_roll ) );
        pVect lost_vel2 =
          min(tact2_roll_vel_dir.magnitude, rfric_loss2 ) * tact2_roll_vel_dir;

        lost_vel += lost_vel2;
      }

    apply_tan_dv(ball1_r,dist,lost_vel);
    if ( b2_real ) apply_tan_dv(ball2_r,dist,lost_vel);
  }
}

///
/// Pairs Pass
///
//
// Resolve ball collisions with each other.

__global__ void pass_pairs
(int prefetch_offset, int schedule_offset, int round_cnt);

__host__ void 
pass_pairs_launch
(dim3 dg, dim3 db, int prefetch_offset, int schedule_offset, int round_cnt)
{
  pass_pairs<<<dg,db>>>(prefetch_offset,schedule_offset,round_cnt);
}

__device__ void
pass_pairs(int prefetch_offset, int schedule_offset, int round_cnt)
{
  const int tid = threadIdx.x;
  const int max_balls_per_thread =
    int( ceil( float(BALLS_PER_BLOCK) / blockDim.x ) );

  // Initialized variables used to access balls_needed and tacts_schedule
  // arrays.
  //
  const int si_block_size = blockIdx.x * max_balls_per_thread * blockDim.x;
  const int si_block_base = prefetch_offset + si_block_size + tid;
  const int sp_block_size = blockIdx.x * round_cnt * blockDim.x;
  const int sp_block_base = schedule_offset + sp_block_size + tid;

  /// Shared memory array holding balls updated by this block.
  //
  __shared__ CUDA_Ball_W sm_balls[BALLS_PER_BLOCK];

  /// Prefetch balls to shared memory.
  //
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      const int m_idx = block_balls_needed[ si_block_base + i * blockDim.x ];
      CUDA_Ball_W& ball = sm_balls[idx];
      ball.m_idx = m_idx;
      if ( m_idx < 0 ) continue;
      ball.velocity = xyz(balls_x.velocity[m_idx]);
      ball.prev_velocity = xyz(balls_x.prev_velocity[m_idx]);
      ball.position = xyz(balls_x.position[m_idx]);
      ball.angular_momentum = xyz(balls_x.angular_momentum[m_idx]);

      int4 tact_counts = balls_x.tact_counts[m_idx];
      ball.collision_count = tact_counts.x;
      ball.contact_count = tact_counts.y;
      ball.debug_pair_calls = tact_counts.z;
    }

  __syncthreads();

  /// Resolve Collisions
  //
  for ( int round=0; round<round_cnt; round++ )
    {
      SM_Idx2 indices = tacts_schedule[ sp_block_base + round * blockDim.x ];

      // Wait for all threads to reach this point (to avoid having
      // two threads operate on the same ball simultaneously).
      //
      __syncthreads();

      if ( indices.x == indices.y ) continue;
      penetration_balls_resolve(sm_balls[indices.x],sm_balls[indices.y],true);
    }

  __syncthreads();

  /// Copy Ball Data to Memory
  //
  for ( int i=0; i<max_balls_per_thread; i++ )
    {
      int idx = tid + i * blockDim.x;
      if ( idx >= BALLS_PER_BLOCK ) continue;
      CUDA_Ball_W& ball = sm_balls[idx];
      const int m_idx = ball.m_idx;
      if ( m_idx < 0 ) continue;

      set_f4(balls_x.velocity[m_idx], ball.velocity);
      set_f4(balls_x.angular_momentum[m_idx], ball.angular_momentum);

      int4 tact_counts;
      tact_counts.x = ball.collision_count;
      tact_counts.y = ball.contact_count;
      tact_counts.z = ball.debug_pair_calls;
      balls_x.tact_counts[m_idx] = tact_counts;
    }
}


///
/// Platform Pass
///
//
// Resolve ball collisions with platform, also update ball position
// and orientation.

__device__ void platform_collision(CUDA_Ball_W& ball);
__global__ void pass_platform(int ball_count);

__host__ hipError_t
cuda_get_attr_plat_pairs
(struct hipFuncAttributes *attr_platform,
 struct hipFuncAttributes *attr_pairs)
{
  // Return attributes of CUDA functions. The code needs the
  // maximum number of threads.
  hipError_t e1 = hipFuncGetAttributes(attr_platform,reinterpret_cast<const void*>(pass_platform));
  if ( e1 ) return e1;
  hipError_t e2 = hipFuncGetAttributes(attr_pairs,reinterpret_cast<const void*>(pass_pairs));
  return e2;
}

__host__ void 
pass_platform_launch
(dim3 dg, dim3 db, int ball_count)
{
  pass_platform<<<dg,db>>>(ball_count);
}

__global__ void
pass_platform(int ball_count)
{
  /// Main CUDA routine for resolving collisions with platform and
  /// updating ball position and orientation.

  // One ball per thread.

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= ball_count ) return;

  CUDA_Ball_W ball;

  /// Copy ball data from memory to local variables.
  //
  //  Local variables hopefully will be in GPU registers, not
  //  slow local memory.
  //
  ball.prev_velocity = xyz(balls_x.prev_velocity[idx]);
  ball.velocity = xyz(balls_x.velocity[idx]) + gravity_accel_dt;
  set_f3(ball.position,balls_x.position[idx]);
  set_f3(ball.angular_momentum, balls_x.angular_momentum[idx]);
  int4 tact_counts = balls_x.tact_counts[idx];
  ball.collision_count = tact_counts.x;
  ball.contact_count = tact_counts.y;

  /// Handle Ball/Platform Collision
  //
  platform_collision(ball);

  /// Update Position and Orientation
  //
  ball.position += delta_t * ball.velocity;
  pNorm axis = mn(ball.angular_momentum);
  balls_x.orientation[idx] =
    quat_mult( balls_x.orientation[idx],
               mq( axis, delta_t * axis.magnitude ) );

  /// Copy other updated data to memory.
  //
  set_f4(balls_x.velocity[idx], ball.velocity);
  set_f4(balls_x.prev_velocity[idx], ball.velocity);
  set_f4(balls_x.angular_momentum[idx], ball.angular_momentum);
  set_f4(balls_x.position[idx],ball.position);
  tact_counts.x = ball.collision_count;
  tact_counts.y = ball.contact_count << 8;
  tact_counts.z = tact_counts.z << 16;
  balls_x.tact_counts[idx] = tact_counts;
}

__device__ void
platform_collision(CUDA_Ball_W& ball)
{
  /// Check if ball in contact with platform, if so apply forces.

  pCoor pos = ball.position;
  bool collision_possible =
    pos.y < opt_ball_radius
    && pos.x >= platform_xmin_mr && pos.x <= platform_xmax_pr
    && pos.z >= platform_zmin_mr && pos.z <= platform_zmax_pr;

  if ( !collision_possible ) return;

  CUDA_Ball_W pball;

  pCoor axis = mc(platform_xmid,0,pos.z);

  // Test for different ways ball can touch platform. If contact
  // is found find position of an artificial platform ball (pball)
  // that touches the real ball at the same place and angle as
  // the platform. This pball will be used for the ball-ball penetration
  // routine, penetration_balls_resolve.

  if ( pos.y > 0 )
    {
      // Possible contact with upper edge of platform.
      //
      pCoor tact
        = mc(pos.x > platform_xmid ? platform_xmax : platform_xmin, 0, pos.z);
      pVect pos_tact = tact - pos;
      float tact_dir_mag_sq = mag_sq(pos_tact);
      if ( tact_dir_mag_sq >= two_r_sq ) return;
      pball.position =
        tact + opt_ball_radius / sqrt(tact_dir_mag_sq) * pos_tact;
    }
  else if ( pos.z > platform_zmax || pos.z < platform_zmin )
    {
      // Possible contact with side (curved) edges of platform.
      //
      pNorm ball_dir = mn(axis,pos);
      if ( ball_dir.mag_sq <= short_xrad_sq ) return;
      const float zedge =
        pos.z > platform_zmax ? platform_zmax : platform_zmin;
      pCoor axis_edge = mc(platform_xmid,0,zedge);
      pCoor tact = axis_edge + platform_xrad * ball_dir;
      pNorm tact_dir = mn(pos,tact);
      if ( tact_dir.mag_sq >= two_r_sq ) return;
      pball.position = tact + opt_ball_radius * tact_dir;
    }
  else
    {
      // Possible contact with surface of platform.
      //
      pNorm tact_dir = mn(axis,pos);
      if ( tact_dir.mag_sq <= short_xrad_sq ) return;
      pball.position = axis + (opt_ball_radius+platform_xrad) * tact_dir;
    }

  // Finish initializing platform ball, and call routine to
  // resolve penetration.
  //
  pVect zero_vec = mv(0,0,0);
  pball.angular_momentum = zero_vec;
  pball.prev_velocity = pball.velocity = zero_vec;
  penetration_balls_resolve(ball,pball,false);
}

