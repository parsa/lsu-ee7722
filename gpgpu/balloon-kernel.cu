#include "hip/hip_runtime.h"
/// LSU EE 7700-1 (Sp 2009), Graphics Processors
//
 /// Balloon Simulation

// $Id:$

 /// Purpose
//
//   Demonstrate use of gpu for physics.


#include "balloon.cuh"

__constant__ CUDA_Tri_Strc* tri_strc;
__constant__ CUDA_Vtx_Strc* vtx_strc;
__constant__ CUDA_Tri_Data* tri_data;
__constant__ float* tower_volumes;
__constant__ float3* centroid_parts;

texture<float4> vtx_data_tex;
texture<float4> tri_data_tex;

__constant__ CUDA_Tri_Work_Strc* tri_work_strc;
__constant__ int tri_work_per_vtx;
__constant__ int tri_work_per_vtx_lg;

__constant__ float volume_cpu;
__constant__ int tri_count;
__constant__ int point_count;
__constant__ bool opt_gravity;

__constant__ float spring_constant;
__constant__ float damping_v;
__constant__ float pressure_factor_coeff;
__constant__ float gas_m_over_temp;

__constant__ float air_resistance;
__constant__ float gas_mass_per_vertex;
__constant__ float air_particle_mass;
__constant__ float gravity_mag;

__constant__ float delta_t;
__constant__ float rep_constant;
__constant__ float point_mass;
__constant__ float point_mass_inv;

__constant__ float platform_xmin;
__constant__ float platform_xmax;
__constant__ float platform_zmin;
__constant__ float platform_zmax;


__device__ int
div_p2_ceil(int num, int den_lg)
{
  const int quot = num >> den_lg;
  return quot << den_lg == num ? quot : quot + 1;
}

__device__ float3
vec_add(float3 a, float3 b){return make_float3(a.x+b.x,a.y+b.y,a.z+b.z);}
__device__ float3
vec_add3(float3 a, float3 b, float3 c) { return vec_add(a,vec_add(b,c)); }
__device__ void vec_addto(float3& a, float3 b)
{
  float3 sum = vec_add(a,b);
  a = sum;
}

__device__ float3
vec_sub(float3 a, float3 b){return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);}

__device__ float3
vec_scale(float s, float3 a) {return make_float3(s*a.x,s*a.y,s*a.z);}

__device__ float
dot(float3 a, float3 b){return a.x*b.x + a.y*b.y + a.z*b.z;}

__device__ float length(float3 a) {return sqrtf(dot(a,a));}

__device__ float3
normalize(float3 a) { return vec_scale(rsqrtf(dot(a,a)),a); }

__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}

__device__ float3
cross3(float3 a, float3 b, float3 c)
{
  float3 ab = vec_sub(a,b);
  float3 cb = vec_sub(c,b);
  return cross(ab,cb);
}

 ///
 /// Reduction Routine
 ///
 // Computes a sum of floats over all threads in the block.
 //
__device__ float
reduce(int block_lg, float *shared_array, float my_value, bool all)
{
  const int tid = threadIdx.x;
  const int block_lg_h = block_lg > 1 ? block_lg >> 1 : block_lg;
  const int block_lg_l = block_lg - block_lg_h;
  const int lower_size = block_lg_l >= 0 ? 1 << block_lg_l : 0;

  float vol_sum = shared_array[tid] = my_value;
  __syncthreads();

  // Round 1
  //
  if ( tid < lower_size )
    {
      // Note: CUDA is not good at unrolling loops or optimizing once
      // unrolled. For that matter, it's not good at scheduling
      // either. (CUDA 2.1)  That's why to loops below are hand unrolled.
      // Thankfully CUDA does optimize out the if statements.
      //
#define ITER(i) vol_sum += shared_array[ (i << block_lg_l ) + tid ]
#define ITER2(i) { ITER(i); ITER(i+1); }
#define ITER4(i) { ITER2(i); ITER2(i+2); }
#define ITER8(i) { ITER4(i); ITER4(i+4); }
      if ( block_lg_h >= 1 ) ITER(1);
      if ( block_lg_h >= 2 ) ITER2(2);
      if ( block_lg_h >= 3 ) ITER4(4);
      if ( block_lg_h >= 4 ) ITER8(8);
      if ( block_lg_h >= 5 ) { ITER8(16); ITER8(24); }
#undef ITER

      shared_array[tid] = vol_sum;
    }

  // Round 2
  //
  __syncthreads();

  if ( tid == 0 )
    {
#define ITER(i) vol_sum += shared_array[i];
      if ( block_lg_l >= 1 ) ITER(1);
      if ( block_lg_l >= 2 ) ITER2(2);
      if ( block_lg_l >= 3 ) ITER4(4);
      if ( block_lg_l >= 4 ) ITER8(8);
      if ( block_lg_l >= 5 ) { ITER8(16); ITER8(24); }
#undef ITER
    }

  if ( !all ) return vol_sum;
  if ( tid == 0 ) shared_array[0] = vol_sum;
  __syncthreads();
  return shared_array[0];
}


 ///
 /// Texture Access Convenience Functions
 ///

// These functions access data via the texture cache and place them in
// appropriate data structures. (Texture cache fetches can not return
// structures and cannot return vectors of length 3.)

__device__ float3 vtx_data_pos(int idx)
{
  const int idx_tex = idx * 3 + 2;
  const float4 pos4 = tex1Dfetch(vtx_data_tex, idx_tex);
  return make_float3(pos4.x,pos4.y,pos4.z);
}

__device__ float3 vtx_data_vel(int idx)
{
  const int idx_tex = idx * 3 + 1;
  const float4 pos4 = tex1Dfetch(vtx_data_tex, idx_tex);
  return make_float3(pos4.x,pos4.y,pos4.z);
}

__device__ float3 tri_data_surface_normal(int idx)
{
  const int idx_tex = idx * 3;
  const float4 sn = tex1Dfetch(tri_data_tex, idx_tex);
  return make_float3(sn.x,sn.y,sn.z);
}

__device__ float3 tri_data_force(int idx, int member)
{
  const int idx_tex_base = idx * 3;
  float4 el, eh;
  switch (member) {
  case 0:                       // force_p
    el = tex1Dfetch(tri_data_tex,idx_tex_base);
    eh = tex1Dfetch(tri_data_tex,idx_tex_base+1);
    return make_float3(el.w,eh.x,eh.y);
  case 1:                       // force_q
    el = tex1Dfetch(tri_data_tex,idx_tex_base+1);
    eh = tex1Dfetch(tri_data_tex,idx_tex_base+2);
    return make_float3(el.z,el.w,eh.x);
  case 2:                       // force_r
    el = tex1Dfetch(tri_data_tex,idx_tex_base+2);
    return make_float3(el.y,el.z,el.w);
  default:                      // Unreachable.
    return make_float3(0,0,0);
  }
}


 ///
 /// Compute Repulsion Force
 ///
//
// Used by one- and two-pass algorithms.
//
__device__ float3
repforce_compute(float3 p_pos, int po_idx)
{
  const float3 po_pos = vtx_data_pos(po_idx);
  const float3 p_to_q = vec_sub(p_pos,po_pos);
  const float mag_sq = dot(p_to_q,p_to_q);
  const float dist_sq_inv = rep_constant / max(0.001,mag_sq);
  const float3 p_to_q_n = normalize(p_to_q);
  return vec_scale(dist_sq_inv, p_to_q_n);
}


///
/// Two-Pass Code
///

__global__ void pass_triangles();
__global__ void pass_vertices(CUDA_Vtx_Data *vtx_data_out);


 __host__ void
pass_triangles_launch
 (dim3 dg, dim3 db, CUDA_Vtx_Data *vtx_data, size_t vtx_data_size)
 {
   size_t offset;
   hipBindTexture(&offset, vtx_data_tex, vtx_data, vtx_data_size);
   pass_triangles<<<dg,db>>>();
 }

__global__ void
pass_triangles()
{
  const int tid = threadIdx.x;
  const int ti = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float volumes[CUDA_TRI_BLOCK_SIZE];
  if ( ti >= tri_count ) volumes[tid] = 0;
  __syncthreads();
  if ( ti >= tri_count ) return;

  const CUDA_Tri_Strc ts = tri_strc[ti];

  const float3 ppos = vtx_data_pos(ts.pi);
  const float3 qpos = vtx_data_pos(ts.qi);
  const float3 rpos = vtx_data_pos(ts.ri);

  const float3 center = vec_scale(1.0/3.0, vec_add3(ppos,qpos,rpos));
  const float3 pqr_cross = cross3(qpos,ppos,rpos);

  const float triangle_area_x2 = length(pqr_cross);
  const float tower_volume_x2 = -pqr_cross.y * center.y;

  float3 force_p = repforce_compute(ppos,ts.pi_opp);
  float3 force_q = repforce_compute(qpos,ts.qi_opp);
  float3 force_r = repforce_compute(rpos,ts.ri_opp);

  const float3 p_to_c = vec_sub(center,ppos);
  const float3 q_to_c = vec_sub(center,qpos);
  const float3 r_to_c = vec_sub(center,rpos);

  const float perimeter = length(p_to_c) + length(q_to_c) + length(r_to_c);

  tri_data[ti].surface_normal = pqr_cross;

  const float length_relaxed = ts.length_relaxed;
  const float eff_length = max(0.0f, perimeter - length_relaxed );
  const float spring_force = eff_length * spring_constant;

  tri_data[ti].force_p = vec_add(force_p, vec_scale(spring_force, p_to_c));
  tri_data[ti].force_q = vec_add(force_q, vec_scale(spring_force, q_to_c));
  tri_data[ti].force_r = vec_add(force_r, vec_scale(spring_force, r_to_c));

  const float vol_sum =
    reduce(CUDA_TRI_BLOCK_LG,volumes,tower_volume_x2 * 0.5f,false);
  if ( threadIdx.x == 0 ) tower_volumes[ blockIdx.x ] = vol_sum;
}

__host__ void
pass_vertices_launch
(dim3 dg, dim3 db, CUDA_Tri_Data *tri_data, CUDA_Vtx_Data *vtx_out,
 size_t tri_data_size)
{
  size_t offset;
  hipBindTexture(&offset, tri_data_tex, tri_data, tri_data_size);
  pass_vertices<<<dg,db>>>(vtx_out);
  hipUnbindTexture(vtx_data_tex);
  hipUnbindTexture(tri_data_tex);
}

__global__ void
pass_vertices(CUDA_Vtx_Data *vtx_data_out)
{
  const int tid = threadIdx.x;
  const int vi = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float volumes[CUDA_VTX_BLOCK_SIZE];

  const int grid_dim_tri = div_p2_ceil(tri_count,CUDA_TRI_BLOCK_LG);
  const int vol_per_thread = div_p2_ceil(grid_dim_tri,CUDA_VTX_BLOCK_LG);
  const int start = tid * vol_per_thread;
  const int stop = min(grid_dim_tri, start + vol_per_thread);
  float my_vol = 0;
  for ( int i=start; i<stop; i++ ) my_vol += tower_volumes[i];
  const float volume = reduce(CUDA_VTX_BLOCK_LG,volumes,my_vol,true);

  const float friction_coefficient = .04;
  const float bounce_factor = 0.0;
  const float mass = 1.0;

  const float3 gravity = make_float3(0.0,-gravity_mag,0.0);

  const float3 pos = vtx_data_pos(vi);
  const float3 vel = vtx_data_vel(vi);
  const CUDA_Vtx_Strc vs = vtx_strc[vi];

  float3 force_spring = make_float3(0.0,0.0,0.0);
  float3 surface_normal_sum = make_float3(0.0,0.0,0.0);

#define TRI_BODY(i)                                                           \
  { const int idx_packed = vs.n##i;                                           \
    if ( idx_packed != -1 )                                                   \
      {                                                                       \
        const int idx_base = idx_packed >> 2;                                 \
        const int idx_force = idx_packed & 0x3;                               \
        const float3 surface_normal_t = tri_data_surface_normal(idx_base);    \
        vec_addto(surface_normal_sum, surface_normal_t);                      \
        const float3 force_x = tri_data_force(idx_base,idx_force);            \
        vec_addto(force_spring, force_x);                                     \
      }}

#if 1
  TRI_BODY(0); TRI_BODY(1); TRI_BODY(2); TRI_BODY(3);
  TRI_BODY(4); TRI_BODY(5); TRI_BODY(6); TRI_BODY(7);
#else
  for ( int i=0; i<VTX_TRI_DEG_MAX; i++ ) TRI_BODY(i);
#endif
#undef TRI_BODY

  float3 surface_normal = vec_scale(1./6., surface_normal_sum);

  float pressure_factor = pressure_factor_coeff / fabs(volume);

  float pressure =
    opt_gravity
    ? pressure_factor * exp( - gas_m_over_temp * pos.y )
    : pressure_factor;
  float air_pressure =
    opt_gravity ? exp( - 0.2f * air_particle_mass * pos.y ) : 1.0;

  float3 force_pressure = vec_scale(air_pressure - pressure, surface_normal);

  float3 force = force_pressure;

  float3 vel_norm = normalize(vel);
  float facing_area = max(0.0f,-dot(vel_norm,surface_normal));
  float3 force_ar = vec_scale( - air_resistance * facing_area, vel);

  float3 gforce = vec_scale(point_mass * mass, gravity);
  vec_addto(force, gforce);

  vec_addto(force, force_ar);

  float3 force_ns = force;

  vec_addto(force, force_spring);

  float mass_wgas_inv_dt =
    delta_t / ( point_mass * mass + gas_mass_per_vertex );

  float3 delta_vns = vec_scale(mass_wgas_inv_dt, force_ns);
  float3 delta_vs = vec_scale(mass_wgas_inv_dt, force_spring);
  float3 delta_v = vec_add(delta_vns, delta_vs);

  float3 pos_next =
    vec_add(pos, vec_scale(delta_t, vec_add( vel,  vec_scale(0.5f,delta_v) )));

  float3 vel_next = vec_add3(vel,  vec_scale(damping_v, delta_vs), delta_vns);

  const bool platform_aligned =
    pos_next.x >= platform_xmin && pos_next.x <= platform_xmax
    && pos_next.z >= platform_zmin && pos_next.z <= platform_zmax;
  const bool above_to_below =  pos_next.y <= 0.0f && pos.y >= 0.0f;

  if ( platform_aligned && above_to_below )
    {
      pos_next.y = 0.0;
      vel_next.y = - bounce_factor * vel_next.y;
      const float f_y =
        min(0.0, gforce.y + force_spring.y - pressure * surface_normal.y);
      const float friction_force = -f_y * friction_coefficient;
      const float delta_v = friction_force * delta_t / ( point_mass * mass );
      const float3 xzvel = make_float3(vel_next.x,0,vel_next.z);
      if ( length(xzvel) <= delta_v ) {
        vel_next.x = 0.0;  vel_next.z = 0.0;
      }
      else
        vec_addto(vel_next, vec_scale( -delta_v, normalize(xzvel) ));
    }

  vtx_data_out[vi].surface_normal = surface_normal;
  vtx_data_out[vi].vel = vel_next;
  vtx_data_out[vi].pos = pos_next;
}



///
/// One-Pass Code
///

struct CUDA_Tri_Shared {
  // Note: make sure GCD(16,sizeof(this)/4) = 1, e.g., not a power of 2!
  float3 center;
  float spring_force;
  float3 surface_normal;
};

__global__ void pass_unified
(CUDA_Vtx_Data *vtx_data_out, float *tv_in, float *tv_out);

__host__ void
pass_unified_launch
(dim3 dg, dim3 db,
 CUDA_Vtx_Data *vtx_data_in, CUDA_Vtx_Data *vtx_out,
 float *tv_in, float *tv_out,
 size_t tri_data_size, size_t vtx_data_size )
{
  size_t offset;
  hipBindTexture(&offset, vtx_data_tex, vtx_data_in, vtx_data_size);
  pass_unified<<<dg,db>>>(vtx_out,tv_in,tv_out);
  hipUnbindTexture(vtx_data_tex);
}

__global__ void
pass_unified
(CUDA_Vtx_Data *vtx_data_out, float *tower_volumes_in, float *tower_volumes_out)
{
  const int tid = threadIdx.x;
  const int vtx_bk_base = __mul24(blockIdx.x , blockDim.x);
  const int vi = vtx_bk_base + tid;
  const int work_idx_base = __mul24(vi , tri_work_per_vtx);
  const int block_lg = CUDA_VTX_BLOCK_LG;
  const int block_size = CUDA_VTX_BLOCK_SIZE;

  __shared__ float volumes[block_size], volumes_read[block_size];
  __shared__ CUDA_Tri_Shared tri_shared[block_size];

  // This routine computes information (area, surface normal, and a
  // force) for several triangles and up to one vertex. Each vertex
  // uses information from up to eight triangles. The triangles that a
  // vertex needs are computed somewhere in the same block, but not
  // necessarily in the same thread, so vertices get the triangle
  // information they need through shared memory.

  ///
  /// Triangle Round
  ///
  // Note that several triangles are processed by a thread.

  float local_volume_x2 = 0;
  float3 force_spring = make_float3(0,0,0);
  float3 surface_normal_sum = make_float3(0,0,0);

  const float3 pos =
    vi < point_count ? vtx_data_pos(vi) : make_float3(0,0,0);

  for ( int i=0; i<tri_work_per_vtx; i++ )
    {
      const CUDA_Tri_Work_Strc ts = tri_work_strc[work_idx_base+i];
      const int ts_pull_i =  ts.pull_i;

      /// Compute information for a triangle (if there is one).
      //
      if ( ts.pi != -1 )
        {
          const float3 ppos = vtx_data_pos(ts.pi);
          const float3 qpos = vtx_data_pos(ts.qi);
          const float3 rpos = vtx_data_pos(ts.ri);

          const float3 center = vec_scale(1.0/3.0, vec_add3(ppos,qpos,rpos));
          const float3 pqr_cross = cross3(qpos,ppos,rpos);
          const float tower_volume_x2 = -pqr_cross.y * center.y;

          const bool use_vol = ts_pull_i & 1;
          if ( use_vol ) local_volume_x2 += tower_volume_x2;

          const float3 p_to_c = vec_sub(center,ppos);
          const float3 q_to_c = vec_sub(center,qpos);
          const float3 r_to_c = vec_sub(center,rpos);
          const float perimeter = length(p_to_c)+length(q_to_c)+length(r_to_c);
          const float length_relaxed = ts.length_relaxed;
          const float eff_length = max(0.0f, perimeter - length_relaxed );
          const float spring_force = eff_length * spring_constant;

          tri_shared[tid].center = center;
          tri_shared[tid].surface_normal = pqr_cross;
          tri_shared[tid].spring_force = spring_force;
        }

      __syncthreads();

      const int pull_i = ts_pull_i >> 1;

      /// Accumulate sum of triangle information computed above.
      //
      // Note that loops are hand unrolled to overcome compiler
      // limitations.
      //
#define VTX_PULL(idx)                                                         \
      if ( idx < pull_i )                                                     \
        {                                                                     \
          const int tri_id = ts.pull_tid_##idx;                               \
          const float3 center = tri_shared[tri_id].center;                    \
          const float3 surface_normal_t = tri_shared[tri_id].surface_normal;  \
          vec_addto(surface_normal_sum,surface_normal_t);                     \
          vec_addto(force_spring,repforce_compute(pos,ts.vi_opp##idx));       \
          const float3 p_to_c = vec_sub(center,pos);                          \
          const float spring_force = tri_shared[tri_id].spring_force;         \
          vec_addto(force_spring, vec_scale(spring_force, p_to_c));           \
        }

      VTX_PULL(0); VTX_PULL(1); VTX_PULL(2); VTX_PULL(3);
    }

  /// Write Volumes
  //
  // Compute sum of volumes of each thread this block, then write to a
  // global array for use in next time step.
  //
  const float vol_sum = reduce(block_lg,volumes,local_volume_x2 * 0.5f,false);
  if ( tid == 0 ) tower_volumes_out[ blockIdx.x ] = vol_sum;

  /// Read Volumes
  //
  // Retrieve volumes of blocks written in the last time step and
  // compute their sum.
  //
  const int grid_dim_vol = gridDim.x;
  const int vol_per_thread = div_p2_ceil(gridDim.x,block_lg);
  const int start = tid * vol_per_thread;
  const int stop = min(grid_dim_vol, start + vol_per_thread);
  const int tid_limit = min(block_size, gridDim.x);
  float my_vol = 0;
  for ( int i=start; i<stop; i++ ) my_vol += tower_volumes_in[i];
  const float volume = reduce(block_lg,volumes_read,my_vol,true);

  ///
  /// Vertex Round
  ///
  // Compute new position and velocity.

  if ( vi >= point_count ) return;

  const float friction_coefficient = .04;
  const float bounce_factor = 0.0;
  const float mass = 1.0;

  const float3 gravity = make_float3(0.0,-gravity_mag,0.0);

  const float3 vel = vtx_data_vel(vi);

  const float3 surface_normal = vec_scale((1./6.), surface_normal_sum);

  float pressure_factor = pressure_factor_coeff / fabs(volume);

  float pressure =
    opt_gravity
    ? pressure_factor * exp( - gas_m_over_temp * pos.y )
    : pressure_factor;
  float air_pressure =
    opt_gravity ? exp( - 0.2f * air_particle_mass * pos.y ) : 1.0;

  float3 force_pressure = vec_scale(air_pressure - pressure, surface_normal);

  float3 force = force_pressure;

  float3 vel_norm = normalize(vel);
  float facing_area = max(0.0f,-dot(vel_norm,surface_normal));
  float3 force_ar = vec_scale( - air_resistance * facing_area, vel);

  float3 gforce = vec_scale(point_mass * mass, gravity);
  vec_addto(force, gforce);

  vec_addto(force, force_ar);

  float3 force_ns = force;

  vec_addto(force, force_spring);

  float mass_wgas_inv_dt =
    delta_t / ( point_mass * mass + gas_mass_per_vertex );

  float3 delta_vns = vec_scale(mass_wgas_inv_dt, force_ns);
  float3 delta_vs = vec_scale(mass_wgas_inv_dt, force_spring);
  float3 delta_v = vec_add(delta_vns, delta_vs);

  float3 pos_next =
    vec_add(pos, vec_scale(delta_t, vec_add( vel,  vec_scale(0.5,delta_v) )));

  float3 vel_next = vec_add3(vel,  vec_scale(damping_v, delta_vs), delta_vns);

  const bool platform_aligned =
    pos_next.x >= platform_xmin && pos_next.x <= platform_xmax
    && pos_next.z >= platform_zmin && pos_next.z <= platform_zmax;
  const bool above_to_below =  pos_next.y <= 0.0f && pos.y >= 0.0f;

  if ( platform_aligned && above_to_below )
    {
      pos_next.y = 0.0;
      vel_next.y = - bounce_factor * vel_next.y;
      const float f_y =
        min(0.0f, gforce.y + force_spring.y - pressure * surface_normal.y);
      const float friction_force = -f_y * friction_coefficient;
      const float delta_v = friction_force * delta_t / ( point_mass * mass );
      const float3 xzvel = make_float3(vel_next.x,0,vel_next.z);
      if ( length(xzvel) <= delta_v ) {
        vel_next.x = 0.0;  vel_next.z = 0.0;
      }
      else
        vec_addto(vel_next, vec_scale( -delta_v, normalize(xzvel) ));
    }

  vtx_data_out[vi].surface_normal = surface_normal;
  vtx_data_out[vi].vel = vel_next;
  vtx_data_out[vi].pos = pos_next;
}
