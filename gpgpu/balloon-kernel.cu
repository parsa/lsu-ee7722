#include "hip/hip_runtime.h"
/// LSU EE 4702-1 (Fall 2011), GPU Programming
//
 /// Balloon Simulation

// $Id:$

 /// Purpose
//
//   Demonstrate use of gpu for physics.


#include "balloon.cuh"

__constant__ CUDA_Tri_Strc* tri_strc;
__constant__ CUDA_Tri_Strc_X tri_strc_x;
__constant__ CUDA_Vtx_Strc* vtx_strc;
__constant__ CUDA_Vtx_Strc_X vtx_strc_x;
__constant__ CUDA_Tri_Data* tri_data;
__constant__ CUDA_Vtx_Data *vtx_data_0, *vtx_data_1;
__constant__ CUDA_Vtx_Data_X vtx_data_x0, vtx_data_x1;
__constant__ float *tower_volumes[2];
__constant__ float3* centroid_parts;

texture<float4> vtx_data_pos_tex;
texture<float4> tri_data_tex;

__constant__ CUDA_Tri_Work_Strc* tri_work_strc;
__constant__ CUDA_Tri_Work_Strc_X tri_work_strc_x;
__constant__ int tri_work_per_vtx;
__constant__ int tri_work_per_vtx_lg;

__constant__ float volume_cpu;
__constant__ int tri_count;
__constant__ int point_count;
__constant__ int tethered_idx;
__constant__ bool opt_gravity;

__constant__ float spring_constant;
__constant__ float damping_v;
__constant__ float pressure_factor_coeff;
__constant__ float gas_m_over_temp;

__constant__ float air_resistance;
__constant__ float gas_mass_per_vertex;
__constant__ float air_particle_mass;
__constant__ float gravity_mag;

__constant__ float delta_t;
__constant__ float rep_constant;
__constant__ float point_mass;
__constant__ float point_mass_inv;

__constant__ float platform_xmin;
__constant__ float platform_xmax;
__constant__ float platform_zmin;
__constant__ float platform_zmax;


__device__ float4 make_float4(float3 f3, float f)
{
  return make_float4(f3.x,f3.y,f3.z,f);
}

__device__ float3 make_float3(float4 f4){return make_float3(f4.x,f4.y,f4.z);}

__device__ int
div_p2_ceil(int num, int den_lg)
{
  const int quot = num >> den_lg;
  return quot << den_lg == num ? quot : quot + 1;
}

__device__ float3
vec_add(float3 a, float3 b){return make_float3(a.x+b.x,a.y+b.y,a.z+b.z);}
__device__ float3
vec_add3(float3 a, float3 b, float3 c) { return vec_add(a,vec_add(b,c)); }
__device__ void vec_addto(float3& a, float3 b)
{
  float3 sum = vec_add(a,b);
  a = sum;
}

__device__ float3
vec_sub(float3 a, float3 b){return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);}

__device__ float3
vec_scale(float s, float3 a) {return make_float3(s*a.x,s*a.y,s*a.z);}

__device__ float
dot(float3 a, float3 b){return a.x*b.x + a.y*b.y + a.z*b.z;}

__device__ float length(float3 a) {return sqrtf(dot(a,a));}

__device__ float3
normalize(float3 a) { return vec_scale(rsqrtf(dot(a,a)),a); }

__device__ float3
cross(float3 a, float3 b)
{
  return make_float3
    ( a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x );
}

__device__ float3
cross3(float3 a, float3 b, float3 c)
{
  float3 ab = vec_sub(a,b);
  float3 cb = vec_sub(c,b);
  return cross(ab,cb);
}

 ///
 /// Reduction Routine
 ///
 // Computes a sum of floats over all threads in the block.
 //
__device__ float
reduce(int block_lg, float *shared_array, float my_value, bool all)
{
  const int tid = threadIdx.x;
  const int block_lg_h = block_lg >> 1;
  const int block_lg_l = block_lg - block_lg_h;
  const int lower_size = 1 << block_lg_l;

  float vol_sum = shared_array[tid] = my_value;
  __syncthreads();

  // Round 1
  //
  if ( tid < lower_size )
    {
      // Note: CUDA is not good at unrolling loops or optimizing once
      // unrolled. For that matter, it's not good at scheduling
      // either. (CUDA 2.1) That's why the two loops below are hand
      // unrolled. Thankfully CUDA does optimize out the if
      // statements.
#define ITER(i) vol_sum += shared_array[ (i << block_lg_l ) + tid ]
#define ITER2(i) { ITER(i); ITER(i+1); }
#define ITER4(i) { ITER2(i); ITER2(i+2); }
#define ITER8(i) { ITER4(i); ITER4(i+4); }
      if ( block_lg_h >= 1 ) ITER(1);
      if ( block_lg_h >= 2 ) ITER2(2);
      if ( block_lg_h >= 3 ) ITER4(4);
      if ( block_lg_h >= 4 ) ITER8(8);
      if ( block_lg_h >= 5 ) { ITER8(16); ITER8(24); }
#undef ITER

      shared_array[tid] = vol_sum;
    }

  // Round 2
  //
  __syncthreads();

  if ( tid == 0 )
    {
#define ITER(i) vol_sum += shared_array[i];
      if ( block_lg_l >= 1 ) ITER(1);
      if ( block_lg_l >= 2 ) ITER2(2);
      if ( block_lg_l >= 3 ) ITER4(4);
      if ( block_lg_l >= 4 ) ITER8(8);
      if ( block_lg_l >= 5 ) { ITER8(16); ITER8(24); }
#undef ITER
    }

  if ( !all ) return vol_sum;
  if ( tid == 0 ) shared_array[0] = vol_sum;
  __syncthreads();
  return shared_array[0];
}


 ///
 /// Texture Access Convenience Functions
 ///

// These functions access data via the texture cache and place them in
// appropriate data structures. (Texture cache fetches can not return
// structures and cannot return vectors of length 3.)

__device__ float3 vtx_data_pos(int idx)
{
  return make_float3(tex1Dfetch(vtx_data_pos_tex, idx));
}

__device__ float3 tri_data_surface_normal(int idx)
{
  return make_float3(tex1Dfetch(tri_data_tex, idx << 2));
}

__device__ float3 tri_data_force(int idx, int member)
{
  return make_float3(tex1Dfetch(tri_data_tex, (idx<<2)+1+member));
}


 ///
 /// Compute Repulsion Force
 ///
//
// Used by one- and two-pass algorithms.
//
__device__ float3
repforce_compute(float3 p_pos, int po_idx)
{
  const float3 po_pos = vtx_data_pos(po_idx);
  const float3 p_to_q = vec_sub(p_pos,po_pos);
  const float mag_sq = dot(p_to_q,p_to_q);
  const float dist_sq_inv = rep_constant / max(0.001,mag_sq);
  const float3 p_to_q_n = normalize(p_to_q);
  return vec_scale(dist_sq_inv, p_to_q_n);
}


///
/// Two-Pass Code
///

__global__ void pass_triangles();
__global__ void pass_vertices(int write_side);


 __host__ void
pass_triangles_launch
 (dim3 dg, dim3 db, int write_size,
  CUDA_Vtx_Data_X *vtx_data_in,
  CUDA_Vtx_Data *vtx_data, size_t vtx_data_size)
 {
   size_t offset;
   hipBindTexture(&offset, vtx_data_pos_tex, vtx_data_in->pos, vtx_data_size);
   pass_triangles<<<dg,db>>>();
 }

__global__ void
pass_triangles()
{
  const int tid = threadIdx.x;
  const int ti = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float volumes[CUDA_TRI_BLOCK_SIZE];
  if ( ti >= tri_count ) volumes[tid] = 0;
  __syncthreads();
  if ( ti >= tri_count ) return;

  const CUDA_Tri_Strc_X_a tsa = tri_strc_x.a[ti];
  const CUDA_Tri_Strc_X_b tsb = tri_strc_x.b[ti];

  const float3 ppos = vtx_data_pos(tsa.pi);
  const float3 qpos = vtx_data_pos(tsa.qi);
  const float3 rpos = vtx_data_pos(tsa.ri);

  const float3 center = vec_scale(1.0/3.0, vec_add3(ppos,qpos,rpos));
  const float3 pqr_cross = cross3(qpos,ppos,rpos);

  const float triangle_area_x2 = length(pqr_cross);
  const float tower_volume_x2 = -pqr_cross.y * center.y;

  float3 force_p = repforce_compute(ppos,tsa.pi_opp);
  float3 force_q = repforce_compute(qpos,tsb.qi_opp);
  float3 force_r = repforce_compute(rpos,tsb.ri_opp);

  const float3 p_to_c = vec_sub(center,ppos);
  const float3 q_to_c = vec_sub(center,qpos);
  const float3 r_to_c = vec_sub(center,rpos);

  const float perimeter = length(p_to_c) + length(q_to_c) + length(r_to_c);

  const float length_relaxed = tri_strc_x.length_relaxed[ti];
  const float eff_length = max(0.0f, perimeter - length_relaxed );
  const float spring_force = eff_length * spring_constant;

  tri_data[ti].surface_normal = make_float4(pqr_cross,0);
  tri_data[ti].force_p =
    make_float4(vec_add(force_p, vec_scale(spring_force, p_to_c)),0);
  tri_data[ti].force_q =
    make_float4(vec_add(force_q, vec_scale(spring_force, q_to_c)),0);
  tri_data[ti].force_r = 
    make_float4(vec_add(force_r, vec_scale(spring_force, r_to_c)),0);

  const float vol_sum =
    reduce(CUDA_TRI_BLOCK_LG,volumes,tower_volume_x2 * 0.5f,false);
  if ( threadIdx.x == 0 ) tower_volumes[0][ blockIdx.x ] = vol_sum;
}

__host__ void
pass_vertices_launch
(dim3 dg, dim3 db, int write_side, CUDA_Tri_Data *tri_data, size_t tri_data_size)
{
  size_t offset;
  hipBindTexture(&offset, tri_data_tex, tri_data, tri_data_size);
  pass_vertices<<<dg,db>>>(write_side);
  hipUnbindTexture(vtx_data_pos_tex);
  hipUnbindTexture(tri_data_tex);
}

__global__ void
pass_vertices(int write_side)
{
  const int tid = threadIdx.x;
  const int vi = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float volumes[CUDA_VTX_BLOCK_SIZE];

  const int grid_dim_tri = div_p2_ceil(tri_count,CUDA_TRI_BLOCK_LG);
  const int vol_per_thread = div_p2_ceil(grid_dim_tri,CUDA_VTX_BLOCK_LG);
  float my_vol = 0;
  for ( int i=0; i<vol_per_thread; i++ )
    my_vol += tower_volumes[0][tid + ( i << CUDA_VTX_BLOCK_LG ) ];

  const float volume = reduce(CUDA_VTX_BLOCK_LG,volumes,my_vol,true);
  if ( vi >= point_count ) return;

  const float friction_coefficient = .04;
  const float bounce_factor = 0.0;
  const float mass = 1.0;

  const float3 gravity = make_float3(0.0,-gravity_mag,0.0);

  CUDA_Vtx_Data_X vtx_data_r = write_side ? vtx_data_x0 : vtx_data_x1;
  const float3 vel = make_float3(vtx_data_r.vel[vi]);
  const float3 pos = make_float3(vtx_data_r.pos[vi]);

  CUDA_Vtx_Strc_X_a vsa = vtx_strc_x.a[vi];
  CUDA_Vtx_Strc_X_b vsb = vtx_strc_x.b[vi];
  const int tri_idx_base = vtx_strc_x.tri_idx_base[vi];
  CUDA_Vtx_Strc vs;
  vs.n0=vsa.n0; vs.n1=vsa.n1; vs.n2=vsa.n2; vs.n3=vsa.n3;
  vs.n4=vsb.n4; vs.n5=vsb.n5; vs.n6=vsb.n6; vs.n7=vsb.n7;

  float3 force_spring = make_float3(0.0,0.0,0.0);
  float3 surface_normal_sum = make_float3(0.0,0.0,0.0);

#define TRI_BODY(i)                                                           \
  { const int idx_packed = vs.n##i;                                           \
    if ( idx_packed )                                                         \
      {                                                                       \
        const int idx_base = tri_idx_base + ( idx_packed >> 2 );              \
        const int idx_force = idx_packed & 0x3;                               \
        const float3 surface_normal_t = tri_data_surface_normal(idx_base);    \
        vec_addto(surface_normal_sum, surface_normal_t);                      \
        const float3 force_x = tri_data_force(idx_base,idx_force);            \
        vec_addto(force_spring, force_x);                                     \
      }}

  TRI_BODY(0); TRI_BODY(1); TRI_BODY(2); TRI_BODY(3);
  TRI_BODY(4); TRI_BODY(5); TRI_BODY(6); TRI_BODY(7);
#undef TRI_BODY

  float3 surface_normal = vec_scale(1./6., surface_normal_sum);

  float pressure_factor = pressure_factor_coeff / fabs(volume);

  float pressure =
    opt_gravity
    ? pressure_factor * exp( - gas_m_over_temp * pos.y )
    : pressure_factor;
  float air_pressure =
    opt_gravity ? exp( - 0.2f * air_particle_mass * pos.y ) : 1.0;

  float3 force_pressure = vec_scale(air_pressure - pressure, surface_normal);

  float3 force = make_float3(0,0,0);
  vec_addto(force_spring, force_pressure);

  float3 vel_norm = normalize(vel);
  float facing_area = max(0.0f,-dot(vel_norm,surface_normal));
  float3 force_ar = vec_scale( - air_resistance * facing_area, vel);

  float3 gforce = vec_scale(point_mass * mass, gravity);
  vec_addto(force, gforce);

  vec_addto(force, force_ar);

  float3 force_ns = force;

  vec_addto(force, force_spring);

  float mass_wgas_inv_dt =
    delta_t / ( point_mass * mass + gas_mass_per_vertex );

  float3 delta_vns = vec_scale(mass_wgas_inv_dt, force_ns);
  float3 delta_vs = vec_scale(mass_wgas_inv_dt, force_spring);
  float3 delta_v = vec_add(delta_vns, delta_vs);

  float3 pos_next =
    vec_add(pos, vec_scale(delta_t, vec_add( vel,  vec_scale(0.5f,delta_v) )));

  float3 vel_next = vec_add3(vel,  vec_scale(damping_v, delta_vs), delta_vns);

  const bool platform_aligned =
    pos_next.x >= platform_xmin && pos_next.x <= platform_xmax
    && pos_next.z >= platform_zmin && pos_next.z <= platform_zmax;
  const bool above_to_below =  pos_next.y <= 0.0f && pos.y >= 0.0f;

  if ( platform_aligned && above_to_below )
    {
      pos_next.y = 0.0;
      vel_next.y = - bounce_factor * vel_next.y;
      const float f_y = min(0.0, gforce.y + force_spring.y );
      const float friction_force = -f_y * friction_coefficient;
      const float delta_v = friction_force * delta_t / ( point_mass * mass );
      const float3 xzvel = make_float3(vel_next.x,0,vel_next.z);
      if ( length(xzvel) <= delta_v ) {
        vel_next.x = 0.0;  vel_next.z = 0.0;
      }
      else
        vec_addto(vel_next, vec_scale( -delta_v, normalize(xzvel) ));
    }

  if ( false && vi == tethered_idx ) { vel_next = vel; pos_next = pos; }

  CUDA_Vtx_Data_X vtx_data_w = write_side ? vtx_data_x1 : vtx_data_x0;
  vtx_data_w.surface_normal[vi] = make_float4(surface_normal,0);
  vtx_data_w.vel[vi] = make_float4(vel_next,0);
  vtx_data_w.pos[vi] = make_float4(pos_next,1);
}



///
/// One-Pass Code
///

struct CUDA_Tri_Shared {
  // Note: make sure GCD(16,sizeof(this)/4) = 1, e.g., not a power of 2!
  float3 center;
  float spring_force;
  float3 surface_normal;
};

__global__ void pass_unified(int write_side);

__host__ void
pass_unified_launch
(dim3 dg, dim3 db, int write_side,
 CUDA_Vtx_Data_X *vtx_data_in, size_t vtx_data_size )
{
  size_t offset;
  hipBindTexture(&offset, vtx_data_pos_tex, vtx_data_in->pos, vtx_data_size);
  pass_unified<<<dg,db>>>(write_side);
  hipUnbindTexture(vtx_data_pos_tex);
}

__global__ void
pass_unified(int write_side)
{
  const int tid = threadIdx.x;
  const int vtx_bk_base = __mul24(blockIdx.x , blockDim.x);
  const int vi = vtx_bk_base + tid;
  const int work_idx_base = __mul24(tri_work_per_vtx, vtx_bk_base) + tid;
  const int block_lg = CUDA_VTX_BLOCK_LG;
  const int block_size = CUDA_VTX_BLOCK_SIZE;
  float* const tower_volumes_in = 
    write_side ? tower_volumes[0] : tower_volumes[1];
  float* const tower_volumes_out =
    write_side ? tower_volumes[1] : tower_volumes[0];

  __shared__ float volumes[block_size], volumes_read[block_size];
  __shared__ CUDA_Tri_Shared tri_shared[block_size];

  // This routine computes information (area, surface normal, and a
  // force) for several triangles and up to one vertex. Each vertex
  // uses information from up to eight triangles. The triangles that a
  // vertex needs are computed somewhere in the same block, but not
  // necessarily in the same thread, so vertices get the triangle
  // information they need through shared memory.

  ///
  /// Triangle Round
  ///
  // Note that several triangles are processed by a thread.

  float local_volume_x2 = 0;
  float3 force_spring = make_float3(0,0,0);
  float3 surface_normal_sum = make_float3(0,0,0);

  CUDA_Vtx_Data_X vtx_data_r = write_side ? vtx_data_x0 : vtx_data_x1;

  const float3 pos =
    vi < point_count ? vtx_data_pos(vi) : make_float3(0,0,0);

  for ( int i=0; i<tri_work_per_vtx; i++ )
    {
      const int widx = work_idx_base + ( i << block_lg );
      CUDA_Tri_Work_Strc_X_a tsa = tri_work_strc_x.a[widx];
      CUDA_Tri_Work_Strc_X_b tsb = tri_work_strc_x.b[widx];
      CUDA_Tri_Work_Strc_X_c tsc = tri_work_strc_x.c[widx];

      const int ts_pull_i =  tsa.pull_i;

      /// Compute information for a triangle (if there is one).
      //
      if ( tsa.pi != tsa.ri || tsa.pi != tsa.qi ) // Compiler Workaround
        {
          const float3 ppos = vtx_data_pos(tsa.pi);
          const float3 qpos = vtx_data_pos(tsa.qi);
          const float3 rpos = vtx_data_pos(tsa.ri);

          const float3 center = vec_scale(1.0/3.0, vec_add3(ppos,qpos,rpos));
          const float3 pqr_cross = cross3(qpos,ppos,rpos);
          const float tower_volume_x2 = -pqr_cross.y * center.y;

          const bool use_vol = ts_pull_i & 1;
          if ( use_vol ) local_volume_x2 += tower_volume_x2;

          const float3 p_to_c = vec_sub(center,ppos);
          const float3 q_to_c = vec_sub(center,qpos);
          const float3 r_to_c = vec_sub(center,rpos);
          const float perimeter = length(p_to_c)+length(q_to_c)+length(r_to_c);
          const float length_relaxed = tri_work_strc_x.length_relaxed[widx];
          const float eff_length = max(0.0f, perimeter - length_relaxed );
          const float spring_force = eff_length * spring_constant;

          tri_shared[tid].center = center;
          tri_shared[tid].surface_normal = pqr_cross;
          tri_shared[tid].spring_force = spring_force;
        }

      __syncthreads();

      const int pull_i = ts_pull_i >> 1;

      /// Accumulate sum of triangle information computed above.
      //
      // Note that loops are hand unrolled to overcome compiler
      // limitations.
      //
#define VTX_PULL(idx)                                                         \
      if ( idx < pull_i )                                                     \
        {                                                                     \
          const int tri_id = tsc.pull_tid_##idx;                              \
          const float3 center = tri_shared[tri_id].center;                    \
          const float3 surface_normal_t = tri_shared[tri_id].surface_normal;  \
          vec_addto(surface_normal_sum,surface_normal_t);                     \
          vec_addto(force_spring,repforce_compute(pos,tsb.vi_opp##idx));      \
          const float3 p_to_c = vec_sub(center,pos);                          \
          const float spring_force = tri_shared[tri_id].spring_force;         \
          vec_addto(force_spring, vec_scale(spring_force, p_to_c));           \
        }

      VTX_PULL(0); VTX_PULL(1); VTX_PULL(2); VTX_PULL(3);
    }

  /// Write Volumes
  //
  // Compute sum of volumes of each thread this block, then write to a
  // global array for use in next time step.
  //
  const float vol_sum = reduce(block_lg,volumes,local_volume_x2 * 0.5f,false);
  if ( tid == 0 ) tower_volumes_out[ blockIdx.x ] = vol_sum;

  /// Read Volumes
  //
  // Retrieve volumes of blocks written in the last time step and
  // compute their sum.
  //
  const int vol_per_thread = div_p2_ceil(gridDim.x,block_lg);
  float my_vol = 0;
  for ( int i=0; i<vol_per_thread; i++ )
    my_vol += tower_volumes_in[tid + ( i << block_lg ) ];
  const float volume = reduce(block_lg,volumes_read,my_vol,true);

  ///
  /// Vertex Round
  ///
  // Compute new position and velocity.

  if ( vi >= point_count ) return;

  const float friction_coefficient = .04;
  const float bounce_factor = 0.0;
  const float mass = 1.0;

  const float3 gravity = make_float3(0.0,-gravity_mag,0.0);

  const float3 vel = make_float3(vtx_data_r.vel[vi]);

  const float3 surface_normal = vec_scale((1./6.), surface_normal_sum);

  float pressure_factor = pressure_factor_coeff / fabs(volume);

  float pressure =
    opt_gravity
    ? pressure_factor * exp( - gas_m_over_temp * pos.y )
    : pressure_factor;
  float air_pressure =
    opt_gravity ? exp( - 0.2f * air_particle_mass * pos.y ) : 1.0;

  float3 force_pressure = vec_scale(air_pressure - pressure, surface_normal);

  float3 force = make_float3(0,0,0);
  vec_addto(force_spring,force_pressure);

  float3 vel_norm = normalize(vel);
  float facing_area = max(0.0f,-dot(vel_norm,surface_normal));
  float3 force_ar = vec_scale( - air_resistance * facing_area, vel);

  float3 gforce = vec_scale(point_mass * mass, gravity);
  vec_addto(force, gforce);

  vec_addto(force, force_ar);

  float3 force_ns = force;

  vec_addto(force, force_spring);

  float mass_wgas_inv_dt =
    delta_t / ( point_mass * mass + gas_mass_per_vertex );

  float3 delta_vns = vec_scale(mass_wgas_inv_dt, force_ns);
  float3 delta_vs = vec_scale(mass_wgas_inv_dt, force_spring);
  float3 delta_v = vec_add(delta_vns, delta_vs);

  float3 pos_next =
    vec_add(pos, vec_scale(delta_t, vec_add( vel,  vec_scale(0.5,delta_v) )));

  float3 vel_next = vec_add3(vel,  vec_scale(damping_v, delta_vs), delta_vns);

  const bool platform_aligned =
    pos_next.x >= platform_xmin && pos_next.x <= platform_xmax
    && pos_next.z >= platform_zmin && pos_next.z <= platform_zmax;
  const bool above_to_below =  pos_next.y <= 0.0f && pos.y >= 0.0f;

  if ( platform_aligned && above_to_below )
    {
      pos_next.y = 0.0;
      vel_next.y = - bounce_factor * vel_next.y;
      const float f_y = min(0.0f, gforce.y + force_spring.y);
      const float friction_force = -f_y * friction_coefficient;
      const float delta_v = friction_force * delta_t / ( point_mass * mass );
      const float3 xzvel = make_float3(vel_next.x,0,vel_next.z);
      if ( length(xzvel) <= delta_v ) {
        vel_next.x = 0.0;  vel_next.z = 0.0;
      }
      else
        vec_addto(vel_next, vec_scale( -delta_v, normalize(xzvel) ));
    }

  CUDA_Vtx_Data_X vtx_data_x = write_side ? vtx_data_x1 : vtx_data_x0;
  vtx_data_x.surface_normal[vi] = make_float4(surface_normal,0);
  vtx_data_x.vel[vi] = make_float4(vel_next,0);
  vtx_data_x.pos[vi] = make_float4(pos_next,1);
}
