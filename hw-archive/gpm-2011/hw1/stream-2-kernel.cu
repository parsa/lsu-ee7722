#include "hip/hip_runtime.h"
/// LSU EE 7700-2 (Sp 2011), GPU Microarchitecture
//
/// Homework 1

 /// See stream-2.cc for instructions.

#include "stream-2.cuh"


// Constants holding array sizes and pointers and coefficients.
//
// Values are set by cuda calls, they don't automatically take values
// of variables in the C program with the same name.
//
__constant__ float v0, v1, v2;
__constant__ int array_size;
__constant__ Vector2* a;
__constant__ float* b;


// Declare Kernels
//
__global__ void dots_loopless();
__global__ void dots_stride_large();
__global__ void dots_stride_small();


__host__ hipError_t
kernels_get_attr
(struct hipFuncAttributes *attr, char **names, int max_count)
{
  // Return information about kernels.
  //
  // Information includes number of registers used.

  struct hipFuncAttributes *attr_stop = attr + max_count;
  hipError_t er = hipSuccess; // Tentative.
#define GETATTR(func)                                                         \
  er = hipFuncGetAttributes(attr,reinterpret_cast<const void*>(func)); *names = #func; attr++; names++;     \
  if ( er || attr == attr_stop ) { *names = NULL; return er; }
  GETATTR(dots_loopless);
  GETATTR(dots_stride_large);
  GETATTR(dots_stride_small);
  *names = NULL;
  return er;
#undef GETATTR
}


 /// Homework 1: The routine below will have to be modfied for several problems.
 // 
__host__ void
dots_launch(dim3 dg, dim3 db)
{
  // Launch the kernel, using the provided configuration (block size, etc).
  //
  //  dots_loopless<<<dg,db>>>();
  dots_stride_large<<<dg,db>>>();
}

// This routine executes on the GPU.
//
__global__ void
dots_loopless()
{
  // Run the kernel for one iteration.  This only works
  // if the total number of threads is equal to the array size.

  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //        Idx within    Idx of       Block size.
  //        a block.      block.
  //        Can be 0      Can be
  //        to block      from 0
  //        size -1       to # of
  //                      blocks.

  // Array size might not be a multiple of block size.
  //
  if ( idx >= array_size ) return;

  b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}


// This routine also executes on the GPU.
//
__global__ void
dots_stride_large()
{
  // Compute a unique index (number) for this thread.
  // This will be used as an array index.
  //
  int idx_start = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;
  int stride = thread_count;

  for ( int idx = idx_start; idx < array_size; idx += stride )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}


 /// Homework 1: Modfiy this routine for one of the problems.
//
__global__ void
dots_stride_small()
{
  // Modify this routine so that a block accesses a contiguous
  // block of data.
  //
  int idx_start = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_count = blockDim.x * gridDim.x;
  int stride = thread_count;

  for ( int idx = idx_start; idx < array_size; idx += stride )
    b[idx] = v0 + v1 * a[idx].x + v2 * a[idx].y;
}
