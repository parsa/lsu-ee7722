#include "hip/hip_runtime.h"
/// LSU EE 7722 (Spring 2014), GPU Microarchitecture
//

 /// Homework 3 (Spring 2014)
 //
 // Assignment in: http://www.ece.lsu.edu/koppel/gp/2014/hw03.pdf
 //

#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <errno.h>
#include <ctype.h>
#include <time.h>
#include <new>
#include <hip/hip_runtime.h>
#include "util.h"


struct App_Common
{
  int length;
  int num_pieces;
  int idx_array_elts;
  int data_array_elts;
  int *idx_array;
  float *data_array;
  float *sum_array;
  float *sum_array_check;

  int *d_idx_array;
  float *d_data_array;
  float *d_sum_array;
};


// In host address space.
App_Common app;

// In device constant address space.
__constant__ App_Common dapp;


const int clength = 8;

extern "C" __global__ void
sums_0()
{
  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
    {
      const int idx_piece_start = piece * clength;
      float sum = 0;
      for ( int i=0; i<clength; i++ )
        sum += dapp.d_data_array[dapp.d_idx_array[idx_piece_start+i]];

      dapp.d_sum_array[piece] = sum;
    }
}

extern "C" __global__ void
sums_1()
{
  // PROBLEM 2 SOLUTION HERE.
  // Reduce noncontiguous access to dapp.d_idx_array.

  return;

  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
    {
      const int idx_piece_start = piece * clength;
      float sum = 0;
      for ( int i=0; i<clength; i++ )
        sum += dapp.d_data_array[dapp.d_idx_array[idx_piece_start+i]];

      dapp.d_sum_array[piece] = sum;
    }
}

extern "C" __global__ void
sums_2()
{
  // PROBLEM 3 SOLUTION HERE.
  // Reduce noncontiguous access to dapp.d_data_array using shared memory.

  return;


  const int thread_count = blockDim.x * gridDim.x;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for ( int piece = tid; piece < dapp.num_pieces; piece += thread_count )
    {
      const int idx_piece_start = piece * clength;
      float sum = 0;
      for ( int i=0; i<clength; i++ )
        sum += dapp.d_data_array[dapp.d_idx_array[idx_piece_start+i]];

      dapp.d_sum_array[piece] = sum;
    }
}

GPU_Info
print_gpu_info()
{
  GPU_Info info;

  // Get information about GPU and its ability to run CUDA.
  //
  int device_count;
  CE( hipGetDeviceCount(&device_count) ); // Get number of GPUs.
  if ( device_count == 0 )
    {
      fprintf(stderr,"No GPU found, exiting.\n");
      exit(1);
    }

  /// Print information about the available GPUs.
  //
  for ( int dev=0; dev<device_count; dev++ )
    {
      info.get_gpu_info(dev);  // Look in file util.h.
      hipDeviceProp_t& cuda_prop = info.cuda_prop;

      printf
        ("GPU %d: %s @ %.2f GHz WITH %d MiB GLOBAL MEM\n",
         dev, cuda_prop.name, cuda_prop.clockRate/1e6,
         int(cuda_prop.totalGlobalMem >> 20));

      printf
        ("GPU %d: CC: %d.%d  MP: %2d  CC/MP: %3d  TH/BL: %4d\n",
         dev, cuda_prop.major, cuda_prop.minor,
         cuda_prop.multiProcessorCount,
         info.cc_per_mp,
         cuda_prop.maxThreadsPerBlock);

      printf
        ("GPU %d: SHARED: %5d B  CONST: %5d B  # REGS: %5d\n",
         dev,
         int(cuda_prop.sharedMemPerBlock), int(cuda_prop.totalConstMem),
         cuda_prop.regsPerBlock);

      printf
        ("GPU %d: L2: %d kiB   MEM to L2: %.1f GB/s  SP %.1f GFLOPS  "
         "OP/ELT %.2f\n",
         dev,
         cuda_prop.l2CacheSize >> 10,
         info.chip_bw_Bps * 1e-9,
         info.chip_sp_flops * 1e-9,
         4 * info.chip_sp_flops / info.chip_bw_Bps);
    }

  // Choose GPU 0 because it's usually the better choice.
  //
  int dev = 0;
  CE(hipSetDevice(dev));
  printf("Using GPU %d\n",dev);
  info.get_gpu_info(dev);

  info.GET_INFO(sums_0);
  info.GET_INFO(sums_1);
  info.GET_INFO(sums_2);

  // Print information about kernel.
  //
  printf("\nCUDA Kernel Resource Usage:\n");

  for ( int i=0; i<info.num_kernels; i++ )
    {
      printf("For %s:\n", info.ki[i].name);
      printf("  %6zd shared, %zd const, %zd loc, %d regs; "
             "%d max threads per block.\n",
             info.ki[i].cfa.sharedSizeBytes,
             info.ki[i].cfa.constSizeBytes,
             info.ki[i].cfa.localSizeBytes,
             info.ki[i].cfa.numRegs,
             info.ki[i].cfa.maxThreadsPerBlock);
    }

  return info;
}

int
main(int argc, char **argv)
{
  // Get info about GPU and each kernel.
  //
  GPU_Info info = print_gpu_info();

  // Examine argument 1, grid size.
  //
  const int arg1_int =
    argc < 2 ? info.cuda_prop.multiProcessorCount : atoi(argv[1]);
  const int num_blocks = abs(arg1_int);

  // Examine argument 2, number of pieces. Fractional values okay.
  //
  app.num_pieces = argc < 3 ? 1 << 20 : int( atof(argv[2]) * (1<<20) );

  if ( num_blocks <= 0 || app.num_pieces <= 0 )
    {
      printf("Usage: %s [ NUM_CUDA_BLOCKS ] [NUMBER OF PIECES]\n",
             argv[0]);
      exit(1);
    }

  app.length = clength;
  app.idx_array_elts = app.num_pieces * app.length;
  app.data_array_elts = 1 << 14;

  const int idx_array_bytes = app.idx_array_elts * sizeof(app.idx_array[0]);
  const int data_array_bytes = app.data_array_elts * sizeof(app.data_array[0]);
  const int sum_array_bytes = app.num_pieces * sizeof(app.sum_array[0]);

  // Allocate storage for CPU copy of data.
  //
  app.idx_array = new int[idx_array_bytes];
  app.data_array = new float[data_array_bytes];
  app.sum_array = new float[sum_array_bytes];
  app.sum_array_check = new float[sum_array_bytes];

  // Allocate storage for GPU copy of data.
  //
  CE( hipMalloc( &app.d_idx_array, idx_array_bytes ) );
  CE( hipMalloc( &app.d_data_array, data_array_bytes ) );
  CE( hipMalloc( &app.d_sum_array, sum_array_bytes ) );

  printf("\nData array size %d elements, index array size %d elements.\n",
         app.data_array_elts, app.idx_array_elts);

  // Initialize input array.
  //
  for ( int i=0; i<app.data_array_elts; i++ )
    {
      app.data_array[i] = drand48();
    }

  for ( int i=0; i<app.idx_array_elts; i++ )
    {
      app.idx_array[i] = random() % app.data_array_elts;
    }

  // Compute correct answer (for checking).
  //
  for ( int p=0; p<app.num_pieces; p++ )
    {
      float sum = 0;
      int idx_base = p * app.length;
      for ( int i=0; i<app.length; i++ )
        sum += app.data_array[app.idx_array[idx_base+i]];
      app.sum_array_check[p] = sum;
    }

  // Prepare events used for timing.
  //
  hipEvent_t gpu_start_ce, gpu_stop_ce;
  CE(hipEventCreate(&gpu_start_ce));
  CE(hipEventCreate(&gpu_stop_ce));

  // Copy input array from CPU to GPU.
  //
  CE( hipMemcpy( app.d_idx_array, app.idx_array, idx_array_bytes,
                  hipMemcpyHostToDevice ) );
  CE( hipMemcpy( app.d_data_array, app.data_array, data_array_bytes,
                  hipMemcpyHostToDevice ) );

  const double data_size =
    idx_array_bytes * data_array_bytes + sum_array_bytes;

  // Stuff needed to print bar graph.
  //
  const int full_width = 49;  // Width of longest bar.
  char **stars = (char**) alloca( info.num_kernels * sizeof(void*) );
  for ( int j=0; j< info.num_kernels; j++ )
    {
      stars[j] = (char*) alloca(full_width+1);
      for ( int i=0; i<full_width; i++ ) stars[j][i] = '0' + j;
      stars[j][full_width] = 0;
    }

  for ( int kernel = 0; kernel < info.num_kernels; kernel++ )
    {
      hipFuncAttributes& cfa = info.ki[kernel].cfa;

      if ( kernel > 0 && cfa.numRegs < 3 )
        {
          printf("Not running kernel %s because code probably not present.\n",
                info.ki[kernel].name);
          continue;
        }

      // Maximum number of warps per block that this GPU can handle
      // for this kernel.
      //
      const int wp_limit = cfa.maxThreadsPerBlock >> 5;

      printf("\nRunning kernel %s which uses %d regs on %d blocks.\n",
             info.ki[kernel].name, cfa.numRegs, num_blocks);

      printf(" %3s %9s  %6s  %5s\n", "Num", "Time", "Data", "" );
      printf(" %3s %9s  %6s  %5s\n", "Wps", "µs", " GB/s", "Pct" );

      for ( int warp_cnt = 2; warp_cnt <= wp_limit; warp_cnt += 2 )
        {
          const int thd_per_block = warp_cnt << 5;

          // Copy App structure to GPU.
          //
          CE( hipMemcpyToSymbol(HIP_SYMBOL( dapp), &app, sizeof(app), 0, hipMemcpyHostToDevice ) );

          // Zero result array (to catch errors when kernel skips elements).
          //
          CE( hipMemset( app.d_sum_array, 0, sum_array_bytes ) );

          // Measure execution time starting "now", which is after data
          // set to GPU.
          //
          CE(hipEventRecord(gpu_start_ce,0));

          // Launch Kernel
          //
          info.ki[kernel].func_ptr<<<num_blocks,thd_per_block>>>();

          // Stop measuring execution time now, which is before is data
          // returned from GPU.
          //
          CE(hipEventRecord(gpu_stop_ce,0));
          CE(hipEventSynchronize(gpu_stop_ce));
          float cuda_time_ms = -1.1;
          CE(hipEventElapsedTime(&cuda_time_ms,gpu_start_ce,gpu_stop_ce));
          const double elapsed_time_s = cuda_time_ms * 0.001;

          // Copy output array from GPU to CPU.
          //
          CE( hipMemcpy
              (app.sum_array, app.d_sum_array, sum_array_bytes,
               hipMemcpyDeviceToHost) );

          int err = 0;
          for ( int e=0; e<app.num_pieces; e++ )
            {
              const float diff =
                fabs( app.sum_array[e] - app.sum_array_check[e] );
              if ( diff > 0.00001 )
                {
                  err++;
                  if ( err < 2 )
                    printf
                      ("Error at array idx %d: %f != %f (correct)\n",
                       e, app.sum_array[e], app.sum_array_check[e]);
                }
            }

          const double frac = data_size / elapsed_time_s / info.chip_bw_Bps;
          const int s_idx = max(0.0,full_width * ( 1 - frac ));

          printf(" %2d  %9.3f  %6.2f %5.1f%%  %s\n",
                 warp_cnt, 1e6 * elapsed_time_s,
                 1e-9 * data_size / elapsed_time_s,
                 100 * frac, &stars[kernel][s_idx] );
        }
    }
}
